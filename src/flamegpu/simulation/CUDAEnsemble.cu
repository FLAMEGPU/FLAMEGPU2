#include "hip/hip_runtime.h"
#include "flamegpu/simulation/CUDAEnsemble.h"

#include <algorithm>
#include <cstdlib>
#include <memory>
#include <thread>
#include <set>
#include <queue>
#include <mutex>
#include <condition_variable>
#include <filesystem>
#include <map>

#include "flamegpu/version.h"
#include "flamegpu/model/ModelDescription.h"
#include "flamegpu/simulation/RunPlanVector.h"
#include "flamegpu/detail/compute_capability.cuh"
#include "flamegpu/detail/SteadyClockTimer.h"
#include "flamegpu/simulation/CUDASimulation.h"
#include "flamegpu/io/StateWriterFactory.h"
#include "flamegpu/simulation/LoggingConfig.h"
#include "flamegpu/simulation/detail/SimRunner.h"
#include "flamegpu/simulation/LogFrame.h"
#include "flamegpu/simulation/detail/SimLogger.h"
#include "flamegpu/detail/cuda.cuh"
#include "flamegpu/io/Telemetry.h"

namespace flamegpu {

CUDAEnsemble::EnsembleConfig::EnsembleConfig()
    : telemetry(flamegpu::io::Telemetry::isEnabled()) {}


CUDAEnsemble::CUDAEnsemble(const ModelDescription& _model, int argc, const char** argv, bool _isSWIG)
    : model(_model.model->clone())
    , isSWIG(_isSWIG) {
    initialise(argc, argv);
}
CUDAEnsemble::~CUDAEnsemble() {
// Call this here incase simulate() exited with an exception
#ifdef _MSC_VER
    if (config.block_standby) {
        // Disable prevention of standby
        SetThreadExecutionState(ES_CONTINUOUS);
    }
#endif
}



unsigned int CUDAEnsemble::simulate(const RunPlanVector &plans) {
#ifdef _MSC_VER
    if (config.block_standby) {
        // This thread requires the system continuously until it exits
        SetThreadExecutionState(ES_CONTINUOUS | ES_SYSTEM_REQUIRED);
    }
#endif
    // Validate that RunPlan model matches CUDAEnsemble model
    if (*plans.environment != this->model->environment->properties) {
        THROW exception::InvalidArgument("RunPlan is for a different ModelDescription, in CUDAEnsemble::simulate()");
    }
    // Validate/init output directories
    if (!config.out_directory.empty()) {
        // Validate out format is right
        config.out_format = io::StateWriterFactory::detectSupportedFileExt(config.out_format);
        if (config.out_format.empty()) {
            THROW exception::InvalidArgument("The out_directory config option also requires the out_format options to be set to a suitable type (e.g. 'json', 'xml'), in CUDAEnsemble::simulate()");
        }
        // Check that output files don't already exist
        if (std::filesystem::exists(config.out_directory)) {
            std::set<std::filesystem::path> exit_files;
            for (unsigned int p = 0; p < plans.size(); ++p) {
                std::filesystem::path exit_path = config.out_directory;
                if (!plans[p].getOutputSubdirectory().empty())
                    exit_path /= std::filesystem::path(plans[p].getOutputSubdirectory());
                exit_path /= std::filesystem::path("exit." + config.out_format);
                exit_files.insert(exit_path);
            }
            if (!config.truncate_log_files) {
                // Step
                for (unsigned int p = 0; p < plans.size(); ++p) {
                    std::filesystem::path step_path = config.out_directory;
                    if (!plans[p].getOutputSubdirectory().empty())
                        step_path /= std::filesystem::path(plans[p].getOutputSubdirectory());
                    step_path /= std::filesystem::path(std::to_string(p) + "." + config.out_format);
                    if (std::filesystem::exists(step_path)) {
                        THROW exception::FileAlreadyExists("Step log file '%s' already exists, in CUDAEnsemble::simulate()", step_path.c_str());
                    }
                }
                // Exit
                for (const auto &exit_path : exit_files) {
                    if (std::filesystem::exists(exit_path)) {
                        THROW exception::FileAlreadyExists("Exit log file '%s' already exists, in CUDAEnsemble::simulate()", exit_path.c_str());
                    }
                }
            } else {
                // Delete pre-existing exit log files
                for (const auto& exit_path : exit_files) {
                    std::filesystem::remove(exit_path);  // Returns false if the file didn't exist
                }
            }
        }
        // Create any missing directories
        try {
            std::filesystem::create_directories(config.out_directory);
        } catch (const std::exception &e) {
            THROW exception::InvalidArgument("Unable to use output directory '%s', in CUDAEnsemble::simulate(): %s", config.out_directory.c_str(), e.what());
        }
        for (const auto &p : plans) {
            const auto subdir = p.getOutputSubdirectory();
            if (!subdir.empty()) {
                std::filesystem::path sub_path = config.out_directory;
                try {
                    sub_path.append(subdir);
                    std::filesystem::create_directories(sub_path);
                } catch (const std::exception &e) {
                    THROW exception::InvalidArgument("Unable to use output subdirectory '%s', in CUDAEnsemble::simulate(): %s", sub_path.generic_string().c_str(), e.what());
                }
            }
        }
    }
    // Purge run logs, and resize ready for new runs
    // Resize means we can setup logs during execution out of order, without risk of list being reallocated
    run_logs.clear();
    run_logs.resize(plans.size());
    // Workout how many devices and runner we will be executing
    int device_count = -1;
    hipError_t cudaStatus = hipGetDeviceCount(&device_count);
    if (cudaStatus != hipSuccess) {
        THROW exception::InvalidCUDAdevice("Error finding CUDA devices!  Do you have a CUDA-capable GPU installed?, in CUDAEnsemble::simulate()");
    }
    if (device_count == 0) {
        THROW exception::InvalidCUDAdevice("Error no CUDA devices found!, in CUDAEnsemble::simulate()");
    }
    for (const int id : config.devices) {
        if (id >= device_count) {
            THROW exception::InvalidCUDAdevice("Requested CUDA device %d is not valid, only %d CUDA devices available!, in CUDAEnsemble::simulate()", id, device_count);
        }
    }

    std::set<int> devices;
    if (config.devices.size()) {
        devices = config.devices;
    } else {
        for (int i = 0; i < device_count; ++i) {
            devices.emplace(i);
        }
    }
    // Check that each device is capable, and init cuda context
    for (auto d = devices.begin(); d != devices.end(); ++d) {
        if (!detail::compute_capability::checkComputeCapability(*d)) {
            fprintf(stderr, "FLAMEGPU2 has not been built with an appropriate compute capability for device %d, this device will not be used.\n", *d);
            d = devices.erase(d);
            --d;
        } else {
            gpuErrchk(hipSetDevice(*d));
            gpuErrchk(flamegpu::detail::cuda::hipFree(nullptr));
        }
    }
    // Return to device 0 (or check original device first?)
    gpuErrchk(hipSetDevice(0));

    // Init runners, devices * concurrent runs
    std::atomic<unsigned int> err_ct = {0};
    std::atomic<unsigned int> next_run = {0};
    const unsigned int TOTAL_RUNNERS = static_cast<unsigned int>(devices.size()) * config.concurrent_runs;
    detail::SimRunner *runners = static_cast<detail::SimRunner *>(malloc(sizeof(detail::SimRunner) * TOTAL_RUNNERS));

    // Log Time (We can't use CUDA events here, due to device resets)
    auto ensemble_timer = detail::SteadyClockTimer();
    ensemble_timer.start();
    // Reset the elapsed time.
    ensemble_elapsed_time = 0.;

    // Logging thread-safety items
    std::queue<unsigned int> log_export_queue;
    std::mutex log_export_queue_mutex;
    std::condition_variable log_export_queue_cdn;
    detail::SimRunner::ErrorDetail fast_err_detail = {};

    // Init with placement new
    {
        unsigned int i = 0;
        for (auto &d : devices) {
            for (unsigned int j = 0; j < config.concurrent_runs; ++j) {
                new (&runners[i++]) detail::SimRunner(model, err_ct, next_run, plans,
                    step_log_config, exit_log_config,
                    d, j,
                    config.verbosity, config.error_level == EnsembleConfig::Fast,
                    run_logs, log_export_queue, log_export_queue_mutex, log_export_queue_cdn, fast_err_detail, TOTAL_RUNNERS, isSWIG);
            }
        }
    }

    // Init log worker
    detail::SimLogger *log_worker = nullptr;
    if (!config.out_directory.empty()) {
        log_worker = new detail::SimLogger(run_logs, plans, config.out_directory, config.out_format, log_export_queue, log_export_queue_mutex, log_export_queue_cdn,
        step_log_config.get(), exit_log_config.get(), step_log_config && step_log_config->log_timing, exit_log_config && exit_log_config->log_timing);
    }

    // Wait for all runners to exit
    for (unsigned int i = 0; i < TOTAL_RUNNERS; ++i) {
        runners[i].thread.join();
        runners[i].~SimRunner();
    }
    // Notify logger to exit
    if (log_worker) {
        {
            std::lock_guard<std::mutex> lck(log_export_queue_mutex);
            log_export_queue.push(UINT_MAX);
        }
        log_export_queue_cdn.notify_one();
        log_worker->thread.join();
        delete log_worker;
        log_worker = nullptr;
    }

    // Record and store the elapsed time
    ensemble_timer.stop();
    ensemble_elapsed_time = ensemble_timer.getElapsedSeconds();

    // Ensemble has finished, print summary
    if (config.verbosity > Verbosity::Quiet) {
        printf("\rCUDAEnsemble completed %u runs successfully!\n", static_cast<unsigned int>(plans.size() - err_ct));
        if (err_ct)
            printf("There were a total of %u errors.\n", err_ct.load());
    }
    if (config.timing || config.verbosity >= Verbosity::Verbose) {
        printf("Ensemble time elapsed: %fs\n", ensemble_elapsed_time);
    }

    // Send Telemetry
    if (config.telemetry) {
        // Generate some payload items
        std::map<std::string, std::string> payload_items;
        payload_items["GPUDevices"] = flamegpu::detail::compute_capability::getDeviceNames(config.devices);
        payload_items["SimTime(s)"] = std::to_string(ensemble_elapsed_time);
        #if defined(__CUDACC_VER_MAJOR__) && defined(__CUDACC_VER_MINOR__) && defined(__CUDACC_VER_PATCH__)
            payload_items["NVCCVersion"] = std::to_string(__CUDACC_VER_MAJOR__) + "." + std::to_string(__CUDACC_VER_MINOR__) + "." + std::to_string(__CUDACC_VER_BUILD__);
        #endif
        // Add the ensemble size to the ensemble telemetry payload
        payload_items["PlansSize"] = std::to_string(plans.size());
        payload_items["ConcurrentRuns"] = std::to_string(config.concurrent_runs);
        // generate telemetry data
        std::string telemetry_data = flamegpu::io::Telemetry::generateData("ensemble-run", payload_items, isSWIG);
        // send the telemetry packet
        bool telemetrySuccess = flamegpu::io::Telemetry::sendData(telemetry_data);
        // If verbose, print either a successful send, or a misc warning.
        if (config.verbosity >= Verbosity::Verbose) {
            if (telemetrySuccess) {
                fprintf(stdout, "Telemetry packet sent to '%s' json was: %s\n", flamegpu::io::Telemetry::TELEMETRY_ENDPOINT, telemetry_data.c_str());
            } else {
                fprintf(stderr, "Warning: Usage statistics for CUDAEnsemble failed to send.\n");
            }
        }
    } else {
        // Encourage users who have opted out to opt back in, unless suppressed.
        if ((config.verbosity > Verbosity::Quiet)) {
            flamegpu::io::Telemetry::encourageUsage();
        }
    }

    // Free memory
    free(runners);

    if (config.error_level == EnsembleConfig::Fast && err_ct.load()) {
        THROW exception::EnsembleError("Run %u failed on device %d, thread %u with exception: \n%s\n",
            fast_err_detail.run_id, fast_err_detail.device_id, fast_err_detail.runner_id, fast_err_detail.exception_string.c_str());
    } else if (config.error_level == EnsembleConfig::Slow && err_ct.load()) {
        THROW exception::EnsembleError("%u/%u runs failed!\n.", err_ct.load(), static_cast<unsigned int>(plans.size()));
    }
#ifdef _MSC_VER
    if (config.block_standby) {
        // Disable prevention of standby
        SetThreadExecutionState(ES_CONTINUOUS);
    }
#endif

    return err_ct.load();
}

void CUDAEnsemble::initialise(int argc, const char** argv) {
    if (!checkArgs(argc, argv)) {
        exit(EXIT_FAILURE);
    }
    // If verbose, output the flamegpu version and seed.
    if (config.verbosity == Verbosity::Verbose) {
        fprintf(stdout, "FLAME GPU %s\n", flamegpu::VERSION_FULL);
        fprintf(stdout, "Ensemble configuration:\n");
        fprintf(stdout, "\tConcurrent runs: %u\n", config.concurrent_runs);
    }
}
int CUDAEnsemble::checkArgs(int argc, const char** argv) {
    // Parse optional args
    int i = 1;
    for (; i < argc; i++) {
        // Get arg as lowercase
        std::string arg(argv[i]);
        std::transform(arg.begin(), arg.end(), arg.begin(), [](unsigned char c) { return std::use_facet< std::ctype<char>>(std::locale()).tolower(c); });
        // -h/--help. Print the help output and exit.
        if (arg.compare("--help") == 0 || arg.compare("-h") == 0) {
            printHelp(argv[0]);
            return false;
        }
        // --concurrent <runs>, Number of concurrent simulations to run per device
        if (arg.compare("--concurrent") == 0 || arg.compare("-c") == 0) {
            if (i + 1 >= argc) {
                fprintf(stderr, "%s requires a trailing argument\n", arg.c_str());
                return false;
            }
            config.concurrent_runs = static_cast<unsigned int>(strtoul(argv[++i], nullptr, 0));
            continue;
        }
        // --devices <string>, comma separated list of uints
        if (arg.compare("--devices") == 0 || arg.compare("-d") == 0) {
            if (i + 1 >= argc) {
                fprintf(stderr, "%s requires a trailing argument\n", arg.c_str());
                return false;
            }
            // Split and parse string
            std::string device_string = argv[++i];
            device_string += ",";  // Append comma, to catch final item
            int max_id = 0;  // Catch max device so we can validate it exists
            size_t pos;
            while ((pos = device_string.find(",")) != std::string::npos) {
                const unsigned int id = static_cast<unsigned int>(strtoul(device_string.substr(0, pos).c_str(), nullptr, 0));
                if (id == 0 && (device_string.length() < 2 || (device_string[0] != '0' || device_string[1] != ','))) {
                    fprintf(stderr, "'%s' is not a valid device index.\n", device_string.substr(0, pos).c_str());
                    printHelp(argv[0]);
                    return false;
                }
                max_id = static_cast<int>(id) > max_id ? id : max_id;
                config.devices.emplace(id);
                device_string.erase(0, pos + 1);
            }
            int ct = -1;
            gpuErrchk(hipGetDeviceCount(&ct));
            if (max_id >= ct) {
                fprintf(stderr, "Device id %u exceeds available CUDA devices %d\n", max_id, ct);
                printHelp(argv[0]);
                return false;
            }
            continue;
        }
        // -o/--out <directory> <filetype>, Quiet FLAME GPU output.
        if (arg.compare("--out") == 0 || arg.compare("-o") == 0) {
            if (i + 2 >= argc) {
                fprintf(stderr, "%s requires two trailing arguments\n", arg.c_str());
                return false;
            }
            // Validate output directory is valid (and recursively create it if necessary)
            try {
                std::filesystem::path out_directory = argv[++i];
                std::filesystem::create_directories(out_directory);
                config.out_directory = out_directory.generic_string();
            } catch (const std::exception &e) {
                // Catch any exceptions, probably std::filesystem::filesystem_error, but other implementation defined errors also possible
                fprintf(stderr, "Unable to use '%s' as output directory:\n%s\n", argv[i], e.what());
                return false;
            }
            // Validate output format is available in io module
            config.out_format = io::StateWriterFactory::detectSupportedFileExt(argv[++i]);
            if (config.out_format.empty()) {
                fprintf(stderr, "'%s' is not a supported output file type.\n", argv[i]);
                return false;
            }
            continue;
        }
        // -q/--quiet, Don't report progress to console.
        if (arg.compare("--quiet") == 0 || arg.compare("-q") == 0) {
            config.verbosity = Verbosity::Quiet;
            continue;
        }
        // -v/--verbose, Report all progress to console.
        if (arg.compare("--verbose") == 0 || arg.compare("-v") == 0) {
            config.verbosity = Verbosity::Verbose;
            continue;
        }
        // -t/--timing, Output timing information to stdout
        if (arg.compare("--timing") == 0 || arg.compare("-t") == 0) {
            config.timing = true;
            continue;
        }
        // -u/--silence-unknown-args, Silence warning for unknown arguments
        if (arg.compare("--silence-unknown-args") == 0 || arg.compare("-u") == 0) {
            config.silence_unknown_args = true;
            continue;
        }
        // -e/--error, Specify the error level
        if (arg.compare("--error") == 0 || arg.compare("-e") == 0) {
            if (i + 1 >= argc) {
                fprintf(stderr, "%s requires a trailing argument\n", arg.c_str());
                return false;
            }
            std::string error_level_string = argv[++i];
            // Shift the trailing arg to lower
            std::transform(error_level_string.begin(), error_level_string.end(), error_level_string.begin(), [](unsigned char c) { return std::use_facet< std::ctype<char>>(std::locale()).tolower(c); });
            if (error_level_string.compare("off") == 0 || error_level_string.compare(std::to_string(EnsembleConfig::Off)) == 0) {
                config.error_level = EnsembleConfig::Off;
            } else if (error_level_string.compare("slow") == 0 || error_level_string.compare(std::to_string(EnsembleConfig::Slow)) == 0) {
                config.error_level = EnsembleConfig::Slow;
            } else if (error_level_string.compare("fast") == 0 || error_level_string.compare(std::to_string(EnsembleConfig::Fast)) == 0) {
                config.error_level = EnsembleConfig::Fast;
            } else {
                fprintf(stderr, "%s is not an appropriate argument for %s\n", error_level_string.c_str(), arg.c_str());
                return false;
            }
            continue;
        }
        // --truncate, Truncate output files
        if (arg.compare("--truncate") == 0) {
            config.truncate_log_files = true;
            continue;
        }
        // --standby Disable the blocking of standby
        if (arg.compare("--standby") == 0) {
#ifdef _MSC_VER
            config.block_standby = false;
#endif
            continue;
        }
        // Warning if not in QUIET verbosity or if silnce-unknown-args is set
        if (!(config.verbosity == flamegpu::Verbosity::Quiet || config.silence_unknown_args))
            fprintf(stderr, "Warning: Unknown argument '%s' passed to Ensemble will be ignored\n", arg.c_str());
    }
    return true;
}
void CUDAEnsemble::printHelp(const char *executable) {
    printf("FLAME GPU %s\n", flamegpu::VERSION_FULL);
    printf("Usage: %s [optional arguments]\n", executable);
    printf("Optional Arguments:\n");
    const char *line_fmt = "%-18s %s\n";
    printf(line_fmt, "-h, --help", "show this help message and exit");
    printf(line_fmt, "-d, --devices <device ids>", "Comma separated list of device ids to be used");
    printf(line_fmt, "", "By default, all available devices will be used.");
    printf(line_fmt, "-c, --concurrent <runs>", "Number of concurrent simulations to run per device");
    printf(line_fmt, "", "By default, 4 will be used.");
    printf(line_fmt, "-o, --out <directory> <filetype>", "Directory and filetype for ensemble outputs");
    printf(line_fmt, "-q, --quiet", "Do not print progress information to console");
    printf(line_fmt, "-v, --verbose", "Print config, progress and timing (-t) information to console");
    printf(line_fmt, "-t, --timing", "Output timing information to stdout");
    printf(line_fmt, "-e, --error <error level>", "The error level 0, 1, 2, off, slow or fast");
    printf(line_fmt, "-u, --silence-unknown-args", "Silence warnings for unknown arguments passed after this flag.");
#ifdef _MSC_VER
    printf(line_fmt, "    --standby", "Allow the machine to enter standby during execution");
#endif
    printf(line_fmt, "", "By default, \"slow\" will be used.");
}
void CUDAEnsemble::setStepLog(const StepLoggingConfig &stepConfig) {
    // Validate ModelDescription matches
    if (*stepConfig.model != *model) {
      THROW exception::InvalidArgument("Model descriptions attached to LoggingConfig and CUDAEnsemble do not match, in CUDAEnsemble::setStepLog()\n");
    }
    // Set internal config
    step_log_config = std::make_shared<StepLoggingConfig>(stepConfig);
}
void CUDAEnsemble::setExitLog(const LoggingConfig &exitConfig) {
    // Validate ModelDescription matches
    if (*exitConfig.model != *model) {
      THROW exception::InvalidArgument("Model descriptions attached to LoggingConfig and CUDAEnsemble do not match, in CUDAEnsemble::setExitLog()\n");
    }
    // Set internal config
    exit_log_config = std::make_shared<LoggingConfig>(exitConfig);
}
const std::vector<RunLog> &CUDAEnsemble::getLogs() {
    return run_logs;
}

}  // namespace flamegpu
