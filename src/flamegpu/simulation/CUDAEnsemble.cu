#include "hip/hip_runtime.h"
#include "flamegpu/simulation/CUDAEnsemble.h"

#include <algorithm>
#include <cstdlib>
#include <memory>
#include <thread>
#include <set>
#include <queue>
#include <mutex>
#include <condition_variable>
#include <filesystem>
#include <map>

#ifdef FLAMEGPU_ENABLE_MPI
#include "flamegpu/simulation/detail/MPIEnsemble.h"
#include "flamegpu/simulation/detail/MPISimRunner.h"
#endif

#include "flamegpu/version.h"
#include "flamegpu/model/ModelDescription.h"
#include "flamegpu/simulation/RunPlanVector.h"
#include "flamegpu/detail/compute_capability.cuh"
#include "flamegpu/detail/SteadyClockTimer.h"
#include "flamegpu/simulation/CUDASimulation.h"
#include "flamegpu/io/StateWriterFactory.h"
#include "flamegpu/simulation/LoggingConfig.h"
#include "flamegpu/simulation/detail/SimRunner.h"
#include "flamegpu/simulation/LogFrame.h"
#include "flamegpu/simulation/detail/SimLogger.h"
#include "flamegpu/detail/cuda.cuh"
#include "flamegpu/io/Telemetry.h"

namespace flamegpu {
CUDAEnsemble::EnsembleConfig::EnsembleConfig()
    : telemetry(flamegpu::io::Telemetry::isEnabled()) {}


CUDAEnsemble::CUDAEnsemble(const ModelDescription& _model, int argc, const char** argv, bool _isSWIG)
    : model(_model.model->clone())
    , isSWIG(_isSWIG) {
    initialise(argc, argv);
}
CUDAEnsemble::~CUDAEnsemble() {
// Call this here incase simulate() exited with an exception
#ifdef _MSC_VER
    if (config.block_standby) {
        // Disable prevention of standby
        SetThreadExecutionState(ES_CONTINUOUS);
    }
#endif
}

unsigned int CUDAEnsemble::simulate(const RunPlanVector& plans) {
#ifdef _MSC_VER
    if (config.block_standby) {
        // This thread requires the system continuously until it exits
        SetThreadExecutionState(ES_CONTINUOUS | ES_SYSTEM_REQUIRED);
    }
#endif
    // Validate that RunPlan model matches CUDAEnsemble model
    if (*plans.environment != this->model->environment->properties) {
        THROW exception::InvalidArgument("RunPlan is for a different ModelDescription, in CUDAEnsemble::simulate()");
    }

#ifdef FLAMEGPU_ENABLE_MPI
    std::unique_ptr<detail::MPIEnsemble> mpi = std::make_unique<detail::MPIEnsemble>(config, static_cast<unsigned int>(plans.size()));
#endif

    // Validate/init output directories
    if (!config.out_directory.empty()
#ifdef FLAMEGPU_ENABLE_MPI
        && (!config.mpi || mpi->world_rank == 0)
#endif
    ) {
        // Validate out format is right
        config.out_format = io::StateWriterFactory::detectSupportedFileExt(config.out_format);
        if (config.out_format.empty()) {
            THROW exception::InvalidArgument("The out_directory config option also requires the out_format options to be set to a suitable type (e.g. 'json', 'xml'), in CUDAEnsemble::simulate()");
        }
        // Check that output files don't already exist
        if (std::filesystem::exists(config.out_directory)) {
            std::set<std::filesystem::path> exit_files;
            for (unsigned int p = 0; p < plans.size(); ++p) {
                std::filesystem::path exit_path = config.out_directory;
                if (!plans[p].getOutputSubdirectory().empty())
                    exit_path /= std::filesystem::path(plans[p].getOutputSubdirectory());
                exit_path /= std::filesystem::path("exit." + config.out_format);
                exit_files.insert(exit_path);
            }
            if (!config.truncate_log_files) {
                // Step
                for (unsigned int p = 0; p < plans.size(); ++p) {
                    std::filesystem::path step_path = config.out_directory;
                    if (!plans[p].getOutputSubdirectory().empty())
                        step_path /= std::filesystem::path(plans[p].getOutputSubdirectory());
                    step_path /= std::filesystem::path(std::to_string(p) + "." + config.out_format);
                    if (std::filesystem::exists(step_path)) {
                        THROW exception::FileAlreadyExists("Step log file '%s' already exists, in CUDAEnsemble::simulate()", step_path.generic_string().c_str());
                    }
                }
                // Exit
                for (const auto &exit_path : exit_files) {
                    if (std::filesystem::exists(exit_path)) {
                        THROW exception::FileAlreadyExists("Exit log file '%s' already exists, in CUDAEnsemble::simulate()", exit_path.generic_string().c_str());
                    }
                }
            } else {
                // Delete pre-existing exit log files
                for (const auto& exit_path : exit_files) {
                    std::filesystem::remove(exit_path);  // Returns false if the file didn't exist
                }
            }
        }
        // Create any missing directories
        try {
            std::filesystem::create_directories(config.out_directory);
        } catch (const std::exception &e) {
            THROW exception::InvalidArgument("Unable to use output directory '%s', in CUDAEnsemble::simulate(): %s", config.out_directory.c_str(), e.what());
        }
        for (const auto &p : plans) {
            const auto subdir = p.getOutputSubdirectory();
            if (!subdir.empty()) {
                std::filesystem::path sub_path = config.out_directory;
                try {
                    sub_path.append(subdir);
                    std::filesystem::create_directories(sub_path);
                } catch (const std::exception &e) {
                    THROW exception::InvalidArgument("Unable to use output subdirectory '%s', in CUDAEnsemble::simulate(): %s", sub_path.generic_string().c_str(), e.what());
                }
            }
        }
    }
    // Purge run logs, and resize ready for new runs
    // Resize means we can setup logs during execution out of order, without risk of list being reallocated
    run_logs.clear();
    // Workout how many devices and runner we will be executing
    // if MPI is enabled, This will throw exceptions if any rank has 0 GPUs visible, prior to device allocation preventing issues where rank 0 would not be participating.
    int device_count = -1;
    hipError_t cudaStatus = hipGetDeviceCount(&device_count);
    if (cudaStatus != hipSuccess) {
        THROW exception::InvalidCUDAdevice("Error finding CUDA devices!  Do you have a CUDA-capable GPU installed?, in CUDAEnsemble::simulate()");
    }
    if (device_count == 0) {
        THROW exception::InvalidCUDAdevice("Error no CUDA devices found!, in CUDAEnsemble::simulate()");
    }
    for (const int id : config.devices) {
        if (id >= device_count) {
            THROW exception::InvalidCUDAdevice("Requested CUDA device %d is not valid, only %d CUDA devices available!, in CUDAEnsemble::simulate()", id, device_count);
        }
    }

    // Select the actual devices to be used, based on user provided gpus, architecture compatibility, and optionally mpi ranks per node.
    // For non-mpi builds / configurations, just use all the devices provided by the user / all visible devices (then check they are valid later)
    // For MPI builds with mpi enabled, load balance the gpus across mpi ranks within the shared memory system. If there are more ranks than gpus, latter ranks will not participate.
    std::set<int> devices;
    // initialise the local devices set to be the non-mpi behaviour, using config.devices or all visible cuda devices
    if (config.devices.size()) {
        devices = config.devices;
    } else {
        // If no devices were specified by the user, use all visible devices but load balance if MPI is in use.
        for (int i = 0; i < device_count; ++i) {
            devices.emplace(i);
        }
    }
#ifdef FLAMEGPU_ENABLE_MPI
    // if MPI is enabled at compile time, use the MPIEnsemble method to assign devices balanced across ranks
    devices = mpi->devicesForThisRank(devices);
#endif  // ifdef FLAMEGPU_ENABLE_MPI

    // Check that each device is capable, and init cuda context
    for (auto d = devices.begin(); d != devices.end(); ++d) {
        if (!detail::compute_capability::checkComputeCapability(*d)) {
            fprintf(stderr, "FLAMEGPU2 has not been built with an appropriate compute capability for device %d, this device will not be used.\n", *d);
            d = devices.erase(d);
            --d;
        } else {
            gpuErrchk(hipSetDevice(*d));
            gpuErrchk(flamegpu::detail::cuda::hipFree(nullptr));
        }
    }
    // Return to device 0 (or check original device first?)
    gpuErrchk(hipSetDevice(0));

    // If there are no devices left (and mpi is not being used), we need to error as the work cannot be executed.
#ifndef FLAMEGPU_ENABLE_MPI
    if (devices.size() == 0) {
        THROW exception::InvalidCUDAdevice("FLAMEGPU2 has not been built with an appropraite compute capability for any devices, unable to continue\n");
    }
#endif  // ifndef FLAMEGPU_ENABLE_MPI

#ifdef FLAMEGPU_ENABLE_MPI
    // Once the number of devices per rank is known, we can create the actual communicator to be used during MPI, so we can warn/error as needed.
    // This rank is participating if it has atleast one device assigned to it.
    // Rank 0 will be participating at this point, otherwise InvalidCUDAdevice would have been thrown
    // This also implies the participating communicator cannot have a size of 0, as atleast one thread must be participating at this point, but throw in that case just in case.
    bool communicatorCreated = mpi->createParticipatingCommunicator(devices.size() > 0);
    // If the communicator failed to be created or is empty for any participating threads, throw. This should never occur.
    if (!communicatorCreated || mpi->getParticipatingCommSize() == 0) {
        THROW exception::EnsembleError("Unable to create MPI communicator. Ensure atleast one GPU is visible.\n");
    }
    // If the world size is not the participating size, issue a warning.that too many threads have been used.
    if (mpi->world_rank == 0 && mpi->world_size != mpi->getParticipatingCommSize() && config.verbosity >= Verbosity::Default) {
        fprintf(stderr, "Warning: MPI Ensemble launched with %d MPI ranks, but only %d ranks have GPUs assigned. %d ranks are unneccesary.\n", mpi->world_size, mpi->getParticipatingCommSize(), mpi->world_size - mpi->getParticipatingCommSize());
        fflush(stderr);
    }
#endif

    const unsigned int TOTAL_RUNNERS = static_cast<unsigned int>(devices.size()) * config.concurrent_runs;

    // Log Time (We can't use CUDA events here, due to device resets)
    auto ensemble_timer = detail::SteadyClockTimer();
    ensemble_timer.start();
    // Reset the elapsed time.
    ensemble_elapsed_time = 0.;

    // Logging thread-safety items
    std::queue<unsigned int> log_export_queue;
    std::mutex log_export_queue_mutex;
    std::condition_variable log_export_queue_cdn;
#ifdef FLAMEGPU_ENABLE_MPI
    // In MPI mode, Rank 0 will collect errors from all ranks
    std::multimap<int, detail::AbstractSimRunner::ErrorDetail> err_detail = {};
#endif
    std::vector<detail::AbstractSimRunner::ErrorDetail> err_detail_local = {};

    // Init log worker
    detail::SimLogger *log_worker = nullptr;
    if (!config.out_directory.empty()) {
        log_worker = new detail::SimLogger(run_logs, plans, config.out_directory, config.out_format, log_export_queue, log_export_queue_mutex, log_export_queue_cdn,
        step_log_config.get(), exit_log_config.get(), step_log_config && step_log_config->log_timing, exit_log_config && exit_log_config->log_timing);
    }

    // In MPI mode, only Rank 0 increments the error counter
    unsigned int err_count = 0;
    if (config.mpi) {
#ifdef FLAMEGPU_ENABLE_MPI
        // Setup MPISimRunners
        detail::MPISimRunner** runners = static_cast<detail::MPISimRunner**>(malloc(sizeof(detail::MPISimRunner*) * TOTAL_RUNNERS));
        std::vector<std::atomic<unsigned int>> err_cts(TOTAL_RUNNERS);
        std::vector<std::atomic<unsigned int>> next_runs(TOTAL_RUNNERS);
        for (unsigned int i = 0; i < TOTAL_RUNNERS; ++i) {
            err_cts[i] = UINT_MAX;
            next_runs[i] = detail::MPISimRunner::Signal::RequestJob;
        }
        {
            unsigned int i = 0;
            for (auto& d : devices) {
                for (unsigned int j = 0; j < config.concurrent_runs; ++j) {
                    runners[i] = new detail::MPISimRunner(model, err_cts[i], next_runs[i], plans,
                        step_log_config, exit_log_config,
                        d, j,
                        config.verbosity,
                        run_logs, log_export_queue, log_export_queue_mutex, log_export_queue_cdn, err_detail_local, TOTAL_RUNNERS, isSWIG);
                    runners[i]->start();
                    ++i;
                }
            }
        }
        // Wait for runners to request work, then communicate via MPI to get assignments
        // If work_rank == 0, also perform the assignments
        if (mpi->world_rank == 0) {
            unsigned int next_run = 0;
            MPI_Status status;
            int flag;
            int mpi_runners_fin = 1;  // Start at 1 because we have always already finished
            // Wait for all runs to have been assigned, and all MPI runners to have been notified of fin
            while (next_run < plans.size() || mpi_runners_fin < mpi->getParticipatingCommSize()) {
                // Check for errors
                const int t_err_count = mpi->receiveErrors(err_detail);
                err_count += t_err_count;
                if (t_err_count && config.error_level == EnsembleConfig::Fast) {
                    // Skip to end to kill workers
                    next_run = plans.size();
                }
                // Check whether local runners require a job assignment
                for (unsigned int i = 0; i < next_runs.size(); ++i) {
                    auto &r = next_runs[i];
                    unsigned int run_id = r.load();
                    if (run_id == detail::MPISimRunner::Signal::RunFailed) {
                        // Retrieve and handle local error detail
                        mpi->retrieveLocalErrorDetail(log_export_queue_mutex, err_detail, err_detail_local, i, devices);
                        ++err_count;
                        if (config.error_level == EnsembleConfig::Fast) {
                            // Skip to end to kill workers
                            next_run = plans.size();
                        }
                        run_id = detail::MPISimRunner::Signal::RequestJob;
                    }
                    if (run_id == detail::MPISimRunner::Signal::RequestJob) {
                        r.store(next_run++);
                        // Print progress to console
                        if (config.verbosity >= Verbosity::Default && next_run <= plans.size()) {
                            fprintf(stdout, "MPI ensemble assigned run %d/%u to rank 0\n", next_run, static_cast<unsigned int>(plans.size()));
                            fflush(stdout);
                        }
                    }
                }
                // Check whether MPI runners require a job assignment
                mpi_runners_fin += mpi->receiveJobRequests(next_run);
                // Yield, rather than hammering the processor
                std::this_thread::yield();
            }
        } else if (mpi->getRankIsParticipating()) {
            // Wait for all runs to have been assigned, and all MPI runners to have been notified of fin. ranks without GPU(s) do not request jobs.
            unsigned int next_run = 0;
            MPI_Status status;
            while (next_run < plans.size()) {
                // Check whether local runners require a job assignment
                for (unsigned int i = 0; i < TOTAL_RUNNERS; ++i) {
                    unsigned int runner_status = next_runs[i].load();
                    if (runner_status == detail::MPISimRunner::Signal::RunFailed) {
                        // Fetch the job id, increment local error counter
                        const unsigned int failed_run_id = err_cts[i].exchange(UINT_MAX);
                        ++err_count;
                        // Retrieve and handle local error detail
                        mpi->retrieveLocalErrorDetail(log_export_queue_mutex, err_detail, err_detail_local, i, devices);
                        runner_status = detail::MPISimRunner::Signal::RequestJob;
                    }
                    if (runner_status == detail::MPISimRunner::Signal::RequestJob) {
                        next_run = mpi->requestJob();
                        // Pass the job to runner that requested it
                        next_runs[i].store(next_run);
                        // Break if assigned job is out of range, work is finished
                        if (next_run >= plans.size()) {
                            break;
                        }
                    }
                }
                std::this_thread::yield();
            }
        }

        // Notify all local runners to exit
        for (unsigned int i = 0; i < TOTAL_RUNNERS; ++i) {
            auto &r = next_runs[i];
            if (r.exchange(plans.size()) == detail::MPISimRunner::Signal::RunFailed) {
                ++err_count;
                // Retrieve and handle local error detail
                mpi->retrieveLocalErrorDetail(log_export_queue_mutex, err_detail, err_detail_local, i, devices);
            }
        }
        // Wait for all runners to exit
        for (unsigned int i = 0; i < TOTAL_RUNNERS; ++i) {
            runners[i]->join();
            delete runners[i];
            if (next_runs[i].load() == detail::MPISimRunner::Signal::RunFailed) {
                ++err_count;
                // Retrieve and handle local error detail
                mpi->retrieveLocalErrorDetail(log_export_queue_mutex, err_detail, err_detail_local, i, devices);
            }
        }
#endif
    } else {
        detail::SimRunner** runners = static_cast<detail::SimRunner**>(malloc(sizeof(detail::SimRunner*) * TOTAL_RUNNERS));
        std::atomic<unsigned int> err_ct = { 0u };
        std::atomic<unsigned int> next_runs = { 0u };
        // Setup SimRunners
        {
            unsigned int i = 0;
            for (auto& d : devices) {
                for (unsigned int j = 0; j < config.concurrent_runs; ++j) {
                    runners[i] = new detail::SimRunner(model, err_ct, next_runs, plans,
                        step_log_config, exit_log_config,
                        d, j,
                        config.verbosity, config.error_level == EnsembleConfig::Fast,
                        run_logs, log_export_queue, log_export_queue_mutex, log_export_queue_cdn, err_detail_local, TOTAL_RUNNERS, isSWIG);
                    runners[i++]->start();
                }
            }
        }
        // Wait for all runners to exit
        for (unsigned int i = 0; i < TOTAL_RUNNERS; ++i) {
            runners[i]->join();
            delete runners[i];
        }
        err_count = err_ct;
    }

    // Notify logger to exit
    if (log_worker) {
        {
            std::lock_guard<std::mutex> lck(log_export_queue_mutex);
            log_export_queue.push(UINT_MAX);
        }
        log_export_queue_cdn.notify_one();
        log_worker->thread.join();
        delete log_worker;
        log_worker = nullptr;
    }

#ifdef FLAMEGPU_ENABLE_MPI
    std::string remote_device_names;
    if (config.mpi) {
        // Ensure all workers have finished before exit
        mpi->worldBarrier();
        // Check whether MPI runners have reported any final errors
        err_count += mpi->receiveErrors(err_detail);
        if (config.telemetry) {
            // All ranks should notify rank 0 of their GPU devices
            remote_device_names = mpi->assembleGPUsString();
        }
    }
#endif
    // Record and store the elapsed time
    ensemble_timer.stop();
    ensemble_elapsed_time = ensemble_timer.getElapsedSeconds();

    // Ensemble has finished, print summary
    if (config.verbosity > Verbosity::Quiet &&
#ifdef FLAMEGPU_ENABLE_MPI
        (!config.mpi || mpi->world_rank == 0) &&
#endif
       (config.error_level != EnsembleConfig::Fast || err_count == 0)) {
        printf("\rCUDAEnsemble completed %u runs successfully!\n", static_cast<unsigned int>(plans.size() - err_count));
        if (err_count)
            printf("There were a total of %u errors.\n", err_count);
    }
    if ((config.timing || config.verbosity >= Verbosity::Verbose) &&
#ifdef FLAMEGPU_ENABLE_MPI
    (!config.mpi || mpi->world_rank == 0) &&
#endif
       (config.error_level != EnsembleConfig::Fast || err_count == 0)) {
        printf("Ensemble time elapsed: %fs\n", ensemble_elapsed_time);
    }

    // Send Telemetry
    if (config.telemetry
#ifdef FLAMEGPU_ENABLE_MPI
       && (!config.mpi || mpi->world_rank == 0)
#endif
    ) {
        // Generate some payload items
        std::map<std::string, std::string> payload_items;
#ifndef FLAMEGPU_ENABLE_MPI
        payload_items["GPUDevices"] = flamegpu::detail::compute_capability::getDeviceNames(config.devices);
#else
        payload_items["GPUDevices"] = flamegpu::detail::compute_capability::getDeviceNames(config.devices) + remote_device_names;
#endif
        payload_items["SimTime(s)"] = std::to_string(ensemble_elapsed_time);
#if defined(__CUDACC_VER_MAJOR__) && defined(__CUDACC_VER_MINOR__) && defined(__CUDACC_VER_BUILD__)
        payload_items["NVCCVersion"] = std::to_string(__CUDACC_VER_MAJOR__) + "." + std::to_string(__CUDACC_VER_MINOR__) + "." + std::to_string(__CUDACC_VER_BUILD__);
#endif
        // Add the ensemble size to the ensemble telemetry payload
        payload_items["PlansSize"] = std::to_string(plans.size());
        payload_items["ConcurrentRuns"] = std::to_string(config.concurrent_runs);
        // Add MPI details to the ensemble telemetry payload
        payload_items["mpi"] = config.mpi ? "true" : "false";
#ifdef FLAMEGPU_ENABLE_MPI
        payload_items["mpi_world_size"] = std::to_string(mpi->world_size);
#endif
        // generate telemetry data
        std::string telemetry_data = flamegpu::io::Telemetry::generateData("ensemble-run", payload_items, isSWIG);
        // send the telemetry packet
        bool telemetrySuccess = flamegpu::io::Telemetry::sendData(telemetry_data);
        // If verbose, print either a successful send, or a misc warning.
        if (config.verbosity >= Verbosity::Verbose) {
            if (telemetrySuccess) {
                fprintf(stdout, "Telemetry packet sent to '%s' json was: %s\n", flamegpu::io::Telemetry::TELEMETRY_ENDPOINT, telemetry_data.c_str());
            } else {
                fprintf(stderr, "Warning: Usage statistics for CUDAEnsemble failed to send.\n");
            }
        }
    } else {
        // Encourage users who have opted out to opt back in, unless suppressed.
        if ((config.verbosity > Verbosity::Quiet)
#ifdef FLAMEGPU_ENABLE_MPI
            && (!config.mpi || mpi->world_rank == 0)
#endif
        ) {
            flamegpu::io::Telemetry::encourageUsage();
        }
    }

#ifdef FLAMEGPU_ENABLE_MPI
    if (config.mpi && mpi->world_rank != 0) {
        // All errors are reported via rank 0
        err_count = 0;
    }
#endif

    if (config.error_level == EnsembleConfig::Fast && err_count) {
        if (config.mpi) {
#ifdef FLAMEGPU_ENABLE_MPI
            for (const auto &e : err_detail) {
                THROW exception::EnsembleError("Run %u failed on rank %d, device %d, thread %u with exception: \n%s\n",
                    e.second.run_id, e.first, e.second.device_id, e.second.runner_id, e.second.exception_string);
            }
#endif
        }
        THROW exception::EnsembleError("Run %u failed on device %d, thread %u with exception: \n%s\n",
            err_detail_local[0].run_id, err_detail_local[0].device_id, err_detail_local[0].runner_id, err_detail_local[0].exception_string);
    } else if (config.error_level == EnsembleConfig::Slow && err_count) {
        THROW exception::EnsembleError("%u/%u runs failed!\n.", err_count, static_cast<unsigned int>(plans.size()));
    }
#ifdef _MSC_VER
    if (config.block_standby) {
        // Disable prevention of standby
        SetThreadExecutionState(ES_CONTINUOUS);
    }
#endif

    return err_count;
}

void CUDAEnsemble::initialise(int argc, const char** argv) {
    if (!checkArgs(argc, argv)) {
        exit(EXIT_FAILURE);
    }
    // If verbose, output the flamegpu version and seed.
    if (config.verbosity == Verbosity::Verbose) {
        fprintf(stdout, "FLAME GPU %s\n", flamegpu::VERSION_FULL);
        fprintf(stdout, "Ensemble configuration:\n");
        fprintf(stdout, "\tConcurrent runs: %u\n", config.concurrent_runs);
    }
}
int CUDAEnsemble::checkArgs(int argc, const char** argv) {
    // Parse optional args
    int i = 1;
    for (; i < argc; i++) {
        // Get arg as lowercase
        std::string arg(argv[i]);
        std::transform(arg.begin(), arg.end(), arg.begin(), [](unsigned char c) { return std::use_facet< std::ctype<char>>(std::locale()).tolower(c); });
        // -h/--help. Print the help output and exit.
        if (arg.compare("--help") == 0 || arg.compare("-h") == 0) {
            printHelp(argv[0]);
            return false;
        }
        // --concurrent <runs>, Number of concurrent simulations to run per device
        if (arg.compare("--concurrent") == 0 || arg.compare("-c") == 0) {
            if (i + 1 >= argc) {
                fprintf(stderr, "%s requires a trailing argument\n", arg.c_str());
                return false;
            }
            config.concurrent_runs = static_cast<unsigned int>(strtoul(argv[++i], nullptr, 0));
            continue;
        }
        // --devices <string>, comma separated list of uints
        if (arg.compare("--devices") == 0 || arg.compare("-d") == 0) {
            if (i + 1 >= argc) {
                fprintf(stderr, "%s requires a trailing argument\n", arg.c_str());
                return false;
            }
            // Split and parse string
            std::string device_string = argv[++i];
            device_string += ",";  // Append comma, to catch final item
            int max_id = 0;  // Catch max device so we can validate it exists
            size_t pos;
            while ((pos = device_string.find(",")) != std::string::npos) {
                const unsigned int id = static_cast<unsigned int>(strtoul(device_string.substr(0, pos).c_str(), nullptr, 0));
                if (id == 0 && (device_string.length() < 2 || (device_string[0] != '0' || device_string[1] != ','))) {
                    fprintf(stderr, "'%s' is not a valid device index.\n", device_string.substr(0, pos).c_str());
                    printHelp(argv[0]);
                    return false;
                }
                max_id = static_cast<int>(id) > max_id ? id : max_id;
                config.devices.emplace(id);
                device_string.erase(0, pos + 1);
            }
            int ct = -1;
            gpuErrchk(hipGetDeviceCount(&ct));
            if (max_id >= ct) {
                fprintf(stderr, "Device id %u exceeds available CUDA devices %d\n", max_id, ct);
                printHelp(argv[0]);
                return false;
            }
            continue;
        }
        // -o/--out <directory> <filetype>, Quiet FLAME GPU output.
        if (arg.compare("--out") == 0 || arg.compare("-o") == 0) {
            if (i + 2 >= argc) {
                fprintf(stderr, "%s requires two trailing arguments\n", arg.c_str());
                return false;
            }
            // Validate output directory is valid (and recursively create it if necessary)
            try {
                std::filesystem::path out_directory = argv[++i];
                std::filesystem::create_directories(out_directory);
                config.out_directory = out_directory.generic_string();
            } catch (const std::exception &e) {
                // Catch any exceptions, probably std::filesystem::filesystem_error, but other implementation defined errors also possible
                fprintf(stderr, "Unable to use '%s' as output directory:\n%s\n", argv[i], e.what());
                return false;
            }
            // Validate output format is available in io module
            config.out_format = io::StateWriterFactory::detectSupportedFileExt(argv[++i]);
            if (config.out_format.empty()) {
                fprintf(stderr, "'%s' is not a supported output file type.\n", argv[i]);
                return false;
            }
            continue;
        }
        // -q/--quiet, Don't report progress to console.
        if (arg.compare("--quiet") == 0 || arg.compare("-q") == 0) {
            config.verbosity = Verbosity::Quiet;
            continue;
        }
        // -v/--verbose, Report all progress to console.
        if (arg.compare("--verbose") == 0 || arg.compare("-v") == 0) {
            config.verbosity = Verbosity::Verbose;
            continue;
        }
        // -t/--timing, Output timing information to stdout
        if (arg.compare("--timing") == 0 || arg.compare("-t") == 0) {
            config.timing = true;
            continue;
        }
        // -u/--silence-unknown-args, Silence warning for unknown arguments
        if (arg.compare("--silence-unknown-args") == 0 || arg.compare("-u") == 0) {
            config.silence_unknown_args = true;
            continue;
        }
        // -e/--error, Specify the error level
        if (arg.compare("--error") == 0 || arg.compare("-e") == 0) {
            if (i + 1 >= argc) {
                fprintf(stderr, "%s requires a trailing argument\n", arg.c_str());
                return false;
            }
            std::string error_level_string = argv[++i];
            // Shift the trailing arg to lower
            std::transform(error_level_string.begin(), error_level_string.end(), error_level_string.begin(), [](unsigned char c) { return std::use_facet< std::ctype<char>>(std::locale()).tolower(c); });
            if (error_level_string.compare("off") == 0 || error_level_string.compare(std::to_string(EnsembleConfig::Off)) == 0) {
                config.error_level = EnsembleConfig::Off;
            } else if (error_level_string.compare("slow") == 0 || error_level_string.compare(std::to_string(EnsembleConfig::Slow)) == 0) {
                config.error_level = EnsembleConfig::Slow;
            } else if (error_level_string.compare("fast") == 0 || error_level_string.compare(std::to_string(EnsembleConfig::Fast)) == 0) {
                config.error_level = EnsembleConfig::Fast;
            } else {
                fprintf(stderr, "%s is not an appropriate argument for %s\n", error_level_string.c_str(), arg.c_str());
                return false;
            }
            continue;
        }
        // --truncate, Truncate output files
        if (arg.compare("--truncate") == 0) {
            config.truncate_log_files = true;
            continue;
        }
        // --standby Disable the blocking of standby
        if (arg.compare("--standby") == 0) {
#ifdef _MSC_VER
            config.block_standby = false;
#endif
            continue;
        }
        // Warning if not in QUIET verbosity or if silence-unknown-args is set
        if (!(config.verbosity == flamegpu::Verbosity::Quiet || config.silence_unknown_args))
            fprintf(stderr, "Warning: Unknown argument '%s' passed to Ensemble will be ignored\n", arg.c_str());
    }
    return true;
}
void CUDAEnsemble::printHelp(const char *executable) {
    printf("FLAME GPU %s\n", flamegpu::VERSION_FULL);
    printf("Usage: %s [optional arguments]\n", executable);
    printf("Optional Arguments:\n");
    const char *line_fmt = "%-18s %s\n";
    printf(line_fmt, "-h, --help", "show this help message and exit");
    printf(line_fmt, "-d, --devices <device ids>", "Comma separated list of device ids to be used");
    printf(line_fmt, "", "By default, all available devices will be used.");
    printf(line_fmt, "-c, --concurrent <runs>", "Number of concurrent simulations to run per device");
    printf(line_fmt, "", "By default, 4 will be used.");
    printf(line_fmt, "-o, --out <directory> <filetype>", "Directory and filetype for ensemble outputs");
    printf(line_fmt, "-q, --quiet", "Do not print progress information to console");
    printf(line_fmt, "-v, --verbose", "Print config, progress and timing (-t) information to console");
    printf(line_fmt, "-t, --timing", "Output timing information to stdout");
    printf(line_fmt, "-e, --error <error level>", "The error level 0, 1, 2, off, slow or fast");
    printf(line_fmt, "", "By default, \"slow\" will be used.");
    printf(line_fmt, "-u, --silence-unknown-args", "Silence warnings for unknown arguments passed after this flag.");
#ifdef _MSC_VER
    printf(line_fmt, "    --standby", "Allow the machine to enter standby during execution");
#endif
}
void CUDAEnsemble::setStepLog(const StepLoggingConfig &stepConfig) {
    // Validate ModelDescription matches
    if (*stepConfig.model != *model) {
      THROW exception::InvalidArgument("Model descriptions attached to LoggingConfig and CUDAEnsemble do not match, in CUDAEnsemble::setStepLog()\n");
    }
    // Set internal config
    step_log_config = std::make_shared<StepLoggingConfig>(stepConfig);
}
void CUDAEnsemble::setExitLog(const LoggingConfig &exitConfig) {
    // Validate ModelDescription matches
    if (*exitConfig.model != *model) {
      THROW exception::InvalidArgument("Model descriptions attached to LoggingConfig and CUDAEnsemble do not match, in CUDAEnsemble::setExitLog()\n");
    }
    // Set internal config
    exit_log_config = std::make_shared<LoggingConfig>(exitConfig);
}
const std::map<unsigned int, RunLog> &CUDAEnsemble::getLogs() {
    return run_logs;
}
}  // namespace flamegpu
