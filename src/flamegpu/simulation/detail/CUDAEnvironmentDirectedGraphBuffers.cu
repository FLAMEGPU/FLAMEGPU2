#include "hip/hip_runtime.h"
#include "flamegpu/simulation/detail/CUDAEnvironmentDirectedGraphBuffers.cuh"

#include <algorithm>

#include "flamegpu/simulation/detail/CUDAAgent.h"
#include "flamegpu/simulation/detail/CUDAErrorChecking.cuh"
#include "flamegpu/simulation/detail/CUDAScatter.cuh"
#include "flamegpu/runtime/detail/curve/HostCurve.cuh"
#include "flamegpu/detail/cuda.cuh"
#ifdef _MSC_VER
#pragma warning(push, 1)
#pragma warning(disable : 4706 4834)
#endif  // _MSC_VER
#ifdef __NVCC_DIAG_PRAGMA_SUPPORT__
#pragma nv_diag_suppress 1719
#else
#pragma diag_suppress 1719
#endif  // __NVCC_DIAG_PRAGMA_SUPPORT__
#include <hipcub/hipcub.hpp>

#ifdef __NVCC_DIAG_PRAGMA_SUPPORT__
#pragma nv_diag_default 1719
#else
#pragma diag_default 1719
#endif  // __NVCC_DIAG_PRAGMA_SUPPORT__
#ifdef _MSC_VER
#pragma warning(pop)
#endif  // _MSC_VER

namespace flamegpu {
namespace detail {

CUDAEnvironmentDirectedGraphBuffers::CUDAEnvironmentDirectedGraphBuffers(const EnvironmentDirectedGraphData& description)
    : graph_description(description)
    , vertex_count(0)
    , edge_count(0)
    , requires_rebuild(false) {
    // Initialise the maps
    for (const auto &v : graph_description.vertexProperties)
        vertex_buffers.emplace(v.first, Buffer{v.second.type_size * v.second.elements});
    for (const auto& e : graph_description.edgeProperties)
        edge_buffers.emplace(e.first, Buffer{e.second.type_size * e.second.elements });
}
CUDAEnvironmentDirectedGraphBuffers::~CUDAEnvironmentDirectedGraphBuffers() {
    deallocateVertexBuffers();
    vertex_buffers.clear();
    deallocateEdgeBuffers();
    edge_buffers.clear();
}
void CUDAEnvironmentDirectedGraphBuffers::registerCurveInstance(const std::shared_ptr<detail::curve::HostCurve>& curve) {
    curve_instances.push_back(std::weak_ptr(curve));
}
void CUDAEnvironmentDirectedGraphBuffers::registerCurveInstance(const std::shared_ptr<detail::curve::CurveRTCHost>& curve) {
    rtc_curve_instances.push_back(std::weak_ptr(curve));
}
void CUDAEnvironmentDirectedGraphBuffers::allocateVertexBuffers(const size_type count, const hipStream_t stream) {
    for (auto& v : graph_description.vertexProperties) {
        auto &vb = vertex_buffers.at(v.first);
        if (!vb.d_ptr) {
            gpuErrchk(hipMalloc(&vb.d_ptr, count * v.second.type_size * v.second.elements));
            // gpuErrchk(hipMalloc(&vb.d_ptr_swap, count * v.second.type_size * v.second.elements));  // Todo: required?
            for (const auto & _curve : curve_instances) {
                if (const auto curve = _curve.lock())
                    curve->setEnvironmentDirectedGraphVertexProperty(graph_description.name, v.first, vb.d_ptr, count);
            }
            for (const auto& _curve : rtc_curve_instances) {
                if (const auto curve = _curve.lock()) {
                    memcpy(curve->getEnvironmentDirectedGraphVertexPropertyCachePtr(graph_description.name, v.first), &vb.d_ptr, sizeof(void*));
                    curve->setEnvironmentDirectedGraphVertexPropertyCount(graph_description.name, v.first, count);
                }
            }
        } else {
            THROW exception::UnknownInternalError("Device buffer already allocated, in CUDAEnvironmentDirectedGraphBuffers::allocateVertexBuffers()");
        }
        if (!vb.h_ptr) {
            vb.h_ptr = malloc(count * v.second.type_size * v.second.elements);
        } else {
            THROW exception::UnknownInternalError("Host buffer already allocated, in CUDAEnvironmentDirectedGraphBuffers::allocateVertexBuffers()");
        }
        vb.ready = Buffer::Both;
    }
    // Min length 4, as pbm_swap is used when building graph
    gpuErrchk(hipMalloc(&d_pbm, sizeof(unsigned int) * std::max<int>(count + 1, 4)));
    gpuErrchk(hipMalloc(&d_pbm_swap, sizeof(unsigned int) * std::max<int>(count + 1, 4)));
    gpuErrchk(hipMalloc(&d_ipbm, sizeof(unsigned int)* std::max<int>(count + 1, 4)));
    // Initialise PBMs incase they doesn't contain edges
    gpuErrchk(hipMemsetAsync(d_pbm, 0, (count + 1) * sizeof(unsigned int), stream));
    gpuErrchk(hipMemsetAsync(d_ipbm, 0, (count + 1) * sizeof(unsigned int), stream));
    for (const auto& _curve : curve_instances) {
        if (const auto curve = _curve.lock()) {
            curve->setEnvironmentDirectedGraphVertexProperty(graph_description.name, GRAPH_VERTEX_PBM_VARIABLE_NAME, d_pbm, 1);
            curve->setEnvironmentDirectedGraphVertexProperty(graph_description.name, GRAPH_VERTEX_IPBM_VARIABLE_NAME, d_ipbm, 1);
            curve->setEnvironmentDirectedGraphVertexProperty(graph_description.name, GRAPH_VERTEX_IPBM_EDGES_VARIABLE_NAME, d_ipbm, 1);  // IPBM needs to point somewhere
            curve->setEnvironmentDirectedGraphVertexProperty(graph_description.name, GRAPH_VERTEX_INDEX_MAP_VARIABLE_NAME, d_vertex_index_map, 1);  // ID map needs to point somewhere
        }
    }
    for (const auto& _curve : rtc_curve_instances) {
        if (const auto curve = _curve.lock()) {
            memcpy(curve->getEnvironmentDirectedGraphVertexPropertyCachePtr(graph_description.name, GRAPH_VERTEX_PBM_VARIABLE_NAME), &d_pbm, sizeof(void*));
            memcpy(curve->getEnvironmentDirectedGraphVertexPropertyCachePtr(graph_description.name, GRAPH_VERTEX_IPBM_VARIABLE_NAME), &d_ipbm, sizeof(void*));
            memcpy(curve->getEnvironmentDirectedGraphVertexPropertyCachePtr(graph_description.name, GRAPH_VERTEX_IPBM_EDGES_VARIABLE_NAME), &d_ipbm, sizeof(void*));  // IPBM needs to point somewhere
            memcpy(curve->getEnvironmentDirectedGraphVertexPropertyCachePtr(graph_description.name, GRAPH_VERTEX_INDEX_MAP_VARIABLE_NAME), &d_ipbm, sizeof(void*));  // ID map needs to point somewhere
            curve->setEnvironmentDirectedGraphVertexPropertyCount(graph_description.name, GRAPH_VERTEX_INDEX_MAP_VARIABLE_NAME, 1);  // 1 because offset is packed at the end
        }
    }
    vertex_count = count;
}
void CUDAEnvironmentDirectedGraphBuffers::allocateEdgeBuffers(const size_type count) {
    for (auto& e : graph_description.edgeProperties) {
        auto& eb = edge_buffers.at(e.first);
        if (!eb.d_ptr) {
            gpuErrchk(hipMalloc(&eb.d_ptr, count * e.second.type_size * e.second.elements));
            gpuErrchk(hipMalloc(&eb.d_ptr_swap, count * e.second.type_size * e.second.elements));
            for (const auto& _curve : curve_instances) {
                if (const auto curve = _curve.lock())
                    curve->setEnvironmentDirectedGraphEdgeProperty(graph_description.name, e.first, eb.d_ptr, count);
            }
            for (const auto& _curve : rtc_curve_instances) {
                if (const auto curve = _curve.lock()) {
                    memcpy(curve->getEnvironmentDirectedGraphEdgePropertyCachePtr(graph_description.name, e.first), &eb.d_ptr, sizeof(void*));
                    curve->setEnvironmentDirectedGraphEdgePropertyCount(graph_description.name, e.first, count);
                }
            }
        } else {
            THROW exception::UnknownInternalError("Device buffer already allocated, in CUDAEnvironmentDirectedGraphBuffers::allocateEdgeBuffers()");
        }
        if (!eb.h_ptr) {
            eb.h_ptr = malloc(count * e.second.type_size * e.second.elements);
        } else {
            THROW exception::UnknownInternalError("Host buffer already allocated, in CUDAEnvironmentDirectedGraphBuffers::allocateEdgeBuffers()");
        }
        eb.ready = Buffer::Both;
    }
    gpuErrchk(hipMalloc(&d_keys, sizeof(uint64_t) * count));
    gpuErrchk(hipMalloc(&d_keys_swap, sizeof(uint64_t) * count));
    gpuErrchk(hipMalloc(&d_vals, sizeof(uint32_t) * (count + 1)));
    gpuErrchk(hipMalloc(&d_vals_swap, sizeof(uint32_t) * (count + 1)));
    gpuErrchk(hipMalloc(&d_ipbm_edges, sizeof(uint32_t) * (count + 1)));
    for (const auto& _curve : curve_instances) {
        if (const auto curve = _curve.lock()) {
            curve->setEnvironmentDirectedGraphVertexProperty(graph_description.name, GRAPH_VERTEX_IPBM_EDGES_VARIABLE_NAME, d_ipbm_edges, 1);
        }
    }
    for (const auto& _curve : rtc_curve_instances) {
        if (const auto curve = _curve.lock()) {
            memcpy(curve->getEnvironmentDirectedGraphVertexPropertyCachePtr(graph_description.name, GRAPH_VERTEX_IPBM_EDGES_VARIABLE_NAME), &d_ipbm_edges, sizeof(void*));
            curve->setEnvironmentDirectedGraphVertexPropertyCount(graph_description.name, GRAPH_VERTEX_INDEX_MAP_VARIABLE_NAME, 1);  // 1 because offset is packed at the end
        }
    }
    edge_count = count;
}
void CUDAEnvironmentDirectedGraphBuffers::deallocateVertexBuffers() {
    for (auto& v : vertex_buffers) {
        if (v.second.d_ptr) {
            gpuErrchk(flamegpu::detail::cuda::hipFree(v.second.d_ptr));
            gpuErrchk(flamegpu::detail::cuda::hipFree(v.second.d_ptr_swap));
            v.second.d_ptr = nullptr;
        }
        if (v.second.h_ptr) {
            free(v.second.h_ptr);
            v.second.h_ptr = nullptr;
        }
    }
    if (d_pbm) {
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_pbm));
        d_pbm = nullptr;
    }
    if (d_pbm_swap) {
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_pbm_swap));
        d_pbm_swap = nullptr;
    }
    if (d_ipbm) {
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_ipbm));
        d_ipbm = nullptr;
    }
    if (d_vertex_index_map) {
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_vertex_index_map));
        d_vertex_index_map = nullptr;
    }
    vertex_count = 0;
    h_vertex_index_map.clear();
}
void CUDAEnvironmentDirectedGraphBuffers::deallocateEdgeBuffers() {
    for (auto& e : edge_buffers) {
        if (e.second.d_ptr) {
            gpuErrchk(flamegpu::detail::cuda::hipFree(e.second.d_ptr));
            gpuErrchk(flamegpu::detail::cuda::hipFree(e.second.d_ptr_swap));
            e.second.d_ptr = nullptr;
        }
        if (e.second.h_ptr) {
            free(e.second.h_ptr);
            e.second.h_ptr = nullptr;
        }
    }
    if (d_keys) {
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_keys));
        d_keys = nullptr;
    }
    if (d_vals) {
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_vals));
        d_vals = nullptr;
    }
    if (d_keys_swap) {
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_keys_swap));
        d_keys_swap = nullptr;
    }
    if (d_vals_swap) {
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_vals_swap));
        d_vals_swap = nullptr;
    }
    if (d_ipbm_edges) {
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_ipbm_edges));
        d_ipbm_edges = nullptr;
    }
    edge_count = 0;
    h_edge_index_map.clear();
}

void CUDAEnvironmentDirectedGraphBuffers::setVertexCount(const size_type count, const hipStream_t stream) {
    if (vertex_count) {
        deallocateVertexBuffers();
    }
    allocateVertexBuffers(count, stream);
    // Default Init host, mark device out of date
    for (auto& v : graph_description.vertexProperties) {
        auto& vb = vertex_buffers.at(v.first);
        vb.ready = Buffer::Host;
        if (v.first == ID_VARIABLE_NAME) {  // ID needs default 0
            memset(vb.h_ptr, ID_NOT_SET, vertex_count * v.second.type_size * v.second.elements);
            continue;
        }
        // Possibly faster if we checked default_value == 0 and memset, but awkward with vague type and lack of template
        for (unsigned int i = 0; i < vertex_count; ++i) {
            // TODO is this just copy-paste junk?
            memcpy(static_cast<char*>(vb.h_ptr) + i * v.second.type_size * v.second.elements, v.second.default_value, v.second.type_size * v.second.elements);
        }
    }
    // Vertex data has been reset, so ID bounds are nolonger valid
    resetVertexIDBounds();
}

void CUDAEnvironmentDirectedGraphBuffers::setEdgeCount(const size_type count) {
    if (edge_count)
        deallocateEdgeBuffers();
    allocateEdgeBuffers(count);
    // Default Init host, mark device out of date
    for (auto& e : graph_description.edgeProperties) {
        auto& eb = edge_buffers.at(e.first);
        eb.ready = Buffer::Host;
        // Possibly faster if we checked default_value == 0 and memset, but awkward with vague type and lack of template
        for (unsigned int i = 0; i < edge_count; ++i) {
            // TODO is this just copy-paste junk?
            memcpy(static_cast<char*>(eb.h_ptr) + i * e.second.type_size * e.second.elements, e.second.default_value, e.second.type_size * e.second.elements);
        }
    }
}
id_t* CUDAEnvironmentDirectedGraphBuffers::getVertexIDBuffer(const hipStream_t stream) {
    size_type element_ct = 1;
    return getVertexPropertyBuffer<id_t>(ID_VARIABLE_NAME, element_ct, stream);
}

__global__ void fillKVPairs(uint32_t *keys, uint32_t *vals, const unsigned int *srcdest, unsigned int count, const unsigned int *idMap, const unsigned int id_offset) {
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < count) {
        // To subsort by destination too, we treat the pair of uint32 as a uint64
        keys[index * 2 + 0] = idMap[srcdest[index * 2 + 0] - id_offset];
        keys[index * 2 + 1] = idMap[srcdest[index * 2 + 1] - id_offset];
        vals[index] = index;
    }
}
__global__ void fillKVPairs_inverted(uint32_t* keys, uint32_t* vals, const unsigned int* srcdest, unsigned int count, const unsigned int *idMap, const unsigned int id_offset) {
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < count) {
        // To subsort by destination too, we treat the pair of uint32 as a uint64
        // To invert we must switch the order of the contained uint32's
        keys[index * 2 + 0] = idMap[srcdest[index * 2 + 1] - id_offset];
        keys[index * 2 + 1] = idMap[srcdest[index * 2 + 0] - id_offset];
        vals[index] = index;
    }
}
__global__ void findBinStart(unsigned int *pbm, uint64_t* keys, unsigned int edge_count, unsigned int vertex_count) {
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < edge_count) {
        // Bins correspond to the first uint32 of the pair
        const uint32_t my_vertex = reinterpret_cast<const uint32_t*>(keys)[(index * 2)+1];
        if (index == 0 || my_vertex != reinterpret_cast<const uint32_t*>(keys)[((index - 1) * 2)+1]) {
            // Store the Index of the first edges for the corresponding vertex
            pbm[my_vertex] = index;
        }
    }
    // 1 thread must init the final cell of the PBM
    if (index == 0) {
        pbm[vertex_count] = edge_count;
    }
}
/**
* This utility class provides a wrapper for `unsigned int *`
* It causes the pointer to iterate in reverse backwards
*/
struct ReverseIterator {
    using difference_type = unsigned int;
    using value_type = unsigned int;
    using pointer = unsigned int*;
    using reference = unsigned int&;
    using iterator_category = std::random_access_iterator_tag;
    __host__ __device__ explicit ReverseIterator(unsigned int* _p) : p(_p) { }

    // __device__ ReverseIterator& operator=(const ReverseIterator& other) = default;
    __device__ ReverseIterator operator++ (int a) { p -= a;  return *this; }
    __device__ ReverseIterator operator++ () { p--;  return *this; }
    __device__ unsigned int &operator *() const { return *p; }
    __device__ ReverseIterator operator+(const int& b) const { return ReverseIterator(p - b); }
    __device__ unsigned int &operator[](int b) const { return *(p-b); }
    unsigned int* p;
};
// Borrowed from CUB DeviceScan docs
struct CustomMin {
    template <typename T>
    HIPCUB_RUNTIME_FUNCTION __forceinline__
        T operator()(const T& a, const T& b) const {
        return (b < a) ? b : a;
    }
};
__global__ void buildIDMap(const id_t *IDsIn, unsigned int *indexOut, const unsigned int count, unsigned int *error_count, unsigned int vertex_id_min, unsigned int vertex_id_max) {
    const unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_index < count) {
        id_t my_thread_id = IDsIn[thread_index];
        // Skip IDs that weren't set
        if (my_thread_id == ID_NOT_SET) {
            atomicInc(error_count + 2, UINT_MAX);
            return;
        }
        if (vertex_id_min <= my_thread_id && my_thread_id <= vertex_id_max) {
            const unsigned int rtn = atomicExch(indexOut + my_thread_id - vertex_id_min, thread_index);
            if (rtn != 0xffffffff) {
                // Report ID collision
                atomicInc(error_count + 0, UINT_MAX);
            }
        } else {
            // Report out of range ID (this should not happen, it's an internal error if it does)
            atomicInc(error_count + 1, UINT_MAX);
        }
    }
}
__global__ void validateSrcDest(id_t *edgeSrcDest, unsigned int *idMap, const unsigned int edge_count, unsigned int *errors, unsigned int vertex_id_min, unsigned int vertex_id_max) {
    const unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_index < edge_count) {
        const id_t my_src_id = edgeSrcDest[thread_index * 2 + 1];
        const id_t my_dest_id = edgeSrcDest[thread_index * 2 + 0];
        if (my_src_id == ID_NOT_SET) {
            atomicInc(errors + 0, UINT_MAX);
        } else if (vertex_id_min <= my_src_id && my_src_id <= vertex_id_max) {
            if (idMap[my_src_id - vertex_id_min] == 0xffffffff) {
                atomicInc(errors + 2, UINT_MAX);
            }
        } else {
            atomicInc(errors + 2, UINT_MAX);
        }
        if (my_dest_id == ID_NOT_SET) {
            atomicInc(errors + 1, UINT_MAX);
        } else if (vertex_id_min <= my_dest_id && my_dest_id <= vertex_id_max) {
            if (idMap[my_dest_id - vertex_id_min] == 0xffffffff) {
                atomicInc(errors + 3, UINT_MAX);
            }
        } else {
            atomicInc(errors + 3, UINT_MAX);
        }
    }
}
__global__ void translateSrcDest(id_t *edgeSrcDest, unsigned int *idMap, const unsigned int edge_count, unsigned int *errors, unsigned int vertex_id_min, unsigned int vertex_id_max) {
    const unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_index < edge_count) {
        const id_t my_src_id = edgeSrcDest[thread_index * 2 + 1];
        const id_t my_dest_id = edgeSrcDest[thread_index * 2 + 0];
        const unsigned int src_id = idMap[my_src_id - vertex_id_min];
        const unsigned int dest_id = idMap[my_dest_id - vertex_id_min];
        edgeSrcDest[thread_index * 2 + 1] = src_id;
        edgeSrcDest[thread_index * 2 + 0] = dest_id;
    }
}
void CUDAEnvironmentDirectedGraphBuffers::syncDevice_async(detail::CUDAScatter& scatter, const unsigned int streamID, const hipStream_t stream) {
    // Copy variable buffers to device
    if (vertex_count) {
        for (auto& v : graph_description.vertexProperties) {
            auto& vb = vertex_buffers.at(v.first);
            if (vb.ready == Buffer::Host) {
                gpuErrchk(hipMemcpyAsync(vb.d_ptr, vb.h_ptr, vertex_count * v.second.type_size * v.second.elements, hipMemcpyHostToDevice, stream));
                vb.ready = Buffer::Both;
            }
        }
    }
    if (edge_count) {
        for (auto& e : graph_description.edgeProperties) {
            auto& eb = edge_buffers.at(e.first);
            if (eb.ready == Buffer::Host) {
                gpuErrchk(hipMemcpyAsync(eb.d_ptr, eb.h_ptr, edge_count * e.second.type_size * e.second.elements, hipMemcpyHostToDevice, stream));
                eb.ready = Buffer::Both;
            }
        }
    }
    if (requires_rebuild && vertex_count && edge_count) {
        // Construct the vertex ID : index map
        {
            if (vertex_id_min == std::numeric_limits<unsigned int>::max() || vertex_id_max == std::numeric_limits<unsigned int>::min()) {
                THROW flamegpu::exception::IDOutOfBounds("No IDs have been set, in CUDAEnvironmentDirectedGraphBuffers::syncDevice_async()");
            }
            const unsigned int ID_RANGE = 1 + vertex_id_max - vertex_id_min;
            if (ID_RANGE < vertex_count) {
                THROW flamegpu::exception::IDNotSet("Not all vertices have been assigned a unique ID, in CUDAEnvironmentDirectedGraphBuffers::syncDevice_async()");
            }
            if (d_vertex_index_map) {
                gpuErrchk(flamegpu::detail::cuda::hipFree(d_vertex_index_map));
            }
            if (hipMalloc(&d_vertex_index_map, sizeof(unsigned int) * (ID_RANGE + 1)) != hipSuccess) {
                THROW flamegpu::exception::OutOfMemory("Out of memory when allocating ID->index map, Vertex IDs cover too wide a range (%u) consider contiguous IDs, in CUDAEnvironmentDirectedGraphBuffers::syncDevice_async()", ID_RANGE);
            }
            // Copy the offset to the end of the map
            gpuErrchk(hipMemcpyAsync(d_vertex_index_map + ID_RANGE, &vertex_id_min, sizeof(unsigned int), hipMemcpyHostToDevice, stream));
            // Add the ID->index map var to curve
            for (const auto& _curve : curve_instances) {
                if (const auto curve = _curve.lock())
                    curve->setEnvironmentDirectedGraphVertexProperty(graph_description.name, GRAPH_VERTEX_INDEX_MAP_VARIABLE_NAME, d_vertex_index_map, ID_RANGE + 1);  // +1 because offset is packed at the end
            }
            for (const auto& _curve : rtc_curve_instances) {
                if (const auto curve = _curve.lock()) {
                    memcpy(curve->getEnvironmentDirectedGraphVertexPropertyCachePtr(graph_description.name, GRAPH_VERTEX_INDEX_MAP_VARIABLE_NAME), &d_vertex_index_map, sizeof(void*));
                    curve->setEnvironmentDirectedGraphVertexPropertyCount(graph_description.name, GRAPH_VERTEX_INDEX_MAP_VARIABLE_NAME, ID_RANGE + 1);  // +1 because offset is packed at the end
                }
            }
            {  // Build the map
                const auto& v_id_b = vertex_buffers.at(ID_VARIABLE_NAME);
                gpuErrchk(hipMemsetAsync(d_vertex_index_map, 0xffffffff, ID_RANGE * sizeof(unsigned int), stream));
                gpuErrchk(hipMemsetAsync(d_pbm_swap, 0, 3 * sizeof(unsigned int), stream));  // We will use spare pbm_swap to count errors, save allocating more memory
                const unsigned int BLOCK_SZ = 512;
                const unsigned int BLOCK_CT = static_cast<unsigned int>(ceil(vertex_count / static_cast<float>(BLOCK_SZ)));
                buildIDMap << <BLOCK_CT, BLOCK_SZ, 0, stream >> > (static_cast<id_t*>(v_id_b.d_ptr), d_vertex_index_map, vertex_count, d_pbm_swap, vertex_id_min, vertex_id_max);
                gpuErrchkLaunch();
                unsigned int err_collision_range[3];
                gpuErrchk(hipMemcpyAsync(err_collision_range, d_pbm_swap, 3 * sizeof(unsigned int), hipMemcpyDeviceToHost, stream));
                gpuErrchk(hipStreamSynchronize(stream));
                if (err_collision_range[2] > 0) {
                    THROW flamegpu::exception::IDNotSet("Graph contains %u vertices which have not had their ID set, in CUDAEnvironmentDirectedGraphBuffers::syncDevice_async()", err_collision_range[2]);
                } else if (err_collision_range[0] > 0) {
                    THROW flamegpu::exception::IDCollision("Graph contains invalid vertex IDs, %u vertices reported ID collisions, vertex IDs must be unique or unset, in CUDAEnvironmentDirectedGraphBuffers::syncDevice_async()", err_collision_range[0]);
                } else if (err_collision_range[1] > 0) {
                    THROW flamegpu::exception::UnknownInternalError("Graph contains invalid vertex IDs, %u vertices reported an ID that does not satisfy %u < ID < %u, in CUDAEnvironmentDirectedGraphBuffers::syncDevice_async()", err_collision_range[1], vertex_id_min, vertex_id_max);
                }
            }
            {  // Validate that edge source/dest pairs correspond to valid IDs
                const auto& e_srcdest_b = edge_buffers.at(GRAPH_SOURCE_DEST_VARIABLE_NAME);
                gpuErrchk(hipMemsetAsync(d_pbm_swap, 0, 4 * sizeof(unsigned int), stream));  // We will use spare pbm_swap to count errors, save allocating more memory
                const unsigned int BLOCK_SZ = 512;
                const unsigned int BLOCK_CT = static_cast<unsigned int>(ceil(edge_count / static_cast<float>(BLOCK_SZ)));
                validateSrcDest << <BLOCK_CT, BLOCK_SZ, 0, stream >> > (static_cast<id_t*>(e_srcdest_b.d_ptr), d_vertex_index_map, edge_count, d_pbm_swap, vertex_id_min, vertex_id_max);
                gpuErrchkLaunch();
                unsigned int err_collision_range[4];  // {src_notset, dest_notset, src_invalid, dest_invalid}
                gpuErrchk(hipMemcpyAsync(err_collision_range, d_pbm_swap, 4 * sizeof(unsigned int), hipMemcpyDeviceToHost, stream));
                gpuErrchk(hipStreamSynchronize(stream));
                if (err_collision_range[0] > 0 || err_collision_range[1] > 0) {
                    THROW flamegpu::exception::IDNotSet("Graph contains %u and %u edges which have not had their source and destinations set respectively, in CUDAEnvironmentDirectedGraphBuffers::syncDevice_async()", err_collision_range[0], err_collision_range[1]);
                } else if (err_collision_range[2] > 0 || err_collision_range[3] > 0) {
                    THROW flamegpu::exception::InvalidID("Graph contains %u and %u edges which have invalid source and destinations set respectively, in CUDAEnvironmentDirectedGraphBuffers::syncDevice_async()", err_collision_range[2], err_collision_range[3]);
                }
            }
        }
        // Rebuild the CSR/VBM (edgesLeaving())
        {
            // Fill Key/Val Pairs
            int blockSize;  // The launch configurator returned block size
            gpuErrchk(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, fillKVPairs, 32, 0));  // Randomly 32
            int gridSize = (edge_count + blockSize - 1) / blockSize;  // Round up according to array size
            fillKVPairs << <gridSize, blockSize, 0, stream >> > (reinterpret_cast<uint32_t*>(d_keys), d_vals, static_cast<unsigned int*>(edge_buffers.at(GRAPH_SOURCE_DEST_VARIABLE_NAME).d_ptr), edge_count, d_vertex_index_map, vertex_id_min);
            gpuErrchkLaunch();
            // Sort Key/Val Pairs according to src->dest
            auto& cub_temp = scatter.CubTemp(streamID);
            size_t temp_req = 0;
            gpuErrchk(hipcub::DeviceRadixSort::SortPairs(nullptr, temp_req, d_keys, d_keys_swap, d_vals, d_vals_swap, edge_count, 0, sizeof(uint64_t) * 8, stream));
            cub_temp.resize(temp_req);
            gpuErrchk(hipcub::DeviceRadixSort::SortPairs(cub_temp.getPtr(), cub_temp.getSize(), d_keys, d_keys_swap, d_vals, d_vals_swap, edge_count, 0, sizeof(uint64_t) * 8, stream));
            // Build PBM (For vertices with edges)
            gpuErrchk(hipMemset(d_pbm, 0xffffffff, (vertex_count + 1) * sizeof(unsigned int)));
            gpuErrchk(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, findBinStart, 32, 0));  // Randomly 32
            gridSize = (edge_count + blockSize - 1) / blockSize;  // Round up according to array size
            findBinStart << <gridSize, blockSize, 0, stream >> > (d_pbm, d_keys_swap, edge_count, vertex_count);
            gpuErrchkLaunch();
            // Build PBM (Fill vertices with no edges)
            temp_req = 0;
            gpuErrchk(hipcub::DeviceScan::InclusiveScan(nullptr, temp_req, ReverseIterator(d_pbm + vertex_count), ReverseIterator(d_pbm_swap + vertex_count), CustomMin(), vertex_count + 1, stream));
            cub_temp.resize(temp_req);
            gpuErrchk(hipcub::DeviceScan::InclusiveScan(cub_temp.getPtr(), cub_temp.getSize(), ReverseIterator(d_pbm + vertex_count), ReverseIterator(d_pbm_swap + vertex_count), CustomMin(), vertex_count + 1, stream));
            // Sort edge variables
            std::vector<detail::CUDAScatter::ScatterData> sd;
            for (auto& edge : edge_buffers) {
                edge.second.swap();
                sd.push_back(detail::CUDAScatter::ScatterData{edge.second.element_size, reinterpret_cast<char*>(edge.second.d_ptr_swap), reinterpret_cast<char*>(edge.second.d_ptr)});
            }
            scatter.scatterPosition_async(streamID, stream, d_vals_swap, sd, edge_count);
            // Swap all the swap pointers, so the junk data is in swap
            std::swap(d_keys, d_keys_swap);
            std::swap(d_vals, d_vals_swap);
            std::swap(d_pbm, d_pbm_swap);
            // Update which buffers curve points to
            for (auto& e : graph_description.edgeProperties) {
                auto& eb = edge_buffers.at(e.first);
                for (const auto& _curve : curve_instances) {
                    if (const auto curve = _curve.lock())
                        curve->setEnvironmentDirectedGraphEdgeProperty(graph_description.name, e.first, eb.d_ptr, edge_count);
                }
                for (const auto& _curve : rtc_curve_instances) {
                    if (const auto curve = _curve.lock())
                        memcpy(curve->getEnvironmentDirectedGraphEdgePropertyCachePtr(graph_description.name, e.first), &eb.d_ptr, sizeof(void*));
                }
                eb.ready = Buffer::Device;
            }
            for (const auto& _curve : curve_instances) {
                if (const auto curve = _curve.lock())
                    curve->setEnvironmentDirectedGraphVertexProperty(graph_description.name, GRAPH_VERTEX_PBM_VARIABLE_NAME, d_pbm, 1);
            }
            for (const auto& _curve : rtc_curve_instances) {
                if (const auto curve = _curve.lock())
                    memcpy(curve->getEnvironmentDirectedGraphVertexPropertyCachePtr(graph_description.name, GRAPH_VERTEX_PBM_VARIABLE_NAME), &d_pbm, sizeof(void*));
            }
        }
        {  // Rebuild the CSC/Inverted VBM (edgesJoining())
            int blockSize;  // The launch configurator returned block size
            gpuErrchk(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, fillKVPairs, 32, 0));  // Randomly 32
            int gridSize = (edge_count + blockSize - 1) / blockSize;  // Round up according to array size
            fillKVPairs_inverted << <gridSize, blockSize, 0, stream >> > (reinterpret_cast<uint32_t*>(d_keys), d_vals, static_cast<unsigned int*>(edge_buffers.at(GRAPH_SOURCE_DEST_VARIABLE_NAME).d_ptr), edge_count, d_vertex_index_map, vertex_id_min);
            gpuErrchkLaunch();
            // Sort Key/Val Pairs according to dest->src
            // Cub temp has already been resized above
            auto& cub_temp = scatter.CubTemp(streamID);
            gpuErrchk(hipcub::DeviceRadixSort::SortPairs(cub_temp.getPtr(), cub_temp.getSize(), d_keys, d_keys_swap, d_vals, d_vals_swap, edge_count, 0, sizeof(uint64_t) * 8, stream));
            // Build inverted PBM (For vertices with edges)
            gpuErrchk(hipMemset(d_ipbm, 0xffffffff, (vertex_count + 1) * sizeof(unsigned int)));
            gpuErrchk(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, findBinStart, 32, 0));  // Randomly 32
            gridSize = (edge_count + blockSize - 1) / blockSize;  // Round up according to array size
            findBinStart << <gridSize, blockSize, 0, stream >> > (d_ipbm, d_keys_swap, edge_count, vertex_count);
            gpuErrchkLaunch();
            // Build inverted PBM (Fill vertices with no edges)
            gpuErrchk(hipcub::DeviceScan::InclusiveScan(cub_temp.getPtr(), cub_temp.getSize(), ReverseIterator(d_ipbm + vertex_count), ReverseIterator(d_pbm_swap + vertex_count), CustomMin(), vertex_count + 1, stream));
            // Swap all the swap pointers, so the junk data is in swap
            std::swap(d_keys, d_keys_swap);
            std::swap(d_ipbm_edges, d_vals_swap);
            std::swap(d_ipbm, d_pbm_swap);
            // Update which buffers curve points to
            for (const auto& _curve : curve_instances) {
                if (const auto curve = _curve.lock()) {
                    curve->setEnvironmentDirectedGraphVertexProperty(graph_description.name, GRAPH_VERTEX_IPBM_VARIABLE_NAME, d_ipbm, 1);
                    curve->setEnvironmentDirectedGraphVertexProperty(graph_description.name, GRAPH_VERTEX_IPBM_EDGES_VARIABLE_NAME, d_ipbm_edges, 1);
                }
            }
            for (const auto& _curve : rtc_curve_instances) {
                if (const auto curve = _curve.lock()) {
                    memcpy(curve->getEnvironmentDirectedGraphVertexPropertyCachePtr(graph_description.name, GRAPH_VERTEX_IPBM_VARIABLE_NAME), &d_ipbm, sizeof(void*));
                    memcpy(curve->getEnvironmentDirectedGraphVertexPropertyCachePtr(graph_description.name, GRAPH_VERTEX_IPBM_EDGES_VARIABLE_NAME), &d_ipbm_edges, sizeof(void*));
                }
            }
        }
        {  // Translate edge source/dest pairs and validate that they correspond to valid IDs
            const auto& e_srcdest_b = edge_buffers.at(GRAPH_SOURCE_DEST_VARIABLE_NAME);
            e_srcdest_b.updateHostBuffer(edge_count, stream);  // Copy back to host, before we translate device IDs
            const unsigned int BLOCK_SZ = 512;
            const unsigned int BLOCK_CT = static_cast<unsigned int>(ceil(edge_count / static_cast<float>(BLOCK_SZ)));
            translateSrcDest << <BLOCK_CT, BLOCK_SZ, 0, stream >> > (static_cast<id_t*>(e_srcdest_b.d_ptr), d_vertex_index_map, edge_count, d_pbm_swap, vertex_id_min, vertex_id_max);
            gpuErrchkLaunch()
            // Rebuild the edge index map
            h_edge_index_map.clear();
            for (unsigned int i = 0; i < edge_count; ++i) {
                h_edge_index_map.emplace(std::pair{static_cast<id_t*>(e_srcdest_b.h_ptr)[i * 2 + 1], static_cast<id_t*>(e_srcdest_b.h_ptr)[i * 2 + 0]}, i);
            }
        }
        requires_rebuild = false;
    }
}

void CUDAEnvironmentDirectedGraphBuffers::Buffer::updateHostBuffer(size_type edge_count, hipStream_t stream) const {
    if (ready == Device) {
        gpuErrchk(hipMemcpyAsync(h_ptr, d_ptr, edge_count * element_size, hipMemcpyDeviceToHost, stream));
        gpuErrchk(hipStreamSynchronize(stream));
        ready = Both;
    }
}
void CUDAEnvironmentDirectedGraphBuffers::resetVertexIDBounds() {
    vertex_id_min = std::numeric_limits<unsigned int>::max();
    vertex_id_max = std::numeric_limits<unsigned int>::min();
}
void CUDAEnvironmentDirectedGraphBuffers::setVertexID(unsigned int vertex_index, id_t vertex_id, hipStream_t stream) {
    if (vertex_index >= vertex_count) {
        THROW exception::OutOfBoundsException("Vertex index exceeds bounds %u >= %u, "
            "in CUDAEnvironmentDirectedGraphBuffers::setVertexID()\n", vertex_index, vertex_count);
    } else if (vertex_id == ID_NOT_SET) {
        THROW exception::IDOutOfBounds("Vertex ID of %u is not valid, "
            "in CUDAEnvironmentDirectedGraphBuffers::setVertexID()\n", ID_NOT_SET);
    }
    // Purge old vertex ID from host map
    auto& vb = vertex_buffers.at(ID_VARIABLE_NAME);
    vb.updateHostBuffer(vertex_count, stream);
    if (static_cast<id_t*>(vb.h_ptr)[vertex_index] != ID_NOT_SET) {
        h_vertex_index_map.erase(static_cast<id_t*>(vb.h_ptr)[vertex_index]);
    }

    // Add new vertex ID to host map (validate it's not already in us)
    const auto find = h_vertex_index_map.find(vertex_id);
    if (find != h_vertex_index_map.end()) {
        THROW exception::IDCollision("ID collision, %u has already been assigned to vertex at index %u, "
            "in CUDAEnvironmentDirectedGraphBuffers::setVertexID()\n", vertex_id, find->second);
    }
    h_vertex_index_map.emplace(vertex_id, vertex_index);

    // Update vertex's ID in buffer
    static_cast<id_t*>(vb.h_ptr)[vertex_index] = vertex_id;
    vb.ready = Buffer::Host;

    // Update range calc (naive, can be wrong if IDs are changed)
    vertex_id_min = std::min(vertex_id_min, vertex_id);
    vertex_id_max = std::max(vertex_id_max, vertex_id);
}
unsigned int CUDAEnvironmentDirectedGraphBuffers::getVertexIndex(id_t vertex_id) const {
    const auto find = h_vertex_index_map.find(vertex_id);
    if (find == h_vertex_index_map.end()) {
        THROW exception::InvalidID("No vertex found with ID %u, in CUDAEnvironmentDirectedGraphBuffers::getVertexIndex()\n", vertex_id);
    }
    return find->second;
}
void CUDAEnvironmentDirectedGraphBuffers::setEdgeSourceDestination(unsigned int edge_index, id_t src_vertex_id, id_t dest_vertex_id) {
    if (edge_index >= edge_count) {
        THROW exception::OutOfBoundsException("Edge index exceeds bounds %u >= %u, "
            "in CUDAEnvironmentDirectedGraphBuffers::setEdgeSourceDestination()\n", edge_index, edge_count);
    } else if (src_vertex_id == ID_NOT_SET) {
        THROW exception::IDOutOfBounds("Source vertex ID of %u is not valid, "
            "in CUDAEnvironmentDirectedGraphBuffers::setEdgeSourceDestination()\n", ID_NOT_SET);
    } else if (dest_vertex_id == ID_NOT_SET) {
        THROW exception::IDOutOfBounds("Destination vertex ID of %u is not valid, "
            "in CUDAEnvironmentDirectedGraphBuffers::setEdgeSourceDestination()\n", ID_NOT_SET);
    }
    // Purge old edge src/dest from host map
    auto& eb = edge_buffers.at(GRAPH_SOURCE_DEST_VARIABLE_NAME);
    // Don't need to update buffer, src_dest is not stored as ID on device
    id_t& edge_dest = static_cast<id_t*>(eb.h_ptr)[edge_index * 2 + 0];
    id_t& edge_src = static_cast<id_t*>(eb.h_ptr)[edge_index * 2 + 1];

    if (edge_src != ID_NOT_SET && edge_dest != ID_NOT_SET) {
        h_edge_index_map.erase({edge_src, edge_dest});
    } else if ((edge_src == ID_NOT_SET) ^ (edge_dest == ID_NOT_SET)) {
        THROW exception::UnknownInternalError("Edge found without both source and destination set, "
            "in CUDAEnvironmentDirectedGraphBuffers::setEdgeSourceDestination()\n");
    }

    // Add new edge ID to host map (validate it's not already in use)
    const auto find = h_edge_index_map.find({src_vertex_id, dest_vertex_id});
    if (find != h_edge_index_map.end()) {
        THROW exception::IDCollision("Edge collision, an edge has already been assigned source %u dest %u at index %u, "
            "in CUDAEnvironmentDirectedGraphBuffers::setEdgeSourceDestination()\n", src_vertex_id, dest_vertex_id, find->second);
    }
    h_edge_index_map.emplace(std::pair{src_vertex_id, dest_vertex_id}, edge_index);

    // Update edge's src dest in buffer
    edge_dest = dest_vertex_id;
    edge_src = src_vertex_id;
    eb.ready = Buffer::Host;

    // Require rebuild before use
    markForRebuild();
}
unsigned int CUDAEnvironmentDirectedGraphBuffers::getEdgeIndex(id_t src_vertex_id, id_t dest_vertex_id) const {
    const auto find = h_edge_index_map.find({src_vertex_id, dest_vertex_id});
    if (find == h_edge_index_map.end()) {
        THROW exception::InvalidID("No edge found with source %u, dest %u, in CUDAEnvironmentDirectedGraphBuffers::getEdgeIndex()\n", src_vertex_id, dest_vertex_id);
    }
    return find->second;
}

unsigned int CUDAEnvironmentDirectedGraphBuffers::createIfNotExistVertex(id_t vertex_id, const hipStream_t stream) {
    if (vertex_id == ID_NOT_SET) {
        THROW exception::IDOutOfBounds("Vertex ID of %u is not valid, "
            "in CUDAEnvironmentDirectedGraphBuffers::createIfNotExistVertex()\n", ID_NOT_SET);
    }
    const auto it = h_vertex_index_map.find(vertex_id);
    if (it != h_vertex_index_map.end()) {
        return it->second;
    }
    if (h_vertex_index_map.size() < vertex_count) {
        const unsigned int vertex_index = static_cast<unsigned int>(h_vertex_index_map.size());
        h_vertex_index_map.emplace(vertex_id, vertex_index);
        // Update vertex's ID in buffer
        auto& vb = vertex_buffers.at(ID_VARIABLE_NAME);
        vb.updateHostBuffer(vertex_count, stream);
        static_cast<id_t*>(vb.h_ptr)[vertex_index] = vertex_id;
        vb.ready = Buffer::Host;
        // Update range calc
        vertex_id_min = std::min(vertex_id_min, vertex_id);
        vertex_id_max = std::max(vertex_id_max, vertex_id);
        return vertex_index;
    }
    THROW exception::OutOfBoundsException("Creating vertex with ID %u would exceed available vertices (%u), "
        "in CUDAEnvironmentDirectedGraphBuffers::createIfNotExistVertex()\n", vertex_id, vertex_count);
}
unsigned int CUDAEnvironmentDirectedGraphBuffers::createIfNotExistEdge(id_t source_vertex_id, id_t dest_vertex_id, const hipStream_t stream) {
    if (source_vertex_id == ID_NOT_SET || dest_vertex_id== ID_NOT_SET) {
        THROW exception::IDOutOfBounds("Vertex ID of %u is not valid, "
            "in CUDAEnvironmentDirectedGraphBuffers::createIfNotExistEdge()\n", ID_NOT_SET);
    }
    const auto it = h_edge_index_map.find({source_vertex_id, dest_vertex_id});
    if (it != h_edge_index_map.end()) {
        return it->second;
    }
    if (h_edge_index_map.size() < edge_count) {
        const unsigned int edge_index = static_cast<unsigned int>(h_edge_index_map.size());
        h_edge_index_map.emplace(std::pair{source_vertex_id, dest_vertex_id}, edge_index);
        // Update vertex's ID in buffer
        auto& eb = edge_buffers.at(GRAPH_SOURCE_DEST_VARIABLE_NAME);
        static_cast<id_t*>(eb.h_ptr)[edge_index * 2 + 0] = dest_vertex_id;
        static_cast<id_t*>(eb.h_ptr)[edge_index * 2 + 1] = source_vertex_id;
        eb.ready = Buffer::Host;
        // Require rebuild before use
        markForRebuild();
        return edge_index;
    }
    THROW exception::OutOfBoundsException("Creating edge with src %u dest %u would exceed available edges (%u), "
        "in CUDAEnvironmentDirectedGraphBuffers::createIfNotExistEdge()\n", source_vertex_id, dest_vertex_id, vertex_count);
}
}  // namespace detail
}  // namespace flamegpu
