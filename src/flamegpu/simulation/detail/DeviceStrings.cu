#include "flamegpu/simulation/detail/DeviceStrings.h"

#include "flamegpu/detail/cuda.cuh"
#include "flamegpu/simulation/detail/CUDAErrorChecking.cuh"

namespace flamegpu {
namespace detail {

DeviceStrings::~DeviceStrings() {
    gpuErrchk(detail::cuda::hipFree(device_buffer));
}
void DeviceStrings::registerDeviceString(const std::string &host_string) {
    if (offsets.find(host_string) == offsets.end()) {
        offsets.emplace(host_string, host_buffer.size());
        host_stream << host_string;
        host_stream << '\0';  // Each string requires a null terminating char
        host_buffer = host_stream.str();
    }
}
const char* DeviceStrings::getDeviceString(const std::string &host_string) {
    if (offsets.find(host_string) == offsets.end()) {
        registerDeviceString(host_string);
    }
    const size_t host_buffer_len = host_buffer.size();
    const ptrdiff_t device_string_offset = offsets.at(host_string);
    // Reallocate device buffer if necessary
    if (!device_buffer || device_buffer_len < host_buffer_len) {
        // Double buffer len in size
        device_buffer_len = device_buffer_len == 0 ? 1024 : device_buffer_len * 2;
        gpuErrchk(hipFree(device_buffer));
        gpuErrchk(hipMalloc(&device_buffer, device_buffer_len));
        device_buffer_occupied = 0;
    }
    // Update device buffer if necessary
    if (device_buffer_occupied < host_buffer_len) {
        gpuErrchk(hipMemcpy(device_buffer, host_buffer.c_str(), host_buffer_len, hipMemcpyHostToDevice));
        device_buffer_occupied = host_buffer_len;
    }
    // Return
    return device_buffer + device_string_offset;
}

}  // namespace detail
}  // namespace flamegpu
