#include "hip/hip_runtime.h"
#include "flamegpu/simulation/detail/RandomManager.cuh"

#include <hip/hip_runtime.h>
#include <>

#include<ctime>

#include <cassert>
#include <cstdio>
#include <algorithm>

#include "flamegpu/detail/hiprand.cuh"
#include "flamegpu/simulation/detail/CUDAErrorChecking.cuh"
#include "flamegpu/simulation/CUDASimulation.h"
#include "flamegpu/detail/cuda.cuh"

namespace flamegpu {
namespace detail {

RandomManager::RandomManager() :
    deviceInitialised(false) {
    reseed(static_cast<uint64_t>(seedFromTime() % UINT_MAX));
}
RandomManager::~RandomManager() {
    free();  // @todo call free/freeDevice not in the constructor! instead just log that?
}
/**
 * Member fns
 */
uint64_t RandomManager::seedFromTime() {
    return static_cast<uint64_t>(time(nullptr));
}

void RandomManager::reseedHost() {
    freeHost();
    host_rng = std::mt19937_64();
    // Reset host random generator/s
    host_rng.seed(mSeed);
}

void RandomManager::reseedDevice() {
    freeDevice();
    // hiprand is initialised on access if length does not match. This would need a second device length?
}

void RandomManager::reseed(const uint64_t seed) {
    // Set the instance's seed to the new value
    mSeed = seed;

    // Apply the new seed to the host
    reseedHost();
    // Apply the new seed to the device.
    reseedDevice();
}

void RandomManager::freeHost() {
    // Release host_max
    if (h_max_random_state) {
        std::free(h_max_random_state);
        h_max_random_state = nullptr;
    }
    h_max_random_size = 0;
}

void RandomManager::freeDevice() {
    // Clear size - length is just for the device portion?
    length = 0;

    if (deviceInitialised) {
        // Set the device's internal size to 0.
        length = 0;
        // Release old random states on the deivce and update pointers.
        if (d_random_state) {
            gpuErrchk(flamegpu::detail::cuda::hipFree(d_random_state));
        }
        d_random_state = nullptr;
    }
}

void RandomManager::free() {
    // Free the host and device.
    freeHost();
    freeDevice();
}

detail::hiprandState *RandomManager::resize(size_type _length, hipStream_t stream) {
    assert(growthModifier > 1.0);
    assert(shrinkModifier > 0.0);
    assert(shrinkModifier <= 1.0);
    auto t_length = length;
    if (length) {
        while (t_length < _length) {
            t_length = static_cast<flamegpu::size_type>(t_length * growthModifier);
            if (shrinkModifier < 1.0f) {
                while (t_length * shrinkModifier > _length) {
                    t_length = static_cast<flamegpu::size_type>(t_length * shrinkModifier);
                }
            }
        }
    } else {  // Special case for first run
        t_length = _length;
    }
    // Don't allow array to go below RandomManager::min_length elements
    t_length = std::max<size_type>(t_length, RandomManager::min_length);
    if (t_length != length)
        resizeDeviceArray(t_length, stream);
    return d_random_state;
}
__global__ void init_curand(detail::hiprandState *d_random_state, unsigned int threadCount, uint64_t seed, flamegpu::size_type offset) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < threadCount)
        hiprand_init(seed, offset + id, 0, &d_random_state[offset + id]);
}
void RandomManager::resizeDeviceArray(const size_type _length, hipStream_t stream) {
    // Mark that the device hsa now been initialised.
    deviceInitialised = true;
    if (_length > h_max_random_size) {
        // Growing array
        detail::hiprandState *t_hd_random_state = nullptr;
        // Allocate new mem to t_hd
        gpuErrchk(hipMalloc(&t_hd_random_state, _length * sizeof(detail::hiprandState)));
        // Copy hd->t_hd[****    ]
        if (d_random_state) {
            gpuErrchk(hipMemcpyAsync(t_hd_random_state, d_random_state, length * sizeof(detail::hiprandState), hipMemcpyDeviceToDevice, stream));
        }
        // Update pointers hd=t_hd
        if (d_random_state) {
            gpuErrchk(flamegpu::detail::cuda::hipFree(d_random_state));
        }
        d_random_state = t_hd_random_state;
        // Init new[    ****]
        if (h_max_random_size > length) {
            // We have part/all host backup, copy to device array
            // Reinit backup[    **  ]
            const size_type copy_len = std::min(h_max_random_size, _length);
            gpuErrchk(hipMemcpyAsync(d_random_state + length, h_max_random_state + length, copy_len * sizeof(detail::hiprandState), hipMemcpyHostToDevice, stream));  // Host not pinned
            length += copy_len;
        }
        if (_length > length) {
            // Init remainder[     **]
            unsigned int initThreads = 512;
            unsigned int initBlocks = ((_length - length) / initThreads) + 1;
            init_curand<<<initBlocks, initThreads, 0,  stream>>>(d_random_state, _length - length, mSeed, length);  // This could be async with above memcpy in diff stream
            gpuErrchkLaunch();
        }
    } else {
        // Shrinking array
        detail::hiprandState *t_hd_random_state = nullptr;
        detail::hiprandState *t_h_max_random_state = nullptr;
        // Allocate new
        gpuErrchk(hipMalloc(&t_hd_random_state, _length * sizeof(detail::hiprandState)));
        // Allocate host backup
        if (length > h_max_random_size)
            t_h_max_random_state = reinterpret_cast<detail::hiprandState*>(malloc(length * sizeof(detail::hiprandState)));
        else
            t_h_max_random_state = h_max_random_state;
        // Copy old->new
        assert(d_random_state);
        gpuErrchk(hipMemcpyAsync(t_hd_random_state, d_random_state, _length * sizeof(detail::hiprandState), hipMemcpyDeviceToDevice, stream));
        // Copy part being shrunk away to host storage (This could be async with above memcpy?)
        gpuErrchk(hipMemcpyAsync(t_h_max_random_state + _length, d_random_state + _length, (length - _length) * sizeof(detail::hiprandState), hipMemcpyDeviceToHost, stream));
        // Release and replace old host ptr
        if (length > h_max_random_size) {
            if (h_max_random_state)
                ::free(h_max_random_state);
            h_max_random_state = t_h_max_random_state;
            h_max_random_size = length;
        }
        // Release old
        if (d_random_state != nullptr) {
            gpuErrchk(flamegpu::detail::cuda::hipFree(d_random_state));
        }
        // Update pointer
        d_random_state = t_hd_random_state;
    }
    // Update length
    length = _length;
    gpuErrchk(hipStreamSynchronize(stream));
}
void RandomManager::setGrowthModifier(float _growthModifier) {
    assert(growthModifier > 1.0);
    RandomManager::growthModifier = _growthModifier;
}
float RandomManager::getGrowthModifier() {
    return RandomManager::growthModifier;
}
void RandomManager::setShrinkModifier(float _shrinkModifier) {
    assert(shrinkModifier > 0.0);
    assert(shrinkModifier <= 1.0);
    RandomManager::shrinkModifier = _shrinkModifier;
}
float RandomManager::getShrinkModifier() {
    return RandomManager::shrinkModifier;
}
flamegpu::size_type RandomManager::size() {
    return length;
}
uint64_t RandomManager::seed() {
    return mSeed;
}
detail::hiprandState *RandomManager::cudaRandomState() {
    return d_random_state;
}

}  // namespace detail
}  // namespace flamegpu
