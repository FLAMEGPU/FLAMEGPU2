#include "flamegpu/simulation/detail/CubTemporaryMemory.cuh"
#include <hip/hip_runtime.h>

#include <cassert>

#include "flamegpu/simulation/detail/CUDAErrorChecking.cuh"
#include "flamegpu/util/nvtx.h"
#include "flamegpu/detail/cuda.cuh"

namespace flamegpu {
namespace detail {

CubTemporaryMemory::CubTemporaryMemory()
  : d_cub_temp(nullptr)
  , d_cub_temp_size(0) { }
CubTemporaryMemory::~CubTemporaryMemory() {
    // @todo - cuda is not allowed in destructor
    if (d_cub_temp) {
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_cub_temp));
        d_cub_temp_size = 0;
    }
}
void CubTemporaryMemory::resize(const size_t newSize) {
    if (newSize > d_cub_temp_size) {
        flamegpu::util::nvtx::Range range{"CubTemporaryMemory::resizeTempStorage"};
        if (d_cub_temp) {
            gpuErrchk(flamegpu::detail::cuda::hipFree(d_cub_temp));
        }
        gpuErrchk(hipMalloc(&d_cub_temp, newSize));
        d_cub_temp_size = newSize;
    }
}

}  // namespace detail
}  // namespace flamegpu
