#include "hip/hip_runtime.h"
#include "flamegpu/simulation/detail/SimLogger.h"

#include <filesystem>

#include "flamegpu/io/LoggerFactory.h"
#include "flamegpu/simulation/RunPlanVector.h"

#ifdef _MSC_VER
#include <windows.h>
#else
#include <pthread.h>
#endif

namespace flamegpu {
namespace detail {

SimLogger::SimLogger(const std::vector<RunLog> &_run_logs,
        const RunPlanVector &_run_plans,
        const std::string &_out_directory,
        const std::string &_out_format,
        std::queue<unsigned int> &_log_export_queue,
        std::mutex &_log_export_queue_mutex,
        std::condition_variable &_log_export_queue_cdn,
        bool _export_step,
        bool _export_exit,
        bool _export_step_time,
        bool _export_exit_time)
    : run_logs(_run_logs)
    , run_plans(_run_plans)
    , out_directory(_out_directory)
    , out_format(_out_format)
    , log_export_queue(_log_export_queue)
    , log_export_queue_mutex(_log_export_queue_mutex)
    , log_export_queue_cdn(_log_export_queue_cdn)
    , export_step(_export_step)
    , export_exit(_export_exit)
    , export_step_time(_export_step_time)
    , export_exit_time(_export_exit_time) {
    this->thread = std::thread(&SimLogger::start, this);
    // Attempt to name the thread
#ifdef _MSC_VER
    std::wstringstream thread_name;
    thread_name << L"SimLogger";
    // HRESULT hr =
    SetThreadDescription(this->thread.native_handle(), thread_name.str().c_str());
    // if (FAILED(hr)) {
    //     fprintf(stderr, "Failed to name thread 'SimLogger'\n");
    // }
#else
    std::stringstream thread_name;
    thread_name << "SimLogger";
    // int hr =
    pthread_setname_np(this->thread.native_handle(), thread_name.str().c_str());
    // if (hr) {
    //     fprintf(stderr, "Failed to name thread 'SimLogger'\n");
    // }
#endif
}
void SimLogger::start() {
    const std::filesystem::path p_out_directory = out_directory;
    unsigned int logs_processed = 0;
    while (logs_processed < run_plans.size()) {
        std::unique_lock<std::mutex> lock(log_export_queue_mutex);
        log_export_queue_cdn.wait(lock, [this]{ return !log_export_queue.empty(); });
        do {
            // Pop item to be logged from queue
            const unsigned int target_log = log_export_queue.front();
            log_export_queue.pop();
            lock.unlock();

            // Check item isn't telling us to exit early
            if (target_log == UINT_MAX) {
                logs_processed = UINT_MAX;
                break;
            }
            // Log items
            if (export_exit) {
                const std::filesystem::path exit_path = p_out_directory / std::filesystem::path(run_plans[target_log].getOutputSubdirectory()) / std::filesystem::path("exit." + out_format);
                const auto exit_logger = io::LoggerFactory::createLogger(exit_path.generic_string(), false, false);
                exit_logger->log(run_logs[target_log], run_plans[target_log], false, true, false, export_exit_time);
            }
            if (export_step) {
                const std::filesystem::path step_path = p_out_directory/std::filesystem::path(run_plans[target_log].getOutputSubdirectory())/std::filesystem::path(std::to_string(target_log)+"."+out_format);
                const auto step_logger = io::LoggerFactory::createLogger(step_path.generic_string(), false, true);
                step_logger->log(run_logs[target_log], run_plans[target_log], true, false, export_step_time, false);
            }
            // Continue
            ++logs_processed;
            lock.lock();
        } while (!log_export_queue.empty());
    }
}

}  // namespace detail
}  // namespace flamegpu
