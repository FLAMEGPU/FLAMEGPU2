#include "flamegpu/simulation/detail/MPISimRunner.h"

#include <utility>

#include "flamegpu/model/ModelData.h"
#include "flamegpu/simulation/CUDASimulation.h"
#include "flamegpu/simulation/RunPlanVector.h"

#ifdef _MSC_VER
#include <windows.h>
#else
#include <pthread.h>
#endif

namespace flamegpu {
namespace detail {

MPISimRunner::MPISimRunner(const std::shared_ptr<const ModelData> _model,
    std::atomic<unsigned int>& _err_ct,
    std::atomic<unsigned int>& _next_run,
    const RunPlanVector& _plans,
    std::shared_ptr<const StepLoggingConfig> _step_log_config,
    std::shared_ptr<const LoggingConfig> _exit_log_config,
    int _device_id,
    unsigned int _runner_id,
    flamegpu::Verbosity _verbosity,
    std::map<unsigned int, RunLog>& _run_logs,
    std::queue<unsigned int>& _log_export_queue,
    std::mutex& _log_export_queue_mutex,
    std::condition_variable& _log_export_queue_cdn,
    std::vector<ErrorDetail>& _err_detail_local,
    const unsigned int _total_runners,
    bool _isSWIG)
    : AbstractSimRunner(
        _model,
        _err_ct,
        _next_run,
        _plans,
        _step_log_config,
        _exit_log_config,
        _device_id,
        _runner_id,
        _verbosity,
        _run_logs,
        _log_export_queue,
        _log_export_queue_mutex,
        _log_export_queue_cdn,
        _err_detail_local,
        _total_runners,
        _isSWIG)
    { }

void MPISimRunner::main() {
    // While there are still plans to process
    while (true) {
        const unsigned int run_id = next_run.load();
        if (run_id < plans.size()) {
            // Process the assigned job
            try {
                runSimulation(run_id);
                if (next_run.exchange(Signal::RequestJob) >= plans.size()) {
                    break;
                }
                // MPI Worker's don't print progress
            } catch(std::exception &e) {
                // log_export_mutex is treated as our protection for race conditions on err_detail
                std::lock_guard<std::mutex> lck(log_export_queue_mutex);
                // Build the error detail (fixed len char array for string)
                // fprintf(stderr, "Fail: run: %u device: %u, runner: %u\n", run_id, device_id, runner_id);  // useful debug, breaks tests
                err_detail.push_back(ErrorDetail{run_id, static_cast<unsigned int>(device_id), runner_id, });
                strncpy(err_detail.back().exception_string, e.what(), sizeof(ErrorDetail::exception_string)-1);
                err_detail.back().exception_string[sizeof(ErrorDetail::exception_string) - 1] = '\0';
                err_ct.store(static_cast<int>(err_detail.size()));
                // Need to notify manager that run failed
                if (next_run.exchange(Signal::RunFailed) >= plans.size()) {
                    break;
                }
            }
        } else if (run_id == Signal::RequestJob || run_id == Signal::RunFailed) {
            std::this_thread::yield();
        } else {
            break;
        }
    }
}

}  // namespace detail
}  // namespace flamegpu
