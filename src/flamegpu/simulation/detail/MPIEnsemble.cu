
#include <hip/hip_runtime.h>
#ifdef FLAMEGPU_ENABLE_MPI
#include "flamegpu/simulation/detail/MPIEnsemble.h"

#include "flamegpu/detail/compute_capability.cuh"

namespace flamegpu {
namespace detail {

MPIEnsemble::MPIEnsemble(const CUDAEnsemble::EnsembleConfig &_config, const unsigned int _total_runs)
    : config(_config)
    , world_rank(queryMPIWorldRank())
    , world_size(queryMPIWorldSize())
    , local_rank(queryMPISharedGroupRank())
    , local_size(queryMPISharedGroupSize())
    , total_runs(_total_runs)
    , MPI_ERROR_DETAIL(AbstractSimRunner::createErrorDetailMPIDatatype())
    , rank_is_participating(false)
    , comm_participating(MPI_COMM_NULL)
    , participating_size(0)
    , participating_rank(-1) { }

int MPIEnsemble::receiveErrors(std::multimap<int, AbstractSimRunner::ErrorDetail> &err_detail) {
    int errCount = 0;
    if (world_rank == 0) {
        MPI_Status status;
        int flag;
        // Check whether MPI runners have reported an error
        MPI_Iprobe(
            MPI_ANY_SOURCE,            // int source
            EnvelopeTag::ReportError,  // int tag
            MPI_COMM_WORLD,            // MPI_Comm communicator
            &flag,                     // int flag
            &status);
        while (flag) {
            // Receive the message
            memset(&status, 0, sizeof(MPI_Status));
            AbstractSimRunner::ErrorDetail e_detail;
            memset(&e_detail, 0, sizeof(AbstractSimRunner::ErrorDetail));
            MPI_Recv(
                &e_detail,                 // void* data
                1,                         // int count
                MPI_ERROR_DETAIL,          // MPI_Datatype datatype (can't use MPI_DATATYPE_NULL)
                MPI_ANY_SOURCE,            // int source
                EnvelopeTag::ReportError,  // int tag
                MPI_COMM_WORLD,            // MPI_Comm communicator
                &status);                  // MPI_Status*
            err_detail.emplace(status.MPI_SOURCE, e_detail);
            ++errCount;
            // Progress flush
            if (config.verbosity >= Verbosity::Default && config.error_level != CUDAEnsemble::EnsembleConfig::Fast) {
                fprintf(stderr, "Warning: Run %u/%u failed on rank %d, device %d, thread %u with exception: \n%s\n",
                    e_detail.run_id + 1, total_runs, status.MPI_SOURCE, e_detail.device_id, e_detail.runner_id, e_detail.exception_string);
                fflush(stderr);
            }
            // Check again
            MPI_Iprobe(MPI_ANY_SOURCE, EnvelopeTag::ReportError, MPI_COMM_WORLD, &flag, &status);
        }
    }
    return errCount;
}
int MPIEnsemble::receiveJobRequests(unsigned int &next_run) {
    int mpi_runners_fin = 0;
    if (world_rank == 0) {
        MPI_Status status;
        int flag;
        MPI_Iprobe(
            MPI_ANY_SOURCE,           // int source
            EnvelopeTag::RequestJob,  // int tag
            MPI_COMM_WORLD,           // MPI_Comm communicator
            &flag,                    // int flag
            &status);                 // MPI_Status*
        while (flag) {
            // Receive the message (kind of redundant as we already have the status and it carrys no data)
            memset(&status, 0, sizeof(MPI_Status));
            MPI_Recv(
                nullptr,                  // void* data
                0,                        // int count
                MPI_CHAR,                 // MPI_Datatype datatype (can't use MPI_DATATYPE_NULL)
                MPI_ANY_SOURCE,           // int source
                EnvelopeTag::RequestJob,  // int tag
                MPI_COMM_WORLD,           // MPI_Comm communicator
                &status);                 // MPI_Status*
            // Respond to the sender with a job assignment
            MPI_Send(
                &next_run,               // void* data
                1,                       // int count
                MPI_UNSIGNED,            // MPI_Datatype datatype
                status.MPI_SOURCE,       // int destination
                EnvelopeTag::AssignJob,  // int tag
                MPI_COMM_WORLD);         // MPI_Comm communicator
            if (next_run >= total_runs) ++mpi_runners_fin;
            ++next_run;
            // Print progress to console
            if (config.verbosity >= Verbosity::Default && next_run <= total_runs) {
                fprintf(stdout, "MPI ensemble assigned run %d/%u to rank %d\n", next_run, total_runs, status.MPI_SOURCE);
                fflush(stdout);
            }
            // Check again
            MPI_Iprobe(MPI_ANY_SOURCE, EnvelopeTag::RequestJob, MPI_COMM_WORLD, &flag, &status);
        }
    }
    return mpi_runners_fin;
}
void MPIEnsemble::sendErrorDetail(AbstractSimRunner::ErrorDetail &e_detail) {
    if (world_rank != 0) {
      MPI_Send(
          &e_detail,                  // void* data
          1,                         // int count
          MPI_ERROR_DETAIL,          // MPI_Datatype datatype (can't use MPI_DATATYPE_NULL)
          0,                         // int destination
          EnvelopeTag::ReportError,  // int tag
          MPI_COMM_WORLD);           // MPI_Comm communicator
    }
}
int MPIEnsemble::requestJob() {
    unsigned int next_run = UINT_MAX;
    if (world_rank != 0) {
        // Send a job request to 0, these have no data
        MPI_Send(
            nullptr,                  // void* data
            0,                        // int count
            MPI_CHAR,                 // MPI_Datatype datatype (can't use MPI_DATATYPE_NULL)
            0,                        // int destination
            EnvelopeTag::RequestJob,  // int tag
            MPI_COMM_WORLD);          // MPI_Comm communicator
        // Wait for a job assignment from 0
        MPI_Status status;
        memset(&status, 0, sizeof(MPI_Status));
        MPI_Recv(
            &next_run,               // void* data
            1,                       // int count
            MPI_UNSIGNED,            // MPI_Datatype datatype
            0,                       // int source
            EnvelopeTag::AssignJob,  // int tag
            MPI_COMM_WORLD,          // MPI_Comm communicator
            &status);                // MPI_Status* status
    }
    return next_run;
}
void MPIEnsemble::worldBarrier() {
    MPI_Barrier(MPI_COMM_WORLD);
}
std::string MPIEnsemble::assembleGPUsString() {
    std::string remote_device_names;
    // One rank per node should notify rank 0 of their GPU devices. other ranks will send an empty message.
    if (world_rank == 0) {
        int bufflen = 256;  // Length of name string in cudaDeviceProp
        char *buff = static_cast<char*>(malloc(bufflen));
        for (int i = 1; i < world_size; ++i) {
            // Receive a message from each rank
            MPI_Status status;
            memset(&status, 0, sizeof(MPI_Status));
            MPI_Probe(
                MPI_ANY_SOURCE,            // int source
                EnvelopeTag::TelemetryDevices,  // int tag
                MPI_COMM_WORLD,            // MPI_Comm communicator
                &status);
            int strlen = 0;
            // Ensure our receive buffer is long enough
            MPI_Get_count(&status, MPI_CHAR, &strlen);
            if (strlen > bufflen) {
                free(buff);
                buff = static_cast<char*>(malloc(strlen));
            }
            MPI_Recv(
                buff,                           // void* data
                strlen,                         // int count
                MPI_CHAR,                       // MPI_Datatype datatype (can't use MPI_DATATYPE_NULL)
                MPI_ANY_SOURCE,                 // int source
                EnvelopeTag::TelemetryDevices,  // int tag
                MPI_COMM_WORLD,                 // MPI_Comm communicator
                &status);                       // MPI_Status*
            if (strlen > 1) {
                remote_device_names.append(", ");
                remote_device_names.append(buff);
            }
        }
        free(buff);
    } else {
        const std::string d_string = local_rank == 0 ? compute_capability::getDeviceNames(config.devices) : "";
        // Send GPU count
        MPI_Send(
            d_string.c_str(),               // void* data
            d_string.length() + 1,          // int count
            MPI_CHAR,                       // MPI_Datatype datatype
            0,                              // int destination
            EnvelopeTag::TelemetryDevices,  // int tag
            MPI_COMM_WORLD);                // MPI_Comm communicator
    }
    worldBarrier();
    return remote_device_names;
}

int MPIEnsemble::queryMPIWorldRank() {
    initMPI();
    int world_rank = -1;
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);
    return world_rank;
}
int MPIEnsemble::queryMPIWorldSize() {
    initMPI();
    int world_size = -1;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    return world_size;
}

int MPIEnsemble::queryMPISharedGroupRank() {
    initMPI();
    int local_rank = -1;
    MPI_Comm group;
    MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, 0, MPI_INFO_NULL, &group);
    MPI_Comm_rank(group, &local_rank);
    return local_rank;
}

int MPIEnsemble::queryMPISharedGroupSize() {
    initMPI();
    int local_size = -1;
    MPI_Comm group;
    MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, 0, MPI_INFO_NULL, &group);
    MPI_Comm_size(group, &local_size);
    return local_size;
}

void MPIEnsemble::initMPI() {
    int flag = 0;
    // MPI can only be init once, for certain test cases we do some initial MPI comms for setup
    MPI_Initialized(&flag);
    if (!flag) {
        // Init MPI, fetch rank and size
        int thread_provided = 0;
        // MPI single means that only the main thread will perform MPI actions
        MPI_Init_thread(NULL, NULL, MPI_THREAD_SINGLE, &thread_provided);
        if (thread_provided != MPI_THREAD_SINGLE) {
            THROW exception::UnknownInternalError("MPI unable to provide MPI_THREAD_SINGLE support");
        }
    }
}
unsigned int MPIEnsemble::getDeviceIndex(const int j, const std::set<int> devices) {
    int i = 0;
    for (auto& d : devices) {
        if (i++ == j)
            return d;
    }
    return j;  // If set is empty, then direct index should be used
}
void MPIEnsemble::retrieveLocalErrorDetail(std::mutex &log_export_queue_mutex, std::multimap<int, AbstractSimRunner::ErrorDetail> &err_detail,
std::vector<detail::AbstractSimRunner::ErrorDetail> &err_detail_local, const int i, std::set<int> devices) {
    // Fetch error detail
    detail::AbstractSimRunner::ErrorDetail e_detail;
    {
        // log_export_mutex is treated as our protection for race conditions on err_detail
        std::lock_guard<std::mutex> lck(log_export_queue_mutex);
        // Fetch corresponding error detail
        bool success = false;
        const unsigned int t_device_id = getDeviceIndex(i / config.concurrent_runs, devices);
        const unsigned int t_runner_id = i % config.concurrent_runs;
        for (auto it = err_detail_local.begin(); it != err_detail_local.end(); ++it) {
            if (it->runner_id == t_runner_id && it->device_id == t_device_id) {
                e_detail = *it;
                if (world_rank == 0) {
                    // Only rank 0 collects error details
                    err_detail.emplace(world_rank, e_detail);
                } else {
                //   fprintf(stderr, "[%d] Purged error  from device %u runner %u\n", world_rank, t_device_id, t_runner_id);  // useful debug, breaks tests
                }
                err_detail_local.erase(it);
                success = true;
                break;
            }
        }
        if (!success) {
            THROW exception::UnknownInternalError("[%d] Management thread failed to locate reported error from device %u runner %u from %u errors, in CUDAEnsemble::simulate()", world_rank, t_device_id, t_runner_id, static_cast<unsigned int>(err_detail_local.size()));
        }
    }
    if (world_rank == 0) {
        // Progress flush
        if (config.verbosity >= Verbosity::Default && config.error_level != CUDAEnsemble::EnsembleConfig::Fast) {
            fprintf(stderr, "Warning: Run %u/%u failed on rank %d, device %d, thread %u with exception: \n%s\n",
                e_detail.run_id + 1, total_runs, world_rank, e_detail.device_id, e_detail.runner_id, e_detail.exception_string);
            fflush(stderr);
        }
    } else {
        // Notify 0 that an error occurred, with the error detail
        sendErrorDetail(e_detail);
    }
}

bool MPIEnsemble::createParticipatingCommunicator(const bool isParticipating) {
    // If the communicator has not yet been created, create it and get the rank and size.
    if (this->comm_participating == MPI_COMM_NULL) {
        // determine if this thread is participating or not, i..e. the colour of the rank
        this->rank_is_participating = isParticipating;
        // Split the world  communicator, if the split fails, abort (this makes the return type not useful tbh.)
        if (MPI_Comm_split(MPI_COMM_WORLD, this->rank_is_participating, this->world_rank, &this->comm_participating) != MPI_SUCCESS) {
            fprintf(stderr, "Error creating communicator\n");
            MPI_Abort(MPI_COMM_WORLD, 1);
            return false;
        }
        // Get the size of the split pariticpating communicator
        MPI_Comm_size(this->comm_participating, &this->participating_size);
        // Get the local rank within the split communicator
        MPI_Comm_rank(this->comm_participating, &this->participating_rank);
    }
    return true;
}

std::set<int> MPIEnsemble::devicesForThisRank(const std::set<int> devicesToSelectFrom, int local_size, int local_rank) {
    // create a vector from teh set to enable direct access.
    std::vector<int> devicesToSelectFromVector = std::vector<int>(devicesToSelectFrom.begin(), devicesToSelectFrom.end());
    int device_count = static_cast<int>(devicesToSelectFrom.size());
    // if there is only a single mpi rank on this shared memory system, assign all devices, or if there are no devices to select from
    if (local_size == 1 || device_count == 0) {
        return devicesToSelectFrom;
    } else if (local_size > 1 && local_size <= device_count) {
        // Otherwise, if there are more than one rank per node, but fewer ranks than gpus, attempt to load balance
        std::set<int> devices;
        // find the balanced number of gpus per rank, and how many will need +1
        int gpusPerRank = device_count / local_size;
        int unallocated = device_count - (gpusPerRank * local_size);
        // Compute the indices of the first and last gpu to be assigned to the current rank, based on how many lower ranks will have +1
        int lowerRanksWithPlus1 = local_rank < unallocated ? local_rank : unallocated;
        int lowerRanksWithPlus0 = std::max(0, local_rank - unallocated);
        int first = (lowerRanksWithPlus1 * (gpusPerRank + 1)) + (lowerRanksWithPlus0 * gpusPerRank);
        int last = local_rank < unallocated ? first + gpusPerRank + 1 : first + gpusPerRank;
        // Assign the devices for this rank
        for (int i = first; i < last; i++) {
            devices.emplace(devicesToSelectFromVector[i]);
        }
        return devices;
    } else {
        // Otherwise, there are more ranks than gpus, so use upto one gpu per rank.
        std::set<int> devices;
        for (const auto & d : devicesToSelectFromVector) {}
        if (local_rank < device_count) {
            devices.emplace(local_rank);
        }
        return devices;
    }
}

std::set<int> MPIEnsemble::devicesForThisRank(const std::set<int> devicesToSelectFrom) {
    return MPIEnsemble::devicesForThisRank(devicesToSelectFrom, this->local_size, this->local_rank);
}

}  // namespace detail
}  // namespace flamegpu
#endif
