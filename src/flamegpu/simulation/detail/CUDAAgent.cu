#include "hip/hip_runtime.h"
#include "flamegpu/simulation/detail/CUDAAgent.h"

#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <fstream>
#include <string>
#include <filesystem>

#ifdef _MSC_VER
#pragma warning(push, 1)
#pragma warning(disable : 4706 4834)
#endif  // _MSC_VER
#ifdef __NVCC_DIAG_PRAGMA_SUPPORT__
#pragma nv_diag_suppress 1719
#else
#pragma diag_suppress 1719
#endif  // __NVCC_DIAG_PRAGMA_SUPPORT__
#include <hipcub/hipcub.hpp>
#ifdef __NVCC_DIAG_PRAGMA_SUPPORT__
#pragma nv_diag_default 1719
#else
#pragma diag_default 1719
#endif  // __NVCC_DIAG_PRAGMA_SUPPORT__
#ifdef _MSC_VER
#pragma warning(pop)
#endif  // _MSC_VER

#include "flamegpu/version.h"
#include "flamegpu/simulation/detail/CUDAFatAgent.h"
#include "flamegpu/simulation/detail/CUDAAgentStateList.h"
#include "flamegpu/simulation/detail/CUDAErrorChecking.cuh"
#include "flamegpu/simulation/CUDASimulation.h"

#include "flamegpu/model/AgentDescription.h"
#include "flamegpu/model/AgentFunctionDescription.h"
#include "flamegpu/runtime/detail/curve/HostCurve.cuh"
#include "flamegpu/runtime/detail/curve/curve_rtc.cuh"
#include "flamegpu/simulation/detail/CUDAScatter.cuh"
#include "flamegpu/detail/compute_capability.cuh"
#include "flamegpu/util/nvtx.h"
#include "flamegpu/runtime/agent/DeviceAgentVector_impl.h"
#include "flamegpu/detail/cuda.cuh"

namespace flamegpu {
namespace detail {

CUDAAgent::CUDAAgent(const AgentData& description, const CUDASimulation &_cudaSimulation)
    : agent_description(description)  // This is a master agent, so it must create a new fat_agent
    , fat_agent(std::make_shared<CUDAFatAgent>(agent_description))  // if we create fat agent, we're index 0
    , fat_index(0)
    , cudaSimulation(_cudaSimulation)
    , TOTAL_AGENT_VARIABLE_SIZE(calcTotalVarSize(description)) {
    // Generate state map from fat_agent
    auto fatstate_map = fat_agent->getStateMap(fat_index);
    for (auto &state : description.states) {
        // Find correct fat state
        auto fatstate = fatstate_map.at(state);
        // Construct a regular state map from this
        auto slimstate = std::make_shared<CUDAAgentStateList>(fatstate, *this, fat_index, agent_description);
        // Store in our map
        state_map.emplace(state, slimstate);
    }
}
CUDAAgent::CUDAAgent(
    const AgentData &description,
    const CUDASimulation &_cudaSimulation,
    const std::unique_ptr<CUDAAgent> &master_agent,
    const std::shared_ptr<SubAgentData> &mapping)
    : agent_description(description)
    , fat_agent(master_agent->getFatAgent())
    , fat_index(fat_agent->getMappedAgentCount())
    , cudaSimulation(_cudaSimulation)
    , TOTAL_AGENT_VARIABLE_SIZE(calcTotalVarSize(description)) {
    // This is next agent to be added to fat_agent, so it takes existing count
    // Pass required info, so fat agent can generate new buffers and mappings
    fat_agent->addSubAgent(agent_description, master_agent->getFatIndex(), mapping);
    // Generate state map from fat_agent
    auto fatstate_map = fat_agent->getStateMap(fat_index);
    for (auto &state : agent_description.states) {
        // Find correct fat state
        auto fatstate = fatstate_map.at(state);
        // Construct a regular state map from this
        auto slimstate = std::make_shared<CUDAAgentStateList>(fatstate, *this, fat_index, agent_description, mapping->states.find(state) != mapping->states.end(), mapping->variables);
        // Store in our map
        state_map.emplace(state, slimstate);
    }
}

void CUDAAgent::mapRuntimeVariables(const AgentFunctionData& func, const unsigned int instance_id) const {
    // check the cuda agent state map to find the correct state list for functions starting state
    auto sm = state_map.find(func.initial_state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
            "in CUDAAgent::mapRuntimeVariables()",
            agent_description.name.c_str(), func.initial_state.c_str());
    }

    const unsigned int agent_count = this->getStateSize(func.initial_state);
    // loop through the agents variables to map each variable name using cuRVE
    for (const auto &mmp : agent_description.variables) {
        // get a device pointer for the agent variable name
        void* d_ptr = sm->second->getVariablePointer(mmp.first);

        // @todo These two blocks are grim, we keep using getRTCHeader() or getCurve(), which does a map lookup

        // Map variables to agent function (these must be mapped before each function execution as the runtime pointer may have changed to the swapping)
        if (!func.rtc_func_name.empty()) {
            auto& rtc_header = getRTCHeader(func.name);
            memcpy(rtc_header.getAgentVariableCachePtr(mmp.first.c_str()), &d_ptr, sizeof(void*));
        } else {
            auto& curve = getCurve(func.name);
            curve.setAgentVariable(mmp.first, d_ptr, agent_count);
        }

        // Map variables to agent function conditions (these must be mapped before each function execution as the runtime pointer may have changed to the swapping)
        if (!func.rtc_func_condition_name.empty()) {
            auto& rtc_header = getRTCHeader(func.name + "_condition");
            memcpy(rtc_header.getAgentVariableCachePtr(mmp.first.c_str()), &d_ptr, sizeof(void*));
        } else if (func.condition) {
            auto& curve = getCurve(func.name + "_condition");
            curve.setAgentVariable(mmp.first, d_ptr, agent_count);
        }
    }
}

void CUDAAgent::setPopulationData(const AgentVector& population, const std::string& state_name, CUDAScatter& scatter, const unsigned int streamId, const hipStream_t stream) {
    // Validate agent state
    auto our_state = state_map.find(state_name);
    if (our_state == state_map.end()) {
        if (state_name == ModelData::DEFAULT_STATE) {
            THROW exception::InvalidAgentState("Agent '%s' does not use the default state, so the state must be passed explicitly, "
                "in CUDAAgent::setPopulationData()",
                population.getAgentName().c_str());
        } else {
            THROW exception::InvalidAgentState("State '%s' was not found in agent '%s', "
                "in CUDAAgent::setPopulationData()",
                state_name.c_str(), population.getAgentName().c_str());
        }
    }
    // Copy population data
    // This call hierarchy validates agent desc matches
    our_state->second->setAgentData(population, scatter, streamId, stream);
    fat_agent->markIDsUnset();
    // Validate that there are no ID collisions
    validateIDCollisions(stream);
}
void CUDAAgent::getPopulationData(AgentVector& population, const std::string& state_name) const {
    // Validate agent state
    auto our_state = state_map.find(state_name);
    if (our_state == state_map.end()) {
        if (state_name == ModelData::DEFAULT_STATE) {
            THROW exception::InvalidAgentState("Agent '%s' does not use the default state, so the state must be passed explicitly, "
                "in CUDAAgent::getPopulationData()",
                state_name.c_str(), population.getAgentName().c_str());
        } else {
            THROW exception::InvalidAgentState("State '%s' was not found in agent '%s', "
                "in CUDAAgent::getPopulationData()",
                state_name.c_str(), population.getAgentName().c_str());
        }
    }
    // Copy population data
    // This call hierarchy validates agent desc matches
    our_state->second->getAgentData(population);
}
__global__ void generateCollisionFlags(const id_t* d_sortedKeys, id_t* d_flagsOut, unsigned int threads, id_t UNSET_FLAG) {
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < threads) {
        const id_t my_id = d_sortedKeys[id];
        if (my_id != UNSET_FLAG && my_id == d_sortedKeys[id+1]) {
            assert(UNSET_FLAG == 0);
            d_flagsOut[id] = 1;  // my_id; // any non-0 value basically
        }
    }
}
void CUDAAgent::validateIDCollisions(hipStream_t stream) const {
    flamegpu::util::nvtx::Range range{"CUDAAgent::validateIDCollisions"};
    // All data is on device, so use a device technique to check for collisions
    // Sort agent IDs, have a simple kernel check for neighbouring ID collisions to set a flag
    // Scan that flag
    // This could be improved by reusing buffers from elsewhere (e.g. StreamResources), rather than making temporary allocations for each method call
    // However, I'm also concerned that a model with agents added to multiple states and no agent birth would then pre-allocate larger buffers than required during execution

    // First count total agents across all states
    unsigned int agentCount = 0;
    for (const auto &s : state_map) {
        agentCount += s.second->getSize();
    }
    if (!agentCount) return;
    // Allocate buffers we will use
    id_t * d_keysIn = nullptr, *d_keysOut = nullptr;
    gpuErrchk(hipMalloc(&d_keysIn, sizeof(id_t) * agentCount));
    gpuErrchk(hipMalloc(&d_keysOut, sizeof(id_t) * agentCount));
    // Copy agent IDs to keysIn buff
    ptrdiff_t buffOffset = 0;
    for (const auto& s : state_map) {
        const unsigned int t_size = s.second->getSize();
        gpuErrchk(hipMemcpyAsync(d_keysIn + buffOffset, s.second->getVariablePointer(ID_VARIABLE_NAME), t_size * sizeof(id_t), hipMemcpyDeviceToDevice, stream));
        buffOffset += t_size;
    }
    // Sort agent ids into d_keysOut
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    gpuErrchk(hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keysIn, d_keysOut, agentCount, 0, sizeof(id_t) * 8, stream));
    gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
    gpuErrchk(hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keysIn, d_keysOut, agentCount, 0, sizeof(id_t) * 8, stream));
    // Reset d_keysIn
    gpuErrchk(hipMemsetAsync(d_keysIn, 0, sizeof(id_t) * agentCount, stream));
    // Launch a kernel to set flags if keys overlap their neighbour
    const unsigned int blockSize = 1024;
    const unsigned int blocks = ((agentCount-1) / blockSize) + 1;
    generateCollisionFlags<<<blocks, blockSize, 0, stream>>>(d_keysOut, d_keysIn, agentCount-1, ID_NOT_SET);
    gpuErrchkLaunch();
    // Check whether any flags were set
    size_t temp_storage_bytes2 = 0;
    gpuErrchk(hipcub::DeviceReduce::Sum(nullptr, temp_storage_bytes2, d_keysIn, d_keysOut, agentCount - 1, stream));
    if (temp_storage_bytes2 > temp_storage_bytes) {
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_temp_storage));
        temp_storage_bytes = temp_storage_bytes2;
        gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
    }
    gpuErrchk(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_keysIn, d_keysOut, agentCount - 1, stream));
    id_t flagsSet = 0;
    gpuErrchk(hipMemcpyAsync(&flagsSet, d_keysOut, sizeof(id_t), hipMemcpyDeviceToHost, stream));
    // Cleanup
    gpuErrchk(flamegpu::detail::cuda::hipFree(d_temp_storage));
    gpuErrchk(flamegpu::detail::cuda::hipFree(d_keysIn));
    gpuErrchk(flamegpu::detail::cuda::hipFree(d_keysOut));
    if (flagsSet) {
        THROW exception::AgentIDCollision("%u agents of type '%s' share an ID with another agent of the same type, "
            "you may need to explicitly reset agent IDs for 1 or more populations before adding them to the CUDASimulation, "
            "in CUDAAgent::validateIDCollisions()\n",
            static_cast<unsigned int>(flagsSet), agent_description.name.c_str());
    }
    gpuErrchk(hipStreamSynchronize(stream));
}
/**
 * Returns the number of alive and active agents in the named state
 */
unsigned int CUDAAgent::getStateSize(const std::string &state) const {
    // check the cuda agent state map to find the correct state list for functions starting state
    const auto &sm = state_map.find(state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found, "
            "in CUDAAgent::getStateSize()",
            agent_description.name.c_str(), state.c_str());
    }
    return sm->second->getSize();
}
/**
 * Returns the number of alive and active agents in the named state
 */
unsigned int CUDAAgent::getStateAllocatedSize(const std::string &state) const {
    // check the cuda agent state map to find the correct state list for functions starting state
    const auto &sm = state_map.find(state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found, "
            "in CUDAAgent::getStateAllocatedSize()",
            agent_description.name.c_str(), state.c_str());
    }
    return sm->second->getAllocatedSize();
}
void CUDAAgent::resizeState(const std::string& state, const unsigned int minimumSize, const bool retainData, const hipStream_t stream) {
    // check the cuda agent state map to find the correct state list
    const auto& sm = state_map.find(state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found, "
            "in CUDAAgent::getStateAllocatedSize()",
            agent_description.name.c_str(), state.c_str());
    }
    sm->second->resize(minimumSize, retainData, stream);
}

void CUDAAgent::setStateAgentCount(const std::string& state, const unsigned int newSize) {
    // check the cuda agent state map to find the correct state list
    const auto& sm = state_map.find(state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found, "
            "in CUDAAgent::getStateAllocatedSize()",
            agent_description.name.c_str(), state.c_str());
    }
    sm->second->setAgentCount(newSize);
}
CAgentDescription CUDAAgent::getAgentDescription() const {
    return CAgentDescription(agent_description.shared_from_this());
}
void *CUDAAgent::getStateVariablePtr(const std::string &state_name, const std::string &variable_name) {
    // check the cuda agent state map to find the correct state list for functions starting state
    const auto &sm = state_map.find(state_name);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found, "
            "in CUDAAgent::getStateVariablePtr()",
            agent_description.name.c_str(), state_name.c_str());
    }
    return sm->second->getVariablePointer(variable_name);
}
void CUDAAgent::processDeath(const AgentFunctionData& func, detail::CUDAScatter &scatter, const unsigned int streamId, const hipStream_t stream) {
    // Optionally process agent death
    if (func.has_agent_death) {
        // Agent death operates on all mapped vars, so handled by fat agent
        fat_agent->processDeath(fat_index, func.initial_state, scatter, streamId, stream);
    }
}
void CUDAAgent::transitionState(const std::string &_src, const std::string &_dest, detail::CUDAScatter &scatter, const unsigned int streamId, const hipStream_t stream) {
    // All mapped vars need to transition too, so handled by fat agent
    fat_agent->transitionState(fat_index, _src, _dest, scatter, streamId, stream);
}
void CUDAAgent::processFunctionCondition(const AgentFunctionData& func, detail::CUDAScatter &scatter, const unsigned int streamId, const hipStream_t stream) {
    // Optionally process function condition
    if ((func.condition) || (!func.rtc_func_condition_name.empty())) {
        // Agent function condition operates on all mapped vars, so handled by fat agent
        fat_agent->processFunctionCondition(fat_index, func.initial_state, scatter, streamId, stream);
    }
}
void CUDAAgent::scatterHostCreation(const std::string &state_name, const unsigned int newSize, char *const d_inBuff, const VarOffsetStruct &offsets, detail::CUDAScatter &scatter, const unsigned int streamId, const hipStream_t stream) {
    auto sm = state_map.find(state_name);
    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
            "in CUDAAgent::scatterHostCreation()",
            agent_description.name.c_str(), state_name.c_str());
    }
    sm->second->scatterHostCreation(newSize, d_inBuff, offsets, scatter, streamId, stream);
}
void CUDAAgent::scatterSort_async(const std::string &state_name, detail::CUDAScatter &scatter, unsigned int streamId, hipStream_t stream) {
    auto sm = state_map.find(state_name);
    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
            "in CUDAAgent::scatterHostCreation()",
            agent_description.name.c_str(), state_name.c_str());
    }
    sm->second->scatterSort_async(scatter, streamId, stream);
}
void CUDAAgent::mapNewRuntimeVariables_async(const CUDAAgent& func_agent, const AgentFunctionData& func, unsigned int maxLen, detail::CUDAScatter &scatter, unsigned int instance_id, hipStream_t stream, unsigned int streamId) {
    // Confirm agent output is set
    if (auto oa = func.agent_output.lock()) {
        // check the cuda agent state map to find the correct state list for functions starting state
        auto sm = state_map.find(func.agent_output_state);

        if (sm == state_map.end()) {
            THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
                "in CUDAAgent::mapNewRuntimeVariables()",
                agent_description.name.c_str(), func.agent_output_state.c_str());
        }
        // Notify scan flag that it might need resizing
        // We need a 3rd array, because a function might combine agent birth, agent death and message output
        scatter.Scan().resize(maxLen, CUDAScanCompaction::AGENT_OUTPUT, streamId);
        // Ensure the scan flag is zeroed
        scatter.Scan().zero_async(CUDAScanCompaction::AGENT_OUTPUT, stream, streamId);

        // Request a buffer for new
        char *d_new_buffer = static_cast<char*>(fat_agent->allocNewBuffer(TOTAL_AGENT_VARIABLE_SIZE, maxLen, agent_description.variables.size()));

        // Store buffer so we can release it later
        {
            std::lock_guard<std::mutex> guard(newBuffsMutex);
            const auto rtn = newBuffs.emplace(func.initial_state, d_new_buffer);
            if (!rtn.second) {
                assert(false);  // Insertion happened (false if element already exists)
            }
        }

        // Init the buffer to default values for variables
        scatter.broadcastInit_async(
            streamId,
            stream,
            agent_description.variables,
            d_new_buffer,
            maxLen, 0);
        // No sync, use of the buffer should be in the same stream

        // loop through the agents variables to map each variable name using cuRVE
        // these must be mapped before each function execution as the runtime pointer may have changed to the swapping
        for (const auto &mmp : agent_description.variables) {
            // get the agent variable size
            const size_t type_size = mmp.second.type_size * mmp.second.elements;

            // get a device pointer for the agent variable name
            void* d_ptr = d_new_buffer;

            // Move the pointer along for next variable
            d_new_buffer += type_size * maxLen;

            // 64 bit align the new buffer start
            if (reinterpret_cast<size_t>(d_new_buffer)%8) {
                d_new_buffer += 8 - (reinterpret_cast<size_t>(d_new_buffer)%8);
            }

            // maximum population num
            if (func.func) {
                auto& curve = func_agent.getCurve(func.name);  // @todo stop map hammering
                curve.setAgentOutputVariable(mmp.first, d_ptr, maxLen);
            } else  {
                auto& rtc_header = func_agent.getRTCHeader(func.name);
                memcpy(rtc_header.getNewAgentVariableCachePtr(mmp.first.c_str()), &d_ptr, sizeof(void*));
            }
        }
    }
}
void CUDAAgent::releaseNewBuffer(const AgentFunctionData& func) {
    // Confirm agent output is set
    if (auto oa = func.agent_output.lock()) {
        // Release new buffer
        {
            std::lock_guard<std::mutex> guard(newBuffsMutex);
            const auto d_buff = newBuffs.find(func.initial_state);
            if (d_buff != newBuffs.end()) {
                fat_agent->freeNewBuffer(d_buff->second);
                newBuffs.erase(d_buff);
            } else {
                assert(false);  // We don't have a new buffer reserved???
            }
        }
    }
}

void CUDAAgent::scatterNew(const AgentFunctionData& func, const unsigned int newSize, detail::CUDAScatter &scatter, const unsigned int streamId, const hipStream_t stream) {
    // Confirm agent output is set
    if (auto oa = func.agent_output.lock()) {
        auto sm = state_map.find(func.agent_output_state);
        if (sm == state_map.end()) {
            THROW exception::InvalidStateName("Agent '%s' does not contain state '%s', "
                "in CUDAAgent::scatterNew()\n",
                agent_description.name.c_str(), func.agent_output_state.c_str());
        }
        // Find new buffer
        void *newBuff = nullptr;
        {
            std::lock_guard<std::mutex> guard(newBuffsMutex);
            const auto d_buff = newBuffs.find(func.initial_state);
            if (d_buff != newBuffs.end()) {
                newBuff = d_buff->second;
            }
        }
        if (!newBuff) {
            THROW exception::InvalidAgentFunc("New buffer not present for function within init state: %s,"
                " in CUDAAgent::scatterNew()\n",
                func.initial_state.c_str());
        }
        unsigned int new_births = sm->second->scatterNew(newBuff, newSize, scatter, streamId, stream);
        fat_agent->notifyDeviceBirths(new_births);
    }
}
void CUDAAgent::clearFunctionCondition(const std::string &state) {
    fat_agent->setConditionState(fat_index, state, 0);
}

void CUDAAgent::addInstantitateRTCFunction(const AgentFunctionData& func, const std::shared_ptr<EnvironmentManager> &env, std::shared_ptr<const detail::CUDAMacroEnvironment> macro_env, bool function_condition) {
    // Generate the dynamic curve header
    detail::curve::CurveRTCHost &curve_header = *rtc_header_map.emplace(function_condition ? func.name + "_condition" : func.name, std::make_unique<detail::curve::CurveRTCHost>()).first->second;

    // set agent function variables in rtc curve
    for (const auto& mmp : func.parent.lock()->variables) {
        curve_header.registerAgentVariable(mmp.first.c_str(), mmp.second.type.name(), mmp.second.type_size, mmp.second.elements);
    }

    // for normal agent function (e.g. not an agent function condition) append messages and agent outputs
    if (!function_condition) {
        // Set input message variables in curve
        if (auto im = func.message_input.lock()) {
            for (auto message_in_var : im->variables) {
                // register message variables using combined hash
                curve_header.registerMessageInVariable(message_in_var.first.c_str(),
                message_in_var.second.type.name(), message_in_var.second.type_size, message_in_var.second.elements, true, false);
            }
        }
        // Set output message variables in curve
        if (auto om = func.message_output.lock()) {
            for (auto message_out_var : om->variables) {
                // register message variables using combined hash
                curve_header.registerMessageOutVariable(message_out_var.first.c_str(),
                message_out_var.second.type.name(), message_out_var.second.type_size, message_out_var.second.elements, false, true);
            }
        }
        // Set agent output variables in curve
        if (auto ao = func.agent_output.lock()) {
            for (auto agent_out_var : ao->variables) {
                // register message variables using combined hash
                curve_header.registerNewAgentVariable(agent_out_var.first.c_str(),
                agent_out_var.second.type.name(), agent_out_var.second.type_size, agent_out_var.second.elements, false, true);
            }
        }
    }

    // Set environment properties in curve (this includes mapped properties)
    {
        const auto &prop_map = env->getPropertiesMap();
        for (const auto &p : prop_map) {
            const char* variableName = p.first.c_str();
            const char* type = p.second.type.name();
            const unsigned int elements = p.second.elements;
            const ptrdiff_t offset = p.second.offset;
            curve_header.registerEnvVariable(variableName, offset, type, p.second.length/elements, elements);
         }
    }

    // Set Environment macro properties in curve
    macro_env->mapRTCVariables(curve_header);

    // Set the agent name/state
    curve_header.registerAgent(this->agent_description.name, func.initial_state);

    std::string header_filename = std::string(func.rtc_func_name).append("_impl");
    if (function_condition)
        header_filename.append("_condition");
    header_filename.append("_curve_rtc_dynamic.h");
    curve_header.setFileName(header_filename);

    // get the dynamically generated header from curve rtc
    const std::string curve_dynamic_header = curve_header.getDynamicHeader(env->getBufferLen());

    // output to disk if FLAMEGPU_OUTPUT_RTC_DYNAMIC_FILES macro is set
#ifdef FLAMEGPU_OUTPUT_RTC_DYNAMIC_FILES
        // create string for agent function implementation
        std::string func_impl = std::string(func.rtc_func_name).append("_impl");
        // curve
        std::ofstream file_curve_rtc_header;
        std::string file_curve_rtc_header_filename = func_impl.c_str();
        if (function_condition)
            file_curve_rtc_header_filename.append("_condition");
        file_curve_rtc_header_filename.append("_curve_rtc_dynamic.h");
        file_curve_rtc_header.open(file_curve_rtc_header_filename);
        // Remove first line as it is the filename, which misaligns profiler
        std::string out_s = curve_dynamic_header;
        out_s.erase(0, out_s.find("\n") + 1);
        file_curve_rtc_header << out_s;
        file_curve_rtc_header.close();
        // agent function
        std::ofstream agent_function_file;
        std::string agent_function_filename = func_impl.c_str();
        if (function_condition)
            agent_function_filename.append("_condition");
        agent_function_filename.append(".cu");
        agent_function_file.open(agent_function_filename);
        // Remove first line as it is the filename, which misaligns profiler
        out_s = func.rtc_source;
        out_s.erase(0, out_s.find("\n") + 1);
        agent_function_file << out_s;
        agent_function_file.close();
#endif

    detail::JitifyCache &jitify = detail::JitifyCache::getInstance();
    // switch between normal agent function and agent function condition
    if (!function_condition) {
        const std::string t_func_impl = std::string(func.rtc_func_name).append("_impl");
        const std::vector<std::string> template_args = { t_func_impl.c_str(), func.message_in_type.c_str(), func.message_out_type.c_str() };
        auto kernel_inst = jitify.loadKernel(func.rtc_func_name, template_args, func.rtc_source, curve_dynamic_header);
        // add kernel instance to map
        rtc_func_map.insert(CUDARTCFuncMap::value_type(func.name, std::move(kernel_inst)));
    } else {
        const std::string t_func_impl = std::string(func.rtc_func_condition_name).append("_cdn_impl");
        const std::vector<std::string> template_args = { t_func_impl.c_str() };
        auto kernel_inst = jitify.loadKernel(func.rtc_func_name + "_condition", template_args, func.rtc_condition_source, curve_dynamic_header);
        // add kernel instance to map
        rtc_func_map.insert(CUDARTCFuncMap::value_type(func.name + "_condition", std::move(kernel_inst)));
    }
}

void CUDAAgent::addInstantitateFunction(const AgentFunctionData& func, const std::shared_ptr<EnvironmentManager>& env, std::shared_ptr<const detail::CUDAMacroEnvironment> macro_env, bool function_condition) {
    // Generate the host curve instance
    std::unique_ptr<detail::curve::HostCurve> curve = std::make_unique<detail::curve::HostCurve>();

    // Initialising values here, removes the need to "unregister" curve values
    // set agent variables in curve
    for (const auto& mmp : func.parent.lock()->variables) {
        curve->registerAgentVariable(mmp.first, mmp.second.type, mmp.second.type_size, mmp.second.elements);
    }

    // for normal agent function (e.g. not an agent function condition) append messages and agent outputs
    if (!function_condition) {
        // Set input message variables in curve
        if (auto im = func.message_input.lock()) {
            for (auto message_in_var : im->variables) {
                curve->registerMessageInputVariable(message_in_var.first, message_in_var.second.type, message_in_var.second.type_size, message_in_var.second.elements);
            }
        }
        // Set output message variables in curve
        if (auto om = func.message_output.lock()) {
            for (auto message_out_var : om->variables) {
                curve->registerMessageOutputVariable(message_out_var.first, message_out_var.second.type, message_out_var.second.type_size, message_out_var.second.elements);
            }
        }
        // Set agent output variables in curve
        if (auto ao = func.agent_output.lock()) {
            for (auto agent_out_var : ao->variables) {
                curve->registerAgentOutputVariable(agent_out_var.first, agent_out_var.second.type, agent_out_var.second.type_size, agent_out_var.second.elements);
            }
        }
    }

    // Set environment properties in curve (this includes mapped properties)
    {
        const auto& prop_map = env->getPropertiesMap();
        for (const auto& p : prop_map) {
            const unsigned int elements = p.second.elements;
            curve->registerSetEnvironmentProperty(p.first, p.second.type, p.second.length / elements, elements, p.second.offset);
        }
    }

    // Set Environment macro properties in curve
    macro_env->registerCurveVariables(*curve);

    // switch between normal agent function and agent function condition, and add to map
    const std::string key_name = function_condition ? func.name + "_condition" : func.name;
    curve_map.insert(std::unordered_map<std::string, std::unique_ptr<detail::curve::HostCurve>>::value_type(key_name, std::move(curve)));
}

const jitify::experimental::KernelInstantiation& CUDAAgent::getRTCInstantiation(const std::string &function_name) const {
    CUDARTCFuncMap::const_iterator mm = rtc_func_map.find(function_name);
    if (mm == rtc_func_map.end()) {
        THROW exception::InvalidAgentFunc("Function name '%s' is not a runtime compiled agent function in agent '%s', "
            "in CUDAAgent::getRTCInstantiation()\n",
            function_name.c_str(), agent_description.name.c_str());
    }

    return *mm->second;
}
detail::curve::CurveRTCHost& CUDAAgent::getRTCHeader(const std::string &function_name) const {
    CUDARTCHeaderMap::const_iterator mm = rtc_header_map.find(function_name);
    if (mm == rtc_header_map.end()) {
        THROW exception::InvalidAgentFunc("Function name '%s' is not a runtime compiled agent function in agent '%s', "
            "in CUDAAgent::getRTCHeader()\n",
            function_name.c_str(), agent_description.name.c_str());
    }

    return *mm->second;
}
detail::curve::HostCurve& CUDAAgent::getCurve(const std::string &function_name) const {
    auto mm = curve_map.find(function_name);
    if (mm == curve_map.end()) {
        THROW exception::InvalidAgentFunc("Function name '%s' is not a (non-rtc) agent function in agent '%s', "
            "in CUDAAgent::getCurve()\n",
            function_name.c_str(), agent_description.name.c_str());
    }

    return *mm->second;
}

const CUDAAgent::CUDARTCFuncMap& CUDAAgent::getRTCFunctions() const {
    return rtc_func_map;
}

void CUDAAgent::initUnmappedVars(detail::CUDAScatter &scatter, const unsigned int streamId, const hipStream_t stream) {
    for (auto &s : state_map) {
        s.second->initUnmappedVars(scatter, streamId, stream);
    }
}
void CUDAAgent::initExcludedVars(const std::string &state, const unsigned int count, const unsigned int offset, CUDAScatter& scatter, const unsigned int streamId, const hipStream_t stream) {
    // check the cuda agent state map to find the correct state list
    const auto& sm = state_map.find(state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found, "
            "in CUDAAgent::initUnmappedVars()",
            agent_description.name.c_str(), state.c_str());
    }
    sm->second->initExcludedVars(count, offset, scatter, streamId, stream);
}
void CUDAAgent::cullUnmappedStates() {
    unsigned int i = 0;
    for (auto &s : state_map) {
        if (!s.second->getIsSubStatelist()) {
            s.second->clear();
            ++i;
        }
    }
    if (i == state_map.size())
        fat_agent->resetIDCounter();
}
void CUDAAgent::cullAllStates() {
    for (auto &s : state_map) {
        s.second->clear();
    }
    fat_agent->resetIDCounter();
}
std::list<std::shared_ptr<VariableBuffer>> CUDAAgent::getUnboundVariableBuffers(const std::string& state) {
    const auto& sm = state_map.find(state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found, "
            "in CUDAAgent::getUnboundVariableBuffers()",
            agent_description.name.c_str(), state.c_str());
    }
    return sm->second->getUnboundVariableBuffers();
}
id_t CUDAAgent::nextID(unsigned int count) {
    return fat_agent->nextID(count);
}
id_t* CUDAAgent::getDeviceNextID() {
    return fat_agent->getDeviceNextID();
}
void CUDAAgent::assignIDs(HostAPI& hostapi, detail::CUDAScatter &scatter, hipStream_t stream, const unsigned int streamId) {
    fat_agent->assignIDs(hostapi, scatter, stream, streamId);
}

void CUDAAgent::setPopulationVec(const std::string& state_name, const std::shared_ptr<DeviceAgentVector_impl>& d_vec) {
    population_dvec[state_name] = d_vec;
}
std::shared_ptr<DeviceAgentVector_impl> CUDAAgent::getPopulationVec(const std::string& state_name) {
    auto find = population_dvec.find(state_name);
    if (find != population_dvec.end())
        return find->second;
    return nullptr;
}
void CUDAAgent::resetPopulationVecs() {
    for (auto &vec : population_dvec) {
        if (vec.second) {
            vec.second->syncChanges();
            vec.second.reset();
        }
    }
    population_dvec.clear();
}

}  // namespace detail
}  // namespace flamegpu
