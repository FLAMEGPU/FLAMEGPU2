#include "hip/hip_runtime.h"
#include "flamegpu/simulation/CUDASimulation.h"


#include <algorithm>
#include <string>
#include <map>

#include "flamegpu/detail/hiprand.cuh"
#include "flamegpu/model/AgentFunctionData.cuh"
#include "flamegpu/model/LayerData.h"
#include "flamegpu/model/AgentDescription.h"
#include "flamegpu/model/SubModelData.h"
#include "flamegpu/model/SubAgentData.h"
#include "flamegpu/runtime/HostAPI.h"
#include "flamegpu/simulation/detail/CUDAScanCompaction.h"
#include "flamegpu/util/nvtx.h"
#include "flamegpu/detail/compute_capability.cuh"
#include "flamegpu/detail/SignalHandlers.h"
#include "flamegpu/detail/wddm.cuh"
#include "flamegpu/detail/SteadyClockTimer.h"
#include "flamegpu/detail/CUDAEventTimer.cuh"
#include "flamegpu/runtime/detail/curve/curve_rtc.cuh"
#include "flamegpu/runtime/HostFunctionCallback.h"
#include "flamegpu/runtime/messaging.h"
#include "flamegpu/simulation/detail/CUDAAgent.h"
#include "flamegpu/simulation/detail/CUDAMessage.h"
#include "flamegpu/simulation/LoggingConfig.h"
#include "flamegpu/simulation/LogFrame.h"
#include "flamegpu/simulation/RunPlan.h"
#include "flamegpu/version.h"
#include "flamegpu/model/AgentFunctionDescription.h"
#include "flamegpu/io/Telemetry.h"
#ifdef FLAMEGPU_VISUALISATION
#include "flamegpu/visualiser/FLAMEGPU_Visualisation.h"
#endif

namespace flamegpu {

namespace {
    // file-scope only variable used to cache the driver mode
    bool deviceUsingWDDM = false;
    // Inlined method in the anonymous namespace to create a new timer, subject to the driver model.
    std::unique_ptr<detail::Timer> getDriverAppropriateTimer(bool is_ensemble) {
        if (!deviceUsingWDDM && !is_ensemble) {
            return std::unique_ptr<detail::Timer>(new detail::CUDAEventTimer());
        } else {
            return std::unique_ptr<detail::Timer>(new detail::SteadyClockTimer());
        }
    }
}  // anonymous namespace

CUDASimulation::CUDASimulation(const ModelDescription& _model, int argc, const char** argv)
    : CUDASimulation(_model.model) {
    if (argc && argv) {
        initialise(argc, argv);
    }
}
CUDASimulation::CUDASimulation(const std::shared_ptr<const ModelData> &_model)
    : Simulation(_model)
    , step_count(0)
    , elapsedSecondsSimulation(0.)
    , elapsedSecondsInitFunctions(0.)
    , elapsedSecondsExitFunctions(0.)
    , elapsedSecondsRTCInitialisation(0.)
    , macro_env(*_model->environment, *this)
    , config({})
    , run_log(std::make_unique<RunLog>())
    , streams(std::vector<hipStream_t>())
    , singletons(nullptr)
    , singletonsInitialised(false)
    , rtcInitialised(false)
    , isPureRTC(detectPureRTC(model)) {
    initOffsetsAndMap();
    // Register the signal handler.
    detail::SignalHandlers::registerSignalHandlers();

    // populate the CUDA agent map
    const auto &am = model->agents;
    // create new cuda agent and add to the map
    for (auto it = am.cbegin(); it != am.cend(); ++it) {
        // insert into map using value_type and store a reference to the map pair
        agent_map.emplace(it->first, std::make_unique<detail::CUDAAgent>(*it->second, *this)).first;
    }

    // populate the CUDA message map
    const auto &mm = model->messages;
    // create new cuda message and add to the map
    for (auto it_m = mm.cbegin(); it_m != mm.cend(); ++it_m) {
        message_map.emplace(it_m->first, std::make_unique<detail::CUDAMessage>(*it_m->second, *this));
    }

    // populate the CUDA submodel map
    const auto &smm = model->submodels;
    // create new cuda message and add to the map
    for (auto it_sm = smm.cbegin(); it_sm != smm.cend(); ++it_sm) {
        submodel_map.emplace(it_sm->first, std::unique_ptr<CUDASimulation>(new CUDASimulation(it_sm->second, this)));
    }

    // Determine which agents will be spatially sorted
    this->determineAgentsToSort();
}
bool CUDASimulation::detectPureRTC(const std::shared_ptr<const ModelData>& _model) {
    const auto& am = _model->agents;
    for (auto it = am.cbegin(); it != am.cend(); ++it) {
        for (const auto& af : it->second->functions) {
            if (af.second->func || af.second->condition)
                return false;
        }
    }
    // TODO: In future this will need to be extended for new forms of device function, e.g. device init
    const auto& as = _model->submodels;
    for (auto it = as.cbegin(); it != as.cend(); ++it) {
        if (!detectPureRTC(it->second->submodel))
          return false;
    }
    return true;
}
CUDASimulation::CUDASimulation(const std::shared_ptr<SubModelData> &submodel_desc, CUDASimulation *master_model)
    : Simulation(submodel_desc, master_model)
    , step_count(0)
    , macro_env(*submodel_desc->submodel->environment, *this)
    , run_log(std::make_unique<RunLog>())
    , streams(std::vector<hipStream_t>())
    , singletons(nullptr)
    , singletonsInitialised(false)
    , rtcInitialised(false)
    , isPureRTC(master_model->isPureRTC) {
    initOffsetsAndMap();
    // Ensure submodel is valid
    if (submodel_desc->submodel->exitConditions.empty() && submodel_desc->submodel->exitConditionCallbacks.empty() && submodel_desc->max_steps == 0) {
        THROW exception::InvalidSubModel("Model '%s' does not contain any exit conditions or exit condition callbacks and submodel '%s' max steps is set to 0, SubModels must exit of their own accord, "
            "in CUDASimulation::CUDASimulation().",
            submodel_desc->submodel->name.c_str(), submodel_desc->name.c_str());
    }

    // populate the CUDA agent map (With SubAgents!)
    const auto &am = model->agents;
    // create new cuda agent and add to the map
    for (auto it = am.cbegin(); it != am.cend(); ++it) {
        // Locate the mapping
        auto _mapping = submodel_desc->subagents.find(it->second->name);
        if (_mapping != submodel_desc->subagents.end()) {
            // Agent is mapped, create subagent
            std::shared_ptr<SubAgentData> &mapping = _mapping->second;
            // Locate the master agent
            std::shared_ptr<AgentData> masterAgentDesc = mapping->masterAgent.lock();
            if (!masterAgentDesc) {
                THROW exception::InvalidParent("Master agent description has expired, in CUDASimulation SubModel constructor.\n");
            }
            std::unique_ptr<detail::CUDAAgent> &masterAgent = master_model->agent_map.at(masterAgentDesc->name);
            agent_map.emplace(it->first, std::make_unique<detail::CUDAAgent>(*it->second, *this, masterAgent, mapping));
        } else {
            // Agent is not mapped, create regular agent
            agent_map.emplace(it->first, std::make_unique<detail::CUDAAgent>(*it->second, *this)).first;
        }
    }  // insert into map using value_type

    // populate the CUDA message map (Sub Messages not currently supported)
    const auto &mm = model->messages;
    // create new cuda message and add to the map
    for (auto it_m = mm.cbegin(); it_m != mm.cend(); ++it_m) {
        message_map.emplace(it_m->first, std::make_unique<detail::CUDAMessage>(*it_m->second, *this));
    }

    // populate the CUDA submodel map
    const auto &smm = model->submodels;
    // create new cuda model and add to the map
    for (auto it_sm = smm.cbegin(); it_sm != smm.cend(); ++it_sm) {
        submodel_map.emplace(it_sm->first, std::unique_ptr<CUDASimulation>(new CUDASimulation(it_sm->second, this)));
    }
    // Submodels all run quiet/not verbose by default
    SimulationConfig().verbosity = Verbosity::Default;
    SimulationConfig().steps = submodel_desc->max_steps;
    CUDAConfig().is_ensemble = true;

    // Determine which agents will be spatially sorted
    this->determineAgentsToSort();
}

CUDASimulation::~CUDASimulation() {
    // Ensure we destruct with the right device, otherwise we could dealloc pointers on the wrong device
    int t_device_id = -1;
    gpuErrchk(hipGetDevice(&t_device_id));
    if (t_device_id != deviceInitialised && deviceInitialised != -1) {
        gpuErrchk(hipSetDevice(deviceInitialised));
    }

    submodel_map.clear();  // Test
    // De-initialise, freeing singletons?
    // @todo - this is unsafe in a destructor as it may invoke cuda commands.
    if (singletonsInitialised) {
        delete singletons;
        singletons = nullptr;
    }

    // We must explicitly delete all cuda members before we cuda device reset
    agent_map.clear();
    message_map.clear();
    submodel_map.clear();
    host_api.reset();
    macro_env.free();
#ifdef FLAMEGPU_VISUALISATION
    visualisation.reset();  // Might want to force destruct this, as user could hold a ModelVis that has shared ptr
#endif

    // Destroy streams, potentially unsafe in a destructor as it will invoke cuda commands.
    // Do this once to re-use existing streams rather than per-step.
    this->destroyStreams();

    // Reset the active device if not the device used for this simulation
    if (t_device_id != deviceInitialised) {
        gpuErrchk(hipSetDevice(t_device_id));
    }
}

void CUDASimulation::initFunctions() {
    flamegpu::util::nvtx::Range range{"CUDASimulation::initFunctions"};
    std::unique_ptr<detail::Timer> initFunctionsTimer(new detail::SteadyClockTimer());
    initFunctionsTimer->start();

    // Execute normal init functions
    for (auto &initFn : model->initFunctions) {
        initFn(this->host_api.get());
    }
    // Execute init function callbacks (python)
    for (auto &initFn : model->initFunctionCallbacks) {
        initFn->run(this->host_api.get());
    }
    // Check if host agent creation was used in init functions
    if (model->initFunctions.size() || model->initFunctionCallbacks.size()) {
        // Sync any device vectors, before performing host agent creation
        for (auto& ca : agent_map) {
            ca.second->resetPopulationVecs();
        }
        processHostAgentCreation(0);
    }

    // Record, store and output the elapsed time of the step.
    initFunctionsTimer->stop();
    this->elapsedSecondsInitFunctions = initFunctionsTimer->getElapsedSeconds();
    if (getSimulationConfig().timing || getSimulationConfig().verbosity >= Verbosity::Verbose) {
        fprintf(stdout, "Init Function Processing time: %.6f s\n", this->elapsedSecondsInitFunctions);
    }
}

void CUDASimulation::exitFunctions() {
    flamegpu::util::nvtx::Range range{"CUDASimulation::exitFunctions"};
    std::unique_ptr<detail::Timer> exitFunctionsTimer(new detail::SteadyClockTimer());
    exitFunctionsTimer->start();

    // Execute exit functions
    for (auto &exitFn : model->exitFunctions) {
        exitFn(this->host_api.get());
    }
    // Execute any exit functions from swig/python
    for (auto &exitFn : model->exitFunctionCallbacks) {
        exitFn->run(this->host_api.get());
    }

    // Record, store and output the elapsed time of the step.
    exitFunctionsTimer->stop();
    this->elapsedSecondsExitFunctions = exitFunctionsTimer->getElapsedSeconds();
    if (getSimulationConfig().timing || getSimulationConfig().verbosity >= Verbosity::Verbose) {
        fprintf(stdout, "Exit Function Processing time: %.6f s\n", this->elapsedSecondsExitFunctions);
    }
}

namespace detail {
template <typename T> struct Dims {
    T x;
    T y;
    T z;
};
}

__global__ void calculateSpatialHashFloat3(float* xyz, unsigned int* binIndex, detail::Dims<float> envMin, detail::Dims<float> envWidth, detail::Dims<unsigned int> gridDim, unsigned int threadCount) {
    const unsigned int TID = blockIdx.x * blockDim.x + threadIdx.x;
    if (TID < threadCount) {
        // Compute hash (effectivley an index for to a bin within the partitioning grid in this case)
        int gridPos[3] = {
            static_cast<int>(floorf(((xyz[TID * 3 + 0] - envMin.x) / envWidth.x) * gridDim.x)),
            static_cast<int>(floorf(((xyz[TID * 3 + 1] - envMin.y) / envWidth.y) * gridDim.y)),
            static_cast<int>(floorf(((xyz[TID * 3 + 2] - envMin.z) / envWidth.z) * gridDim.z))
        };

        // Compute and set the bin index
        unsigned int bindex;

        bindex = (unsigned int)(
            (gridPos[2] * gridDim.x * gridDim.y +   // z
                (gridPos[1] * gridDim.x) +              // y
                gridPos[0]));                           // x

        binIndex[TID] = bindex;
    }
}
__global__ void calculateSpatialHashFloat2(float* xy, unsigned int* binIndex, detail::Dims<float> envMin, detail::Dims<float> envWidth, detail::Dims<unsigned int> gridDim, unsigned int threadCount) {
    const unsigned int TID = blockIdx.x * blockDim.x + threadIdx.x;
    if (TID < threadCount) {
        // Compute hash (effectivley an index for to a bin within the partitioning grid in this case)
        int gridPos[3] = {
            static_cast<int>(floorf(((xy[TID * 2 + 0] - envMin.x) / envWidth.x) * gridDim.x)),
            static_cast<int>(floorf(((xy[TID * 2 + 1] - envMin.y) / envWidth.y) * gridDim.y)),
            0
        };

        // Compute and set the bin index
        unsigned int bindex;

        bindex = (unsigned int)(
            (gridPos[1] * gridDim.x) +              // y
            gridPos[0]);                            // x

        binIndex[TID] = bindex;
    }
}
__global__ void calculateSpatialHash(float* x, float* y, float* z, unsigned int* binIndex, detail::Dims<float> envMin, detail::Dims<float> envWidth, detail::Dims<unsigned int> gridDim, unsigned int threadCount) {
    const unsigned int TID = blockIdx.x * blockDim.x + threadIdx.x;
    if (TID < threadCount) {
        // Compute hash (effectivley an index for to a bin within the partitioning grid in this case)
        int gridPos[3] = {
            static_cast<int>(floorf(((x[TID]-envMin.x) / envWidth.x)*gridDim.x)),
            static_cast<int>(floorf(((y[TID]-envMin.y) / envWidth.y)*gridDim.y)),
            0
        };

        // If 3D, set 3rd component
        if (z) {
            gridPos[2] = static_cast<int>(floorf(((z[TID]-envMin.z) / envWidth.z)*gridDim.z));
        }

        // Compute and set the bin index
        unsigned int bindex;

        if (z) {
            bindex = (unsigned int)(
            (gridPos[2] * gridDim.x * gridDim.y +   // z
            (gridPos[1] * gridDim.x) +              // y
            gridPos[0]));                           // x

        } else {
            bindex = (unsigned int)(
            (gridPos[1] * gridDim.x) +              // y
            gridPos[0]);                            // x
        }

        binIndex[TID] = bindex;
    }
}

void CUDASimulation::determineAgentsToSort() {
    const auto& am = model->agents;

    // Iterate agents and then agent functions to find any functions which use spatial messaging
    for (auto it = am.cbegin(); it != am.cend(); ++it) {
        const auto& mf = it->second->functions;
        for (auto it_f = mf.cbegin(); it_f != mf.cend(); ++it_f) {
            if (auto ptr = it_f->second->message_input.lock()) {
                // Check if this agent function uses 3D spatial messages
                if (ptr->getSortingType() == flamegpu::MessageSortingType::spatial3D) {
                    // Agent uses spatial, check it has correct variables
                    CAgentDescription ad(it->second);
                    if (ad.hasVariable("x") && ad.hasVariable("y") && ad.hasVariable("z")) {
                        auto& x = it->second->variables.at("x");
                        auto& y = it->second->variables.at("y");
                        auto& z = it->second->variables.at("z");
                        if (x.type == std::type_index(typeid(float)) && x.elements == 1 &&
                            y.type == std::type_index(typeid(float)) && y.elements == 1 &&
                            z.type == std::type_index(typeid(float)) && z.elements == 1) {
                            sortTriggers3D.insert(it_f->first);
                        }
                    } else if (ad.hasVariable("xyz")) {
                        auto& xyz = it->second->variables.at("xyz");
                        if (xyz.type == std::type_index(typeid(float)) && xyz.elements == 3) {
                            sortTriggers3D.insert(it_f->first);
                        }
                    }
                }

                // Check if this agent function uses 2D spatial messages
                if (ptr->getSortingType() == flamegpu::MessageSortingType::spatial2D) {
                    // Agent uses spatial, check it has correct variables
                    CAgentDescription ad(it->second);
                    if (ad.hasVariable("x") && ad.hasVariable("y")) {
                        auto& x = it->second->variables.at("x");
                        auto& y = it->second->variables.at("y");
                        if (x.type == std::type_index(typeid(float)) && x.elements == 1 &&
                            y.type == std::type_index(typeid(float)) && y.elements == 1) {
                            sortTriggers2D.insert(it_f->first);
                        }
                    } else if (ad.hasVariable("xy")) {
                        auto& xy = it->second->variables.at("xy");
                        if (xy.type == std::type_index(typeid(float)) && xy.elements == 2) {
                            sortTriggers2D.insert(it_f->first);
                        }
                    }
                }
            }
        }
    }
}


void CUDASimulation::spatialSortAgent_async(const std::string& funcName, const std::string& agentName, const std::string& state, const int mode, hipStream_t stream, unsigned int streamId) {
    // Fetch the appropriate message name
    detail::CUDAAgent& cuda_agent = getCUDAAgent(agentName);

    const unsigned int state_list_size = cuda_agent.getStateSize(state);
    // Can't sort no agents
    if (!state_list_size)
        return;

    const CAgentDescription cudaAgentData(cuda_agent.getAgentDescription());
    auto funcData = cudaAgentData.getFunction(funcName);
    if (!funcData.hasMessageInput()) {
        THROW exception::InvalidAgentFunc("Function %s registered for auto-spatial sorting but input message type not found!\n", funcName.c_str());
    }
    std::string messageName = funcData.getMessageInput().getName();
    MessageBruteForce::Data* msgData = model->messages.at(messageName).get();

    // Get the spatial metadata
    float radius;
    detail::Dims<float> envMin {};
    detail::Dims<float> envMax {};
    detail::Dims<float> envWidth {};
    detail::Dims<unsigned int> gridDim {};

    if (auto messageSpatialData2D = dynamic_cast<MessageSpatial2D::Data*>(msgData)) {
        radius = messageSpatialData2D->radius;
        envMin = {messageSpatialData2D->minX, messageSpatialData2D->minY, 0.0f};
        envMax = {messageSpatialData2D->maxX, messageSpatialData2D->maxY, 0.0f};
    } else if (auto messageSpatialData3D = dynamic_cast<MessageSpatial3D::Data*>(msgData)) {
        radius = messageSpatialData3D->radius;
        envMin = {messageSpatialData3D->minX, messageSpatialData3D->minY, messageSpatialData3D->minZ};
        envMax = {messageSpatialData3D->maxX, messageSpatialData3D->maxY, messageSpatialData3D->maxZ};
    } else {
        radius = 0.0f;
        envMin = {0.0f, 0.0f, 0.0f};
        envMax = {0.0f, 0.0f, 0.0f};
    }
    if (radius > 0.0f) {
        envWidth = {(envMax.x-envMin.x), (envMax.y-envMin.y), (envMax.z-envMin.z)};
        gridDim = {
            envWidth.x ? static_cast<unsigned int>(ceilf(envWidth.x / radius)) : 1,
            envWidth.y ? static_cast<unsigned int>(ceilf(envWidth.y / radius)) : 1,
            envWidth.z ? static_cast<unsigned int>(ceilf(envWidth.z / radius)) : 1
        };
    }


    // Any agent in this list is guaranteed to have x, y, z (or xyz vec versions) and _auto_sort_bin_index vars - used in the computation of spatial hash
    // TODO: User could supply alternatives to "x", "y", "z" to use alternative variables?
    void* xPtr = nullptr, *yPtr = nullptr, *zPtr = nullptr;
    void* xyPtr = nullptr, * xyzPtr = nullptr;
    if (mode == Agent3D && cudaAgentData.hasVariable("xyz")) {
        xyzPtr = cuda_agent.getStateVariablePtr(state, "xyz");
    } else if (mode == Agent2D && cudaAgentData.hasVariable("xy")) {
        xyPtr = cuda_agent.getStateVariablePtr(state, "xy");
    } else {
        xPtr = cuda_agent.getStateVariablePtr(state, "x");
        yPtr = cuda_agent.getStateVariablePtr(state, "y");
        zPtr = mode == Agent3D ? cuda_agent.getStateVariablePtr(state, "z") : 0;
    }

    void* binIndexPtr = cuda_agent.getStateVariablePtr(state, "_auto_sort_bin_index");

    // Compute occupancy
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, calculateSpatialHash, 0, state_list_size);

    //! Round up according to CUDAAgent state list size
    gridSize = (state_list_size + blockSize - 1) / blockSize;

    unsigned int sm_size = 0;
#if !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
    auto *error_buffer = this->singletons->exception.getDevicePtr(streamId, stream);
    sm_size = sizeof(error_buffer);
#endif

    // Launch kernel
    if (xyzPtr) {
        calculateSpatialHashFloat3<<<gridSize, blockSize, sm_size, stream>>>(reinterpret_cast<float*>(xyzPtr),
            reinterpret_cast<unsigned int*>(binIndexPtr),
            envMin,
            envWidth,
            gridDim,
            state_list_size);
    } else if (xyPtr) {
        calculateSpatialHashFloat2<<<gridSize, blockSize, sm_size, stream>>>(reinterpret_cast<float*>(xyPtr),
            reinterpret_cast<unsigned int*>(binIndexPtr),
            envMin,
            envWidth,
            gridDim,
            state_list_size);
    } else {
        calculateSpatialHash<<<gridSize, blockSize, sm_size, stream>>>(reinterpret_cast<float*>(xPtr),
            reinterpret_cast<float*>(yPtr),
            reinterpret_cast<float*>(zPtr),
            reinterpret_cast<unsigned int*>(binIndexPtr),
            envMin,
            envWidth,
            gridDim,
            state_list_size);
    }
    gpuErrchkLaunch();

    assert(host_api);
    // Calculate max bit (hipcub::DeviceRadixSort end bit is exclusive and 0-indexed)
    // https://math.stackexchange.com/a/160299/126129
    const int max_bit = static_cast<int>(floor(log2(gridDim.x * gridDim.y * gridDim.z))) + 1;
    host_api->agent(agentName, state).sort_async<unsigned int>("_auto_sort_bin_index", HostAgentAPI::Asc, 0, max_bit, stream, streamId);
}

bool CUDASimulation::step() {
    flamegpu::util::nvtx::Range range{std::string("CUDASimulation::step " + std::to_string(step_count)).c_str()};
    // Ensure singletons have been initialised
    initialiseSingletons();

    // Time the individual step, using a CUDAEventTimer if possible, else a steadyClockTimer.
    std::unique_ptr<detail::Timer> stepTimer = getDriverAppropriateTimer(getCUDAConfig().is_ensemble);
    stepTimer->start();

    // Init any unset agent IDs
    this->assignAgentIDs();

    // If verbose, print the step number.
    if (getSimulationConfig().verbosity == Verbosity::Verbose) {
        fprintf(stdout, "Processing Simulation Step %u\n", step_count);
    }


    // Ensure there are enough streams to execute the layer.
    // Taking into consideration if in-layer concurrency is disabled or not.
    unsigned int nStreams = getMaximumLayerWidth();
    this->createStreams(nStreams);

    // Reset message list flags
    for (auto m =  message_map.begin(); m != message_map.end(); ++m) {
        m->second->setTruncateMessageListFlag();
    }

    // Execute each layer of the simulation.
    unsigned int layerIndex = 0;
    for (auto& layer : model->layers) {
        // Execute the individual layer
        stepLayer(layer, layerIndex);
        // Increment counter
        ++layerIndex;
    }

    // Run the step functions (including pyhton.)
    stepStepFunctions();

    // Run the exit conditons, detecting wheter or not any we
    bool exitRequired = this->stepExitConditions();

    // Set message counts to zero, and set flags to update state of non-persistent message lists
    for (auto &a : message_map) {
        if (!a.second->getMessageData().persistent) {
            a.second->setMessageCount(0);
            a.second->setTruncateMessageListFlag();
            a.second->setPBMConstructionRequiredFlag();
        }
    }

    // Record, store and output the elapsed time of the step.
    stepTimer->stop();
    float stepMilliseconds = stepTimer->getElapsedSeconds();
    this->elapsedSecondsPerStep.push_back(stepMilliseconds);
    if (getSimulationConfig().timing || getSimulationConfig().verbosity >= Verbosity::Verbose) {
        // Resolution is 0.5 microseconds, so print to 1 us.
        fprintf(stdout, "Step %d Processing time: %.6f s\n", this->step_count, stepMilliseconds);
    }

    // Update step count at the end of the step - when it has completed.
    incrementStepCounter();
    // Update the log for the step.
    processStepLog(this->elapsedSecondsPerStep.back());
    // Return false if any exit condition's passed.
    return !exitRequired;
}

void CUDASimulation::stepLayer(const std::shared_ptr<LayerData>& layer, const unsigned int layerIndex) {
    flamegpu::util::nvtx::Range range{std::string("stepLayer " + std::to_string(layerIndex)).c_str()};

    std::string message_name;

    // If the layer contains a sub model, it can only execute the sub model.
    if (layer->sub_model) {
        this->synchronizeAllStreams();
        auto &sm = submodel_map.at(layer->sub_model->name);
        sm->resetStepCounter();
        sm->simulate();
        sm->reset(true);
        // Next layer, this layer cannot also contain agent functions
        // Ensure synchronisation has occurred.
        this->synchronizeAllStreams();
        return;
    }

    // Track stream index
    int streamIdx = 0;
    // Sum the total number of threads being launched in the layer
    unsigned int totalThreads = 0;

    // Sync the environment once per layer (incase Host Fns, or submodel have changed it)
    singletons->environment->updateDevice_async(getStream(0));

    // Spatially sort the agents
    for (const auto &func_des : layer->agent_functions) {
        auto func_agent = func_des->parent.lock();
        if ((func_agent->sortPeriod != 0) && (step_count % func_agent->sortPeriod == 0)) {
            if (sortTriggers3D.find(func_des->name) != sortTriggers3D.end()) {
                this->spatialSortAgent_async(func_des->name, func_agent->name, func_des->initial_state, Agent3D, getStream(streamIdx), streamIdx);
            } else if (sortTriggers2D.find(func_des->name) != sortTriggers2D.end()) {
                this->spatialSortAgent_async(func_des->name, func_agent->name, func_des->initial_state, Agent2D, getStream(streamIdx), streamIdx);
            }
        }
        ++streamIdx;
    }
    // No explicit sync, sorts should be in same stream as eventual kernel launch (digging deep, the underlying scatter method does have a sync though)
    streamIdx = 0;

    // Map agent memory
    for (const auto &func_des : layer->agent_functions) {
        if ((func_des->condition) || (!func_des->rtc_func_condition_name.empty())) {
            auto func_agent = func_des->parent.lock();
            flamegpu::util::nvtx::Range condition_range{std::string("condition map " + func_agent->name + "::" + func_des->name).c_str()};
            const detail::CUDAAgent& cuda_agent = getCUDAAgent(func_agent->name);

            const unsigned int state_list_size = cuda_agent.getStateSize(func_des->initial_state);
            if (state_list_size == 0) {
                ++streamIdx;
                continue;
            }
            singletons->scatter.Scan().resize(state_list_size, detail::CUDAScanCompaction::AGENT_DEATH, streamIdx);

            // Configure runtime access of the functions variables within the FLAME_API object
            cuda_agent.mapRuntimeVariables(*func_des, instance_id);

            // Zero the scan flag that will be written to
            singletons->scatter.Scan().zero_async(detail::CUDAScanCompaction::AGENT_DEATH, getStream(streamIdx), streamIdx);
            // No sync, this occurs in same stream as dependent kernel launch

            // Push function's RTC cache to device if using RTC
            if (!func_des->rtc_func_condition_name.empty()) {
                auto &rtc_header = cuda_agent.getRTCHeader(func_des->name + "_condition");
                // Sync EnvManager's RTC cache with RTC header's cache
                rtc_header.updateEnvCache(singletons->environment->getHostBuffer(), singletons->environment->getBufferLen());
                // Push RTC header's cache to device
                rtc_header.updateDevice_async(cuda_agent.getRTCInstantiation(func_des->name + "_condition"), getStream(streamIdx));
            } else {
                auto& curve = cuda_agent.getCurve(func_des->name + "_condition");
                curve.updateDevice_async(this->getStream(streamIdx));
            }
            // No sync, kernel launch should be in same stream

            totalThreads += state_list_size;
        }
        ++streamIdx;
    }

    // If any condition kernel needs to be executed, do so, by checking the number of threads from before.
    if (totalThreads > 0) {
        // Ensure RandomManager is the correct size to accommodate all threads to be launched
        detail::hiprandState *d_rng = singletons->rng.resize(totalThreads, getStream(0));
        // Track which stream to use for concurrency
        streamIdx = 0;
        // Sum the total number of threads being launched in the layer, for rng offsetting.
        totalThreads = 0;
        // Launch function condition kernels
        for (const auto &func_des : layer->agent_functions) {
            if ((func_des->condition) || (!func_des->rtc_func_condition_name.empty())) {
                auto func_agent = func_des->parent.lock();
                flamegpu::util::nvtx::Range condition_range{std::string("condition " + func_agent->name + "::" + func_des->name).c_str()};
                if (!func_agent) {
                    THROW exception::InvalidAgentFunc("Agent function condition refers to expired agent.");
                }
                std::string agent_name = func_agent->name;
                std::string func_name = func_des->name;

                const detail::CUDAAgent& cuda_agent = getCUDAAgent(agent_name);

                const unsigned int state_list_size = cuda_agent.getStateSize(func_des->initial_state);
                if (state_list_size == 0) {
                    ++streamIdx;
                    continue;
                }

                int blockSize = 0;  // The launch configurator returned block size
                int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
                int gridSize = 0;  // The actual grid size needed, based on input size

                //  Agent function condition kernel wrapper args
                detail::hiprandState *t_rng = d_rng + totalThreads;
                unsigned int *scanFlag_agentDeath = this->singletons->scatter.Scan().Config(detail::CUDAScanCompaction::Type::AGENT_DEATH, streamIdx).d_ptrs.scan_flag;
#if !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
                auto *error_buffer = this->singletons->exception.getDevicePtr(streamIdx, this->getStream(streamIdx));
#endif
                // switch between normal and RTC agent function condition
                if (func_des->condition) {
                    // calculate the grid block size for agent function condition
                    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, func_des->condition, 0, state_list_size);

                    //! Round up according to CUDAAgent state list size
                    gridSize = (state_list_size + blockSize - 1) / blockSize;
                    (func_des->condition) << <gridSize, blockSize, 0, this->getStream(streamIdx) >> > (
#if !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
                    error_buffer,
#endif
                    cuda_agent.getCurve(func_des->name + "_condition").getDevicePtr(),
                    static_cast<const char *>(this->singletons->environment->getDeviceBuffer()),
                    state_list_size,
                    t_rng,
                    scanFlag_agentDeath);
                    gpuErrchkLaunch();
                } else {  // RTC function
                    std::string func_condition_identifier = func_name + "_condition";
                    // get instantiation
                    const jitify::experimental::KernelInstantiation& instance = cuda_agent.getRTCInstantiation(func_condition_identifier);
                    // calculate the grid block size for main agent function
                    hipFunction_t cu_func = (hipFunction_t)instance;
                    hipModuleOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cu_func, 0, state_list_size);
                    //! Round up according to CUDAAgent state list size
                    gridSize = (state_list_size + blockSize - 1) / blockSize;
                    // launch the kernel
                    hipError_t a = instance.configure(gridSize, blockSize, 0, this->getStream(streamIdx)).launch({
#if !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
                        reinterpret_cast<void*>(&error_buffer),
#endif
                        const_cast<void *>(reinterpret_cast<const void*>(&state_list_size)),
                        reinterpret_cast<void*>(&t_rng),
                        reinterpret_cast<void*>(&scanFlag_agentDeath) });
                    if (a != hipError_t::hipSuccess) {
                        const char* err_str = nullptr;
                        hipDrvGetErrorString(a, &err_str);
                        THROW exception::InvalidAgentFunc("There was a problem launching the runtime agent function condition '%s': %s", func_des->rtc_func_condition_name.c_str(), err_str);
                    }
                    gpuErrchkLaunch();
                }

                totalThreads += state_list_size;
            }
            ++streamIdx;
        }
    }

    // Track stream index
    streamIdx = 0;
    // Unmap agent memory, apply condition.
    for (const auto &func_des : layer->agent_functions) {
        if ((func_des->condition) || (!func_des->rtc_func_condition_name.empty())) {
            auto func_agent = func_des->parent.lock();
            if (!func_agent) {
                THROW exception::InvalidAgentFunc("Agent function condition refers to expired agent.");
            }
            flamegpu::util::nvtx::Range unmap_range{std::string("condition unmap " + func_agent->name + "::" + func_des->name).c_str()};
            detail::CUDAAgent& cuda_agent = getCUDAAgent(func_agent->name);

            // Skip if no agents in the input state
            const unsigned int state_list_size = cuda_agent.getStateSize(func_des->initial_state);
            if (state_list_size == 0) {
                ++streamIdx;
                continue;
            }

#if !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
            // Error check after unmap vars
            this->singletons->exception.checkError("condition " + func_des->name, streamIdx, this->getStream(streamIdx));
#endif
            // Process agent function condition
            cuda_agent.processFunctionCondition(*func_des, this->singletons->scatter, streamIdx, this->getStream(streamIdx));
        }
        // Increment the stream tracker.
        ++streamIdx;
    }

    streamIdx = 0;
    // Sum the total number of threads being launched in the layer
    totalThreads = 0;
    // for each func function - Loop through to do all mapping of agent and message variables
    for (const auto &func_des : layer->agent_functions) {
        auto func_agent = func_des->parent.lock();
        if (!func_agent) {
            THROW exception::InvalidAgentFunc("Agent function refers to expired agent.");
        }
        flamegpu::util::nvtx::Range map_range{std::string("map" + func_agent->name + "::" + func_des->name).c_str()};

        const detail::CUDAAgent& cuda_agent = getCUDAAgent(func_agent->name);
        const unsigned int state_list_size = cuda_agent.getStateSize(func_des->initial_state);
        if (state_list_size == 0) {
            ++streamIdx;
            continue;
        }
        // Resize death flag array if necessary
        singletons->scatter.Scan().resize(state_list_size, detail::CUDAScanCompaction::AGENT_DEATH, streamIdx);

        // check if a function has an input message
        if (auto im = func_des->message_input.lock()) {
            std::string inpMessage_name = im->name;
            detail::CUDAMessage& cuda_message = getCUDAMessage(inpMessage_name);
            // Construct PBM here if required!!
            cuda_message.buildIndex(this->singletons->scatter, streamIdx, this->getStream(streamIdx));  // This is synchronous.
            // Map variables after, as index building can swap arrays
            cuda_message.mapReadRuntimeVariables(*func_des, cuda_agent);
        }

        // check if a function has an output message
        if (auto om = func_des->message_output.lock()) {
            std::string outpMessage_name = om->name;
            detail::CUDAMessage& cuda_message = getCUDAMessage(outpMessage_name);
            // Resize message list if required
            const unsigned int existingMessages = cuda_message.getTruncateMessageListFlag() ? 0 : cuda_message.getMessageCount();
            cuda_message.resize(existingMessages + state_list_size, this->singletons->scatter, getStream(streamIdx), streamIdx, existingMessages);  // This could have it's internal syncs delayed
            cuda_message.mapWriteRuntimeVariables(*func_des, cuda_agent, state_list_size, getStream(streamIdx));
            singletons->scatter.Scan().resize(state_list_size, detail::CUDAScanCompaction::MESSAGE_OUTPUT, streamIdx);
            // Zero the scan flag that will be written to
            if (func_des->message_output_optional)
                singletons->scatter.Scan().zero_async(detail::CUDAScanCompaction::MESSAGE_OUTPUT, getStream(streamIdx), streamIdx);
                // No Sync, any subsequent use should be in same stream
        }

        // check if a function has an output agent
        if (auto oa = func_des->agent_output.lock()) {
            // This will act as a reserve word
            // which is added to variable hashes for agent creation on device
            detail::CUDAAgent& output_agent = getCUDAAgent(oa->name);

            // Map vars with curve (this allocates/requests enough new buffer space if an existing version is not available/suitable)
            output_agent.mapNewRuntimeVariables_async(cuda_agent, *func_des, state_list_size, this->singletons->scatter, instance_id, getStream(streamIdx), streamIdx);
            // No Sync, any subsequent use should be in same stream
        }

        // Configure runtime access of the functions variables within the FLAME_API object
        cuda_agent.mapRuntimeVariables(*func_des, instance_id);

        // Zero the scan flag that will be written to
        if (func_des->has_agent_death) {
            singletons->scatter.Scan().zero_async(detail::CUDAScanCompaction::AGENT_DEATH, getStream(streamIdx), streamIdx);
            // No Sync, any subsequent use should be in same stream
        }

        // Push function's RTC cache to device if using RTC
        if (!func_des->rtc_func_name.empty()) {
            auto& rtc_header = cuda_agent.getRTCHeader(func_des->name);
            // Sync EnvManager's RTC cache with RTC header's cache
            rtc_header.updateEnvCache(singletons->environment->getHostBuffer(), singletons->environment->getBufferLen());
            // Push RTC header's cache to device
            rtc_header.updateDevice_async(cuda_agent.getRTCInstantiation(func_des->name), getStream(streamIdx));
        } else {
            auto& curve = cuda_agent.getCurve(func_des->name);
            curve.updateDevice_async(this->getStream(streamIdx));
        }
        // No sync, kernel launch should be in the same stream

        // Count total threads being launched
        totalThreads += cuda_agent.getStateSize(func_des->initial_state);
        ++streamIdx;
    }

    // If any kernel needs to be executed, do so, by checking the number of threads from before.
    if (totalThreads > 0) {
        // Ensure RandomManager is the correct size to accommodate all threads to be launched
        detail::hiprandState *d_rng = singletons->rng.resize(totalThreads, getStream(0));
        // Total threads is now used to provide kernel launches an offset to thread-safe thread-index
        totalThreads = 0;
        streamIdx = 0;

        // for each func function - Loop through to launch all agent functions
        for (const auto &func_des : layer->agent_functions) {
            auto func_agent = func_des->parent.lock();
            if (!func_agent) {
                THROW exception::InvalidAgentFunc("Agent function refers to expired agent.");
            }
            flamegpu::util::nvtx::Range func_range{std::string(func_agent->name + "::" + func_des->name).c_str()};
            const void *d_in_messagelist_metadata = nullptr;
            const void *d_out_messagelist_metadata = nullptr;
            id_t *d_agentOut_nextID = nullptr;
            std::string agent_name = func_agent->name;
            std::string func_name = func_des->name;

            // check if a function has an input message
            if (auto im = func_des->message_input.lock()) {
                std::string inpMessage_name = im->name;
                const detail::CUDAMessage& cuda_message = getCUDAMessage(inpMessage_name);

                d_in_messagelist_metadata = cuda_message.getMetaDataDevicePtr();
            }

            // check if a function has an output message
            if (auto om = func_des->message_output.lock()) {
                std::string outpMessage_name = om->name;
                const detail::CUDAMessage& cuda_message = getCUDAMessage(outpMessage_name);

                d_out_messagelist_metadata = cuda_message.getMetaDataDevicePtr();
            }

            // check if a function has agent output
            if (auto oa = func_des->agent_output.lock()) {
                detail::CUDAAgent& output_agent = getCUDAAgent(oa->name);
                d_agentOut_nextID = output_agent.getDeviceNextID();
            }

            const detail::CUDAAgent& cuda_agent = getCUDAAgent(agent_name);

            const unsigned int state_list_size = cuda_agent.getStateSize(func_des->initial_state);
            if (state_list_size == 0) {
                ++streamIdx;
                continue;
            }

            int blockSize = 0;  // The launch configurator returned block size
            int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
            int gridSize = 0;  // The actual grid size needed, based on input size

            // Agent function kernel wrapper args
            detail::hiprandState *t_rng = d_rng + totalThreads;
            unsigned int *scanFlag_agentDeath = func_des->has_agent_death ? this->singletons->scatter.Scan().Config(detail::CUDAScanCompaction::Type::AGENT_DEATH, streamIdx).d_ptrs.scan_flag : nullptr;
            unsigned int *scanFlag_messageOutput = this->singletons->scatter.Scan().Config(detail::CUDAScanCompaction::Type::MESSAGE_OUTPUT, streamIdx).d_ptrs.scan_flag;
            unsigned int *scanFlag_agentOutput = this->singletons->scatter.Scan().Config(detail::CUDAScanCompaction::Type::AGENT_OUTPUT, streamIdx).d_ptrs.scan_flag;
    #if !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
            auto *error_buffer = this->singletons->exception.getDevicePtr(streamIdx, this->getStream(streamIdx));
    #endif

            if (func_des->func) {   // compile time specified agent function launch
                // calculate the grid block size for main agent function
                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, func_des->func, 0, state_list_size);
                //! Round up according to CUDAAgent state list size
                gridSize = (state_list_size + blockSize - 1) / blockSize;

                (func_des->func) << <gridSize, blockSize, 0, this->getStream(streamIdx) >> > (
    #if !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
                    error_buffer,
    #endif
                    cuda_agent.getCurve(func_des->name).getDevicePtr(),
                    static_cast<const char*>(this->singletons->environment->getDeviceBuffer()),
                    d_agentOut_nextID,
                    state_list_size,
                    d_in_messagelist_metadata,
                    d_out_messagelist_metadata,
                    t_rng,
                    scanFlag_agentDeath,
                    scanFlag_messageOutput,
                    scanFlag_agentOutput);
                gpuErrchkLaunch();
            } else {      // assume this is a runtime specified agent function
                // get instantiation
                const jitify::experimental::KernelInstantiation& instance = cuda_agent.getRTCInstantiation(func_name);
                // calculate the grid block size for main agent function
                hipFunction_t cu_func = (hipFunction_t)instance;
                hipModuleOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cu_func, 0, state_list_size);
                //! Round up according to CUDAAgent state list size
                gridSize = (state_list_size + blockSize - 1) / blockSize;
                // launch the kernel
                hipError_t a = instance.configure(gridSize, blockSize, 0, this->getStream(streamIdx)).launch({
#if !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
                    reinterpret_cast<void*>(&error_buffer),
#endif
                    reinterpret_cast<void*>(&d_agentOut_nextID),
                    const_cast<void*>(reinterpret_cast<const void*>(&state_list_size)),
                    const_cast<void*>(reinterpret_cast<const void*>(&d_in_messagelist_metadata)),
                    const_cast<void*>(reinterpret_cast<const void*>(&d_out_messagelist_metadata)),
                    const_cast<void*>(reinterpret_cast<const void*>(&t_rng)),
                    reinterpret_cast<void*>(&scanFlag_agentDeath),
                    reinterpret_cast<void*>(&scanFlag_messageOutput),
                    reinterpret_cast<void*>(&scanFlag_agentOutput)});
                if (a != hipError_t::hipSuccess) {
                    const char* err_str = nullptr;
                    hipDrvGetErrorString(a, &err_str);
                    THROW exception::InvalidAgentFunc("There was a problem launching the runtime agent function '%s': %s", func_name.c_str(), err_str);
                }
                gpuErrchkLaunch();
            }
            totalThreads += state_list_size;
            ++streamIdx;
        }
    }

    streamIdx = 0;
    // for each func function - Loop through to un-map all agent and message variables
    for (const auto &func_des : layer->agent_functions) {
        auto func_agent = func_des->parent.lock();
        if (!func_agent) {
            THROW exception::InvalidAgentFunc("Agent function refers to expired agent.");
        }
        flamegpu::util::nvtx::Range unmap_range{std::string("unmap" + func_agent->name + "::" + func_des->name).c_str()};
        detail::CUDAAgent& cuda_agent = getCUDAAgent(func_agent->name);

        const unsigned int state_list_size = cuda_agent.getStateSize(func_des->initial_state);
        // If agent function wasn't executed, these are redundant
        if (state_list_size > 0) {
            // check if a function has an output message
            if (auto om = func_des->message_output.lock()) {
                std::string outpMessage_name = om->name;
                detail::CUDAMessage& cuda_message = getCUDAMessage(outpMessage_name);
                cuda_message.swap(func_des->message_output_optional, state_list_size, this->singletons->scatter, getStream(streamIdx), streamIdx);
                cuda_message.clearTruncateMessageListFlag();
                cuda_message.setPBMConstructionRequiredFlag();
            }

            // Process agent death (has agent death check is handled by the method)
            // This MUST occur before agent_output, as if agent_output triggers resize then scan_flag for death will be purged
            cuda_agent.processDeath(*func_des, this->singletons->scatter, streamIdx, this->getStream(streamIdx));

            // Process agent state transition (Longer term merge this with process death?)
            cuda_agent.transitionState(func_des->initial_state, func_des->end_state, this->singletons->scatter, streamIdx, this->getStream(streamIdx));
        }

        // Process agent function condition
        cuda_agent.clearFunctionCondition(func_des->initial_state);

        // If agent function wasn't executed, these are redundant
        if (state_list_size > 0) {
            // check if a function has an output agent
            if (auto oa = func_des->agent_output.lock()) {
                // This will act as a reserve word
                // which is added to variable hashes for agent creation on device
                detail::CUDAAgent& output_agent = getCUDAAgent(oa->name);
                // Scatter the agent birth
                output_agent.scatterNew(*func_des, state_list_size, this->singletons->scatter, streamIdx, this->getStream(streamIdx));
                output_agent.releaseNewBuffer(*func_des);
            }

#if !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
            // Error check after unmap vars
            // This means that curve is cleaned up before we throw exception (mostly prevents curve being polluted if we catch and handle errors)
            this->singletons->exception.checkError(func_des->name, streamIdx, this->getStream(streamIdx));
#endif
        }

        ++streamIdx;
    }

    // Synchronise to ensure that device memory is in a goood state prior to host layer functions? This can potentially be removed
    this->synchronizeAllStreams();

    // Execute the host functions.
    layerHostFunctions(layer, layerIndex);

#if !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
    // Reset macro-environment read-write flags
    // Note this does not synchronise threads, it relies on synchronizeAllStreams() post host fns
    macro_env.resetFlagsAsync(streams);
#endif

    // Synchronise  after the host layer functions to ensure that the device is up to date? This can potentially be removed.
    this->synchronizeAllStreams();
}

void CUDASimulation::layerHostFunctions(const std::shared_ptr<LayerData>& layer, const unsigned int layerIndex) {
    flamegpu::util::nvtx::Range range{"CUDASimulation::stepHostFunctions"};
    // Execute all host functions attached to layer
    // TODO: Concurrency?
    assert(host_api);
    for (auto &stepFn : layer->host_functions) {
        flamegpu::util::nvtx::Range hostfn_range{"hostFunc"};
        stepFn(this->host_api.get());
    }
    // Execute all host function callbacks attached to layer
    for (auto &stepFn : layer->host_functions_callbacks) {
        flamegpu::util::nvtx::Range hostfncallback_range{"hostFunc_swig"};
        stepFn->run(this->host_api.get());
    }
    // If we have host layer functions, we might have host agent creation
    if (layer->host_functions.size() || (layer->host_functions_callbacks.size())) {
        // Sync any device vectors, before performing host agent creation
        for (auto& ca : agent_map) {
            ca.second->resetPopulationVecs();
        }
        // @todo - What is the most appropriate stream to use here?
        processHostAgentCreation(0);
    }
}

void CUDASimulation::stepStepFunctions() {
    flamegpu::util::nvtx::Range range{"CUDASimulation::step::StepFunctions"};
    // Execute step functions
    for (auto &stepFn : model->stepFunctions) {
        flamegpu::util::nvtx::Range step_range{"stepFunc"};
        stepFn(this->host_api.get());
    }
    // Execute step function callbacks
    for (auto &stepFn : model->stepFunctionCallbacks) {
        flamegpu::util::nvtx::Range callback_range{"stepFunc_swig"};
        stepFn->run(this->host_api.get());
    }
    // If we have step functions, we might have host agent creation
    if (model->stepFunctions.size() || model->stepFunctionCallbacks.size()) {
        // Sync any device vectors, before performing host agent creation
        for (auto &ca : agent_map) {
            ca.second->resetPopulationVecs();
        }
        processHostAgentCreation(0);
    }
}

bool CUDASimulation::stepExitConditions() {
    flamegpu::util::nvtx::Range range{"CUDASimulation::stepExitConditions"};
    // Track if any exit conditions were successful. Use this to control return code and skipsteps.
    // early returning makes timing/stepCounter logic more complicated.
    bool exitConditionExit = false;

    // Execute exit conditions
    for (auto &exitCdns : model->exitConditions) {
        if (exitCdns(this->host_api.get()) == EXIT) {
            #ifdef FLAMEGPU_VISUALISATION
                if (visualisation) {
                    visualisation->updateBuffers(step_count+1);
                }
            #endif
            // Set the flag, and bail out of the exit condition loop.
            exitConditionExit = true;
            break;
        }
    }
    // Execute exit condition callbacks
    if (!exitConditionExit) {
        for (auto &exitCdns : model->exitConditionCallbacks) {
            if (exitCdns->run(this->host_api.get()) == EXIT) {
                #ifdef FLAMEGPU_VISUALISATION
                if (visualisation) {
                    visualisation->updateBuffers(step_count+1);
                }
                #endif
                // Set the flag, and bail out of the exit condition loop.
                exitConditionExit = true;
                break;
            }
        }
    }
    // No need for this if any exit conditions passed.
    if (!exitConditionExit) {
        // If we have exit conditions functions, we might have host agent creation
        if (model->exitConditions.size() || model->exitConditionCallbacks.size()) {
            processHostAgentCreation(0);
        }

        #ifdef FLAMEGPU_VISUALISATION
            if (visualisation) {
                visualisation->updateBuffers(step_count+1);
            }
        #endif
    }
    return exitConditionExit;
}

void CUDASimulation::simulate() {
    flamegpu::util::nvtx::Range range{"CUDASimulation::simulate"};

    // Ensure there is work to do.
    if (agent_map.size() == 0) {
        THROW exception::InvalidCudaAgentMapSize("Simulation has no agents, in CUDASimulation::simulate().");
    }

    // Ensure singletons have been initialised
    initialiseSingletons();

    // Create the event timing object, using an appropriate timer implementation.
    std::unique_ptr<detail::Timer> simulationTimer = getDriverAppropriateTimer(getCUDAConfig().is_ensemble);
    simulationTimer->start();

    // Create as many streams as required
    unsigned int nStreams = getMaximumLayerWidth();
    this->createStreams(nStreams);

    // Init any unset agent IDs
    this->assignAgentIDs();

    // Reinitialise any unmapped agent variables
    if (submodel) {
        int streamIdx = 0;
        for (auto &a : agent_map) {
            a.second->initUnmappedVars(this->singletons->scatter, streamIdx, this->getStream(streamIdx));
            streamIdx++;
        }
    }

    // Reset the class' elapsed time value.
    this->elapsedSecondsSimulation = 0.f;
    this->elapsedSecondsPerStep.clear();
    if (getSimulationConfig().steps > 0) {
        this->elapsedSecondsPerStep.reserve(getSimulationConfig().steps);
    }

    // Execute init functions
    this->initFunctions();

    // Reset and log initial state to step log 0
    resetLog();
    processStepLog(this->elapsedSecondsRTCInitialisation + this->elapsedSecondsInitFunctions);

    #ifdef FLAMEGPU_VISUALISATION
    // Pre step-loop visualisation update
    if (visualisation) {
        visualisation->updateBuffers();
    }
    visualiser::ModelVis mv(visualisation);
    #endif

    // Run the required number of simulation steps.
    for (unsigned int i = 0; getSimulationConfig().steps == 0 ? true : i < getSimulationConfig().steps; i++) {
        // Run the step
        bool continueSimulation = step();
        if (!continueSimulation) {
            break;
        }
        #ifdef FLAMEGPU_VISUALISATION

        // Special case, if steps == 0 and visualisation has been closed
        if (getSimulationConfig().steps == 0 &&
            visualisation && !mv.isRunning()) {
            mv.join();  // Vis exists in separate thread, make sure it has actually exited
            break;
        }
        #endif
    }

    // Exit functions
    this->exitFunctions();

    // Sync visualistaion after the exit functions
    #ifdef FLAMEGPU_VISUALISATION
    if (visualisation) {
        visualisation->updateBuffers();
    }
    #endif

    // Record, store and output the elapsed simulation time
    simulationTimer->stop();
    elapsedSecondsSimulation = simulationTimer->getElapsedSeconds();
    if (getSimulationConfig().timing || getSimulationConfig().verbosity >= Verbosity::Verbose) {
        // Resolution is 0.5 microseconds, so print to 1 us.
        fprintf(stdout, "Total Processing time: %.6f s\n", elapsedSecondsSimulation);
    }
    processExitLog();

    // Send Telemetry
    if (getSimulationConfig().telemetry) {
        // Generate some payload items
        std::map<std::string, std::string> payload_items;
        payload_items["GPUDevices"] = flamegpu::detail::compute_capability::getDeviceName(deviceInitialised);
        payload_items["SimTime(s)"] = std::to_string(elapsedSecondsSimulation);
        #if defined(__CUDACC_VER_MAJOR__) && defined(__CUDACC_VER_MINOR__) && defined(__CUDACC_VER_PATCH__)
            payload_items["NVCCVersion"] = std::to_string(__CUDACC_VER_MAJOR__) + "." + std::to_string(__CUDACC_VER_MINOR__) + "." + std::to_string(__CUDACC_VER_BUILD__);
        #endif
        // generate telemtry data
        std::string telemetry_data = flamegpu::io::Telemetry::generateData("simulation-run", payload_items);
        // send
        if (!flamegpu::io::Telemetry::sendData(telemetry_data)) {
            if ((getSimulationConfig().verbosity > Verbosity::Verbose))
                fprintf(stderr, "Warning: Usage statistics for CUDASimulation failed to send.\n");
        }
        // print
        if ((getSimulationConfig().verbosity >= Verbosity::Verbose)) {
            fprintf(stdout, "Telemetry packet sent to '%s' json was: %s\n", flamegpu::io::Telemetry::TELEMETRY_ENDPOINT, telemetry_data.c_str());
        }
    } else {
        // Occasional hinting of telemetry if not in use (and not Quiet and not testing mode)
        if ((getSimulationConfig().verbosity > Verbosity::Quiet))
            flamegpu::io::Telemetry::encourageUsage();
    }

    // Export logs
    if (!SimulationConfig().step_log_file.empty())
        exportLog(SimulationConfig().step_log_file, true, false, step_log_config && step_log_config->log_timing, false);
    if (!SimulationConfig().exit_log_file.empty())
        exportLog(SimulationConfig().exit_log_file, false, true, false, exit_log_config && exit_log_config->log_timing);
    if (!SimulationConfig().common_log_file.empty())
        exportLog(SimulationConfig().common_log_file, true, true, step_log_config && step_log_config->log_timing, exit_log_config && exit_log_config->log_timing);
}


void CUDASimulation::simulate(const RunPlan& plan) {
    // Validate that RunPlan is for same ModelDesc
    // RunPlan only holds a copy of env, so we must compare those
    if (*plan.environment != model->environment->properties) {
        THROW exception::InvalidArgument("RunPlan's associated environment does not match the ModelDescription's environment, "
        "in CUDASimulation::simulate(RunPlan)\n");
    }
    // Backup config
    const uint64_t t_random_seed = SimulationConfig().random_seed;
    const unsigned int t_steps = SimulationConfig().steps;
    // Temp override config
    SimulationConfig().steps = plan.getSteps();
    SimulationConfig().random_seed = plan.getRandomSimulationSeed();
    // Ensure singletons have been initialised (so env actually exists in mgr)
    initialiseSingletons();
    // Override environment properties
    for (auto& ovrd : plan.property_overrides) {
        singletons->environment->setPropertyDirect(ovrd.first, static_cast<char *>(ovrd.second.ptr));
    }
    // Call regular simulate
    simulate();
    // Reset config
    SimulationConfig().random_seed = t_random_seed;
    SimulationConfig().steps = t_steps;
}

void CUDASimulation::reset(bool submodelReset) {
    // Reset step counter
    resetStepCounter();

    if (singletonsInitialised) {
        // Reset environment properties
        singletons->environment->resetModel(*model->environment);

        // Reseed random, unless performing submodel reset
        if (!submodelReset) {
            singletons->rng.reseed(getSimulationConfig().random_seed);
        }
    }

    // Cull agents
    if (submodel) {
        // Submodels only want to reset unmapped states, otherwise they will break parent model
        for (auto &a : agent_map) {
            a.second->cullUnmappedStates();
        }
    } else {
        for (auto &a : agent_map) {
            a.second->cullAllStates();
        }
    }

    // Cull messagelists
    for (auto &a : message_map) {
        a.second->setMessageCount(0);
        a.second->setTruncateMessageListFlag();
        a.second->setPBMConstructionRequiredFlag();
    }


    // Trigger reset in all submodels, propagation is not necessary when performing submodel reset
    if (!submodelReset) {
        for (auto &s : submodel_map) {
            s.second->reset(false);
        }
    }

    // Reset any timing data.
    this->elapsedSecondsSimulation = 0.f;
    this->elapsedSecondsPerStep.clear();
}

void CUDASimulation::setPopulationData(AgentVector& population, const std::string& state_name) {
    // Ensure singletons have been initialised
    initialiseSingletons();
    flamegpu::util::nvtx::Range range{"CUDASimulation::setPopulationData()"};
    auto it = agent_map.find(population.getAgentName());
    if (it == agent_map.end()) {
        THROW exception::InvalidAgent("Agent '%s' was not found, "
            "in CUDASimulation::setPopulationData()",
            population.getAgentName().c_str());
    }
    // This call hierarchy validates agent desc matches and state is valid
    it->second->setPopulationData(population, state_name, this->singletons->scatter, 0, getStream(0));  // Streamid shouldn't matter here
#ifdef FLAMEGPU_VISUALISATION
    if (visualisation) {
        visualisation->updateBuffers();
    }
#endif
    gpuErrchk(hipDeviceSynchronize());
    agent_ids_have_init = false;
}
void CUDASimulation::getPopulationData(AgentVector& population, const std::string& state_name) {
    // Ensure singletons have been initialised
    initialiseSingletons();
    flamegpu::util::nvtx::Range range{"CUDASimulation::getPopulationData()"};
    gpuErrchk(hipDeviceSynchronize());
    auto it = agent_map.find(population.getAgentName());
    if (it == agent_map.end()) {
        THROW exception::InvalidAgent("Agent '%s' was not found, "
            "in CUDASimulation::setPopulationData()",
            population.getAgentName().c_str());
    }
    // This call hierarchy validates agent desc matches and state is valid
    it->second->getPopulationData(population, state_name);
    gpuErrchk(hipDeviceSynchronize());
}

detail::CUDAAgent& CUDASimulation::getCUDAAgent(const std::string& agent_name) const {
    CUDAAgentMap::const_iterator it;
    it = agent_map.find(agent_name);

    if (it == agent_map.end()) {
        THROW exception::InvalidCudaAgent("CUDA agent ('%s') not found, in CUDASimulation::getCUDAAgent().",
            agent_name.c_str());
    }

    return *(it->second);
}

detail::AgentInterface& CUDASimulation::getAgent(const std::string& agent_name) {
    // Ensure singletons have been initialised
    initialiseSingletons();

    auto it = agent_map.find(agent_name);

    if (it == agent_map.end()) {
        THROW exception::InvalidCudaAgent("CUDA agent ('%s') not found, in CUDASimulation::getAgent().",
            agent_name.c_str());
    }

    return *(it->second);
}

detail::CUDAMessage& CUDASimulation::getCUDAMessage(const std::string& message_name) const {
    CUDAMessageMap::const_iterator it;
    it = message_map.find(message_name);

    if (it == message_map.end()) {
        THROW exception::InvalidCudaMessage("CUDA message ('%s') not found, in CUDASimulation::getCUDAMessage().",
            message_name.c_str());
    }

    return *(it->second);
}

void CUDASimulation::setStepLog(const StepLoggingConfig &stepConfig) {
    // Validate ModelDescription matches
    if (*stepConfig.model != *model) {
        THROW exception::InvalidArgument("Model descriptions attached to LoggingConfig and CUDASimulation do not match, in CUDASimulation::setStepLog()\n");
    }
    // Set internal config
    step_log_config = std::make_shared<StepLoggingConfig>(stepConfig);
}
void CUDASimulation::setExitLog(const LoggingConfig &exitConfig) {
    // Validate ModelDescription matches
    if (*exitConfig.model != *model) {
        THROW exception::InvalidArgument("Model descriptions attached to LoggingConfig and CUDASimulation do not match, in CUDASimulation::setExitLog()\n");
    }
    // Set internal config
    exit_log_config = std::make_shared<LoggingConfig>(exitConfig);
}

bool CUDASimulation::checkArgs_derived(int argc, const char** argv, int &i) {
    // Get arg as lowercase
    std::string arg(argv[i]);
    std::transform(arg.begin(), arg.end(), arg.begin(), [](unsigned char c) { return std::use_facet< std::ctype<char>>(std::locale()).tolower(c); });
    // -device <uint>, Uses the specified cuda device, defaults to 0
    if ((arg.compare("--device") == 0 || arg.compare("-d") == 0) && argc > i+1) {
        config.device_id = static_cast<unsigned int>(strtoul(argv[++i], nullptr, 0));
        return true;
    }
    return false;
}

void CUDASimulation::printHelp_derived() {
    const char *line_fmt = "%-18s %s\n";
    printf("CUDA Model Optional Arguments:\n");
    printf(line_fmt, "-d, --device", "GPU index");
}

void CUDASimulation::applyConfig_derived() {
    flamegpu::util::nvtx::Range range{"applyConfig_derived"};

    // Handle console_mode
#ifdef FLAMEGPU_VISUALISATION
    if (visualisation) {
        visualiser::ModelVis mv(visualisation);
        if (getSimulationConfig().console_mode) {
            mv.deactivate();
        } else {
            visualisation->updateRandomSeed();
        }
    }
#endif


    hipError_t cudaStatus;
    int device_count;

    // default device
    cudaStatus = hipGetDeviceCount(&device_count);

    if (cudaStatus != hipSuccess) {
        THROW exception::InvalidCUDAdevice("Error finding CUDA devices!  Do you have a CUDA-capable GPU installed?");
    }
    if (device_count == 0) {
        THROW exception::InvalidCUDAdevice("Error no CUDA devices found!");
    }

    // Select device
    if (config.device_id >= device_count) {
        THROW exception::InvalidCUDAdevice("Error setting CUDA device to '%d', only %d available!", config.device_id, device_count);
    }
    if (deviceInitialised !=- 1 && deviceInitialised != config.device_id) {
        THROW exception::InvalidCUDAdevice("Unable to set CUDA device to '%d' after the CUDASimulation has already initialised on device '%d'.", config.device_id, deviceInitialised);
    }

    // Check the compute capability of the device, throw an exception if not valid for the executable.
    if (!detail::compute_capability::checkComputeCapability(static_cast<int>(config.device_id))) {
        int min_cc = detail::compute_capability::minimumCompiledComputeCapability();
        int cc = detail::compute_capability::getComputeCapability(static_cast<int>(config.device_id));
        THROW exception::InvalidCUDAComputeCapability("Error application compiled for CUDA Compute Capability %d and above. Device %u is compute capability %d. Rebuild for SM_%d.", min_cc, config.device_id, cc, cc);
    }

    cudaStatus = hipSetDevice(static_cast<int>(config.device_id));
    if (cudaStatus != hipSuccess) {
        THROW exception::InvalidCUDAdevice("Unknown error setting CUDA device to '%d'. (%d available)", config.device_id, device_count);
    }
    // Call hipFree to initialise the context early
    gpuErrchk(hipFree(nullptr));

    // Apply changes to submodels
    for (auto &sm : submodel_map) {
        // We're not actually going to use this value, but it might be useful there later
        // Calling apply config a second time would reinit GPU, which might clear existing gpu allocations etc
        sm.second->CUDAConfig().device_id = config.device_id;
    }

    // Initialise singletons once a device has been selected.
    initialiseSingletons();

    // We init Random through submodel hierarchy after singletons
    reseed(getSimulationConfig().random_seed);
}

void CUDASimulation::reseed(const uint64_t seed) {
    SimulationConfig().random_seed = seed;
    singletons->rng.reseed(seed);

    // Propagate to submodels
    int i = 7;
    for (auto &sm : submodel_map) {
        // Pass random seed on to submodels
        sm.second->singletons->rng.reseed(getSimulationConfig().random_seed * i * 23);
        // Mutate seed
        i *= 13;
    }
}

void CUDASimulation::initialiseSingletons() {
    // Only do this once.
    if (!singletonsInitialised) {
        // If the device has not been specified, also check the compute capability is OK
        // Check the compute capability of the device, throw an exception if not valid for the executable.
        if (!detail::compute_capability::checkComputeCapability(static_cast<int>(config.device_id))) {
            int min_cc = detail::compute_capability::minimumCompiledComputeCapability();
            int cc = detail::compute_capability::getComputeCapability(static_cast<int>(config.device_id));
            THROW exception::InvalidCUDAComputeCapability("Error application compiled for CUDA Compute Capability %d and above. Device %u is compute capability %d. Rebuild for SM_%d.", min_cc, config.device_id, cc, cc);
        }
        gpuErrchk(hipGetDevice(&deviceInitialised));
        // Get references to all required singleton and store in the instance.
        singletons = new Singletons((!submodel)?
            detail::EnvironmentManager::create(*model->environment) :
            detail::EnvironmentManager::create(*model->environment, mastermodel->singletons->environment, *submodel->subenvironment));

        // Reinitialise random for this simulation instance
        singletons->rng.reseed(getSimulationConfig().random_seed);

        hipStream_t stream_0 = getStream(0);

        // Pass created RandomManager to host api
        host_api = std::make_unique<HostAPI>(*this, singletons->rng, singletons->scatter, agentOffsets, agentData, singletons->environment, macro_env, 0, stream_0);  // Host fns are currently all serial

        for (auto &cm : message_map) {
            cm.second->init(singletons->scatter, 0, stream_0);
        }

        // Populate the environment properties
        if (!submodel) {
            macro_env.init(stream_0);
        } else {
            macro_env.init(*submodel->subenvironment, mastermodel->macro_env, stream_0);
        }

        // Propagate singleton init to submodels
        for (auto &sm : submodel_map) {
            sm.second->initialiseSingletons();
        }

        // Store the WDDM/TCC driver mode status, for timer class decisions. Result is cached in the anon namespace to avoid multiple queries
        deviceUsingWDDM = detail::wddm::deviceIsWDDM();

#ifdef FLAMEGPU_VISUALISATION
        if (visualisation) {
            visualisation->updateRandomSeed();  // Incase user hasn't triggered applyConfig()
            visualisation->registerEnvProperties();
        }
#endif

        singletonsInitialised = true;
    } else {
        int t = -1;
        gpuErrchk(hipGetDevice(&t));
        if (t != deviceInitialised) {
            THROW exception::CUDAError("CUDASimulation initialised on device %d, but stepped on device %d.\n", deviceInitialised, t);
        }
    }
    // Populate the environment properties
    initEnvironmentMgr();

    // Ensure there are enough streams to execute the layer.
    // Taking into consideration if in-layer concurrency is disabled or not.
    unsigned int nStreams = getMaximumLayerWidth();
    this->createStreams(nStreams);

    // Ensure RTC is set up.
    initialiseRTC();
}

void CUDASimulation::initialiseRTC() {
    // Only do this once.
    if (!rtcInitialised) {
        flamegpu::util::nvtx::Range range{"CUDASimulation::initialiseRTC"};
        std::unique_ptr<detail::Timer> rtcTimer(new detail::SteadyClockTimer());
        rtcTimer->start();
        // Build any RTC functions
        const auto& am = model->agents;
        // iterate agents and then agent functions to find any rtc functions or function conditions
        for (auto it = am.cbegin(); it != am.cend(); ++it) {
            auto a_it = agent_map.find(it->first);
            const auto& mf = it->second->functions;
            for (auto it_f = mf.cbegin(); it_f != mf.cend(); ++it_f) {
                // check rtc source to see if this is a RTC function
                if (!it_f->second->rtc_source.empty()) {
                    // create CUDA agent RTC function by calling addInstantitateRTCFunction on CUDAAgent with AgentFunctionData
                    a_it->second->addInstantitateRTCFunction(*it_f->second, singletons->environment, macro_env);
                } else {
                    // Init curve for non-rtc functions
                    a_it->second->addInstantitateFunction(*it_f->second, singletons->environment, macro_env);
                }
                // check rtc source to see if the function condition is an rtc condition
                if (!it_f->second->rtc_condition_source.empty()) {
                    // create CUDA agent RTC function condition by calling addInstantitateRTCFunction on CUDAAgent with AgentFunctionData
                    a_it->second->addInstantitateRTCFunction(*it_f->second, singletons->environment, macro_env, true);
                } else if (it_f->second->condition) {
                    // Init curve for non-rtc function conditionss
                    a_it->second->addInstantitateFunction(*it_f->second, singletons->environment, macro_env, true);
                }
            }
        }

        rtcInitialised = true;

        // Record, store and output the elapsed time of the step.
        rtcTimer->stop();
        this->elapsedSecondsRTCInitialisation = rtcTimer->getElapsedSeconds();
        if (getSimulationConfig().timing) {
            fprintf(stdout, "RTC Initialisation Processing time: %.6f s\n", this->elapsedSecondsRTCInitialisation);
        }
    }
}

CUDASimulation::Config &CUDASimulation::CUDAConfig() {
    return config;
}
const CUDASimulation::Config &CUDASimulation::getCUDAConfig() const {
    return config;
}
#ifdef FLAMEGPU_VISUALISATION
visualiser::ModelVis CUDASimulation::getVisualisation() {
    if (!visualisation)
        visualisation = std::make_shared<visualiser::ModelVisData>(*this);
    return visualiser::ModelVis(visualisation);
}
#endif

unsigned int CUDASimulation::getStepCounter() {
    return step_count;
}
void CUDASimulation::resetStepCounter() {
    step_count = 0;
}

void CUDASimulation::initOffsetsAndMap() {
    const auto &md = getModelDescription();
    // Build offsets
    agentOffsets.clear();
    for (const auto &agent : md.agents) {
        agentOffsets.emplace(agent.first, VarOffsetStruct(agent.second->variables));
    }
    // Build data
    agentData.clear();
    for (const auto &agent : md.agents) {
        AgentDataBufferStateMap agent_states;
        for (const auto&state : agent.second->states)
            agent_states.emplace(state, AgentDataBuffer());
        agentData.emplace(agent.first, std::move(agent_states));
    }
}

void CUDASimulation::processHostAgentCreation(const unsigned int streamId) {
    size_t t_bufflen = 0;
    char *t_buff = nullptr;
    char *dt_buff = nullptr;
    // For each agent type
    for (auto &agent : agentData) {
        // We need size of agent
        const VarOffsetStruct &offsets = agentOffsets.at(agent.first);
        // For each state within the agent
        for (auto &state : agent.second) {
            // If the buffer has data
            if (state.second.size()) {
                size_t size_req = offsets.totalSize * state.second.size();
                {  // Ensure we have enough temp memory
                    if (size_req > t_bufflen) {
                        if (t_buff) {
                            free(t_buff);
                            gpuErrchk(flamegpu::detail::cuda::hipFree(dt_buff));
                        }
                        t_buff = reinterpret_cast<char*>(malloc(size_req));
                        gpuErrchk(hipMalloc(&dt_buff, size_req));
                        t_bufflen = size_req;
                    }
                }
                // Copy buffer memory into a single block
                for (unsigned int i = 0; i < state.second.size(); ++i) {
                    memcpy(t_buff + (i*offsets.totalSize), state.second[i].data, offsets.totalSize);
                }
                // Copy t_buff to device
                gpuErrchk(hipMemcpyAsync(dt_buff, t_buff, size_req, hipMemcpyHostToDevice, this->getStream(streamId)));
                // Scatter to device
                auto &cudaagent = agent_map.at(agent.first);
                cudaagent->scatterHostCreation(state.first, static_cast<unsigned int>(state.second.size()), dt_buff, offsets, this->singletons->scatter, streamId, this->getStream(streamId));
                // Clear buffer
                state.second.clear();
            }
        }
    }
    // Release temp memory
    if (t_buff) {
        free(t_buff);
        gpuErrchk(flamegpu::detail::cuda::hipFree(dt_buff));
    }
}

void CUDASimulation::incrementStepCounter() {
    this->step_count++;
    this->singletons->environment->setProperty<unsigned int>("_stepCount", this->step_count);
}

double CUDASimulation::getElapsedTimeSimulation() const {
    // Get the value
    return this->elapsedSecondsSimulation;
}

double CUDASimulation::getElapsedTimeInitFunctions() const {
    // Get the value
    return this->elapsedSecondsInitFunctions;
}

double CUDASimulation::getElapsedTimeExitFunctions() const {
    // Get the value
    return this->elapsedSecondsExitFunctions;
}
double CUDASimulation::getElapsedTimeRTCInitialisation() const {
    // Get the value
    return this->elapsedSecondsRTCInitialisation;
}

std::vector<double> CUDASimulation::getElapsedTimeSteps() const {
    // returns a copy of the timing vector, to avoid mutabililty issues. This should not be called in a performacne intensive part of the application.
    std::vector<double> rtn = this->elapsedSecondsPerStep;
    return rtn;
}

double CUDASimulation::getElapsedTimeStep(unsigned int step) const {
    if (step > this->elapsedSecondsPerStep.size()) {
        THROW exception::OutOfBoundsException("getElapsedTimeStep out of bounds.\n");
    }
    return this->elapsedSecondsPerStep.at(step);
}

void CUDASimulation::initEnvironmentMgr() {
    if (!singletons || !singletons->environment) {
        THROW exception::UnknownInternalError("CUDASimulation::initEnvironmentMgr() called before singletons member initialised.");
    }

    // Set any properties loaded from file during arg parse stage
    for (const auto &prop : env_init) {
        const std::string np = prop.first;
        const auto it = singletons->environment->properties.find(np);
        if (it == singletons->environment->properties.end()) {
            THROW exception::InvalidEnvProperty("Environment init data contains unexpected environment property '%s', "
                "in CUDASimulation::initEnvironmentMgr()\n", prop.first.c_str());
        }
        if (prop.second.type != it->second.type) {
            THROW exception::InvalidEnvProperty("Environment init data contains environment property '%s' with type mismatch '%s' != '%s', "
                "this should have been caught during file parsing, "
                "in CUDASimulation::initEnvironmentMgr()\n", prop.first.c_str(), prop.second.type.name(), it->second.type.name());
        } else if (prop.second.elements != it->second.elements) {
            THROW exception::InvalidEnvProperty("Environment init data contains environment property '%s' with type length mismatch '%u' != '%u', "
                "this should have been caught during file parsing, "
                "in CUDASimulation::initEnvironmentMgr()\n", prop.first.c_str(), prop.second.elements, it->second.elements);
        } else {
            singletons->environment->setPropertyDirect(np, static_cast<char*>(prop.second.ptr));
        }
    }
    // Clear init
    env_init.clear();
}
void CUDASimulation::resetLog() {
    // Track previous device id, so we can avoid costly request for device properties if not required
    static int previous_device_id = -1;
    run_log->step.clear();
    run_log->exit = ExitLogFrame();
    run_log->random_seed = SimulationConfig().random_seed;
    run_log->step_log_frequency = step_log_config ? step_log_config->frequency : 0;
    if (run_log->performance_specs.device_name.empty() || CUDAConfig().device_id != previous_device_id) {
        hipDeviceProp_t d_props = {};
        gpuErrchk(hipGetDeviceProperties(&d_props, CUDAConfig().device_id));
        run_log->performance_specs.device_name = d_props.name;
        previous_device_id = CUDAConfig().device_id;
    }
    gpuErrchk(hipDeviceGetAttribute(&run_log->performance_specs.device_cc_major, hipDeviceAttributeComputeCapabilityMajor, CUDAConfig().device_id));
    gpuErrchk(hipDeviceGetAttribute(&run_log->performance_specs.device_cc_minor,  hipDeviceAttributeComputeCapabilityMinor, CUDAConfig().device_id));
    gpuErrchk(hipRuntimeGetVersion(&run_log->performance_specs.cuda_version));
#if !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
    run_log->performance_specs.seatbelts = true;
#else
    run_log->performance_specs.seatbelts = false;
#endif
    run_log->performance_specs.flamegpu_version = VERSION_FULL;
}
void CUDASimulation::processStepLog(const double step_time_seconds) {
    if (!step_log_config)
        return;
    if (step_count % step_log_config->frequency != 0)
        return;
    // Iterate members of step log to build the step log frame
    std::map<std::string, detail::Any> environment_log;
    for (const auto &prop_name : step_log_config->environment) {
        // Fetch the named environment prop
        environment_log.emplace(prop_name, singletons->environment->getPropertyAny(prop_name));
    }
    std::map<util::StringPair, std::pair<std::map<LoggingConfig::NameReductionFn, detail::Any>, unsigned int>> agents_log;
    for (const auto &name_state : step_log_config->agents) {
        // Create the named sub map
        const std::string &agent_name = name_state.first.first;
        const std::string &agent_state = name_state.first.second;
        HostAgentAPI host_agent = host_api->agent(agent_name, agent_state);
        auto &agent_state_log = agents_log.emplace(name_state.first, std::make_pair(std::map<LoggingConfig::NameReductionFn, detail::Any>(), UINT_MAX)).first->second;
        // Log individual variable reductions
        for (const auto &name_reduction : *name_state.second.first) {
            // Perform the corresponding reduction
            auto result = name_reduction.function(host_agent, name_reduction.name);
            // Store the result
            agent_state_log.first.emplace(name_reduction, std::move(result));
        }
        // Log count of agents in state
        if (name_state.second.second) {
            agent_state_log.second = host_api->agent(agent_name, agent_state).count();
        }
    }

    // Append to step log
    run_log->step.push_back(StepLogFrame(std::move(environment_log), std::move(agents_log), step_count));
    run_log->step.back().step_time = step_time_seconds;
}

void CUDASimulation::processExitLog() {
    if (!exit_log_config)
        return;
    // Iterate members of step log to build the step log frame
    std::map<std::string, detail::Any> environment_log;
    for (const auto &prop_name : exit_log_config->environment) {
        // Fetch the named environment prop
        environment_log.emplace(prop_name, singletons->environment->getPropertyAny(prop_name));
    }
    std::map<util::StringPair, std::pair<std::map<LoggingConfig::NameReductionFn, detail::Any>, unsigned int>> agents_log;
    for (const auto &name_state : exit_log_config->agents) {
        // Create the named sub map
        const std::string &agent_name = name_state.first.first;
        const std::string &agent_state = name_state.first.second;
        HostAgentAPI host_agent = host_api->agent(agent_name, agent_state);
        auto &agent_state_log = agents_log.emplace(name_state.first, std::make_pair(std::map<LoggingConfig::NameReductionFn, detail::Any>(), UINT_MAX)).first->second;
        // Log individual variable reductions
        for (const auto &name_reduction : *name_state.second.first) {
            // Perform the corresponding reduction
            auto result = name_reduction.function(host_agent, name_reduction.name);
            // Store the result
            agent_state_log.first.emplace(name_reduction, std::move(result));
        }
        // Log count of agents in state
        if (name_state.second.second) {
            agent_state_log.second = host_api->agent(agent_name, agent_state).count();
        }
    }

    // Set Log
    run_log->exit = ExitLogFrame(std::move(environment_log), std::move(agents_log), step_count);
    // Add the timing info
    run_log->exit.rtc_time = getElapsedTimeRTCInitialisation();
    run_log->exit.init_time = getElapsedTimeInitFunctions();
    run_log->exit.exit_time = getElapsedTimeExitFunctions();
    run_log->exit.total_time = getElapsedTimeSimulation();
}
const RunLog &CUDASimulation::getRunLog() const {
    return *run_log;
}

void CUDASimulation::createStreams(const unsigned int nStreams) {
    // There should always be atleast 1 stream, as some tests require the 0th stream even when there is no concurrent work to be done.
    unsigned int totalStreams = std::max(nStreams, 1u);
    while (streams.size() < totalStreams) {
        hipStream_t stream = 0;
        gpuErrchk(hipStreamCreate(&stream));
        streams.push_back(stream);
    }
}

hipStream_t CUDASimulation::getStream(const unsigned int n) {
    // Return the appropriate stream, unless concurrency is disabled in which case always stream 0.
    if (this->streams.size() <= n) {
        unsigned int nStreams = getMaximumLayerWidth();
        this->createStreams(nStreams);
    }

    if (getCUDAConfig().inLayerConcurrency && n < streams.size()) {
        return streams.at(n);
    } else {
        return streams.at(0);
    }
}

void CUDASimulation::destroyStreams() {
    // early exit if there are no streams to reset
    if (streams.size() == 0) {
        return;
    }
    /*
    This method is called by ~CUDASimulation(), which may be after a device reset and / or CUDA shutdown (if static, or if GC'd by python implementation)
    hipStreamDestroy and hipStreamQuery under linux with CUDA 11.8 (and potentialy others) would occasionally segfault after a reset, so it's error code could not be relied on to check if the cudaStream was valid for the current primary context or not.
    Instead, we can use the cudaDriverAPI to check the primary context is correct / valid for the device, and if it is attempt to destory the stream. If it is not, we can assume the device has been reset or CUDA has been shutdown, so the stream has already been destroyed.
    */
    bool ctxIsActive = flamegpu::detail::cuda::cuDevicePrimaryContextIsActive(deviceInitialised);
    if (ctxIsActive) {
        // Destroy streams.
        for (auto stream : streams) {
            gpuErrchk(hipStreamDestroy(stream));
        }
    }
    streams.clear();
}

void CUDASimulation::synchronizeAllStreams() {
    // Sync streams.
    for (auto stream : streams) {
        gpuErrchk(hipStreamSynchronize(stream));
    }
}

std::shared_ptr<detail::EnvironmentManager> CUDASimulation::getEnvironment() const {
    if (singletons)
        return singletons->environment;
    return nullptr;
}
void CUDASimulation::assignAgentIDs() {
    flamegpu::util::nvtx::Range range{"CUDASimulation::assignAgentIDs"};
    if (!agent_ids_have_init) {
        // Ensure singletons have been initialised
        initialiseSingletons();

        for (auto &a : agent_map) {
            a.second->assignIDs(*host_api, singletons->scatter, getStream(0), 0);  // This could be made concurrent, 1 stream per agent
        }
        agent_ids_have_init = true;
    }
}

}  // namespace flamegpu
