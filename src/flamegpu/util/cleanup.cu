#include "flamegpu/util/cleanup.h"

#include <hip/hip_runtime.h>
#include "flamegpu/simulation/detail/CUDAErrorChecking.cuh"
#include "flamegpu/detail/JitifyCache.h"

namespace flamegpu {
namespace util {

void cleanup() {
    int originalDevice = 0;
    gpuErrchk(hipGetDevice(&originalDevice));
    // Reset all cuda devices for memcheck / profiling purposes.
    int devices = 0;
    gpuErrchk(hipGetDeviceCount(&devices));
    // @todo - this would be better to be only devices touched by flamegpu since the last call to cleanup.
    for (int device = 0; device < devices; ++device) {
        gpuErrchk(hipSetDevice(device));
        gpuErrchk(hipDeviceReset());
    }
    // resume the old device, but do not create a new context via reset or memsets
    gpuErrchk(hipSetDevice(originalDevice));
}

void clearRTCDiskCache() {
    detail::JitifyCache::clearDiskCache();
}

}  // namespace util
}  // namespace flamegpu
