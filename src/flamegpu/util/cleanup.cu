#include "flamegpu/util/cleanup.h"

#include <hip/hip_runtime.h>
#include "flamegpu/gpu/detail/CUDAErrorChecking.cuh"
#include "flamegpu/util/detail/JitifyCache.h"

namespace flamegpu {
namespace util {

void cleanup() {
    // Reset all cuda devices for memcheck / profiling purposes.
    int devices = 0;
    gpuErrchk(hipGetDeviceCount(&devices));
    for (int device = 0; device < devices; ++device) {
        gpuErrchk(hipSetDevice(device));
        gpuErrchk(hipDeviceReset());
    }
}

void clearRTCDiskCache() {
    detail::JitifyCache::clearDiskCache();
}

}  // namespace util
}  // namespace flamegpu
