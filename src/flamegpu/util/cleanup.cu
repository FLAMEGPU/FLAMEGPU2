#include "flamegpu/util/cleanup.h"

#ifdef FLAMEGPU_ENABLE_MPI
#include <mpi.h>
#endif

#include <hip/hip_runtime.h>
#include "flamegpu/simulation/detail/CUDAErrorChecking.cuh"
#include "flamegpu/detail/JitifyCache.h"

namespace flamegpu {
namespace util {

void cleanup() {
#ifdef FLAMEGPU_ENABLE_MPI
    int init_flag = 0;
    int fin_flag = 0;
    // MPI can only be init and finalized once
    MPI_Initialized(&init_flag);
    MPI_Finalized(&fin_flag);
    if (init_flag && !fin_flag) {
        MPI_Finalize();
    }
#endif
    int originalDevice = 0;
    gpuErrchk(hipGetDevice(&originalDevice));
    // Reset all cuda devices for memcheck / profiling purposes.
    int devices = 0;
    gpuErrchk(hipGetDeviceCount(&devices));
    // @todo - this would be better to be only devices touched by flamegpu since the last call to cleanup.
    for (int device = 0; device < devices; ++device) {
        gpuErrchk(hipSetDevice(device));
        gpuErrchk(hipDeviceReset());
    }
    // resume the old device, but do not create a new context via reset or memsets
    gpuErrchk(hipSetDevice(originalDevice));
}

void clearRTCDiskCache() {
    detail::JitifyCache::clearDiskCache();
}

}  // namespace util
}  // namespace flamegpu
