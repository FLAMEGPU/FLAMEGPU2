#include "hip/hip_runtime.h"
#include "flamegpu/util/detail/JitifyCache.h"

#include <cassert>
#include <regex>
#include <array>

#include "flamegpu/version.h"
#include "flamegpu/exception/FLAMEGPUException.h"
#include "flamegpu/util/detail/compute_capability.cuh"
#include "flamegpu/util/nvtx.h"

// If MSVC earlier than VS 2019
#if defined(_MSC_VER) && _MSC_VER < 1920
#include <filesystem>
using std::tr2::sys::temp_directory_path;
using std::tr2::sys::exists;
using std::tr2::sys::current_path;
using std::tr2::sys::path;
using std::tr2::sys::directory_iterator;
#else
// VS2019 requires this macro, as building pre c++17 cant use std::filesystem
#define _SILENCE_EXPERIMENTAL_FILESYSTEM_DEPRECATION_WARNING
#include <experimental/filesystem>
using std::experimental::filesystem::v1::temp_directory_path;
using std::experimental::filesystem::v1::exists;
using std::experimental::filesystem::v1::current_path;
using std::experimental::filesystem::v1::path;
using std::experimental::filesystem::v1::directory_iterator;
#endif

using jitify::detail::hash_combine;
using jitify::detail::hash_larson64;

namespace flamegpu {
namespace util {
namespace detail {

namespace {
/**
 * Returns the tmp dir for storing cache files
 * Defined here to avoid filesystem includes being in header
 */
path getTMP() {
    static path result;
    if (result.empty()) {
        path tmp =  std::getenv("FLAMEGPU_TMP_DIR") ? std::getenv("FLAMEGPU_TMP_DIR") : temp_directory_path();
        // Create the $tmp/flamegpu/jitifycache(/debug) folder hierarchy
        if (!::exists(tmp) && !create_directory(tmp)) {
            THROW exception::InvalidFilePath("Directory '%s' does not exist and cannot be created by JitifyCache.", tmp.generic_string().c_str());
        }
        if (!std::getenv("FLAMEGPU_TMP_DIR")) {
            tmp /= "flamegpu";
            if (!::exists(tmp)) {
                create_directory(tmp);
            }
        }
        tmp /= "jitifycache";
        if (!::exists(tmp)) {
            create_directory(tmp);
        }
#ifdef _DEBUG
        tmp /= "debug";
        if (!::exists(tmp)) {
            create_directory(tmp);
        }
#endif
        result = tmp;
    }
    return result;
}
/**
 * Returns the user-defined include directories
 */
std::vector<path> getIncludeDirs() {
    static std::vector<path> rtn;
    if (rtn.empty()) {
        if (std::getenv("FLAMEGPU_RTC_INCLUDE_DIRS")) {
            const std::string s = std::getenv("FLAMEGPU_RTC_INCLUDE_DIRS");
            // Split the string by ; (windows), : (linux)
#if defined(_MSC_VER)
            std::string delimiter = ";";
#else
            std::string delimiter = ":";
#endif
            size_t start = 0, end = s.find(delimiter);
            std::string token;
            do {
                path p = s.substr(start, end - start);
                if (!p.empty()) {
                    rtn.push_back(p);
                }
                start = end + delimiter.length();
            } while ((end = s.find(delimiter, start))!= std::string::npos);
        } else {
            rtn.push_back(current_path());
        }
    }
    return rtn;
}
std::string loadFile(const path &filepath) {
    std::ifstream ifs;
    ifs.open(filepath, std::ifstream::binary);
    if (!ifs)
    return "";
    // get length of file
    ifs.seekg(0, ifs.end);
    const std::streamoff length = ifs.tellg();
    ifs.seekg(0, ifs.beg);
    std::string rtn;
    rtn.resize(length);
    char *buffer = &rtn[0];
    ifs.read(buffer, length);
    ifs.close();
    return rtn;
}

/**
 * Find the cuda include directory.
 * Throws exceptions if it can not be found.
 * @return the path to the CUDA include directory.
 */
std::string getCUDAIncludeDir() {
    // Define an array of environment variables to check in order
    std::array<const std::string, 2> ENV_VARS { "CUDA_PATH", "CUDA_HOME" };
    std::string cuda_include_dir_str = "";
    for (const auto& env_var : ENV_VARS) {
        std::string env_value = std::getenv(env_var.c_str()) ? std::getenv(env_var.c_str()) : "";
        if (!env_value.empty()) {
            path check_path = path(env_value) / "include/";
            // Use try catch to suppress file permission exceptions etc
            try {
                if (exists(check_path)) {
                    cuda_include_dir_str = check_path.string();
                    break;
                }
            } catch (...) { }
            // Throw if the value is not empty, but it does not exist. Outside the try catch excplicityly.
            THROW flamegpu::exception::InvalidFilePath("Error environment variable %s (%s) does not contain a valid CUDA include directory", env_var.c_str(), env_value.c_str());
        }
    }
    // If none of the search enviornmental variables were useful, throw an exception.
    if (cuda_include_dir_str.empty()) {
        THROW exception::InvalidFilePath("Error could not find CUDA include directory. Please specify using the CUDA_PATH environment variable");
    }
    return cuda_include_dir_str;
}

/**
 * Get the FLAME GPU include directory via the environment variables. 
 * @param env_var_used modified to return the name of the environment variable which was used, if any.
 * @return the FLAME GPU 2+ include directory.
 */
std::string getFLAMEGPUIncludeDir(std::string &env_var_used) {
    // Define an array of environment variables to check
    std::array<const std::string, 2> ENV_VARS { "FLAMEGPU_INC_DIR", "FLAMEGPU2_INC_DIR" };
    std::string include_dir_str = "";
    // Iterate the array of environment variables to check for the version header.
    for (const auto& env_var : ENV_VARS) {
        // If the environment variable exists
        std::string env_value = std::getenv(env_var.c_str()) ? std::getenv(env_var.c_str()) : "";
        // If it's a value, check if the path exists, and if any expected files are found.
        if (!env_value.empty()) {
            path check_file = path(env_value) / "flamegpu/flamegpu.h";
            // Use try catch to suppress file permission exceptions etc
            try {
                if (exists(check_file)) {
                    include_dir_str = env_value;
                    env_var_used = env_var;
                    break;
                }
            } catch (...) { }
            // Throw if the value is not empty, but it does not exist. Outside the try catch excplicityly.
            THROW flamegpu::exception::InvalidFilePath("Error environment variable %s (%s) does not contain flamegpu/flamegpu.h. Please correct this environment variable.", env_var.c_str(), env_value.c_str());
        }
    }

    // If no appropriate environmental variables were found, check upwards for N levels (assuming the default filestructure is in use)
    if (include_dir_str.empty()) {
        // Start with the current working directory
        path test_dir(".");
        // Try multiple levels of directory, to see if we can find include/flamegpu/flamegpu.h
        const unsigned int LEVELS = 5;
        for (unsigned int level = 0; level < LEVELS; level++) {
            // If break out the loop if the test_dir directory does not exist.
            if (!exists(test_dir)) {
                break;
            }
            // Check file assuming flamegpu is the root cmake project
            path check_file = test_dir;
            check_file /= "include/flamegpu/flamegpu.h";
            // Use try catch to suppress file permission exceptions etc
            try {
                if (exists(check_file)) {
                    test_dir /= "include";
                    include_dir_str = test_dir.string();
                    break;
                }
            } catch (...) { }
            // Check file assuming a standalone example is the root cmake project
            // We want to see if we can find the build directory
            for (auto& p : directory_iterator(test_dir)) {
                if (is_directory(p)) {
                    check_file = p.path();
                    check_file /= "_deps/flamegpu2-src/include/flamegpu/version.h";
                    // Use try catch to suppress file permission exceptions etc
                    try {
                        if (exists(check_file)) {
                            test_dir = p.path();
                            test_dir /= "_deps/flamegpu2-src/include";
                            include_dir_str = test_dir.string();
                            goto break_flamegpu_inc_dir_loop;  // Break out of nested loop
                        }
                    } catch (...) { }
                }
            }
            // Go up a level for next iteration
            test_dir /= "..";
        }
break_flamegpu_inc_dir_loop:
        // If still not found, throw.
        if (include_dir_str.empty()) {
            // @todo - more appropriate exception?
            THROW flamegpu::exception::InvalidAgentFunc("Error compiling runtime agent function: Unable to automatically determine include directory and FLAMEGPU_INC_DIR environment variable not set");
        }
    }
    return include_dir_str;
}

/**
 * Confirm that include directory version header matches the version of the static library.
 * This only compares up to the pre-release version number. Build metadata is only used for the RTC cache.
 * @param flamegpuIncludeDir path to the flamegpu include directory to check.
 * @return boolean indicator of success.
 */
bool confirmFLAMEGPUHeaderVersion(const std::string flamegpuIncludeDir, const std::string envVariable) {
    static bool header_version_confirmed = false;

    if (!header_version_confirmed) {
        std::string fileHash;
        std::string fileVersionMacro;
        std::string fileVersionPrerelease;
        // Open version.h
        path version_file = path(flamegpuIncludeDir) /= "flamegpu/version.h";
        std::ifstream vFile(version_file);
        if (vFile.is_open()) {
            // Use a regular expression to match the FLAMEGPU_VERSION number macro against lines in the file.
            std::regex macroPattern("^#define FLAMEGPU_VERSION ([0-9]+)$");
            std::regex prereleasePattern("^static constexpr char VERSION_PRERELEASE\\[\\] = \"(.*)\";$");
            std::smatch match;
            std::string line;
            bool extractedMacro = false;
            bool extractedPrerelease = false;
            while (std::getline(vFile, line)) {
                if (std::regex_search(line, match, macroPattern)) {
                    fileVersionMacro = match[1];
                    extractedMacro = true;
                } else if (std::regex_search(line, match, prereleasePattern)) {
                    fileVersionPrerelease = match[1];
                    extractedPrerelease = true;
                }
                if (extractedMacro && extractedPrerelease) {
                    break;
                }
            }
            vFile.close();
            if (!extractedMacro || !extractedPrerelease) {
                THROW exception::VersionMismatch("Could not extract RTC header version information.\n");
            }
        }
        // Confirm that the version matches, else throw an exception.
        if (fileVersionMacro == std::to_string(flamegpu::VERSION) && fileVersionPrerelease == std::string(flamegpu::VERSION_PRERELEASE)) {
            header_version_confirmed = true;
        } else {
            THROW exception::VersionMismatch("RTC header version (%s, %s) does not match version flamegpu library was built with (%s, %s). Set the environment variable %s to the correct include directory.\n",
                fileVersionMacro.c_str(), fileVersionPrerelease.c_str(),
                std::to_string(flamegpu::VERSION).c_str(), flamegpu::VERSION_PRERELEASE,
                envVariable.c_str());
        }
    }
    return header_version_confirmed;
}

}  // namespace

std::mutex JitifyCache::instance_mutex;
std::unique_ptr<KernelInstantiation> JitifyCache::compileKernel(const std::string &func_name, const std::vector<std::string> &template_args, const std::string &kernel_src, const std::string &dynamic_header) {
    NVTX_RANGE("JitifyCache::compileKernel");
    // find and validate the cuda include directory via CUDA_PATH or CUDA_HOME.
    static const std::string cuda_include_dir = getCUDAIncludeDir();
    // find and validate the the flamegpu include directory
    static std::string flamegpu_include_dir_envvar;
    static const std::string flamegpu_include_dir = getFLAMEGPUIncludeDir(flamegpu_include_dir_envvar);
    // verify that the include directory contains the correct headers.
    confirmFLAMEGPUHeaderVersion(flamegpu_include_dir, flamegpu_include_dir_envvar);

     // vector of compiler options for jitify
    std::vector<std::string> options;
    std::vector<std::string> headers;

    // fpgu include directory
    options.push_back(std::string("-I" + std::string(flamegpu_include_dir)));

    // cuda include directory (via CUDA_PATH)
    options.push_back(std::string("-I" + cuda_include_dir));

    // Add user specified include paths
    for (const auto &p : getIncludeDirs())
        options.push_back(std::string("-I" + p.generic_string()));

#ifdef USE_GLM
    // GLM headers increase build time ~5x, so only enable glm if user is using it
    if (kernel_src.find("glm") != std::string::npos) {
        options.push_back(std::string("-I") + GLM_PATH);
        options.push_back(std::string("-DUSE_GLM"));
    }
#endif

    // Set the compilation architecture target if it was successfully detected.
    int currentDeviceIdx = 0;
    hipError_t status = hipGetDevice(&currentDeviceIdx);
    if (status == hipSuccess) {
        int arch = compute_capability::getComputeCapability(currentDeviceIdx);
        options.push_back(std::string("--gpu-architecture=compute_" + std::to_string(arch)));
    }

    // If CUDA is compiled with -G (--device-debug) forward it to the compiler, otherwise forward lineinfo for profiling.
#if defined(__CUDACC_DEBUG__)
    options.push_back("--device-debug");
#else
    options.push_back("--generate-line-info");
#endif

    // If DEBUG is defined, forward it
#if defined(DEBUG)
    options.push_back("-DDEBUG");
#endif

    // If NDEBUG is defined, forward it, this should disable asserts in device code.
#if defined(NDEBUG)
    options.push_back("-DNDEBUG");
#endif

// pass the c++ language dialect. It may be better to explicitly pass this from CMake.
#if defined(__cplusplus) && __cplusplus > 201700L && defined(__CUDACC_VER_MAJOR__) && __CUDACC_VER_MAJOR__ >= 11
    options.push_back("--std=c++17");
#elif defined(__cplusplus) && __cplusplus > 201400L
    options.push_back("--std=c++14");
#endif

    // If SEATBELTS is defined and false, forward it as off, otherwise forward it as on.
#if !defined(SEATBELTS) || SEATBELTS
    options.push_back("--define-macro=SEATBELTS=1");
#else
    options.push_back("--define-macro=SEATBELTS=0");
#endif

    // hip/hip_runtime.h
    std::string include_cuda_h;
    include_cuda_h = "--pre-include=" + cuda_include_dir + "/hip/hip_runtime.h";
    options.push_back(include_cuda_h);

    // get the dynamically generated header from curve rtc
    headers.push_back(dynamic_header);

    // cassert header (to remove remaining warnings) TODO: Ask Jitify to implement safe version of this
    std::string cassert_h = "cassert\n";
    headers.push_back(cassert_h);

    // jitify to create program (with compilation settings)
    try {
        auto program = jitify::experimental::Program(kernel_src, headers, options);
        assert(template_args.size() == 1 || template_args.size() == 3);  // Add this assertion incase template args change
        auto kernel = program.kernel(template_args.size() > 1 ? "flamegpu::agent_function_wrapper" : "flamegpu::agent_function_condition_wrapper");
        return std::make_unique<KernelInstantiation>(kernel, template_args);
    } catch (std::runtime_error const&) {
        // jitify does not have a method for getting compile logs so rely on JITIFY_PRINT_LOG defined in cmake
        THROW exception::InvalidAgentFunc("Error compiling runtime agent function (or function condition) ('%s'): function had compilation errors (see std::cout), "
            "in JitifyCache::buildProgram().",
            func_name.c_str());
    }
}

std::unique_ptr<KernelInstantiation> JitifyCache::loadKernel(const std::string &func_name, const std::vector<std::string> &template_args, const std::string &kernel_src, const std::string &dynamic_header) {
    NVTX_RANGE("JitifyCache::loadKernel");
    std::lock_guard<std::mutex> lock(cache_mutex);
    // Detect current compute capability=
    int currentDeviceIdx = 0;
    hipError_t status = hipGetDevice(&currentDeviceIdx);
    const std::string arch = std::to_string((status == hipSuccess) ? compute_capability::getComputeCapability(currentDeviceIdx) : 0);
    status = hipRuntimeGetVersion(&currentDeviceIdx);
    const std::string cuda_version = std::to_string((status == hipSuccess) ? currentDeviceIdx : 0);
    const std::string seatbelts = std::to_string(SEATBELTS);
    // Cat kernel, dynamic header, header version
    const std::string long_reference = kernel_src + dynamic_header;  // Don't need to include rest, they are explicit in short reference/filename
    // Generate short reference string
    // Would prefer to use a proper hash, e.g. md5(reference_string), but that requires extra dependencies
    const std::string short_reference =
        cuda_version + "_" +
        arch + "_" +
        seatbelts + "_" +
        std::string(flamegpu::VERSION_FULL) + "_" +
        // Use jitify hash methods for consistent hashing between OSs
        std::to_string(hash_combine(hash_larson64(kernel_src.c_str()), hash_larson64(dynamic_header.c_str())));
    // Does a copy with the right reference exist in memory?
    if (use_memory_cache) {
        const auto it = cache.find(short_reference);
        if (it != cache.end()) {
            // Check long reference
            if (it->second.long_reference == long_reference) {
                return std::make_unique<KernelInstantiation>(KernelInstantiation::deserialize(it->second.serialised_kernelinst));
            }
        }
    }
    // Does a copy with the right reference exist on disk?
    const path cache_file = getTMP() / short_reference;
    const path reference_file = cache_file.parent_path() / path(cache_file.filename().string() + ".ref");
    if (use_disk_cache && exists(cache_file)) {
        // Load the long reference for the cache file
        const std::string file_long_reference = loadFile(reference_file);
        if (file_long_reference == long_reference) {
            // Load the cache file
            const std::string serialised_kernelinst = loadFile(cache_file);
            if (!serialised_kernelinst.empty()) {
                // Add it to cache for later loads
                cache.emplace(short_reference, CachedProgram{long_reference, serialised_kernelinst});
                // Deserialize and return program
                return std::make_unique<KernelInstantiation>(KernelInstantiation::deserialize(serialised_kernelinst));
            }
        }
    }
    // Kernel has not yet been cached
    {
        // Build kernel
        auto kernelinst = compileKernel(func_name, template_args, kernel_src, dynamic_header);
        // Add it to cache for later loads
        const std::string serialised_kernelinst = use_memory_cache || use_disk_cache ? kernelinst->serialize() : "";
        if (use_memory_cache) {
            cache.emplace(short_reference, CachedProgram{long_reference, serialised_kernelinst});
        }
        // Save it to disk
        if (use_disk_cache) {
            std::ofstream ofs(cache_file, std::ofstream::out | std::ofstream::binary | std::ofstream::trunc);
            if (ofs) {
                ofs << serialised_kernelinst;
                ofs.close();
            }
            ofs = std::ofstream(reference_file, std::ofstream::out | std::ofstream::binary | std::ofstream::trunc);
            if (ofs) {
                ofs << long_reference;
                ofs.close();
            }
        }
        return kernelinst;
    }
}
void JitifyCache::useMemoryCache(bool yesno) {
    std::lock_guard<std::mutex> lock(cache_mutex);
    use_memory_cache = yesno;
}
void JitifyCache::useDiskCache(bool yesno) {
    std::lock_guard<std::mutex> lock(cache_mutex);
    use_disk_cache = yesno;
}
bool JitifyCache::useMemoryCache() const {
    std::lock_guard<std::mutex> lock(cache_mutex);
    return use_memory_cache;
}
bool JitifyCache::useDiskCache() const {
    std::lock_guard<std::mutex> lock(cache_mutex);
    return use_disk_cache;
}
void JitifyCache::clearMemoryCache() {
    std::lock_guard<std::mutex> lock(cache_mutex);
    cache.clear();
}
void JitifyCache::clearDiskCache() {
    std::lock_guard<std::mutex> lock(cache_mutex);
    const path tmp_dir = getTMP();
    for (const auto & entry : directory_iterator(tmp_dir)) {
        if (is_regular_file(entry.path())) {
            remove(entry.path());
        }
    }
}
JitifyCache::JitifyCache()
    : use_memory_cache(true)
#ifndef DISABLE_RTC_DISK_CACHE
    , use_disk_cache(true) { }
#else
    , use_disk_cache(false) { }
#endif
JitifyCache& JitifyCache::getInstance() {
    auto lock = std::unique_lock<std::mutex>(instance_mutex);  // Mutex to protect from two threads triggering the static instantiation concurrently
    static JitifyCache instance;  // Instantiated on first use.
    return instance;
}

}  // namespace detail
}  // namespace util
}  // namespace flamegpu
