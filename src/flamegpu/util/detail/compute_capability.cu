#include <hip/hiprtc.h>

#include <cassert>

#include "flamegpu/util/detail/compute_capability.cuh"
#include "flamegpu/gpu/detail/CUDAErrorChecking.cuh"


namespace flamegpu {
namespace util {
namespace detail {

int compute_capability::getComputeCapability(int deviceIndex) {
    int major = 0;
    int minor = 0;

    // Throw an exception if the deviceIndex is negative.
    if (deviceIndex < 0) {
        THROW exception::InvalidCUDAdevice();
    }

    // Ensure deviceIndex is valid.
    int deviceCount = 0;
    gpuErrchk(hipGetDeviceCount(&deviceCount));
    if (deviceIndex >= deviceCount) {
        // Throw an excpetion if the device index is bad.
        THROW exception::InvalidCUDAdevice();
    }
    // Load device attributes
    gpuErrchk(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, deviceIndex));
    gpuErrchk(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, deviceIndex));
    // Compute the arch integer value.
    int arch = (10 * major) + minor;
    return arch;
}

int compute_capability::minimumCompiledComputeCapability() {
    #if defined(MIN_CUDA_ARCH)
        return MIN_CUDA_ARCH;
    #else
        // Return 0 as a default minimum?
        return 0;
    #endif
}

bool compute_capability::checkComputeCapability(int deviceIndex) {
    // If the compile time minimum architecture is defined, fetch the device's compute capability and check that the executable (probably) supports this device.
    if (getComputeCapability(deviceIndex) < minimumCompiledComputeCapability()) {
        return false;
    } else {
        return true;
    }
}

std::vector<int> compute_capability::getNVRTCSupportedComputeCapabilties() {
// NVRTC included with CUDA 11.2+ includes methods to query the supported architectures and CUDA from 11.2+
// Also changes the soname rules such that nvrtc.11.2.so is vald for all nvrtc >= 11.2, and libnvrtc.12.so for CUDA 12.x etc, so this is different at runtime not compile time for future versions, so use the methods
#if (__CUDACC_VER_MAJOR__ > 11) || ((__CUDACC_VER_MAJOR__ == 11) && __CUDACC_VER_MINOR__ >= 2)
    hiprtcResult nvrtcStatus = HIPRTC_SUCCESS;
    int nvrtcNumSupportedArchs = 0;
    // Query the number of architecture flags supported by this nvrtc, to allocate enough memory
    nvrtcStatus = nvrtcGetNumSupportedArchs(&nvrtcNumSupportedArchs);
    if (nvrtcStatus == HIPRTC_SUCCESS && nvrtcNumSupportedArchs > 0) {
        // prepare a large enough std::vector for the results
        std::vector<int> nvrtcSupportedArchs = std::vector<int>(nvrtcNumSupportedArchs);
        assert(nvrtcSupportedArchs.size() >= nvrtcNumSupportedArchs);
        nvrtcStatus = nvrtcGetSupportedArchs(nvrtcSupportedArchs.data());
        if (nvrtcStatus == HIPRTC_SUCCESS) {
            // Return the populated std::vector, this should be RVO'd
            return nvrtcSupportedArchs;
        }
    }
    // If any of the above functions failed, we have no idea what arch's are supported, so assume none are?
    return {};
// Older CUDA's do not support this, but this is simple to hard-code for CUDA 11.0/11.1  (and our deprected CUDA 10.x).
// CUDA 11.1 suports 35 to 86
#elif (__CUDACC_VER_MAJOR__ == 11) && __CUDACC_VER_MINOR__ == 1
    return {35, 37, 50, 52, 53, 60, 61, 62, 70, 72, 75, 80, 86};
// CUDA 11.0 supports 35 to 80
#elif (__CUDACC_VER_MAJOR__ == 11) && __CUDACC_VER_MINOR__ == 0
    return {35, 37, 50, 52, 53, 60, 61, 62, 70, 72, 75, 80};
// CUDA 10.x supports 30 to 75
#elif (__CUDACC_VER_MAJOR__ >= 10)
    return {30, 32, 35, 37, 50, 52, 53, 60, 61, 62, 70, 72, 75};
// This should be all cases for FLAME GPU 2, but leave the fallback branch just in case
#else
    return {};
#endif
}

int compute_capability::selectAppropraiteComputeCapability(const int target, const std::vector<int>& architectures) {
    int maxArch = 0;
    for (const int &arch : architectures) {
        if (arch <= target && arch > maxArch) {
            maxArch = arch;
            // The vector is in ascending order, so we can potentially early exit
            if (arch == target) {
                return target;
            }
        }
    }
    return maxArch;
}

}  // namespace detail
}  // namespace util
}  // namespace flamegpu
