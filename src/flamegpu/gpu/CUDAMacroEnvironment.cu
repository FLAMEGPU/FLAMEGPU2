#include "flamegpu/gpu/CUDAMacroEnvironment.h"

#include "flamegpu/model/EnvironmentDescription.h"
#include "flamegpu/gpu/CUDASimulation.h"
#include "flamegpu/model/AgentFunctionData.cuh"
#include "flamegpu/model/SubEnvironmentData.h"
#include "flamegpu/runtime/detail/curve/curve_rtc.cuh"
#include "flamegpu/util/detail/cuda.cuh"

namespace flamegpu {

CUDAMacroEnvironment::CUDAMacroEnvironment(const EnvironmentDescription& description, const CUDASimulation& _cudaSimulation)
    : cudaSimulation(_cudaSimulation) {
    for (const auto &p : description.getMacroPropertiesMap()) {
        properties.emplace(p.first, MacroEnvProp(p.second.type, p.second.type_size, p.second.elements));
    }
}

void CUDAMacroEnvironment::init(hipStream_t stream) {
    for (auto &prop : properties) {
        if (!prop.second.d_ptr) {
            size_t buffer_size = prop.second.type_size
                                     * prop.second.elements[0]
                                     * prop.second.elements[1]
                                     * prop.second.elements[2]
                                     * prop.second.elements[3];
#if !defined(SEATBELTS) || SEATBELTS
            buffer_size += sizeof(unsigned int);  // Extra uint is used as read-write flag by seatbelts
#endif
            gpuErrchk(hipMalloc(&prop.second.d_ptr, buffer_size));
            gpuErrchk(hipMemsetAsync(prop.second.d_ptr, 0, buffer_size, stream));
        }
    }
    gpuErrchk(hipStreamSynchronize(stream));
}

void CUDAMacroEnvironment::init(const SubEnvironmentData& mapping, const CUDAMacroEnvironment &master_macro_env, hipStream_t stream) {
    // Map local properties
    for (auto& prop : properties) {
        if (!prop.second.d_ptr) {
            auto sub = mapping.macro_properties.find(prop.first);
            if (sub == mapping.macro_properties.end()) {
                // If it's a local macro property
                    size_t buffer_size = prop.second.type_size
                        * prop.second.elements[0]
                        * prop.second.elements[1]
                        * prop.second.elements[2]
                        * prop.second.elements[3];
#if !defined(SEATBELTS) || SEATBELTS
                    buffer_size += sizeof(unsigned int);  // Extra uint is used as read-write flag by seatbelts
#endif
                    gpuErrchk(hipMalloc(&prop.second.d_ptr, buffer_size));
                    gpuErrchk(hipMemsetAsync(prop.second.d_ptr, 0, buffer_size, stream));
            } else {
                // If it's a mapped sub macro property
                auto mmp = master_macro_env.properties.find(sub->second);
                if (mmp != master_macro_env.properties.end()
                    && mmp->second.d_ptr
                    && mmp->second.elements == prop.second.elements
                    && mmp->second.type == prop.second.type) {
                    prop.second.d_ptr = mmp->second.d_ptr;
                    prop.second.is_sub = true;
                } else {
                    THROW exception::UnknownInternalError("Unable to initialise mapped macro property '%s' to '%s', this should not have failed, "
                    "in CUDAMacroEnvironment::init()\n",
                    prop.first.c_str(), sub->second.c_str());
                }
            }
        }
    }
    gpuErrchk(hipStreamSynchronize(stream));
}
void CUDAMacroEnvironment::free() {
    for (auto& prop : properties) {
        if (prop.second.d_ptr) {
            if (!prop.second.is_sub) {
                gpuErrchk(flamegpu::util::detail::cuda::hipFree(prop.second.d_ptr));
            }
            prop.second.d_ptr = nullptr;
        }
    }
}
void CUDAMacroEnvironment::registerCurveVariables(detail::curve::HostCurve& curve) const {
    for (const auto& p : properties) {
        const unsigned int total_elements = p.second.elements[0] * p.second.elements[1] * p.second.elements[2] * p.second.elements[3];
        curve.registerSetMacroEnvironmentProperty(p.first, p.second.type, p.second.type_size, total_elements, p.second.d_ptr);
    }
}
void CUDAMacroEnvironment::mapRTCVariables(detail::curve::CurveRTCHost& curve_header) const {
    for (const auto &p : properties) {
        curve_header.registerEnvMacroProperty(p.first.c_str(), p.second.d_ptr, p.second.type.name(), p.second.type_size, p.second.elements);
    }
}
void CUDAMacroEnvironment::unmapRTCVariables(detail::curve::CurveRTCHost& curve_header) const {
    for (const auto &p : properties) {
        curve_header.unregisterEnvMacroProperty(p.first.c_str());
    }
}
#if !defined(SEATBELTS) || SEATBELTS
void CUDAMacroEnvironment::resetFlagsAsync(const std::vector<hipStream_t> &streams) {
    unsigned int i = 0;
    for (const auto& prop : properties) {
        if (prop.second.d_ptr) {
            const size_t buffer_size = prop.second.type_size
                * prop.second.elements[0]
                * prop.second.elements[1]
                * prop.second.elements[2]
                * prop.second.elements[3];
            gpuErrchk(hipMemsetAsync(static_cast<char*>(prop.second.d_ptr) + buffer_size, 0 , sizeof(unsigned int), streams[i++%streams.size()]));
        }
    }
    // Disable the sync here, users must sync themselves
    // if (properties.size()) {
    //     gpuErrchk(hipDeviceSynchronize());
    // }
}
bool CUDAMacroEnvironment::getDeviceReadFlag(const std::string& property_name) {
    const auto prop = properties.find(property_name);
    if (prop == properties.end()) {
        THROW flamegpu::exception::InvalidEnvProperty("The environment macro property '%s' was not found, "
            "in CUDAMacroEnvironment::getDeviceReadFlag()\n",
            property_name.c_str());
    }
    const size_t buffer_size = prop->second.type_size
        * prop->second.elements[0]
        * prop->second.elements[1]
        * prop->second.elements[2]
        * prop->second.elements[3];
    unsigned int ret = 0;
    gpuErrchk(hipMemcpy(&ret, static_cast<char*>(prop->second.d_ptr) + buffer_size, sizeof(unsigned int), hipMemcpyDeviceToHost));
    return (ret & 1u << 0);
}
bool CUDAMacroEnvironment::getDeviceWriteFlag(const std::string& property_name) {
    const auto prop = properties.find(property_name);
    if (prop == properties.end()) {
        THROW flamegpu::exception::InvalidEnvProperty("The environment macro property '%s' was not found, "
            "in CUDAMacroEnvironment::getDeviceReadFlag()\n",
            property_name.c_str());
    }
    const size_t buffer_size = prop->second.type_size
        * prop->second.elements[0]
        * prop->second.elements[1]
        * prop->second.elements[2]
        * prop->second.elements[3];
    unsigned int ret = 0;
    gpuErrchk(hipMemcpy(&ret, static_cast<char*>(prop->second.d_ptr) + buffer_size, sizeof(unsigned int), hipMemcpyDeviceToHost));
    return (ret & 1u << 1);
}
unsigned int CUDAMacroEnvironment::getDeviceRWFlags(const std::string& property_name) {
    const auto prop = properties.find(property_name);
    if (prop == properties.end()) {
        THROW flamegpu::exception::InvalidEnvProperty("The environment macro property '%s' was not found, "
            "in CUDAMacroEnvironment::getDeviceReadFlag()\n",
            property_name.c_str());
    }
    const size_t buffer_size = prop->second.type_size
        * prop->second.elements[0]
        * prop->second.elements[1]
        * prop->second.elements[2]
        * prop->second.elements[3];
    unsigned int ret = 0;
    gpuErrchk(hipMemcpy(&ret, static_cast<char*>(prop->second.d_ptr) + buffer_size, sizeof(unsigned int), hipMemcpyDeviceToHost));
    return ret;
}
#endif
}  // namespace flamegpu
