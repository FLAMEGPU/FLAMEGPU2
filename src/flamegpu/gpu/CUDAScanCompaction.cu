#include <cassert>

#include "flamegpu/gpu/CUDAScanCompaction.h"
#include "flamegpu/gpu/CUDAErrorChecking.h"
#include "flamegpu/gpu/CUDAAgentModel.h"


namespace flamegpu_internal {
namespace CUDAScanCompaction {
    /**
    * These will remain unallocated until used
    * They exist so that the correct array can be used with only the stream index known
    */
    __device__ CUDAScanCompactionPtrs ds_configs[MAX_TYPES][MAX_STREAMS];
    /**
    * Host mirror of ds_configs
    */
    CUDAScanCompactionConfig hd_configs[MAX_TYPES][MAX_STREAMS];

}  // namespace CUDAScanCompaction
}  // namespace flamegpu_internal


__host__ void CUDAScanCompactionConfig::free_scan_flag() {
    if (d_ptrs.scan_flag) {
        gpuErrchk(hipFree(d_ptrs.scan_flag));
    }
    if (d_ptrs.position) {
        gpuErrchk(hipFree(d_ptrs.position));
    }
}

__host__ void CUDAScanCompactionConfig::zero_scan_flag() {
    if (d_ptrs.position) {
        gpuErrchk(hipMemset(d_ptrs.position, 0, scan_flag_len * sizeof(unsigned int)));
    }
    if (d_ptrs.scan_flag) {
        gpuErrchk(hipMemset(d_ptrs.scan_flag, 0, scan_flag_len * sizeof(unsigned int)));
    }
}

__host__ void CUDAScanCompactionConfig::resize_scan_flag(const unsigned int& count, const CUDAAgentModel& model) {
    if (count + 1 > scan_flag_len) {
        free_scan_flag();
        gpuErrchk(hipMalloc(&d_ptrs.scan_flag, (count + 1) * sizeof(unsigned int)));  // +1 so we can get the total from the scan
        gpuErrchk(hipMalloc(&d_ptrs.position, (count + 1) * sizeof(unsigned int)));  // +1 so we can get the total from the scan
        // Calculate offset of this object from start of array, then divide by size of this object, and multiply by size of device object

        scan_flag_len = count + 1;
    }
    // TODO: This can be moved back in side the check once CUDAScanCompation is no longer a singleton
    ptrdiff_t output_dist = (std::distance(reinterpret_cast<char*>(flamegpu_internal::CUDAScanCompaction::hd_configs), reinterpret_cast<char*>(this)) / sizeof(CUDAScanCompactionConfig)) * sizeof(CUDAScanCompactionPtrs);
    // gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(flamegpu_internal::CUDAScanCompaction::ds_configs), &this->d_ptrs, sizeof(CUDAScanCompactionPtrs), output_dist));
    // call the RTC safe version of cudamemcpy
    model.RTCSafeCudaMemcpyToSymbol(flamegpu_internal::CUDAScanCompaction::ds_configs, "flamegpu_internal::CUDAScanCompaction::ds_configs", &this->d_ptrs, sizeof(CUDAScanCompactionPtrs), output_dist);
}


void flamegpu_internal::CUDAScanCompaction::resize(const unsigned int& newCount, const flamegpu_internal::CUDAScanCompaction::Type& type, const unsigned int& streamId, const CUDAAgentModel &model) {
    assert(streamId < MAX_STREAMS);
    assert(type < MAX_TYPES);
    hd_configs[type][streamId].resize_scan_flag(newCount, model);
}


void flamegpu_internal::CUDAScanCompaction::zero(const flamegpu_internal::CUDAScanCompaction::Type& type, const unsigned int& streamId) {
    assert(streamId < MAX_STREAMS);
    assert(type < MAX_TYPES);
    hd_configs[type][streamId].zero_scan_flag();
}


