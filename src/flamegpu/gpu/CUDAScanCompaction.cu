#include <cassert>

#include "flamegpu/gpu/CUDAScanCompaction.h"
#include "flamegpu/gpu/detail/CUDAErrorChecking.cuh"
#include "flamegpu/gpu/CUDASimulation.h"

namespace flamegpu {

/**
 * CUDAScanCompaction methods
 */
void CUDAScanCompaction::resize(const unsigned int newCount, const Type& type, const unsigned int streamId) {
    assert(streamId < MAX_STREAMS);
    assert(type < MAX_TYPES);
    configs[type][streamId].resize_scan_flag(newCount);
}

void CUDAScanCompaction::zero_async(const Type& type, hipStream_t stream, unsigned int streamId) {
    assert(streamId < MAX_STREAMS);
    assert(type < MAX_TYPES);
    configs[type][streamId].zero_scan_flag_async(stream);
}

const CUDAScanCompactionConfig &CUDAScanCompaction::getConfig(const Type& type, const unsigned int streamId) {
    return configs[type][streamId];
}
CUDAScanCompactionConfig &CUDAScanCompaction::Config(const Type& type, const unsigned int streamId) {
    return configs[type][streamId];
}
/**
 *
 */
CUDAScanCompactionConfig::~CUDAScanCompactionConfig() {
    free_scan_flag();
}
void CUDAScanCompactionConfig::free_scan_flag() {
    if (d_ptrs.scan_flag) {
        gpuErrchk(hipFree(d_ptrs.scan_flag));
        d_ptrs.scan_flag = nullptr;
    }
    if (d_ptrs.position) {
        gpuErrchk(hipFree(d_ptrs.position));
        d_ptrs.position = nullptr;
    }
}

void CUDAScanCompactionConfig::zero_scan_flag_async(hipStream_t stream) {
    if (d_ptrs.position) {
        gpuErrchk(hipMemsetAsync(d_ptrs.position, 0, scan_flag_len * sizeof(unsigned int), stream));
    }
    if (d_ptrs.scan_flag) {
        gpuErrchk(hipMemsetAsync(d_ptrs.scan_flag, 0, scan_flag_len * sizeof(unsigned int), stream));
    }
}

void CUDAScanCompactionConfig::resize_scan_flag(const unsigned int count) {
    if (count + 1 > scan_flag_len) {
        free_scan_flag();
        gpuErrchk(hipMalloc(&d_ptrs.scan_flag, (count + 1) * sizeof(unsigned int)));  // +1 so we can get the total from the scan
        gpuErrchk(hipMalloc(&d_ptrs.position, (count + 1) * sizeof(unsigned int)));  // +1 so we can get the total from the scan
        scan_flag_len = count + 1;
    }
}

}  // namespace flamegpu
