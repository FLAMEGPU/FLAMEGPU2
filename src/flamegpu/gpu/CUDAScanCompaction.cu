#include <cassert>

#include "flamegpu/gpu/CUDAScanCompaction.h"
#include "flamegpu/gpu/CUDAErrorChecking.h"
#include "flamegpu/gpu/CUDAAgentModel.h"

/**
 * CUDAScanCompaction methods
 */
void CUDAScanCompaction::purge() {
    memset(configs, 0, sizeof(configs));
}

void CUDAScanCompaction::resize(const unsigned int& newCount, const Type& type, const unsigned int& streamId) {
    assert(streamId < MAX_STREAMS);
    assert(type < MAX_TYPES);
    configs[type][streamId].resize_scan_flag(newCount);
}

void CUDAScanCompaction::zero(const Type& type, const unsigned int& streamId) {
    assert(streamId < MAX_STREAMS);
    assert(type < MAX_TYPES);
    configs[type][streamId].zero_scan_flag();
}

const CUDAScanCompactionConfig &CUDAScanCompaction::getConfig(const Type& type, const unsigned int& streamId) {
    return configs[type][streamId];
}
CUDAScanCompactionConfig &CUDAScanCompaction::Config(const Type& type, const unsigned int& streamId) {
    return configs[type][streamId];
}
/**
 *
 */
CUDAScanCompactionConfig::~CUDAScanCompactionConfig() {
    free_scan_flag();
}
void CUDAScanCompactionConfig::free_scan_flag() {
    if (d_ptrs.scan_flag) {
        gpuErrchk(hipFree(d_ptrs.scan_flag));
        d_ptrs.scan_flag = nullptr;
    }
    if (d_ptrs.position) {
        gpuErrchk(hipFree(d_ptrs.position));
        d_ptrs.position = nullptr;
    }
}

void CUDAScanCompactionConfig::zero_scan_flag() {
    if (d_ptrs.position) {
        gpuErrchk(hipMemset(d_ptrs.position, 0, scan_flag_len * sizeof(unsigned int)));
    }
    if (d_ptrs.scan_flag) {
        gpuErrchk(hipMemset(d_ptrs.scan_flag, 0, scan_flag_len * sizeof(unsigned int)));
    }
}

void CUDAScanCompactionConfig::resize_scan_flag(const unsigned int& count) {
    if (count + 1 > scan_flag_len) {
        free_scan_flag();
        gpuErrchk(hipMalloc(&d_ptrs.scan_flag, (count + 1) * sizeof(unsigned int)));  // +1 so we can get the total from the scan
        gpuErrchk(hipMalloc(&d_ptrs.position, (count + 1) * sizeof(unsigned int)));  // +1 so we can get the total from the scan
        scan_flag_len = count + 1;
    }
}
