/**
* @file CUDAMessage.cpp
* @authors
* @date
* @brief
*
* @see
* @warning
*/

#include <hip/hip_runtime.h>
#include <>

#include "flamegpu/gpu/CUDAMessage.h"
#include "flamegpu/gpu/CUDAMessageList.h"
#include "flamegpu/gpu/CUDAErrorChecking.h"

#include "flamegpu/model/MessageDescription.h"
#include "flamegpu/model/AgentFunctionDescription.h"
#include "flamegpu/runtime/cuRVE/curve.h"
#include "flamegpu/model/AgentDescription.h"
#include "flamegpu/pop/AgentPopulation.h"

#ifdef _MSC_VER
#pragma warning(push, 3)
#include <hipcub/hipcub.hpp>
#pragma warning(pop)
#else
#include <hipcub/hipcub.hpp>
#endif

/**
* CUDAMessage class
* @brief allocates the hash table/list for message variables and copy the list to device
*/
CUDAMessage::CUDAMessage(const MessageData& description)
    : message_description(description)
    , message_count(0)
    , max_list_size(0)
    , curve(Curve::getInstance()) {
    // resize(0); // Think this call is redundant
}

/**
 * A destructor.
 * @brief Destroys the CUDAMessage object
 */
CUDAMessage::~CUDAMessage(void) {
}

/**
* @brief Returns message description
* @param none
* @return MessageDescription object
*/
const MessageData& CUDAMessage::getMessageDescription() const {
    return message_description;
}

/**
* @brief Sets initial message data to zero by allocating memory for message lists
* @param empty
* @return none
*/

void CUDAMessage::resize(unsigned int newSize, const unsigned int &streamId) {
    // Only grow currently
    max_list_size = max_list_size < 2 ? 2 : max_list_size;
    if (newSize > max_list_size) {
        while (max_list_size < newSize) {
            max_list_size = static_cast<unsigned int>(max_list_size * 1.5);
        }
        // This drops old message data
        message_list = std::unique_ptr<CUDAMessageList>(new CUDAMessageList(*this));
        flamegpu_internal::CUDAScanCompaction::resizeMessages(max_list_size, streamId);

// #ifdef _DEBUG
        /**set the message list to zero*/
        zeroAllMessageData();
// #endif
    }
    message_count = newSize;  // This will be reduced down after function call if optional
}


/**
* @brief Returns the maximum list size
* @param none
* @return maximum size list that is equal to the maximum list size
* @note may want to change this to maximum population size
*/
unsigned int CUDAMessage::getMaximumListSize() const {
    return max_list_size;
}
unsigned int CUDAMessage::getMessageCount() const {
    return message_count;
}

/**
* @brief Sets all message variable data to zero
* @param none
* @return none
*/
void CUDAMessage::zeroAllMessageData() {
    message_list->zeroMessageData();
}

/**
@bug message_name is input or output, run some tests to see which one is correct
*/
void CUDAMessage::mapReadRuntimeVariables(const AgentFunctionData& func) const {
    // check that the message list has been allocated
    if (!message_list) {
        THROW InvalidMessageData("Error: Initial message list for message '%s' has not been allocated, "
            "in CUDAMessage::mapRuntimeVariables()",
            message_description.name.c_str());
    }

    const std::string message_name = message_description.name;

    const Curve::VariableHash message_hash = curve.variableRuntimeHash(message_name.c_str());
    const Curve::VariableHash agent_hash = curve.variableRuntimeHash(func.parent.lock()->name.c_str());
    const Curve::VariableHash func_hash = curve.variableRuntimeHash(func.name.c_str());
    // loop through the message variables to map each variable name using cuRVE
    for (const auto &mmp : message_description.variables) {
        // get a device pointer for the message variable name
        void* d_ptr = message_list->getReadMessageListVariablePointer(mmp.first);

        // map using curve
        Curve::VariableHash var_hash = curve.variableRuntimeHash(mmp.first.c_str());

        // get the message variable size
        size_t size = mmp.second.type_size;

       // maximum population size
        unsigned int length = this->getMessageCount();  // check to see if it is equal to pop
        curve.registerVariableByHash(var_hash + agent_hash + func_hash + message_hash, d_ptr, size, length);
    }
}
void CUDAMessage::mapWriteRuntimeVariables(const AgentFunctionData& func) const {
    // check that the message list has been allocated
    if (!message_list) {
        THROW InvalidMessageData("Error: Initial message list for message '%s' has not been allocated, "
            "in CUDAMessage::mapRuntimeVariables()",
            message_description.name.c_str());
    }

    const std::string message_name = message_description.name;

    const Curve::VariableHash message_hash = curve.variableRuntimeHash(message_name.c_str());
    const Curve::VariableHash agent_hash = curve.variableRuntimeHash(func.parent.lock()->name.c_str());
    const Curve::VariableHash func_hash = curve.variableRuntimeHash(func.name.c_str());
    // loop through the message variables to map each variable name using cuRVE
    for (const auto &mmp : message_description.variables) {
        // get a device pointer for the message variable name
        void* d_ptr = message_list->getWriteMessageListVariablePointer(mmp.first);

        // map using curve
        Curve::VariableHash var_hash = curve.variableRuntimeHash(mmp.first.c_str());

        // get the message variable size
        size_t size = mmp.second.type_size;

        // maximum population size
        unsigned int length = this->getMessageCount();  // check to see if it is equal to pop
        curve.registerVariableByHash(var_hash + agent_hash + func_hash + message_hash, d_ptr, size, length);
    }
}

void CUDAMessage::unmapRuntimeVariables(const AgentFunctionData& func) const {
    const std::string message_name = message_description.name;

    const Curve::VariableHash message_hash = curve.variableRuntimeHash(message_name.c_str());
    const Curve::VariableHash agent_hash = curve.variableRuntimeHash(func.parent.lock()->name.c_str());
    const Curve::VariableHash func_hash = curve.variableRuntimeHash(func.name.c_str());
    // loop through the message variables to map each variable name using cuRVE
    for (const auto &mmp : message_description.variables) {
        // get a device pointer for the message variable name
        // void* d_ptr = message_list->getMessageListVariablePointer(mmp.first);

        // unmap using curve
        Curve::VariableHash var_hash = curve.variableRuntimeHash(mmp.first.c_str());
        curve.unregisterVariableByHash(var_hash + agent_hash + func_hash + message_hash);
    }
}
void CUDAMessage::swap(bool isOptional, const unsigned int &streamId) {
    if (isOptional && message_description.optional_outputs > 0) {
        if (message_count > flamegpu_internal::CUDAScanCompaction::hd_message_configs[streamId].cub_temp_size_max_list_size) {
            if (flamegpu_internal::CUDAScanCompaction::hd_message_configs[streamId].hd_cub_temp) {
                gpuErrchk(hipFree(flamegpu_internal::CUDAScanCompaction::hd_message_configs[streamId].hd_cub_temp));
            }
            flamegpu_internal::CUDAScanCompaction::hd_message_configs[streamId].cub_temp_size = 0;
            hipcub::DeviceScan::ExclusiveSum(
                nullptr,
                flamegpu_internal::CUDAScanCompaction::hd_message_configs[streamId].cub_temp_size,
                flamegpu_internal::CUDAScanCompaction::hd_message_configs[streamId].d_ptrs.scan_flag,
                flamegpu_internal::CUDAScanCompaction::hd_message_configs[streamId].d_ptrs.position,
                max_list_size + 1);
            gpuErrchk(hipMalloc(&flamegpu_internal::CUDAScanCompaction::hd_message_configs[streamId].hd_cub_temp, flamegpu_internal::CUDAScanCompaction::hd_message_configs[streamId].cub_temp_size));
            flamegpu_internal::CUDAScanCompaction::hd_message_configs[streamId].cub_temp_size_max_list_size = max_list_size;
        }
        hipcub::DeviceScan::ExclusiveSum(
            flamegpu_internal::CUDAScanCompaction::hd_message_configs[streamId].hd_cub_temp,
            flamegpu_internal::CUDAScanCompaction::hd_message_configs[streamId].cub_temp_size,
            flamegpu_internal::CUDAScanCompaction::hd_message_configs[streamId].d_ptrs.scan_flag,
            flamegpu_internal::CUDAScanCompaction::hd_message_configs[streamId].d_ptrs.position,
            message_count + 1);
        // Scatter
        // Update count
        message_count = message_list->scatter(streamId);
    } else {
        message_list->swap();
    }
}
