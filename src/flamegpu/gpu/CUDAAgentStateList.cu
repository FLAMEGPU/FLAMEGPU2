#include "hip/hip_runtime.h"
#include "flamegpu/gpu/CUDAAgentStateList.h"

#include <hip/hip_runtime.h>
#include <>

#include "flamegpu/gpu/CUDAAgent.h"
#include "flamegpu/gpu/CUDAErrorChecking.h"
#include "flamegpu/pop/AgentStateMemory.h"
#include "flamegpu/model/AgentDescription.h"
#include "flamegpu/gpu/CUDAScatter.h"
#include "flamegpu/runtime/flamegpu_host_new_agent_api.h"
#include "flamegpu/exception/FGPUException.h"

#ifdef _MSC_VER
#pragma warning(push, 3)
#include <hipcub/hipcub.hpp>
#pragma warning(pop)
#else
#include <hipcub/hipcub.hpp>
#endif

CUDAAgentStateList::CUDAAgentStateList(
    const std::shared_ptr<CUDAFatAgentStateList> &fat_list,
    CUDAAgent& cuda_agent,
    const unsigned int &_fat_index,
    const AgentData& description,
    bool _isSubStateList)
    : fat_index(_fat_index)
    , agent(cuda_agent)
    , parent_list(fat_list)
    , isSubStateList(_isSubStateList) {
    // For each agent variable, take a copy of the shared pointer, store it
    for (auto var : description.variables) {
        variables.emplace(var.first, fat_list->getVariableBuffer(fat_index, var.first));
    }
}
CUDAAgentStateList::CUDAAgentStateList(
    const std::shared_ptr<CUDAFatAgentStateList> &fat_list,
    CUDAAgent& cuda_agent,
    const unsigned int &_fat_index,
    const AgentData& description,
    bool _isSubStateList,
    const SubAgentData::Mapping &varMap)
    : CUDAAgentStateList(fat_list, cuda_agent, _fat_index, description, _isSubStateList) {
    // Build a list of variables not present in the mapping
    // These are not mapped to parent agent, therefore they must be reset when CUDAAgentModel::simulate() is called
    for (auto var : variables) {
        if (varMap.find(var.first)== varMap.end()) {
            unmappedBuffers.push_back(var.second);
        }
    }
}
void CUDAAgentStateList::resize(const unsigned int &minimumSize, const bool &retainData) {
    parent_list->resize(minimumSize, retainData);
}
unsigned int CUDAAgentStateList::getSize() const {
    return parent_list->getSize();
}
/**
 * Returns the maximum number of agents that can be stored based on the current buffer allocations
 */
unsigned int CUDAAgentStateList::getAllocatedSize() const {
    return parent_list->getAllocatedSize();
}
void *CUDAAgentStateList::getVariablePointer(const std::string &variable_name) {
    // check the cuda agent state map to find the correct state list for functions starting state
    auto var = variables.find(variable_name);

    if (var == variables.end()) {
        THROW InvalidAgentVar("Error: Agent ('%s') variable ('%s') was not found "
            "in CUDAAgentStateList::getVariablePointer()",
            agent.getAgentDescription().name.c_str(), variable_name.c_str());
    }

    return var->second->data_condition;
}
void CUDAAgentStateList::setAgentData(const AgentStateMemory &data, CUDAScatter &scatter, const unsigned int &streamId) {
    // check that we are using the same agent description
    if (!data.isSameDescription(agent.getAgentDescription())) {
        THROW InvalidCudaAgentDesc("Agent State memory has different description to CUDA Agent ('%s'), "
            "in CUDAAgentStateList::setAgentData().",
            agent.getAgentDescription().name.c_str());
    }
    // Check our internal state matches or exceeds the size of the state in the agent pop
    // This will return if list already correct size
    const unsigned int data_count = data.getStateListSize();
    if (data_count) {
        parent_list->resize(data_count, false);  // FALSE=Do not retain existing data
        // Initialise any buffers in the fat_agent which aren't part of the agent description
        std::set<std::shared_ptr<VariableBuffer>> exclusionSet;
        for (auto &a : variables)
            exclusionSet.insert(a.second);
        parent_list->initVariables(exclusionSet, data_count, 0, scatter, streamId);
        // Copy across the required data host->device
        for (auto &_var : variables) {
            // get the variable size from agent description
            const auto &var = agent.getAgentDescription().variables.at(_var.first);
            const size_t var_size = var.type_size;
            const unsigned int  var_elements = var.elements;

            // get the vector
            const GenericMemoryVector &m_vec = data.getReadOnlyMemoryVector(_var.first);

            // get pointer to vector data
            const void * v_data = m_vec.getReadOnlyDataPtr();

            // copy the host data to the GPU
            gpuErrchk(hipMemcpy(_var.second->data, v_data, var_elements * var_size * data_count, hipMemcpyHostToDevice));
        }
    }
    // Update alive count etc
    parent_list->setAgentCount(data_count);
}
void CUDAAgentStateList::getAgentData(AgentStateMemory &data) {
    // check that we are using the same agent description
    if (!data.isSameDescription(agent.getAgentDescription())) {
        THROW InvalidCudaAgentDesc("Agent State memory has different description to CUDA Agent ('%s'), "
            "in CUDAAgentStateList::getAgentData().",
            agent.getAgentDescription().name.c_str());
    }
    const unsigned int data_count = getSize();
    if (data_count) {
        // Check the output buffer has been resized
        if (data.getPopulationCapacity() < data_count) {
            THROW InvalidMemoryCapacity("AgentStateMemory must be resized before passing to CUDAAgentStateList::getAgentData()\n");
        }
        // Copy across the required data device->host
        for (auto &_var : variables) {
            // get the variable size from agent description
            const auto &var = agent.getAgentDescription().variables.at(_var.first);
            const size_t var_size = var.type_size;
            const unsigned int  var_elements = var.elements;

            // get the vector
            GenericMemoryVector &m_vec = data.getMemoryVector(_var.first);

            // get pointer to vector data
            void * v_data = m_vec.getDataPtr();

            // copy the host data to the GPU
            gpuErrchk(hipMemcpy(v_data, _var.second->data, var_elements * var_size * data_count, hipMemcpyDeviceToHost));
        }
    }

    // Update alive count etc
    data.overrideStateListSize(data_count);
}
void CUDAAgentStateList::scatterHostCreation(const unsigned int &newSize, char *const d_inBuff, const VarOffsetStruct &offsets, CUDAScatter &scatter, const unsigned int &streamId) {
    // Resize agent list if required
    parent_list->resize(parent_list->getSizeWithDisabled() + newSize, true);
    // Build scatter data
    std::vector<CUDAScatter::ScatterData> sd;
    for (const auto &v : variables) {
        // In this case, in is the location of first variable, but we step by inOffsetData.totalSize
        char *in_p = reinterpret_cast<char*>(d_inBuff) + offsets.vars.at(v.first).offset;
        char *out_p = reinterpret_cast<char*>(v.second->data);
        sd.push_back({ v.second->type_size * v.second->elements, in_p, out_p });
    }
    // Scatter to device
    scatter.scatterNewAgents(streamId,
        sd,
        offsets.totalSize,
        newSize,
        parent_list->getSize());
    // Initialise any buffers in the fat_agent which aren't part of the current agent description
    // TODO: This does redundant inits, it only needs to initialise parent/master agent variables which are not mapped
    //       Sub variables will already be init everytime the submodel is executed.
    std::set<std::shared_ptr<VariableBuffer>> exclusionSet;
    for (auto &a : variables)
        exclusionSet.insert(a.second);
    parent_list->initVariables(exclusionSet, newSize, parent_list->getSize(), scatter, streamId);
    // Update number of alive agents
    parent_list->setAgentCount(parent_list->getSize() + newSize);
}
void CUDAAgentStateList::scatterNew(void * d_newBuff, const unsigned int &newSize, CUDAScatter &scatter, const unsigned int &streamId) {
    if (newSize) {
        CUDAScanCompactionConfig &scanCfg = scatter.Scan().Config(CUDAScanCompaction::Type::AGENT_OUTPUT, streamId);
        // Perform scan
        if (newSize > scanCfg.cub_temp_size_max_list_size) {
            if (scanCfg.hd_cub_temp) {
                gpuErrchk(hipFree(scanCfg.hd_cub_temp));
            }
            scanCfg.cub_temp_size = 0;
            gpuErrchk(hipcub::DeviceScan::ExclusiveSum(
                nullptr,
                scanCfg.cub_temp_size,
                scanCfg.d_ptrs.scan_flag,
                scanCfg.d_ptrs.position,
                newSize + 1));
            gpuErrchk(hipMalloc(&scanCfg.hd_cub_temp,
                scanCfg.cub_temp_size));
            scanCfg.cub_temp_size_max_list_size = newSize;
        }
        gpuErrchk(hipcub::DeviceScan::ExclusiveSum(
            scanCfg.hd_cub_temp,
            scanCfg.cub_temp_size,
            scanCfg.d_ptrs.scan_flag,
            scanCfg.d_ptrs.position,
            newSize + 1));
        // Resize if necessary
        // @todo? this could be improved by checking scan result for the actual size, rather than max size)
        resize(parent_list->getSizeWithDisabled() + newSize, true);
        // Build scatter data
        char * d_var = static_cast<char*>(d_newBuff);

        std::vector<CUDAScatter::ScatterData> scatterdata;
        for (const auto &v : variables) {
            char *in_p = reinterpret_cast<char*>(d_var);
            char *out_p = reinterpret_cast<char*>(v.second->data_condition);
            scatterdata.push_back({ v.second->type_size * v.second->elements, in_p, out_p });
            // Prep pointer for next var
            d_var += v.second->type_size * v.second->elements * newSize;
            // 64 bit align the new buffer start
            if (reinterpret_cast<size_t>(d_var)%8) {
                d_var += 8 - (reinterpret_cast<size_t>(d_var)%8);
            }
        }
        // Perform scatter
        const unsigned int new_births = scatter.scatter(
            streamId,
            CUDAScatter::Type::AGENT_OUTPUT,
            scatterdata,
            newSize, parent_list->getSizeWithDisabled());
        if (new_births == 0) return;
        // Initialise any buffers in the fat_agent which aren't part of the current agent description
        // TODO: This does redundant inits, it only needs to initialise parent/master agent variables which are not mapped
        //       Sub variables will already be init everytime the submodel is executed.
        std::set<std::shared_ptr<VariableBuffer>> exclusionSet;
        for (auto &a : variables)
            exclusionSet.insert(a.second);
        parent_list->initVariables(exclusionSet, newSize, parent_list->getSize(), scatter, streamId);
        // Update number of alive agents
        parent_list->setAgentCount(parent_list->getSize() + new_births);
    }
}
bool CUDAAgentStateList::getIsSubStatelist() {
    return isSubStateList;
}
void CUDAAgentStateList::initUnmappedVars(CUDAScatter &scatter, const unsigned int &streamId) {
    assert(parent_list->getSizeWithDisabled() == parent_list->getSize());
    if (parent_list->getSize()) {
        assert(isSubStateList);
        // If unmappedBuffers is not empty, perform broadcast init
        if (unmappedBuffers.size()) {
            scatter.broadcastInit(streamId, unmappedBuffers, parent_list->getSize(), 0);
        }
    }
}
void CUDAAgentStateList::clear() {
    parent_list->setAgentCount(0, true);
}
