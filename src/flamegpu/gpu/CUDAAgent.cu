#include "hip/hip_runtime.h"
#include "flamegpu/gpu/CUDAAgent.h"

#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <fstream>
#include <string>
// If MSVC earlier than VS 2019
#if defined(_MSC_VER) && _MSC_VER < 1920
#include <filesystem>
using std::tr2::sys::exists;
using std::tr2::sys::path;
#else
// VS2019 requires this macro, as building pre c++17 cant use std::filesystem
#define _SILENCE_EXPERIMENTAL_FILESYSTEM_DEPRECATION_WARNING
#include <experimental/filesystem>
using std::experimental::filesystem::v1::exists;
using std::experimental::filesystem::v1::path;
#endif
#ifdef _MSC_VER
#pragma warning(push, 1)
#pragma warning(disable : 4706 4834)
#include <hipcub/hipcub.hpp>
#pragma warning(pop)
#else
#include <hipcub/hipcub.hpp>
#endif

#include "flamegpu/version.h"
#include "flamegpu/gpu/CUDAFatAgent.h"
#include "flamegpu/gpu/CUDAAgentStateList.h"
#include "flamegpu/gpu/detail/CUDAErrorChecking.cuh"
#include "flamegpu/gpu/CUDASimulation.h"

#include "flamegpu/model/AgentDescription.h"
#include "flamegpu/model/AgentFunctionDescription.h"
#include "flamegpu/runtime/detail/curve/curve.cuh"
#include "flamegpu/runtime/detail/curve/curve_rtc.cuh"
#include "flamegpu/gpu/CUDAScatter.cuh"
#include "flamegpu/util/detail/compute_capability.cuh"
#include "flamegpu/util/nvtx.h"
#include "flamegpu/pop/DeviceAgentVector_impl.h"

namespace flamegpu {

CUDAAgent::CUDAAgent(const AgentData& description, const CUDASimulation &_cudaSimulation)
    : agent_description(description)  // This is a master agent, so it must create a new fat_agent
    , fat_agent(std::make_shared<CUDAFatAgent>(agent_description))  // if we create fat agent, we're index 0
    , fat_index(0)
    , cudaSimulation(_cudaSimulation)
    , TOTAL_AGENT_VARIABLE_SIZE(calcTotalVarSize(description)) {
    // Generate state map from fat_agent
    auto fatstate_map = fat_agent->getStateMap(fat_index);
    for (auto &state : description.states) {
        // Find correct fat state
        auto fatstate = fatstate_map.at(state);
        // Construct a regular state map from this
        auto slimstate = std::make_shared<CUDAAgentStateList>(fatstate, *this, fat_index, agent_description);
        // Store in our map
        state_map.emplace(state, slimstate);
    }
}
CUDAAgent::CUDAAgent(
    const AgentData &description,
    const CUDASimulation &_cudaSimulation,
    const std::unique_ptr<CUDAAgent> &master_agent,
    const std::shared_ptr<SubAgentData> &mapping)
    : agent_description(description)
    , fat_agent(master_agent->getFatAgent())
    , fat_index(fat_agent->getMappedAgentCount())
    , cudaSimulation(_cudaSimulation)
    , TOTAL_AGENT_VARIABLE_SIZE(calcTotalVarSize(description)) {
    // This is next agent to be added to fat_agent, so it takes existing count
    // Pass required info, so fat agent can generate new buffers and mappings
    fat_agent->addSubAgent(agent_description, master_agent->getFatIndex(), mapping);
    // Generate state map from fat_agent
    auto fatstate_map = fat_agent->getStateMap(fat_index);
    for (auto &state : agent_description.states) {
        // Find correct fat state
        auto fatstate = fatstate_map.at(state);
        // Construct a regular state map from this
        auto slimstate = std::make_shared<CUDAAgentStateList>(fatstate, *this, fat_index, agent_description, mapping->states.find(state) != mapping->states.end(), mapping->variables);
        // Store in our map
        state_map.emplace(state, slimstate);
    }
}

void CUDAAgent::mapRuntimeVariables(const AgentFunctionData& func, const unsigned int &instance_id) const {
    // check the cuda agent state map to find the correct state list for functions starting state
    auto sm = state_map.find(func.initial_state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
            "in CUDAAgent::mapRuntimeVariables()",
            agent_description.name.c_str(), func.initial_state.c_str());
    }

    const detail::curve::Curve::VariableHash agent_hash = detail::curve::Curve::variableRuntimeHash(agent_description.name.c_str());
    const detail::curve::Curve::VariableHash func_hash = detail::curve::Curve::variableRuntimeHash(func.name.c_str());
    auto &curve = detail::curve::Curve::getInstance();
    const unsigned int agent_count = this->getStateSize(func.initial_state);
    // loop through the agents variables to map each variable name using cuRVE
    for (const auto &mmp : agent_description.variables) {
        // get a device pointer for the agent variable name
        void* d_ptr = sm->second->getVariablePointer(mmp.first);

        // map using curve
        const detail::curve::Curve::VariableHash var_hash = detail::curve::Curve::variableRuntimeHash(mmp.first.c_str());

        // get the agent variable size
        const size_t type_size = mmp.second.type_size * mmp.second.elements;

        // maximum population num
        if (func.func || func.condition) {
#ifdef _DEBUG
            const detail::curve::Curve::Variable cv = curve.registerVariableByHash(var_hash + agent_hash + func_hash + instance_id, d_ptr, type_size, agent_count);
            if (cv != static_cast<int>((var_hash + agent_hash + func_hash + instance_id)%detail::curve::Curve::MAX_VARIABLES)) {
                fprintf(stderr, "detail::curve::Curve Warning: Agent Function '%s' Variable '%s' has a collision and may work improperly.\n", func.name.c_str(), mmp.first.c_str());
            }
#else
            curve.registerVariableByHash(var_hash + agent_hash + func_hash + instance_id, d_ptr, type_size, agent_count);
#endif
        }
        // Map RTC variables to agent function (these must be mapped before each function execution as the runtime pointer may have changed to the swapping)
        if (!func.rtc_func_name.empty()) {
            // Copy data to rtc header cache
            auto& rtc_header = getRTCHeader(func.name);
            memcpy(rtc_header.getAgentVariableCachePtr(mmp.first.c_str()), &d_ptr, sizeof(void*));
        }

        // Map RTC variables to agent function conditions (these must be mapped before each function execution as the runtime pointer may have changed to the swapping)
        if (!func.rtc_func_condition_name.empty()) {
            // Copy data to rtc header cache
            std::string func_name = func.name + "_condition";
            auto& rtc_header = getRTCHeader(func_name);
            memcpy(rtc_header.getAgentVariableCachePtr(mmp.first.c_str()), &d_ptr, sizeof(void*));
        }
    }
}

void CUDAAgent::unmapRuntimeVariables(const AgentFunctionData& func, const unsigned int &instance_id) const {
    // Skip if RTC
    if (!(func.func || func.condition))
        return;
    // check the cuda agent state map to find the correct state list for functions starting state
    const auto &sm = state_map.find(func.initial_state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
            "in CUDAAgent::unmapRuntimeVariables()",
            agent_description.name.c_str(), func.initial_state.c_str());
    }

    const detail::curve::Curve::VariableHash agent_hash = detail::curve::Curve::variableRuntimeHash(agent_description.name.c_str());
    const detail::curve::Curve::VariableHash func_hash = detail::curve::Curve::variableRuntimeHash(func.name.c_str());
    // loop through the agents variables to map each variable name using cuRVE
    for (const auto &mmp : agent_description.variables) {
        // get a device pointer for the agent variable name
        // void* d_ptr = sm->second->getAgentListVariablePointer(mmp.first);

        // unmap using curve
        const detail::curve::Curve::VariableHash var_hash = detail::curve::Curve::variableRuntimeHash(mmp.first.c_str());
        detail::curve::Curve::getInstance().unregisterVariableByHash(var_hash + agent_hash + func_hash + instance_id);
    }

    // No current need to unmap RTC variables as they are specific to the agent functions and thus do not persist beyond the scope of a single function
}

void CUDAAgent::setPopulationData(const AgentVector& population, const std::string& state_name, CUDAScatter& scatter, const unsigned int& streamId, const hipStream_t& stream) {
    // Validate agent state
    auto our_state = state_map.find(state_name);
    if (our_state == state_map.end()) {
        if (state_name == ModelData::DEFAULT_STATE) {
            THROW exception::InvalidAgentState("Agent '%s' does not use the default state, so the state must be passed explicitly, "
                "in CUDAAgent::setPopulationData()",
                state_name.c_str(), population.getAgentName().c_str());
        } else {
            THROW exception::InvalidAgentState("State '%s' was not found in agent '%s', "
                "in CUDAAgent::setPopulationData()",
                state_name.c_str(), population.getAgentName().c_str());
        }
    }
    // Copy population data
    // This call hierarchy validates agent desc matches
    our_state->second->setAgentData(population, scatter, streamId, stream);
    fat_agent->markIDsUnset();
    // Validate that there are no ID collisions
    validateIDCollisions(stream);
}
void CUDAAgent::getPopulationData(AgentVector& population, const std::string& state_name) const {
    // Validate agent state
    auto our_state = state_map.find(state_name);
    if (our_state == state_map.end()) {
        if (state_name == ModelData::DEFAULT_STATE) {
            THROW exception::InvalidAgentState("Agent '%s' does not use the default state, so the state must be passed explicitly, "
                "in CUDAAgent::getPopulationData()",
                state_name.c_str(), population.getAgentName().c_str());
        } else {
            THROW exception::InvalidAgentState("State '%s' was not found in agent '%s', "
                "in CUDAAgent::getPopulationData()",
                state_name.c_str(), population.getAgentName().c_str());
        }
    }
    // Copy population data
    // This call hierarchy validates agent desc matches
    our_state->second->getAgentData(population);
}
__global__ void generateCollisionFlags(const id_t* d_sortedKeys, id_t* d_flagsOut, unsigned int threads, id_t UNSET_FLAG) {
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < threads) {
        const id_t my_id = d_sortedKeys[id];
        if (my_id != UNSET_FLAG && my_id == d_sortedKeys[id+1]) {
            assert(UNSET_FLAG == 0);
            d_flagsOut[id] = 1;  // my_id; // any non-0 value basically
        }
    }
}
void CUDAAgent::validateIDCollisions(hipStream_t stream) const {
    NVTX_RANGE("CUDAAgent::validateIDCollisions");
    // All data is on device, so use a device technique to check for collisions
    // Sort agent IDs, have a simple kernel check for neighbouring ID collisions to set a flag
    // Scan that flag
    // This could be improved by reusing buffers from elsewhere (e.g. StreamResources), rather than making temporary allocations for each method call
    // However, I'm also concerned that a model with agents added to multiple states and no agent birth would then pre-allocate larger buffers than required during execution

    // First count total agents across all states
    unsigned int agentCount = 0;
    for (const auto &s : state_map) {
        agentCount += s.second->getSize();
    }
    if (!agentCount) return;
    // Allocate buffers we will use
    id_t * d_keysIn = nullptr, *d_keysOut = nullptr;
    gpuErrchk(hipMalloc(&d_keysIn, sizeof(id_t) * agentCount));
    gpuErrchk(hipMalloc(&d_keysOut, sizeof(id_t) * agentCount));
    // Copy agent IDs to keysIn buff
    ptrdiff_t buffOffset = 0;
    for (const auto& s : state_map) {
        const unsigned int t_size = s.second->getSize();
        gpuErrchk(hipMemcpyAsync(d_keysIn + buffOffset, s.second->getVariablePointer(ID_VARIABLE_NAME), t_size * sizeof(id_t), hipMemcpyDeviceToDevice, stream));
        buffOffset += t_size;
    }
    // Sort agent ids into d_keysOut
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    gpuErrchk(hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keysIn, d_keysOut, agentCount, 0, sizeof(id_t) * 8, stream));
    gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
    gpuErrchk(hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keysIn, d_keysOut, agentCount, 0, sizeof(id_t) * 8, stream));
    // Reset d_keysIn
    gpuErrchk(hipMemsetAsync(d_keysIn, 0, sizeof(id_t) * agentCount, stream));
    // Launch a kernel to set flags if keys overlap their neighbour
    const unsigned int blockSize = 1024;
    const unsigned int blocks = ((agentCount-1) / blockSize) + 1;
    generateCollisionFlags<<<blocks, blockSize, 0, stream>>>(d_keysOut, d_keysIn, agentCount-1, ID_NOT_SET);
    gpuErrchkLaunch();
    // Check whether any flags were set
    size_t temp_storage_bytes2 = 0;
    gpuErrchk(hipcub::DeviceReduce::Sum(nullptr, temp_storage_bytes2, d_keysIn, d_keysOut, agentCount - 1, stream));
    if (temp_storage_bytes2 > temp_storage_bytes) {
        gpuErrchk(hipFree(d_temp_storage));
        temp_storage_bytes = temp_storage_bytes2;
        gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
    }
    gpuErrchk(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_keysIn, d_keysOut, agentCount - 1, stream));
    id_t flagsSet = 0;
    gpuErrchk(hipMemcpyAsync(&flagsSet, d_keysOut, sizeof(id_t), hipMemcpyDeviceToHost, stream));
    // Cleanup
    gpuErrchk(hipFree(d_temp_storage));
    gpuErrchk(hipFree(d_keysIn));
    gpuErrchk(hipFree(d_keysOut));
    if (flagsSet) {
        THROW exception::AgentIDCollision("%u agents of type '%s' share an ID with another agent of the same type, "
            "you may need to explicitly reset agent IDs for 1 or more populations before adding them to the CUDASimulation, "
            "in CUDAAgent::validateIDCollisions()\n",
            static_cast<unsigned int>(flagsSet), agent_description.name.c_str());
    }
    gpuErrchk(hipStreamSynchronize(stream));
}
/**
 * Returns the number of alive and active agents in the named state
 */
unsigned int CUDAAgent::getStateSize(const std::string &state) const {
    // check the cuda agent state map to find the correct state list for functions starting state
    const auto &sm = state_map.find(state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found, "
            "in CUDAAgent::getStateSize()",
            agent_description.name.c_str(), state.c_str());
    }
    return sm->second->getSize();
}
/**
 * Returns the number of alive and active agents in the named state
 */
unsigned int CUDAAgent::getStateAllocatedSize(const std::string &state) const {
    // check the cuda agent state map to find the correct state list for functions starting state
    const auto &sm = state_map.find(state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found, "
            "in CUDAAgent::getStateAllocatedSize()",
            agent_description.name.c_str(), state.c_str());
    }
    return sm->second->getAllocatedSize();
}
void CUDAAgent::resizeState(const std::string& state, const unsigned int minimumSize, const bool retainData, const hipStream_t stream) {
    // check the cuda agent state map to find the correct state list
    const auto& sm = state_map.find(state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found, "
            "in CUDAAgent::getStateAllocatedSize()",
            agent_description.name.c_str(), state.c_str());
    }
    sm->second->resize(minimumSize, retainData, stream);
}

void CUDAAgent::setStateAgentCount(const std::string& state, const unsigned int& newSize) {
    // check the cuda agent state map to find the correct state list
    const auto& sm = state_map.find(state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found, "
            "in CUDAAgent::getStateAllocatedSize()",
            agent_description.name.c_str(), state.c_str());
    }
    sm->second->setAgentCount(newSize);
}
const AgentData &CUDAAgent::getAgentDescription() const {
    return agent_description;
}
void *CUDAAgent::getStateVariablePtr(const std::string &state_name, const std::string &variable_name) {
    // check the cuda agent state map to find the correct state list for functions starting state
    const auto &sm = state_map.find(state_name);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found, "
            "in CUDAAgent::getStateVariablePtr()",
            agent_description.name.c_str(), state_name.c_str());
    }
    return sm->second->getVariablePointer(variable_name);
}
void CUDAAgent::processDeath(const AgentFunctionData& func, CUDAScatter &scatter, const unsigned int &streamId, const hipStream_t &stream) {
    // Optionally process agent death
    if (func.has_agent_death) {
        // Agent death operates on all mapped vars, so handled by fat agent
        fat_agent->processDeath(fat_index, func.initial_state, scatter, streamId, stream);
    }
}
void CUDAAgent::transitionState(const std::string &_src, const std::string &_dest, CUDAScatter &scatter, const unsigned int &streamId, const hipStream_t &stream) {
    // All mapped vars need to transition too, so handled by fat agent
    fat_agent->transitionState(fat_index, _src, _dest, scatter, streamId, stream);
}
void CUDAAgent::processFunctionCondition(const AgentFunctionData& func, CUDAScatter &scatter, const unsigned int &streamId, const hipStream_t &stream) {
    // Optionally process function condition
    if ((func.condition) || (!func.rtc_func_condition_name.empty())) {
        // Agent function condition operates on all mapped vars, so handled by fat agent
        fat_agent->processFunctionCondition(fat_index, func.initial_state, scatter, streamId, stream);
    }
}
void CUDAAgent::scatterHostCreation(const std::string &state_name, const unsigned int &newSize, char *const d_inBuff, const VarOffsetStruct &offsets, CUDAScatter &scatter, const unsigned int &streamId, const hipStream_t &stream) {
    auto sm = state_map.find(state_name);
    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
            "in CUDAAgent::scatterHostCreation()",
            agent_description.name.c_str(), state_name.c_str());
    }
    sm->second->scatterHostCreation(newSize, d_inBuff, offsets, scatter, streamId, stream);
}
void CUDAAgent::scatterSort_async(const std::string &state_name, CUDAScatter &scatter, unsigned int streamId, hipStream_t stream) {
    auto sm = state_map.find(state_name);
    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
            "in CUDAAgent::scatterHostCreation()",
            agent_description.name.c_str(), state_name.c_str());
    }
    sm->second->scatterSort_async(scatter, streamId, stream);
}
void CUDAAgent::mapNewRuntimeVariables_async(const CUDAAgent& func_agent, const AgentFunctionData& func, unsigned int maxLen, CUDAScatter &scatter, unsigned int instance_id, hipStream_t stream, unsigned int streamId) {
    // Confirm agent output is set
    if (auto oa = func.agent_output.lock()) {
        // check the cuda agent state map to find the correct state list for functions starting state
        auto sm = state_map.find(func.agent_output_state);

        if (sm == state_map.end()) {
            THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
                "in CUDAAgent::mapNewRuntimeVariables()",
                agent_description.name.c_str(), func.agent_output_state.c_str());
        }
        // Notify scan flag that it might need resizing
        // We need a 3rd array, because a function might combine agent birth, agent death and message output
        scatter.Scan().resize(maxLen, CUDAScanCompaction::AGENT_OUTPUT, streamId);
        // Ensure the scan flag is zeroed
        scatter.Scan().zero_async(CUDAScanCompaction::AGENT_OUTPUT, stream, streamId);

        // Request a buffer for new
        char *d_new_buffer = static_cast<char*>(fat_agent->allocNewBuffer(TOTAL_AGENT_VARIABLE_SIZE, maxLen, agent_description.variables.size()));

        // Store buffer so we can release it later
        {
            std::lock_guard<std::mutex> guard(newBuffsMutex);
            const auto rtn = newBuffs.emplace(func.initial_state, d_new_buffer);
            if (!rtn.second) {
                assert(false);  // Insertion happened (false if element already exists)
            }
        }

        // Init the buffer to default values for variables
        scatter.broadcastInit_async(
            streamId,
            stream,
            agent_description.variables,
            d_new_buffer,
            maxLen, 0);
        // No sync, use of the buffer should be in the same stream

        // Map variables to curve
        const detail::curve::Curve::VariableHash _agent_birth_hash = detail::curve::Curve::variableRuntimeHash("_agent_birth");
        const detail::curve::Curve::VariableHash func_hash = detail::curve::Curve::variableRuntimeHash(func.name.c_str());
        auto &curve = detail::curve::Curve::getInstance();
        // loop through the agents variables to map each variable name using cuRVE
        for (const auto &mmp : agent_description.variables) {
            // map using curve
            const detail::curve::Curve::VariableHash var_hash = detail::curve::Curve::variableRuntimeHash(mmp.first.c_str());

            // get the agent variable size
            const size_t type_size = mmp.second.type_size * mmp.second.elements;

            // get a device pointer for the agent variable name
            void* d_ptr = d_new_buffer;

            // Move the pointer along for next variable
            d_new_buffer += type_size * maxLen;

            // 64 bit align the new buffer start
            if (reinterpret_cast<size_t>(d_new_buffer)%8) {
                d_new_buffer += 8 - (reinterpret_cast<size_t>(d_new_buffer)%8);
            }

            // maximum population num
            if (func.func) {
#ifdef _DEBUG
                const detail::curve::Curve::Variable cv = curve.registerVariableByHash(var_hash + (_agent_birth_hash ^ func_hash) + instance_id, d_ptr, type_size, maxLen);
                if (cv != static_cast<int>((var_hash + (_agent_birth_hash ^ func_hash) + instance_id)%detail::curve::Curve::MAX_VARIABLES)) {
                    fprintf(stderr, "detail::curve::Curve Warning: Agent Function '%s' New Agent Variable '%s' has a collision and may work improperly.\n", func.name.c_str(), mmp.first.c_str());
                }
#else
                curve.registerVariableByHash(var_hash + (_agent_birth_hash ^ func_hash) + instance_id, d_ptr, type_size, maxLen);
#endif
            } else  {
                // Map RTC variables (these must be mapped before each function execution as the runtime pointer may have changed to the swapping)
                // Copy data to rtc header cache
                auto& rtc_header = func_agent.getRTCHeader(func.name);
                memcpy(rtc_header.getNewAgentVariableCachePtr(mmp.first.c_str()), &d_ptr, sizeof(void*));
            }
        }
    }
}
void CUDAAgent::unmapNewRuntimeVariables(const AgentFunctionData& func, const unsigned int &instance_id) {
    // Confirm agent output is set
    if (auto oa = func.agent_output.lock()) {
        // Release new buffer
        {
            std::lock_guard<std::mutex> guard(newBuffsMutex);
            const auto d_buff = newBuffs.find(func.initial_state);
            if (d_buff != newBuffs.end()) {
                fat_agent->freeNewBuffer(d_buff->second);
                newBuffs.erase(d_buff);
            } else {
                assert(false);  // We don't have a new buffer reserved???
            }
        }
        // Skip if RTC
        if (!func.func)
            return;
        // Unmap curve
        const detail::curve::Curve::VariableHash _agent_birth_hash = detail::curve::Curve::variableRuntimeHash("_agent_birth");
        const detail::curve::Curve::VariableHash func_hash = detail::curve::Curve::variableRuntimeHash(func.name.c_str());
        auto &curve = detail::curve::Curve::getInstance();
        // loop through the agents variables to map each variable name using cuRVE
        for (const auto &mmp : agent_description.variables) {
            // unmap using curve
            const detail::curve::Curve::VariableHash var_hash = detail::curve::Curve::variableRuntimeHash(mmp.first.c_str());
            curve.unregisterVariableByHash(var_hash + (_agent_birth_hash ^ func_hash) + instance_id);

            // no need to unmap RTC variables
        }
    }
}

void CUDAAgent::scatterNew(const AgentFunctionData& func, const unsigned int &newSize, CUDAScatter &scatter, const unsigned int &streamId, const hipStream_t &stream) {
    // Confirm agent output is set
    if (auto oa = func.agent_output.lock()) {
        auto sm = state_map.find(func.agent_output_state);
        if (sm == state_map.end()) {
            THROW exception::InvalidStateName("Agent '%s' does not contain state '%s', "
                "in CUDAAgent::scatterNew()\n",
                agent_description.name.c_str(), func.agent_output_state.c_str());
        }
        // Find new buffer
        void *newBuff = nullptr;
        {
            std::lock_guard<std::mutex> guard(newBuffsMutex);
            const auto d_buff = newBuffs.find(func.initial_state);
            if (d_buff != newBuffs.end()) {
                newBuff = d_buff->second;
            }
        }
        if (!newBuff) {
            THROW exception::InvalidAgentFunc("New buffer not present for function within init state: %s,"
                " in CUDAAgent::scatterNew()\n",
                func.initial_state.c_str());
        }
        unsigned int new_births = sm->second->scatterNew(newBuff, newSize, scatter, streamId, stream);
        fat_agent->notifyDeviceBirths(new_births);
    }
}
void CUDAAgent::clearFunctionCondition(const std::string &state) {
    fat_agent->setConditionState(fat_index, state, 0);
}

void CUDAAgent::addInstantitateRTCFunction(const AgentFunctionData& func, const CUDAMacroEnvironment &macro_env, bool function_condition) {
    // Generate the dynamic curve header
    detail::curve::CurveRTCHost &curve_header = *rtc_header_map.emplace(function_condition ? func.name + "_condition" : func.name, std::make_unique<detail::curve::CurveRTCHost>()).first->second;

    // set agent function variables in rtc curve
    for (const auto& mmp : func.parent.lock()->variables) {
        curve_header.registerAgentVariable(mmp.first.c_str(), mmp.second.type.name(), mmp.second.type_size, mmp.second.elements);
    }

    // for normal agent function (e.g. not an agent function condition) append messages and agent outputs
    if (!function_condition) {
        // Set input message variables in curve
        if (auto im = func.message_input.lock()) {
            for (auto message_in_var : im->variables) {
                // register message variables using combined hash
                curve_header.registerMessageInVariable(message_in_var.first.c_str(),
                message_in_var.second.type.name(), message_in_var.second.type_size, message_in_var.second.elements, true, false);
            }
        }
        // Set output message variables in curve
        if (auto om = func.message_output.lock()) {
            for (auto message_out_var : om->variables) {
                // register message variables using combined hash
                curve_header.registerMessageOutVariable(message_out_var.first.c_str(),
                message_out_var.second.type.name(), message_out_var.second.type_size, message_out_var.second.elements, false, true);
            }
        }
        // Set agent output variables in curve
        if (auto ao = func.agent_output.lock()) {
            for (auto agent_out_var : ao->variables) {
                // register message variables using combined hash
                curve_header.registerNewAgentVariable(agent_out_var.first.c_str(),
                agent_out_var.second.type.name(), agent_out_var.second.type_size, agent_out_var.second.elements, false, true);
            }
        }
    }

    // Set Environment variables in curve
    {
        // Scope the mutex
        auto lock = EnvironmentManager::getInstance().getSharedLock();
        const auto &prop_map = EnvironmentManager::getInstance().getPropertiesMap();
        for (const auto &p : prop_map) {
            if (p.first.first == cudaSimulation.getInstanceID()) {
                const char* variableName = p.first.second.c_str();
                const char* type = p.second.type.name();
                unsigned int elements = p.second.elements;
                ptrdiff_t offset = p.second.rtc_offset;
                curve_header.registerEnvVariable(variableName, offset, type, p.second.length/elements, elements);
            }
        }
        // Set mapped environment variables in curve
        for (const auto &mp : EnvironmentManager::getInstance().getMappedProperties()) {
            if (mp.first.first == cudaSimulation.getInstanceID()) {
                auto p = prop_map.at(mp.second.masterProp);
                const char* variableName = mp.second.masterProp.second.c_str();
                const char* type = p.type.name();
                unsigned int elements = p.elements;
                ptrdiff_t offset = p.rtc_offset;
                curve_header.registerEnvVariable(variableName, offset, type, p.length/elements, elements);
            }
        }
    }

    // Set Environment macro properties in curve
    macro_env.mapRTCVariables(curve_header);

    std::string header_filename = std::string(func.rtc_func_name).append("_impl");
    if (function_condition)
        header_filename.append("_condition");
    header_filename.append("_curve_rtc_dynamic.h");
    curve_header.setFileName(header_filename);

    // get the dynamically generated header from curve rtc
    const std::string curve_dynamic_header = curve_header.getDynamicHeader();

    // output to disk if OUTPUT_RTC_DYNAMIC_FILES macro is set
#ifdef OUTPUT_RTC_DYNAMIC_FILES
        // create string for agent function implementation
        std::string func_impl = std::string(func.rtc_func_name).append("_impl");
        // curve
        std::ofstream file_curve_rtc_header;
        std::string file_curve_rtc_header_filename = func_impl.c_str();
        if (function_condition)
            file_curve_rtc_header_filename.append("_condition");
        file_curve_rtc_header_filename.append("_curve_rtc_dynamic.h");
        file_curve_rtc_header.open(file_curve_rtc_header_filename);
        // Remove first line as it is the filename, which misaligns profiler
        std::string out_s = curve_dynamic_header;
        out_s.erase(0, out_s.find("\n") + 1);
        file_curve_rtc_header << out_s;
        file_curve_rtc_header.close();
        // agent function
        std::ofstream agent_function_file;
        std::string agent_function_filename = func_impl.c_str();
        if (function_condition)
            agent_function_filename.append("_condition");
        agent_function_filename.append(".cu");
        agent_function_file.open(agent_function_filename);
        // Remove first line as it is the filename, which misaligns profiler
        out_s = func.rtc_source;
        out_s.erase(0, out_s.find("\n") + 1);
        agent_function_file << out_s;
        agent_function_file.close();
#endif

    util::detail::JitifyCache &jitify = util::detail::JitifyCache::getInstance();
    // switch between normal agent function and agent function condition
    if (!function_condition) {
        const std::string t_func_impl = std::string(func.rtc_func_name).append("_impl");
        const std::vector<std::string> template_args = { t_func_impl.c_str(), func.message_in_type.c_str(), func.message_out_type.c_str() };
        auto kernel_inst = jitify.loadKernel(func.rtc_func_name, template_args, func.rtc_source, curve_dynamic_header);
        // add kernel instance to map
        rtc_func_map.insert(CUDARTCFuncMap::value_type(func.name, std::move(kernel_inst)));
    } else {
        const std::string t_func_impl = std::string(func.rtc_func_condition_name).append("_cdn_impl");
        const std::vector<std::string> template_args = { t_func_impl.c_str() };
        auto kernel_inst = jitify.loadKernel(func.rtc_func_name + "_condition", template_args, func.rtc_condition_source, curve_dynamic_header);
        // add kernel instance to map
        rtc_func_map.insert(CUDARTCFuncMap::value_type(func.name + "_condition", std::move(kernel_inst)));
    }
}

const jitify::experimental::KernelInstantiation& CUDAAgent::getRTCInstantiation(const std::string &function_name) const {
    CUDARTCFuncMap::const_iterator mm = rtc_func_map.find(function_name);
    if (mm == rtc_func_map.end()) {
        THROW exception::InvalidAgentFunc("Function name '%s' is not a runtime compiled agent function in agent '%s', "
            "in CUDAAgent::getRTCInstantiation()\n",
            function_name.c_str(), agent_description.name.c_str());
    }

    return *mm->second;
}
detail::curve::CurveRTCHost& CUDAAgent::getRTCHeader(const std::string& function_name) const {
    CUDARTCHeaderMap::const_iterator mm = rtc_header_map.find(function_name);
    if (mm == rtc_header_map.end()) {
        THROW exception::InvalidAgentFunc("Function name '%s' is not a runtime compiled agent function in agent '%s', "
            "in CUDAAgent::getRTCHeader()\n",
            function_name.c_str(), agent_description.name.c_str());
    }

    return *mm->second;
}

const CUDAAgent::CUDARTCFuncMap& CUDAAgent::getRTCFunctions() const {
    return rtc_func_map;
}

void CUDAAgent::initUnmappedVars(CUDAScatter &scatter, const unsigned int &streamId, const hipStream_t &stream) {
    for (auto &s : state_map) {
        s.second->initUnmappedVars(scatter, streamId, stream);
    }
}
void CUDAAgent::initExcludedVars(const std::string &state, const unsigned int&count, const unsigned int&offset, CUDAScatter& scatter, const unsigned int& streamId, const hipStream_t& stream) {
    // check the cuda agent state map to find the correct state list
    const auto& sm = state_map.find(state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found, "
            "in CUDAAgent::initUnmappedVars()",
            agent_description.name.c_str(), state.c_str());
    }
    sm->second->initExcludedVars(count, offset, scatter, streamId, stream);
}
void CUDAAgent::cullUnmappedStates() {
    unsigned int i = 0;
    for (auto &s : state_map) {
        if (!s.second->getIsSubStatelist()) {
            s.second->clear();
            ++i;
        }
    }
    if (i == state_map.size())
        fat_agent->resetIDCounter();
}
void CUDAAgent::cullAllStates() {
    for (auto &s : state_map) {
        s.second->clear();
    }
    fat_agent->resetIDCounter();
}
std::list<std::shared_ptr<VariableBuffer>> CUDAAgent::getUnboundVariableBuffers(const std::string& state) {
    const auto& sm = state_map.find(state);

    if (sm == state_map.end()) {
        THROW exception::InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found, "
            "in CUDAAgent::getUnboundVariableBuffers()",
            agent_description.name.c_str(), state.c_str());
    }
    return sm->second->getUnboundVariableBuffers();
}
id_t CUDAAgent::nextID(unsigned int count) {
    return fat_agent->nextID(count);
}
id_t* CUDAAgent::getDeviceNextID() {
    return fat_agent->getDeviceNextID();
}
void CUDAAgent::assignIDs(HostAPI& hostapi, CUDAScatter &scatter, hipStream_t stream, const unsigned int streamId) {
    fat_agent->assignIDs(hostapi, scatter, stream, streamId);
}

void CUDAAgent::setPopulationVec(const std::string& state_name, const std::shared_ptr<DeviceAgentVector_impl>& d_vec) {
    population_dvec[state_name] = d_vec;
}
std::shared_ptr<DeviceAgentVector_impl> CUDAAgent::getPopulationVec(const std::string& state_name) {
    auto find = population_dvec.find(state_name);
    if (find != population_dvec.end())
        return find->second;
    return nullptr;
}
void CUDAAgent::resetPopulationVecs() {
    for (auto &vec : population_dvec) {
        if (vec.second) {
            vec.second->syncChanges();
            vec.second.reset();
        }
    }
    population_dvec.clear();
}

}  // namespace flamegpu
