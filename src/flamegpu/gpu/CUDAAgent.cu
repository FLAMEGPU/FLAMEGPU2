/**
* @file CUDAAgent.cpp
* @authors Paul
* @date
* @brief
*
* @see
* @warning
*/

#include <hip/hip_runtime.h>
#include <>

#include "flamegpu/gpu/CUDAAgent.h"
#include "flamegpu/gpu/CUDAAgentStateList.h"
#include "flamegpu/gpu/CUDAErrorChecking.h"

#include "flamegpu/model/AgentDescription.h"
#include "flamegpu/pop/AgentPopulation.h"
#include "flamegpu/runtime/cuRVE/curve.h"

/**
* CUDAAgent class
* @brief allocates the hash table/list for agent variables and copy the list to device
*/
CUDAAgent::CUDAAgent(const AgentDescription& description) : agent_description(description), state_map(), max_list_size(0) {
}

/**
 * A destructor.
 * @brief Destroys the CUDAAgent object
 */
CUDAAgent::~CUDAAgent(void) {
}


/**
* @brief Returns agent description
* @param none
* @return AgentDescription object
*/
const AgentDescription& CUDAAgent::getAgentDescription() const {
    return agent_description;
}

/**
* @brief Sets initial population data by allocating memory for each state list by creating a new agent state list
* @param AgentPopulation object
* @return none
*/
void CUDAAgent::setInitialPopulationData(const AgentPopulation& population) {
    // check that the initial population data has not already been set
    if (!state_map.empty()) {
        THROW InvalidPopulationData("Error: Initial population data for agent '%s' already set. "
            "In CUDAAgent::setInitialPopulationData()",
            population.getAgentName().c_str());
    }
    // set the maximum population state size
    max_list_size = population.getMaximumStateListCapacity();

    // Make sure population uses same agent description as was used to initialise the agent CUDAAgent
    if (&(population.getAgentDescription()) != &agent_description) {
        THROW InvalidPopulationData("Error: Initial Population has a different agent description ('%s') "
            "to that which was used to initialise the CUDAAgent ('%s'). "
            "In CUDAAgent::setInitialPopulationData()",
            population.getAgentName().c_str(),
            agent_description.getName().c_str());
    }
    // create map of device state lists by traversing the state list
    const StateMap& sm = agent_description.getStateMap();
    for (const StateMapPair& s : sm) {
        // allocate memory for each state list by creating a new Agent State List
        state_map.insert(CUDAStateMap::value_type(s.first, std::unique_ptr<CUDAAgentStateList>( new CUDAAgentStateList(*this))));
    }

    /**set the population data*/
    setPopulationData(population);
}

/**
* @brief Sets the population data
* @param AgentPopulation object
* @return none
*/
void CUDAAgent::setPopulationData(const AgentPopulation& population) {
    // check that the gpu state lists have been initialised by a previous call to setInitialPopulationData
    if (state_map.empty()) {
        THROW InvalidPopulationData("Error: Initial population data for agent '%s' not allocated. "
            "Have you called setInitialPopulationData()? "
            "In CUDAAgent::setPopulationData()",
            population.getAgentName().c_str());
    }
    // check that the population maximums do not exceed the current maximum (as their will not be enough GPU memory to hold it)
    if (population.getMaximumStateListCapacity() > max_list_size) {
        THROW InvalidPopulationData("Error: Maximum population size for agent '%s' exceeds allocation. "
            "In CUDAAgent::setPopulationData()",
            population.getAgentName().c_str());
    }
    // Make sure population uses same agent description as was used to initialise the agent CUDAAgent
    const std::string agent_name = agent_description.getName();
    if (&(population.getAgentDescription()) != &agent_description) {
        THROW InvalidPopulationData("Error: Initial Population has a different agent description ('%s') "
            "to that which was used to initialise the CUDAAgent ('%s'). "
            "In CUDAAgent::setPopulationData()",
            population.getAgentName().c_str(),
            agent_description.getName().c_str());
    }
    /**set all population data to zero*/
    zeroAllStateVariableData();

    /**copy all population data to correct state map*/
    const StateMap& sm = agent_description.getStateMap();
    for (const StateMapPair& s : sm) {
        // get an associated CUDA statemap pair
        CUDAStateMap::iterator i = state_map.find(s.first);

        /**check that the CUDAAgentStateList was found (should ALWAYS be the case)*/
        if (i == state_map.end()) {
            THROW InvalidMapEntry("Error: failed to find memory allocated for agent ('%s') state ('%s') "
                "In CUDAAgent::setPopulationData() ",
                "This should never happen!",
                population.getAgentName().c_str(), s.first.c_str());
        }
        // copy the data from the population state memory to the state_maps CUDAAgentStateList
        i->second->setAgentData(population.getReadOnlyStateMemory(i->first));
    }
}

void CUDAAgent::getPopulationData(AgentPopulation& population) {
    // check that the gpu state lists have been initialised by a previous call to setInitialPopulationData
    if (state_map.empty()) {
        THROW InvalidPopulationData("Error: Initial population data for agent '%s' not allocated. "
            "Have you called getPopulationData()? "
            "In CUDAAgent::setPopulationData()",
            population.getAgentName().c_str());
    }
    // check that the population maximums do not exceed the current maximum (as their will not be enough GPU memory to hold it)
    if (population.getMaximumStateListCapacity() < max_list_size) {
        THROW InvalidPopulationData("Error: Maximum population size for agent '%s' exceeds allocation. "
            "In CUDAAgent::getPopulationData()",
            population.getAgentName().c_str());
    }
    // Make sure population uses same agent description as was used to initialise the agent CUDAAgent
    const std::string agent_name = agent_description.getName();
    if (&(population.getAgentDescription()) != &agent_description) {
        THROW InvalidPopulationData("Error: Initial Population has a different agent description ('%s') "
            "to that which was used to initialise the CUDAAgent ('%s'). "
            "In CUDAAgent::setPopulationData()",
            population.getAgentName().c_str(),
            agent_description.getName().c_str());
    }
    /* copy all population from correct state maps */
    const StateMap& sm = agent_description.getStateMap();
    for (const StateMapPair& s : sm) {
        // get an associated CUDA statemap pair
        CUDAStateMap::iterator i = state_map.find(s.first);

        /**check that the CUDAAgentStateList was found (should ALWAYS be the case)*/
        if (i == state_map.end()) {
            THROW InvalidMapEntry("Error: failed to find memory allocated for agent ('%s') state ('%s') "
                "In CUDAAgent::setPopulationData() ",
                "This should never happen!",
                population.getAgentName().c_str(), s.first.c_str());
        }
        // copy the data from the population state memory to the state_maps CUDAAgentStateList
        i->second->getAgentData(population.getStateMemory(i->first));
    }
}

/**
* @brief Returns the maximum list size
* @param none
* @return maximum size list that is equal to the maxmimum population size
*/
unsigned int CUDAAgent::getMaximumListSize() const {
    return max_list_size;
}

/**
* @brief Sets all state variable data to zero
* It loops through sate maps and resets the values
* @param none
* @return none
* @warning zeroAgentData
*/
void CUDAAgent::zeroAllStateVariableData() {
    // loop through state maps and reset the values
    for (CUDAStateMapPair& s : state_map) {
        s.second->zeroAgentData();
    }
}

// this is done for all the variables for now.
void CUDAAgent::mapRuntimeVariables(const AgentFunctionDescription& func) const {
    // check the cuda agent state map to find the correct state list for functions starting state
    CUDAStateMap::const_iterator sm = state_map.find(func.getInitialState());

    if (sm == state_map.end()) {
        THROW InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
            "in CUDAAgent::mapRuntimeVariables()",
            agent_description.getName().c_str(), func.getInitialState().c_str());
    }

    // loop through the agents variables to map each variable name using cuRVE
    for (MemoryMapPair mmp : agent_description.getMemoryMap()) {
        // get a device pointer for the agent variable name
        void* d_ptr = sm->second->getAgentListVariablePointer(mmp.first);

        // map using curve
        CurveVariableHash var_hash = curveVariableRuntimeHash(mmp.first.c_str());
        CurveVariableHash agent_hash = curveVariableRuntimeHash(func.getParent().getName().c_str());
        CurveVariableHash func_hash = curveVariableRuntimeHash(func.getName().c_str());

        // get the agent variable size
        size_t size;
        size = agent_description.getAgentVariableSize(mmp.first.c_str());

       // maximum population num
        unsigned int length = this->getMaximumListSize();

        curveRegisterVariableByHash(var_hash + agent_hash + func_hash, d_ptr, size, length);
    }
}

void CUDAAgent::unmapRuntimeVariables(const AgentFunctionDescription& func) const {
    // check the cuda agent state map to find the correct state list for functions starting state
    CUDAStateMap::const_iterator sm = state_map.find(func.getInitialState());

    if (sm == state_map.end()) {
        THROW InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
            "in CUDAAgent::unmapRuntimeVariables()",
            agent_description.getName().c_str(), func.getInitialState().c_str());
    }

    // loop through the agents variables to map each variable name using cuRVE
    for (MemoryMapPair mmp : agent_description.getMemoryMap()) {
        // get a device pointer for the agent variable name
        // void* d_ptr = sm->second->getAgentListVariablePointer(mmp.first);

        // unmap using curve
        CurveVariableHash var_hash = curveVariableRuntimeHash(mmp.first.c_str());
        CurveVariableHash agent_hash = curveVariableRuntimeHash(func.getParent().getName().c_str());
        CurveVariableHash func_hash = curveVariableRuntimeHash(func.getName().c_str());

        curveUnregisterVariableByHash(var_hash + agent_hash + func_hash);
    }
}


const std::unique_ptr<CUDAAgentStateList> &CUDAAgent::getAgentStateList(const std::string &state_name) const {
    // check the cuda agent state map to find the correct state list for functions starting state
    CUDAStateMap::const_iterator sm = state_map.find(state_name);

    if (sm == state_map.end()) {
        THROW InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
            "in CUDAAgent::getAgentStateList()",
            agent_description.getName().c_str(), state_name.c_str());
    }
    return sm->second;
}
