/**
* @file CUDAAgent.cpp
* @authors Paul
* @date
* @brief
*
* @see
* @warning
*/

#include <hip/hip_runtime.h>
#include <>

#include "flamegpu/gpu/CUDAAgent.h"
#include "flamegpu/gpu/CUDAAgentStateList.h"
#include "flamegpu/gpu/CUDAErrorChecking.h"

#include "flamegpu/model/AgentDescription.h"
#include "flamegpu/model/AgentFunctionDescription.h"
#include "flamegpu/pop/AgentPopulation.h"
#include "flamegpu/runtime/cuRVE/curve.h"
#include "flamegpu/gpu/CUDAScatter.h"

#ifdef _MSC_VER
#pragma warning(push, 3)
#include <hipcub/hipcub.hpp>
#pragma warning(pop)
#else
#include <hipcub/hipcub.hpp>
#endif

/**
* CUDAAgent class
* @brief allocates the hash table/list for agent variables and copy the list to device
*/
CUDAAgent::CUDAAgent(const AgentData& description)
    : agent_description(description)
    , state_map()
    , max_list_size(0) {
    // Regen new empty state_map
    for (const std::string &s : agent_description.states) {
        // allocate memory for each state list by creating a new Agent State List
        state_map.insert(CUDAStateMap::value_type(s, std::unique_ptr<CUDAAgentStateList>(new CUDAAgentStateList(*this))));
    }
}

/**
 * A destructor.
 * @brief Destroys the CUDAAgent object
 */
CUDAAgent::~CUDAAgent(void) {
}


/**
* @brief Returns agent description
* @param none
* @return AgentDescription object
*/
const AgentData& CUDAAgent::getAgentDescription() const {
    return agent_description;
}

void CUDAAgent::resize(const unsigned int &newSize, const unsigned int &streamId) {
    // Only grow currently
    max_list_size = max_list_size < 2 ? 2 : max_list_size;
    if (newSize > max_list_size) {
        while (max_list_size < newSize) {
            max_list_size = static_cast<unsigned int>(max_list_size * 1.5);
        }
        // Resize all items in the statemap
        for (auto &state : state_map) {
            state.second->resize();  // It auto pulls size from this->max_list_size
        }
    }
    // Notify scan flag that it might need resizing
    flamegpu_internal::CUDAScanCompaction::resize(max_list_size, flamegpu_internal::CUDAScanCompaction::AGENT_DEATH, streamId);
}
/**
* @brief Sets the population data
* @param AgentPopulation object
* @return none
*/
void CUDAAgent::setPopulationData(const AgentPopulation& population) {
    // check that the gpu state lists have been initialised by a previous call to setInitialPopulationData
    if (state_map.empty()) {
        // create map of device state lists by traversing the state list
        for (const std::string &s : agent_description.states) {
            // allocate memory for each state list by creating a new Agent State List
            state_map.insert(CUDAStateMap::value_type(s, std::unique_ptr<CUDAAgentStateList>(new CUDAAgentStateList(*this))));
        }
    }
    // Make sure population uses same agent description as was used to initialise the agent CUDAAgent
    const std::string agent_name = agent_description.name;
    if ((population.getAgentDescription()) != agent_description) {
        THROW InvalidPopulationData("Error: Initial Population has a different agent description ('%s') "
            "to that which was used to initialise the CUDAAgent ('%s'). "
            "In CUDAAgent::setPopulationData()",
            population.getAgentName().c_str(),
            agent_description.name.c_str());
    }
    // check that the population maximums do not exceed the current maximum (as their will not be enough GPU memory to hold it)
    if (population.getMaximumStateListCapacity() > max_list_size) {
        // Resize the population exactly, setPopData is a whole population movement, no need for greedy resize
        // Unlikely to add new agents during simulation

        // Update capacity
        max_list_size = population.getMaximumStateListCapacity();
        // Drop old state_map
        state_map.clear();
        // Regen new state_map
        for (const std::string &s : agent_description.states) {
            // allocate memory for each state list by creating a new Agent State List
            state_map.insert(CUDAStateMap::value_type(s, std::unique_ptr<CUDAAgentStateList>(new CUDAAgentStateList(*this))));
        }
    }
    /**set all population data to zero*/
    zeroAllStateVariableData();

    /**copy all population data to correct state map*/
    const std::set<std::string> &sm = agent_description.states;
    for (const std::string &s : sm) {
        // get an associated CUDA statemap pair
        CUDAStateMap::iterator i = state_map.find(s);

        /**check that the CUDAAgentStateList was found (should ALWAYS be the case)*/
        if (i == state_map.end()) {
            THROW InvalidMapEntry("Error: failed to find memory allocated for agent ('%s') state ('%s') "
                "In CUDAAgent::setPopulationData() ",
                "This should never happen!",
                population.getAgentName().c_str(), s.c_str());
        }
        // copy the data from the population state memory to the state_maps CUDAAgentStateList
        i->second->setAgentData(population.getReadOnlyStateMemory(i->first));
    }
}

void CUDAAgent::getPopulationData(AgentPopulation& population) {
    // check that the gpu state lists have been initialised by a previous call to setInitialPopulationData
    if (state_map.empty()) {
        THROW InvalidPopulationData("Error: Initial population data for agent '%s' not allocated. "
            "Have you called getPopulationData()? "
            "In CUDAAgent::setPopulationData()",
            population.getAgentName().c_str());
    }
    // Make sure population uses same agent description as was used to initialise the agent CUDAAgent
    const std::string agent_name = agent_description.name;
    if (population.getAgentDescription() != agent_description) {
        THROW InvalidPopulationData("Error: Initial Population has a different agent description ('%s') "
            "to that which was used to initialise the CUDAAgent ('%s'). "
            "In CUDAAgent::setPopulationData()",
            population.getAgentName().c_str(),
            agent_description.name.c_str());
    }
    // Resize population if it is too small
    if (population.getMaximumStateListCapacity() < getMaximumListSize())
        population.setStateListCapacity(getMaximumListSize());

    /* copy all population from correct state maps */
    const std::set<std::string> &sm = agent_description.states;
    for (const std::string &s : sm) {
        // get an associated CUDA statemap pair
        CUDAStateMap::iterator i = state_map.find(s);

        /**check that the CUDAAgentStateList was found (should ALWAYS be the case)*/
        if (i == state_map.end()) {
            THROW InvalidMapEntry("Error: failed to find memory allocated for agent ('%s') state ('%s') "
                "In CUDAAgent::setPopulationData() ",
                "This should never happen!",
                population.getAgentName().c_str(), s.c_str());
        }
        // check that the population maximums do not exceed the current maximum (as their will not be enough GPU memory to hold it)
        if (population.getMaximumStateListCapacity() < i->second->getCUDAStateListSize()) {
            // This should be redundant
            THROW InvalidPopulationData("Error: Maximum population size for agent '%s' exceeds allocation. "
                "In CUDAAgent::getPopulationData()",
                population.getAgentName().c_str());
        }
        // copy the data from the population state memory to the state_maps CUDAAgentStateList
        i->second->getAgentData(population.getStateMemory(i->first));
    }
}

/**
* @brief Returns the maximum list size
* @param none
* @return maximum size list that is equal to the maxmimum population size
*/
unsigned int CUDAAgent::getMaximumListSize() const {
    return max_list_size;
}

/**
* @brief Sets all state variable data to zero
* It loops through sate maps and resets the values
* @param none
* @return none
* @warning zeroAgentData
*/
void CUDAAgent::zeroAllStateVariableData() {
    // loop through state maps and reset the values
    for (CUDAStateMapPair& s : state_map) {
        s.second->zeroAgentData();
    }
}

// this is done for all the variables for now.
void CUDAAgent::mapRuntimeVariables(const AgentFunctionData& func, const std::string &state) const {
    // check the cuda agent state map to find the correct state list for functions starting state
    CUDAStateMap::const_iterator sm = state_map.find(state);

    if (sm == state_map.end()) {
        THROW InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
            "in CUDAAgent::mapRuntimeVariables()",
            agent_description.name.c_str(), func.initial_state.c_str());
    }

    const Curve::VariableHash agent_hash = Curve::getInstance().variableRuntimeHash(agent_description.name.c_str());
    const Curve::VariableHash func_hash = Curve::getInstance().variableRuntimeHash(func.name.c_str());
    // loop through the agents variables to map each variable name using cuRVE
    for (const auto &mmp : agent_description.variables) {
        // get a device pointer for the agent variable name
        void* d_ptr = sm->second->getAgentListVariablePointer(mmp.first);

        // map using curve
        const Curve::VariableHash var_hash = Curve::getInstance().variableRuntimeHash(mmp.first.c_str());

        // get the agent variable size
        size_t size = mmp.second.type_size;

       // maximum population num
        unsigned int length = this->getMaximumListSize();
        Curve::getInstance().registerVariableByHash(var_hash + agent_hash + func_hash, d_ptr, size, length);
    }
}

void CUDAAgent::unmapRuntimeVariables(const AgentFunctionData& func) const {
    // check the cuda agent state map to find the correct state list for functions starting state
    CUDAStateMap::const_iterator sm = state_map.find(func.initial_state);

    if (sm == state_map.end()) {
        THROW InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
            "in CUDAAgent::unmapRuntimeVariables()",
            agent_description.name.c_str(), func.initial_state.c_str());
    }

    const Curve::VariableHash agent_hash = Curve::getInstance().variableRuntimeHash(agent_description.name.c_str());
    const Curve::VariableHash func_hash = Curve::getInstance().variableRuntimeHash(func.name.c_str());
    // loop through the agents variables to map each variable name using cuRVE
    for (const auto &mmp : agent_description.variables) {
        // get a device pointer for the agent variable name
        // void* d_ptr = sm->second->getAgentListVariablePointer(mmp.first);

        // unmap using curve
        const Curve::VariableHash var_hash = Curve::getInstance().variableRuntimeHash(mmp.first.c_str());
        Curve::getInstance().unregisterVariableByHash(var_hash + agent_hash + func_hash);
    }
}

void CUDAAgent::processDeath(const AgentFunctionData& func, const unsigned int &streamId) {
    if (func.has_agent_death) {  // Optionally process agent death
        // check the cuda agent state map to find the correct state list for functions starting state
        CUDAStateMap::const_iterator sm = state_map.find(func.initial_state);

        unsigned int agent_count = sm->second->getCUDAStateListSize();
        // Resize cub (if required)
        if (agent_count > flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].cub_temp_size_max_list_size) {
            if (flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].hd_cub_temp) {
                gpuErrchk(hipFree(flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].hd_cub_temp));
            }
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].cub_temp_size = 0;
            hipcub::DeviceScan::ExclusiveSum(
                nullptr,
                flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].cub_temp_size,
                flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].d_ptrs.scan_flag,
                flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].d_ptrs.position,
                max_list_size + 1);
            gpuErrchk(hipMalloc(&flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].hd_cub_temp,
                flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].cub_temp_size));
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].cub_temp_size_max_list_size = max_list_size;
        }
        hipcub::DeviceScan::ExclusiveSum(
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].hd_cub_temp,
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].cub_temp_size,
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].d_ptrs.scan_flag,
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].d_ptrs.position,
            agent_count + 1);

        // Scatter
        sm->second->scatter(streamId, CUDAAgentStateList::ScatterMode::Death);
    }
}
const std::unique_ptr<CUDAAgentStateList> &CUDAAgent::getAgentStateList(const std::string &state_name) const {
    // check the cuda agent state map to find the correct state list for functions starting state
    CUDAStateMap::const_iterator sm = state_map.find(state_name);

    if (sm == state_map.end()) {
        THROW InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
            "in CUDAAgent::getAgentStateList()",
            agent_description.name.c_str(), state_name.c_str());
    }
    return sm->second;
}

void* CUDAAgent::getStateVariablePtr(const std::string& state_name, const std::string& variable_name) {
    return getAgentStateList(state_name)->getAgentListVariablePointer(variable_name);
}

ModelData::size_type CUDAAgent::getStateSize(const std::string& state_name) const {
    return getAgentStateList(state_name)->getCUDAStateListSize();
}

void CUDAAgent::processFunctionCondition(const AgentFunctionData& func, const unsigned int &streamId) {
    if (func.condition) {  // Optionally process agent death
        // check the cuda agent state map to find the correct state list for functions starting state
        CUDAStateMap::const_iterator sm = state_map.find(func.initial_state);

        unsigned int agent_count = sm->second->getCUDAStateListSize();
        // Resize cub (if required)
        if (agent_count > flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].cub_temp_size_max_list_size) {
            if (flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].hd_cub_temp) {
                gpuErrchk(hipFree(flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].hd_cub_temp));
            }
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].cub_temp_size = 0;
            hipcub::DeviceScan::ExclusiveSum(
                nullptr,
                flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].cub_temp_size,
                flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].d_ptrs.scan_flag,
                flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].d_ptrs.position,
                max_list_size + 1);
            gpuErrchk(hipMalloc(&flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].hd_cub_temp,
                flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].cub_temp_size));
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].cub_temp_size_max_list_size = max_list_size;
        }
        // Perform scan
        hipcub::DeviceScan::ExclusiveSum(
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].hd_cub_temp,
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].cub_temp_size,
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].d_ptrs.scan_flag,
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].d_ptrs.position,
            agent_count + 1);
        gpuErrchkLaunch();
        // Use scan results to sort false agents into start of list (and don't swap buffers)
        const unsigned int conditionFailCount = sm->second->scatter(streamId, 0, CUDAAgentStateList::FunctionCondition);
        // Invert scan
        CUDAScatter::InversionIterator ii = CUDAScatter::InversionIterator(flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].d_ptrs.scan_flag);
        hipMemset(flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].d_ptrs.position, 0, sizeof(unsigned int)*(agent_count + 1));
        hipcub::DeviceScan::ExclusiveSum(
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].hd_cub_temp,
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].cub_temp_size,
            ii,
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_DEATH][streamId].d_ptrs.position,
            agent_count + 1);
        gpuErrchkLaunch();
        // Use inverted scan results to sort true agents into end of list (and swap buffers)
        const unsigned int conditionpassCount = sm->second->scatter(streamId, conditionFailCount, CUDAAgentStateList::FunctionCondition2);
        assert(agent_count == conditionpassCount + conditionFailCount);
        // Set agent function condition state
        sm->second->setConditionState(conditionFailCount);
    }
}
void CUDAAgent::clearFunctionConditionState(const std::string &state) {
        // check the cuda agent state map to find the correct state list for functions starting state
        CUDAStateMap::const_iterator sm = state_map.find(state);
        sm->second->setConditionState(0);
}

void CUDAAgent::transitionState(const std::string &_src, const std::string &_dest, const unsigned int &streamId) {
    if (_src != _dest) {
        CUDAStateMap::const_iterator src = state_map.find(_src);
        CUDAStateMap::const_iterator dest = state_map.find(_dest);
        if (src == state_map.end()) {
            THROW InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
                "in CUDAAgent::transition_state()",
                agent_description.name.c_str(), _src.c_str());
        }
        if (dest == state_map.end()) {
            THROW InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
                "in CUDAAgent::transition_state()",
                agent_description.name.c_str(), _dest.c_str());
        }
        // If src list is empty we can skip
        if (src->second->getCUDATrueStateListSize() == 0)
            return;
        // If dest list is empty and we are not in an gent function condition, we can swap the lists
        if (dest->second->getCUDATrueStateListSize() == 0 && src->second->getCUDAStateListSize() == src->second->getCUDATrueStateListSize()) {
            swap(state_map.at(_src), state_map.at(_dest));
            assert(state_map.find(_src)->second->getCUDAStateListSize() == 0);
        } else {  // Otherwise we must perform a scatter all operation
            auto &cs = CUDAScatter::getInstance(streamId);
            cs.scatterAll(agent_description.variables, src->second->getReadList(), dest->second->getReadList(), src->second->getCUDAStateListSize(), dest->second->getCUDAStateListSize());
            // Update list sizes
            dest->second->setCUDAStateListSize(dest->second->getCUDAStateListSize() + src->second->getCUDAStateListSize());
            src->second->setCUDAStateListSize(0);
        }
    }
}
void CUDAAgent::mapNewRuntimeVariables(const AgentFunctionData& func, const unsigned int &maxLen) const {
    // check the cuda agent state map to find the correct state list for functions starting state
    CUDAStateMap::const_iterator sm = state_map.find(func.agent_output_state);

    if (sm == state_map.end()) {
        THROW InvalidCudaAgentState("Error: Agent ('%s') state ('%s') was not found "
            "in CUDAAgent::mapNewRuntimeVariables()",
            agent_description.name.c_str(), func.agent_output_state.c_str());
    }

    const Curve::VariableHash _agent_birth_hash = Curve::getInstance().variableRuntimeHash("_agent_birth");
    const Curve::VariableHash func_hash = Curve::getInstance().variableRuntimeHash(func.name.c_str());
    // loop through the agents variables to map each variable name using cuRVE
    for (const auto &mmp : agent_description.variables) {
        // get a device pointer for the agent variable name
        void* d_ptr = sm->second->getAgentNewListVariablePointer(mmp.first);

        // map using curve
        const Curve::VariableHash var_hash = Curve::getInstance().variableRuntimeHash(mmp.first.c_str());

        // get the agent variable size
        size_t size = mmp.second.type_size;

        // maximum population num
        Curve::getInstance().registerVariableByHash(var_hash + _agent_birth_hash + func_hash, d_ptr, size, maxLen);
    }
}

void CUDAAgent::unmapNewRuntimeVariables(const AgentFunctionData& func) const {
    const Curve::VariableHash _agent_birth_hash = Curve::getInstance().variableRuntimeHash("_agent_birth");
    const Curve::VariableHash func_hash = Curve::getInstance().variableRuntimeHash(func.name.c_str());
    // loop through the agents variables to map each variable name using cuRVE
    for (const auto &mmp : agent_description.variables) {
        // get a device pointer for the agent variable name
        // void* d_ptr = sm->second->getAgentListVariablePointer(mmp.first);

        // unmap using curve
        const Curve::VariableHash var_hash = Curve::getInstance().variableRuntimeHash(mmp.first.c_str());
        Curve::getInstance().unregisterVariableByHash(var_hash + _agent_birth_hash + func_hash);
    }
}
void CUDAAgent::resizeNew(const AgentFunctionData& func, const unsigned int &newSize, const unsigned int &streamId) {
    // Confirm agent output is set
    if (auto oa = func.agent_output.lock()) {
        // Resize new list of state_map
        auto sm = state_map.find(func.agent_output_state);
        if (sm != state_map.end()) {
            sm->second->resizeNewList(newSize);
        } else {
            THROW InvalidStateName("Agent '%s' does not contain state '%s', "
                "in CUDAAgent::resizeNew()\n",
                agent_description.name.c_str(), func.agent_output_state.c_str());
        }
        // Fill new list with default values
        sm->second->initNew(newSize, streamId);
        // Notify scan flag that it might need resizing
        // We need a 3rd array, because a function might combine agent birth, agent death and message output
        flamegpu_internal::CUDAScanCompaction::resize(newSize, flamegpu_internal::CUDAScanCompaction::AGENT_OUTPUT, streamId);
    }
}

void CUDAAgent::scatterNew(const std::string state, const unsigned int &newSize, const unsigned int &streamId) {
    auto sm = state_map.find(state);
    if (sm == state_map.end()) {
        THROW InvalidStateName("Agent '%s' does not contain state '%s', "
            "in CUDAAgent::scatterNew()\n",
            agent_description.name.c_str(), state.c_str());
    }
    // Perform scan
    if (newSize > flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_OUTPUT][streamId].cub_temp_size_max_list_size) {
        if (flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_OUTPUT][streamId].hd_cub_temp) {
            gpuErrchk(hipFree(flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_OUTPUT][streamId].hd_cub_temp));
        }
        flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_OUTPUT][streamId].cub_temp_size = 0;
        hipcub::DeviceScan::ExclusiveSum(
            nullptr,
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_OUTPUT][streamId].cub_temp_size,
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_OUTPUT][streamId].d_ptrs.scan_flag,
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_OUTPUT][streamId].d_ptrs.position,
            newSize + 1);
        gpuErrchk(hipMalloc(&flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_OUTPUT][streamId].hd_cub_temp,
            flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_OUTPUT][streamId].cub_temp_size));
        flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_OUTPUT][streamId].cub_temp_size_max_list_size = max_list_size;
    }
    hipcub::DeviceScan::ExclusiveSum(
        flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_OUTPUT][streamId].hd_cub_temp,
        flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_OUTPUT][streamId].cub_temp_size,
        flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_OUTPUT][streamId].d_ptrs.scan_flag,
        flamegpu_internal::CUDAScanCompaction::hd_configs[flamegpu_internal::CUDAScanCompaction::Type::AGENT_OUTPUT][streamId].d_ptrs.position,
        newSize + 1);
    // Resize d_list if necessary
    if (sm->second->getCUDATrueStateListSize() + newSize > max_list_size) {
        resize(sm->second->getCUDATrueStateListSize() + newSize, streamId);
    }
    // Scatter
    if (newSize)
        sm->second->scatterNew(newSize, streamId);
}
