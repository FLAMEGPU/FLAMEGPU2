#include <hip/hip_runtime.h>
#include <>

#include "flamegpu/gpu/CUDAMessageList.h"

#include "flamegpu/gpu/CUDAMessage.h"
#include "flamegpu/gpu/detail/CUDAErrorChecking.cuh"
#include "flamegpu/runtime/messaging/MessageBruteForce/MessageBruteForceHost.h"
#include "flamegpu/gpu/CUDAScatter.cuh"
#include "flamegpu/util/detail/cuda.cuh"

namespace flamegpu {

/**
* CUDAMessageList class
* @brief populates CUDA message map
*/
CUDAMessageList::CUDAMessageList(CUDAMessage& cuda_message, CUDAScatter &scatter, hipStream_t stream, unsigned int streamId)
    : message(cuda_message) {
    // allocate message lists
    allocateDeviceMessageList(d_list);
    allocateDeviceMessageList(d_swap_list);
    zeroDeviceMessageList_async(d_list, stream);
    zeroDeviceMessageList_async(d_swap_list, stream);
    gpuErrchk(hipStreamSynchronize(stream));
}

/**
 * A destructor.
 * @brief Destroys the CUDAMessageList object
 */
CUDAMessageList::~CUDAMessageList() {
    cleanupAllocatedData();
}

void CUDAMessageList::cleanupAllocatedData() {
    // clean up
    releaseDeviceMessageList(d_list);
    releaseDeviceMessageList(d_swap_list);
}

void CUDAMessageList::allocateDeviceMessageList(CUDAMessageMap &memory_map) {
    // we use the  messages memory map to iterate the  message variables and do allocation within our GPU hash map
    const auto &mem = message.getMessageData().variables;

    // for each variable allocate a device array and add to map
    for (const auto &mm : mem) {
        // get the variable name
        std::string var_name = mm.first;

        // get the variable size from  message description
        size_t var_size = mm.second.type_size * mm.second.elements;

        // do the device allocation
        void * d_ptr;

#ifdef UNIFIED_GPU_MEMORY
        // unified memory allocation
        gpuErrchk(hipMallocManaged(reinterpret_cast<void**>(&d_ptr), var_size *  message.getMaximumListSize()))
#else
        // non unified memory allocation
        gpuErrchk(hipMalloc(reinterpret_cast<void**>(&d_ptr), var_size * message.getMaximumListSize()));
#endif

        // store the pointer in the map
        memory_map.insert(CUDAMessageMap::value_type(var_name, d_ptr));
    }
}
void CUDAMessageList::resize(CUDAScatter& scatter, hipStream_t stream, unsigned int streamId, unsigned int keep_len) {
    // Release d_swap_list, we don't retain this data
    releaseDeviceMessageList(d_swap_list);
    // Allocate the new d_list
    CUDAMessageMap d_list_old;
    std::swap(d_list, d_list_old);
    allocateDeviceMessageList(d_list);
    if (keep_len && keep_len <= message.getMessageCount()) {
        // Copy data from d_list_old to d_list
        // Note, if keep_len exceeds length of d_swap_list_old, this will crash
        scatter.scatterAll(streamId,
            stream,
            message.getMessageData().variables,
            d_list_old, d_list,
            keep_len,
            0);
    }
    // Release d_list_old
    releaseDeviceMessageList(d_list_old);
    // Allocate the new d_swap_list
    allocateDeviceMessageList(d_swap_list);
    // Zero any new buffers with undefined data
    zeroDeviceMessageList_async(d_list, stream, keep_len);
    zeroDeviceMessageList_async(d_swap_list, stream);
    gpuErrchk(hipStreamSynchronize(stream));
}

void CUDAMessageList::releaseDeviceMessageList(CUDAMessageMap& memory_map) {
    // for each device pointer in the cuda memory map we need to free these
    for (const auto &mm : memory_map) {
        // free the memory on the device
        gpuErrchk(flamegpu::util::detail::cuda::hipFree(mm.second));
    }
    memory_map.clear();
}

void CUDAMessageList::zeroDeviceMessageList_async(CUDAMessageMap& memory_map, hipStream_t stream, unsigned int skip_offset) {
    if (skip_offset >= message.getMaximumListSize())
        return;
    // for each device pointer in the cuda memory map set the values to 0
    for (const auto &mm : memory_map) {
        // get the variable size from message description
        const auto var = message.getMessageData().variables.at(mm.first);
        const size_t var_size = var.type_size * var.elements;

        // set the memory to zero
        gpuErrchk(hipMemsetAsync(static_cast<char*>(mm.second) + (var_size * skip_offset), 0, var_size * (message.getMaximumListSize() - skip_offset), stream));
    }
}

void* CUDAMessageList::getReadMessageListVariablePointer(std::string variable_name) {
    CUDAMessageMap::iterator mm = d_list.find(variable_name);
    if (mm == d_list.end()) {
        THROW exception::InvalidMessageVar("Variable '%s' was not found in message '%s', "
          "in CUDAMessageList::getReadMessageListVariablePointer()",
          variable_name.c_str(), message.getMessageData().name.c_str());
    }

    return mm->second;
}
void* CUDAMessageList::getWriteMessageListVariablePointer(std::string variable_name) {
    CUDAMessageMap::iterator mm = d_swap_list.find(variable_name);
    if (mm == d_swap_list.end()) {
        THROW exception::InvalidMessageVar("Variable '%s' was not found in message '%s', "
            "in CUDAMessageList::getWriteMessageListVariablePointer()",
            variable_name.c_str(), message.getMessageData().name.c_str());
    }

    return mm->second;
}

void CUDAMessageList::zeroMessageData(hipStream_t stream) {
    zeroDeviceMessageList_async(d_list, stream);
    zeroDeviceMessageList_async(d_swap_list, stream);
    gpuErrchk(hipStreamSynchronize(stream));
}


void CUDAMessageList::swap() {
    std::swap(d_list, d_swap_list);
}

unsigned int CUDAMessageList::scatter(unsigned int newCount, CUDAScatter &scatter, hipStream_t stream, unsigned int streamId, bool append) {
    if (append) {
        unsigned int oldCount = message.getMessageCount();
        return oldCount + scatter.scatter(streamId,
            stream,
            CUDAScatter::Type::MESSAGE_OUTPUT,
            message.getMessageData().variables,
            d_swap_list, d_list,
            newCount,
            oldCount);
    } else {
        return scatter.scatter(streamId,
            stream,
            CUDAScatter::Type::MESSAGE_OUTPUT,
            message.getMessageData().variables,
            d_swap_list, d_list,
            newCount,
            0);
    }
}
unsigned int CUDAMessageList::scatterAll(unsigned int newCount, CUDAScatter &scatter, hipStream_t stream, unsigned int streamId) {
    unsigned int oldCount = message.getMessageCount();
    return oldCount + scatter.scatterAll(streamId,
        stream,
        message.getMessageData().variables,
        d_swap_list, d_list,
        newCount,
        oldCount);
}

}  // namespace flamegpu
