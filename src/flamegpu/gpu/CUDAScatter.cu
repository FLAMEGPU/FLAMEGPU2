#include "hip/hip_runtime.h"
#include "flamegpu/gpu/CUDAScatter.h"

#include <hip/hip_runtime.h>
#include <vector>
#include <cassert>

#include "flamegpu/gpu/CUDAErrorChecking.h"
#include "flamegpu/gpu/CUDAScanCompaction.h"
#include "flamegpu/runtime/flamegpu_host_new_agent_api.h"

unsigned int CUDAScatter::simulationInstances = 0;

CUDAScatter::CUDAScatter()
    : d_data(nullptr)
    , data_len(0) {
}
CUDAScatter::~CUDAScatter() {
    free();
}
void CUDAScatter::free() {
    if (d_data) {
        gpuErrchk(hipFree(d_data));
    }
    d_data = nullptr;
    data_len = 0;
}

void CUDAScatter::resize(const unsigned int &newLen) {
    if (newLen > data_len) {
        if (d_data) {
            gpuErrchk(hipFree(d_data));
        }
        gpuErrchk(hipMalloc(&d_data, newLen * sizeof(ScatterData)));
        data_len = newLen;
    }
}

template <typename T>
__global__ void scatter_generic(
    unsigned int threadCount,
    T scan_flag,
    unsigned int *position,
    CUDAScatter::ScatterData *scatter_data,
    const unsigned int scatter_len,
    const unsigned int out_index_offset = 0) {
    // global thread index
    int index = (blockIdx.x*blockDim.x) + threadIdx.x;

    if (index >= threadCount) return;

    // if optional message is to be written
    if (scan_flag[index] == 1) {
        int output_index = position[index];
        for (unsigned int i = 0; i < scatter_len; ++i) {
            memcpy(scatter_data[i].out + ((out_index_offset + output_index) * scatter_data[i].typeLen), scatter_data[i].in + (index * scatter_data[i].typeLen), scatter_data[i].typeLen);
        }
    }
}
__global__ void scatter_all_generic(
    unsigned int threadCount,
    CUDAScatter::ScatterData *scatter_data,
    const unsigned int scatter_len,
    const unsigned int out_index_offset = 0) {
    // global thread index
    int index = (blockIdx.x*blockDim.x) + threadIdx.x;

    if (index >= threadCount) return;
    for (unsigned int i = 0; i < scatter_len; ++i) {
        memcpy(scatter_data[i].out + ((out_index_offset + index) * scatter_data[i].typeLen), scatter_data[i].in + (index * scatter_data[i].typeLen), scatter_data[i].typeLen);
    }
}

unsigned int CUDAScatter::scatter(
    Type messageOrAgent,
    const VariableMap &vars,
    const std::map<std::string, void*> &in,
    const std::map<std::string, void*> &out,
    const unsigned int &itemCount,
    const unsigned int &out_index_offset,
    const bool &invert_scan_flag) {
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    // calculate the grid block size for main agent function
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, scatter_generic<unsigned int*>, 0, itemCount);
    //! Round up according to CUDAAgent state list size
    gridSize = (itemCount + blockSize - 1) / blockSize;
    // for each variable, scatter from swap to regular
    std::vector<ScatterData> sd;
    for (const auto &v : vars) {
        char *in_p = reinterpret_cast<char*>(in.at(v.first));
        char *out_p = reinterpret_cast<char*>(out.at(v.first));
        sd.push_back({ v.second.type_size, in_p, out_p });
    }
    resize(static_cast<unsigned int>(sd.size()));
    // Important that sd.size() is still used here, incase allocated len (data_len) is bigger
    gpuErrchk(hipMemcpy(d_data, sd.data(), sizeof(ScatterData) * sd.size(), hipMemcpyHostToDevice));
    if (invert_scan_flag) {
        scatter_generic << <gridSize, blockSize >> > (
            itemCount,
            InversionIterator(flamegpu_internal::CUDAScanCompaction::hd_configs[messageOrAgent][streamId].d_ptrs.scan_flag),
            flamegpu_internal::CUDAScanCompaction::hd_configs[messageOrAgent][streamId].d_ptrs.position,
            d_data, static_cast<unsigned int>(sd.size()),
            out_index_offset);
    } else {
        scatter_generic << <gridSize, blockSize >> > (
            itemCount,
            flamegpu_internal::CUDAScanCompaction::hd_configs[messageOrAgent][streamId].d_ptrs.scan_flag,
            flamegpu_internal::CUDAScanCompaction::hd_configs[messageOrAgent][streamId].d_ptrs.position,
            d_data, static_cast<unsigned int>(sd.size()),
            out_index_offset);
    }
    gpuErrchkLaunch();
    // Update count of live agents
    unsigned int rtn = 0;
    gpuErrchk(hipMemcpy(&rtn, flamegpu_internal::CUDAScanCompaction::hd_configs[messageOrAgent][streamId].d_ptrs.position + itemCount, sizeof(unsigned int), hipMemcpyDeviceToHost));
    return rtn;
}

unsigned int CUDAScatter::scatterAll(
    const VariableMap &vars,
    const std::map<std::string, void*> &in,
    const std::map<std::string, void*> &out,
    const unsigned int &itemCount,
    const unsigned int &out_index_offset) {
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size

                       // calculate the grid block size for main agent function
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, scatter_all_generic, 0, itemCount);
    //! Round up according to CUDAAgent state list size
    gridSize = (itemCount + blockSize - 1) / blockSize;
    // for each variable, scatter from swap to regular
    std::vector<ScatterData> sd;
    for (const auto &v : vars) {
        char *in_p = reinterpret_cast<char*>(in.at(v.first));
        char *out_p = reinterpret_cast<char*>(out.at(v.first));
        sd.push_back({ v.second.type_size, in_p, out_p });
    }
    resize(static_cast<unsigned int>(sd.size()));
    // Important that sd.size() is still used here, incase allocated len (data_len) is bigger
    gpuErrchk(hipMemcpy(d_data, sd.data(), sizeof(ScatterData) * sd.size(), hipMemcpyHostToDevice));
    scatter_all_generic << <gridSize, blockSize >> > (
        itemCount,
        d_data, static_cast<unsigned int>(sd.size()),
        out_index_offset);
    gpuErrchkLaunch();
    // Update count of live agents
    return itemCount;
}

__global__ void pbm_reorder_generic(
    const unsigned int threadCount,
    const unsigned int * __restrict__ bin_index,
    const unsigned int * __restrict__ bin_sub_index,
    const unsigned int * __restrict__ pbm,
    CUDAScatter::ScatterData *scatter_data,
    const unsigned int scatter_len) {
    // global thread index
    int index = (blockIdx.x*blockDim.x) + threadIdx.x;

    if (index >= threadCount) return;

    const unsigned int sorted_index = pbm[bin_index[index]] + bin_sub_index[index];

    // if optional message is to be written
    for (unsigned int i = 0; i < scatter_len; ++i) {
        memcpy(scatter_data[i].out + (sorted_index * scatter_data[i].typeLen), scatter_data[i].in + (index * scatter_data[i].typeLen), scatter_data[i].typeLen);
    }
}

void CUDAScatter::pbm_reorder(
    const VariableMap &vars,
    const std::map<std::string, void*> &in,
    const std::map<std::string, void*> &out,
    const unsigned int &itemCount,
    const unsigned int *d_bin_index,
    const unsigned int *d_bin_sub_index,
    const unsigned int *d_pbm) {
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size

                       // calculate the grid block size for main agent function
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, pbm_reorder_generic, 0, itemCount);
    //! Round up according to CUDAAgent state list size
    gridSize = (itemCount + blockSize - 1) / blockSize;
    // for each variable, scatter from swap to regular
    std::vector<ScatterData> sd;
    for (const auto &v : vars) {
        char *in_p = reinterpret_cast<char*>(in.at(v.first));
        char *out_p = reinterpret_cast<char*>(out.at(v.first));
        sd.push_back({ v.second.type_size, in_p, out_p });
    }
    resize(static_cast<unsigned int>(sd.size()));
    // Important that sd.size() is still used here, incase allocated len (data_len) is bigger
    gpuErrchk(hipMemcpy(d_data, sd.data(), sizeof(ScatterData) * sd.size(), hipMemcpyHostToDevice));
    pbm_reorder_generic <<<gridSize, blockSize>>> (
            itemCount,
            d_bin_index,
            d_bin_sub_index,
            d_pbm,
            d_data, static_cast<unsigned int>(sd.size()));
    gpuErrchkLaunch();
}

/**
 * Scatter kernel for host agent creation
 * Input data is stored in AoS, and translated to SoA for device
 * @param threadCount Total number of threads required
 * @param agent_size The total size of an agent's variables in memory, for stepping through input array
 * @param scatter_data Scatter data array location in memory
 * @param scatter_len Length of scatter data array
 * @parma out_index_offset The number of agents already in the output array (so that they are not overwritten)
 */
__global__ void scatter_new_agents(
    unsigned int threadCount,
    const unsigned int agent_size,
    CUDAScatter::ScatterData *scatter_data,
    const unsigned int scatter_len,
    const unsigned int out_index_offset) {
    // global thread index
    int index = (blockIdx.x*blockDim.x) + threadIdx.x;

    if (index >= threadCount) return;

    // Which variable are we outputting
    const unsigned int var_out = index % scatter_len;
    const unsigned int agent_index = index / scatter_len;

    // if optional message is to be written
    char * const in_ptr = scatter_data[var_out].in + (agent_index * agent_size);
    char * const out_ptr = scatter_data[var_out].out + ((out_index_offset + agent_index) * scatter_data[var_out].typeLen);
    memcpy(out_ptr, in_ptr, scatter_data[var_out].typeLen);
}
void CUDAScatter::scatterNewAgents(
    const VariableMap &vars,
    const std::map<std::string, void*> &out,
    void *d_in_buff,
    const VarOffsetStruct &inOffsetData,
    const unsigned int &inCount,
    const unsigned int outIndexOffset) {
    // 1 thread per agent variable
    const unsigned int threadCount = static_cast<unsigned int>(inOffsetData.vars.size()) * inCount;
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size

    // calculate the grid block size for main agent function
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, scatter_new_agents, 0, threadCount);
    //! Round up according to CUDAAgent state list size
    gridSize = (threadCount + blockSize - 1) / blockSize;
    // for each variable, scatter from swap to regular
    std::vector<ScatterData> sd;
    for (const auto &v : vars) {
        // In this case, in is the location of first variable, but we step by inOffsetData.totalSize
        char *in_p = reinterpret_cast<char*>(d_in_buff) + inOffsetData.vars.at(v.first).offset;
        char *out_p = reinterpret_cast<char*>(out.at(v.first));
        sd.push_back({ v.second.type_size, in_p, out_p });
    }
    resize(static_cast<unsigned int>(sd.size()));
    // Important that sd.size() is still used here, incase allocated len (data_len) is bigger
    gpuErrchk(hipMemcpy(d_data, sd.data(), sizeof(ScatterData) * sd.size(), hipMemcpyHostToDevice));
    scatter_new_agents << <gridSize, blockSize >> > (
        threadCount,
        static_cast<unsigned int>(inOffsetData.totalSize),
        d_data, static_cast<unsigned int>(sd.size()),
        outIndexOffset);
    gpuErrchkLaunch();
}
void CUDAScatter::increaseSimCounter() {
    simulationInstances++;
}
void CUDAScatter::decreaseSimCounter() {
    simulationInstances--;
    if (simulationInstances == 0) {
        for (unsigned int i = 0; i < flamegpu_internal::CUDAScanCompaction::MAX_STREAMS; ++i) {
            getInstance(i).free();
        }
    }
}
