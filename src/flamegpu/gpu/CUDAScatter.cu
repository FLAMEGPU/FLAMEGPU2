#include "hip/hip_runtime.h"
#include "flamegpu/gpu/CUDAScatter.h"

#include <hip/hip_runtime.h>
#include <vector>
#include <cassert>

#include "flamegpu/gpu/CUDAErrorChecking.h"
#include "flamegpu/gpu/CUDAScanCompaction.h"
#include "flamegpu/runtime/flamegpu_host_new_agent_api.h"

#ifdef _MSC_VER
#pragma warning(push, 3)
#include <hipcub/hipcub.hpp>
#pragma warning(pop)
#else
#include <hipcub/hipcub.hpp>
#endif

unsigned int CUDAScatter::simulationInstances = 0;

CUDAScatter::CUDAScatter()
    : d_data(nullptr)
    , data_len(0) {
}
CUDAScatter::~CUDAScatter() {
    /* @note - Do not clear cuda memory in the destructor of singletons.
     This is because order of static destruction in c++ is undefined
     So the cuda driver is not guaranteed to still exist when the static is destroyed.
     As this is only ever destroyed at exit time, it's not a real memory leak either.
    */
    // free();
}
void CUDAScatter::free() {
    if (d_data) {
        gpuErrchk(hipFree(d_data));
    }
    d_data = nullptr;
    data_len = 0;
}

void CUDAScatter::purge() {
    d_data = nullptr;
    data_len = 0;
}

void CUDAScatter::resize(const unsigned int &newLen) {
    if (newLen > data_len) {
        if (d_data) {
            gpuErrchk(hipFree(d_data));
        }
        gpuErrchk(hipMalloc(&d_data, newLen * sizeof(ScatterData)));
        data_len = newLen;
    }
}

template <typename T>
__global__ void scatter_generic(
    unsigned int threadCount,
    T scan_flag,
    unsigned int *position,
    CUDAScatter::ScatterData *scatter_data,
    const unsigned int scatter_len,
    const unsigned int out_index_offset = 0,
    const unsigned int scatter_all_count = 0) {
    // global thread index
    int index = (blockIdx.x*blockDim.x) + threadIdx.x;

    if (index >= threadCount) return;

    // if optional message is to be written
    if (index < scatter_all_count || scan_flag[index - scatter_all_count] == 1) {
        int output_index = index < scatter_all_count ? index : scatter_all_count + position[index - scatter_all_count];
        for (unsigned int i = 0; i < scatter_len; ++i) {
            memcpy(scatter_data[i].out + ((out_index_offset + output_index) * scatter_data[i].typeLen), scatter_data[i].in + (index * scatter_data[i].typeLen), scatter_data[i].typeLen);
        }
    }
}
__global__ void scatter_all_generic(
    unsigned int threadCount,
    CUDAScatter::ScatterData *scatter_data,
    const unsigned int scatter_len,
    const unsigned int out_index_offset = 0) {
    // global thread index
    int index = (blockIdx.x*blockDim.x) + threadIdx.x;

    if (index >= threadCount) return;
    for (unsigned int i = 0; i < scatter_len; ++i) {
        memcpy(scatter_data[i].out + ((out_index_offset + index) * scatter_data[i].typeLen), scatter_data[i].in + (index * scatter_data[i].typeLen), scatter_data[i].typeLen);
    }
}

unsigned int CUDAScatter::scatter(
    Type messageOrAgent,
    const VariableMap &vars,
    const std::map<std::string, void*> &in,
    const std::map<std::string, void*> &out,
    const unsigned int &itemCount,
    const unsigned int &out_index_offset,
    const bool &invert_scan_flag,
    const unsigned int &scatter_all_count) {
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
    // calculate the grid block size for main agent function
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, scatter_generic<unsigned int*>, 0, itemCount));
    //! Round up according to CUDAAgent state list size
    gridSize = (itemCount + blockSize - 1) / blockSize;
    // for each variable, scatter from swap to regular
    std::vector<ScatterData> sd;
    for (const auto &v : vars) {
        char *in_p = reinterpret_cast<char*>(in.at(v.first));
        char *out_p = reinterpret_cast<char*>(out.at(v.first));
        sd.push_back({ v.second.type_size * v.second.elements, in_p, out_p });
    }
    resize(static_cast<unsigned int>(sd.size()));
    // Important that sd.size() is still used here, incase allocated len (data_len) is bigger
    gpuErrchk(hipMemcpy(d_data, sd.data(), sizeof(ScatterData) * sd.size(), hipMemcpyHostToDevice));
    if (invert_scan_flag) {
        scatter_generic << <gridSize, blockSize >> > (
            itemCount,
            InversionIterator(flamegpu_internal::CUDAScanCompaction::hd_configs[messageOrAgent][streamId].d_ptrs.scan_flag),
            flamegpu_internal::CUDAScanCompaction::hd_configs[messageOrAgent][streamId].d_ptrs.position,
            d_data, static_cast<unsigned int>(sd.size()),
            out_index_offset, scatter_all_count);
    } else {
        scatter_generic << <gridSize, blockSize >> > (
            itemCount,
            flamegpu_internal::CUDAScanCompaction::hd_configs[messageOrAgent][streamId].d_ptrs.scan_flag,
            flamegpu_internal::CUDAScanCompaction::hd_configs[messageOrAgent][streamId].d_ptrs.position,
            d_data, static_cast<unsigned int>(sd.size()),
            out_index_offset, scatter_all_count);
    }
    gpuErrchkLaunch();
    // Update count of live agents
    unsigned int rtn = 0;
    gpuErrchk(hipMemcpy(&rtn, flamegpu_internal::CUDAScanCompaction::hd_configs[messageOrAgent][streamId].d_ptrs.position + itemCount - scatter_all_count, sizeof(unsigned int), hipMemcpyDeviceToHost));
    return rtn + scatter_all_count;
}

unsigned int CUDAScatter::scatterAll(
    const VariableMap &vars,
    const std::map<std::string, void*> &in,
    const std::map<std::string, void*> &out,
    const unsigned int &itemCount,
    const unsigned int &out_index_offset) {
    if (!itemCount)
        return itemCount;  // No work to do
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size

                       // calculate the grid block size for main agent function
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, scatter_all_generic, 0, itemCount));
    //! Round up according to CUDAAgent state list size
    gridSize = (itemCount + blockSize - 1) / blockSize;
    // for each variable, scatter from swap to regular
    std::vector<ScatterData> sd;
    for (const auto &v : vars) {
        char *in_p = reinterpret_cast<char*>(in.at(v.first));
        char *out_p = reinterpret_cast<char*>(out.at(v.first));
        sd.push_back({ v.second.type_size * v.second.elements, in_p, out_p });
    }
    resize(static_cast<unsigned int>(sd.size()));
    // Important that sd.size() is still used here, incase allocated len (data_len) is bigger
    gpuErrchk(hipMemcpy(d_data, sd.data(), sizeof(ScatterData) * sd.size(), hipMemcpyHostToDevice));
    scatter_all_generic << <gridSize, blockSize >> > (
        itemCount,
        d_data, static_cast<unsigned int>(sd.size()),
        out_index_offset);
    gpuErrchkLaunch();
    // Update count of live agents
    return itemCount;
}

__global__ void pbm_reorder_generic(
    const unsigned int threadCount,
    const unsigned int * __restrict__ bin_index,
    const unsigned int * __restrict__ bin_sub_index,
    const unsigned int * __restrict__ pbm,
    CUDAScatter::ScatterData *scatter_data,
    const unsigned int scatter_len) {
    // global thread index
    int index = (blockIdx.x*blockDim.x) + threadIdx.x;

    if (index >= threadCount) return;

    const unsigned int sorted_index = pbm[bin_index[index]] + bin_sub_index[index];

    // if optional message is to be written
    for (unsigned int i = 0; i < scatter_len; ++i) {
        memcpy(scatter_data[i].out + (sorted_index * scatter_data[i].typeLen), scatter_data[i].in + (index * scatter_data[i].typeLen), scatter_data[i].typeLen);
    }
}

void CUDAScatter::pbm_reorder(
    const VariableMap &vars,
    const std::map<std::string, void*> &in,
    const std::map<std::string, void*> &out,
    const unsigned int &itemCount,
    const unsigned int *d_bin_index,
    const unsigned int *d_bin_sub_index,
    const unsigned int *d_pbm) {
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size

                       // calculate the grid block size for main agent function
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, pbm_reorder_generic, 0, itemCount));
    //! Round up according to CUDAAgent state list size
    gridSize = (itemCount + blockSize - 1) / blockSize;
    // for each variable, scatter from swap to regular
    std::vector<ScatterData> sd;
    for (const auto &v : vars) {
        char *in_p = reinterpret_cast<char*>(in.at(v.first));
        char *out_p = reinterpret_cast<char*>(out.at(v.first));
        sd.push_back({ v.second.type_size * v.second.elements, in_p, out_p });
    }
    resize(static_cast<unsigned int>(sd.size()));
    // Important that sd.size() is still used here, incase allocated len (data_len) is bigger
    gpuErrchk(hipMemcpy(d_data, sd.data(), sizeof(ScatterData) * sd.size(), hipMemcpyHostToDevice));
    pbm_reorder_generic <<<gridSize, blockSize>>> (
            itemCount,
            d_bin_index,
            d_bin_sub_index,
            d_pbm,
            d_data, static_cast<unsigned int>(sd.size()));
    gpuErrchkLaunch();
}

/**
 * Scatter kernel for host agent creation
 * Input data is stored in AoS, and translated to SoA for device
 * @param threadCount Total number of threads required
 * @param agent_size The total size of an agent's variables in memory, for stepping through input array
 * @param scatter_data Scatter data array location in memory
 * @param scatter_len Length of scatter data array
 * @param out_index_offset The number of agents already in the output array (so that they are not overwritten)
 */
__global__ void scatter_new_agents(
    const unsigned int threadCount,
    const unsigned int agent_size,
    CUDAScatter::ScatterData *scatter_data,
    const unsigned int scatter_len,
    const unsigned int out_index_offset) {
    // global thread index
    int index = (blockIdx.x*blockDim.x) + threadIdx.x;

    if (index >= threadCount) return;

    // Which variable are we outputting
    const unsigned int var_out = index % scatter_len;
    const unsigned int agent_index = index / scatter_len;

    // if optional message is to be written
    char * const in_ptr = scatter_data[var_out].in + (agent_index * agent_size);
    char * const out_ptr = scatter_data[var_out].out + ((out_index_offset + agent_index) * scatter_data[var_out].typeLen);
    memcpy(out_ptr, in_ptr, scatter_data[var_out].typeLen);
}
void CUDAScatter::scatterNewAgents(
    const VariableMap &vars,
    const std::map<std::string, void*> &out,
    void *d_in_buff,
    const VarOffsetStruct &inOffsetData,
    const unsigned int &inCount,
    const unsigned int outIndexOffset) {
    // 1 thread per agent variable
    const unsigned int threadCount = static_cast<unsigned int>(inOffsetData.vars.size()) * inCount;
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size

    // calculate the grid block size for main agent function
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, scatter_new_agents, 0, threadCount));
    //! Round up according to CUDAAgent state list size
    gridSize = (threadCount + blockSize - 1) / blockSize;
    // for each variable, scatter from swap to regular
    std::vector<ScatterData> sd;
    for (const auto &v : vars) {
        // In this case, in is the location of first variable, but we step by inOffsetData.totalSize
        char *in_p = reinterpret_cast<char*>(d_in_buff) + inOffsetData.vars.at(v.first).offset;
        char *out_p = reinterpret_cast<char*>(out.at(v.first));
        sd.push_back({ v.second.type_size * v.second.elements, in_p, out_p });
    }
    resize(static_cast<unsigned int>(sd.size()));
    // Important that sd.size() is still used here, incase allocated len (data_len) is bigger
    gpuErrchk(hipMemcpy(d_data, sd.data(), sizeof(ScatterData) * sd.size(), hipMemcpyHostToDevice));
    scatter_new_agents << <gridSize, blockSize >> > (
        threadCount,
        static_cast<unsigned int>(inOffsetData.totalSize),
        d_data, static_cast<unsigned int>(sd.size()),
        outIndexOffset);
    gpuErrchkLaunch();
}
/**
* Broadcast kernel for initialising agent variables to default on device
* Input data is stored pointed directly do by scatter_data and translated to SoA for device
* @param threadCount Total number of threads required
* @param scatter_data Scatter data array location in memory
* @param scatter_len Length of scatter data array
* @param out_index_offset The number of agents already in the output array (so that they are not overwritten)
*/
__global__ void broadcastInitKernel(
    const unsigned int threadCount,
    CUDAScatter::ScatterData *scatter_data,
    const unsigned int scatter_len,
    const unsigned int out_index_offset) {
    // global thread index
    int index = (blockIdx.x*blockDim.x) + threadIdx.x;

    if (index >= threadCount) return;

    // Which variable are we outputting
    const unsigned int var_out = index % scatter_len;
    const unsigned int agent_index = index / scatter_len;
    const unsigned int type_len = scatter_data[var_out].typeLen;
    // if optional message is to be written
    char * const in_ptr = scatter_data[var_out].in;
    char * const out_ptr = scatter_data[var_out].out + ((out_index_offset + agent_index) * type_len);
    memcpy(out_ptr, in_ptr, type_len);
}
void CUDAScatter::broadcastInit(
    const VariableMap &vars,
    const std::map<std::string, void*> &out,
    const unsigned int &inCount,
    const unsigned int outIndexOffset) {
    // 1 thread per agent variable
    const unsigned int threadCount = static_cast<unsigned int>(vars.size()) * inCount;
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size

    // calculate the grid block size for main agent function
    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, broadcastInitKernel, 0, threadCount));
    //! Round up according to CUDAAgent state list size
    gridSize = (threadCount + blockSize - 1) / blockSize;
    // Calculate memory usage (crudely in multiples of ScatterData)
    std::vector<ScatterData> sd;
    ptrdiff_t offset = 0;
    for (const auto &v : vars) {
        offset += v.second.type_size * v.second.elements;
    }
    resize(static_cast<unsigned int>(vars.size() + (offset /sizeof(ScatterData)) + sizeof(ScatterData)));
    // Build scatter data structure
    offset = 0;
    for (const auto &v : vars) {
        // In this case, in is the location of first variable, but we step by inOffsetData.totalSize
        char *in_p = reinterpret_cast<char*>(d_data) + offset;
        offset += v.second.type_size * v.second.elements;
        char *out_p = reinterpret_cast<char*>(out.at(v.first));
        sd.push_back({ v.second.type_size * v.second.elements, in_p, out_p });
    }
    // Build init data
    char *default_data = reinterpret_cast<char*>(malloc(offset));
    offset = 0;
    for (const auto &v : vars) {
        memcpy(default_data + offset, v.second.default_value, v.second.type_size * v.second.elements);
        offset += v.second.type_size * v.second.elements;
    }
    // Important that sd.size() is still used here, incase allocated len (data_len) is bigger
    gpuErrchk(hipMemcpy(d_data, default_data, offset, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_data + offset, sd.data(), sizeof(ScatterData) * sd.size(), hipMemcpyHostToDevice));
    ::free(default_data);
    broadcastInitKernel <<<gridSize, blockSize>>> (
        threadCount,
        d_data + offset, static_cast<unsigned int>(sd.size()),
        outIndexOffset);
    gpuErrchkLaunch();
}

__global__ void reorder_array_messages(
    const unsigned int threadCount,
    const unsigned int array_length,
    const unsigned int *d_position,
    unsigned int *d_write_flag,
    CUDAScatter::ScatterData *scatter_data,
    const unsigned int scatter_len
) {
    // global thread index
    int index = (blockIdx.x*blockDim.x) + threadIdx.x;

    if (index >= threadCount) return;

    const unsigned int output_index = d_position[index];
    assert(output_index < array_length);  // This or fail silently

    for (unsigned int i = 0; i < scatter_len; ++i) {
        memcpy(scatter_data[i].out + (output_index * scatter_data[i].typeLen), scatter_data[i].in + (index * scatter_data[i].typeLen), scatter_data[i].typeLen);
    }
    // Set err check flag
    atomicInc(d_write_flag + output_index, UINT_MAX);
}
void CUDAScatter::arrayMessageReorder(
    const VariableMap &vars,
    const std::map<std::string, void*> &in,
    const std::map<std::string, void*> &out,
    const unsigned int &itemCount,
    const unsigned int &array_length,
    unsigned int *d_write_flag) {
    if (itemCount > array_length) {
        THROW ArrayMessageWriteConflict("Too many messages output for array message structure (%u > %u).\n", itemCount, array_length);
    }
    int blockSize = 0;  // The launch configurator returned block size
    int minGridSize = 0;  // The minimum grid size needed to achieve the // maximum occupancy for a full device // launch
    int gridSize = 0;  // The actual grid size needed, based on input size
                       // calculate the grid block size for main agent function
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, reorder_array_messages, 0, itemCount);
    //! Round up according to CUDAAgent state list size
    gridSize = (itemCount + blockSize - 1) / blockSize;
    unsigned int *d_position = nullptr;
    // Build AoS -> AoS list
    std::vector<ScatterData> sd;
    for (const auto &v : vars) {
        if (v.first != "___INDEX") {
            char *in_p = reinterpret_cast<char*>(in.at(v.first));
            char *out_p = reinterpret_cast<char*>(out.at(v.first));
            sd.push_back({ v.second.type_size * v.second.elements, in_p, out_p });
        } else {  // Special case, log index var
            d_position = reinterpret_cast<unsigned int*>(in.at(v.first));
            d_write_flag = d_write_flag ? d_write_flag : reinterpret_cast<unsigned int*>(out.at(v.first));
        }
    }
    assert(d_position);  // Not an array message, lacking ___INDEX var
    resize(static_cast<unsigned int>(sd.size()));
    // Important that sd.size() is still used here, incase allocated len (data_len) is bigger
    gpuErrchk(hipMemcpy(d_data, sd.data(), sizeof(ScatterData) * sd.size(), hipMemcpyHostToDevice));
    reorder_array_messages << <gridSize, blockSize >> > (
        itemCount, array_length,
        d_position, d_write_flag,
        d_data, static_cast<unsigned int>(sd.size()));
    gpuErrchkLaunch();
    // Check d_write_flag for dupes
    size_t t_data_len = data_len;
    hipcub::DeviceReduce::Max(d_data, t_data_len, d_write_flag, d_position, array_length);
    unsigned int maxBinSize = 0;
    gpuErrchk(hipMemcpy(&maxBinSize, d_position, sizeof(unsigned int), hipMemcpyDeviceToHost));
    if (maxBinSize > 1) {
        // Too many messages for single element of array
        // Report bad ones
        unsigned int *hd_write_flag = (unsigned int *)malloc(sizeof(unsigned int) * array_length);
        gpuErrchk(hipMemcpy(hd_write_flag, d_write_flag, sizeof(unsigned int)* array_length, hipMemcpyDeviceToHost));
        for (unsigned int i = 0; i < array_length; ++i) {
            if (hd_write_flag[i] > 1)
                fprintf(stderr, "Array messagelist contains %u messages at index %u!\n", hd_write_flag[i], i);
        }
        THROW ArrayMessageWriteConflict("Multiple threads output array messages to the same index, see stderr.\n");
    }
}
void CUDAScatter::increaseSimCounter() {
    simulationInstances++;
}
void CUDAScatter::decreaseSimCounter() {
    simulationInstances--;
    if (simulationInstances == 0) {
        for (unsigned int i = 0; i < flamegpu_internal::CUDAScanCompaction::MAX_STREAMS; ++i) {
            getInstance(i).free();
        }
    }
}
