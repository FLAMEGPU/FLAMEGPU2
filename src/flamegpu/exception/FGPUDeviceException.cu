// This define forces the header to include the DeviceException::subformat() template specialisation for const char *
// It needs to be in the header so that RTC can also use it
#define FGPUDEVICEEXCEPTION_CU
#include "flamegpu/exception/FGPUDeviceException.h"

#include "flamegpu/gpu/CUDAErrorChecking.h"
#ifndef NO_SEATBELTS

DeviceExceptionManager::DeviceExceptionManager()
    : d_buffer()
    , hd_buffer() {
    memset (&d_buffer, 0, sizeof(d_buffer));
    memset (&hd_buffer, 0, sizeof(hd_buffer));
}
DeviceExceptionManager::~DeviceExceptionManager() {
    for (auto &i : d_buffer) {
        gpuErrchk(hipFree(i));
    }
}
DeviceExceptionBuffer *DeviceExceptionManager::getDevicePtr(const unsigned int &streamId) {
    if (streamId >= CUDAScanCompaction::MAX_STREAMS) {
        THROW OutOfBoundsException("Stream id %u is out of bounds, %u >= %u, "
        "in FGPUDeviceException::getDevicePtr()\n", streamId, streamId, CUDAScanCompaction::MAX_STREAMS);
    }
    if (!d_buffer[streamId]) {
        gpuErrchk(hipMalloc(&d_buffer[streamId], sizeof(DeviceExceptionBuffer)));
    }
    gpuErrchk(hipDeviceSynchronize());
    // Memset and return buffer
    gpuErrchk(hipMemset(d_buffer[streamId], 0, sizeof(DeviceExceptionBuffer)));
    memset(&hd_buffer[streamId], 0, sizeof(DeviceExceptionBuffer));
    return d_buffer[streamId];
}
void DeviceExceptionManager::checkError(const std::string &function, const unsigned int &streamId) {
    if (streamId >= CUDAScanCompaction::MAX_STREAMS) {
        THROW OutOfBoundsException("Stream id %u is out of bounds, %u >= %u, "
        "in FGPUDeviceException::checkError()\n", streamId, streamId, CUDAScanCompaction::MAX_STREAMS);
    }
    if (d_buffer[streamId]) {
        // Grab buffer from device
        hipMemcpy(&hd_buffer[streamId], d_buffer[streamId], sizeof(DeviceExceptionBuffer), hipMemcpyDeviceToHost);
        // If there is a reported error count
        if (hd_buffer[streamId].error_count) {
            std::string location_string = getLocationString(hd_buffer[streamId]);
            std::string error_string = getErrorString(hd_buffer[streamId]);
            throw DeviceError(
            "Device function '%s' reported %u errors.\nFirst error:\n%s:\n%s",
            function.c_str(), hd_buffer[streamId].error_count, location_string.c_str(), error_string.c_str());
        }
    } else {
        THROW OutOfBoundsException("FGPUDeviceExceptionBuffer for stream %u has not been allocated, "
        "in FGPUDeviceException::checkError()\n", streamId, streamId, CUDAScanCompaction::MAX_STREAMS);
    }
}
std::string DeviceExceptionManager::getLocationString(const DeviceExceptionBuffer &b) {
    char buff[DeviceExceptionBuffer::OUT_STRING_LEN];
    snprintf(buff, DeviceExceptionBuffer::OUT_STRING_LEN, "%s(%u)[%u,%u,%u][%u,%u,%u]",
        b.file_path, b.line_no,
        b.block_id[0], b.block_id[1], b.block_id[2],
        b.thread_id[0], b.thread_id[1], b.thread_id[2]);
    return buff;
}
std::string DeviceExceptionManager::getErrorString(const DeviceExceptionBuffer &b) {
    /**
     * This buffer is used to copy sub-format strings into before we send them to snprintf
     * This saves us needing to set the final+1 char to '\0'
     */
    char temp_buffer[DeviceExceptionBuffer::FORMAT_BUFF_LEN];
    /**
     * This is the buffer into which we construct the string to be returned
     */
    char out_buffer[DeviceExceptionBuffer::OUT_STRING_LEN];
    memset(out_buffer, 0, DeviceExceptionBuffer::FORMAT_BUFF_LEN);
    // Progress through b.format_string
    unsigned int format_buffer_index = 0;
    // Progress through out_buffer
    unsigned int out_index = 0;
    // Progress through b.format_args_sizes
    unsigned int arg_no = 0;
    // Progress through b.format_args
    unsigned int arg_offset = 0;
    // Whilst there is still work to be done, we are still in range of format string and all other structures used
    while (b.format_string[format_buffer_index] != '\0' &&
           format_buffer_index < DeviceExceptionBuffer::FORMAT_BUFF_LEN &&
           out_index < DeviceExceptionBuffer::FORMAT_BUFF_LEN &&
           arg_no < DeviceExceptionBuffer::MAX_ARGS) {
        // If we find the start of a sub format string
        if (b.format_string[format_buffer_index] == '%') {
            // Find the next sub format start, or end of entire format string
            unsigned int format_end = format_buffer_index + 1;
            char format_type = '\0';
            while (b.format_string[format_end] != '%' &&
                  b.format_string[format_end] != '\0' &&
                  format_end < DeviceExceptionBuffer::FORMAT_BUFF_LEN) {
                // Detect the format type, we will use this later
                if (format_type == '\0') {
                    switch (b.format_string[format_end]) {
                        // This is every format specifier supported by the printf family of functions
                        case 'd':
                        case 'i':
                        case 'u':
                        case 'o':
                        case 'x':
                        case 'X':
                        case 'f':
                        case 'e':
                        case 'g':
                        case 'G':
                        case 'a':
                        case 'A':
                        case 'c':
                        case 's':
                        case 'p':
                        case 'n':
                            format_type = b.format_string[format_end];
                            break;
                    }
                }
                ++format_end;
            }
            // Sub format string bounds have been found
            // Copy the sub format string into a temporary buffer
            memset(temp_buffer, 0, DeviceExceptionBuffer::FORMAT_BUFF_LEN);
            memcpy(temp_buffer, b.format_string + format_buffer_index, format_end - format_buffer_index);
            // Now send this substring to the formatter to process
            // Cast it to the correct type first
            // (This assumes snprintf never returns negative)
            switch (format_type) {
                case 'd':
                case 'i': {
                    // Signed integer
                    if (b.format_args_sizes[arg_no] == 4) {
                        out_index += snprintf(out_buffer + out_index, DeviceExceptionBuffer::OUT_STRING_LEN - out_index, temp_buffer, *reinterpret_cast<const int32_t*>(b.format_args+arg_offset));
                    } else {
                        out_index += snprintf(out_buffer + out_index, DeviceExceptionBuffer::OUT_STRING_LEN - out_index, temp_buffer, *reinterpret_cast<const int64_t*>(b.format_args+arg_offset));
                    }
                    break;
                }
                case 'u':
                case 'o':
                case 'x':
                case 'X': {
                    // Unsigned integer
                    if (b.format_args_sizes[arg_no] == 4) {
                        out_index += snprintf(out_buffer + out_index, DeviceExceptionBuffer::OUT_STRING_LEN - out_index, temp_buffer, *reinterpret_cast<const uint32_t*>(b.format_args+arg_offset));
                    } else {
                        out_index += snprintf(out_buffer + out_index, DeviceExceptionBuffer::OUT_STRING_LEN - out_index, temp_buffer, *reinterpret_cast<const uint64_t*>(b.format_args+arg_offset));
                    }
                    break;
                }
                case 'f':
                case 'e':
                case 'g':
                case 'G':
                case 'a':
                case 'A': {
                    // Floating point
                    if (b.format_args_sizes[arg_no] == 4) {
                        out_index += snprintf(out_buffer + out_index, DeviceExceptionBuffer::OUT_STRING_LEN - out_index, temp_buffer, *reinterpret_cast<const float*>(b.format_args+arg_offset));
                    } else {
                        out_index += snprintf(out_buffer + out_index, DeviceExceptionBuffer::OUT_STRING_LEN - out_index, temp_buffer, *reinterpret_cast<const double*>(b.format_args+arg_offset));
                    }
                    break;
                }
                case 'c': {
                    // Char
                    out_index += snprintf(out_buffer + out_index, DeviceExceptionBuffer::OUT_STRING_LEN - out_index, temp_buffer, *reinterpret_cast<const char*>(b.format_args+arg_offset));
                    break;
                }
                case 's': {
                    // Char string
                    out_index += snprintf(out_buffer + out_index, DeviceExceptionBuffer::OUT_STRING_LEN - out_index, temp_buffer, reinterpret_cast<const char*>(b.format_args+arg_offset));
                    break;
                }
                case 'p': {
                    // Pointer
                    out_index += snprintf(out_buffer + out_index, DeviceExceptionBuffer::OUT_STRING_LEN - out_index, temp_buffer, reinterpret_cast<const void*>(b.format_args+arg_offset));
                    break;
                }
                case 'n': {
                    // No of chars written (signed pointer to have value written back to)
                    if (b.format_args_sizes[arg_no] == 4) {
                        out_index += snprintf(out_buffer + out_index, DeviceExceptionBuffer::OUT_STRING_LEN - out_index, temp_buffer, reinterpret_cast<const int32_t*>(b.format_args+arg_offset));
                    } else {
                        out_index += snprintf(out_buffer + out_index, DeviceExceptionBuffer::OUT_STRING_LEN - out_index, temp_buffer, reinterpret_cast<const int64_t*>(b.format_args+arg_offset));
                    }
                    break;
                }
            }
            // Update arg counters
            arg_offset += b.format_args_sizes[arg_no];
            ++arg_no;
            // Update pointer into main format string and continue loop
            format_buffer_index = format_end;
        } else {
            // Copy the single char
            // This will only happen until we hit first sub format string
            out_buffer[out_index] = b.format_string[format_buffer_index];
            ++out_index;
            ++format_buffer_index;
        }
    }
    return out_buffer;
}

#endif  // NO_SEATBELTS
