#include <hip/hip_runtime.h>

#include <cstdio>
#include <cassert>
#include <map>
#include <memory>

#include "flamegpu/runtime/detail/curve/curve.cuh"


#include "flamegpu/gpu/detail/CUDAErrorChecking.cuh"
#include "flamegpu/util/nvtx.h"

namespace flamegpu {
namespace detail {
namespace curve {

std::mutex Curve::instance_mutex;

/* header implementations */
__host__ Curve::Curve()
    : deviceInitialised(false)
    , d_curve_table(nullptr) {
}
__host__ Curve::~Curve() {
    if (d_curve_table) {
        gpuErrchk(hipFree(d_curve_table));
        d_curve_table = nullptr;
    }
}
__host__ void Curve::purge(hipStream_t stream) {
    auto lock = std::unique_lock<std::shared_timed_mutex>(mutex);
    if (d_curve_table) {
        // gpuErrchk(hipFree(d_curve_table));  // This fails if called after device reset
        d_curve_table = nullptr;
    }
    deviceInitialised = false;
    initialiseDevice(stream);
}
__host__ void Curve::initialiseDevice(hipStream_t stream) {
    // Don't lock mutex here, do it in the calling method
    if (!deviceInitialised) {
        // get a host pointer to d_hashes and d_variables
        gpuErrchk(hipMalloc(&d_curve_table, sizeof(CurveTable)));

        // set values of hash table to 0 on host and device
        memset(h_curve_table.hashes, 0, sizeof(unsigned int)*MAX_VARIABLES);
        memset(h_curve_table.lengths, 0, sizeof(unsigned int)*MAX_VARIABLES);
        memset(h_curve_table.sizes, 0, sizeof(size_t)*MAX_VARIABLES);

        // initialise data to 0 on device
        gpuErrchk(hipMemsetAsync(d_curve_table, 0, sizeof(CurveTable), stream));
    }
    deviceInitialised = true;
}

__host__ Curve::VariableHash Curve::variableRuntimeHash(const char* str) {
    // Method is static, does not require mutex
    const size_t length = std::strlen(str) + 1;
    unsigned int hash = 2166136261u;

    for (size_t i = 0; i < length; ++i) {
        hash ^= *str++;
        hash *= 16777619u;
    }
    return hash;
}
__host__ Curve::VariableHash Curve::variableRuntimeHash(unsigned int num) {
    // Method is static, does not require mutex
    return variableRuntimeHash(std::to_string(num).c_str());
}

__host__ Curve::Variable Curve::getVariableHandle(VariableHash variable_hash) {
    // Method is static, does not require mutex
    unsigned int n = 0;
    unsigned int i = (variable_hash) % MAX_VARIABLES;

    while (h_curve_table.hashes[i] != EMPTY_FLAG) {
        if (h_curve_table.hashes[i] == variable_hash) {
            return i;
        }
        n += 1;
        if (n >= MAX_VARIABLES) {
            break;
        }
        i += 1;
        if (i >= MAX_VARIABLES) {
            i = 0;
        }
    }
    return UNKNOWN_VARIABLE;
}

__host__ Curve::Variable Curve::registerVariableByHash(VariableHash variable_hash, void * d_ptr, size_t size, unsigned int length) {
    auto lock = std::unique_lock<std::shared_timed_mutex>(mutex);
    return _registerVariableByHash(variable_hash, d_ptr, size, length);
}
__host__ Curve::Variable Curve::_registerVariableByHash(VariableHash variable_hash, void * d_ptr, size_t size, unsigned int length) {
    // Do not lock mutex here, do it in the calling method
    unsigned int n = 0;
    assert(variable_hash != EMPTY_FLAG);
    assert(variable_hash != DELETED_FLAG);
    unsigned int i = (variable_hash) % MAX_VARIABLES;
    while (h_curve_table.hashes[i] != EMPTY_FLAG && h_curve_table.hashes[i] != DELETED_FLAG) {
        n += 1;
        if (n >= MAX_VARIABLES) {
            return UNKNOWN_VARIABLE;
        }
        i += 1;
        if (i >= MAX_VARIABLES) {
            i = 0;
        }
    }

    h_curve_table.hashes[i] = variable_hash;

    // make a host copy of the pointer
    h_curve_table.variables[i] = reinterpret_cast<char*>(d_ptr);

    // set the size of the data type
    h_curve_table.sizes[i] = size;

    // set the length of variable
    h_curve_table.lengths[i] = length;

    return i;
}
__host__ int Curve::size() const {
    auto lock = std::shared_lock<std::shared_timed_mutex>(mutex);
    return _size();
}
__host__ int Curve::_size() const {
    int rtn = 0;
    for (unsigned int hash : h_curve_table.hashes) {
        if (hash != EMPTY_FLAG && hash != DELETED_FLAG)
            rtn++;
    }
    return rtn;
}
/**
 * TODO: Does un-registering imply that other variable with collisions will no longer be found. I.e. do you need to re-register all other variable when one is removed.
 */
__host__ void Curve::unregisterVariableByHash(VariableHash variable_hash) {
    auto lock = std::unique_lock<std::shared_timed_mutex>(mutex);
    _unregisterVariableByHash(variable_hash);
}
__host__ void Curve::_unregisterVariableByHash(VariableHash variable_hash) {
    // Do not lock mutex here, do it in the calling method
    // get the curve variable
    Variable cv = getVariableHandle(variable_hash);

    // error checking
    if (cv == UNKNOWN_VARIABLE) {
        THROW exception::CurveException("Cannot unregister '%u', hash not found within curve table.", variable_hash);
    }

    // clear hash location on host and copy hash to device
    h_curve_table.hashes[cv] = DELETED_FLAG;

    // set a host pointer to nullptr and copy to the device
    h_curve_table.variables[cv] = 0;

    // set the empty size to 0
    h_curve_table.sizes[cv] = 0;

    // set the length of variable to 0
    h_curve_table.lengths[cv] = 0;
}
__host__ void Curve::updateDevice(const hipStream_t stream) {
    auto lock = std::shared_lock<std::shared_timed_mutex>(mutex);
    NVTX_RANGE("Curve::updateDevice()");
    // Initialise the device (if required)
    assert(deviceInitialised);  // No reason for this to ever fail. Purge calls init device
    // Copy
    gpuErrchk(hipMemcpyAsync(d_curve_table, &h_curve_table, sizeof(CurveTable), hipMemcpyHostToDevice, stream));
    gpuErrchk(hipStreamSynchronize(stream));
}
__host__ const Curve::CurveTable *Curve::getDevicePtr() const {
    return d_curve_table;
}

Curve& Curve::getInstance() {
    auto lock = std::unique_lock<std::mutex>(instance_mutex);  // Mutex to protect from two threads triggering the static instantiation concurrently
    static std::map<int, std::unique_ptr<Curve>> instances = {};  // Instantiated on first use.
    int device_id = -1;
    gpuErrchk(hipGetDevice(&device_id));
    // Can't use operator[] here, constructor is private
    const auto f = instances.find(device_id);
    if (f != instances.end())
        return *f->second;
    return *(instances.emplace(device_id, std::unique_ptr<Curve>(new Curve())).first->second);
}

}  // namespace curve
}  // namespace detail
}  // namespace flamegpu
