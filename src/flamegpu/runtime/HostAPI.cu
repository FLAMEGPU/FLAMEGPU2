#include "flamegpu/runtime/HostAPI.h"

#include <map>
#include "flamegpu/runtime/agent/HostAgentAPI.cuh"
#include "flamegpu/model/ModelDescription.h"
#include "flamegpu/simulation/Simulation.h"
#include "flamegpu/util/nvtx.h"
#include "flamegpu/simulation/CUDASimulation.h"
#include "flamegpu/detail/cuda.cuh"

namespace flamegpu {

HostAPI::HostAPI(CUDASimulation &_agentModel,
    detail::RandomManager& rng,
    detail::CUDAScatter &_scatter,
    const AgentOffsetMap &_agentOffsets,
    AgentDataMap &_agentData,
    const std::shared_ptr<detail::EnvironmentManager>& env,
    const std::shared_ptr<detail::CUDAMacroEnvironment>& macro_env,
    CUDADirectedGraphMap &directed_graph_map,
    const unsigned int _streamId,
    hipStream_t _stream)
    : random(rng)
    , environment(_agentModel, env, macro_env, directed_graph_map, _scatter, _streamId, _stream)
    , agentModel(_agentModel)
    , d_output_space(nullptr)
    , d_output_space_size(0)
    , agentOffsets(_agentOffsets)
    , agentData(_agentData)
    , scatter(_scatter)
    , streamId(_streamId)
    , stream(_stream) { }

HostAPI::~HostAPI() {
    // @todo - cuda is not allowed in destructor
    if (d_output_space_size) {
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_output_space));
        d_output_space_size = 0;
    }
}

HostAgentAPI HostAPI::agent(const std::string &agent_name, const std::string &state_name) {
    auto agt = agentData.find(agent_name);
    if (agt == agentData.end()) {
        THROW exception::InvalidAgent("Agent '%s' was not found in model description hierarchy.\n", agent_name.c_str());
    }
    auto state = agt->second.find(state_name);
    if (state == agt->second.end()) {
        THROW exception::InvalidAgentState("Agent '%s' in model description hierarchy does not contain state '%s'.\n", agent_name.c_str(), state_name.c_str());
    }
    return HostAgentAPI(*this, agentModel.getCUDAAgent(agent_name), state_name, agentOffsets.at(agent_name), state->second);
}

unsigned int HostAPI::getStepCounter() const {
    return agentModel.getStepCounter();
}
unsigned int HostAPI::getEnsembleRunIndex() const {
    return agentModel.getCUDAConfig().ensemble_run_id;
}

}  // namespace flamegpu
