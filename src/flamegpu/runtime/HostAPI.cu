#include "flamegpu/runtime/HostAPI.h"
#include "flamegpu/runtime/HostAgentAPI.cuh"
#include "flamegpu/model/ModelDescription.h"
#include "flamegpu/sim/Simulation.h"
#include "flamegpu/util/nvtx.h"
#include "flamegpu/gpu/CUDASimulation.h"

namespace flamegpu {

HostAPI::HostAPI(CUDASimulation &_agentModel,
    RandomManager& rng,
    CUDAScatter &_scatter,
    const AgentOffsetMap &_agentOffsets,
    AgentDataMap &_agentData,
    const std::shared_ptr<EnvironmentManager>& env,
    CUDAMacroEnvironment &macro_env,
    const unsigned int _streamId,
    hipStream_t _stream)
    : random(rng)
    , environment(_agentModel.getInstanceID(), env, macro_env)
    , agentModel(_agentModel)
    , d_output_space(nullptr)
    , d_output_space_size(0)
    , agentOffsets(_agentOffsets)
    , agentData(_agentData)
    , scatter(_scatter)
    , streamId(_streamId)
    , stream(_stream) { }

HostAPI::~HostAPI() {
    // @todo - cuda is not allowed in destructor
    if (d_output_space_size) {
        gpuErrchk(hipFree(d_output_space));
        d_output_space_size = 0;
    }
}

HostAgentAPI HostAPI::agent(const std::string &agent_name, const std::string &state_name) {
    auto agt = agentData.find(agent_name);
    if (agt == agentData.end()) {
        THROW exception::InvalidAgent("Agent '%s' was not found in model description hierarchy.\n", agent_name.c_str());
    }
    auto state = agt->second.find(state_name);
    if (state == agt->second.end()) {
        THROW exception::InvalidAgentState("Agent '%s' in model description hierarchy does not contain state '%s'.\n", agent_name.c_str(), state_name.c_str());
    }
    return HostAgentAPI(*this, agentModel.getAgent(agent_name), state_name, agentOffsets.at(agent_name), state->second);
}

/**
 * Access the current stepCount
 * Sepearate implementation to avoid dependency loop with cuda agent model.
 * @return the current step count, 0 indexed unsigned.
 */
unsigned int HostAPI::getStepCounter() const {
    return agentModel.getStepCounter();
}

}  // namespace flamegpu
