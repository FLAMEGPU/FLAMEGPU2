#include "hip/hip_runtime.h"
#include "flamegpu/runtime/utility/RandomManager.cuh"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>

#include<ctime>

#include <cassert>
#include <cstdio>
#include <algorithm>

#include "flamegpu/gpu/CUDAErrorChecking.h"
#include "flamegpu/gpu/CUDAAgentModel.h"

RandomManager::RandomManager() :
    deviceInitialised(false) {
    reseed(static_cast<unsigned int>(seedFromTime() % UINT_MAX));
}
RandomManager::~RandomManager() {
    free();  // @todo call free/freeDevice not in the constructor! instead just log that?
}
void RandomManager::purge() {
    length = 0;
    d_random_state = nullptr;
}
/**
 * Member fns
 */
uint64_t RandomManager::seedFromTime() {
    return static_cast<uint64_t>(time(nullptr));
}

void RandomManager::reseedHost() {
    freeHost();
    host_rng = std::mt19937();
    // Reset host random generator/s
    host_rng.seed(mSeed);
}

void RandomManager::reseedDevice() {
    freeDevice();
    // hiprand is initialised on access if length does not match. This would need a second device length?
}

void RandomManager::reseed(const unsigned int &seed) {
    // Set the instance's seed to the new value
    mSeed = seed;

    // Apply the new seed to the host
    reseedHost();
    // Apply the new seed to the device.
    reseedDevice();
}

void RandomManager::freeHost() {
    // Release host_max
    if (h_max_random_state) {
        std::free(h_max_random_state);
        h_max_random_state = nullptr;
    }
    h_max_random_size = 0;
}

void RandomManager::freeDevice() {
    // Clear size - length is just for the device portion?
    length = 0;

    if (deviceInitialised) {
        // Set the device's internal size to 0.
        length = 0;
        // Release old random states on the deivce and update pointers.
        if (d_random_state) {
            gpuErrchk(hipFree(d_random_state));
        }
        d_random_state = nullptr;
    }
}

void RandomManager::free() {
    // Free the host and device.
    freeHost();
    freeDevice();
}

hiprandState *RandomManager::resize(const size_type &_length) {
    assert(growthModifier > 1.0);
    assert(shrinkModifier > 0.0);
    assert(shrinkModifier <= 1.0);
    auto t_length = length;
    if (length) {
        while (t_length < _length) {
            t_length = static_cast<RandomManager::size_type>(t_length * growthModifier);
            if (shrinkModifier < 1.0f) {
                while (t_length * shrinkModifier > _length) {
                    t_length = static_cast<RandomManager::size_type>(t_length * shrinkModifier);
                }
            }
        }
    } else {  // Special case for first run
        t_length = _length;
    }
    // Don't allow array to go below RandomManager::min_length elements
    t_length = std::max<size_type>(t_length, RandomManager::min_length);
    if (t_length != length)
        resizeDeviceArray(t_length);
    return d_random_state;
}
__global__ void init_curand(hiprandState *d_random_state, unsigned int threadCount, uint64_t seed, RandomManager::size_type offset) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < threadCount)
        hiprand_init(seed, offset + id, 0, &d_random_state[offset + id]);
}
void RandomManager::resizeDeviceArray(const size_type &_length) {
    // Mark that the device hsa now been initialised.
    deviceInitialised = true;
    if (_length > h_max_random_size) {
        // Growing array
        hiprandState *t_hd_random_state = nullptr;
        // Allocate new mem to t_hd
        gpuErrchk(hipMalloc(&t_hd_random_state, _length * sizeof(hiprandState)));
        // Copy hd->t_hd[****    ]
        if (d_random_state) {
            gpuErrchk(hipMemcpy(t_hd_random_state, d_random_state, length * sizeof(hiprandState), hipMemcpyDeviceToDevice));
        }
        // Update pointers hd=t_hd
        if (d_random_state) {
            gpuErrchk(hipFree(d_random_state));
        }
        d_random_state = t_hd_random_state;
        // Init new[    ****]
        if (h_max_random_size > length) {
            // We have part/all host backup, copy to device array
            // Reinit backup[    **  ]
            const size_type copy_len = std::min(h_max_random_size, _length);
            gpuErrchk(hipMemcpy(d_random_state + length, h_max_random_state + length, copy_len * sizeof(hiprandState), hipMemcpyHostToDevice));
            length += copy_len;
        }
        if (_length > length) {
            // Init remainder[     **]
            unsigned int initThreads = 512;
            unsigned int initBlocks = ((_length - length) / initThreads) + 1;
            init_curand<<<initBlocks, initThreads>>>(d_random_state, _length - length, mSeed, length);  // This could be async with above memcpy?
            gpuErrchkLaunch();
        }
    } else {
        // Shrinking array
        hiprandState *t_hd_random_state = nullptr;
        hiprandState *t_h_max_random_state = nullptr;
        // Allocate new
        gpuErrchk(hipMalloc(&t_hd_random_state, _length * sizeof(hiprandState)));
        // Allocate host backup
        if (length > h_max_random_size)
            t_h_max_random_state = reinterpret_cast<hiprandState *>(malloc(length * sizeof(hiprandState)));
        else
            t_h_max_random_state = h_max_random_state;
        // Copy old->new
        assert(d_random_state);
        gpuErrchk(hipMemcpy(t_hd_random_state, d_random_state, _length * sizeof(hiprandState), hipMemcpyDeviceToDevice));
        // Copy part being shrunk away to host storage (This could be async with above memcpy?)
        gpuErrchk(hipMemcpy(t_h_max_random_state + _length, d_random_state + _length, (length - _length) * sizeof(hiprandState), hipMemcpyDeviceToHost));
        // Release and replace old host ptr
        if (length > h_max_random_size) {
            if (h_max_random_state)
                ::free(h_max_random_state);
            h_max_random_state = t_h_max_random_state;
            h_max_random_size = length;
        }
        // Release old
        if (d_random_state != nullptr) {
            gpuErrchk(hipFree(d_random_state));
        }
        // Update pointer
        d_random_state = t_hd_random_state;
    }
    // Update length
    length = _length;
}
void RandomManager::setGrowthModifier(float _growthModifier) {
    assert(growthModifier > 1.0);
    RandomManager::growthModifier = _growthModifier;
}
float RandomManager::getGrowthModifier() {
    return RandomManager::growthModifier;
}
void RandomManager::setShrinkModifier(float _shrinkModifier) {
    assert(shrinkModifier > 0.0);
    assert(shrinkModifier <= 1.0);
    RandomManager::shrinkModifier = _shrinkModifier;
}
float RandomManager::getShrinkModifier() {
    return RandomManager::shrinkModifier;
}
RandomManager::size_type RandomManager::size() {
    return length;
}
uint64_t RandomManager::seed() {
    return mSeed;
}
hiprandState *RandomManager::cudaRandomState() {
    return d_random_state;
}
