#include "hip/hip_runtime.h"
#include "flamegpu/runtime/utility/RandomManager.cuh"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>

#include<ctime>

#include <cassert>
#include <cstdio>
#include <algorithm>

#include "flamegpu/gpu/CUDAErrorChecking.h"
#include "flamegpu/gpu/CUDAAgentModel.h"

/**
 * Internal namespace to hide __device__ declarations from modeller
 */
namespace flamegpu_internal {
    /**
     * Device array holding hiprand states
     * They should always be initialised
     */
    __device__ hiprandState *d_random_state;
    /**
     * Device copy of the length of d_random_state
     */
    __device__ RandomManager::size_type d_random_size;
    /**
     * Host mirror of d_random_state
     */
    hiprandState *hd_random_state;
    /**
     * Host mirror of d_random_size
     */
    RandomManager::size_type hd_random_size;
}  // namespace flamegpu_internal

unsigned int RandomManager::simulationInstances = 0;

RandomManager::RandomManager() :
    deviceInitialised(false) {
    reseed(static_cast<unsigned int>(seedFromTime() % UINT_MAX));
}
RandomManager::~RandomManager() {
    // free(); // @todo call free/freeDevice not in the constructor! instead just log that?
}
void RandomManager::purge() {
    flamegpu_internal::hd_random_size = 0;
    flamegpu_internal::hd_random_state = nullptr;
}
/**
 * Member fns
 */
uint64_t RandomManager::seedFromTime() {
    return static_cast<uint64_t>(time(nullptr));
}

void RandomManager::reseedHost() {
    freeHost();
    host_rng = std::mt19937();
    // Reset host random generator/s
    host_rng.seed(mSeed);
}

void RandomManager::reseedDevice() {
    freeDevice();
    // hiprand is initialised on access if length does not match. This would need a second device lenght?
}

void RandomManager::reseed(const unsigned int &seed) {
    // Set the instance's seed to the new value
    RandomManager::mSeed = seed;

    // Apply the new seed to the host
    reseedHost();
    // Apply the new seed to the device.
    reseedDevice();
}

void RandomManager::freeHost() {
    // Release host_max
    if (h_max_random_state) {
        std::free(h_max_random_state);
        h_max_random_state = nullptr;
    }
    h_max_random_size = 0;
}

void RandomManager::freeDevice() {
    // Clear size - length is just for the device portion?
    length = 0;

    if (deviceInitialised) {
        // Set the device's internal size to 0.
        flamegpu_internal::hd_random_size = 0;
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(flamegpu_internal::d_random_size), &flamegpu_internal::hd_random_size, sizeof(RandomManager::size_type)));
        // Release old random states on the deivce and update pointers.
        if (flamegpu_internal::hd_random_state) {
            gpuErrchk(hipFree(flamegpu_internal::hd_random_state));
        }
        flamegpu_internal::hd_random_state = nullptr;
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(flamegpu_internal::d_random_state), &flamegpu_internal::hd_random_state, sizeof(hiprandState*)))
    }
}

void RandomManager::free() {
    // Free the host and device.
    freeHost();
    freeDevice();
}

bool RandomManager::resize(const size_type &_length, const CUDAAgentModel& model) {
    assert(growthModifier > 1.0);
    assert(shrinkModifier > 0.0);
    assert(shrinkModifier <= 1.0);
    auto t_length = length;
    if (length) {
        while (t_length < _length) {
            t_length = static_cast<RandomManager::size_type>(t_length * growthModifier);
            if (shrinkModifier < 1.0f) {
                while (t_length * shrinkModifier > _length) {
                    t_length = static_cast<RandomManager::size_type>(t_length * shrinkModifier);
                }
            }
        }
    } else {  // Special case for first run
        t_length = _length;
    }
    // Don't allow array to go below RandomManager::min_length elements
    t_length = std::max<size_type>(t_length, RandomManager::min_length);
    if (t_length != length)
        resizeDeviceArray(t_length, model);
    else if (length != flamegpu_internal::hd_random_size)
        resizeDeviceArray(length, model);
    return t_length != length;
}
__global__ void init_curand(unsigned int threadCount, uint64_t seed, RandomManager::size_type offset) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < threadCount)
        hiprand_init(seed, offset + id, 0, &flamegpu_internal::d_random_state[offset + id]);
}
void RandomManager::resizeDeviceArray(const size_type &_length, const CUDAAgentModel& model) {
    // Mark that the device hsa now been initialised.
    deviceInitialised = true;
    if (_length > h_max_random_size) {
        // Growing array
        hiprandState *t_hd_random_state = nullptr;
        // Allocate new mem to t_hd
        gpuErrchk(hipMalloc(&t_hd_random_state, _length * sizeof(hiprandState)));
        // Copy hd->t_hd[****    ]
        if (flamegpu_internal::hd_random_state) {
            gpuErrchk(hipMemcpy(t_hd_random_state, flamegpu_internal::hd_random_state, length * sizeof(hiprandState), hipMemcpyDeviceToDevice));
        }
        // Update pointers hd=t_hd
        if (flamegpu_internal::hd_random_state) {
            gpuErrchk(hipFree(flamegpu_internal::hd_random_state));
        }
        flamegpu_internal::hd_random_state = t_hd_random_state;
        // perform safe copy to symbol by ensuring that the runtime library symbol AND each RTC function symbol is also updated
        void* d_ptr;
        gpuErrchk(hipGetSymbolAddress(&d_ptr, HIP_SYMBOL(flamegpu_internal::d_random_state)));
        model.RTCSafeCudaMemcpyToSymbolAddress(d_ptr, "flamegpu_internal::d_random_state", &flamegpu_internal::hd_random_state, sizeof(hiprandState*));
        // Init new[    ****]
        if (h_max_random_size > length) {
            // We have part/all host backup, copy to device array
            // Reinit backup[    **  ]
            size_type copy_len = std::min(h_max_random_size, _length);
            gpuErrchk(hipMemcpy(flamegpu_internal::hd_random_state + length, h_max_random_state + length, copy_len * sizeof(hiprandState), hipMemcpyHostToDevice));
            length += copy_len;
        }
        if (_length > length) {
            // Init remainder[     **]
            unsigned int initThreads = 512;
            unsigned int initBlocks = ((_length - length) / initThreads) + 1;
            init_curand<<<initBlocks, initThreads>>>(_length - length, mSeed, length);  // This could be async with above memcpy?
            gpuErrchkLaunch();
        }
    } else {
        // Shrinking array
        hiprandState *t_hd_random_state = nullptr;
        hiprandState *t_h_max_random_state = nullptr;
        // Allocate new
        gpuErrchk(hipMalloc(&t_hd_random_state, _length * sizeof(hiprandState)));
        // Allocate host backup
        if (length > h_max_random_size)
            t_h_max_random_state = reinterpret_cast<hiprandState *>(malloc(length * sizeof(hiprandState)));
        else
            t_h_max_random_state = h_max_random_state;
        // Copy old->new
        assert(flamegpu_internal::hd_random_state);
        gpuErrchk(hipMemcpy(t_hd_random_state, flamegpu_internal::hd_random_state, _length * sizeof(hiprandState), hipMemcpyDeviceToDevice));
        // Copy part being shrunk away to host storage (This could be async with above memcpy?)
        gpuErrchk(hipMemcpy(t_h_max_random_state + _length, flamegpu_internal::hd_random_state + _length, (length - _length) * sizeof(hiprandState), hipMemcpyDeviceToHost));
        // Release and replace old host ptr
        if (length > h_max_random_size) {
            if (h_max_random_state)
                ::free(h_max_random_state);
            h_max_random_state = t_h_max_random_state;
            h_max_random_size = length;
        }
        // Update pointers
        flamegpu_internal::hd_random_state = t_hd_random_state;
        // perform safe copy to symbol by ensuring that the runtime library symbol AND each RTC function symbol is also updated
        void* d_ptr;
        gpuErrchk(hipGetSymbolAddress(&d_ptr, HIP_SYMBOL(flamegpu_internal::d_random_state)));
        model.RTCSafeCudaMemcpyToSymbolAddress(d_ptr, "flamegpu_internal::d_random_state", &flamegpu_internal::hd_random_state, sizeof(hiprandState*));
        // Release old
        if (flamegpu_internal::hd_random_state != nullptr) {
            gpuErrchk(hipFree(flamegpu_internal::hd_random_state));
        }
    }
    // Update length
    length = _length;
    flamegpu_internal::hd_random_size = _length;
    // perform safe copy to symbol by ensuring that the runtime library symbol AND each RTC function symbol is also updated
    void* d_ptr;
    gpuErrchk(hipGetSymbolAddress(&d_ptr, HIP_SYMBOL(flamegpu_internal::d_random_size)));
    model.RTCSafeCudaMemcpyToSymbolAddress(d_ptr, "flamegpu_internal::d_random_size", &flamegpu_internal::hd_random_size, sizeof(RandomManager::size_type));
}
void RandomManager::setGrowthModifier(float _growthModifier) {
    assert(growthModifier > 1.0);
    RandomManager::growthModifier = _growthModifier;
}
float RandomManager::getGrowthModifier() {
    return RandomManager::growthModifier;
}
void RandomManager::setShrinkModifier(float _shrinkModifier) {
    assert(shrinkModifier > 0.0);
    assert(shrinkModifier <= 1.0);
    RandomManager::shrinkModifier = _shrinkModifier;
}
float RandomManager::getShrinkModifier() {
    return RandomManager::shrinkModifier;
}
RandomManager::size_type RandomManager::size() {
    return length;
}
uint64_t RandomManager::seed() {
    return mSeed;
}

void RandomManager::increaseSimCounter() {
    simulationInstances++;
}
void RandomManager::decreaseSimCounter() {
    simulationInstances--;
    if (simulationInstances == 0)
        free();
}
