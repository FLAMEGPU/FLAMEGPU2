#include "hip/hip_runtime.h"
#include "Random.cuh"

#include<ctime>

#include <cassert>
#include <cstdio>
#include <algorithm>

#include "./hip/hip_runtime.h"
#include "./hiprand/hiprand_kernel.h"
#include "./"

/**
 * Internal namespace to hide __device__ declarations from modeller
 */
namespace flamegpu_internal {
    __device__ hiprandState *d_random_state;
    __device__ Random::size_type d_random_size;
    hiprandState *hd_random_state;
    Random::size_type hd_random_size;
}
/**
 * Static member vars
 */
uint64_t Random::mSeed = 0;
Random::size_type Random::length = 0;
Random::size_type Random::min_length = 1024;
float Random::growthModifier = 1.5;
float Random::shrinkModifier = 1.0;
hiprandState *Random::h_max_random_state = nullptr;
Random::size_type Random::h_max_random_size = 0;
/**
 * Member fns
 */
uint64_t Random::seedFromTime() {
    return static_cast<uint64_t>(time(nullptr));
}
void Random::init(const uint64_t &seed) {
    Random::mSeed = seed;
    free();
}
void Random::free() {
    // Clear size
    length = 0;
    flamegpu_internal::hd_random_size = 0;
    if (hipMemcpyToSymbol(HIP_SYMBOL(flamegpu_internal::d_random_size), &flamegpu_internal::hd_random_size, sizeof(Random::size_type)) != hipSuccess)
        printf("(%s:%d) CUDA Error initialising hiprand.", __FILE__, __LINE__);
    // Release old
    if (flamegpu_internal::hd_random_state != nullptr && hipFree(flamegpu_internal::hd_random_state) != hipSuccess)
        printf("(%s:%d) CUDA Error Random::~Random().", __FILE__, __LINE__);
    // Update pointers
    flamegpu_internal::hd_random_state = nullptr;
    if (hipMemcpyToSymbol(HIP_SYMBOL(flamegpu_internal::d_random_state), &flamegpu_internal::hd_random_state, sizeof(hiprandState*)) != hipSuccess)
        printf("(%s:%d) CUDA Error Random::~Random().", __FILE__, __LINE__);
    // Release host_max
    if (h_max_random_state)
        ::free(h_max_random_state);
    h_max_random_size = 0;
}

bool Random::resize(const size_type &_length) {
    assert(growthModifier > 1.0);
    assert(shrinkModifier > 0.0);
    assert(shrinkModifier <= 1.0);
    auto t_length = length;
    if (length) {
        while (t_length < _length) {
            t_length = static_cast<Random::size_type>(t_length * growthModifier);
            if (shrinkModifier < 1.0f) {
                while (t_length * shrinkModifier > _length) {
                    t_length = static_cast<Random::size_type>(t_length * shrinkModifier);
                }
            }
        }
    } else {  // Special case for first run
        t_length = _length;
    }
    // Don't allow array to go below Random::min_length elements
    t_length = std::max<size_type>(t_length, Random::min_length);
    if (t_length != length)
        resizeDeviceArray(t_length);
    return t_length != length;
}
__global__ void init_curand(unsigned int threadCount, uint64_t seed, Random::size_type offset) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < threadCount)
        hiprand_init(seed, offset + id, 0, &flamegpu_internal::d_random_state[offset + id]);
}
void Random::resizeDeviceArray(const size_type &_length) {
    if (_length > length) {
        // Growing array
        hiprandState *t_hd_random_state = nullptr;
        // Allocate new mem to t_hd
        if (hipMalloc(&t_hd_random_state, _length * sizeof(hiprandState)) != hipSuccess)
            printf("(%s:%d) CUDA Error Random::resizeDeviceArray().", __FILE__, __LINE__);
        // Copy hd->t_hd[****    ]
        if (flamegpu_internal::hd_random_state)
            if (hipMemcpy(t_hd_random_state, flamegpu_internal::hd_random_state, length * sizeof(hiprandState), hipMemcpyDeviceToDevice))
                printf("(%s:%d) CUDA Error Random::resizeDeviceArray().", __FILE__, __LINE__);
        // Update pointers hd=t_hd
        if (flamegpu_internal::hd_random_state)
            if (hipFree(flamegpu_internal::hd_random_state) != hipSuccess)
                printf("(%s:%d) CUDA Error Random::resizeDeviceArray().", __FILE__, __LINE__);
        flamegpu_internal::hd_random_state = t_hd_random_state;
        if (hipMemcpyToSymbol(HIP_SYMBOL(flamegpu_internal::d_random_state), &flamegpu_internal::hd_random_state, sizeof(hiprandState*)) != hipSuccess)
            printf("(%s:%d) CUDA Error Random::resizeDeviceArray().", __FILE__, __LINE__);
        // Init new[    ****]
        if (h_max_random_size > length) {
            // We have part/all host backup, copy to device array
            // Reinit backup[    **  ]
            size_type copy_len = std::min(h_max_random_size, _length);
            if (hipMemcpy(flamegpu_internal::hd_random_state + length, h_max_random_state + length, copy_len * sizeof(hiprandState), hipMemcpyHostToDevice))
                printf("(%s:%d) CUDA Error Random::resizeDeviceArray().", __FILE__, __LINE__);
            length += copy_len;
        }
        if (_length > length) {
            // Init remainder[     **]
            unsigned int initThreads = 512;
            unsigned int initBlocks = (_length - length / initThreads) + 1;
            init_curand<<<initBlocks, initThreads>>>(_length - length, mSeed, length);  // This could be async with above memcpy?
        }
    } else {
        // Shrinking array
        hiprandState *t_hd_random_state = nullptr;
        hiprandState *t_h_max_random_state = nullptr;
        // Allocate new
        if (hipMalloc(&t_hd_random_state, _length * sizeof(hiprandState)) != hipSuccess)
            printf("(%s:%d) CUDA Error Random::resizeDeviceArray().", __FILE__, __LINE__);
        // Allocate host backup
        if (length > h_max_random_size)
            t_h_max_random_state = reinterpret_cast<hiprandState *>(malloc(length * sizeof(hiprandState)));
        else
            t_h_max_random_state = h_max_random_state;
        // Copy old->new
        assert(flamegpu_internal::hd_random_state);
        if (hipMemcpy(t_hd_random_state, flamegpu_internal::hd_random_state, _length * sizeof(hiprandState), hipMemcpyDeviceToDevice))
            printf("(%s:%d) CUDA Error Random::resizeDeviceArray().", __FILE__, __LINE__);
        // Copy part being shrunk away to host storage (This could be async with above memcpy?)
        if (hipMemcpy(t_h_max_random_state + _length, flamegpu_internal::hd_random_state + _length, (length - _length) * sizeof(hiprandState), hipMemcpyDeviceToHost))
            printf("(%s:%d) CUDA Error Random::resizeDeviceArray().", __FILE__, __LINE__);
        // Release and replace old host ptr
        if (length > h_max_random_size) {
            if (h_max_random_state)
                ::free(h_max_random_state);
            h_max_random_state = t_h_max_random_state;
            h_max_random_size = length;
        }
        // Update pointers
        flamegpu_internal::hd_random_state = t_hd_random_state;
        if (hipMemcpyToSymbol(HIP_SYMBOL(flamegpu_internal::d_random_state), &flamegpu_internal::hd_random_state, sizeof(hiprandState*)) != hipSuccess)
            printf("(%s:%d) CUDA Error Random::resizeDeviceArray().", __FILE__, __LINE__);
        // Release old
        if (flamegpu_internal::hd_random_state != nullptr && hipFree(flamegpu_internal::hd_random_state) != hipSuccess)
            printf("(%s:%d) CUDA Error Random::resizeDeviceArray().", __FILE__, __LINE__);
    }
    // Update length
    length = _length;
    flamegpu_internal::hd_random_size = _length;
    if (hipMemcpyToSymbol(HIP_SYMBOL(flamegpu_internal::d_random_size), &flamegpu_internal::hd_random_size, sizeof(Random::size_type)) != hipSuccess)
        printf("(%s:%d) CUDA Error initialising hiprand.", __FILE__, __LINE__);
}
void Random::setGrowthModifier(float _growthModifier) {
    assert(growthModifier > 1.0);
    Random::growthModifier = _growthModifier;
}
float Random::getGrowthModifier() {
    return Random::growthModifier;
}
void Random::setShrinkModifier(float _shrinkModifier) {
    assert(shrinkModifier > 0.0);
    assert(shrinkModifier <= 1.0);
    Random::shrinkModifier = _shrinkModifier;
}
float Random::getShrinkModifier() {
    return Random::shrinkModifier;
}
Random::size_type Random::size() {
    return length;
}
uint64_t Random::seed() {
    return mSeed;
}
