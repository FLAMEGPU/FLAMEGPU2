#include "hip/hip_runtime.h"
#include "flamegpu/runtime/messaging/MessageSpatial3D/MessageSpatial3DHost.h"
#include "flamegpu/runtime/messaging/MessageSpatial3D/MessageSpatial3DDevice.cuh"
#include "flamegpu/detail/cuda.cuh"
#include "flamegpu/simulation/detail/CUDAScatter.cuh"

#ifdef _MSC_VER
#pragma warning(push, 1)
#pragma warning(disable : 4706 4834)
#endif  // _MSC_VER
#ifdef __NVCC_DIAG_PRAGMA_SUPPORT__
#pragma nv_diag_suppress 1719
#else
#pragma diag_suppress 1719
#endif  // __NVCC_DIAG_PRAGMA_SUPPORT__
#include <hipcub/hipcub.hpp>
#ifdef __NVCC_DIAG_PRAGMA_SUPPORT__
#pragma nv_diag_default 1719
#else
#pragma diag_default 1719
#endif  // __NVCC_DIAG_PRAGMA_SUPPORT__
#ifdef _MSC_VER
#pragma warning(pop)
#endif  // _MSC_VER

namespace flamegpu {
MessageSpatial3D::CUDAModelHandler::CUDAModelHandler(detail::CUDAMessage &a)
  : MessageSpecialisationHandler()
  , sim_message(a) {
    flamegpu::util::nvtx::Range range{"Spatial3D::CUDAModelHandler"};
    const Data &d = (const Data &)a.getMessageData();
    hd_data.radius = d.radius;
    hd_data.min[0] = d.minX;
    hd_data.min[1] = d.minY;
    hd_data.min[2] = d.minZ;
    hd_data.max[0] = d.maxX;
    hd_data.max[1] = d.maxY;
    hd_data.max[2] = d.maxZ;
    binCount = 1;
    for (unsigned int axis = 0; axis < 3; ++axis) {
        hd_data.environmentWidth[axis] = hd_data.max[axis] - hd_data.min[axis];
        hd_data.gridDim[axis] = static_cast<unsigned int>(ceil(hd_data.environmentWidth[axis] / hd_data.radius));
        binCount *= hd_data.gridDim[axis];
    }
    // Device allocation occurs in allocateMetaDataDevicePtr rather than the constructor.
}

__global__ void atomicHistogram3D(
    const MessageSpatial3D::MetaData *md,
    unsigned int* bin_index,
    unsigned int* bin_sub_index,
    unsigned int *pbm_counts,
    unsigned int message_count,
    const float * __restrict__ x,
    const float * __restrict__ y,
    const float * __restrict__ z) {
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    // Kill excess threads
    if (index >= message_count) return;

    MessageSpatial3D::GridPos3D gridPos = getGridPosition3D(md, x[index], y[index], z[index]);
    unsigned int hash = getHash3D(md, gridPos);
    bin_index[index] = hash;
    unsigned int bin_idx = atomicInc((unsigned int*)&pbm_counts[hash], 0xFFFFFFFF);
    bin_sub_index[index] = bin_idx;
}

void MessageSpatial3D::CUDAModelHandler::init(detail::CUDAScatter &, unsigned int, hipStream_t stream) {
    allocateMetaDataDevicePtr(stream);
    // Set PBM to 0
    gpuErrchk(hipMemsetAsync(hd_data.PBM, 0x00000000, (binCount + 1) * sizeof(unsigned int), stream));
    gpuErrchk(hipStreamSynchronize(stream));  // This could probably be skipped/delayed safely
}

void MessageSpatial3D::CUDAModelHandler::allocateMetaDataDevicePtr(hipStream_t stream) {
    if (d_data == nullptr) {
        gpuErrchk(hipMalloc(&d_histogram, (binCount + 1) * sizeof(unsigned int)));
        gpuErrchk(hipMalloc(&hd_data.PBM, (binCount + 1) * sizeof(unsigned int)));
        gpuErrchk(hipMalloc(&d_data, sizeof(MetaData)));
        gpuErrchk(hipMemcpyAsync(d_data, &hd_data, sizeof(MetaData), hipMemcpyHostToDevice, stream));
        gpuErrchk(hipStreamSynchronize(stream));
        resizeCubTemp(stream);
    }
}

void MessageSpatial3D::CUDAModelHandler::freeMetaDataDevicePtr() {
    if (d_data != nullptr) {
        d_CUB_temp_storage_bytes = 0;
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_CUB_temp_storage));
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_histogram));
        gpuErrchk(flamegpu::detail::cuda::hipFree(hd_data.PBM));
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_data));
        d_CUB_temp_storage = nullptr;
        d_histogram = nullptr;
        hd_data.PBM = nullptr;
        d_data = nullptr;
        if (d_keys) {
            d_keys_vals_storage_bytes = 0;
            gpuErrchk(flamegpu::detail::cuda::hipFree(d_keys));
            gpuErrchk(flamegpu::detail::cuda::hipFree(d_vals));
            d_keys = nullptr;
            d_vals = nullptr;
        }
    }
}

void MessageSpatial3D::CUDAModelHandler::buildIndex(detail::CUDAScatter &scatter, unsigned int streamId, hipStream_t stream) {
    flamegpu::util::nvtx::Range range{"MessageSpatial3D::CUDAModelHandler::buildIndex"};
    const unsigned int MESSAGE_COUNT = this->sim_message.getMessageCount();
    if (!MESSAGE_COUNT) {
        gpuErrchk(hipMemsetAsync(hd_data.PBM, 0x00000000, (binCount + 1) * sizeof(unsigned int), stream));
        gpuErrchk(hipStreamSynchronize(stream));
        return;
    }
    resizeKeysVals(this->sim_message.getMaximumListSize());  // Resize based on allocated amount rather than message count
    {  // Build atomic histogram
        gpuErrchk(hipMemsetAsync(d_histogram, 0x00000000, (binCount + 1) * sizeof(unsigned int), stream));
        int blockSize;  // The launch configurator returned block size
        gpuErrchk(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, atomicHistogram3D, 32, 0));  // Randomly 32
                                                                                                         // Round up according to array size
        int gridSize = (MESSAGE_COUNT + blockSize - 1) / blockSize;
        atomicHistogram3D <<<gridSize, blockSize, 0, stream >>>(d_data, d_keys, d_vals, d_histogram, MESSAGE_COUNT,
            reinterpret_cast<float*>(this->sim_message.getReadPtr("x")),
            reinterpret_cast<float*>(this->sim_message.getReadPtr("y")),
            reinterpret_cast<float*>(this->sim_message.getReadPtr("z")));
    }
    {  // Scan (sum), to finalise PBM
        gpuErrchk(hipcub::DeviceScan::ExclusiveSum(d_CUB_temp_storage, d_CUB_temp_storage_bytes, d_histogram, hd_data.PBM, binCount + 1, stream));
    }
    {  // Reorder messages
       // Copy messages from d_messages to d_messages_swap, in hash order
        scatter.pbm_reorder(streamId, stream, this->sim_message.getMessageData().variables, this->sim_message.getReadList(), this->sim_message.getWriteList(), MESSAGE_COUNT, d_keys, d_vals, hd_data.PBM);
        this->sim_message.swap();  // Stream id is unused here
        gpuErrchk(hipStreamSynchronize(stream));  // Not striclty neceesary while pbm_reorder is synchronous.
    }
    {  // Fill PBM and Message Texture Buffers
       // gpuErrchk(hipBindTexture(nullptr, d_texMessages, d_agents, sizeof(glm::vec4) * MESSAGE_COUNT));
       // gpuErrchk(hipBindTexture(nullptr, d_texPBM, d_PBM, sizeof(unsigned int) * (binCount + 1)));
    }
}

void MessageSpatial3D::CUDAModelHandler::resizeCubTemp(hipStream_t stream) {
    size_t bytesCheck = 0;
    gpuErrchk(hipcub::DeviceScan::ExclusiveSum(nullptr, bytesCheck, hd_data.PBM, d_histogram, binCount + 1, stream));
    if (bytesCheck > d_CUB_temp_storage_bytes) {
        if (d_CUB_temp_storage) {
            gpuErrchk(flamegpu::detail::cuda::hipFree(d_CUB_temp_storage));
        }
        d_CUB_temp_storage_bytes = bytesCheck;
        gpuErrchk(hipMalloc(&d_CUB_temp_storage, d_CUB_temp_storage_bytes));
    }
}

void MessageSpatial3D::CUDAModelHandler::resizeKeysVals(const unsigned int newSize) {
    size_t bytesCheck = newSize * sizeof(unsigned int);
    if (bytesCheck > d_keys_vals_storage_bytes) {
        if (d_keys) {
            gpuErrchk(flamegpu::detail::cuda::hipFree(d_keys));
            gpuErrchk(flamegpu::detail::cuda::hipFree(d_vals));
        }
        d_keys_vals_storage_bytes = bytesCheck;
        gpuErrchk(hipMalloc(&d_keys, d_keys_vals_storage_bytes));
        gpuErrchk(hipMalloc(&d_vals, d_keys_vals_storage_bytes));
    }
}
/// <summary>
/// CDescription
/// </summary>
MessageSpatial3D::CDescription::CDescription(std::shared_ptr<Data> data)
    : MessageSpatial2D::CDescription(std::move(std::static_pointer_cast<MessageSpatial2D::Data>(data))) { }
MessageSpatial3D::CDescription::CDescription(std::shared_ptr<const Data> data)
    : CDescription(std::move(std::const_pointer_cast<Data>(data))) { }

bool MessageSpatial3D::CDescription::operator==(const CDescription& rhs) const {
    return *this->message == *rhs.message;  // Compare content is functionally the same
}
bool MessageSpatial3D::CDescription::operator!=(const CDescription& rhs) const {
    return !(*this == rhs);
}
/**
 * Const accessors
 */
float MessageSpatial3D::CDescription::getMinZ() const {
    return std::static_pointer_cast<Data>(message)->minZ;
}
float MessageSpatial3D::CDescription::getMaxZ() const {
    return std::static_pointer_cast<Data>(message)->maxZ;
}

/// <summary>
/// Description
/// </summary>
MessageSpatial3D::Description::Description(std::shared_ptr<Data> data)
    : CDescription(data) { }
/**
 * Accessors
 */
void MessageSpatial3D::Description::setMinZ(const float z) {
    if (!isnan(std::static_pointer_cast<Data>(message)->maxZ) &&
        z >= std::static_pointer_cast<Data>(message)->maxZ) {
        THROW exception::InvalidArgument("Spatial messaging min z bound must be lower than max bound, %f !< %f", z, std::static_pointer_cast<Data>(message)->maxZ);
    }
    std::static_pointer_cast<Data>(message)->minZ = z;
}
void MessageSpatial3D::Description::setMin(const float x, const float y, const float z) {
    if (!isnan(std::static_pointer_cast<Data>(message)->maxX) &&
        x >= std::static_pointer_cast<Data>(message)->maxX) {
        THROW exception::InvalidArgument("Spatial messaging min x bound must be lower than max bound, %f !< %f", x, std::static_pointer_cast<Data>(message)->maxX);
    }
    if (!isnan(std::static_pointer_cast<Data>(message)->maxY) &&
        y >= std::static_pointer_cast<Data>(message)->maxY) {
        THROW exception::InvalidArgument("Spatial messaging min y bound must be lower than max bound, %f !< %f", y, std::static_pointer_cast<Data>(message)->maxY);
    }
    if (!isnan(std::static_pointer_cast<Data>(message)->maxZ) &&
        z >= std::static_pointer_cast<Data>(message)->maxZ) {
        THROW exception::InvalidArgument("Spatial messaging min z bound must be lower than max bound, %f !< %f", z, std::static_pointer_cast<Data>(message)->maxZ);
    }
    std::static_pointer_cast<Data>(message)->minX = x;
    std::static_pointer_cast<Data>(message)->minY = y;
    std::static_pointer_cast<Data>(message)->minZ = z;
}
void MessageSpatial3D::Description::setMaxZ(const float z) {
    if (!isnan(std::static_pointer_cast<Data>(message)->minZ) &&
        z <= std::static_pointer_cast<Data>(message)->minZ) {
        THROW exception::InvalidArgument("Spatial messaging max z bound must be greater than min bound, %f !> %f", z, std::static_pointer_cast<Data>(message)->minZ);
    }
    std::static_pointer_cast<Data>(message)->maxZ = z;
}
void MessageSpatial3D::Description::setMax(const float x, const float y, const float z) {
    if (!isnan(std::static_pointer_cast<Data>(message)->minX) &&
        x <= std::static_pointer_cast<Data>(message)->minX) {
        THROW exception::InvalidArgument("Spatial messaging max x bound must be greater than min bound, %f !> %f", x, std::static_pointer_cast<Data>(message)->minX);
    }
    if (!isnan(std::static_pointer_cast<Data>(message)->minY) &&
        y <= std::static_pointer_cast<Data>(message)->minY) {
        THROW exception::InvalidArgument("Spatial messaging max y bound must be greater than min bound, %f !> %f", y, std::static_pointer_cast<Data>(message)->minY);
    }
    if (!isnan(std::static_pointer_cast<Data>(message)->minZ) &&
        z <= std::static_pointer_cast<Data>(message)->minZ) {
        THROW exception::InvalidArgument("Spatial messaging max z bound must be greater than min bound, %f !> %f", z, std::static_pointer_cast<Data>(message)->minZ);
    }
    std::static_pointer_cast<Data>(message)->maxX = x;
    std::static_pointer_cast<Data>(message)->maxY = y;
    std::static_pointer_cast<Data>(message)->maxZ = z;
}

/// <summary>
/// Data
/// </summary>
MessageSpatial3D::Data::Data(std::shared_ptr<const ModelData> model, const std::string &message_name)
    : MessageSpatial2D::Data(model, message_name)
    , minZ(NAN)
    , maxZ(NAN) {
    // MessageSpatial3D has x/y/z variables by default (x/y are inherited)
    variables.emplace("z", Variable(std::array<typename detail::type_decode<float>::type_t, 1>{}));
}
MessageSpatial3D::Data::Data(std::shared_ptr<const ModelData> model, const Data &other)
    : MessageSpatial2D::Data(model, other)
    , minZ(other.minZ)
    , maxZ(other.maxZ) {
    if (isnan(minZ)) {
        THROW exception::InvalidMessage("Environment minimum z bound has not been set in spatial message '%s'\n", other.name.c_str());
    }
    if (isnan(maxZ)) {
        THROW exception::InvalidMessage("Environment maximum z bound has not been set in spatial message '%s'\n", other.name.c_str());
    }
}
MessageSpatial3D::Data *MessageSpatial3D::Data::clone(const std::shared_ptr<const ModelData> &newParent) {
    return new Data(newParent, *this);
}
std::unique_ptr<MessageSpecialisationHandler> MessageSpatial3D::Data::getSpecialisationHander(detail::CUDAMessage &owner) const {
    return std::unique_ptr<MessageSpecialisationHandler>(new CUDAModelHandler(owner));
}
std::type_index MessageSpatial3D::Data::getType() const { return std::type_index(typeid(MessageSpatial3D)); }

flamegpu::MessageSortingType MessageSpatial3D::Data::getSortingType() const {
    return flamegpu::MessageSortingType::spatial3D;
}

}  // namespace flamegpu
