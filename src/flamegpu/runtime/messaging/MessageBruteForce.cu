#include "flamegpu/runtime/messaging/MessageBruteForce/MessageBruteForceHost.h"
#include "flamegpu/runtime/messaging/MessageBruteForce/MessageBruteForceDevice.cuh"
#include "flamegpu/model/AgentDescription.h"  // Used by Move-Assign
#include "flamegpu/gpu/CUDAMessage.h"

namespace flamegpu {

void MessageBruteForce::CUDAModelHandler::init(CUDAScatter &, unsigned int, hipStream_t stream) {
    allocateMetaDataDevicePtr(stream);
    // Allocate messages
    hd_metadata.length = 0;  // This value should already be 0
    gpuErrchk(hipMemcpyAsync(d_metadata, &hd_metadata, sizeof(MetaData), hipMemcpyHostToDevice, stream));
    gpuErrchk(hipStreamSynchronize(stream));  // This could probably be skipped/delayed safely
}

void MessageBruteForce::CUDAModelHandler::allocateMetaDataDevicePtr(hipStream_t stream) {
    if (d_metadata == nullptr) {
        gpuErrchk(hipMalloc(&d_metadata, sizeof(MetaData)));
    }
}

void MessageBruteForce::CUDAModelHandler::freeMetaDataDevicePtr() {
    if (d_metadata != nullptr) {
        gpuErrchk(hipFree(d_metadata));
    }
    d_metadata = nullptr;
}

void MessageBruteForce::CUDAModelHandler::buildIndex(CUDAScatter &, unsigned int, hipStream_t stream) {
    unsigned int newLength = this->sim_message.getMessageCount();
    if (newLength != hd_metadata.length) {
        hd_metadata.length = newLength;
        gpuErrchk(hipMemcpyAsync(d_metadata, &hd_metadata, sizeof(MetaData), hipMemcpyHostToDevice, stream));  // Not Pinned
        gpuErrchk(hipStreamSynchronize(stream));  // This could probably be skipped/delayed safely if in the right stream
    }
}

MessageBruteForce::Data::Data(const std::shared_ptr<const ModelData> &model, const std::string &message_name)
    : description(new Description(model, this))
    , name(message_name)
    , optional_outputs(0) { }
MessageBruteForce::Data::~Data() {}
MessageBruteForce::Data::Data(const std::shared_ptr<const ModelData> &model, const MessageBruteForce::Data &other)
    : variables(other.variables)
    , description(model ? new Description(model, this) : nullptr)
    , name(other.name)
    , optional_outputs(other.optional_outputs) { }
MessageBruteForce::Data *MessageBruteForce::Data::clone(const std::shared_ptr<const ModelData> &newParent) {
    return new MessageBruteForce::Data(newParent, *this);
}
bool MessageBruteForce::Data::operator==(const MessageBruteForce::Data& rhs) const {
    if (this == &rhs)  // They point to same object
        return true;
    if (name == rhs.name
        && variables.size() == rhs.variables.size()) {
            {  // Compare variables
                for (auto &v : variables) {
                    auto _v = rhs.variables.find(v.first);
                    if (_v == rhs.variables.end())
                        return false;
                    if (v.second.type_size != _v->second.type_size
                        || v.second.type != _v->second.type
                        || v.second.elements != _v->second.elements)
                        return false;
                }
            }
            return true;
    }
    return false;
}
bool MessageBruteForce::Data::operator!=(const MessageBruteForce::Data& rhs) const {
    return !operator==(rhs);
}

std::unique_ptr<MessageSpecialisationHandler> MessageBruteForce::Data::getSpecialisationHander(CUDAMessage &owner) const {
    return std::unique_ptr<MessageSpecialisationHandler>(new MessageBruteForce::CUDAModelHandler(owner));
}

flamegpu::MessageSortingType flamegpu::MessageBruteForce::Data::getSortingType() const {
    return flamegpu::MessageSortingType::none;
}

// Used for the MessageBruteForce::Data::getType() type and derived methods
std::type_index MessageBruteForce::Data::getType() const { return std::type_index(typeid(MessageBruteForce)); }


/**
* Constructors
*/
MessageBruteForce::Description::Description(const std::shared_ptr<const ModelData> &_model, Data *const description)
    : model(_model)
    , message(description) { }

bool MessageBruteForce::Description::operator==(const MessageBruteForce::Description& rhs) const {
    return *this->message == *rhs.message;  // Compare content is functionally the same
}
bool MessageBruteForce::Description::operator!=(const MessageBruteForce::Description& rhs) const {
    return !(*this == rhs);
}

/**
* Const Accessors
*/
std::string MessageBruteForce::Description::getName() const {
    return message->name;
}

const std::type_index& MessageBruteForce::Description::getVariableType(const std::string &variable_name) const {
    auto f = message->variables.find(variable_name);
    if (f != message->variables.end()) {
        return f->second.type;
    }
    THROW exception::InvalidMessageVar("Message ('%s') does not contain variable '%s', "
        "in MessageDescription::getVariableType().",
        message->name.c_str(), variable_name.c_str());
}
size_t MessageBruteForce::Description::getVariableSize(const std::string &variable_name) const {
    auto f = message->variables.find(variable_name);
    if (f != message->variables.end()) {
        return f->second.type_size;
    }
    THROW exception::InvalidMessageVar("Message ('%s') does not contain variable '%s', "
        "in MessageDescription::getVariableSize().",
        message->name.c_str(), variable_name.c_str());
}
flamegpu::size_type MessageBruteForce::Description::getVariableLength(const std::string &variable_name) const {
    auto f = message->variables.find(variable_name);
    if (f != message->variables.end()) {
        return f->second.elements;
    }
    THROW exception::InvalidAgentVar("Message ('%s') does not contain variable '%s', "
        "in MessageBruteForce::getVariableLength().",
        message->name.c_str(), variable_name.c_str());
}
flamegpu::size_type MessageBruteForce::Description::getVariablesCount() const {
    // Downcast, will never have more than UINT_MAX variables
    return static_cast<flamegpu::size_type>(message->variables.size());
}
bool MessageBruteForce::Description::hasVariable(const std::string &variable_name) const {
    return message->variables.find(variable_name) != message->variables.end();
}

}  // namespace flamegpu
