#include "flamegpu/runtime/messaging/MessageBruteForce/MessageBruteForceHost.h"
#include "flamegpu/runtime/messaging/MessageBruteForce/MessageBruteForceDevice.cuh"
#include "flamegpu/model/AgentDescription.h"  // Used by Move-Assign
#include "flamegpu/gpu/CUDAMessage.h"
#include "flamegpu/util/detail/cuda.cuh"

namespace flamegpu {
void MessageBruteForce::CUDAModelHandler::init(CUDAScatter &, unsigned int, hipStream_t stream) {
    allocateMetaDataDevicePtr(stream);
    // Allocate messages
    hd_metadata.length = 0;  // This value should already be 0
    gpuErrchk(hipMemcpyAsync(d_metadata, &hd_metadata, sizeof(MetaData), hipMemcpyHostToDevice, stream));
    gpuErrchk(hipStreamSynchronize(stream));  // This could probably be skipped/delayed safely
}

void MessageBruteForce::CUDAModelHandler::allocateMetaDataDevicePtr(hipStream_t stream) {
    if (d_metadata == nullptr) {
        gpuErrchk(hipMalloc(&d_metadata, sizeof(MetaData)));
    }
}

void MessageBruteForce::CUDAModelHandler::freeMetaDataDevicePtr() {
    if (d_metadata != nullptr) {
        gpuErrchk(flamegpu::util::detail::cuda::hipFree(d_metadata));
    }
    d_metadata = nullptr;
}

void MessageBruteForce::CUDAModelHandler::buildIndex(CUDAScatter &, unsigned int, hipStream_t stream) {
    unsigned int newLength = this->sim_message.getMessageCount();
    if (newLength != hd_metadata.length) {
        hd_metadata.length = newLength;
        gpuErrchk(hipMemcpyAsync(d_metadata, &hd_metadata, sizeof(MetaData), hipMemcpyHostToDevice, stream));  // Not Pinned
        gpuErrchk(hipStreamSynchronize(stream));  // This could probably be skipped/delayed safely if in the right stream
    }
}

/// <summary>
///  Data
/// </summary>
MessageBruteForce::Data::Data(std::shared_ptr<const ModelData> _model, const std::string &message_name)
    : model(_model)
    , name(message_name)
    , optional_outputs(0) { }
MessageBruteForce::Data::Data(std::shared_ptr<const ModelData> _model, const MessageBruteForce::Data &other)
    : model(_model)
    , variables(other.variables)
    , name(other.name)
    , optional_outputs(other.optional_outputs) { }
MessageBruteForce::Data *MessageBruteForce::Data::clone(const std::shared_ptr<const ModelData> &newParent) {
    return new MessageBruteForce::Data(newParent, *this);
}
bool MessageBruteForce::Data::operator==(const MessageBruteForce::Data& rhs) const {
    if (this == &rhs)  // They point to same object
        return true;
    if (name == rhs.name
        // && model.lock() == rhs.model.lock()  // Don't check weak pointers
        && variables.size() == rhs.variables.size()) {
            {  // Compare variables
                for (auto &v : variables) {
                    auto _v = rhs.variables.find(v.first);
                    if (_v == rhs.variables.end())
                        return false;
                    if (v.second.type_size != _v->second.type_size
                        || v.second.type != _v->second.type
                        || v.second.elements != _v->second.elements)
                        return false;
                }
            }
            return true;
    }
    return false;
}
bool MessageBruteForce::Data::operator!=(const MessageBruteForce::Data& rhs) const {
    return !operator==(rhs);
}

std::unique_ptr<MessageSpecialisationHandler> MessageBruteForce::Data::getSpecialisationHander(CUDAMessage &owner) const {
    return std::unique_ptr<MessageSpecialisationHandler>(new MessageBruteForce::CUDAModelHandler(owner));
}

flamegpu::MessageSortingType flamegpu::MessageBruteForce::Data::getSortingType() const {
    return flamegpu::MessageSortingType::none;
}

// Used for the MessageBruteForce::Data::getType() type and derived methods
std::type_index MessageBruteForce::Data::getType() const { return std::type_index(typeid(MessageBruteForce)); }


/// <summary>
///  CDescription
/// </summary>
MessageBruteForce::CDescription::CDescription(std::shared_ptr<Data> data)
    : message(std::move(data)) { }
MessageBruteForce::CDescription::CDescription(std::shared_ptr<const Data> data)
    : message(std::move(std::const_pointer_cast<Data>(data))) { }

bool MessageBruteForce::CDescription::operator==(const CDescription& rhs) const {
    return *this->message == *rhs.message;  // Compare content is functionally the same
}
bool MessageBruteForce::CDescription::operator!=(const CDescription& rhs) const {
    return !(*this == rhs);
}

/**
 * Const Accessors
 */
std::string MessageBruteForce::CDescription::getName() const {
    return message->name;
}

const std::type_index& MessageBruteForce::CDescription::getVariableType(const std::string& variable_name) const {
    auto f = message->variables.find(variable_name);
    if (f != message->variables.end()) {
        return f->second.type;
    }
    THROW exception::InvalidMessageVar("Message ('%s') does not contain variable '%s', "
        "in MessageDescription::getVariableType().",
        message->name.c_str(), variable_name.c_str());
}
size_t MessageBruteForce::CDescription::getVariableSize(const std::string& variable_name) const {
    auto f = message->variables.find(variable_name);
    if (f != message->variables.end()) {
        return f->second.type_size;
    }
    THROW exception::InvalidMessageVar("Message ('%s') does not contain variable '%s', "
        "in MessageDescription::getVariableSize().",
        message->name.c_str(), variable_name.c_str());
}
flamegpu::size_type MessageBruteForce::CDescription::getVariableLength(const std::string& variable_name) const {
    auto f = message->variables.find(variable_name);
    if (f != message->variables.end()) {
        return f->second.elements;
    }
    THROW exception::InvalidAgentVar("Message ('%s') does not contain variable '%s', "
        "in MessageBruteForce::getVariableLength().",
        message->name.c_str(), variable_name.c_str());
}
flamegpu::size_type MessageBruteForce::CDescription::getVariablesCount() const {
    // Downcast, will never have more than UINT_MAX variables
    return static_cast<flamegpu::size_type>(message->variables.size());
}
bool MessageBruteForce::CDescription::hasVariable(const std::string& variable_name) const {
    return message->variables.find(variable_name) != message->variables.end();
}

/// <summary>
///  Description
/// </summary>
MessageBruteForce::Description::Description(std::shared_ptr<Data> data)
    : CDescription(std::move(data)) { }

}  // namespace flamegpu
