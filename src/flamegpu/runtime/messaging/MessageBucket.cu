#include "hip/hip_runtime.h"
#include "flamegpu/runtime/messaging/MessageBucket.h"

#ifdef _MSC_VER
#pragma warning(push, 1)
#pragma warning(disable : 4706 4834)
#endif  // _MSC_VER
#ifdef __NVCC_DIAG_PRAGMA_SUPPORT__
#pragma nv_diag_suppress 1719
#else
#pragma diag_suppress 1719
#endif  // __NVCC_DIAG_PRAGMA_SUPPORT__
#include <hipcub/hipcub.hpp>
#ifdef __NVCC_DIAG_PRAGMA_SUPPORT__
#pragma nv_diag_default 1719
#else
#pragma diag_default 1719
#endif  // __NVCC_DIAG_PRAGMA_SUPPORT__
#ifdef _MSC_VER
#pragma warning(pop)
#endif  // _MSC_VER

#include "flamegpu/model/AgentDescription.h"
#include "flamegpu/simulation/detail/CUDAMessage.h"
#include "flamegpu/simulation/detail/CUDAScatter.cuh"
#include "flamegpu/util/nvtx.h"

#include "flamegpu/runtime/messaging/MessageBucket/MessageBucketHost.h"
// #include "flamegpu/runtime/messaging/MessageBucket/MessageBucketDevice.cuh"
#include "flamegpu/detail/cuda.cuh"

namespace flamegpu {
MessageBucket::CUDAModelHandler::CUDAModelHandler(detail::CUDAMessage &a)
    : MessageSpecialisationHandler()
    , sim_message(a) {
    flamegpu::util::nvtx::Range range{"MessageBucket::CUDAModelHandler::CUDAModelHandler"};
    const Data &d = (const Data &)a.getMessageData();
    hd_data.min = d.lowerBound;
    // Here we convert it so that upperBound is one greater than the final valid index
    hd_data.max = d.upperBound + 1;
    bucketCount = d.upperBound - d.lowerBound  + 1;
}
MessageBucket::CUDAModelHandler::~CUDAModelHandler() { }

__global__ void atomicHistogram1D(
    const MessageBucket::MetaData *md,
    unsigned int* bin_index,
    unsigned int* bin_sub_index,
    unsigned int *pbm_counts,
    unsigned int message_count,
    const IntT * __restrict__ key) {
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    // Kill excess threads
    if (index >= message_count) return;

    const unsigned int hash = key[index] - md->min;
    bin_index[index] = hash;
    unsigned int bin_idx = atomicInc((unsigned int*)&pbm_counts[hash], 0xFFFFFFFF);
    bin_sub_index[index] = bin_idx;
}

void MessageBucket::CUDAModelHandler::init(detail::CUDAScatter &, unsigned int, hipStream_t stream) {
    allocateMetaDataDevicePtr(stream);
    // Set PBM to 0
    gpuErrchk(hipMemsetAsync(hd_data.PBM, 0x00000000, (bucketCount + 1) * sizeof(unsigned int), stream));
    gpuErrchk(hipStreamSynchronize(stream));
}

void MessageBucket::CUDAModelHandler::allocateMetaDataDevicePtr(hipStream_t stream) {
    if (d_data == nullptr) {
        gpuErrchk(hipMalloc(&d_histogram, (bucketCount + 1) * sizeof(unsigned int)));
        gpuErrchk(hipMalloc(&hd_data.PBM, (bucketCount + 1) * sizeof(unsigned int)));
        gpuErrchk(hipMalloc(&d_data, sizeof(MetaData)));
        gpuErrchk(hipMemcpyAsync(d_data, &hd_data, sizeof(MetaData), hipMemcpyHostToDevice, stream));
        gpuErrchk(hipStreamSynchronize(stream));
        resizeCubTemp();
    }
}

void MessageBucket::CUDAModelHandler::freeMetaDataDevicePtr() {
    if (d_data != nullptr) {
        d_CUB_temp_storage_bytes = 0;
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_CUB_temp_storage));
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_histogram));
        gpuErrchk(flamegpu::detail::cuda::hipFree(hd_data.PBM));
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_data));
        d_CUB_temp_storage = nullptr;
        d_histogram = nullptr;
        hd_data.PBM = nullptr;
        d_data = nullptr;
        if (d_keys) {
            d_keys_vals_storage_bytes = 0;
            gpuErrchk(flamegpu::detail::cuda::hipFree(d_keys));
            gpuErrchk(flamegpu::detail::cuda::hipFree(d_vals));
            d_keys = nullptr;
            d_vals = nullptr;
        }
    }
}

void MessageBucket::CUDAModelHandler::buildIndex(detail::CUDAScatter &scatter, unsigned int streamId, hipStream_t stream) {
    flamegpu::util::nvtx::Range range{"MessageBucket::CUDAModelHandler::buildIndex"};
    // Cuda operations all occur within the stream, so only a final sync is required.s
    const unsigned int MESSAGE_COUNT = this->sim_message.getMessageCount();
    resizeKeysVals(this->sim_message.getMaximumListSize());  // Resize based on allocated amount rather than message count
    {  // Build atomic histogram
        gpuErrchk(hipMemsetAsync(d_histogram, 0x00000000, (bucketCount + 1) * sizeof(unsigned int), stream));
        int blockSize;  // The launch configurator returned block size
        gpuErrchk(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, atomicHistogram1D, 32, 0));  // Randomly 32
                                                                                                         // Round up according to array size
        int gridSize = (MESSAGE_COUNT + blockSize - 1) / blockSize;
        atomicHistogram1D <<<gridSize, blockSize, 0, stream >>>(d_data, d_keys, d_vals, d_histogram, MESSAGE_COUNT,
            reinterpret_cast<IntT*>(this->sim_message.getReadPtr("_key")));
    }
    {  // Scan (sum), to finalise PBM
        gpuErrchk(hipcub::DeviceScan::ExclusiveSum(d_CUB_temp_storage, d_CUB_temp_storage_bytes, d_histogram, hd_data.PBM, bucketCount + 1, stream));
    }
    {  // Reorder messages
       // Copy messages from d_messages to d_messages_swap, in hash order
        scatter.pbm_reorder(streamId, stream, this->sim_message.getMessageData().variables, this->sim_message.getReadList(), this->sim_message.getWriteList(), MESSAGE_COUNT, d_keys, d_vals, hd_data.PBM);
        this->sim_message.swap();
        gpuErrchk(hipStreamSynchronize(stream));  // Not strictly necessary while pbm_reorder is synchronous.
    }
    {  // Fill PBM and Message Texture Buffers
       // gpuErrchk(hipBindTexture(nullptr, d_texMessages, d_agents, sizeof(glm::vec4) * MESSAGE_COUNT));
       // gpuErrchk(hipBindTexture(nullptr, d_texPBM, d_PBM, sizeof(unsigned int) * (bucketCount + 1)));
    }
}

void MessageBucket::CUDAModelHandler::resizeCubTemp() {
    size_t bytesCheck = 0;
    gpuErrchk(hipcub::DeviceScan::ExclusiveSum(nullptr, bytesCheck, hd_data.PBM, d_histogram, bucketCount + 1));
    if (bytesCheck > d_CUB_temp_storage_bytes) {
        if (d_CUB_temp_storage) {
            gpuErrchk(flamegpu::detail::cuda::hipFree(d_CUB_temp_storage));
        }
        d_CUB_temp_storage_bytes = bytesCheck;
        gpuErrchk(hipMalloc(&d_CUB_temp_storage, d_CUB_temp_storage_bytes));
    }
}

void MessageBucket::CUDAModelHandler::resizeKeysVals(const unsigned int newSize) {
    size_t bytesCheck = newSize * sizeof(unsigned int);
    if (bytesCheck > d_keys_vals_storage_bytes) {
        if (d_keys) {
            gpuErrchk(flamegpu::detail::cuda::hipFree(d_keys));
            gpuErrchk(flamegpu::detail::cuda::hipFree(d_vals));
        }
        d_keys_vals_storage_bytes = bytesCheck;
        gpuErrchk(hipMalloc(&d_keys, d_keys_vals_storage_bytes));
        gpuErrchk(hipMalloc(&d_vals, d_keys_vals_storage_bytes));
    }
}

/// <summary>
/// CDescription
/// </summary>
MessageBucket::CDescription::CDescription(std::shared_ptr<Data> data)
    : MessageBruteForce::CDescription(std::move(std::static_pointer_cast<MessageBruteForce::Data>(data))) { }
MessageBucket::CDescription::CDescription(std::shared_ptr<const Data> data)
    : CDescription(std::move(std::const_pointer_cast<Data>(data))) { }

bool MessageBucket::CDescription::operator==(const CDescription& rhs) const {
    return *this->message == *rhs.message;  // Compare content is functionally the same
}
bool MessageBucket::CDescription::operator!=(const CDescription& rhs) const {
    return !(*this == rhs);
}
/**
 * Const accessors
 */
IntT MessageBucket::CDescription::getLowerBound() const {
    return std::static_pointer_cast<Data>(message)->lowerBound;
}
IntT MessageBucket::CDescription::getUpperBound() const {
    return std::static_pointer_cast<Data>(message)->upperBound;
}

/// <summary>
/// Description
/// </summary>
MessageBucket::Description::Description(std::shared_ptr<Data> data)
    : CDescription(data) { }
/**
 * Accessors
 */
void MessageBucket::Description::setLowerBound(const IntT min) {
    if (std::static_pointer_cast<Data>(message)->upperBound != std::numeric_limits<IntT>::max() &&
        min >= std::static_pointer_cast<Data>(message)->upperBound) {
        THROW exception::InvalidArgument("Bucket messaging minimum bound must be lower than upper bound, %lld !< %lld.", min, static_cast<int64_t>(std::static_pointer_cast<Data>(message)->upperBound));
    }
    std::static_pointer_cast<Data>(message)->lowerBound = min;
}
void MessageBucket::Description::setUpperBound(const IntT max) {
    if (max <= std::static_pointer_cast<Data>(message)->lowerBound) {
        THROW exception::InvalidArgument("Bucket messaging upperBound bound must be greater than lower bound, %lld !> %lld.", static_cast<int64_t>(max), static_cast<int64_t>(std::static_pointer_cast<Data>(message)->lowerBound));
    }
    std::static_pointer_cast<Data>(message)->upperBound = max;
}
void MessageBucket::Description::setBounds(const IntT min, const IntT max) {
    if (max <= min) {
        THROW exception::InvalidArgument("Bucket messaging upperBound bound must be greater than lower bound, %lld !> %lld.", static_cast<int64_t>(max), static_cast<int64_t>(min));
    }
    std::static_pointer_cast<Data>(message)->lowerBound = min;
    std::static_pointer_cast<Data>(message)->upperBound = max;
}
/// <summary>
/// Data
/// </summary>
MessageBucket::Data::Data(std::shared_ptr<const ModelData> model, const std::string &message_name)
    : MessageBruteForce::Data(model, message_name)
    , lowerBound(0)
    , upperBound(std::numeric_limits<IntT>::max()) {
    variables.emplace("_key", Variable(1, static_cast<IntT>(0)));
}
MessageBucket::Data::Data(std::shared_ptr<const ModelData> model, const Data &other)
    : MessageBruteForce::Data(model, other)
    , lowerBound(other.lowerBound)
    , upperBound(other.upperBound) {
    if (lowerBound == std::numeric_limits<IntT>::max()) {
        THROW exception::InvalidMessage("Minimum bound has not been set for bucket message '%s.", other.name.c_str());
    }
    if (upperBound == std::numeric_limits<IntT>::max()) {
        THROW exception::InvalidMessage("Maximum bound has not been set for bucket message '%s.", other.name.c_str());
    }
}
MessageBucket::Data *MessageBucket::Data::clone(const std::shared_ptr<const ModelData> &newParent) {
    return new Data(newParent, *this);
}
std::unique_ptr<MessageSpecialisationHandler> MessageBucket::Data::getSpecialisationHander(detail::CUDAMessage &owner) const {
    return std::unique_ptr<MessageSpecialisationHandler>(new CUDAModelHandler(owner));
}
std::type_index MessageBucket::Data::getType() const { return std::type_index(typeid(MessageBucket)); }




}  // namespace flamegpu
