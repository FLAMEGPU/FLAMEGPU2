#include "hip/hip_runtime.h"
#include "flamegpu/runtime/messaging/Spatial3D.h"

#include "flamegpu/gpu/CUDAScatter.h"
#ifdef _MSC_VER
#pragma warning(push, 3)
#include <hipcub/hipcub.hpp>
#pragma warning(pop)
#else
#include <hipcub/hipcub.hpp>
#endif

__device__ __forceinline__ MsgSpatial3D::GridPos3D getGridPosition(const MsgSpatial3D::MetaData *md, float x, float y, float z) {
    // Clamp each grid coord to 0<=x<dim
    int gridPos[3] = {
        static_cast<int>(floor((x / md->environmentWidth[0])*md->gridDim[0])),
        static_cast<int>(floor((y / md->environmentWidth[1])*md->gridDim[1])),
        static_cast<int>(floor((z / md->environmentWidth[2])*md->gridDim[2]))
    };
    MsgSpatial3D::GridPos3D rtn = {
        gridPos[0] < 0 ? 0 : (gridPos[0] >= md->gridDim[0] ? static_cast<int>(md->gridDim[0] - 1) : gridPos[0]),
        gridPos[1] < 0 ? 0 : (gridPos[1] >= md->gridDim[1] ? static_cast<int>(md->gridDim[1] - 1) : gridPos[1]),
        gridPos[2] < 0 ? 0 : (gridPos[2] >= md->gridDim[2] ? static_cast<int>(md->gridDim[2] - 1) : gridPos[2])
    };
    return rtn;
}
__device__ __forceinline__ unsigned int getHash(const MsgSpatial3D::MetaData *md, const MsgSpatial3D::GridPos3D &xyz) {
    // Bound gridPos to gridDimensions
    unsigned int gridPos[3] = {
        xyz.x < 0 ? 0 : (xyz.x >= md->gridDim[0] - 1 ? md->gridDim[0] - 1 : xyz.x),  // Only x should ever be out of bounds here
        xyz.y,  // xyz.y < 0 ? 0 : (xyz.y >= md->gridDim[1] - 1 ? md->gridDim[1] - 1 : xyz.y),
        xyz.z,  // xyz.z < 0 ? 0 : (xyz.z >= md->gridDim[2] - 1 ? md->gridDim[2] - 1 : xyz.z)
    };
    // Compute hash (effectivley an index for to a bin within the partitioning grid in this case)
    return (unsigned int)(
        (gridPos[2] * md->gridDim[0] * md->gridDim[1]) +   // z
        (gridPos[1] * md->gridDim[0]) +                    // y
        gridPos[0]);                                      // x
}

__device__ void MsgSpatial3D::Out::setLocation(const float &x, const float &y, const float &z) const {
    unsigned int index = (blockDim.x * blockIdx.x) + threadIdx.x;  // + d_message_count;

    // set the variables using curve
    Curve::setVariable<float>("x", combined_hash, x, index);
    Curve::setVariable<float>("y", combined_hash, y, index);
    Curve::setVariable<float>("z", combined_hash, z, index);

    // Set scan flag incase the message is optional
    flamegpu_internal::CUDAScanCompaction::ds_message_configs[streamId].scan_flag[index] = 1;
}

__device__ MsgSpatial3D::In::Filter::Filter(const MetaData* _metadata, const Curve::NamespaceHash &_combined_hash, const float& x, const float& y, const float& z)
    : metadata(_metadata)
    , combined_hash(_combined_hash) {
    loc[0] = x;
    loc[1] = y;
    loc[2] = z;
    cell = getGridPosition(_metadata, x, y, z);
}
__device__ MsgSpatial3D::In::Filter::Message& MsgSpatial3D::In::Filter::Message::operator++() {
    cell_index++;
    bool move_strip = cell_index >= cell_index_max;
    while (move_strip) {
        nextStrip();
        cell_index = 0;
        cell_index_max = 1;
        if (relative_cell[0] < 2) {
            // Calculate the strips start and end hash
            int absolute_cell[2] = { _parent.cell.y + relative_cell[0], _parent.cell.z + relative_cell[1] };
            // Skip the strip if it is completely out of bounds
            if (absolute_cell[0] >= 0 && absolute_cell[1] >= 0 && absolute_cell[0] < _parent.metadata->gridDim[1] && absolute_cell[1] < _parent.metadata->gridDim[2]) {
                unsigned int start_hash = getHash(_parent.metadata, { _parent.cell.x - 1, absolute_cell[0], absolute_cell[1] });
                unsigned int end_hash = getHash(_parent.metadata, { _parent.cell.x + 1, absolute_cell[0], absolute_cell[1] });
                // Lookup start and end indicies from PBM
                cell_index = _parent.metadata->PBM[start_hash];
                cell_index_max = _parent.metadata->PBM[end_hash + 1];
            } else {
                // Goto next strip
                // Don't update move_strip
                continue;
            }
        }
        move_strip = cell_index >= cell_index_max;
    }
    return *this;
}


__global__ void atomicHistogram3D(
    const MsgSpatial3D::MetaData *md,
    unsigned int* bin_index,
    unsigned int* bin_sub_index,
    unsigned int *pbm_counts,
    unsigned int message_count,
    const float * __restrict__ x,
    const float * __restrict__ y,
    const float * __restrict__ z) {
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    // Kill excess threads
    if (index >= message_count) return;

    MsgSpatial3D::GridPos3D gridPos = getGridPosition(md, x[index], y[index], z[index]);
    unsigned int hash = getHash(md, gridPos);
    bin_index[index] = hash;
    unsigned int bin_idx = atomicInc((unsigned int*)&pbm_counts[hash], 0xFFFFFFFF);
    bin_sub_index[index] = bin_idx;
}

void MsgSpatial3D::CUDAModelHandler::buildIndex() {
    const unsigned int MESSAGE_COUNT = this->sim_message.getMessageCount();
    resizeKeysVals(this->sim_message.getMaximumListSize());  // Resize based on allocated amount rather than message count
    {  // Build atomic histogram
        gpuErrchk(hipMemset(d_histogram, 0x00000000, (binCount + 1) * sizeof(unsigned int)));
        int blockSize;  // The launch configurator returned block size
        gpuErrchk(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, atomicHistogram3D, 32, 0));  // Randomly 32
                                                                                                         // Round up according to array size
        int gridSize = (MESSAGE_COUNT + blockSize - 1) / blockSize;
        atomicHistogram3D << <gridSize, blockSize >> >(d_data, d_keys, d_vals, d_histogram, MESSAGE_COUNT,
            reinterpret_cast<float*>(this->sim_message.getReadPtr("x")),
            reinterpret_cast<float*>(this->sim_message.getReadPtr("y")),
            reinterpret_cast<float*>(this->sim_message.getReadPtr("z")));
        gpuErrchk(hipDeviceSynchronize());
    }
    {  // Scan (sum), to finalise PBM
        hipcub::DeviceScan::ExclusiveSum(d_CUB_temp_storage, d_CUB_temp_storage_bytes, d_histogram, hd_data.PBM, binCount + 1);
    }
    {  // Reorder messages
       // Copy messages from d_messages to d_messages_swap, in hash order
        auto &cs = CUDAScatter::getInstance(0);  // Choose proper stream_id in future!
        cs.pbm_reorder(this->sim_message.getMessageDescription().variables, this->sim_message.getReadList(), this->sim_message.getWriteList(), MESSAGE_COUNT, d_keys, d_vals, hd_data.PBM);
        this->sim_message.swap(false, 0);  // Stream id is unused here
    }
    {  // Fill PBM and Message Texture Buffers
       // gpuErrchk(hipBindTexture(nullptr, d_texMessages, d_agents, sizeof(glm::vec4) * MESSAGE_COUNT));
       // gpuErrchk(hipBindTexture(nullptr, d_texPBM, d_PBM, sizeof(unsigned int) * (binCount + 1)));
    }
}

void MsgSpatial3D::CUDAModelHandler::resizeCubTemp() {
    size_t bytesCheck = 0;
    hipcub::DeviceScan::ExclusiveSum(nullptr, bytesCheck, hd_data.PBM, d_histogram, binCount + 1);
    if (bytesCheck > d_CUB_temp_storage_bytes) {
        if (d_CUB_temp_storage) {
            gpuErrchk(hipFree(d_CUB_temp_storage));
        }
        d_CUB_temp_storage_bytes = bytesCheck;
        gpuErrchk(hipMalloc(&d_CUB_temp_storage, d_CUB_temp_storage_bytes));
    }
}

void MsgSpatial3D::CUDAModelHandler::resizeKeysVals(const unsigned int &newSize) {
    size_t bytesCheck = newSize * sizeof(unsigned int);
    if (bytesCheck > d_keys_vals_storage_bytes) {
        if (d_keys) {
            gpuErrchk(hipFree(d_keys));
            gpuErrchk(hipFree(d_vals));
        }
        d_keys_vals_storage_bytes = bytesCheck;
        gpuErrchk(hipMalloc(&d_keys, d_keys_vals_storage_bytes));
        gpuErrchk(hipMalloc(&d_vals, d_keys_vals_storage_bytes));
    }
}
