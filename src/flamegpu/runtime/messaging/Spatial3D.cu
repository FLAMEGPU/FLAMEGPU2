#include "hip/hip_runtime.h"
#include "flamegpu/runtime/messaging/Spatial3D/Spatial3DHost.h"
#include "flamegpu/runtime/messaging/Spatial3D/Spatial3DDevice.h"

#include "flamegpu/gpu/CUDAScatter.h"
#ifdef _MSC_VER
#pragma warning(push, 1)
#pragma warning(disable : 4706 4834)
#include <hipcub/hipcub.hpp>
#pragma warning(pop)
#else
#include <hipcub/hipcub.hpp>
#endif



MsgSpatial3D::CUDAModelHandler::CUDAModelHandler(CUDAMessage &a)
  : MsgSpecialisationHandler()
  , sim_message(a) {
    NVTX_RANGE("Spatial3D::CUDAModelHandler");
    const Data &d = (const Data &)a.getMessageDescription();
    hd_data.radius = d.radius;
    hd_data.min[0] = d.minX;
    hd_data.min[1] = d.minY;
    hd_data.min[2] = d.minZ;
    hd_data.max[0] = d.maxX;
    hd_data.max[1] = d.maxY;
    hd_data.max[2] = d.maxZ;
    binCount = 1;
    for (unsigned int axis = 0; axis < 3; ++axis) {
        hd_data.environmentWidth[axis] = hd_data.max[axis] - hd_data.min[axis];
        hd_data.gridDim[axis] = static_cast<unsigned int>(ceil(hd_data.environmentWidth[axis] / hd_data.radius));
        binCount *= hd_data.gridDim[axis];
    }
    // Device allocation occurs in allocateMetaDataDevicePtr rather than the constructor.
}

__global__ void atomicHistogram3D(
    const MsgSpatial3D::MetaData *md,
    unsigned int* bin_index,
    unsigned int* bin_sub_index,
    unsigned int *pbm_counts,
    unsigned int message_count,
    const float * __restrict__ x,
    const float * __restrict__ y,
    const float * __restrict__ z) {
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    // Kill excess threads
    if (index >= message_count) return;

    MsgSpatial3D::GridPos3D gridPos = getGridPosition3D(md, x[index], y[index], z[index]);
    unsigned int hash = getHash3D(md, gridPos);
    bin_index[index] = hash;
    unsigned int bin_idx = atomicInc((unsigned int*)&pbm_counts[hash], 0xFFFFFFFF);
    bin_sub_index[index] = bin_idx;
}

void MsgSpatial3D::CUDAModelHandler::init(CUDAScatter &, const unsigned int &) {
    allocateMetaDataDevicePtr();
    // Set PBM to 0
    gpuErrchk(hipMemset(hd_data.PBM, 0x00000000, (binCount + 1) * sizeof(unsigned int)));
}

void MsgSpatial3D::CUDAModelHandler::allocateMetaDataDevicePtr() {
    if (d_data == nullptr) {
        gpuErrchk(hipMalloc(&d_histogram, (binCount + 1) * sizeof(unsigned int)));
        gpuErrchk(hipMalloc(&hd_data.PBM, (binCount + 1) * sizeof(unsigned int)));
        gpuErrchk(hipMalloc(&d_data, sizeof(MetaData)));
        gpuErrchk(hipMemcpy(d_data, &hd_data, sizeof(MetaData), hipMemcpyHostToDevice));
        resizeCubTemp();
    }
}

void MsgSpatial3D::CUDAModelHandler::freeMetaDataDevicePtr() {
    if (d_data != nullptr) {
        d_CUB_temp_storage_bytes = 0;
        gpuErrchk(hipFree(d_CUB_temp_storage));
        gpuErrchk(hipFree(d_histogram));
        gpuErrchk(hipFree(hd_data.PBM));
        gpuErrchk(hipFree(d_data));
        d_CUB_temp_storage = nullptr;
        d_histogram = nullptr;
        hd_data.PBM = nullptr;
        d_data = nullptr;
        if (d_keys) {
            d_keys_vals_storage_bytes = 0;
            gpuErrchk(hipFree(d_keys));
            gpuErrchk(hipFree(d_vals));
            d_keys = nullptr;
            d_vals = nullptr;
        }
    }
}

void MsgSpatial3D::CUDAModelHandler::buildIndex(CUDAScatter &scatter, const unsigned int &streamId) {
    const unsigned int MESSAGE_COUNT = this->sim_message.getMessageCount();
    resizeKeysVals(this->sim_message.getMaximumListSize());  // Resize based on allocated amount rather than message count
    {  // Build atomic histogram
        gpuErrchk(hipMemset(d_histogram, 0x00000000, (binCount + 1) * sizeof(unsigned int)));
        int blockSize;  // The launch configurator returned block size
        gpuErrchk(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, atomicHistogram3D, 32, 0));  // Randomly 32
                                                                                                         // Round up according to array size
        int gridSize = (MESSAGE_COUNT + blockSize - 1) / blockSize;
        atomicHistogram3D << <gridSize, blockSize >> >(d_data, d_keys, d_vals, d_histogram, MESSAGE_COUNT,
            reinterpret_cast<float*>(this->sim_message.getReadPtr("x")),
            reinterpret_cast<float*>(this->sim_message.getReadPtr("y")),
            reinterpret_cast<float*>(this->sim_message.getReadPtr("z")));
        gpuErrchk(hipDeviceSynchronize());
    }
    {  // Scan (sum), to finalise PBM
        gpuErrchk(hipcub::DeviceScan::ExclusiveSum(d_CUB_temp_storage, d_CUB_temp_storage_bytes, d_histogram, hd_data.PBM, binCount + 1));
    }
    {  // Reorder messages
       // Copy messages from d_messages to d_messages_swap, in hash order
        scatter.pbm_reorder(streamId, this->sim_message.getMessageDescription().variables, this->sim_message.getReadList(), this->sim_message.getWriteList(), MESSAGE_COUNT, d_keys, d_vals, hd_data.PBM);
        this->sim_message.swap();  // Stream id is unused here
    }
    {  // Fill PBM and Message Texture Buffers
       // gpuErrchk(hipBindTexture(nullptr, d_texMessages, d_agents, sizeof(glm::vec4) * MESSAGE_COUNT));
       // gpuErrchk(hipBindTexture(nullptr, d_texPBM, d_PBM, sizeof(unsigned int) * (binCount + 1)));
    }
}

void MsgSpatial3D::CUDAModelHandler::resizeCubTemp() {
    size_t bytesCheck = 0;
    gpuErrchk(hipcub::DeviceScan::ExclusiveSum(nullptr, bytesCheck, hd_data.PBM, d_histogram, binCount + 1));
    if (bytesCheck > d_CUB_temp_storage_bytes) {
        if (d_CUB_temp_storage) {
            gpuErrchk(hipFree(d_CUB_temp_storage));
        }
        d_CUB_temp_storage_bytes = bytesCheck;
        gpuErrchk(hipMalloc(&d_CUB_temp_storage, d_CUB_temp_storage_bytes));
    }
}

void MsgSpatial3D::CUDAModelHandler::resizeKeysVals(const unsigned int &newSize) {
    size_t bytesCheck = newSize * sizeof(unsigned int);
    if (bytesCheck > d_keys_vals_storage_bytes) {
        if (d_keys) {
            gpuErrchk(hipFree(d_keys));
            gpuErrchk(hipFree(d_vals));
        }
        d_keys_vals_storage_bytes = bytesCheck;
        gpuErrchk(hipMalloc(&d_keys, d_keys_vals_storage_bytes));
        gpuErrchk(hipMalloc(&d_vals, d_keys_vals_storage_bytes));
    }
}

MsgSpatial3D::Data::Data(const std::shared_ptr<const ModelData> &model, const std::string &message_name)
    : MsgSpatial2D::Data(model, message_name)
    , minZ(NAN)
    , maxZ(NAN) {
    description = std::unique_ptr<Description>(new Description(model, this));
    description->newVariable<float>("z");
}
MsgSpatial3D::Data::Data(const std::shared_ptr<const ModelData> &model, const Data &other)
    : MsgSpatial2D::Data(model, other)
    , minZ(other.minZ)
    , maxZ(other.maxZ) {
    description = std::unique_ptr<Description>(model ? new Description(model, this) : nullptr);
    if (isnan(minZ)) {
        THROW InvalidMessage("Environment minimum z bound has not been set in spatial message '%s'\n", other.name.c_str());
    }
    if (isnan(maxZ)) {
        THROW InvalidMessage("Environment maximum z bound has not been set in spatial message '%s'\n", other.name.c_str());
    }
}
MsgSpatial3D::Data *MsgSpatial3D::Data::clone(const std::shared_ptr<const ModelData> &newParent) {
    return new Data(newParent, *this);
}
std::unique_ptr<MsgSpecialisationHandler> MsgSpatial3D::Data::getSpecialisationHander(CUDAMessage &owner) const {
    return std::unique_ptr<MsgSpecialisationHandler>(new CUDAModelHandler(owner));
}
std::type_index MsgSpatial3D::Data::getType() const { return std::type_index(typeid(MsgSpatial3D)); }

MsgSpatial3D::Description::Description(const std::shared_ptr<const ModelData> &_model, Data *const data)
    : MsgBruteForce::Description(_model, data) { }

void MsgSpatial3D::Description::setRadius(const float &r) {
    if (r <= 0) {
        THROW InvalidArgument("Spatial messaging radius must be a positive value, %f is not valid.", r);
    }
    reinterpret_cast<Data *>(message)->radius = r;
}
void MsgSpatial3D::Description::setMinX(const float &x) {
    if (!isnan(reinterpret_cast<Data *>(message)->maxX) &&
        x >= reinterpret_cast<Data *>(message)->maxX) {
        THROW InvalidArgument("Spatial messaging min x bound must be lower than max bound, %f !< %f", x, reinterpret_cast<Data *>(message)->maxX);
    }
    reinterpret_cast<Data *>(message)->minX = x;
}
void MsgSpatial3D::Description::setMinY(const float &y) {
    if (!isnan(reinterpret_cast<Data *>(message)->maxY) &&
        y >= reinterpret_cast<Data *>(message)->maxY) {
        THROW InvalidArgument("Spatial messaging min bound must be lower than max bound, %f !< %f", y, reinterpret_cast<Data *>(message)->maxY);
    }
    reinterpret_cast<Data *>(message)->minY = y;
}
void MsgSpatial3D::Description::setMinZ(const float &z) {
    if (!isnan(reinterpret_cast<Data *>(message)->maxZ) &&
        z >= reinterpret_cast<Data *>(message)->maxZ) {
        THROW InvalidArgument("Spatial messaging min z bound must be lower than max bound, %f !< %f", z, reinterpret_cast<Data *>(message)->maxZ);
    }
    reinterpret_cast<Data *>(message)->minZ = z;
}
void MsgSpatial3D::Description::setMin(const float &x, const float &y, const float &z) {
    if (!isnan(reinterpret_cast<Data *>(message)->maxX) &&
        x >= reinterpret_cast<Data *>(message)->maxX) {
        THROW InvalidArgument("Spatial messaging min x bound must be lower than max bound, %f !< %f", x, reinterpret_cast<Data *>(message)->maxX);
    }
    if (!isnan(reinterpret_cast<Data *>(message)->maxY) &&
        y >= reinterpret_cast<Data *>(message)->maxY) {
        THROW InvalidArgument("Spatial messaging min y bound must be lower than max bound, %f !< %f", y, reinterpret_cast<Data *>(message)->maxY);
    }
    if (!isnan(reinterpret_cast<Data *>(message)->maxZ) &&
        z >= reinterpret_cast<Data *>(message)->maxZ) {
        THROW InvalidArgument("Spatial messaging min z bound must be lower than max bound, %f !< %f", z, reinterpret_cast<Data *>(message)->maxZ);
    }
    reinterpret_cast<Data *>(message)->minX = x;
    reinterpret_cast<Data *>(message)->minY = y;
    reinterpret_cast<Data *>(message)->minZ = z;
}
void MsgSpatial3D::Description::setMaxX(const float &x) {
    if (!isnan(reinterpret_cast<Data *>(message)->minX) &&
        x <= reinterpret_cast<Data *>(message)->minX) {
        THROW InvalidArgument("Spatial messaging max x bound must be greater than min bound, %f !> %f", x, reinterpret_cast<Data *>(message)->minX);
    }
    reinterpret_cast<Data *>(message)->maxX = x;
}
void MsgSpatial3D::Description::setMaxY(const float &y) {
    if (!isnan(reinterpret_cast<Data *>(message)->minY) &&
        y <= reinterpret_cast<Data *>(message)->minY) {
        THROW InvalidArgument("Spatial messaging max y bound must be greater than min bound, %f !> %f", y, reinterpret_cast<Data *>(message)->minY);
    }
    reinterpret_cast<Data *>(message)->maxY = y;
}
void MsgSpatial3D::Description::setMaxZ(const float &z) {
    if (!isnan(reinterpret_cast<Data *>(message)->minZ) &&
        z <= reinterpret_cast<Data *>(message)->minZ) {
        THROW InvalidArgument("Spatial messaging max z bound must be greater than min bound, %f !> %f", z, reinterpret_cast<Data *>(message)->minZ);
    }
    reinterpret_cast<Data *>(message)->maxZ = z;
}
void MsgSpatial3D::Description::setMax(const float &x, const float &y, const float &z) {
    if (!isnan(reinterpret_cast<Data *>(message)->minX) &&
        x <= reinterpret_cast<Data *>(message)->minX) {
        THROW InvalidArgument("Spatial messaging max x bound must be greater than min bound, %f !> %f", x, reinterpret_cast<Data *>(message)->minX);
    }
    if (!isnan(reinterpret_cast<Data *>(message)->minY) &&
        y <= reinterpret_cast<Data *>(message)->minY) {
        THROW InvalidArgument("Spatial messaging max y bound must be greater than min bound, %f !> %f", y, reinterpret_cast<Data *>(message)->minY);
    }
    if (!isnan(reinterpret_cast<Data *>(message)->minZ) &&
        z <= reinterpret_cast<Data *>(message)->minZ) {
        THROW InvalidArgument("Spatial messaging max z bound must be greater than min bound, %f !> %f", z, reinterpret_cast<Data *>(message)->minZ);
    }
    reinterpret_cast<Data *>(message)->maxX = x;
    reinterpret_cast<Data *>(message)->maxY = y;
    reinterpret_cast<Data *>(message)->maxZ = z;
}

float MsgSpatial3D::Description::getRadius() const {
    return reinterpret_cast<Data *>(message)->radius;
}
float MsgSpatial3D::Description::getMinX() const {
    return reinterpret_cast<Data *>(message)->minX;
}
float MsgSpatial3D::Description::getMinY() const {
    return reinterpret_cast<Data *>(message)->minY;
}
float MsgSpatial3D::Description::getMinZ() const {
    return reinterpret_cast<Data *>(message)->minZ;
}
float MsgSpatial3D::Description::getMaxX() const {
    return reinterpret_cast<Data *>(message)->maxX;
}
float MsgSpatial3D::Description::getMaxY() const {
    return reinterpret_cast<Data *>(message)->maxY;
}
float MsgSpatial3D::Description::getMaxZ() const {
    return reinterpret_cast<Data *>(message)->maxZ;
}
