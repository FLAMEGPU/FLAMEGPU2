#include "flamegpu/runtime/messaging/MessageArray2D.h"
#include "flamegpu/model/AgentDescription.h"  // Used by Move-Assign
#include "flamegpu/gpu/CUDAMessage.h"
#include "flamegpu/gpu/CUDAScatter.cuh"

#include "flamegpu/runtime/messaging/MessageArray2D/MessageArray2DHost.h"
// #include "flamegpu/runtime/messaging/MessageArray2D/MessageArray2DDevice.cuh"

namespace flamegpu {

/**
 * Constructor
 * Allocates memory on device for message list length
 * @param a Parent CUDAMessage, used to access message settings, data ptrs etc
 */
MessageArray2D::CUDAModelHandler::CUDAModelHandler(CUDAMessage &a)
    : MessageSpecialisationHandler()
    , d_metadata(nullptr)
    , sim_message(a)
    , d_write_flag(nullptr)
    , d_write_flag_len(0) {
    const Data& d = static_cast<const Data &>(a.getMessageDescription());
    memcpy(&hd_metadata.dimensions, d.dimensions.data(), d.dimensions.size() * sizeof(unsigned int));
    hd_metadata.length = d.dimensions[0] * d.dimensions[1];
}

void MessageArray2D::CUDAModelHandler::init(CUDAScatter &scatter, unsigned int streamId, hipStream_t stream) {
    allocateMetaDataDevicePtr(stream);
    // Allocate messages
    this->sim_message.resize(hd_metadata.length, scatter, stream, streamId);
    this->sim_message.setMessageCount(hd_metadata.length);
    // Zero the output arrays
    auto &read_list = this->sim_message.getReadList();
    auto &write_list = this->sim_message.getWriteList();
    for (auto &var : this->sim_message.getMessageDescription().variables) {
        // Elements is harmless, futureproof for arrays support
        // hd_metadata.length is used, as message array can be longer than message count
        gpuErrchk(hipMemsetAsync(write_list.at(var.first), 0, var.second.type_size * var.second.elements * hd_metadata.length));
        gpuErrchk(hipMemsetAsync(read_list.at(var.first), 0, var.second.type_size * var.second.elements * hd_metadata.length));
    }
    gpuErrchk(hipStreamSynchronize(stream));
}
void MessageArray2D::CUDAModelHandler::allocateMetaDataDevicePtr(hipStream_t stream) {
    if (d_metadata == nullptr) {
        gpuErrchk(hipMalloc(&d_metadata, sizeof(MetaData)));
        gpuErrchk(hipMemcpyAsync(d_metadata, &hd_metadata, sizeof(MetaData), hipMemcpyHostToDevice));
        gpuErrchk(hipStreamSynchronize(stream));
    }
}

void MessageArray2D::CUDAModelHandler::freeMetaDataDevicePtr() {
    if (d_metadata != nullptr) {
        gpuErrchk(hipFree(d_metadata));
    }
    d_metadata = nullptr;

    if (d_write_flag) {
        gpuErrchk(hipFree(d_write_flag));
    }
    d_write_flag = nullptr;
    d_write_flag_len = 0;
}
void MessageArray2D::CUDAModelHandler::buildIndex(CUDAScatter &scatter, unsigned int streamId, hipStream_t stream) {
    const unsigned int MESSAGE_COUNT = this->sim_message.getMessageCount();
    // Zero the output arrays
    auto &read_list = this->sim_message.getReadList();
    auto &write_list = this->sim_message.getWriteList();
    for (auto &var : this->sim_message.getMessageDescription().variables) {
        // Elements is harmless, futureproof for arrays support
        // hd_metadata.length is used, as message array can be longer than message count
        gpuErrchk(hipMemsetAsync(write_list.at(var.first), 0, var.second.type_size * var.second.elements * hd_metadata.length, stream));
    }

    // Reorder messages
    unsigned int *t_d_write_flag = nullptr;
    if (MESSAGE_COUNT > hd_metadata.length) {
        // Use internal memory for d_write_flag
        if (d_write_flag_len < MESSAGE_COUNT) {
            // Increase length
            if (d_write_flag) {
                gpuErrchk(hipFree(d_write_flag));
            }
            d_write_flag_len = static_cast<unsigned int>(MESSAGE_COUNT * 1.1f);
            gpuErrchk(hipMalloc(&d_write_flag, sizeof(unsigned int) * d_write_flag_len));
        }
        t_d_write_flag = d_write_flag;
    }
    scatter.arrayMessageReorder(streamId, stream, this->sim_message.getMessageDescription().variables, read_list, write_list, MESSAGE_COUNT, hd_metadata.length, t_d_write_flag);
    this->sim_message.swap();
    // Reset message count back to full array length
    // Array message exposes not output messages as 0
    if (MESSAGE_COUNT != hd_metadata.length)
        this->sim_message.setMessageCount(hd_metadata.length);
    // Detect errors
    // TODO
    gpuErrchk(hipStreamSynchronize(stream));  // Redundant: Array msg reorder has a sync
}


MessageArray2D::Data::Data(const std::shared_ptr<const ModelData> &model, const std::string &message_name)
    : MessageBruteForce::Data(model, message_name)
    , dimensions({ 0, 0 }) {
    description = std::unique_ptr<MessageArray2D::Description>(new MessageArray2D::Description(model, this));
    variables.emplace("___INDEX", Variable(1, size_type()));
}
MessageArray2D::Data::Data(const std::shared_ptr<const ModelData> &model, const Data &other)
    : MessageBruteForce::Data(model, other)
    , dimensions(other.dimensions) {
    description = std::unique_ptr<MessageArray2D::Description>(model ? new MessageArray2D::Description(model, this) : nullptr);
    if (dimensions[0] == 0 || dimensions[1] == 0) {
        THROW exception::InvalidMessage("All dimensions must be ABOVE zero in array2D message '%s'\n", other.name.c_str());
    }
}
MessageArray2D::Data *MessageArray2D::Data::clone(const std::shared_ptr<const ModelData> &newParent) {
    return new Data(newParent, *this);
}
std::unique_ptr<MessageSpecialisationHandler> MessageArray2D::Data::getSpecialisationHander(CUDAMessage &owner) const {
    return std::unique_ptr<MessageSpecialisationHandler>(new CUDAModelHandler(owner));
}
std::type_index MessageArray2D::Data::getType() const { return std::type_index(typeid(MessageArray2D)); }


MessageArray2D::Description::Description(const std::shared_ptr<const ModelData> &_model, Data *const data)
    : MessageBruteForce::Description(_model, data) { }

void MessageArray2D::Description::setDimensions(const size_type len_x, const size_type len_y) {
    setDimensions({ len_x , len_y });
}
void MessageArray2D::Description::setDimensions(const std::array<size_type, 2> &dims) {
    if (dims[0] == 0 || dims[1] == 0) {
        THROW exception::InvalidArgument("All dimensions must be above zero in array2D message.\n");
    }
    reinterpret_cast<Data *>(message)->dimensions = dims;
}
std::array<MessageArray2D::size_type, 2> MessageArray2D::Description::getDimensions() const {
    return reinterpret_cast<Data *>(message)->dimensions;
}
MessageArray2D::size_type MessageArray2D::Description::getDimX() const {
    return reinterpret_cast<Data *>(message)->dimensions[0];
}
MessageArray2D::size_type MessageArray2D::Description::getDimY() const {
    return reinterpret_cast<Data *>(message)->dimensions[1];
}

}  // namespace flamegpu
