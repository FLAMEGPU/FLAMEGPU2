#include "flamegpu/runtime/messaging/MessageArray3D.h"
#include "flamegpu/model/AgentDescription.h"  // Used by Move-Assign
#include "flamegpu/gpu/CUDAMessage.h"
#include "flamegpu/gpu/CUDAScatter.cuh"

#include "flamegpu/runtime/messaging/MessageArray3D/MessageArray3DHost.h"
// #include "flamegpu/runtime/messaging/MessageArray3D/MessageArray3DDevice.cuh"

namespace flamegpu {

/**
 * Constructor
 * Allocates memory on device for message list length
 * @param a Parent CUDAMessage, used to access message settings, data ptrs etc
 */
MessageArray3D::CUDAModelHandler::CUDAModelHandler(CUDAMessage &a)
    : MessageSpecialisationHandler()
    , d_metadata(nullptr)
    , sim_message(a)
    , d_write_flag(nullptr)
    , d_write_flag_len(0) {
    const Data& d = static_cast<const Data &>(a.getMessageDescription());
    memcpy(&hd_metadata.dimensions, d.dimensions.data(), d.dimensions.size() * sizeof(unsigned int));
    hd_metadata.length = d.dimensions[0] * d.dimensions[1] * d.dimensions[2];
}

void MessageArray3D::CUDAModelHandler::init(CUDAScatter &scatter, unsigned int streamId, hipStream_t stream) {
    allocateMetaDataDevicePtr(stream);
    // Allocate messages
    this->sim_message.resize(hd_metadata.length, scatter, stream, streamId);
    this->sim_message.setMessageCount(hd_metadata.length);
    // Zero the output arrays
    auto &read_list = this->sim_message.getReadList();
    auto &write_list = this->sim_message.getWriteList();
    for (auto &var : this->sim_message.getMessageDescription().variables) {
        // Elements is harmless, futureproof for arrays support
        // hd_metadata.length is used, as message array can be longer than message count
        gpuErrchk(hipMemsetAsync(write_list.at(var.first), 0, var.second.type_size * var.second.elements * hd_metadata.length, stream));
        gpuErrchk(hipMemsetAsync(read_list.at(var.first), 0, var.second.type_size * var.second.elements * hd_metadata.length, stream));
    }
    gpuErrchk(hipStreamSynchronize(stream));
}
void MessageArray3D::CUDAModelHandler::allocateMetaDataDevicePtr(hipStream_t stream) {
    if (d_metadata == nullptr) {
        gpuErrchk(hipMalloc(&d_metadata, sizeof(MetaData)));
        gpuErrchk(hipMemcpyAsync(d_metadata, &hd_metadata, sizeof(MetaData), hipMemcpyHostToDevice, stream));
        gpuErrchk(hipStreamSynchronize(stream));
    }
}

void MessageArray3D::CUDAModelHandler::freeMetaDataDevicePtr() {
    if (d_metadata != nullptr) {
        gpuErrchk(hipFree(d_metadata));
    }
    d_metadata = nullptr;

    if (d_write_flag) {
        gpuErrchk(hipFree(d_write_flag));
    }
    d_write_flag = nullptr;
    d_write_flag_len = 0;
}
void MessageArray3D::CUDAModelHandler::buildIndex(CUDAScatter &scatter, unsigned int streamId, hipStream_t stream) {
    const unsigned int MESSAGE_COUNT = this->sim_message.getMessageCount();
    // Zero the output arrays
    auto &read_list = this->sim_message.getReadList();
    auto &write_list = this->sim_message.getWriteList();
    for (auto &var : this->sim_message.getMessageDescription().variables) {
        // Elements is harmless, futureproof for arrays support
        // hd_metadata.length is used, as message array can be longer than message count
        gpuErrchk(hipMemsetAsync(write_list.at(var.first), 0, var.second.type_size * var.second.elements * hd_metadata.length, stream));
    }

    // Reorder messages
    unsigned int *t_d_write_flag = nullptr;
    if (MESSAGE_COUNT > hd_metadata.length) {
        // Use internal memory for d_write_flag
        if (d_write_flag_len < MESSAGE_COUNT) {
            // Increase length
            if (d_write_flag) {
                gpuErrchk(hipFree(d_write_flag));
            }
            d_write_flag_len = static_cast<unsigned int>(MESSAGE_COUNT * 1.1f);
            gpuErrchk(hipMalloc(&d_write_flag, sizeof(unsigned int) * d_write_flag_len));
        }
        t_d_write_flag = d_write_flag;
    }
    scatter.arrayMessageReorder(streamId, stream, this->sim_message.getMessageDescription().variables, read_list, write_list, MESSAGE_COUNT, hd_metadata.length, t_d_write_flag);
    this->sim_message.swap();
    // Reset message count back to full array length
    // Array message exposes not output messages as 0
    if (MESSAGE_COUNT != hd_metadata.length)
        this->sim_message.setMessageCount(hd_metadata.length);
    // Detect errors
    // TODO
    gpuErrchk(hipStreamSynchronize(stream));  // Redundant: Array msg reorder has a sync
}


MessageArray3D::Data::Data(const std::shared_ptr<const ModelData> &model, const std::string &message_name)
    : MessageBruteForce::Data(model, message_name)
    , dimensions({0, 0, 0}) {
    description = std::unique_ptr<MessageArray3D::Description>(new MessageArray3D::Description(model, this));
    variables.emplace("___INDEX", Variable(1, size_type()));
}
MessageArray3D::Data::Data(const std::shared_ptr<const ModelData> &model, const Data &other)
    : MessageBruteForce::Data(model, other)
    , dimensions(other.dimensions) {
    description = std::unique_ptr<MessageArray3D::Description>(model ? new MessageArray3D::Description(model, this) : nullptr);
    if (dimensions[0] == 0 || dimensions[1] == 0 || dimensions[2] == 0) {
        THROW exception::InvalidMessage("All dimensions must be above zero in array3D message '%s'\n", other.name.c_str());
    }
}
MessageArray3D::Data *MessageArray3D::Data::clone(const std::shared_ptr<const ModelData> &newParent) {
    return new Data(newParent, *this);
}
std::unique_ptr<MessageSpecialisationHandler> MessageArray3D::Data::getSpecialisationHander(CUDAMessage &owner) const {
    return std::unique_ptr<MessageSpecialisationHandler>(new CUDAModelHandler(owner));
}
std::type_index MessageArray3D::Data::getType() const { return std::type_index(typeid(MessageArray3D)); }


MessageArray3D::Description::Description(const std::shared_ptr<const ModelData> &_model, Data *const data)
    : MessageBruteForce::Description(_model, data) { }

void MessageArray3D::Description::setDimensions(const size_type& len_x, const size_type& len_y, const size_type& len_z) {
    setDimensions({ len_x , len_y, len_z});
}
void MessageArray3D::Description::setDimensions(const std::array<size_type, 3> &dims) {
    if (dims[0] == 0 || dims[1] == 0 || dims[2] == 0) {
        THROW exception::InvalidArgument("All dimensions must be above zero in array3D message.\n");
    }
    reinterpret_cast<Data *>(message)->dimensions = dims;
}
std::array<MessageArray3D::size_type, 3> MessageArray3D::Description::getDimensions() const {
    return reinterpret_cast<Data *>(message)->dimensions;
}
MessageArray2D::size_type MessageArray3D::Description::getDimX() const {
    return reinterpret_cast<Data *>(message)->dimensions[0];
}
MessageArray2D::size_type MessageArray3D::Description::getDimY() const {
    return reinterpret_cast<Data *>(message)->dimensions[1];
}
MessageArray2D::size_type MessageArray3D::Description::getDimZ() const {
    return reinterpret_cast<Data *>(message)->dimensions[2];
}

}  // namespace flamegpu
