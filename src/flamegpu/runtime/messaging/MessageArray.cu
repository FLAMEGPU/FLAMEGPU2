#include "flamegpu/runtime/messaging/MessageArray.h"
#include "flamegpu/model/AgentDescription.h"  // Used by Move-Assign
#include "flamegpu/simulation/detail/CUDAMessage.h"
#include "flamegpu/simulation/detail/CUDAScatter.cuh"

#include "flamegpu/runtime/messaging/MessageArray/MessageArrayHost.h"
// #include "flamegpu/runtime/messaging/MessageArray/MessageArrayDevice.cuh"
#include "flamegpu/detail/cuda.cuh"

namespace flamegpu {

/**
 * Constructor
 * Allocates memory on device for message list length
 * @param a Parent CUDAMessage, used to access message settings, data ptrs etc
 */
MessageArray::CUDAModelHandler::CUDAModelHandler(detail::CUDAMessage &a)
    : MessageSpecialisationHandler()
    , d_metadata(nullptr)
    , sim_message(a)
    , d_write_flag(nullptr)
    , d_write_flag_len(0) {
    const Data& d = static_cast<const Data &>(a.getMessageData());
    hd_metadata.length = d.length;
}

void MessageArray::CUDAModelHandler::init(detail::CUDAScatter &scatter, unsigned int streamId, hipStream_t stream) {
    allocateMetaDataDevicePtr(stream);
    // Allocate messages
    this->sim_message.resize(hd_metadata.length, scatter, stream, streamId);
    this->sim_message.setMessageCount(hd_metadata.length);
    // Zero the output arrays
    auto &read_list = this->sim_message.getReadList();
    auto &write_list = this->sim_message.getWriteList();
    for (auto &var : this->sim_message.getMessageData().variables) {
        // Elements is harmless, futureproof for arrays support
        // hd_metadata.length is used, as message array can be longer than message count
        gpuErrchk(hipMemsetAsync(write_list.at(var.first), 0, var.second.type_size * var.second.elements * hd_metadata.length, stream));
        gpuErrchk(hipMemsetAsync(read_list.at(var.first), 0, var.second.type_size * var.second.elements * hd_metadata.length, stream));
    }
    gpuErrchk(hipStreamSynchronize(stream));
}
void MessageArray::CUDAModelHandler::allocateMetaDataDevicePtr(hipStream_t stream) {
    if (d_metadata == nullptr) {
        gpuErrchk(hipMalloc(&d_metadata, sizeof(MetaData)));
        gpuErrchk(hipMemcpyAsync(d_metadata, &hd_metadata, sizeof(MetaData), hipMemcpyHostToDevice, stream));
        gpuErrchk(hipStreamSynchronize(stream));
    }
}

void MessageArray::CUDAModelHandler::freeMetaDataDevicePtr() {
    if (d_metadata != nullptr) {
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_metadata));
    }
    d_metadata = nullptr;

    if (d_write_flag) {
        gpuErrchk(flamegpu::detail::cuda::hipFree(d_write_flag));
    }
    d_write_flag = nullptr;
    d_write_flag_len = 0;
}
void MessageArray::CUDAModelHandler::buildIndex(detail::CUDAScatter &scatter, unsigned int streamId, hipStream_t stream) {
    const unsigned int MESSAGE_COUNT = this->sim_message.getMessageCount();
    // Zero the output arrays
    auto &read_list = this->sim_message.getReadList();
    auto &write_list = this->sim_message.getWriteList();
    for (auto &var : this->sim_message.getMessageData().variables) {
        // Elements is harmless, futureproof for arrays support
        // hd_metadata.length is used, as message array can be longer than message count
        gpuErrchk(hipMemsetAsync(write_list.at(var.first), 0, var.second.type_size * var.second.elements * hd_metadata.length, stream));
    }

    // Reorder messages
    unsigned int *t_d_write_flag = nullptr;
    if (MESSAGE_COUNT > hd_metadata.length) {
        // Use internal memory for d_write_flag
        if (d_write_flag_len < MESSAGE_COUNT) {
            // Increase length
            if (d_write_flag) {
                gpuErrchk(flamegpu::detail::cuda::hipFree(d_write_flag));
            }
            d_write_flag_len = static_cast<unsigned int>(MESSAGE_COUNT * 1.1f);
            gpuErrchk(hipMalloc(&d_write_flag, sizeof(unsigned int) * d_write_flag_len));
        }
        t_d_write_flag = d_write_flag;
    }
    scatter.arrayMessageReorder(streamId, stream, this->sim_message.getMessageData().variables, read_list, write_list, MESSAGE_COUNT, hd_metadata.length, t_d_write_flag);
    this->sim_message.swap();
    // Reset message count back to full array length
    // Array message exposes not output messages as 0
    if (MESSAGE_COUNT != hd_metadata.length)
        this->sim_message.setMessageCount(hd_metadata.length);
    // Detect errors
    // TODO
    gpuErrchk(hipStreamSynchronize(stream));  // Redundant: Array msg reorder has a sync
}

/// <summary>
/// CDescription
/// </summary>
MessageArray::CDescription::CDescription(std::shared_ptr<Data> data)
    : MessageBruteForce::CDescription(std::move(std::static_pointer_cast<MessageBruteForce::Data>(data))) { }
MessageArray::CDescription::CDescription(std::shared_ptr<const Data> data)
    : CDescription(std::move(std::const_pointer_cast<Data>(data))) { }

bool MessageArray::CDescription::operator==(const CDescription& rhs) const {
    return *this->message == *rhs.message;  // Compare content is functionally the same
}
bool MessageArray::CDescription::operator!=(const CDescription& rhs) const {
    return !(*this == rhs);
}
/**
 * Const accessors
 */
flamegpu::size_type MessageArray::CDescription::getLength() const {
    return std::static_pointer_cast<Data>(message)->length;
}

/// <summary>
/// Description
/// </summary>
MessageArray::Description::Description(std::shared_ptr<Data> data)
    : CDescription(data) { }
/**
 * Accessors
 */
void MessageArray::Description::setLength(const size_type len) {
    if (len == 0) {
        THROW exception::InvalidArgument("Array messaging length must not be zero.\n");
    }
    std::static_pointer_cast<Data>(message)->length = len;
}

/// <summary>
/// Data
/// </summary>
MessageArray::Data::Data(std::shared_ptr<const ModelData> model, const std::string &message_name)
    : MessageBruteForce::Data(model, message_name)
    , length(0) {
    variables.emplace("___INDEX", Variable(1, size_type()));
}
MessageArray::Data::Data(std::shared_ptr<const ModelData> model, const Data &other)
    : MessageBruteForce::Data(model, other)
    , length(other.length) {
    if (length == 0) {
        THROW exception::InvalidMessage("Length must not be zero in array message '%s'\n", other.name.c_str());
    }
}
MessageArray::Data *MessageArray::Data::clone(const std::shared_ptr<const ModelData> &newParent) {
    return new Data(newParent, *this);
}
std::unique_ptr<MessageSpecialisationHandler> MessageArray::Data::getSpecialisationHander(detail::CUDAMessage &owner) const {
    return std::unique_ptr<MessageSpecialisationHandler>(new CUDAModelHandler(owner));
}
std::type_index MessageArray::Data::getType() const { return std::type_index(typeid(MessageArray)); }

}  // namespace flamegpu
