#include "hip/hip_runtime.h"
#include "flamegpu/runtime/messaging/MessageSpatial2D.h"

#ifdef _MSC_VER
#pragma warning(push, 1)
#pragma warning(disable : 4706 4834)
#include <hipcub/hipcub.hpp>
#pragma warning(pop)
#else
#include <hipcub/hipcub.hpp>
#endif

#include "flamegpu/runtime/messaging.h"
#include "flamegpu/runtime/messaging/MessageSpatial2D/MessageSpatial2DHost.h"
#include "flamegpu/runtime/messaging/MessageSpatial2D/MessageSpatial2DDevice.cuh"
#include "flamegpu/gpu/CUDAMessage.h"
#include "flamegpu/gpu/CUDAScatter.cuh"
#include "flamegpu/util/nvtx.h"
#include "flamegpu/util/detail/cuda.cuh"

namespace flamegpu {

MessageSpatial2D::CUDAModelHandler::CUDAModelHandler(CUDAMessage &a)
    : MessageSpecialisationHandler()
    , sim_message(a) {
    NVTX_RANGE("MessageSpatial2D::CUDAModelHandler::CUDAModelHandler");
    const Data &d = (const Data &)a.getMessageDescription();
    hd_data.radius = d.radius;
    hd_data.min[0] = d.minX;
    hd_data.min[1] = d.minY;
    hd_data.max[0] = d.maxX;
    hd_data.max[1] = d.maxY;
    binCount = 1;
    for (unsigned int axis = 0; axis < 2; ++axis) {
        hd_data.environmentWidth[axis] = hd_data.max[axis] - hd_data.min[axis];
        hd_data.gridDim[axis] = static_cast<unsigned int>(ceil(hd_data.environmentWidth[axis] / hd_data.radius));
        binCount *= hd_data.gridDim[axis];
    }
}
MessageSpatial2D::CUDAModelHandler::~CUDAModelHandler() { }
__global__ void atomicHistogram2D(
    const MessageSpatial2D::MetaData *md,
    unsigned int* bin_index,
    unsigned int* bin_sub_index,
    unsigned int *pbm_counts,
    unsigned int message_count,
    const float * __restrict__ x,
    const float * __restrict__ y) {
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    // Kill excess threads
    if (index >= message_count) return;

    MessageSpatial2D::GridPos2D gridPos = getGridPosition2D(md, x[index], y[index]);
    unsigned int hash = getHash2D(md, gridPos);
    bin_index[index] = hash;
    unsigned int bin_idx = atomicInc((unsigned int*)&pbm_counts[hash], 0xFFFFFFFF);
    bin_sub_index[index] = bin_idx;
}

void MessageSpatial2D::CUDAModelHandler::init(CUDAScatter &, unsigned int, hipStream_t stream) {
    allocateMetaDataDevicePtr(stream);
    // Set PBM to 0
    gpuErrchk(hipMemsetAsync(hd_data.PBM, 0x00000000, (binCount + 1) * sizeof(unsigned int), stream));
    gpuErrchk(hipStreamSynchronize(stream));  // This could probably be skipped/delayed safely
}

void MessageSpatial2D::CUDAModelHandler::allocateMetaDataDevicePtr(hipStream_t stream) {
    if (d_data == nullptr) {
        gpuErrchk(hipMalloc(&d_histogram, (binCount + 1) * sizeof(unsigned int)));
        gpuErrchk(hipMalloc(&hd_data.PBM, (binCount + 1) * sizeof(unsigned int)));
        gpuErrchk(hipMalloc(&d_data, sizeof(MetaData)));
        gpuErrchk(hipMemcpyAsync(d_data, &hd_data, sizeof(MetaData), hipMemcpyHostToDevice, stream));
        gpuErrchk(hipStreamSynchronize(stream));
        resizeCubTemp(stream);
    }
}

void MessageSpatial2D::CUDAModelHandler::freeMetaDataDevicePtr() {
    if (d_data != nullptr) {
        d_CUB_temp_storage_bytes = 0;
        gpuErrchk(flamegpu::util::detail::cuda::hipFree(d_CUB_temp_storage));
        gpuErrchk(flamegpu::util::detail::cuda::hipFree(d_histogram));
        gpuErrchk(flamegpu::util::detail::cuda::hipFree(hd_data.PBM));
        gpuErrchk(flamegpu::util::detail::cuda::hipFree(d_data));
        d_CUB_temp_storage = nullptr;
        d_histogram = nullptr;
        hd_data.PBM = nullptr;
        d_data = nullptr;
        if (d_keys) {
            d_keys_vals_storage_bytes = 0;
            gpuErrchk(flamegpu::util::detail::cuda::hipFree(d_keys));
            gpuErrchk(flamegpu::util::detail::cuda::hipFree(d_vals));
            d_keys = nullptr;
            d_vals = nullptr;
        }
    }
}

void MessageSpatial2D::CUDAModelHandler::buildIndex(CUDAScatter &scatter, unsigned int streamId, hipStream_t stream) {
    NVTX_RANGE("MessageSpatial2D::CUDAModelHandler::buildIndex");
    const unsigned int MESSAGE_COUNT = this->sim_message.getMessageCount();
    resizeKeysVals(this->sim_message.getMaximumListSize());  // Resize based on allocated amount rather than message count
    {  // Build atomic histogram
        gpuErrchk(hipMemsetAsync(d_histogram, 0x00000000, (binCount + 1) * sizeof(unsigned int), stream));
        int blockSize;  // The launch configurator returned block size
        gpuErrchk(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, atomicHistogram2D, 32, 0));  // Randomly 32
                                                                                                         // Round up according to array size
        int gridSize = (MESSAGE_COUNT + blockSize - 1) / blockSize;
        atomicHistogram2D <<<gridSize, blockSize, 0, stream >>>(d_data, d_keys, d_vals, d_histogram, MESSAGE_COUNT,
            reinterpret_cast<float*>(this->sim_message.getReadPtr("x")),
            reinterpret_cast<float*>(this->sim_message.getReadPtr("y")));
    }
    {  // Scan (sum), to finalise PBM
        gpuErrchk(hipcub::DeviceScan::ExclusiveSum(d_CUB_temp_storage, d_CUB_temp_storage_bytes, d_histogram, hd_data.PBM, binCount + 1, stream));
    }
    {  // Reorder messages
       // Copy messages from d_messages to d_messages_swap, in hash order
        scatter.pbm_reorder(streamId, stream, this->sim_message.getMessageDescription().variables, this->sim_message.getReadList(), this->sim_message.getWriteList(), MESSAGE_COUNT, d_keys, d_vals, hd_data.PBM);
        this->sim_message.swap();
        gpuErrchk(hipStreamSynchronize(stream));  // Not striclty neceesary while pbm_reorder is synchronous.
    }
    {  // Fill PBM and Message Texture Buffers
       // gpuErrchk(hipBindTexture(nullptr, d_texMessages, d_agents, sizeof(glm::vec4) * MESSAGE_COUNT));
       // gpuErrchk(hipBindTexture(nullptr, d_texPBM, d_PBM, sizeof(unsigned int) * (binCount + 1)));
    }
}

void MessageSpatial2D::CUDAModelHandler::resizeCubTemp(hipStream_t stream) {
    size_t bytesCheck = 0;
    gpuErrchk(hipcub::DeviceScan::ExclusiveSum(nullptr, bytesCheck, hd_data.PBM, d_histogram, binCount + 1, stream));
    if (bytesCheck > d_CUB_temp_storage_bytes) {
        if (d_CUB_temp_storage) {
            gpuErrchk(flamegpu::util::detail::cuda::hipFree(d_CUB_temp_storage));
        }
        d_CUB_temp_storage_bytes = bytesCheck;
        gpuErrchk(hipMalloc(&d_CUB_temp_storage, d_CUB_temp_storage_bytes));
    }
}

void MessageSpatial2D::CUDAModelHandler::resizeKeysVals(const unsigned int newSize) {
    size_t bytesCheck = newSize * sizeof(unsigned int);
    if (bytesCheck > d_keys_vals_storage_bytes) {
        if (d_keys) {
            gpuErrchk(flamegpu::util::detail::cuda::hipFree(d_keys));
            gpuErrchk(flamegpu::util::detail::cuda::hipFree(d_vals));
        }
        d_keys_vals_storage_bytes = bytesCheck;
        gpuErrchk(hipMalloc(&d_keys, d_keys_vals_storage_bytes));
        gpuErrchk(hipMalloc(&d_vals, d_keys_vals_storage_bytes));
    }
}


MessageSpatial2D::Data::Data(const std::shared_ptr<const ModelData> &model, const std::string &message_name)
    : MessageBruteForce::Data(model, message_name)
    , radius(NAN)
    , minX(NAN)
    , minY(NAN)
    , maxX(NAN)
    , maxY(NAN) {
    description = std::unique_ptr<MessageSpatial2D::Description>(new MessageSpatial2D::Description(model, this));
    description->newVariable<float>("x");
    description->newVariable<float>("y");
}
MessageSpatial2D::Data::Data(const std::shared_ptr<const ModelData> &model, const Data &other)
    : MessageBruteForce::Data(model, other)
    , radius(other.radius)
    , minX(other.minX)
    , minY(other.minY)
    , maxX(other.maxX)
    , maxY(other.maxY) {
    description = std::unique_ptr<MessageSpatial2D::Description>(model ? new MessageSpatial2D::Description(model, this) : nullptr);
    if (isnan(radius)) {
        THROW exception::InvalidMessage("Radius has not been set in spatial message '%s'.", other.name.c_str());
    }
    if (isnan(minX)) {
        THROW exception::InvalidMessage("Environment minimum x bound has not been set in spatial message '%s.", other.name.c_str());
    }
    if (isnan(minY)) {
        THROW exception::InvalidMessage("Environment minimum y bound has not been set in spatial message '%s'.", other.name.c_str());
    }
    if (isnan(maxX)) {
        THROW exception::InvalidMessage("Environment maximum x bound has not been set in spatial message '%s'.", other.name.c_str());
    }
    if (isnan(maxY)) {
        THROW exception::InvalidMessage("Environment maximum y bound has not been set in spatial message '%s'.", other.name.c_str());
    }
}
MessageSpatial2D::Data *MessageSpatial2D::Data::clone(const std::shared_ptr<const ModelData> &newParent) {
    return new Data(newParent, *this);
}
std::unique_ptr<MessageSpecialisationHandler> MessageSpatial2D::Data::getSpecialisationHander(CUDAMessage &owner) const {
    return std::unique_ptr<MessageSpecialisationHandler>(new CUDAModelHandler(owner));
}
std::type_index MessageSpatial2D::Data::getType() const { return std::type_index(typeid(MessageSpatial2D)); }

flamegpu::MessageSortingType MessageSpatial2D::Data::getSortingType() const {
    return flamegpu::MessageSortingType::spatial2D;
}

MessageSpatial2D::Description::Description(const std::shared_ptr<const ModelData> &_model, Data *const data)
    : MessageBruteForce::Description(_model, data) { }

void MessageSpatial2D::Description::setRadius(const float r) {
    if (r <= 0) {
        THROW exception::InvalidArgument("Spatial messaging radius must be a positive value, %f is not valid.", r);
    }
    reinterpret_cast<Data *>(message)->radius = r;
}
void MessageSpatial2D::Description::setMinX(const float x) {
    if (!isnan(reinterpret_cast<Data *>(message)->maxX) &&
        x >= reinterpret_cast<Data *>(message)->maxX) {
        THROW exception::InvalidArgument("Spatial messaging minimum bound must be lower than max bound, %f !< %f.", x, reinterpret_cast<Data *>(message)->maxX);
    }
    reinterpret_cast<Data *>(message)->minX = x;
}
void MessageSpatial2D::Description::setMinY(const float y) {
    if (!isnan(reinterpret_cast<Data *>(message)->maxY) &&
        y >= reinterpret_cast<Data *>(message)->maxY) {
        THROW exception::InvalidArgument("Spatial messaging minimum bound must be lower than max bound, %f !< %f.", y, reinterpret_cast<Data *>(message)->maxY);
    }
    reinterpret_cast<Data *>(message)->minY = y;
}
void MessageSpatial2D::Description::setMin(const float x, const float y) {
    if (!isnan(reinterpret_cast<Data *>(message)->maxX) &&
        x >= reinterpret_cast<Data *>(message)->maxX) {
        THROW exception::InvalidArgument("Spatial messaging minimum bound must be lower than max bound, %f !< %f.", x, reinterpret_cast<Data *>(message)->maxX);
    }
    if (!isnan(reinterpret_cast<Data *>(message)->maxY) &&
        y >= reinterpret_cast<Data *>(message)->maxY) {
        THROW exception::InvalidArgument("Spatial messaging minimum bound must be lower than max bound, %f !< %f.", y, reinterpret_cast<Data *>(message)->maxY);
    }
    reinterpret_cast<Data *>(message)->minX = x;
    reinterpret_cast<Data *>(message)->minY = y;
}
void MessageSpatial2D::Description::setMaxX(const float x) {
    if (!isnan(reinterpret_cast<Data *>(message)->minX) &&
        x <= reinterpret_cast<Data *>(message)->minX) {
        THROW exception::InvalidArgument("Spatial messaging max x bound must be greater than min bound, %f !> %f.", x, reinterpret_cast<Data *>(message)->minX);
    }
    reinterpret_cast<Data *>(message)->maxX = x;
}
void MessageSpatial2D::Description::setMaxY(const float y) {
    if (!isnan(reinterpret_cast<Data *>(message)->minY) &&
        y <= reinterpret_cast<Data *>(message)->minY) {
        THROW exception::InvalidArgument("Spatial messaging max y bound must be greater than min bound, %f !> %f.", y, reinterpret_cast<Data *>(message)->minY);
    }
    reinterpret_cast<Data *>(message)->maxY = y;
}
void MessageSpatial2D::Description::setMax(const float x, const float y) {
    if (!isnan(reinterpret_cast<Data *>(message)->minX) &&
        x <= reinterpret_cast<Data *>(message)->minX) {
        THROW exception::InvalidArgument("Spatial messaging max x bound must be greater than min bound, %f !> %f.", x, reinterpret_cast<Data *>(message)->minX);
    }
    if (!isnan(reinterpret_cast<Data *>(message)->minY) &&
        y <= reinterpret_cast<Data *>(message)->minY) {
        THROW exception::InvalidArgument("Spatial messaging max y bound must be greater than min bound, %f !> %f.", y, reinterpret_cast<Data *>(message)->minY);
    }
    reinterpret_cast<Data *>(message)->maxX = x;
    reinterpret_cast<Data *>(message)->maxY = y;
}

float MessageSpatial2D::Description::getRadius() const {
    return reinterpret_cast<Data *>(message)->radius;
}
float MessageSpatial2D::Description::getMinX() const {
    return reinterpret_cast<Data *>(message)->minX;
}
float MessageSpatial2D::Description::getMinY() const {
    return reinterpret_cast<Data *>(message)->minY;
}
float MessageSpatial2D::Description::getMaxX() const {
    return reinterpret_cast<Data *>(message)->maxX;
}
float MessageSpatial2D::Description::getMaxY() const {
    return reinterpret_cast<Data *>(message)->maxY;
}

}  // namespace flamegpu
