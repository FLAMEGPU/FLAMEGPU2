#include "flamegpu/runtime/environment/HostEnvironment.cuh"

#include <filesystem>
#include <fstream>
#include <iterator>
#include <numeric>
#include <vector>

#include "flamegpu/io/StateWriter.h"
#include "flamegpu/io/StateWriterFactory.h"
#include "flamegpu/io/StateReader.h"
#include "flamegpu/io/StateReaderFactory.h"
#include "flamegpu/simulation/CUDASimulation.h"

namespace flamegpu {
HostEnvironment::HostEnvironment(CUDASimulation &_simulation, std::shared_ptr<detail::EnvironmentManager> env, std::shared_ptr<detail::CUDAMacroEnvironment> _macro_env,
    CUDADirectedGraphMap& _directed_graph_map, detail::CUDAScatter& _scatter, const unsigned int _streamID, const hipStream_t _stream)
    : env_mgr(std::move(env))
    , macro_env(std::move(_macro_env))
    , directed_graph_map(_directed_graph_map)
    , instance_id(_simulation.getInstanceID())
    , simulation(_simulation)
    , scatter(_scatter)
    , streamID(_streamID)
    , stream(_stream) { }

void HostEnvironment::importMacroProperty(const std::string& property_name, const std::string& file_path) const {
    // Validate the property exists
    const auto &m_props = macro_env->getPropertiesMap();
    const auto &m_prop = m_props.find(property_name);
    if (m_prop == m_props.end()) {
        THROW exception::InvalidEnvProperty("The environment macro property '%s' was not found within the model description, in HostEnvironment::importMacroProperty().", property_name.c_str());
    }
    const unsigned int m_prop_elements = std::accumulate(m_prop->second.elements.begin(), m_prop->second.elements.end(), 1, std::multiplies<unsigned int>());
    try {
        io::StateReader *read__ = io::StateReaderFactory::createReader(file_path);
        read__->parse(file_path, simulation.getModelDescription().shared_from_this(), Verbosity::Quiet);
        std::unordered_map<std::string, std::vector<char>> macro_init;
        read__->getMacroEnvironment(macro_init);
        // Validate the property exists within macro_init
        const auto &l_prop = macro_init.find(property_name);
        if (l_prop == macro_init.end()) {
            THROW exception::InvalidEnvProperty("The environment macro property '%s' was not found within the input file '%s'.", property_name.c_str(), file_path.c_str());
        }
        // Check the length validates
        if (l_prop->second.size() != m_prop_elements * m_prop->second.type_size) {
            THROW exception::InvalidInputFile("Length of input file '%s's environment macro property '%s'  does not match, (%u != %u), in HostEnvironment::importMacroProperty()",
                file_path.c_str(), property_name.c_str(), static_cast<unsigned int>(l_prop->second.size()), static_cast<unsigned int>(m_prop_elements * m_prop->second.type_size));
        }
        gpuErrchk(hipMemcpyAsync(m_prop->second.d_ptr, l_prop->second.data(), l_prop->second.size(), hipMemcpyHostToDevice, stream));
    } catch (const exception::UnsupportedFileType&) {
        const std::string extension = std::filesystem::path(file_path).extension().string();
        if (extension == ".bin") {
            // Additionally support raw binary dump
            // Read the file
            std::ifstream input(file_path, std::ios::binary);
            std::vector buffer(std::istreambuf_iterator<char>(input), {});
            // Check the length validates
            if (buffer.size() != m_prop_elements * m_prop->second.type_size) {
                THROW exception::InvalidInputFile("Length of binary input file '%s' does not match the environment macro property '%s', (%u != %u), in HostEnvironment::importMacroProperty()",
                    file_path.c_str(), property_name.c_str(), static_cast<unsigned int>(buffer.size()), static_cast<unsigned int>(m_prop_elements * m_prop->second.type_size));
            }
            // Update the property
            gpuErrchk(hipMemcpyAsync(m_prop->second.d_ptr, buffer.data(), buffer.size(), hipMemcpyHostToDevice, stream));
        } else {
            throw;
        }
    }
    gpuErrchk(hipStreamSynchronize(stream));
    // If macro property exists in cache sync cache
    if (const auto cache = macro_env->getHostPropertyMetadata(property_name)) {
        cache->force_download();
    }
}
void HostEnvironment::exportMacroProperty(const std::string& property_name, const std::string& file_path, bool pretty_print) const {
    // If macro property exists in cache sync cache
    if (const auto cache = macro_env->getHostPropertyMetadata(property_name)) {
        cache->upload();
    }
    try {
        io::StateWriter* write__ = io::StateWriterFactory::createWriter(file_path);
        write__->beginWrite(file_path, pretty_print);
        write__->writeMacroEnvironment(macro_env, { property_name });
        write__->endWrite();
    }
    catch (const exception::UnsupportedFileType&) {
        const std::string extension = std::filesystem::path(file_path).extension().string();
        if (extension == ".bin") {
            // Additionally support raw binary dump
            // Validate the property exists
            const auto& m_props = macro_env->getPropertiesMap();
            const auto& m_prop = m_props.find(property_name);
            if (m_prop == m_props.end()) {
                THROW exception::InvalidEnvProperty("The environment macro property '%s' was not found within the model description, in HostEnvironment::exportMacroProperty().", property_name.c_str());
            }
            // Check the file doesn't already exist
            if (std::filesystem::exists(file_path)) {
                THROW exception::FileAlreadyExists("The binary output file '%s' already exists, in HostEnvironment::exportMacroProperty().", file_path.c_str());
            }
            // Copy the data to a temporary buffer on host
            const unsigned int m_prop_elements = std::accumulate(m_prop->second.elements.begin(), m_prop->second.elements.end(), 1, std::multiplies<unsigned int>());
            std::vector<char> buffer;
            buffer.resize(m_prop_elements * m_prop->second.type_size);
            gpuErrchk(hipMemcpyAsync(buffer.data(), m_prop->second.d_ptr, m_prop_elements * m_prop->second.type_size, hipMemcpyDeviceToHost, stream));
            gpuErrchk(hipStreamSynchronize(stream));
            // Output to file
            std::ofstream output(file_path, std::ios::binary);
            output.write(buffer.data(), buffer.size());
        } else {
            throw;
        }
    }
}

HostEnvironmentDirectedGraph HostEnvironment::getDirectedGraph(const std::string& name) const {
    const auto rt = directed_graph_map.find(name);
    if (rt != directed_graph_map.end())
        return HostEnvironmentDirectedGraph(rt->second, stream, scatter, streamID);
    THROW exception::InvalidGraphName("Directed Graph with name '%s' was not found, "
        "in HostEnvironment::getDirectedGraph()",
        name.c_str());
}

}  // namespace flamegpu
