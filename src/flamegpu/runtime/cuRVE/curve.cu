#include <hip/hip_runtime.h>

#include <cstdio>
#include <cassert>
#include <map>
#include <memory>

#include "flamegpu/runtime/cuRVE/curve.h"


#include "flamegpu/gpu/CUDAErrorChecking.h"
#include "flamegpu/util/nvtx.h"

namespace curve_internal {
    __constant__ Curve::VariableHash d_hashes[Curve::MAX_VARIABLES];  // Device array of the hash values of registered variables
    __device__ char* d_variables[Curve::MAX_VARIABLES];               // Device array of pointer to device memory addresses for variable storage
    __constant__ size_t d_sizes[Curve::MAX_VARIABLES];                // Device array of the types of registered variables
    __constant__ unsigned int d_lengths[Curve::MAX_VARIABLES];        // Device array of the length of registered variables (i.e: vector length)

    __device__ Curve::DeviceError d_curve_error;
    Curve::HostError h_curve_error;
}  // namespace curve_internal

std::mutex Curve::instance_mutex;

/* header implementations */
__host__ Curve::Curve() :
    deviceInitialised(false) {
    // Initialise some host variables.
    curve_internal::h_curve_error  = ERROR_NO_ERRORS;
}
__host__ void Curve::purge() {
    auto lock = std::unique_lock<std::shared_timed_mutex>(mutex);
    deviceInitialised = false;
    curve_internal::h_curve_error = ERROR_NO_ERRORS;
    initialiseDevice();
}
__host__ void Curve::initialiseDevice() {
    // Don't lock mutex here, do it in the calling method
    if (!deviceInitialised) {
        unsigned int *_d_hashes;
        char** _d_variables;
        unsigned int* _d_lengths;
        size_t* _d_sizes;

        // get a host pointer to d_hashes and d_variables
        gpuErrchk(hipGetSymbolAddress(reinterpret_cast<void **>(&_d_hashes), curve_internal::d_hashes));
        gpuErrchk(hipGetSymbolAddress(reinterpret_cast<void **>(&_d_variables), curve_internal::d_variables));
        gpuErrchk(hipGetSymbolAddress(reinterpret_cast<void **>(&_d_lengths), curve_internal::d_lengths));
        gpuErrchk(hipGetSymbolAddress(reinterpret_cast<void **>(&_d_sizes), curve_internal::d_sizes));

        // set values of hash table to 0 on host and device
        memset(h_hashes, 0, sizeof(unsigned int)*MAX_VARIABLES);
        memset(h_lengths, 0, sizeof(unsigned int)*MAX_VARIABLES);
        memset(h_sizes, 0, sizeof(size_t)*MAX_VARIABLES);

        // initialise data to 0 on device
        gpuErrchk(hipMemset(_d_hashes, 0, sizeof(unsigned int)*MAX_VARIABLES));
        gpuErrchk(hipMemset(_d_variables, 0, sizeof(void*)*MAX_VARIABLES));
        gpuErrchk(hipMemset(_d_lengths, 0, sizeof(unsigned int)*MAX_VARIABLES));
        gpuErrchk(hipMemset(_d_sizes, 0, sizeof(size_t)*MAX_VARIABLES));
    }
    deviceInitialised = true;
}

__host__ Curve::VariableHash Curve::variableRuntimeHash(const char* str) {
    // Method is static, does not require mutex
    const size_t length = std::strlen(str) + 1;
    unsigned int hash = 2166136261u;

    for (size_t i = 0; i < length; ++i) {
        hash ^= *str++;
        hash *= 16777619u;
    }
    return hash;
}
__host__ Curve::VariableHash Curve::variableRuntimeHash(unsigned int num) {
    // Method is static, does not require mutex
    return variableRuntimeHash(std::to_string(num).c_str());
}

__host__ Curve::Variable Curve::getVariableHandle(VariableHash variable_hash) {
    // Method is static, does not require mutex
    unsigned int n = 0;
    unsigned int i = (variable_hash) % MAX_VARIABLES;

    while (h_hashes[i] != EMPTY_FLAG) {
        if (h_hashes[i] == variable_hash) {
            return i;
        }
        n += 1;
        if (n >= MAX_VARIABLES) {
            break;
        }
        i += 1;
        if (i >= MAX_VARIABLES) {
            i = 0;
        }
    }
    return UNKNOWN_VARIABLE;
}

__host__ Curve::Variable Curve::registerVariableByHash(VariableHash variable_hash, void * d_ptr, size_t size, unsigned int length) {
    auto lock = std::unique_lock<std::shared_timed_mutex>(mutex);
    return _registerVariableByHash(variable_hash, d_ptr, size, length);
}
__host__ Curve::Variable Curve::_registerVariableByHash(VariableHash variable_hash, void * d_ptr, size_t size, unsigned int length) {
    // Do not lock mutex here, do it in the calling method
    unsigned int n = 0;
    assert(variable_hash != EMPTY_FLAG);
    assert(variable_hash != DELETED_FLAG);
    unsigned int i = (variable_hash) % MAX_VARIABLES;
    while (h_hashes[i] != EMPTY_FLAG && h_hashes[i] != DELETED_FLAG) {
        n += 1;
        if (n >= MAX_VARIABLES) {
            curve_internal::h_curve_error = ERROR_TOO_MANY_VARIABLES;
            return UNKNOWN_VARIABLE;
        }
        i += 1;
        if (i >= MAX_VARIABLES) {
            i = 0;
        }
    }

    h_hashes[i] = variable_hash;

    // make a host copy of the pointer
    h_d_variables[i] = d_ptr;

    // set the size of the data type
    h_sizes[i] = size;

    // set the length of variable
    h_lengths[i] = length;

    return i;
}
__host__ int Curve::size() const {
    auto lock = std::shared_lock<std::shared_timed_mutex>(mutex);
    return _size();
}
__host__ int Curve::_size() const {
    int rtn = 0;
    for (unsigned int hash : h_hashes) {
        if (hash != EMPTY_FLAG && hash != DELETED_FLAG)
            rtn++;
    }
    return rtn;
}
/**
 * TODO: Does un-registering imply that other variable with collisions will no longer be found. I.e. do you need to re-register all other variable when one is removed.
 */
__host__ void Curve::unregisterVariableByHash(VariableHash variable_hash) {
    auto lock = std::unique_lock<std::shared_timed_mutex>(mutex);
    _unregisterVariableByHash(variable_hash);
}
__host__ void Curve::_unregisterVariableByHash(VariableHash variable_hash) {
    // Do not lock mutex here, do it in the calling method
    // get the curve variable
    Variable cv = getVariableHandle(variable_hash);

    // error checking
    if (cv == UNKNOWN_VARIABLE) {
        THROW CurveException("Cannot unregister '%u', hash not found within curve table.", variable_hash);
    }

    // clear hash location on host and copy hash to device
    h_hashes[cv] = DELETED_FLAG;

    // set a host pointer to nullptr and copy to the device
    h_d_variables[cv] = 0;

    // set the empty size to 0
    h_sizes[cv] = 0;

    // set the length of variable to 0
    h_lengths[cv] = 0;
}
__host__ void Curve::updateDevice() {
    auto lock = std::shared_lock<std::shared_timed_mutex>(mutex);
    NVTX_RANGE("Curve::updateDevice()");
    // Initialise the device (if required)
    assert(deviceInitialised);  // No reason for this to ever fail. Purge calls init device
    // Copy
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curve_internal::d_hashes), h_hashes, sizeof(unsigned int) * MAX_VARIABLES));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curve_internal::d_variables), h_d_variables, sizeof(void*) * MAX_VARIABLES));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curve_internal::d_sizes), h_sizes, sizeof(size_t) * MAX_VARIABLES));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curve_internal::d_lengths), h_lengths, sizeof(unsigned int) * MAX_VARIABLES));
}

/* errors */
void __host__ Curve::printLastHostError(const char* file, const char* function, const int line) {
    // Do not lock mutex here, do it in the calling method
    if (curve_internal::h_curve_error != ERROR_NO_ERRORS) {
        printf("%s.%s.%d: cuRVE Host Error %d (%s)\n", file, function, line, (unsigned int)curve_internal::h_curve_error, getHostErrorString(curve_internal::h_curve_error));
    }
}

void __host__ Curve::printErrors(const char* file, const char* function, const int line) {
    auto lock = std::unique_lock<std::shared_timed_mutex>(mutex);
    // Initialise the device (if required)
    initialiseDevice();

    DeviceError d_curve_error_local;

    printLastHostError(file, function, line);

    // check device errors
    gpuErrchk(hipMemcpyFromSymbol(&d_curve_error_local, HIP_SYMBOL(curve_internal::d_curve_error), sizeof(DeviceError)));
    if (d_curve_error_local != DEVICE_ERROR_NO_ERRORS) {
        printf("%s.%s.%d: cuRVE Device Error %d (%s)\n", file, function, line, (unsigned int)d_curve_error_local, getDeviceErrorString(d_curve_error_local));
    }
}
__host__ const char* Curve::getHostErrorString(HostError e) {
    // Do not lock mutex here, do it in the calling method
    switch (e) {
    case(ERROR_NO_ERRORS):
        return "No cuRVE errors";
    case(ERROR_UNKNOWN_VARIABLE):
        return "Unknown cuRVE variable";
    case(ERROR_TOO_MANY_VARIABLES):
        return "Too many cuRVE variables";
    default:
        return "Unspecified cuRVE error";
    }
}
__host__ Curve::HostError Curve::getLastHostError() {
    auto lock = std::shared_lock<std::shared_timed_mutex>(mutex);
    return curve_internal::h_curve_error;
}
__host__ void Curve::clearErrors() {
    auto lock = std::unique_lock<std::shared_timed_mutex>(mutex);
    // Initialise the device (if required)
    initialiseDevice();

    DeviceError curve_error_none;

    curve_error_none = DEVICE_ERROR_NO_ERRORS;
    curve_internal::h_curve_error  = ERROR_NO_ERRORS;

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curve_internal::d_curve_error), &curve_error_none, sizeof(DeviceError)));
}

__host__ unsigned int Curve::checkHowManyMappedItems() {
    auto lock = std::shared_lock<std::shared_timed_mutex>(mutex);
    unsigned int rtn = 0;
    for (unsigned int i = 0; i < MAX_VARIABLES; ++i)
        if (h_hashes[i] != EMPTY_FLAG && h_hashes[i] != DELETED_FLAG)
            rtn++;
    return rtn;
}
Curve& Curve::getInstance() {
    auto lock = std::unique_lock<std::mutex>(instance_mutex);  // Mutex to protect from two threads triggering the static instantiation concurrently
    static std::map<int, std::unique_ptr<Curve>> instances = {};  // Instantiated on first use.
    int device_id = -1;
    gpuErrchk(hipGetDevice(&device_id));
    // Can't use operator[] here, constructor is private
    const auto f = instances.find(device_id);
    if (f != instances.end())
        return *f->second;
    return *(instances.emplace(device_id, std::unique_ptr<Curve>(new Curve())).first->second);
}
