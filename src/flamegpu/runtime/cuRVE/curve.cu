#include <hip/hip_runtime.h>

#include <cstdio>
#include <cassert>

#include "flamegpu/runtime/cuRVE/curve.h"
#include "flamegpu/gpu/CUDAErrorChecking.h"

namespace curve_internal {
    __constant__ Curve::NamespaceHash d_namespace;
    __constant__ Curve::VariableHash d_hashes[Curve::MAX_VARIABLES];  // Device array of the hash values of registered variables
    __device__ char* d_variables[Curve::MAX_VARIABLES];               // Device array of pointer to device memory addresses for variable storage
    __constant__ int d_states[Curve::MAX_VARIABLES];                  // Device array of the states of registered variables
    __constant__ size_t d_sizes[Curve::MAX_VARIABLES];                // Device array of the types of registered variables
    __constant__ unsigned int d_lengths[Curve::MAX_VARIABLES];        // Device array of the length of registered variables (i.e: vector length)

    __device__ Curve::DeviceError d_curve_error;
    Curve::HostError h_curve_error;
}  // namespace curve_internal

/* header implementations */
__host__ Curve::Curve() :
    deviceInitialised(false) {
    // Initialise some host variables.
    curve_internal::h_curve_error  = ERROR_NO_ERRORS;
}
__host__ void Curve::purge() {
    deviceInitialised = false;
    curve_internal::h_curve_error = ERROR_NO_ERRORS;
    initialiseDevice();
}
__host__ void Curve::initialiseDevice() {
    if (!deviceInitialised) {
        unsigned int *_d_hashes;
        char** _d_variables;
        int* _d_states;
        unsigned int* _d_lengths;
        size_t* _d_sizes;

        // namespace
        h_namespace = NAMESPACE_NONE;
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curve_internal::d_namespace), &h_namespace, sizeof(unsigned int)));

        // get a host pointer to d_hashes and d_variables
        gpuErrchk(hipGetSymbolAddress(reinterpret_cast<void **>(&_d_hashes), curve_internal::d_hashes));
        gpuErrchk(hipGetSymbolAddress(reinterpret_cast<void **>(&_d_variables), curve_internal::d_variables));
        gpuErrchk(hipGetSymbolAddress(reinterpret_cast<void **>(&_d_states), curve_internal::d_states));
        gpuErrchk(hipGetSymbolAddress(reinterpret_cast<void **>(&_d_lengths), curve_internal::d_lengths));
        gpuErrchk(hipGetSymbolAddress(reinterpret_cast<void **>(&_d_sizes), curve_internal::d_sizes));

        // set values of hash table to 0 on host and device
        memset(h_hashes, 0, sizeof(unsigned int)*MAX_VARIABLES);
        memset(h_lengths, 0, sizeof(unsigned int)*MAX_VARIABLES);
        memset(h_states, 0, sizeof(int)*MAX_VARIABLES);
        memset(h_sizes, 0, sizeof(size_t)*MAX_VARIABLES);

        // initialise data to 0 on device
        gpuErrchk(hipMemset(_d_hashes, 0, sizeof(unsigned int)*MAX_VARIABLES));
        gpuErrchk(hipMemset(_d_variables, 0, sizeof(void*)*MAX_VARIABLES));
        gpuErrchk(hipMemset(_d_states, VARIABLE_DISABLED, sizeof(int)*MAX_VARIABLES));
        gpuErrchk(hipMemset(_d_lengths, 0, sizeof(unsigned int)*MAX_VARIABLES));
        gpuErrchk(hipMemset(_d_sizes, 0, sizeof(size_t)*MAX_VARIABLES));
    }
    deviceInitialised = true;
}

__host__ Curve::VariableHash Curve::variableRuntimeHash(const char* str) {
    const size_t length = std::strlen(str) + 1;
    unsigned int hash = 2166136261u;

    for (size_t i = 0; i < length; ++i) {
        hash ^= *str++;
        hash *= 16777619u;
    }
    return hash;
}
__host__ Curve::VariableHash Curve::variableRuntimeHash(unsigned int num) {
    return variableRuntimeHash(std::to_string(num).c_str());
}

__host__ Curve::Variable Curve::getVariableHandle(VariableHash variable_hash) {
    variable_hash += h_namespace;
    unsigned int n = 0;
    unsigned int i = (variable_hash) % MAX_VARIABLES;

    while (h_hashes[i] != EMPTY_FLAG) {
        if (h_hashes[i] == variable_hash) {
            return i;
        }
        n += 1;
        if (n >= MAX_VARIABLES) {
            break;
        }
        i += 1;
        if (i >= MAX_VARIABLES) {
            i = 0;
        }
    }
    return UNKNOWN_VARIABLE;
}

__host__ Curve::Variable Curve::registerVariableByHash(VariableHash variable_hash, void * d_ptr, size_t size, unsigned int length) {
    unsigned int n = 0;
    variable_hash += h_namespace;
    assert(variable_hash != EMPTY_FLAG);
    assert(variable_hash != DELETED_FLAG);
    unsigned int i = (variable_hash) % MAX_VARIABLES;

    while (h_hashes[i] != EMPTY_FLAG && h_hashes[i] != DELETED_FLAG) {
        n += 1;
        if (n >= MAX_VARIABLES) {
            curve_internal::h_curve_error = ERROR_TOO_MANY_VARIABLES;
            return UNKNOWN_VARIABLE;
        }
        i += 1;
        if (i >= MAX_VARIABLES) {
            i = 0;
        }
    }
    h_hashes[i] = variable_hash;

    // make a host copy of the pointer
    h_d_variables[i] = d_ptr;

    // set the state to enabled
    h_states[i] = VARIABLE_ENABLED;

    // set the size of the data type
    h_sizes[i] = size;

    // set the length of variable
    h_lengths[i] = length;

    return i;
}

/**
 * TODO: Does un-registering imply that other variable with collisions will no longer be found. I.e. do you need to re-register all other variable when one is removed.
 */
__host__ void Curve::unregisterVariableByHash(VariableHash variable_hash) {
    // get the curve variable
    Variable cv = getVariableHandle(variable_hash);

    // error checking
    if (cv == UNKNOWN_VARIABLE) {
        curve_internal::h_curve_error = ERROR_UNKNOWN_VARIABLE;
        return;
    }

    // clear hash location on host and copy hash to device
    h_hashes[cv] = DELETED_FLAG;

    // set a host pointer to nullptr and copy to the device
    h_d_variables[cv] = 0;

    // return the state to disabled
    h_states[cv] = VARIABLE_DISABLED;

    // set the empty size to 0
    h_sizes[cv] = 0;

    // set the length of variable to 0
    h_lengths[cv] = 0;
}
__host__ void Curve::updateDevice() {
    // Initialise the device (if required)
    initialiseDevice();
    // Copy
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curve_internal::d_hashes), h_hashes, sizeof(unsigned int) * MAX_VARIABLES));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curve_internal::d_variables), h_d_variables, sizeof(void*) * MAX_VARIABLES));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curve_internal::d_states), h_states, sizeof(int) * MAX_VARIABLES));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curve_internal::d_sizes), h_sizes, sizeof(size_t) * MAX_VARIABLES));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curve_internal::d_lengths), h_lengths, sizeof(unsigned int) * MAX_VARIABLES));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curve_internal::d_namespace), &h_namespace, sizeof(unsigned int)));
}

__host__ void Curve::disableVariableByHash(VariableHash variable_hash) {
    Variable cv = getVariableHandle(variable_hash);

    // error checking
    if (cv == UNKNOWN_VARIABLE) {
        curve_internal::h_curve_error = ERROR_UNKNOWN_VARIABLE;
        return;
    }

    h_states[cv] = VARIABLE_DISABLED;
}
__host__ void Curve::enableVariableByHash(VariableHash variable_hash) {
    Variable cv = getVariableHandle(variable_hash);

    // error checking
    if (cv == UNKNOWN_VARIABLE) {
        curve_internal::h_curve_error = ERROR_UNKNOWN_VARIABLE;
        return;
    }

    h_states[cv] = VARIABLE_ENABLED;
}
__host__ void Curve::setNamespaceByHash(NamespaceHash namespace_hash) {
    h_namespace = namespace_hash;
}

__host__ void Curve::setDefaultNamespace() {
    h_namespace = NAMESPACE_NONE;
}

/* errors */
void __host__ Curve::printLastHostError(const char* file, const char* function, const int line) {
    if (curve_internal::h_curve_error != ERROR_NO_ERRORS) {
        printf("%s.%s.%d: cuRVE Host Error %d (%s)\n", file, function, line, (unsigned int)curve_internal::h_curve_error, getHostErrorString(curve_internal::h_curve_error));
    }
}

void __host__ Curve::printErrors(const char* file, const char* function, const int line) {
    // Initialise the device (if required)
    initialiseDevice();

    DeviceError d_curve_error_local;

    printLastHostError(file, function, line);

    // check device errors
    gpuErrchk(hipMemcpyFromSymbol(&d_curve_error_local, HIP_SYMBOL(curve_internal::d_curve_error), sizeof(DeviceError)));
    if (d_curve_error_local != DEVICE_ERROR_NO_ERRORS) {
        printf("%s.%s.%d: cuRVE Device Error %d (%s)\n", file, function, line, (unsigned int)d_curve_error_local, getDeviceErrorString(d_curve_error_local));
    }
}
__host__ const char* Curve::getHostErrorString(HostError e) {
    switch (e) {
    case(ERROR_NO_ERRORS):
        return "No cuRVE errors";
    case(ERROR_UNKNOWN_VARIABLE):
        return "Unknown cuRVE variable";
    case(ERROR_TOO_MANY_VARIABLES):
        return "Too many cuRVE variables";
    default:
        return "Unspecified cuRVE error";
    }
}
__host__ Curve::HostError Curve::getLastHostError() {
    return curve_internal::h_curve_error;
}
__host__ void Curve::clearErrors() {
    // Initialise the device (if required)
    initialiseDevice();

    DeviceError curve_error_none;

    curve_error_none = DEVICE_ERROR_NO_ERRORS;
    curve_internal::h_curve_error  = ERROR_NO_ERRORS;

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curve_internal::d_curve_error), &curve_error_none, sizeof(DeviceError)));
}

__host__ unsigned int Curve::checkHowManyMappedItems() {
    unsigned int rtn = 0;
    for (unsigned int i = 0; i < MAX_VARIABLES; ++i)
        if (h_hashes[i] != EMPTY_FLAG && h_hashes[i] != DELETED_FLAG)
            rtn++;
    return rtn;
}
