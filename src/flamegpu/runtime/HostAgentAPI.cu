#include "hip/hip_runtime.h"
#include "flamegpu/runtime/HostAgentAPI.h"
#include "flamegpu/pop/DeviceAgentVector_impl.h"

namespace flamegpu {

HostAgentAPI::~HostAgentAPI() {
    if (population) {
        population->syncChanges();
        population.reset();
    }
}

HostNewAgentAPI HostAgentAPI::newAgent() {
    // Create the agent in our backing data structure
    newAgentData.emplace_back(NewAgentStorage(agentOffsets, agent.nextID(1)));
    // Point the returned object to the created agent
    return HostNewAgentAPI(newAgentData.back());
}

unsigned HostAgentAPI::count() {
    if (population) {
        // If the user has a DeviceAgentVector out, use that instead
        return population->size();
    }
    return agent.getStateSize(stateName);
}

__global__ void initToThreadIndex(unsigned int *output, unsigned int threadCount) {
    const unsigned int TID = blockIdx.x * blockDim.x + threadIdx.x;
    if (TID < threadCount) {
        output[TID] = TID;
    }
}

void HostAgentAPI::fillTIDArray(unsigned int *buffer, const unsigned int &threadCount, const hipStream_t &stream) {
    initToThreadIndex<<<(threadCount/512)+1, 512, 0, stream>>>(buffer, threadCount);
    gpuErrchkLaunch();
}

__global__ void sortBuffer_kernel(char *dest, char*src, unsigned int *position, size_t typeLen, unsigned int threadCount) {
    const unsigned int TID = blockIdx.x * blockDim.x + threadIdx.x;
    if (TID < threadCount) {
        memcpy(dest + TID * typeLen, src + position[TID] * typeLen, typeLen);
    }
}

void HostAgentAPI::sortBuffer(void *dest, void*src, unsigned int *position, const size_t &typeLen, const unsigned int &threadCount, const hipStream_t &stream) {
    sortBuffer_kernel<<<(threadCount/512)+1, 512, 0, stream >>>(static_cast<char*>(dest), static_cast<char*>(src), position, typeLen, threadCount);
    gpuErrchkLaunch();
}

DeviceAgentVector HostAgentAPI::getPopulationData() {
    // Create and return a new AgentVector
    if (!population) {
        population = std::make_shared<DeviceAgentVector_impl>(static_cast<CUDAAgent&>(agent), stateName, agentOffsets, newAgentData, api.scatter, api.streamId, api.stream);
    }
    return *population;
}

}  // namespace flamegpu
