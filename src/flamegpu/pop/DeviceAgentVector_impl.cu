#include "hip/hip_runtime.h"
#include "flamegpu/pop/DeviceAgentVector_impl.h"
#include "flamegpu/gpu/CUDAAgent.h"

namespace flamegpu {

DeviceAgentVector_impl::DeviceAgentVector_impl(CUDAAgent& _cuda_agent, const std::string &_cuda_agent_state,
    const VarOffsetStruct& _agentOffsets, std::vector<NewAgentStorage>& _newAgentData,
    CUDAScatter& _scatter, const unsigned int& _streamId, const hipStream_t& _stream)
    : AgentVector(_cuda_agent.getAgentDescription(), 0)
    , unbound_buffers_has_changed(false)
    , known_device_buffer_size(_cuda_agent.getStateSize(_cuda_agent_state))
    , cuda_agent(_cuda_agent)
    , cuda_agent_state(_cuda_agent_state)
    , agentOffsets(_agentOffsets)
    , newAgentData(_newAgentData)
    , scatter(_scatter)
    , streamId(_streamId)
    , stream(_stream) {
    // Create an empty AgentVector and initialise it manually
    // For each variable create an uninitialised array of variable data
    _size = known_device_buffer_size;
    internal_resize(_size, false);
    // Mark all variables as Invalid
    for (const auto& v : agent->variables)
        invalid_variables.insert(v.first);
    // Grab the unbound variable buffers from the CUDAFatAgentStateList
    // Leave their host counterparts de-allocated until required
    {
        const auto buffs = cuda_agent.getUnboundVariableBuffers(cuda_agent_state);
        for (auto &d_buff : buffs)
            unbound_buffers.emplace_back(d_buff);
        unbound_host_buffer_invalid = true;
    }
}

void DeviceAgentVector_impl::syncChanges() {
    // Resize device buffers if necessary
    const unsigned int old_allocated_size = cuda_agent.getStateAllocatedSize(cuda_agent_state);
    if (_size > old_allocated_size) {
        const unsigned int old_size = cuda_agent.getStateSize(cuda_agent_state);
        // Resize the underlying variable buffers for this agent state and retain variable data
        cuda_agent.resizeState(cuda_agent_state, _size, true);  // @todo Don't retain data for mapped buffers?
        // Init agent data for any variables of newly created agents which are only present in a parent model
        const unsigned int new_allocated_size = cuda_agent.getStateAllocatedSize(cuda_agent_state);
        // This call does not use streams properly internally
        cuda_agent.initExcludedVars(cuda_agent_state, new_allocated_size - old_size, old_size, scatter, streamId, stream);
    }
    _requireLength();
    // Copy all changes back to device
    for (const auto &ch : change_detail) {
        auto &v = agent->variables.at(ch.first);
        // Copy back variable data into each array
        const char* host_src = static_cast<const char*>(_data->at(ch.first)->getDataPtr());
        char* device_dest = static_cast<char*>(cuda_agent.getStateVariablePtr(cuda_agent_state, ch.first));
        const size_t copy_offset = ch.second.first * v.type_size * v.elements;
        const size_t copy_len = (ch.second.second - ch.second.first) * v.type_size * v.elements;
        gpuErrchk(hipMemcpyAsync(device_dest + copy_offset, host_src + copy_offset, copy_len, hipMemcpyHostToDevice, stream));
    }
    change_detail.clear();
    // Copy all unbound buffes
    if (unbound_buffers_has_changed) {
        if (unbound_host_buffer_size != _size) {
            THROW exception::InvalidOperation("Unbound buffers have gone out of sync, in DeviceAgentVector::syncChanges().\n");
        }
        for (auto &buff : unbound_buffers) {
            const size_t variable_size = buff.device->type_size * buff.device->elements;
            gpuErrchk(hipMemcpyAsync(buff.device->data, buff.host, unbound_host_buffer_size * variable_size, hipMemcpyHostToDevice, stream));
        }
        unbound_buffers_has_changed = false;
    }
    gpuErrchk(hipStreamSynchronize(stream));
    // Update CUDAAgent statelist size
    cuda_agent.setStateAgentCount(cuda_agent_state, _size);
}
void DeviceAgentVector_impl::purgeCache() {
    _size = cuda_agent.getStateSize(cuda_agent_state);
    // All variables are now invalid
    for (const auto& v : agent->variables)
        invalid_variables.insert(v.first);
    // Mark all unbound host buffers as requiring update
    unbound_host_buffer_invalid = false;
    unbound_host_buffer_size = 0;
    known_device_buffer_size = cuda_agent.getStateSize(cuda_agent_state);
    unbound_buffers_has_changed = false;
}

void DeviceAgentVector_impl::initUnboundBuffers() {
    if (!_capacity)
      return;
    const unsigned int device_len = cuda_agent.getStateSize(cuda_agent_state);
    const unsigned int copy_len = _size < device_len ? _size : device_len;
    // Resize to match _capacity
    for (auto &buff : unbound_buffers) {
        if (buff.host) {
            THROW exception::InvalidOperation("Host buffer is already allocated, in DeviceAgentVector::initUnboundBuffers().\n");
        }
        // Alloc
        const size_t var_size = buff.device->type_size * buff.device->elements;
        buff.host = static_cast<char*>(malloc(_capacity * var_size));
        // DtH memcpy
        gpuErrchk(hipMemcpyAsync(buff.host, buff.device->data, copy_len * var_size, hipMemcpyDeviceToHost, stream));
        // Not sure this will ever happen, but better safe
        for (unsigned int i = device_len; i < _size; ++i) {
            // We have unknown agents, default init them
            memcpy(buff.host + i * var_size, buff.device->default_value, var_size);
        }
    }
    gpuErrchk(hipStreamSynchronize(stream));
    unbound_host_buffer_capacity = _capacity;
    unbound_host_buffer_size = copy_len;
    unbound_buffers_has_changed = true;  // Probably not required, but if they are being init, high chance they're going to be changed
    unbound_host_buffer_invalid = false;
}
void DeviceAgentVector_impl::reinitUnboundBuffers() {
    const unsigned int device_len = cuda_agent.getStateSize(cuda_agent_state);
    const unsigned int copy_len = _size;
    if (device_len > _size) {
        THROW exception::InvalidOperation("Unexpected state, in DeviceAgentVector::reinitUnboundBuffers()\n");
    }
    // Resize to match _capacity
    for (auto& buff : unbound_buffers) {
        if (!buff.host) {
            THROW exception::InvalidOperation("Host buffer is not already allocated, in DeviceAgentVector::reinitUnboundBuffers().\n");
        }
        const size_t var_size = buff.device->type_size * buff.device->elements;
        if (unbound_host_buffer_capacity < _capacity) {
            free(buff.host);
            // Alloc
            buff.host = static_cast<char*>(malloc(_capacity * var_size));
        }
        // DtH memcpy
        gpuErrchk(hipMemcpyAsync(buff.host, buff.device->data, copy_len * var_size, hipMemcpyDeviceToHost, stream));
        // Not sure this will ever happen, but better safe
        for (unsigned int i = device_len; i < _size; ++i) {
            // We have unknown agents, default init them
            memcpy(buff.host + i * var_size, buff.device->default_value, var_size);
        }
    }
    gpuErrchk(hipStreamSynchronize(stream));
    unbound_host_buffer_capacity = unbound_host_buffer_capacity < _capacity ?_capacity : unbound_host_buffer_capacity;
    unbound_host_buffer_size = copy_len;
    unbound_buffers_has_changed = true;  // Probably not required, but if they are being init, high chance they're going to be changed
    unbound_host_buffer_invalid = false;
}
void DeviceAgentVector_impl::resizeUnboundBuffers(const unsigned int& new_capacity, bool init) {
    // Resize to match agent_count
    for (auto& buff : unbound_buffers) {
        if (!buff.host) {
            THROW exception::InvalidOperation("Not setup to resize before init");
        }
        // Alloc new buff
        const size_t var_size = buff.device->type_size * buff.device->elements;
        char *t = static_cast<char*>(malloc(new_capacity * var_size));
        // Copy data across
        const unsigned int copy_len = _size < unbound_host_buffer_capacity ? _size : unbound_host_buffer_capacity;
        memcpy(t, buff.host, copy_len * var_size);
        // Free old
        free(buff.host);
        // Replace old ptr
        buff.host = t;
        if (init) {
            for (unsigned int i = unbound_host_buffer_capacity; i < new_capacity; ++i) {
                // We have unknown agents, default init them
                memcpy(buff.host + i * var_size, buff.device->default_value, var_size);
            }
        }
    }
    unbound_host_buffer_capacity = new_capacity;
    // unbound_host_buffer_size = agent_count;  // This would only make sense for init, but consisent behaviour is better
    unbound_buffers_has_changed = true;  // Probably not required, but if they are resized, high chance theyre going to change
}

void DeviceAgentVector_impl::_insert(size_type pos, size_type count) {
    if (!count)
        return;
    // Init ID for all the inserted agents
    {
        auto d = _data->find(ID_VARIABLE_NAME);
        if (d != _data->end()) {
            _require(ID_VARIABLE_NAME);
            id_t *h_ptr = static_cast<id_t*>(d->second->getDataPtr());
            for (unsigned int i = pos; i < pos + count; ++i) {
                // Always assign ID, as AgentVector should reset these to unset, but this saves us checking
                // if (h_ptr[i] == ID_NOT_SET) {
                    h_ptr[i] = cuda_agent.nextID();
                // }
            }
            _changedAfter(ID_VARIABLE_NAME, pos);
        } else {
            THROW exception::InvalidOperation("Internal agent ID variable was not found, "
                "in DeviceAgentVector_impl._insert().");
        }
    }
    // No unbound buffers, return
    if (unbound_buffers.empty())
        return;
    // Unbound buffers first use, init
    // This updates unbound_host_buffer_size to match known_device_buffer_size
    if (!unbound_host_buffer_capacity)
        initUnboundBuffers();
    // Resizes unbound buffers if necessary
    const size_type new_size = known_device_buffer_size + count;
    if (new_size > unbound_host_buffer_capacity) {
        resizeUnboundBuffers(_capacity, false);
        // Init new agents that won't be init by the replacement below
        for (auto& buff : unbound_buffers) {
            const size_t variable_size = buff.device->type_size * buff.device->elements;
            for (unsigned int i = new_size; i < _capacity; ++i) {
                memcpy(buff.host + i * variable_size, buff.device->default_value, variable_size);
            }
        }
    }
    if (unbound_host_buffer_invalid) {
        // Redownload unbound buffers from device
        reinitUnboundBuffers();
    }
    //  Move all items behind pos, then init all the newly inserted
    for (auto& buff : unbound_buffers) {
        const size_t variable_size = buff.device->type_size * buff.device->elements;
        // Move all items after this index backwards count places
        for (unsigned int i = known_device_buffer_size - 1; i >= pos; --i) {
            // Copy items individually, incase the src and destination overlap
            memcpy(buff.host + (i + count) * variable_size, buff.host + i * variable_size, variable_size);
        }
        // Default init the inserted variables
        for (unsigned int i = pos; i < pos + count; ++i) {
            memcpy(buff.host + i * variable_size, buff.device->default_value, variable_size);
        }
    }
    // Update size
    unbound_buffers_has_changed = true;
    unbound_host_buffer_size = new_size;
    known_device_buffer_size = _size;
    if (unbound_host_buffer_size != _size) {
        THROW exception::InvalidOperation("Unbound buffers have gone out of sync, in DeviceAgentVector::_insert().\n");
    }
    // Update change detail for all variables
    for (const auto& v : agent->variables) {
        // Does it exist in change map
        auto change = change_detail.find(v.first);
        if (change == change_detail.end()) {
            change_detail.emplace(v.first, std::pair<size_type, size_type>{pos, _size});
        } else {
            // Inclusive min bound
            change->second.first = change->second.first > pos ? pos : change->second.first;
            // Exclusive max bound
            change->second.second = _size;
        }
    }
}
void DeviceAgentVector_impl::_erase(size_type pos, size_type count) {
    // No unbound buffers, return
    if (unbound_buffers.empty() || !count)
        return;
    // Unbound buffers first use, init
    if (!unbound_host_buffer_capacity)
        initUnboundBuffers();
    if (unbound_host_buffer_invalid) {
        // Redownload unbound buffers from device
        reinitUnboundBuffers();
    }
    const size_type new_size = known_device_buffer_size - count;
    const size_type copy_start = pos + count;
    for (auto& buff : unbound_buffers) {
        const size_t variable_size = buff.device->type_size * buff.device->elements;
        // Move all items after this index forwards count places
        for (unsigned int i = copy_start; i < unbound_host_buffer_size; ++i) {
            // Copy items individually, incase the src and destination overlap
            memcpy(buff.host + (i - count) * variable_size, buff.host + i * variable_size, variable_size);
        }
        // Default init the empty variables at the end
        for (unsigned int i = new_size; i < known_device_buffer_size; ++i) {
            memcpy(buff.host + i * variable_size, buff.device->default_value, variable_size);
        }
    }
    // Update size
    unbound_buffers_has_changed = true;
    unbound_host_buffer_size = new_size;
    known_device_buffer_size = _size;
    if (unbound_host_buffer_size != _size) {
        THROW exception::InvalidOperation("Unbound buffers have gone out of sync, in DeviceAgentVector::_erase().\n");
    }
    // Update change detail for all variables
    for (const auto &v : agent->variables) {
        // Does it exist in change map
        auto change = change_detail.find(v.first);
        if (change == change_detail.end()) {
            change_detail.emplace(v.first, std::pair<size_type, size_type>{pos, _size});
        } else {
            // Inclusive min bound
            change->second.first = change->second.first > pos ? pos : change->second.first;
            // Exclusive max bound
            change->second.second = _size;
        }
    }
}


void DeviceAgentVector_impl::_changed(const std::string& variable_name, size_type pos) {
    // Check the variable exists
    auto var = agent->variables.find(variable_name);
    if (var == agent->variables.end()) {
        THROW exception::InvalidAgentVar("Variable %s was not found, "
            "in DeviceAgentVector::_changed()\n",
            variable_name.c_str());
    }
    // Does it exist in change map
    auto change = change_detail.find(variable_name);
    if (change == change_detail.end()) {
        change_detail.emplace(variable_name, std::pair<size_type, size_type>{pos, pos + 1});
    } else {
        // Inclusive min bound
        change->second.first = change->second.first > pos ? pos : change->second.first;
        // Exclusive max bound
        change->second.second = change->second.second <= pos ? pos + 1 : change->second.second;
    }
}
void DeviceAgentVector_impl::_changedAfter(const std::string& variable_name, size_type pos) {
    // Check the variable exists
    auto var = agent->variables.find(variable_name);
    if (var == agent->variables.end()) {
        THROW exception::InvalidAgentVar("Variable %s was not found, "
            "in DeviceAgentVector::_changed()\n",
            variable_name.c_str());
    }
    // Does it exist in change map
    auto change = change_detail.find(variable_name);
    if (change == change_detail.end()) {
        change_detail.emplace(variable_name, std::pair<size_type, size_type>{pos, _size});
    } else {
        // Inclusive min bound
        change->second.first = change->second.first > pos ? pos : change->second.first;
        // Exclusive max bound
        change->second.second = _size;
    }
}
void DeviceAgentVector_impl::_require(const std::string& variable_name) const {
    if (invalid_variables.find(variable_name) !=invalid_variables.end()) {
        const auto& v = agent->variables.at(variable_name);
        // Copy back variable data into array
        void* host_dest = _data->at(variable_name)->getDataPtr();
        const void* device_src = cuda_agent.getStateVariablePtr(cuda_agent_state, variable_name);
        gpuErrchk(hipMemcpyAsync(host_dest, device_src, _size * v.type_size * v.elements, hipMemcpyDeviceToHost, stream));
        if (_capacity > _size) {
            // Default-init remaining buffer space
            const auto it = _data->find(variable_name);
            const size_t variable_size = v.type_size * v.elements;
            char* t_data = static_cast<char*>(it->second->getDataPtr());
            for (unsigned int i = _size; i < _capacity; ++i) {
                memcpy(t_data + i * variable_size, v.default_value, variable_size);
            }
        }
        // The invalid variable is now current
        invalid_variables.erase(variable_name);
        gpuErrchk(hipStreamSynchronize(stream));
    }
}
void DeviceAgentVector_impl::_requireAll() const {
    for (const auto& vn : invalid_variables) {
        const auto &v = agent->variables.at(vn);
        // Copy back variable data into array
        void* host_dest = _data->at(vn)->getDataPtr();
        const void* device_src = cuda_agent.getStateVariablePtr(cuda_agent_state, vn);
        gpuErrchk(hipMemcpyAsync(host_dest, device_src, _size * v.type_size * v.elements, hipMemcpyDeviceToHost, stream));
    }
    // Perform the cuda ops in a separate loop to host inits, gives a slight bit of time to eat latency
    for (const auto& vn : invalid_variables) {
        if (_capacity > _size) {
            const auto& v = agent->variables.at(vn);
            // Default-init remaining buffer space
            const auto it = _data->find(vn);
            const size_t variable_size = v.type_size * v.elements;
            char* t_data = static_cast<char*>(it->second->getDataPtr());
            for (unsigned int i = _size; i < _capacity; ++i) {
                memcpy(t_data + i * variable_size, v.default_value, variable_size);
            }
        }
    }
    // All invalid variables are now current
    invalid_variables.clear();
    gpuErrchk(hipStreamSynchronize(stream));
}
void DeviceAgentVector_impl::_requireLength() const {
    /**
     * This method is a nightmare, as it needs to be const, so can't call non-const untility methods
     * Copy the implementations was bad, so I just decided to abuse const cast instead
     */
    if (newAgentData.empty())
        return;
    if (_size + newAgentData.size() > _capacity) {
        // BEGIN: Re implementation of AgentVector::resize(size_type, bool)
        // Can't call it here, as would have huge knock-on effects to which methods can/can't be const
        const_cast<DeviceAgentVector_impl*>(this)->internal_resize(_size + static_cast<size_type>(newAgentData.size()), false);
        // END: Re implementation of AgentVector::resize(size_type, bool)
    }
    _requireAll();
    // Check if host new agent has any agents
    for (auto &newAgent : newAgentData) {
        // Manually insert them to device agent vector
        for (auto &v : agentOffsets.vars) {
            char* dst = static_cast<char*>(_data->at(v.first)->getDataPtr()) + _size * v.second.len;
            const char * src = newAgent.data + v.second.offset;
            memcpy(dst, src, v.second.len);
        }
        // Increase size
        ++_size;
    }
    // This updates unbound buffers
    // BEGIN: Re implementation of DeviceAgentVector_t::_insert(size_type, size_type)
    // Can't call it here, as would have huge knock-on effects to which methods can/can't be const
    const_cast<DeviceAgentVector_impl*>(this)->_insert(_size - static_cast<size_type>(newAgentData.size()), static_cast<size_type>(newAgentData.size()));
    // END: Re implementation of DeviceAgentVector_t::_insert(size_type, size_type)
    newAgentData.clear();
}


}  // namespace flamegpu
