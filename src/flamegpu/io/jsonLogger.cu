#include "hip/hip_runtime.h"
#include "flamegpu/io/jsonLogger.h"

#include <rapidjson/writer.h>
#include <rapidjson/prettywriter.h>
#include <rapidjson/stringbuffer.h>
#include <iostream>
#include <fstream>
#include <string>

#include "flamegpu/sim/RunPlan.h"
#include "flamegpu/sim/LogFrame.h"

jsonLogger::jsonLogger(const std::string &outPath, bool _prettyPrint, bool _truncateFile)
    : out_path(outPath)
    , prettyPrint(_prettyPrint)
    , truncateFile(_truncateFile) { }

void jsonLogger::log(const RunLog &log, const RunPlan &plan, bool logSteps, bool logExit) const {
  logCommon(log, &plan, false, logSteps, logExit);
}
void jsonLogger::log(const RunLog &log, bool logConfig, bool logSteps, bool logExit) const {
  logCommon(log, nullptr, logConfig, logSteps, logExit);
}

template<typename T>
void jsonLogger::writeAny(T &writer, const Any &value, const unsigned int &elements) const {
    // Output value
    if (elements > 1) {
        writer.StartArray();
    }
    // Loop through elements, to construct array
    for (unsigned int el = 0; el < elements; ++el) {
        if (value.type == std::type_index(typeid(float))) {
            writer.Double(static_cast<const float*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(double))) {
            writer.Double(static_cast<const double*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(int64_t))) {
            writer.Int64(static_cast<const int64_t*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(uint64_t))) {
            writer.Uint64(static_cast<const uint64_t*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(int32_t))) {
            writer.Int(static_cast<const int32_t*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(uint32_t))) {
            writer.Uint(static_cast<const uint32_t*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(int16_t))) {
            writer.Int(static_cast<const int16_t*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(uint16_t))) {
            writer.Uint(static_cast<const uint16_t*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(int8_t))) {
            writer.Int(static_cast<int32_t>(static_cast<const int8_t*>(value.ptr)[el]));  // Char outputs weird if being used as an integer
        } else if (value.type == std::type_index(typeid(uint8_t))) {
            writer.Uint(static_cast<uint32_t>(static_cast<const uint8_t*>(value.ptr)[el]));  // Char outputs weird if being used as an integer
        } else if (value.type == std::type_index(typeid(char))) {
            writer.Int(static_cast<int32_t>(static_cast<const char*>(value.ptr)[el]));  // Char outputs weird if being used as an integer
        } else {
            THROW RapidJSONError("Attempting to export value of unsupported type '%s', "
                "in jsonLogger::writeAny()\n", value.type.name());
        }
    }
    if (elements > 1) {
        writer.EndArray();
    }
}
template<typename T>
void jsonLogger::writeLogFrame(T &writer, const LogFrame &frame) const {
    writer.StartObject();
    {
        // Add static items
        writer.Key("step_index");
        writer.Uint(frame.getStepCount());
        if (frame.getEnvironment().size()) {
            // Add dynamic environment values
            writer.Key("environment");
            writer.StartObject();
            {
                for (const auto &prop : frame.getEnvironment()) {
                    writer.Key(prop.first.c_str());
                    // Log value
                    writeAny(writer, prop.second, prop.second.elements);
                }
            }
            writer.EndObject();
        }

        if (frame.getAgents().size()) {
            // Add dynamic agent values
            writer.Key("agents");
            writer.StartObject();
            {
                // This assumes that sort order places all agents of same name, different state consecutively
                std::string current_agent;
                for (const auto &agent : frame.getAgents()) {
                    // Start/end new agent
                    if (current_agent != agent.first.first) {
                        if (!current_agent.empty())
                            writer.EndObject();
                        current_agent = agent.first.first;
                        writer.Key(current_agent.c_str());
                        writer.StartObject();
                    }
                    // Start new state
                    writer.Key(agent.first.second.c_str());
                    writer.StartObject();
                    {
                        // Log agent count if provided
                        if (agent.second.second != UINT_MAX) {
                            writer.Key("count");
                            writer.Uint(agent.second.second);
                        }
                        if (agent.second.first.size()) {
                            writer.Key("variables");
                            writer.StartObject();
                            // This assumes that sort order places all variables of same name, different reduction consecutively
                            std::string current_variable;
                            // Log each reduction
                            for (auto &var : agent.second.first) {
                                // Start/end new variable
                                if (current_variable != var.first.name) {
                                    if (!current_variable.empty())
                                        writer.EndObject();
                                    current_variable = var.first.name;
                                    writer.Key(current_variable.c_str());
                                    writer.StartObject();
                                }
                                // Build name key for the variable
                                writer.Key(LoggingConfig::toString(var.first.reduction));
                                // Log value
                                writeAny(writer, var.second, 1);
                            }
                            if (!current_variable.empty())
                                writer.EndObject();
                            writer.EndObject();
                        }
                    }
                    writer.EndObject();
                }
                if (!current_agent.empty())
                    writer.EndObject();
            }
            writer.EndObject();
        }
    }
    writer.EndObject();
}

template<typename T>
void jsonLogger::logConfig(T &writer, const RunLog &log) const {
    writer.Key("config");
    writer.StartObject();
    {
        writer.Key("random_seed");
        writer.Uint(log.getRandomSeed());
    }
    writer.EndObject();
}
template<typename T>
void jsonLogger::logConfig(T &writer, const RunPlan &plan) const {
    writer.Key("config");
    writer.StartObject();
    {
        // Add static items
        writer.Key("random_seed");
        writer.Uint(plan.getRandomSimulationSeed());
        writer.Key("steps");
        writer.Uint(plan.getSteps());
        // Add dynamic environment overrides
        writer.Key("environment");
        writer.StartObject();
        {
            for (const auto &prop : plan.property_overrides) {
                const EnvironmentDescription::PropData &env_prop = plan.environment->at(prop.first);
                writer.Key(prop.first.c_str());
                writeAny(writer, prop.second, env_prop.data.elements);
            }
        }
        writer.EndObject();
    }
    writer.EndObject();
}
template<typename T>
void jsonLogger::logSteps(T &writer, const RunLog &log) const {
    writer.Key("steps");
    writer.StartArray();
    {
        for (const auto &step : log.getStepLog()) {
            writeLogFrame(writer, step);
        }
    }
    writer.EndArray();
}
template<typename T>
void jsonLogger::logExit(T &writer, const RunLog &log) const {
    writer.Key("exit");
    writeLogFrame(writer, log.getExitLog());
}

template<typename T>
void jsonLogger::logCommon(T &writer, const RunLog &log, const RunPlan *plan, bool doLogConfig, bool doLogSteps, bool doLogExit) const {
    // Begin json output object
    writer->StartObject();
    {
        // Log config
        if (plan) {
            logConfig(*writer, *plan);
        } else if (doLogConfig) {
            logConfig(*writer, log);
        }

        // Log step log
        if (doLogSteps) {
            logSteps(*writer, log);
        }

        // Log exit log
        if (doLogExit) {
            logExit(*writer, log);
        }
    }
    // End Json file
    writer->EndObject();
}
void jsonLogger::logCommon(const RunLog &log, const RunPlan *plan, bool doLogConfig, bool doLogSteps, bool doLogExit) const {
    // Init writer
    rapidjson::StringBuffer s;
    if (prettyPrint) {
        // rapidjson::Writer doesn't have virtual methods, so can't pass rapidjson::PrettyWriter around as ptr to rapidjson::writer
        rapidjson::PrettyWriter<rapidjson::StringBuffer>* writer = new rapidjson::PrettyWriter<rapidjson::StringBuffer>(s);
        writer->SetIndent('\t', 1);
        logCommon(writer, log, plan, doLogConfig, doLogSteps, doLogExit);
        delete writer;
    } else {
        rapidjson::Writer<rapidjson::StringBuffer> *writer = new rapidjson::Writer<rapidjson::StringBuffer>(s);
        logCommon(writer, log, plan, doLogConfig, doLogSteps, doLogExit);
        delete writer;
    }
    // Perform output
    std::ofstream out(out_path, truncateFile ? std::ofstream::trunc : std::ofstream::app);
    if (!out.is_open()) {
        THROW RapidJSONError("Unable to open file '%s' for writing\n", out_path.c_str());
    }

    out << s.GetString();
    out << "\n";
    out.close();
}
