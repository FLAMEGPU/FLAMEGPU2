#include "hip/hip_runtime.h"
#include "flamegpu/io/XMLStateWriter.h"

#include <numeric>
#include <sstream>
#include "tinyxml2/tinyxml2.h"              // downloaded from https:// github.com/leethomason/tinyxml2, the list of xml parsers : http:// lars.ruoff.free.fr/xmlcpp/
#include "flamegpu/exception/FLAMEGPUException.h"
#include "flamegpu/model/AgentDescription.h"
#include "flamegpu/simulation/CUDASimulation.h"
#include "flamegpu/simulation/AgentVector.h"
#include "flamegpu/simulation/detail/EnvironmentManager.cuh"

namespace flamegpu {
namespace io {

#ifndef XMLCheckResult
 /**
  * Macro function for converting a tinyxml2 return code to an exception
  * @param a_eResult The tinyxml2 return code
  */
#define XMLCheckResult(a_eResult) if (a_eResult != tinyxml2::XML_SUCCESS) { exception::FLAMEGPUException::setLocation(__FILE__, __LINE__);\
    switch (a_eResult) { \
    case tinyxml2::XML_ERROR_FILE_NOT_FOUND : \
    case tinyxml2::XML_ERROR_FILE_COULD_NOT_BE_OPENED : \
        THROW exception::InvalidInputFile("TinyXML error: File could not be opened.\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_FILE_READ_ERROR : \
        THROW exception::InvalidInputFile("TinyXML error: File could not be read.\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_PARSING_ELEMENT : \
    case tinyxml2::XML_ERROR_PARSING_ATTRIBUTE : \
    case tinyxml2::XML_ERROR_PARSING_TEXT : \
    case tinyxml2::XML_ERROR_PARSING_CDATA : \
    case tinyxml2::XML_ERROR_PARSING_COMMENT : \
    case tinyxml2::XML_ERROR_PARSING_DECLARATION : \
    case tinyxml2::XML_ERROR_PARSING_UNKNOWN : \
    case tinyxml2::XML_ERROR_PARSING : \
        THROW exception::TinyXMLError("TinyXML error: Error parsing file.\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_EMPTY_DOCUMENT : \
        THROW exception::TinyXMLError("TinyXML error: XML_ERROR_EMPTY_DOCUMENT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_MISMATCHED_ELEMENT : \
        THROW exception::TinyXMLError("TinyXML error: XML_ERROR_MISMATCHED_ELEMENT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_CAN_NOT_CONVERT_TEXT : \
        THROW exception::TinyXMLError("TinyXML error: XML_CAN_NOT_CONVERT_TEXT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_NO_TEXT_NODE : \
        THROW exception::TinyXMLError("TinyXML error: XML_NO_TEXT_NODE\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ELEMENT_DEPTH_EXCEEDED : \
        THROW exception::TinyXMLError("TinyXML error: XML_ELEMENT_DEPTH_EXCEEDED\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_COUNT : \
        THROW exception::TinyXMLError("TinyXML error: XML_ERROR_COUNT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_NO_ATTRIBUTE: \
        THROW exception::TinyXMLError("TinyXML error: XML_NO_ATTRIBUTE\n Error code: %d", a_eResult); \
    case tinyxml2::XML_WRONG_ATTRIBUTE_TYPE : \
        THROW exception::TinyXMLError("TinyXML error: XML_WRONG_ATTRIBUTE_TYPE\n Error code: %d", a_eResult); \
    default: \
        THROW exception::TinyXMLError("TinyXML error: Unrecognised error code\n Error code: %d", a_eResult); \
    } \
}
#endif

XMLStateWriter::XMLStateWriter()
    : StateWriter() {}
void XMLStateWriter::beginWrite(const std::string &output_file, bool pretty_print) {
    this->outputPath = output_file;
    this->prettyPrint = pretty_print;
    if (doc || pRoot) {
        THROW exception::UnknownInternalError("Writing already active, in XMLStateWriter::beginWrite()");
    }
    doc = std::make_unique<tinyxml2::XMLDocument>();
    // Begin Json file
    pRoot = doc->NewElement("states");
    doc->InsertFirstChild(pRoot);

    // Clear flags
    this->config_written = false;
    this->stats_written = false;
    this->environment_written = false;
    this->macro_environment_written = false;
    this->agents_written = false;
}
void XMLStateWriter::endWrite() {
    if (!doc || !pRoot) {
        THROW exception::UnknownInternalError("Writing not active, in XMLStateWriter::endWrite()");
    }

    // End Json file
    tinyxml2::XMLError errorId = doc->SaveFile(outputPath.c_str(), !prettyPrint);
    XMLCheckResult(errorId);

    pRoot = nullptr;
    doc.reset();
}


void XMLStateWriter::writeConfig(const Simulation *sim_instance) {
    if (!doc || !pRoot) {
        THROW exception::UnknownInternalError("beginWrite() must be called before writeConfig(), in XMLStateWriter::writeConfig()");
    } else if (config_written) {
        THROW exception::UnknownInternalError("writeConfig() can only be called once per write session, in XMLStateWriter::writeConfig()");
    }

    // Output config elements
    tinyxml2::XMLElement *pElement = doc->NewElement("config");
    {
        // Sim config
        tinyxml2::XMLElement *pSimCfg = doc->NewElement("simulation");
        {
            const auto &sim_cfg = sim_instance->getSimulationConfig();
            tinyxml2::XMLElement *pListElement = nullptr;
            // Input file
            pListElement = doc->NewElement("input_file");
            pListElement->SetText(sim_cfg.input_file.c_str());
            pSimCfg->InsertEndChild(pListElement);
            // Step log file
            pListElement = doc->NewElement("step_log_file");
            pListElement->SetText(sim_cfg.step_log_file.c_str());
            pSimCfg->InsertEndChild(pListElement);
            // Exit log file
            pListElement = doc->NewElement("exit_log_file");
            pListElement->SetText(sim_cfg.exit_log_file.c_str());
            pSimCfg->InsertEndChild(pListElement);
            // Common log file
            pListElement = doc->NewElement("common_log_file");
            pListElement->SetText(sim_cfg.common_log_file.c_str());
            pSimCfg->InsertEndChild(pListElement);
            // Truncate log files
            pListElement = doc->NewElement("truncate_log_files");
            pListElement->SetText(sim_cfg.truncate_log_files);
            pSimCfg->InsertEndChild(pListElement);
            // Random seed
            pListElement = doc->NewElement("random_seed");
            pListElement->SetText(sim_cfg.random_seed);
            pSimCfg->InsertEndChild(pListElement);
            // Steps
            pListElement = doc->NewElement("steps");
            pListElement->SetText(sim_cfg.steps);
            pSimCfg->InsertEndChild(pListElement);
            // Verbose output
            pListElement = doc->NewElement("verbosity");
            pListElement->SetText(static_cast<unsigned int>(sim_cfg.verbosity));
            pSimCfg->InsertEndChild(pListElement);
            // Timing Output
            pListElement = doc->NewElement("timing");
            pListElement->SetText(sim_cfg.timing);
            pSimCfg->InsertEndChild(pListElement);
#ifdef FLAMEGPU_VISUALISATION
            // Console Mode
            pListElement = doc->NewElement("console_mode");
            pListElement->SetText(sim_cfg.console_mode);
            pSimCfg->InsertEndChild(pListElement);
#endif
        }
        pElement->InsertEndChild(pSimCfg);

        // Cuda config
        if (auto *cudamodel_instance = dynamic_cast<const CUDASimulation*>(sim_instance)) {
            tinyxml2::XMLElement *pCUDACfg = doc->NewElement("cuda");
            {
                const auto &cuda_cfg = cudamodel_instance->getCUDAConfig();
                tinyxml2::XMLElement *pListElement = nullptr;
                // Device ID
                pListElement = doc->NewElement("device_id");
                pListElement->SetText(cuda_cfg.device_id);
                pCUDACfg->InsertEndChild(pListElement);
                // inLayerConcurrency
                pListElement = doc->NewElement("inLayerConcurrency");
                pListElement->SetText(cuda_cfg.inLayerConcurrency);
                pCUDACfg->InsertEndChild(pListElement);
            }
            pElement->InsertEndChild(pCUDACfg);
        }
    }
    pRoot->InsertEndChild(pElement);

    config_written = true;
}
void XMLStateWriter::writeStats(unsigned int iterations) {
    if (!doc || !pRoot) {
        THROW exception::UnknownInternalError("beginWrite() must be called before writeStats(), in XMLStateWriter::writeStats()");
    } else if (stats_written) {
        THROW exception::UnknownInternalError("writeStats() can only be called once per write session, in XMLStateWriter::writeStats()");
    }

    // Redundant for FLAMEGPU1 backwards compatibility
    tinyxml2::XMLElement *pElement = doc->NewElement("itno");
    pElement->SetText(iterations);
    pRoot->InsertEndChild(pElement);

    // Output stats elements
    pElement = doc->NewElement("stats");
    {
        tinyxml2::XMLElement *pListElement = nullptr;
        // Input file
        pListElement = doc->NewElement("step_count");
        pListElement->SetText(iterations);
        pElement->InsertEndChild(pListElement);
    }
    pRoot->InsertEndChild(pElement);

    stats_written = true;
}

void XMLStateWriter::writeEnvironment(const std::shared_ptr<const detail::EnvironmentManager>& env_manager) {
    if (!doc || !pRoot) {
        THROW exception::UnknownInternalError("beginWrite() must be called before writeEnvironment(), in XMLStateWriter::writeEnvironment()");
    } else if (environment_written) {
        THROW exception::UnknownInternalError("writeEnvironment() can only be called once per write session, in XMLStateWriter::writeEnvironment()");
    }

    tinyxml2::XMLElement *pElement = doc->NewElement("environment");
    if (env_manager) {
        const char* env_buffer = reinterpret_cast<const char*>(env_manager->getHostBuffer());
        // for each environment property
        for (auto &a : env_manager->getPropertiesMap()) {
            tinyxml2::XMLElement* pListElement = doc->NewElement(a.first.c_str());
            pListElement->SetAttribute("type", a.second.type.name());
            // Output properties
            std::stringstream ss;
            // Loop through elements, to construct csv string
            for (unsigned int el = 0; el < a.second.elements; ++el) {
                if (a.second.type == std::type_index(typeid(float))) {
                    ss << *reinterpret_cast<const float*>(env_buffer + a.second.offset + (el * sizeof(float)));
                } else if (a.second.type == std::type_index(typeid(double))) {
                    ss << *reinterpret_cast<const double*>(env_buffer + a.second.offset + (el * sizeof(double)));
                } else if (a.second.type == std::type_index(typeid(int64_t))) {
                    ss << *reinterpret_cast<const int64_t*>(env_buffer + a.second.offset + (el * sizeof(int64_t)));
                } else if (a.second.type == std::type_index(typeid(uint64_t))) {
                    ss << *reinterpret_cast<const uint64_t*>(env_buffer + a.second.offset + (el * sizeof(uint64_t)));
                } else if (a.second.type == std::type_index(typeid(int32_t))) {
                    ss << *reinterpret_cast<const int32_t*>(env_buffer + a.second.offset + (el * sizeof(int32_t)));
                } else if (a.second.type == std::type_index(typeid(uint32_t))) {
                    ss << *reinterpret_cast<const uint32_t*>(env_buffer + a.second.offset + (el * sizeof(uint32_t)));
                } else if (a.second.type == std::type_index(typeid(int16_t))) {
                    ss << *reinterpret_cast<const int16_t*>(env_buffer + a.second.offset + (el * sizeof(int16_t)));
                } else if (a.second.type == std::type_index(typeid(uint16_t))) {
                    ss << *reinterpret_cast<const uint16_t*>(env_buffer + a.second.offset + (el * sizeof(uint16_t)));
                } else if (a.second.type == std::type_index(typeid(int8_t))) {
                    ss << static_cast<int32_t>(*reinterpret_cast<const int8_t*>(env_buffer + a.second.offset + (el * sizeof(int8_t))));  // Char outputs weird if being used as an integer
                } else if (a.second.type == std::type_index(typeid(uint8_t))) {
                    ss << static_cast<uint32_t>(*reinterpret_cast<const uint8_t*>(env_buffer + a.second.offset + (el * sizeof(uint8_t))));  // Char outputs weird if being used as an integer
                } else {
                    THROW exception::TinyXMLError("Model contains environment property '%s' of unsupported type '%s', "
                        "in XMLStateWriter::writeEnvironment()\n", a.first.c_str(), a.second.type.name());
                }
                if (el + 1 != a.second.elements)
                    ss << ",";
            }
            pListElement->SetText(ss.str().c_str());
            pElement->InsertEndChild(pListElement);
        }
    }
    pRoot->InsertEndChild(pElement);

    environment_written = true;
}
void XMLStateWriter::writeMacroEnvironment(const std::shared_ptr<const detail::CUDAMacroEnvironment>& macro_env, std::initializer_list<std::string> filter) {
    if (!doc || !pRoot) {
        THROW exception::UnknownInternalError("beginWrite() must be called before writeMacroEnvironment(), in XMLStateWriter::writeMacroEnvironment()");
    } else if (macro_environment_written) {
        THROW exception::UnknownInternalError("writeMacroEnvironment() can only be called once per write session, in XMLStateWriter::writeMacroEnvironment()");
    }

    tinyxml2::XMLElement *pElement = doc->NewElement("macro_environment");
    if (macro_env) {
        const std::map<std::string, detail::CUDAMacroEnvironment::MacroEnvProp>& m_properties = macro_env->getPropertiesMap();
        for (const auto &_filter : filter) {
            if (m_properties.find(_filter) == m_properties.end()) {
                THROW exception::InvalidEnvProperty("Macro property '%s' specified in filter does not exist, in XMLStateWriter::writeMacroEnvironment()", _filter.c_str());
            }
        }
        std::set<std::string> filter_set = filter;
        // Calculate largest buffer in map
        size_t max_len = 0;
        for (const auto& [_, prop] : m_properties) {
            max_len = std::max(max_len, std::accumulate(prop.elements.begin(), prop.elements.end(), 1, std::multiplies<unsigned int>()) * prop.type_size);
        }
        if (max_len) {
            // Allocate temp buffer
            char* const t_buffer = static_cast<char*>(malloc(max_len));
            // Write out each array (all are written out as 1D arrays for simplicity given variable dimensions)
            for (const auto& [name, prop] : m_properties) {
                if (!filter_set.empty() && filter_set.find(name) == filter_set.end())
                    continue;
                // Copy data
                const size_t element_ct = std::accumulate(prop.elements.begin(), prop.elements.end(), 1, std::multiplies<unsigned int>());
                gpuErrchk(hipMemcpy(t_buffer, prop.d_ptr, element_ct * prop.type_size, hipMemcpyDeviceToHost));

                tinyxml2::XMLElement* pListElement = doc->NewElement(name.c_str());
                pListElement->SetAttribute("type", prop.type.name());

                // Loop through dimensions to construct dimensions string
                // Clip trailing 1 dimensions
                std::stringstream ss;
                size_t sum = 1;
                for (size_t j = 0; j < prop.elements.size(); ++j) {
                    ss << prop.elements[j];
                    sum *= prop.elements[j];
                    if (sum == element_ct)
                        break;
                    ss << ",";
                }
                pListElement->SetAttribute("dimensions", ss.str().c_str());
                ss.str("");
                ss.clear();

                // Output elements
                // Loop through elements, to construct csv string
                for (size_t i = 0; i < element_ct; ++i) {
                    if (prop.type == std::type_index(typeid(float))) {
                        ss << *reinterpret_cast<const float*>(t_buffer + i * sizeof(float));
                    } else if (prop.type == std::type_index(typeid(double))) {
                        ss << *reinterpret_cast<const double*>(t_buffer + i * sizeof(double));
                    } else if (prop.type == std::type_index(typeid(int64_t))) {
                        ss << *reinterpret_cast<const int64_t*>(t_buffer + i * sizeof(int64_t));
                    } else if (prop.type == std::type_index(typeid(uint64_t))) {
                        ss << *reinterpret_cast<const uint64_t*>(t_buffer + i * sizeof(uint64_t));
                    } else if (prop.type == std::type_index(typeid(int32_t))) {
                        ss << *reinterpret_cast<const int32_t*>(t_buffer + i * sizeof(int32_t));
                    } else if (prop.type == std::type_index(typeid(uint32_t))) {
                        ss << *reinterpret_cast<const uint32_t*>(t_buffer + i * sizeof(uint32_t));
                    } else if (prop.type == std::type_index(typeid(int16_t))) {
                        ss << *reinterpret_cast<const int16_t*>(t_buffer + i * sizeof(int16_t));
                    } else if (prop.type == std::type_index(typeid(uint16_t))) {
                        ss << *reinterpret_cast<const uint16_t*>(t_buffer + i * sizeof(uint16_t));
                    } else if (prop.type == std::type_index(typeid(int8_t))) {
                        ss << static_cast<int32_t>(*reinterpret_cast<const int8_t*>(t_buffer + i * sizeof(int8_t)));  // Char outputs weird if being used as an integer
                    } else if (prop.type == std::type_index(typeid(uint8_t))) {
                        ss << static_cast<uint32_t>(*reinterpret_cast<const uint8_t*>(t_buffer + i * sizeof(uint8_t)));  // Char outputs weird if being used as an integer
                    } else {
                        THROW exception::TinyXMLError("Model contains macro environment property '%s' of unsupported type '%s', "
                            "in XMLStateWriter::writeMacroEnvironment()\n", name.c_str(), prop.type.name());
                    }
                    if (i + 1 != element_ct)
                        ss << ",";
                }
                pListElement->SetText(ss.str().c_str());
                pElement->InsertEndChild(pListElement);
            }
            // Release temp buffer
            free(t_buffer);
        }
    }
    pRoot->InsertEndChild(pElement);

    macro_environment_written = true;
}
void XMLStateWriter::writeAgents(const util::StringPairUnorderedMap<std::shared_ptr<const AgentVector>>& agents_map) {
    if (!doc || !pRoot) {
        THROW exception::UnknownInternalError("beginWrite() must be called before writeAgents(), in XMLStateWriter::writeAgents()");
    } else if (agents_written) {
        THROW exception::UnknownInternalError("writeAgents() can only be called once per write session, in XMLStateWriter::writeAgents()");
    }

    // for each agent types
    for (const auto &[key, vec] : agents_map) {
        // For each agent state
        const std::string &agent_name = key.first;
        const std::string &state_name = key.second;

        unsigned int populationSize = vec->size();
        if (populationSize) {
            for (unsigned int i = 0; i < populationSize; ++i) {
                // Create vars block
                tinyxml2::XMLElement * pXagentElement = doc->NewElement("xagent");

                const AgentVector::CAgent instance = vec->at(i);
                const VariableMap &mm = vec->getVariableMetaData();

                // Add agent's name to block
                tinyxml2::XMLElement * pXagentNameElement = doc->NewElement("name");
                pXagentNameElement->SetText(agent_name.c_str());
                pXagentElement->InsertEndChild(pXagentNameElement);
                // Add state's name to block
                tinyxml2::XMLElement * pStateNameElement = doc->NewElement("state");
                pStateNameElement->SetText(state_name.c_str());
                pXagentElement->InsertEndChild(pStateNameElement);

                // for each variable
                for (auto iter_mm = mm.begin(); iter_mm != mm.end(); ++iter_mm) {
                    const std::string variable_name = iter_mm->first;

                    tinyxml2::XMLElement* pListElement = doc->NewElement(variable_name.c_str());
                    if (i == 0)
                        pListElement->SetAttribute("type", iter_mm->second.type.name());

                    // Output properties
                    std::stringstream ss;
                    // Loop through elements, to construct csv string
                    for (unsigned int el = 0; el < iter_mm->second.elements; ++el) {
                        if (iter_mm->second.type == std::type_index(typeid(float))) {
                            ss << instance.getVariable<float>(variable_name, el);
                        } else if (iter_mm->second.type == std::type_index(typeid(double))) {
                            ss << instance.getVariable<double>(variable_name, el);
                        } else if (iter_mm->second.type == std::type_index(typeid(int64_t))) {
                            ss << instance.getVariable<int64_t>(variable_name, el);
                        } else if (iter_mm->second.type == std::type_index(typeid(uint64_t))) {
                            ss << instance.getVariable<uint64_t>(variable_name, el);
                        } else if (iter_mm->second.type == std::type_index(typeid(int32_t))) {
                            ss << instance.getVariable<int32_t>(variable_name, el);
                        } else if (iter_mm->second.type == std::type_index(typeid(uint32_t))) {
                            ss << instance.getVariable<uint32_t>(variable_name, el);
                        } else if (iter_mm->second.type == std::type_index(typeid(int16_t))) {
                            ss << instance.getVariable<int16_t>(variable_name, el);
                        } else if (iter_mm->second.type == std::type_index(typeid(uint16_t))) {
                            ss << instance.getVariable<uint16_t>(variable_name, el);
                        } else if (iter_mm->second.type == std::type_index(typeid(int8_t))) {
                            ss << static_cast<int32_t>(instance.getVariable<int8_t>(variable_name, el));  // Char outputs weird if being used as an integer
                        } else if (iter_mm->second.type == std::type_index(typeid(uint8_t))) {
                            ss << static_cast<uint32_t>(instance.getVariable<uint8_t>(variable_name, el));  // Char outputs weird if being used as an integer
                        } else {
                            THROW exception::TinyXMLError("Agent '%s' contains variable '%s' of unsupported type '%s', "
                                "in XMLStateWriter::writeFullModelState()\n", agent_name.c_str(), variable_name.c_str(), iter_mm->second.type.name());
                        }
                        if (el + 1 != iter_mm->second.elements)
                            ss << ",";
                    }
                    pListElement->SetText(ss.str().c_str());
                    pXagentElement->InsertEndChild(pListElement);
                }
                // Insert xagent block into doc root
                pRoot->InsertEndChild(pXagentElement);
            }
        }  // if state has agents
    }

    agents_written = true;
}
}  // namespace io
}  // namespace flamegpu
