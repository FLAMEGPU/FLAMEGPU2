#include "hip/hip_runtime.h"
#include "flamegpu/io/JSONLogger.h"

#include <iostream>
#include <fstream>
#include <string>

#include "flamegpu/simulation/RunPlan.h"
#include "flamegpu/simulation/LogFrame.h"

namespace flamegpu {
namespace io {

JSONLogger::JSONLogger(const std::string &outPath, bool _prettyPrint, bool _truncateFile)
    : out_path(outPath)
    , prettyPrint(_prettyPrint)
    , truncateFile(_truncateFile) { }

void JSONLogger::log(const RunLog &log, const RunPlan &plan, bool logSteps, bool logExit, bool logStepTime, bool logExitTime) const {
  logCommon(log, &plan, false, logSteps, logExit, logStepTime, logExitTime);
}
void JSONLogger::log(const RunLog &log, bool logConfig, bool logSteps, bool logExit, bool logStepTime, bool logExitTime) const {
  logCommon(log, nullptr, logConfig, logSteps, logExit, logStepTime, logExitTime);
}

void JSONLogger::writeAny(nlohmann::ordered_json& j, const detail::Any &value, const unsigned int elements) const {
    if (elements == 1) {
        if (value.type == std::type_index(typeid(float))) {
            j = static_cast<const float*>(value.ptr)[0];
        } else if (value.type == std::type_index(typeid(double))) {
            j = static_cast<const double*>(value.ptr)[0];
        } else if (value.type == std::type_index(typeid(int64_t))) {
            j = static_cast<const int64_t*>(value.ptr)[0];
        } else if (value.type == std::type_index(typeid(uint64_t))) {
            j = static_cast<const uint64_t*>(value.ptr)[0];
        } else if (value.type == std::type_index(typeid(int32_t))) {
            j = static_cast<const int32_t*>(value.ptr)[0];
        } else if (value.type == std::type_index(typeid(uint32_t))) {
            j = static_cast<const uint32_t*>(value.ptr)[0];
        } else if (value.type == std::type_index(typeid(int16_t))) {
            j = static_cast<const int16_t*>(value.ptr)[0];
        } else if (value.type == std::type_index(typeid(uint16_t))) {
            j = static_cast<const uint16_t*>(value.ptr)[0];
        } else if (value.type == std::type_index(typeid(int8_t))) {
            j = static_cast<int32_t>(static_cast<const int8_t*>(value.ptr)[0]);  // Char outputs weird if being used as an integer
        } else if (value.type == std::type_index(typeid(uint8_t))) {
            j = static_cast<uint32_t>(static_cast<const uint8_t*>(value.ptr)[0]);  // Char outputs weird if being used as an integer
        } else if (value.type == std::type_index(typeid(char))) {
            j = static_cast<int32_t>(static_cast<const char*>(value.ptr)[0]);  // Char outputs weird if being used as an integer
        } else {
            THROW exception::JSONError("Attempting to export value of unsupported type '%s', "
                "in JSONLogger::writeAny()\n", value.type.name());
        }
        return;
    }
    // Loop through elements, to construct array
    for (unsigned int el = 0; el < elements; ++el) {
        if (value.type == std::type_index(typeid(float))) {
            j.emplace_back(static_cast<const float*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(double))) {
            j.emplace_back(static_cast<const double*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(int64_t))) {
            j.emplace_back(static_cast<const int64_t*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(uint64_t))) {
            j.emplace_back(static_cast<const uint64_t*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(int32_t))) {
            j.emplace_back(static_cast<const int32_t*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(uint32_t))) {
            j.emplace_back(static_cast<const uint32_t*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(int16_t))) {
            j.emplace_back(static_cast<const int16_t*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(uint16_t))) {
            j.emplace_back(static_cast<const uint16_t*>(value.ptr)[el]);
        } else if (value.type == std::type_index(typeid(int8_t))) {
            j.emplace_back(static_cast<int32_t>(static_cast<const int8_t*>(value.ptr)[el]));  // Char outputs weird if being used as an integer
        } else if (value.type == std::type_index(typeid(uint8_t))) {
            j.emplace_back(static_cast<uint32_t>(static_cast<const uint8_t*>(value.ptr)[el]));  // Char outputs weird if being used as an integer
        } else if (value.type == std::type_index(typeid(char))) {
            j.emplace_back(static_cast<int32_t>(static_cast<const char*>(value.ptr)[el]));  // Char outputs weird if being used as an integer
        } else {
            THROW exception::JSONError("Attempting to export value of unsupported type '%s', "
                "in JSONLogger::writeAny()\n", value.type.name());
        }
    }
}
nlohmann::ordered_json JSONLogger::writeLogFrame(const StepLogFrame& frame, bool logTime) const {
    nlohmann::ordered_json j;
    if (logTime) {
        j["step_time"] = frame.getStepTime();
    }
    writeCommonLogFrame(j, frame);
    return j;
}
nlohmann::ordered_json JSONLogger::writeLogFrame(const ExitLogFrame& frame, bool logTime) const {
    nlohmann::ordered_json j;
    if (logTime) {
        j["rtc_time"] = frame.getRTCTime();
        j["init_time"] = frame.getInitTime();
        j["exit_time"] = frame.getExitTime();
        j["total_time"] = frame.getTotalTime();
    }
    writeCommonLogFrame(j, frame);
    return j;
}
void JSONLogger::writeCommonLogFrame(nlohmann::ordered_json& j, const LogFrame &frame) const {
    // Add static items
    j["step_index"] = frame.getStepCount();
    if (frame.getEnvironment().size()) {
        // Add dynamic environment values
        nlohmann::ordered_json j_env;
        for (const auto &prop : frame.getEnvironment()) {
            j_env[prop.first] = {};
            // Log value
            writeAny(j_env[prop.first], prop.second, prop.second.elements);
        }
        j["env"] = j_env;
    }

    if (frame.getAgents().size()) {
        // Add dynamic agent values
        nlohmann::ordered_json j_agents = {};
        for (const auto &agent : frame.getAgents()) {
            nlohmann::ordered_json j_t_agent;
            // Log agent count if provided
            if (agent.second.second != UINT_MAX) {
                j_t_agent["count"][agent.second.second];
            }
            if (agent.second.first.size()) {
                j_t_agent["variables"];
                // This assumes that sort order places all variables of same name, different reduction consecutively
                std::string current_variable;
                // Log each reduction
                for (auto &var : agent.second.first) {
                    // Log value
                    writeAny(j_t_agent["variables"][var.first.name][LoggingConfig::toString(var.first.reduction)], var.second, 1);
                }
            }
            j_agents[agent.first.first][agent.first.second].push_back(j_t_agent);
        }
        j["agents"] = j_agents;
    }
}

nlohmann::ordered_json JSONLogger::logConfig(const RunLog &log) const {
    nlohmann::ordered_json j;
    j["random_seed"] = log.getRandomSeed();
    return j;
}
nlohmann::ordered_json JSONLogger::logConfig(const RunPlan &plan) const {
    nlohmann::ordered_json j;
    // Add static items
    j["random_seed"] = plan.getRandomSimulationSeed();
    j["steps"] = plan.getSteps();
    // Add dynamic environment overrides
    nlohmann::ordered_json dyn_j;
    for (const auto& prop : plan.property_overrides) {
        const EnvironmentData::PropData& env_prop = plan.environment->at(prop.first);
        dyn_j[prop.first] = {};
        writeAny(dyn_j[prop.first], prop.second, env_prop.data.elements);
    }
    j["environment"] = dyn_j;
    return j;
}
nlohmann::ordered_json JSONLogger::logPerformanceSpecs(const RunLog& log) const {
    nlohmann::ordered_json j;
    // Add static items
    j["device_name"] = log.getPerformanceSpecs().device_name;
    j["device_cc_major"] = log.getPerformanceSpecs().device_cc_major;
    j["device_cc_minor"] = log.getPerformanceSpecs().device_cc_minor;
    j["cuda_version"] = log.getPerformanceSpecs().cuda_version;
    j["seatbelts"] = log.getPerformanceSpecs().seatbelts;
    j["flamegpu_version"] = log.getPerformanceSpecs().flamegpu_version;
    return j;
}
void JSONLogger::logSteps(nlohmann::ordered_json& j, const RunLog &log, bool logTime) const {
    j["steps"] = {};
    for (const auto &step : log.getStepLog()) {
        j["steps"].push_back(writeLogFrame(step, logTime));
    }
}
void JSONLogger::logExit(nlohmann::ordered_json& j, const RunLog &log, bool logTime) const {
    j["exit"] = writeLogFrame(log.getExitLog(), logTime);
}

void JSONLogger::logCommon(nlohmann::ordered_json &j, const RunLog &log, const RunPlan *plan, bool doLogConfig, bool doLogSteps, bool doLogExit, bool doLogStepTime, bool doLogExitTime) const {
    // Log config
    if (plan) {
        j["config"] = logConfig(*plan);
    } else if (doLogConfig) {
        j["config"] = logConfig(log);
    }
    if (doLogStepTime || doLogExitTime) {
        j["performance_specs"] = logPerformanceSpecs(log);
    }

    // Log step log
    if (doLogSteps) {
        logSteps(j, log, doLogStepTime);
    }

    // Log exit log
    if (doLogExit) {
        logExit(j, log, doLogExitTime);
    }
}
void JSONLogger::logCommon(const RunLog &log, const RunPlan *plan, bool doLogConfig, bool doLogSteps, bool doLogExit, bool doLogStepTime, bool doLogExitTime) const {
    // Init writer
    nlohmann::ordered_json j;
    logCommon(j, log, plan, doLogConfig, doLogSteps, doLogExit, doLogStepTime, doLogExitTime);
    // Perform output
    std::ofstream out(out_path, std::ios::binary | (truncateFile ? std::ofstream::trunc : std::ofstream::app));
    if (!out.is_open()) {
        THROW exception::JSONError("Unable to open file '%s' for writing\n", out_path.c_str());
    }
    if (prettyPrint) {
        out << std::setw(4);
    }
    out << j;
    out.close();
}

}  // namespace io
}  // namespace flamegpu
