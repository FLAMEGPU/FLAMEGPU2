#include "hip/hip_runtime.h"
#include "flamegpu/io/XMLLogger.h"

#include <sstream>

#include "tinyxml2/tinyxml2.h"              // downloaded from https:// github.com/leethomason/tinyxml2, the list of xml parsers : http:// lars.ruoff.free.fr/xmlcpp/

#include "flamegpu/simulation/RunPlan.h"
#include "flamegpu/simulation/LogFrame.h"

namespace flamegpu {
namespace io {

#ifndef XMLCheckResult
#define XMLCheckResult(a_eResult) if (a_eResult != tinyxml2::XML_SUCCESS) { exception::FLAMEGPUException::setLocation(__FILE__, __LINE__);\
    switch (a_eResult) { \
    case tinyxml2::XML_ERROR_FILE_NOT_FOUND : \
    case tinyxml2::XML_ERROR_FILE_COULD_NOT_BE_OPENED : \
        throw exception::InvalidInputFile("TinyXML error: File could not be opened.\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_FILE_READ_ERROR : \
        throw exception::InvalidInputFile("TinyXML error: File could not be read.\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_PARSING_ELEMENT : \
    case tinyxml2::XML_ERROR_PARSING_ATTRIBUTE : \
    case tinyxml2::XML_ERROR_PARSING_TEXT : \
    case tinyxml2::XML_ERROR_PARSING_CDATA : \
    case tinyxml2::XML_ERROR_PARSING_COMMENT : \
    case tinyxml2::XML_ERROR_PARSING_DECLARATION : \
    case tinyxml2::XML_ERROR_PARSING_UNKNOWN : \
    case tinyxml2::XML_ERROR_PARSING : \
        throw exception::TinyXMLError("TinyXML error: Error parsing file.\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_EMPTY_DOCUMENT : \
        throw exception::TinyXMLError("TinyXML error: XML_ERROR_EMPTY_DOCUMENT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_MISMATCHED_ELEMENT : \
        throw exception::TinyXMLError("TinyXML error: XML_ERROR_MISMATCHED_ELEMENT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_CAN_NOT_CONVERT_TEXT : \
        throw exception::TinyXMLError("TinyXML error: XML_CAN_NOT_CONVERT_TEXT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_NO_TEXT_NODE : \
        throw exception::TinyXMLError("TinyXML error: XML_NO_TEXT_NODE\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ELEMENT_DEPTH_EXCEEDED : \
        throw exception::TinyXMLError("TinyXML error: XML_ELEMENT_DEPTH_EXCEEDED\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_COUNT : \
        throw exception::TinyXMLError("TinyXML error: XML_ERROR_COUNT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_NO_ATTRIBUTE: \
        throw exception::TinyXMLError("TinyXML error: XML_NO_ATTRIBUTE\n Error code: %d", a_eResult); \
    case tinyxml2::XML_WRONG_ATTRIBUTE_TYPE : \
        throw exception::TinyXMLError("TinyXML error: XML_WRONG_ATTRIBUTE_TYPE\n Error code: %d", a_eResult); \
    default: \
        throw exception::TinyXMLError("TinyXML error: Unrecognised error code\n Error code: %d", a_eResult); \
    } \
}
#endif

XMLLogger::XMLLogger(const std::string &outPath, bool _prettyPrint, bool _truncateFile)
    : out_path(outPath)
    , prettyPrint(_prettyPrint)
    , truncateFile(_truncateFile) { }

void XMLLogger::log(const RunLog &log, const RunPlan &plan, bool logSteps, bool logExit, bool logStepTime, bool logExitTime) const {
  logCommon(log, &plan, false, logSteps, logExit, logStepTime, logExitTime);
}
void XMLLogger::log(const RunLog &log, bool logConfig, bool logSteps, bool logExit, bool logStepTime, bool logExitTime) const {
  logCommon(log, nullptr, logConfig, logSteps, logExit, logStepTime, logExitTime);
}

void XMLLogger::logCommon(const RunLog &log, const RunPlan *plan, bool doLogConfig, bool doLogSteps, bool doLogExit, bool doLogStepTime, bool doLogExitTime) const {
    tinyxml2::XMLDocument doc;

    tinyxml2::XMLNode * pRoot = doc.NewElement("log");
    doc.InsertFirstChild(pRoot);

    // Log config
    if (plan) {
        pRoot->InsertEndChild(logConfig(doc, *plan));
    } else if (doLogConfig) {
        pRoot->InsertEndChild(logConfig(doc, log));
    }

    // Log performance specs
    if (doLogStepTime || doLogExitTime) {
        pRoot->InsertEndChild(logPerformanceSpecs(doc, log));
    }

    // Log step log
    if (doLogSteps) {
        pRoot->InsertEndChild(logSteps(doc, log, doLogStepTime));
    }

    // Log exit log
    if (doLogExit) {
        pRoot->InsertEndChild(logExit(doc, log, doLogExitTime));
    }
    // export
    FILE *fptr = fopen(out_path.c_str(), truncateFile ? "w" : "a");
    if (fptr == nullptr) {
        THROW exception::TinyXMLError("Unable to open file '%s' for writing\n", out_path.c_str());
    }
    XMLCheckResult(doc.SaveFile(fptr, !prettyPrint));
    fwrite("\n", sizeof(char), 1, fptr);
    fclose(fptr);
}

tinyxml2::XMLNode *XMLLogger::logConfig(tinyxml2::XMLDocument &doc, const RunLog &log) const {
    tinyxml2::XMLElement *pConfigElement = doc.NewElement("config");
    {
        tinyxml2::XMLElement *pListElement;
        pListElement = doc.NewElement("random_seed");
        pListElement->SetText(log.getRandomSeed());
        pConfigElement->InsertEndChild(pListElement);
    }
    return pConfigElement;
}
tinyxml2::XMLNode *XMLLogger::logConfig(tinyxml2::XMLDocument &doc, const RunPlan &plan) const {
    tinyxml2::XMLElement *pConfigElement = doc.NewElement("config");
    {
        tinyxml2::XMLElement *pListElement;
        // Add static items
        pListElement = doc.NewElement("random_seed");
        pListElement->SetText(plan.getRandomSimulationSeed());
        pConfigElement->InsertEndChild(pListElement);
        pListElement = doc.NewElement("steps");
        pListElement->SetText(plan.getSteps());
        pConfigElement->InsertEndChild(pListElement);
        // Add dynamic environment overrides
        tinyxml2::XMLElement *pEnvElement = doc.NewElement("environment");
        {
            for (const auto &prop : plan.property_overrides) {
                const EnvironmentData::PropData &env_prop = plan.environment->at(prop.first);
                pListElement = doc.NewElement(prop.first.c_str());
                writeAny(pListElement, prop.second, env_prop.data.elements);
                pEnvElement->InsertEndChild(pListElement);
            }
        }
        pConfigElement->InsertEndChild(pEnvElement);
    }
    return pConfigElement;
}
tinyxml2::XMLNode* XMLLogger::logPerformanceSpecs(tinyxml2::XMLDocument& doc, const RunLog& log) const {
    tinyxml2::XMLElement* pConfigElement = doc.NewElement("performance_specs");
    {
        tinyxml2::XMLElement* pListElement;
        // Add static items
        pListElement = doc.NewElement("device_name");
        pListElement->SetText(log.getPerformanceSpecs().device_name.c_str());
        pConfigElement->InsertEndChild(pListElement);
        pListElement = doc.NewElement("device_cc_major");
        pListElement->SetText(log.getPerformanceSpecs().device_cc_major);
        pConfigElement->InsertEndChild(pListElement);
        pListElement = doc.NewElement("device_cc_minor");
        pListElement->SetText(log.getPerformanceSpecs().device_cc_minor);
        pConfigElement->InsertEndChild(pListElement);
        pListElement = doc.NewElement("cuda_version");
        pListElement->SetText(log.getPerformanceSpecs().cuda_version);
        pConfigElement->InsertEndChild(pListElement);
        pListElement = doc.NewElement("seatbelts");
        pListElement->SetText(log.getPerformanceSpecs().seatbelts);
        pConfigElement->InsertEndChild(pListElement);
        pListElement = doc.NewElement("flamegpu_version");
        pListElement->SetText(log.getPerformanceSpecs().flamegpu_version.c_str());
        pConfigElement->InsertEndChild(pListElement);
    }
    return pConfigElement;
}
tinyxml2::XMLNode *XMLLogger::logSteps(tinyxml2::XMLDocument &doc, const RunLog &log, bool logTime) const {
    tinyxml2::XMLElement *pStepsElement = doc.NewElement("steps");
    {
        for (const auto &step : log.getStepLog()) {
            pStepsElement->InsertEndChild(writeLogFrame(doc, step, logTime));
        }
    }
    return pStepsElement;
}
tinyxml2::XMLNode *XMLLogger::logExit(tinyxml2::XMLDocument &doc, const RunLog &log, bool logTime) const {
    return writeLogFrame(doc, log.getExitLog(), logTime);
}
tinyxml2::XMLNode* XMLLogger::writeLogFrame(tinyxml2::XMLDocument& doc, const StepLogFrame& frame, bool logTime) const {
    tinyxml2::XMLElement* pFrameElement = doc.NewElement("step");
    {
        if (logTime) {
            tinyxml2::XMLElement* pListElement;
            pListElement = doc.NewElement("step_time");
            pListElement->SetText(frame.getStepTime());
            pFrameElement->InsertEndChild(pListElement);
        }
        writeCommonLogFrame(doc, pFrameElement, frame);
    }
    return pFrameElement;
}
tinyxml2::XMLNode* XMLLogger::writeLogFrame(tinyxml2::XMLDocument & doc, const ExitLogFrame & frame, bool logTime) const {
    tinyxml2::XMLElement* pFrameElement = doc.NewElement("exit");
    {
        if (logTime) {
            tinyxml2::XMLElement* pListElement;
            pListElement = doc.NewElement("rtc_time");
            pListElement->SetText(frame.getRTCTime());
            pFrameElement->InsertEndChild(pListElement);
            pListElement = doc.NewElement("init_time");
            pListElement->SetText(frame.getInitTime());
            pFrameElement->InsertEndChild(pListElement);
            pListElement = doc.NewElement("exit_time");
            pListElement->SetText(frame.getExitTime());
            pFrameElement->InsertEndChild(pListElement);
            pListElement = doc.NewElement("total_time");
            pListElement->SetText(frame.getTotalTime());
            pFrameElement->InsertEndChild(pListElement);
        }
        writeCommonLogFrame(doc, pFrameElement, frame);
    }
    return pFrameElement;
}
void XMLLogger::writeCommonLogFrame(tinyxml2::XMLDocument &doc, tinyxml2::XMLElement* pFrameElement, const LogFrame & frame) const {
    tinyxml2::XMLElement *pListElement;
    // Add static items
    pListElement = doc.NewElement("step_index");
    pListElement->SetText(frame.getStepCount());
    pFrameElement->InsertEndChild(pListElement);
    // Add dynamic environment values
    if (frame.getEnvironment().size()) {
        tinyxml2::XMLElement *pEnvElement = doc.NewElement("environment");
        {
            for (const auto &prop : frame.getEnvironment()) {
                pListElement = doc.NewElement(prop.first.c_str());
                writeAny(pListElement, prop.second, prop.second.elements);
                pEnvElement->InsertEndChild(pListElement);
            }
        }
        pFrameElement->InsertEndChild(pEnvElement);
    }

    if (frame.getAgents().size()) {
        // Add dynamic agent values
        tinyxml2::XMLElement *pAgentsElement = doc.NewElement("agents");
        {
            // This assumes that sort order places all agents of same name, different state consecutively
            std::string current_agent;
            tinyxml2::XMLElement *pAgentsItemElement = nullptr;
            for (const auto &agent : frame.getAgents()) {
                // Start/end new agent
                if (current_agent != agent.first.first) {
                    if (!current_agent.empty())
                        pAgentsElement->InsertEndChild(pAgentsItemElement);
                    current_agent = agent.first.first;
                    pAgentsItemElement = doc.NewElement(current_agent.c_str());
                }
                // Start new state
                tinyxml2::XMLElement *pStateElement = doc.NewElement(agent.first.second.c_str());
                {
                    // Log agent count if provided
                    if (agent.second.second != UINT_MAX) {
                        tinyxml2::XMLElement *pCountElement = doc.NewElement("count");
                        pCountElement->SetText(agent.second.second);
                        pStateElement->InsertEndChild(pCountElement);
                    }
                    if (agent.second.first.size()) {
                        tinyxml2::XMLElement *pVariablesBlock = doc.NewElement("variables");
                        // This assumes that sort order places all variables of same name, different reduction consecutively
                        std::string current_variable;
                        tinyxml2::XMLElement *pVariableElement = nullptr;
                        // Log each reduction
                        for (auto &var : agent.second.first) {
                            // Start/end new variable
                            if (current_variable != var.first.name) {
                                if (!current_variable.empty())
                                    pVariablesBlock->InsertEndChild(pVariableElement);
                                current_variable = var.first.name;
                                pVariableElement = doc.NewElement(current_variable.c_str());
                            }
                            // Build name key for the variable & log value
                            tinyxml2::XMLElement *pValueElement = doc.NewElement(LoggingConfig::toString(var.first.reduction));
                            writeAny(pValueElement, var.second, 1);
                            pVariableElement->InsertEndChild(pValueElement);
                        }
                        if (!current_variable.empty())
                            pVariablesBlock->InsertEndChild(pVariableElement);
                        pStateElement->InsertEndChild(pVariablesBlock);
                    }
                }
                pAgentsItemElement->InsertEndChild(pStateElement);
            }
            if (!current_agent.empty())
                pAgentsElement->InsertEndChild(pAgentsItemElement);
        }
        pFrameElement->InsertEndChild(pAgentsElement);
    }
}

void XMLLogger::writeAny(tinyxml2::XMLElement *pElement, const detail::Any &value, const unsigned int elements) const {
    std::stringstream ss;
    // Loop through elements, to construct csv string
    for (unsigned int el = 0; el < elements; ++el) {
        if (value.type == std::type_index(typeid(float))) {
            ss << static_cast<const float*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(double))) {
             ss << static_cast<const double*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(int64_t))) {
            ss << static_cast<const int64_t*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(uint64_t))) {
             ss << static_cast<const uint64_t*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(int32_t))) {
            ss << static_cast<const int32_t*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(uint32_t))) {
             ss << static_cast<const uint32_t*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(int16_t))) {
             ss << static_cast<const int16_t*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(uint16_t))) {
             ss << static_cast<const uint16_t*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(int8_t))) {
            ss << static_cast<int32_t>(static_cast<const int8_t*>(value.ptr)[el]);  // Char outputs weird if being used as an integer
        } else if (value.type == std::type_index(typeid(uint8_t))) {
            ss << static_cast<uint32_t>(static_cast<const uint8_t*>(value.ptr)[el]);  // Char outputs weird if being used as an integer
        } else if (value.type == std::type_index(typeid(char))) {
            ss << static_cast<int32_t>(static_cast<const char*>(value.ptr)[el]);  // Char outputs weird if being used as an integer
        } else {
            THROW exception::TinyXMLError("Attempting to export value of unsupported type '%s', "
                "in XMLLogger::writeAny()\n", value.type.name());
       }
        if (el + 1 != elements)
            ss << ",";
    }
    pElement->SetText(ss.str().c_str());
}

}  // namespace io
}  // namespace flamegpu
