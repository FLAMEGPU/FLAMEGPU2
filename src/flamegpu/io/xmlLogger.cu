#include "hip/hip_runtime.h"
#include "flamegpu/io/xmlLogger.h"

#include <sstream>

#include "tinyxml2/tinyxml2.h"              // downloaded from https:// github.com/leethomason/tinyxml2, the list of xml parsers : http:// lars.ruoff.free.fr/xmlcpp/

#include "flamegpu/sim/RunPlan.h"
#include "flamegpu/sim/LogFrame.h"

#ifndef XMLCheckResult
#define XMLCheckResult(a_eResult) if (a_eResult != tinyxml2::XML_SUCCESS) { FGPUException::setLocation(__FILE__, __LINE__);\
    switch (a_eResult) { \
    case tinyxml2::XML_ERROR_FILE_NOT_FOUND : \
    case tinyxml2::XML_ERROR_FILE_COULD_NOT_BE_OPENED : \
        throw InvalidInputFile("TinyXML error: File could not be opened.\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_FILE_READ_ERROR : \
        throw InvalidInputFile("TinyXML error: File could not be read.\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_PARSING_ELEMENT : \
    case tinyxml2::XML_ERROR_PARSING_ATTRIBUTE : \
    case tinyxml2::XML_ERROR_PARSING_TEXT : \
    case tinyxml2::XML_ERROR_PARSING_CDATA : \
    case tinyxml2::XML_ERROR_PARSING_COMMENT : \
    case tinyxml2::XML_ERROR_PARSING_DECLARATION : \
    case tinyxml2::XML_ERROR_PARSING_UNKNOWN : \
    case tinyxml2::XML_ERROR_PARSING : \
        throw TinyXMLError("TinyXML error: Error parsing file.\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_EMPTY_DOCUMENT : \
        throw TinyXMLError("TinyXML error: XML_ERROR_EMPTY_DOCUMENT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_MISMATCHED_ELEMENT : \
        throw TinyXMLError("TinyXML error: XML_ERROR_MISMATCHED_ELEMENT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_CAN_NOT_CONVERT_TEXT : \
        throw TinyXMLError("TinyXML error: XML_CAN_NOT_CONVERT_TEXT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_NO_TEXT_NODE : \
        throw TinyXMLError("TinyXML error: XML_NO_TEXT_NODE\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ELEMENT_DEPTH_EXCEEDED : \
        throw TinyXMLError("TinyXML error: XML_ELEMENT_DEPTH_EXCEEDED\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_COUNT : \
        throw TinyXMLError("TinyXML error: XML_ERROR_COUNT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_NO_ATTRIBUTE: \
        throw TinyXMLError("TinyXML error: XML_NO_ATTRIBUTE\n Error code: %d", a_eResult); \
    case tinyxml2::XML_WRONG_ATTRIBUTE_TYPE : \
        throw TinyXMLError("TinyXML error: XML_WRONG_ATTRIBUTE_TYPE\n Error code: %d", a_eResult); \
    default: \
        throw TinyXMLError("TinyXML error: Unrecognised error code\n Error code: %d", a_eResult); \
    } \
}
#endif

xmlLogger::xmlLogger(const std::string &outPath, bool _prettyPrint, bool _truncateFile)
    : out_path(outPath)
    , prettyPrint(_prettyPrint)
    , truncateFile(_truncateFile) { }

void xmlLogger::log(const RunLog &log, const RunPlan &plan, bool logSteps, bool logExit) const {
  logCommon(log, &plan, false, logSteps, logExit);
}
void xmlLogger::log(const RunLog &log, bool logConfig, bool logSteps, bool logExit) const {
  logCommon(log, nullptr, logConfig, logSteps, logExit);
}

void xmlLogger::logCommon(const RunLog &log, const RunPlan *plan, bool doLogConfig, bool doLogSteps, bool doLogExit) const {
    tinyxml2::XMLDocument doc;

    tinyxml2::XMLNode * pRoot = doc.NewElement("log");
    doc.InsertFirstChild(pRoot);

    // Log config
    if (plan) {
        pRoot->InsertEndChild(logConfig(doc, *plan));
    } else if (doLogConfig) {
        pRoot->InsertEndChild(logConfig(doc, log));
    }

    // Log step log
    if (doLogSteps) {
        pRoot->InsertEndChild(logSteps(doc, log));
    }

    // Log exit log
    if (doLogExit) {
        pRoot->InsertEndChild(logExit(doc, log));
    }
    // export
    FILE *fptr = fopen(out_path.c_str(), truncateFile ? "w" : "a");
    if (fptr == nullptr) {
        THROW TinyXMLError("Unable to open file '%s' for writing\n", out_path.c_str());
    }
    XMLCheckResult(doc.SaveFile(fptr, !prettyPrint));
    fwrite("\n", sizeof(char), 1, fptr);
    fclose(fptr);
}

tinyxml2::XMLNode *xmlLogger::logConfig(tinyxml2::XMLDocument &doc, const RunLog &log) const {
    tinyxml2::XMLElement *pConfigElement = doc.NewElement("config");
    {
        tinyxml2::XMLElement *pListElement;
        pListElement = doc.NewElement("random_seed");
        pListElement->SetText(log.getRandomSeed());
        pConfigElement->InsertEndChild(pListElement);
    }
    return pConfigElement;
}
tinyxml2::XMLNode *xmlLogger::logConfig(tinyxml2::XMLDocument &doc, const RunPlan &plan) const {
    tinyxml2::XMLElement *pConfigElement = doc.NewElement("config");
    {
        tinyxml2::XMLElement *pListElement;
        // Add static items
        pListElement = doc.NewElement("random_seed");
        pListElement->SetText(plan.getRandomSimulationSeed());
        pConfigElement->InsertEndChild(pListElement);
        pListElement = doc.NewElement("steps");
        pListElement->SetText(plan.getSteps());
        pConfigElement->InsertEndChild(pListElement);
        // Add dynamic environment overrides
        tinyxml2::XMLElement *pEnvElement = doc.NewElement("environment");
        {
            for (const auto &prop : plan.property_overrides) {
                const EnvironmentDescription::PropData &env_prop = plan.environment->at(prop.first);
                pListElement = doc.NewElement(prop.first.c_str());
                writeAny(pListElement, prop.second, env_prop.data.elements);
                pEnvElement->InsertEndChild(pListElement);
            }
        }
        pConfigElement->InsertEndChild(pEnvElement);
    }
    return pConfigElement;
}
tinyxml2::XMLNode *xmlLogger::logSteps(tinyxml2::XMLDocument &doc, const RunLog &log) const {
    tinyxml2::XMLElement *pStepsElement = doc.NewElement("steps");
    {
        for (const auto &step : log.getStepLog()) {
            pStepsElement->InsertEndChild(writeLogFrame(doc, step));
        }
    }
    return pStepsElement;
}
tinyxml2::XMLNode *xmlLogger::logExit(tinyxml2::XMLDocument &doc, const RunLog &log) const {
    tinyxml2::XMLElement *pExitElement = doc.NewElement("exit");
    pExitElement->InsertEndChild(writeLogFrame(doc, log.getExitLog()));
    return pExitElement;
}

tinyxml2::XMLNode *xmlLogger::writeLogFrame(tinyxml2::XMLDocument &doc, const LogFrame &frame) const {
    tinyxml2::XMLElement *pFrameElement = doc.NewElement("step");
    {
        tinyxml2::XMLElement *pListElement;
        // Add static items
        pListElement = doc.NewElement("step_index");
        pListElement->SetText(frame.getStepCount());
        pFrameElement->InsertEndChild(pListElement);
        // Add dynamic environment values
        if (frame.getEnvironment().size()) {
            tinyxml2::XMLElement *pEnvElement = doc.NewElement("environment");
            {
                for (const auto &prop : frame.getEnvironment()) {
                    pListElement = doc.NewElement(prop.first.c_str());
                    writeAny(pListElement, prop.second, prop.second.elements);
                    pEnvElement->InsertEndChild(pListElement);
                }
            }
            pFrameElement->InsertEndChild(pEnvElement);
        }

        if (frame.getAgents().size()) {
            // Add dynamic agent values
            tinyxml2::XMLElement *pAgentsElement = doc.NewElement("agents");
            {
                // This assumes that sort order places all agents of same name, different state consecutively
                std::string current_agent;
                tinyxml2::XMLElement *pAgentsItemElement = nullptr;
                for (const auto &agent : frame.getAgents()) {
                    // Start/end new agent
                    if (current_agent != agent.first.first) {
                        if (!current_agent.empty())
                            pAgentsElement->InsertEndChild(pAgentsItemElement);
                        current_agent = agent.first.first;
                        pAgentsItemElement = doc.NewElement(current_agent.c_str());
                    }
                    // Start new state
                    tinyxml2::XMLElement *pStateElement = doc.NewElement(agent.first.second.c_str());
                    {
                        // Log agent count if provided
                        if (agent.second.second != UINT_MAX) {
                            tinyxml2::XMLElement *pCountElement = doc.NewElement("count");
                            pCountElement->SetText(agent.second.second);
                            pStateElement->InsertEndChild(pCountElement);
                        }
                        if (agent.second.first.size()) {
                            tinyxml2::XMLElement *pVariablesBlock = doc.NewElement("variables");
                            // This assumes that sort order places all variables of same name, different reduction consecutively
                            std::string current_variable;
                            tinyxml2::XMLElement *pVariableElement = nullptr;
                            // Log each reduction
                            for (auto &var : agent.second.first) {
                                // Start/end new variable
                                if (current_variable != var.first.name) {
                                    if (!current_variable.empty())
                                        pVariablesBlock->InsertEndChild(pVariableElement);
                                    current_variable = var.first.name;
                                    pVariableElement = doc.NewElement(current_variable.c_str());
                                }
                                // Build name key for the variable & log value
                                tinyxml2::XMLElement *pValueElement = doc.NewElement(LoggingConfig::toString(var.first.reduction));
                                writeAny(pValueElement, var.second, 1);
                                pVariableElement->InsertEndChild(pValueElement);
                            }
                            if (!current_variable.empty())
                                pVariablesBlock->InsertEndChild(pVariableElement);
                            pStateElement->InsertEndChild(pVariablesBlock);
                        }
                    }
                    pAgentsItemElement->InsertEndChild(pStateElement);
                }
                if (!current_agent.empty())
                    pAgentsElement->InsertEndChild(pAgentsItemElement);
            }
            pFrameElement->InsertEndChild(pAgentsElement);
        }
    }
    return pFrameElement;
}

void xmlLogger::writeAny(tinyxml2::XMLElement *pElement, const Any &value, const unsigned int &elements) const {
    std::stringstream ss;
    // Loop through elements, to construct csv string
    for (unsigned int el = 0; el < elements; ++el) {
        if (value.type == std::type_index(typeid(float))) {
            ss << static_cast<const float*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(double))) {
             ss << static_cast<const double*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(int64_t))) {
            ss << static_cast<const int64_t*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(uint64_t))) {
             ss << static_cast<const uint64_t*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(int32_t))) {
            ss << static_cast<const int32_t*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(uint32_t))) {
             ss << static_cast<const uint32_t*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(int16_t))) {
             ss << static_cast<const int16_t*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(uint16_t))) {
             ss << static_cast<const uint16_t*>(value.ptr)[el];
        } else if (value.type == std::type_index(typeid(int8_t))) {
            ss << static_cast<int32_t>(static_cast<const int8_t*>(value.ptr)[el]);  // Char outputs weird if being used as an integer
        } else if (value.type == std::type_index(typeid(uint8_t))) {
            ss << static_cast<uint32_t>(static_cast<const uint8_t*>(value.ptr)[el]);  // Char outputs weird if being used as an integer
        } else if (value.type == std::type_index(typeid(char))) {
            ss << static_cast<int32_t>(static_cast<const char*>(value.ptr)[el]);  // Char outputs weird if being used as an integer
        } else {
            THROW TinyXMLError("Attempting to export value of unsupported type '%s', "
                "in xmlLogger::writeAny()\n", value.type.name());
       }
        if (el + 1 != elements)
            ss << ",";
    }
    pElement->SetText(ss.str().c_str());
}
