#include "flamegpu/io/StateReader.h"

namespace flamegpu {
namespace io {

void StateReader::resetCache() {
    simulation_config.clear();
    cuda_config.clear();
    env_init.clear();
    macro_env_init.clear();
    agents_map.clear();
}
void StateReader::getFullModelState(
    Simulation::Config &s_cfg,
    std::unordered_map<std::string, detail::Any> &environment_init,
    std::unordered_map<std::string, std::vector<char>> &macro_environment_init,
    util::StringPairUnorderedMap<std::shared_ptr<AgentVector>> &agents_init) {
    getSimulationConfig(s_cfg);
    getEnvironment(environment_init);
    getMacroEnvironment(macro_environment_init);
    getAgents(agents_init);
}

#define MAP_GET(out, map, name, typ) out.name = map.find(#name) == map.end()?out.name:std::any_cast<typ>(map.at(#name))

void StateReader::getSimulationConfig(Simulation::Config &cfg) {
    if (input_filepath.empty()) {
        THROW exception::InvalidOperation("Input file has not been parsed, in StateReader::getSimulationConfig()");
    }
    // if (!simulation_config) {
    //     THROW exception::InvalidInputFile("Input file %s did not contain an simulation config, in StateReader::getSimulationConfig()", input_filepath.c_str());
    // }
    // Set all the items manually
    MAP_GET(cfg, simulation_config, input_file, std::string);
    MAP_GET(cfg, simulation_config, step_log_file, std::string);
    MAP_GET(cfg, simulation_config, exit_log_file, std::string);
    MAP_GET(cfg, simulation_config, common_log_file, std::string);
    MAP_GET(cfg, simulation_config, truncate_log_files, bool);
    MAP_GET(cfg, simulation_config, random_seed, uint64_t);
    MAP_GET(cfg, simulation_config, steps, unsigned int);
    MAP_GET(cfg, simulation_config, verbosity, Verbosity);
    MAP_GET(cfg, simulation_config, timing, bool);
    MAP_GET(cfg, simulation_config, silence_unknown_args, bool);
    MAP_GET(cfg, simulation_config, telemetry, bool);
#ifdef FLAMEGPU_VISUALISATION
    MAP_GET(cfg, simulation_config, console_mode, bool);
#endif
}
void StateReader::getCUDAConfig(CUDASimulation::Config &cfg) {
    if (input_filepath.empty()) {
        THROW exception::InvalidOperation("Input file has not been parsed, in StateReader::getCUDAConfig()");
    }
    // if (!cuda_config) {
    //     THROW exception::InvalidInputFile("Input file %s did not contain an CUDA config, in StateReader::getCUDAConfig()", input_filepath.c_str());
    // }
    // Set all the items manually
    MAP_GET(cfg, cuda_config, device_id, int);
    MAP_GET(cfg, cuda_config, inLayerConcurrency, bool);
}
void StateReader::getEnvironment(std::unordered_map<std::string, detail::Any> &environment_init) {
    if (input_filepath.empty()) {
        THROW exception::InvalidOperation("Input file has not been parsed, in StateReader::getEnvironment()");
    }
    // if (env_init.empty()) {
    //     THROW exception::InvalidInputFile("Input file %s did not contain any environment properties, in StateReader::getEnvironment()", input_filepath.c_str());
    // }
    for (const auto& [key, val] : env_init) {
        environment_init.erase(key);
        environment_init.emplace(key, val);
    }
}
void StateReader::getMacroEnvironment(std::unordered_map<std::string, std::vector<char>> &macro_environment_init) {
    if (input_filepath.empty()) {
        THROW exception::InvalidOperation("Input file has not been parsed, in StateReader::getEnvironment()");
    }
    // if (macro_env_init.empty()) {
    //     THROW exception::InvalidInputFile("Input file %s did not contain any macro environment properties, in StateReader::getMacroEnvironment()", input_filepath.c_str());
    // }
    for (const auto& [key, val] : macro_env_init) {
        macro_environment_init.insert_or_assign(key, val);
    }
}
void StateReader::getAgents(util::StringPairUnorderedMap<std::shared_ptr<AgentVector>> &agents_init) {
    if (input_filepath.empty()) {
        THROW exception::InvalidOperation("Input file has not been parsed, in StateReader::getEnvironment()");
    }
    // if (agents_map.empty()) {
    //     THROW exception::InvalidInputFile("Input file %s did not contain any agents, in StateReader::getMacroEnvironment()", input_filepath.c_str());
    // }
    for (const auto& [key, val] : agents_map) {
        agents_init.insert_or_assign(key, val);
    }
}

}  // namespace io
}  // namespace flamegpu
