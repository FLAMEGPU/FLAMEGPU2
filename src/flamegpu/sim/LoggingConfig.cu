#include "flamegpu/sim/LoggingConfig.h"

#include "flamegpu/sim/AgentLoggingConfig.h"
#include "flamegpu/model/ModelDescription.h"
#include "flamegpu/model/ModelData.h"
#include "flamegpu/model/AgentData.h"

LoggingConfig::LoggingConfig(const ModelDescription &_model)
    :model(_model.model->clone()) { }
LoggingConfig::LoggingConfig(const ModelData &_model)
    :model(_model.clone()) { }
LoggingConfig::LoggingConfig(const LoggingConfig &other)
    : model(other.model->clone())
    , environment(other.environment)
    , agents(other.agents) { }
AgentLoggingConfig LoggingConfig::agent(const std::string &agent_name, const std::string &agent_state) {
    // Validate the agent state combination exists
    auto model_agent_it = model->agents.find(agent_name);
    if (model_agent_it == model->agents.end()) {
        THROW InvalidAgentName("Agent '%s' was not found in the model description, "
            "in LoggingConfig::agent()\n",
            agent_name.c_str());
    }
    if (model_agent_it->second->states.find(agent_state) == model_agent_it->second->states.end()) {
        THROW InvalidAgentState("State '%s' was not found within agent '%s' in the model description, "
            "in LoggingConfig::agent()\n",
            agent_state.c_str(), agent_name.c_str());
    }
    NameStatePair name = std::make_pair(agent_name, agent_state);
    auto agent_it = agents.find(name);
    if (agent_it== agents.end())
        agent_it = agents.emplace(name, std::make_pair(std::make_shared<std::set<NameReductionFn>>(), false)).first;
    return AgentLoggingConfig(model_agent_it->second, agent_it->second);
}

void LoggingConfig::logEnvironment(const std::string &property_name) {
    // Validate the environment property exists
    auto env_map = model->environment->getPropertiesMap();
    if (env_map.find(property_name) == env_map.end()) {
        THROW InvalidEnvProperty("Environment property '%s' was not found in the model description, "
            "in LoggingConfig::logEnvironment()\n",
            property_name.c_str());
    }
    // Log the property
    if (!environment.emplace(property_name).second) {
        THROW InvalidEnvProperty("Environment property '%s' has already been marked for logging, "
            "in LoggingConfig::logEnvironment()\n",
            property_name.c_str());
    }
}
StepLoggingConfig::StepLoggingConfig(const ModelDescription &model)
    : LoggingConfig(model)
    , frequency(1) { }
StepLoggingConfig::StepLoggingConfig(const ModelData &model)
    : LoggingConfig(model)
    , frequency(1) { }
StepLoggingConfig::StepLoggingConfig(const StepLoggingConfig &other)
    : LoggingConfig(other)
    , frequency(other.frequency) { }
StepLoggingConfig::StepLoggingConfig(const LoggingConfig &other)
    : LoggingConfig(other)
    , frequency(1) { }
void StepLoggingConfig::setFrequency(const unsigned int &steps) {
    frequency = steps;
}
