#include "hip/hip_runtime.h"
#include "flamegpu/detail/JitifyCache.h"

#include <hip/hiprtc.h>

#include <cassert>
#include <regex>
#include <array>
#include <filesystem>
#include <vector>
#include <memory>
#include <string>
#include <unordered_map>
#include <cstdio>

#include "jitify/jitify2.hpp"

#include "flamegpu/version.h"
#include "flamegpu/exception/FLAMEGPUException.h"
#include "flamegpu/detail/compute_capability.cuh"
#include "flamegpu/util/nvtx.h"


namespace flamegpu {
namespace detail {

namespace {
/**
 * Returns the tmp dir for storing cache files
 * Defined here to avoid filesystem includes being in header
 */
std::filesystem::path getTMP() {
    static std::filesystem::path result;
    if (result.empty()) {
        std::filesystem::path tmp =  std::getenv("FLAMEGPU_TMP_DIR") ? std::getenv("FLAMEGPU_TMP_DIR") : std::filesystem::temp_directory_path();
        // Create the $tmp/flamegpu/jitifycache(/debug) folder hierarchy
        if (!std::filesystem::exists(tmp) && !std::filesystem::create_directories(tmp)) {
            THROW exception::InvalidFilePath("Directory '%s' does not exist and cannot be created by JitifyCache.", tmp.generic_string().c_str());
        }
        if (!std::getenv("FLAMEGPU_TMP_DIR")) {
            tmp /= "flamegpu";
            if (!std::filesystem::exists(tmp)) {
                std::filesystem::create_directories(tmp);
            }
        }
        tmp /= "jitifycache";
        if (!std::filesystem::exists(tmp)) {
            std::filesystem::create_directories(tmp);
        }
#ifdef _DEBUG
        tmp /= "debug";
        if (!std::filesystem::exists(tmp)) {
            std::filesystem::create_directories(tmp);
        }
#endif
        result = tmp;
    }
    return result;
}
/**
 * Returns the user-defined include directories
 */
std::vector<std::filesystem::path> getIncludeDirs() {
    static std::vector<std::filesystem::path> rtn;
    if (rtn.empty()) {
        if (std::getenv("FLAMEGPU_RTC_INCLUDE_DIRS")) {
            const std::string s = std::getenv("FLAMEGPU_RTC_INCLUDE_DIRS");
            // Split the string by ; (windows), : (linux)
#if defined(_MSC_VER)
            std::string delimiter = ";";
#else
            std::string delimiter = ":";
#endif
            size_t start = 0, end = s.find(delimiter);
            std::string token;
            do {
                std::filesystem::path p = s.substr(start, end - start);
                if (!p.empty()) {
                    rtn.push_back(p);
                }
                start = end + delimiter.length();
            } while ((end = s.find(delimiter, start))!= std::string::npos);
        } else {
            rtn.push_back(std::filesystem::current_path());
        }
    }
    return rtn;
}

std::string loadFile(const std::filesystem::path &filepath) {
    std::ifstream ifs;
    ifs.open(filepath, std::ifstream::binary);
    if (!ifs)
        return "";
    // get length of file
    ifs.seekg(0, ifs.end);
    const std::streamoff length = ifs.tellg();
    ifs.seekg(0, ifs.beg);
    std::string rtn;
    rtn.resize(length);
    char *buffer = &rtn[0];
    ifs.read(buffer, length);
    ifs.close();
    return rtn;
}

/**
 * Find the cuda include directory.
 * Throws exceptions if it can not be found.
 * @return the path to the CUDA include directory.
 */
std::string getCUDAIncludeDir() {
    // Define an array of environment variables to check in order
    std::array<const std::string, 2> ENV_VARS { "CUDA_PATH", "CUDA_HOME" };
    std::string cuda_include_dir_str = "";
    for (const auto& env_var : ENV_VARS) {
        std::string env_value = std::getenv(env_var.c_str()) ? std::getenv(env_var.c_str()) : "";
        if (!env_value.empty()) {
            std::filesystem::path check_path = std::filesystem::path(env_value) / "include/";
            // Use try catch to suppress file permission exceptions etc
            try {
                if (std::filesystem::exists(check_path)) {
                    cuda_include_dir_str = check_path.string();
                    break;
                }
            } catch (...) { }
            // Throw if the value is not empty, but it does not exist. Outside the try catch excplicityly.
            THROW flamegpu::exception::InvalidFilePath("Error environment variable %s (%s) does not contain a valid CUDA include directory", env_var.c_str(), env_value.c_str());
        }
    }
    // If none of the search enviornmental variables were useful, throw an exception.
    if (cuda_include_dir_str.empty()) {
        THROW exception::InvalidFilePath("Error could not find CUDA include directory. Please specify using the CUDA_PATH environment variable");
    }
    return cuda_include_dir_str;
}

/**
 * Get the FLAME GPU include directory via the environment variables. 
 * @param env_var_used modified to return the name of the environment variable which was used, if any.
 * @return the FLAME GPU 2+ include directory.
 */
std::string getFLAMEGPUIncludeDir(std::string &env_var_used) {
    // Define an array of environment variables to check
    std::array<const std::string, 2> ENV_VARS { "FLAMEGPU_INC_DIR", "FLAMEGPU2_INC_DIR" };
    std::string include_dir_str = "";
    // Iterate the array of environment variables to check for the version header.
    for (const auto& env_var : ENV_VARS) {
        // If the environment variable exists
        std::string env_value = std::getenv(env_var.c_str()) ? std::getenv(env_var.c_str()) : "";
        // If it's a value, check if the path exists, and if any expected files are found.
        if (!env_value.empty()) {
            std::filesystem::path check_file = std::filesystem::path(env_value) / "flamegpu/flamegpu.h";
            // Use try catch to suppress file permission exceptions etc
            try {
                if (std::filesystem::exists(check_file)) {
                    include_dir_str = env_value;
                    env_var_used = env_var;
                    break;
                }
            } catch (...) { }
            // Throw if the value is not empty, but it does not exist. Outside the try catch explicitly.
            THROW flamegpu::exception::InvalidFilePath("Error environment variable %s (%s) does not contain flamegpu/flamegpu.h. Please correct this environment variable.", env_var.c_str(), env_value.c_str());
        }
    }

    // If no appropriate environmental variables were found, check upwards for N levels (assuming the default file structure is in use)
    if (include_dir_str.empty()) {
        // Start with the current working directory
        std::filesystem::path test_dir(".");
        // Try multiple levels of directory, to see if we can find include/flamegpu/flamegpu.h
        const unsigned int LEVELS = 5;
        for (unsigned int level = 0; level < LEVELS; level++) {
            // If break out the loop if the test_dir directory does not exist.
            if (!std::filesystem::exists(test_dir)) {
                break;
            }
            // Check file assuming flamegpu is the root cmake project
            std::filesystem::path check_file = test_dir;
            check_file /= "include/flamegpu/flamegpu.h";
            // Use try catch to suppress file permission exceptions etc
            try {
                if (std::filesystem::exists(check_file)) {
                    test_dir /= "include";
                    include_dir_str = test_dir.string();
                    break;
                }
            } catch (...) { }
            // Check file assuming a standalone example is the root cmake project
            // We want to see if we can find the build directory
            for (auto& p : std::filesystem::directory_iterator(test_dir)) {
                if (std::filesystem::is_directory(p)) {
                    check_file = p.path();
                    check_file /= "_deps/flamegpu2-src/include/flamegpu/version.h";
                    // Use try catch to suppress file permission exceptions etc
                    try {
                        if (exists(check_file)) {
                            test_dir = p.path();
                            test_dir /= "_deps/flamegpu2-src/include";
                            include_dir_str = test_dir.string();
                            goto break_flamegpu_inc_dir_loop;  // Break out of nested loop
                        }
                    } catch (...) { }
                }
            }
            // Go up a level for next iteration
            test_dir /= "..";
        }
break_flamegpu_inc_dir_loop:
        // If still not found, throw.
        if (include_dir_str.empty()) {
            // @todo - more appropriate exception?
            THROW flamegpu::exception::InvalidAgentFunc("Error compiling runtime agent function: Unable to automatically determine include directory and FLAMEGPU_INC_DIR environment variable not set");
        }
    }
    return std::filesystem::absolute(include_dir_str).generic_string();
}

#ifdef FLAMEGPU_USE_GLM
/**
 * Get the GLM include directory via the environment variables.
 * @return the GLM include directory.
 */
std::string getGLMIncludeDir() {
    const std::string env_var = "FLAMEGPU_GLM_INC_DIR";
    const std::string test_file = "glm/glm.hpp";
    // Check the environment variable to see whether glm/glm.hpp exists
    {
        // If the environment variable exists
        std::string env_value = std::getenv(env_var.c_str()) ? std::getenv(env_var.c_str()) : "";
        // If it's a value, check if the path exists, and if any expected files are found.
        if (!env_value.empty()) {
            std::filesystem::path check_file = std::filesystem::path(env_value) / test_file;
            // Use try catch to suppress file permission exceptions etc
            try {
                if (std::filesystem::exists(check_file)) {
                    return env_value;
                }
            }
            catch (...) {}
            // Throw if the value is not empty, but it does not exist. Outside the try catch explicitly.
            THROW flamegpu::exception::InvalidFilePath("Error environment variable %s (%s) does not contain %s. Please correct this environment variable.", env_var.c_str(), env_value.c_str(), test_file.c_str());
        }
    }

    // If no appropriate environmental variables were found, check the compile time path to GLM
    std::filesystem::path check_file = std::filesystem::path(FLAMEGPU_GLM_PATH) / test_file;
    // Use try catch to suppress file permission exceptions etc
    try {
        if (std::filesystem::exists(check_file)) {
            return FLAMEGPU_GLM_PATH;
        }
    }
    catch (...) {}
    // Throw if header wasn't found. Outside the try catch explicitly.
    THROW flamegpu::exception::InvalidAgentFunc("Error compiling runtime agent function: Unable to automatically determine location of GLM include directory and %s environment variable not set", env_var.c_str());
}
#endif

/**
 * Confirm that include directory version header matches the version of the static library.
 * This only compares up to the pre-release version number. Build metadata is only used for the RTC cache.
 * @param flamegpuIncludeDir path to the flamegpu include directory to check.
 * @return boolean indicator of success.
 */
bool confirmFLAMEGPUHeaderVersion(const std::string &flamegpuIncludeDir, const std::string &envVariable) {
    static bool header_version_confirmed = false;

    if (!header_version_confirmed) {
        std::string fileHash;
        std::string fileVersionMacro;
        std::string fileVersionPrerelease;
        // Open version.h
        std::filesystem::path version_file = std::filesystem::path(flamegpuIncludeDir) /= "flamegpu/version.h";
        std::ifstream vFile(version_file);
        if (vFile.is_open()) {
            // Use a regular expression to match the FLAMEGPU_VERSION number macro against lines in the file.
            std::regex macroPattern("^#define FLAMEGPU_VERSION ([0-9]+)$");
            std::regex prereleasePattern("^static constexpr char VERSION_PRERELEASE\\[\\] = \"(.*)\";$");
            std::smatch match;
            std::string line;
            bool extractedMacro = false;
            bool extractedPrerelease = false;
            while (std::getline(vFile, line)) {
                if (std::regex_search(line, match, macroPattern)) {
                    fileVersionMacro = match[1];
                    extractedMacro = true;
                } else if (std::regex_search(line, match, prereleasePattern)) {
                    fileVersionPrerelease = match[1];
                    extractedPrerelease = true;
                }
                if (extractedMacro && extractedPrerelease) {
                    break;
                }
            }
            vFile.close();
            if (!extractedMacro || !extractedPrerelease) {
                THROW exception::VersionMismatch("Could not extract RTC header version information.\n");
            }
        }
        // Confirm that the version matches, else throw an exception.
        if (fileVersionMacro == std::to_string(flamegpu::VERSION) && fileVersionPrerelease == std::string(flamegpu::VERSION_PRERELEASE)) {
            header_version_confirmed = true;
        } else {
            THROW exception::VersionMismatch("RTC header version (%s, %s) does not match version flamegpu library was built with (%s, %s). Set the environment variable %s to the correct include directory.\n",
                fileVersionMacro.c_str(), fileVersionPrerelease.c_str(),
                std::to_string(flamegpu::VERSION).c_str(), flamegpu::VERSION_PRERELEASE,
                envVariable.c_str());
        }
    }
    return header_version_confirmed;
}

}  // namespace

std::mutex JitifyCache::instance_mutex;
std::unique_ptr<jitify2::LinkedProgramData> JitifyCache::buildProgram(
    const std::string &func_name,
    const std::vector<std::string> &template_args,
    const std::string &kernel_src,
    const std::string &dynamic_header,
    const std::string &name_expression) {
    flamegpu::util::nvtx::Range range{"JitifyCache::preprocessKernel"};
    // find and validate the cuda include directory via CUDA_PATH or CUDA_HOME.
    static const std::string cuda_include_dir = getCUDAIncludeDir();
    // find and validate the the flamegpu include directory
    static std::string flamegpu_include_dir_envvar;
    static const std::string flamegpu_include_dir = getFLAMEGPUIncludeDir(flamegpu_include_dir_envvar);
    // verify that the include directory contains the correct headers.
    confirmFLAMEGPUHeaderVersion(flamegpu_include_dir, flamegpu_include_dir_envvar);

     // vector of compiler options for jitify
    std::vector<std::string> options;
    std::unordered_map<std::string, std::string> headers;

    // fpgu include directory
    options.push_back(std::string("-I" + std::string(flamegpu_include_dir)));

    // cuda include directory (via CUDA_PATH)
    options.push_back(std::string("-I" + cuda_include_dir));

    // Add user specified include paths
    for (const auto &p : getIncludeDirs())
        options.push_back(std::string("-I" + p.generic_string()));

#ifdef FLAMEGPU_USE_GLM
    // GLM headers increase build time ~5x, so only enable glm if user is using it
    if (kernel_src.find("glm") != std::string::npos) {
        static const std::string glm_include_dir = getGLMIncludeDir();
        options.push_back(std::string("-I") + glm_include_dir);
        options.push_back(std::string("-DFLAMEGPU_USE_GLM"));
    }
#endif

    // Forward the hiprand Engine request
#if defined(FLAMEGPU_CURAND_MRG32k3a)
    options.push_back(std::string("-DFLAMEGPU_CURAND_MRG32k3a"));
#elif defined(FLAMEGPU_CURAND_Philox4_32_10)
    options.push_back(std::string("-DFLAMEGPU_CURAND_Philox4_32_10"));
#elif defined(FLAMEGPU_CURAND_XORWOW)
    options.push_back(std::string("-DFLAMEGPU_CURAND_XORWOW"));
#endif

    // Set the cuda compuate capability architecture to optimize / generate for, based on the values supported by the current dynamiclaly linked nvrtc and the device in question.
    std::vector<int> nvrtcArchitectures = detail::compute_capability::getNVRTCSupportedComputeCapabilties();
    if (nvrtcArchitectures.size()) {
        int currentDeviceIdx = 0;
        if (hipSuccess == hipGetDevice(&currentDeviceIdx)) {
            int arch = compute_capability::getComputeCapability(currentDeviceIdx);
            int maxSupportedArch = compute_capability::selectAppropraiteComputeCapability(arch, nvrtcArchitectures);
            // only set a nvrtc compilation flag if a usable value was found
            if (maxSupportedArch != 0) {
                options.push_back(std::string("--gpu-architecture=compute_" + std::to_string(maxSupportedArch)));
            } else {
                // This branch should never be taken
                // Rather than throwing an exception which users cannot catch and reover from, assert instead. This will just result in not targetting a specific arch.
                assert(false);
            }
        }
    }

    // If CUDA is compiled with -G (--device-debug) forward it to the compiler, otherwise forward lineinfo for profiling.
#if defined(__CUDACC_DEBUG__)
    options.push_back("--device-debug");
#else
    options.push_back("--generate-line-info");
#endif

    // If DEBUG is defined, forward it
#if defined(DEBUG)
    options.push_back("-DDEBUG");
#endif

    // If NDEBUG is defined, forward it, this should disable asserts in device code.
#if defined(NDEBUG)
    options.push_back("-DNDEBUG");
#endif

// pass the c++ language dialect. It may be better to explicitly pass this from CMake.
#if defined(__cplusplus) && __cplusplus > 201700L && defined(__CUDACC_VER_MAJOR__) && __CUDACC_VER_MAJOR__ >= 11
    options.push_back("--std=c++17");
#endif

    // If FLAMEGPU_SEATBELTS is defined and false, forward it as off, otherwise forward it as on.
#if !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
    options.push_back("--define-macro=FLAMEGPU_SEATBELTS=1");
#else
    options.push_back("--define-macro=FLAMEGPU_SEATBELTS=0");
#endif

    // get the dynamically generated header from curve rtc
    headers.emplace("dynamic/curve_rtc_dynamic.h", dynamic_header);

    // jitify to create program (with compilation settings)
    const std::string program_name = func_name + "_program";  // Does this name actually matter?
    jitify2::PreprocessedProgram program = jitify2::Program(program_name, kernel_src, headers)->preprocess(options);
    if (!program.ok()) {
        const jitify2::ErrorMsg& compile_error = program.error();
        fprintf(stderr, "Failed to load program for agent function (condition) '%s', log:\n%s",
            func_name.c_str(), compile_error.c_str());
        THROW exception::InvalidAgentFunc("Error loading agent function (or function condition) ('%s'): function had compilation errors:\n%s",
            func_name.c_str(), compile_error.c_str());
    }
    // Compile
    jitify2::CompiledProgram compiled_program = program->compile({ name_expression });
    if (!compiled_program.ok()) {
        const jitify2::ErrorMsg& compile_error = compiled_program.error();
        fprintf(stderr, "Failed to compile agent function (condition) '%s', log:\n%s",
            func_name.c_str(), compile_error.c_str());
        THROW exception::InvalidAgentFunc("Error loading agent function (or function condition) ('%s'): function had compilation errors:\n%s",
            func_name.c_str(), compile_error.c_str());
    }
    // Link
    jitify2::LinkedProgram linked_program = compiled_program->link();
    if (!linked_program.ok()) {
        const jitify2::ErrorMsg& link_error = linked_program.error();
        fprintf(stderr, "Failed to link agent function (condition) '%s', log:\n%s",
            func_name.c_str(), link_error.c_str());
        THROW exception::InvalidAgentFunc("Error loading agent function (or function condition) ('%s'): function had link errors:\n%s",
            func_name.c_str(), link_error.c_str());
    }
    return std::make_unique<jitify2::LinkedProgramData>(linked_program.value());
}
std::unique_ptr<jitify2::KernelData> JitifyCache::loadKernel(const std::string &func_name, const std::vector<std::string> &template_args, const std::string &kernel_src, const std::string &dynamic_header) {
    flamegpu::util::nvtx::Range range{"JitifyCache::loadKernel"};
    std::lock_guard<std::mutex> lock(cache_mutex);
    // Detect current compute capability=
    int currentDeviceIdx = 0;
    hipError_t status = hipGetDevice(&currentDeviceIdx);
    const std::string arch = std::to_string((status == hipSuccess) ? compute_capability::getComputeCapability(currentDeviceIdx) : 0);
    status = hipRuntimeGetVersion(&currentDeviceIdx);
    const std::string cuda_version = std::to_string((status == hipSuccess) ? currentDeviceIdx : 0);
    const std::string seatbelts = std::to_string(FLAMEGPU_SEATBELTS);
    // Cat kernel, dynamic header, header version
    const std::string long_reference = kernel_src + dynamic_header;  // Don't need to include rest, they are explicit in short reference/filename
    // Generate short reference string
    // Would prefer to use a proper hash, e.g. md5(reference_string), but that requires extra dependencies
    const std::string short_reference =
        cuda_version + "_" +
        arch + "_" +
        seatbelts + "_" +
        std::string(flamegpu::VERSION_FULL) + "_" +
#ifdef FLAMEGPU_USE_GLM
        "glm_" +
#endif
#if defined(FLAMEGPU_CURAND_MRG32k3a)
        "MRG_" +
#elif defined(FLAMEGPU_CURAND_Philox4_32_10)
        "PHILOX_" +
#elif defined(FLAMEGPU_CURAND_XORWOW)
        "XORWOW_" +
#endif
        // Use jitify hash methods for consistent hashing between OSs
        jitify2::detail::sha256(kernel_src + dynamic_header);
    std::unique_ptr<jitify2::LinkedProgramData> linked_program;
    // Does a copy with the right reference exist in memory?
    if (use_memory_cache) {
        const auto it = cache.find(short_reference);
        if (it != cache.end()) {
            // Check long reference
            if (it->second.long_reference == long_reference) {
                // Deserialize and return program
                jitify2::LinkedProgram prog = jitify2::LinkedProgram::deserialize(it->second.serialised_program);
                if (prog.ok()) {
                    linked_program = std::make_unique<jitify2::LinkedProgramData>(prog.value());
                }
                // Fail silently and try to build code
            }
        }
    }
    // Does a copy with the right reference exist on disk?
    const std::filesystem::path cache_file = getTMP() / short_reference;
    const std::filesystem::path reference_file = cache_file.parent_path() / std::filesystem::path(cache_file.filename().string() + ".ref");
    if (!linked_program && use_disk_cache && std::filesystem::exists(cache_file)) {
        // Load the long reference for the cache file
        const std::string file_long_reference = loadFile(reference_file);
        if (file_long_reference == long_reference) {
            // Load the cache file
            const std::string serialised_kernelinst = loadFile(cache_file);
            if (!serialised_kernelinst.empty()) {
                // Add it to cache for later loads
                cache.emplace(short_reference, CachedProgram{long_reference, serialised_kernelinst});
                // Deserialize and return program
                jitify2::LinkedProgram prog = jitify2::LinkedProgram::deserialize(serialised_kernelinst);
                if (prog.ok()) {
                    linked_program = std::make_unique<jitify2::LinkedProgramData>(prog.value());
                }
                // Fail silently and try to build code
            }
        }
    }
    // Build the name of the template configuration to be instantiated
    std::stringstream name_expression;
    if (template_args.size() == 1) {
        name_expression << "flamegpu::agent_function_condition_wrapper<";
        name_expression << template_args[0];
        name_expression << ">";
    } else if (template_args.size() == 3) {
        name_expression << "flamegpu::agent_function_wrapper<";
        name_expression << template_args[0] << "," << template_args[1] << "," << template_args[2];
        name_expression << ">";
    } else {
        THROW exception::UnknownInternalError("Unexpected AgentFunction template arg count!");
    }
    // Kernel has not yet been cached
    if (!linked_program) {
        // Build kernel
        linked_program = buildProgram(func_name, template_args, kernel_src, dynamic_header, name_expression.str());
        // Add it to cache for later loads
        const std::string serialised_program = use_memory_cache || use_disk_cache ? linked_program->serialize() : "";
        if (use_memory_cache) {
            cache.emplace(short_reference, CachedProgram{long_reference, serialised_program });
        }
        // Save it to disk
        if (use_disk_cache) {
            std::ofstream ofs(cache_file, std::ofstream::out | std::ofstream::binary | std::ofstream::trunc);
            if (ofs) {
                ofs << serialised_program;
                ofs.close();
            }
            ofs = std::ofstream(reference_file, std::ofstream::out | std::ofstream::binary | std::ofstream::trunc);
            if (ofs) {
                ofs << long_reference;
                ofs.close();
            }
        }
    }
    jitify2::LoadedProgram loaded_program = linked_program->load();
    if (!loaded_program.ok()) {
        const jitify2::ErrorMsg& compile_error = loaded_program.error();
        fprintf(stderr, "Failed to load program for agent function (condition) '%s' into memory, log:\n%s",
            func_name.c_str(), compile_error.c_str());
        THROW exception::InvalidAgentFunc("Error loading agent function (or function condition) ('%s'): function had errors (see std::out), "
            "in JitifyCache::loadKernel().",
            func_name.c_str());
    }
    jitify2::Kernel loaded_kernel = loaded_program->get_kernel(name_expression.str());
    if (loaded_kernel.ok()) {
        return std::make_unique<jitify2::KernelData>(loaded_kernel.value());
    }
    const jitify2::ErrorMsg& compile_error = loaded_kernel.error();
    fprintf(stderr, "Failed to compile and link agent function (condition) '%s', log:\n%s",
        func_name.c_str(), compile_error.c_str());
    THROW exception::InvalidAgentFunc("Error compiling runtime agent function (or function condition) ('%s'): function had compilation errors (see std::cout), "
        "in JitifyCache::loadKernel().",
        func_name.c_str());
}
void JitifyCache::useMemoryCache(bool yesno) {
    std::lock_guard<std::mutex> lock(cache_mutex);
    use_memory_cache = yesno;
}
void JitifyCache::useDiskCache(bool yesno) {
    std::lock_guard<std::mutex> lock(cache_mutex);
    use_disk_cache = yesno;
}
bool JitifyCache::useMemoryCache() const {
    std::lock_guard<std::mutex> lock(cache_mutex);
    return use_memory_cache;
}
bool JitifyCache::useDiskCache() const {
    std::lock_guard<std::mutex> lock(cache_mutex);
    return use_disk_cache;
}
void JitifyCache::clearMemoryCache() {
    std::lock_guard<std::mutex> lock(cache_mutex);
    cache.clear();
}
void JitifyCache::clearDiskCache() {
    const std::filesystem::path tmp_dir = getTMP();
    for (const auto & entry : std::filesystem::directory_iterator(tmp_dir)) {
        if (std::filesystem::is_regular_file(entry.path())) {
            remove(entry.path());
        }
    }
}
JitifyCache::JitifyCache()
    : use_memory_cache(true)
#ifndef FLAMEGPU_DISABLE_RTC_DISK_CACHE
    , use_disk_cache(true) { }
#else
    , use_disk_cache(false) { }
#endif
JitifyCache& JitifyCache::getInstance() {
    auto lock = std::unique_lock<std::mutex>(instance_mutex);  // Mutex to protect from two threads triggering the static instantiation concurrently
    static JitifyCache instance;  // Instantiated on first use.
    return instance;
}

}  // namespace detail
}  // namespace flamegpu
