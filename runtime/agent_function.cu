#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>

#include "../flame_functions_api.h"

__global__ void agent_function_wrapper(const char* func_name, FLAMEGPU_AGENT_FUNCTION_POINTER func)
{
	//create a new device FLAME_GPU instance
	FLAMEGPU_API *api = new FLAMEGPU_API();

	//call the user specified device function
	func(api);

	//do something with the return value to set a flag for deletion
}