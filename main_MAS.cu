/******************************************************************************
 * main_MAS.cu is a host function that prepares data array and passes it to the CUDA kernel.
 * This main_MAS.cu would either be specified by a user or automatically generated from the model.xml.
 * Each of the API functions will have a 121 mapping with XML elements
 * The API is very similar to FLAME 2. The directory structure and general project is set out very similarly.

 * Multi Agent model example

 ******************************************************************************
 * Author  Paul Richmond, Mozhgan Kabiri Chimeh
 * Date    Feb 2017
 *****************************************************************************/

#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "runtime/flame_api.h"


using namespace std;


/* must be compiled separately using FLAME GPU builder
 * This will generate object files for different architecture targets as well as ptx info for each agent function (registers, memory use etc.)
 * http://stackoverflow.com/questions/12388207/interpreting-output-of-ptxas-options-v
 */


FLAMEGPU_AGENT_FUNCTION(output_func)
{
    printf("Hello from output_func\n");
    float x = FLAMEGPU->getVariable<float>("x");
    float y = FLAMEGPU->getVariable<float>("y");
    printf("x = %f, y = %f\n", x, y);
    FLAMEGPU->setVariable<float>("x", x + 3);
    x = FLAMEGPU->getVariable<float>("x");
    printf("x after set = %f\n", x);
    return ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(input_func)
{
   printf("Hello from input_func\n");
    float x = FLAMEGPU->getVariable<float>("x");
    float y = FLAMEGPU->getVariable<float>("y");
    printf("x = %f, y = %f\n", x, y);
    FLAMEGPU->setVariable<float>("x", x + 2);
    x = FLAMEGPU->getVariable<float>("x");
    printf("x after set = %f\n", x);
    return ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(add_func)
{
   //printf("Hello from add_func\n");
    float x = FLAMEGPU->getVariable<float>("x");
    float y = FLAMEGPU->getVariable<float>("y");
   printf("-y = %f, x = %f\n", y, x);
    FLAMEGPU->setVariable<float>("y", y + x);
    y = FLAMEGPU->getVariable<float>("y");
   printf("-y after set = %f\n", y);
    return ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(subtract_func)
{
    //printf("Hello from subtract_func\n");
    float x = FLAMEGPU->getVariable<float>("x");
    float y = FLAMEGPU->getVariable<float>("y");
    printf("y = %f, x = %f\n", y, x);
    FLAMEGPU->setVariable<float>("y", x - y);
    y = FLAMEGPU->getVariable<float>("y");
    printf("y after set = %f\n", y);
    return ALIVE;
}

int main(void)
{
    /* Multi agent model */
    ModelDescription flame_model("circles_model");

    AgentDescription circle1_agent("circle1");
    circle1_agent.addAgentVariable<float>("x");
    circle1_agent.addAgentVariable<float>("y");

    AgentDescription circle2_agent("circle2");
    circle2_agent.addAgentVariable<float>("x");
    circle2_agent.addAgentVariable<float>("y");

    //same name ?
    //MessageDescription location1_message("location");
    //location1_message.addVariable<float>("x");
    //location1_message.addVariable<float>("y");

    //MessageDescription location2_message("location");
    //location2_message.addVariable<float>("x");
    //location2_message.addVariable<float>("y");


    AgentFunctionDescription output_data("output_data");
    //AgentFunctionOutput output_location("location");
    //output_data.addOutput(output_location);
    output_data.setFunction(&output_func);
    circle1_agent.addAgentFunction(output_data);

    AgentFunctionDescription input_data("input_data");
    //AgentFunctionInput input_location("location");
    //input_data.addInput(input_location);
    input_data.setFunction(&input_func);
    circle2_agent.addAgentFunction(input_data);

    AgentFunctionDescription add_data("add_data");
    //add_data.addInput(input_location);
    add_data.setFunction(&add_func);
    circle1_agent.addAgentFunction(add_data);

    AgentFunctionDescription subtract_data("subtract_data");
    //subtract_data.addInput(input_location);
    subtract_data.setFunction(&subtract_func);
    circle2_agent.addAgentFunction(subtract_data);


    //model
    //flame_model.addMessage(location1_message);
    flame_model.addAgent(circle1_agent);

   // flame_model.addMessage(location2_message);
    flame_model.addAgent(circle2_agent);
#define SIZE 10
    AgentPopulation population1(circle1_agent, SIZE);
    for (int i=0; i< SIZE; i++)
    {
        AgentInstance instance = population1.getNextInstance("default");
        instance.setVariable<float>("x", i*0.1f);
        instance.setVariable<float>("y", i*0.1f);
    }

    AgentPopulation population2(circle2_agent, SIZE);
    for (int i=0; i< SIZE; i++)
    {
        AgentInstance instance = population2.getNextInstance("default");
        instance.setVariable<float>("x", i*0.2f);
        instance.setVariable<float>("y", i*0.2f);
    }

    Simulation simulation(flame_model);

    SimulationLayer output_layer(simulation, "output_layer");
    output_layer.addAgentFunction("output_data");
    simulation.addSimulationLayer(output_layer);

    SimulationLayer input_layer(simulation, "input_layer");
    input_layer.addAgentFunction("input_data");
    simulation.addSimulationLayer(input_layer);

    //multiple functions per simulation layer (from different agents)
    SimulationLayer concurrent_layer(simulation, "concurrent_layer");
    concurrent_layer.addAgentFunction("add_data");
    concurrent_layer.addAgentFunction("subtract_data");
    simulation.addSimulationLayer(concurrent_layer);

    simulation.setSimulationSteps(1);

    /* Run the model */
    CUDAAgentModel cuda_model(flame_model);

    cuda_model.setInitialPopulationData(population1);
    cuda_model.setInitialPopulationData(population2);

    cuda_model.addSimulation(simulation);

    cuda_model.step(simulation);

    cuda_model.getPopulationData(population1);
    cuda_model.getPopulationData(population2);

    return 0;
}

