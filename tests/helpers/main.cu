#include <hip/hip_runtime.h>
#include <cstdio>
#include "gtest/gtest.h"


GTEST_API_ int main(int argc, char **argv) {
  printf("Running main() from %s\n", __FILE__);
  testing::InitGoogleTest(&argc, argv);
  auto rtn = RUN_ALL_TESTS();
  hipDeviceReset();
  return rtn;
}
