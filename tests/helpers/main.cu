#include <hip/hip_runtime.h>
#include <cstdio>
#include <map>

#include "flamegpu/gpu/CUDASimulation.h"
#include "gtest/gtest.h"
#include "helpers/device_initialisation.h"
#include "flamegpu/io/Telemetry.h"


GTEST_API_ int main(int argc, char **argv) {
    // Get the current status of telemetry, to control if test suite results shold be submit or not
    const bool telemetryEnabled = flamegpu::io::Telemetry::isEnabled();
    // Disable telemetry for simulation / ensemble objects in the test suite.
    flamegpu::io::Telemetry::disable();
    // Suppress the notice about telemetry.
    flamegpu::io::Telemetry::suppressNotice();
    // Time the cuda agent model initialisation, to check it creates the context.
    flamegpu::tests::timeCUDASimulationContextCreationTest();
    // Run the main google test body
    printf("Running main() from %s\n", __FILE__);
    testing::InitGoogleTest(&argc, argv);
    auto rtn = RUN_ALL_TESTS();
    // Reset all cuda devices for memcheck / profiling purposes.
    int devices = 0;
    gpuErrchk(hipGetDeviceCount(&devices));
    if (devices > 0) {
        for (int device = 0; device < devices; ++device) {
            gpuErrchk(hipSetDevice(device));
            gpuErrchk(hipDeviceReset());
        }
    }

    return rtn;
}
