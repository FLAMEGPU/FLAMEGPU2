#include "hip/hip_runtime.h"
#include "flamegpu/flame_api.h"

#include "gtest/gtest.h"

/* 
    Test suite to check that RTC behaves corrrectly with the various methods of using namespaces.
 */

namespace flamegpu {
namespace test_rtc_namespaces {
const unsigned int AGENT_COUNT = 32;

// All agent functions require specialising the Message type, regardless of wheter it is None or not, so not point testing without message output being involved.

// Test using explicitly namespaced types. ie. flamegpu::ALIVE
const char* msg_out_func_explicit = R"###(
FLAMEGPU_AGENT_FUNCTION(msg_out_func_explicit, flamegpu::MsgNone, flamegpu::MsgBruteForce) {
    FLAMEGPU->message_out.setVariable("x", FLAMEGPU->getVariable<int>("x"));
    return flamegpu::ALIVE;
}
)###";

const char* msg_in_func_explicit = R"###(
FLAMEGPU_AGENT_FUNCTION(msg_in_func_explicit, flamegpu::MsgBruteForce, flamegpu::MsgNone) {
    int sum = 0;
    for (auto& message : FLAMEGPU->message_in) {
        const int x = message.getVariable<int>("x");
        sum += x;
    }
    FLAMEGPU->setVariable<int>("sum", sum);
    return flamegpu::ALIVE;
}
)###";


TEST(RTCNamespaceTest, AgentFunctionsExplicit) {
    ModelDescription m("model");
    MsgBruteForce::Description& msg = m.newMessage("message_x");
    msg.newVariable<int>("x");
    AgentDescription& a = m.newAgent("agent");
    a.newVariable<int>("x");
    a.newVariable<int>("sum");
    a.newVariable<int>("product");
    AgentFunctionDescription& fo = a.newRTCFunction("msg_out_func_explicit", msg_out_func_explicit);
    fo.setMessageOutput(msg);
    AgentFunctionDescription& fi = a.newRTCFunction("msg_in_func_explicit", msg_in_func_explicit);
    fi.setMessageInput(msg);
    std::default_random_engine rng(static_cast<unsigned int>(time(nullptr)));
    std::uniform_int_distribution<int> dist(-3, 3);
    AgentVector pop(a, (unsigned int)AGENT_COUNT);
    int sum = 0;
    for (AgentVector::Agent ai : pop) {
        const int x = dist(rng);
        sum += x;
        ai.setVariable<int>("x", x);
    }
    LayerDescription& lo = m.newLayer("output_layer");
    lo.addAgentFunction(fo);
    LayerDescription& li = m.newLayer("input_layer");
    li.addAgentFunction(fi);
    CUDASimulation c(m);
    c.SimulationConfig().steps = 1;
    c.setPopulationData(pop);
    c.simulate();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        ASSERT_EQ(ai.getVariable<int>("sum"), sum);
    }
}

// Test via the using declarations, i.e. using flamegpu::ALIVE
const char* msg_out_func_declaration = R"###(
using flamegpu::ALIVE;
using flamegpu::MsgNone;
using flamegpu::MsgBruteForce;
FLAMEGPU_AGENT_FUNCTION(msg_out_func_declaration, MsgNone, MsgBruteForce) {
    FLAMEGPU->message_out.setVariable("x", FLAMEGPU->getVariable<int>("x"));
    return ALIVE;
}
)###";

const char* msg_in_func_declaration = R"###(
using flamegpu::ALIVE;
using flamegpu::MsgNone;
using flamegpu::MsgBruteForce;
FLAMEGPU_AGENT_FUNCTION(msg_in_func_declaration, MsgBruteForce, MsgNone) {
    int sum = 0;
    for (auto& message : FLAMEGPU->message_in) {
        const int x = message.getVariable<int>("x");
        sum += x;
    }
    FLAMEGPU->setVariable<int>("sum", sum);
    return ALIVE;
}
)###";


TEST(RTCNamespaceTest, AgentFunctionsDeclaration) {
    ModelDescription m("model");
    MsgBruteForce::Description& msg = m.newMessage("message_x");
    msg.newVariable<int>("x");
    AgentDescription& a = m.newAgent("agent");
    a.newVariable<int>("x");
    a.newVariable<int>("sum");
    a.newVariable<int>("product");
    AgentFunctionDescription& fo = a.newRTCFunction("msg_out_func_declaration", msg_out_func_declaration);
    fo.setMessageOutput(msg);
    AgentFunctionDescription& fi = a.newRTCFunction("msg_in_func_declaration", msg_in_func_declaration);
    fi.setMessageInput(msg);
    std::default_random_engine rng(static_cast<unsigned int>(time(nullptr)));
    std::uniform_int_distribution<int> dist(-3, 3);
    AgentVector pop(a, (unsigned int)AGENT_COUNT);
    int sum = 0;
    for (AgentVector::Agent ai : pop) {
        const int x = dist(rng);
        sum += x;
        ai.setVariable<int>("x", x);
    }
    LayerDescription& lo = m.newLayer("output_layer");
    lo.addAgentFunction(fo);
    LayerDescription& li = m.newLayer("input_layer");
    li.addAgentFunction(fi);
    CUDASimulation c(m);
    c.SimulationConfig().steps = 1;
    c.setPopulationData(pop);
    c.simulate();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        ASSERT_EQ(ai.getVariable<int>("sum"), sum);
    }
}

// Test via the using directives (which outside of rtc will be a lint failure), i.e. using namespace flamegpu
const char* msg_out_func_directive = R"###(
using namespace flamegpu;
FLAMEGPU_AGENT_FUNCTION(msg_out_func_directive, MsgNone, MsgBruteForce) {
    FLAMEGPU->message_out.setVariable("x", FLAMEGPU->getVariable<int>("x"));
    return ALIVE;
}
)###";

const char* msg_in_func_directive = R"###(
using namespace flamegpu;
FLAMEGPU_AGENT_FUNCTION(msg_in_func_directive, MsgBruteForce, MsgNone) {
    int sum = 0;
    for (auto& message : FLAMEGPU->message_in) {
        const int x = message.getVariable<int>("x");
        sum += x;
    }
    FLAMEGPU->setVariable<int>("sum", sum);
    return ALIVE;
}
)###";


TEST(RTCNamespaceTest, AgentFunctionsDirective) {
    ModelDescription m("model");
    MsgBruteForce::Description& msg = m.newMessage("message_x");
    msg.newVariable<int>("x");
    AgentDescription& a = m.newAgent("agent");
    a.newVariable<int>("x");
    a.newVariable<int>("sum");
    a.newVariable<int>("product");
    AgentFunctionDescription& fo = a.newRTCFunction("msg_out_func_directive", msg_out_func_directive);
    fo.setMessageOutput(msg);
    AgentFunctionDescription& fi = a.newRTCFunction("msg_in_func_directive", msg_in_func_directive);
    fi.setMessageInput(msg);
    std::default_random_engine rng(static_cast<unsigned int>(time(nullptr)));
    std::uniform_int_distribution<int> dist(-3, 3);
    AgentVector pop(a, (unsigned int)AGENT_COUNT);
    int sum = 0;
    for (AgentVector::Agent ai : pop) {
        const int x = dist(rng);
        sum += x;
        ai.setVariable<int>("x", x);
    }
    LayerDescription& lo = m.newLayer("output_layer");
    lo.addAgentFunction(fo);
    LayerDescription& li = m.newLayer("input_layer");
    li.addAgentFunction(fi);
    CUDASimulation c(m);
    c.SimulationConfig().steps = 1;
    c.setPopulationData(pop);
    c.simulate();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        ASSERT_EQ(ai.getVariable<int>("sum"), sum);
    }
}



/* Test by embedding within the flamegpu namespace. (Users shouldn't be doing this, but it is an option.)
This option requires additional steps for NVRTC to correctly compile things.
https://docs.nvidia.com/cuda/nvrtc/index.html#accessing-lowered-names
__global__, __constant__ and __device__ in namespaces need to be expliciltly made available via  hiprtcAddNameExpression
This is a lot of additional complexity to support edge case behaviour, that we do not want to encourage anyway, so disabling this test. */

/* const char* msg_out_func_named = R"###(
namespace flamegpu {
FLAMEGPU_AGENT_FUNCTION(msg_out_func_named, MsgNone, MsgBruteForce) {
    FLAMEGPU->message_out.setVariable("x", FLAMEGPU->getVariable<int>("x"));
    return ALIVE;
}
}  // namespace flamegpu
)###";

const char* msg_in_func_named = R"###(
namespace flamegpu {
FLAMEGPU_AGENT_FUNCTION(msg_in_func_named, MsgBruteForce, MsgNone) {
    int sum = 0;
    for (auto& message : FLAMEGPU->message_in) {
        const int x = message.getVariable<int>("x");
        sum += x;
    }
    FLAMEGPU->setVariable<int>("sum", sum);
    return ALIVE;
}
}  // namespace flamegpu
)###";


TEST(RTCNamespaceTest, AgentFunctionsNamed) {
    ModelDescription m("model");
    MsgBruteForce::Description& msg = m.newMessage("message_x");
    msg.newVariable<int>("x");
    AgentDescription& a = m.newAgent("agent");
    a.newVariable<int>("x");
    a.newVariable<int>("sum");
    a.newVariable<int>("product");
    AgentFunctionDescription& fo = a.newRTCFunction("msg_out_func_named", msg_out_func_named);
    fo.setMessageOutput(msg);
    AgentFunctionDescription& fi = a.newRTCFunction("msg_in_func_named", msg_in_func_named);
    fi.setMessageInput(msg);
    std::default_random_engine rng(static_cast<unsigned int>(time(nullptr)));
    std::uniform_int_distribution<int> dist(-3, 3);
    AgentVector pop(a, (unsigned int)AGENT_COUNT);
    int sum = 0;
    for (AgentVector::Agent ai : pop) {
        const int x = dist(rng);
        sum += x;
        ai.setVariable<int>("x", x);
    }
    LayerDescription& lo = m.newLayer("output_layer");
    lo.addAgentFunction(fo);
    LayerDescription& li = m.newLayer("input_layer");
    li.addAgentFunction(fi);
    CUDASimulation c(m);
    c.SimulationConfig().steps = 1;
    c.setPopulationData(pop);
    c.simulate();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        ASSERT_EQ(ai.getVariable<int>("sum"), sum);
    }
}
 */

// Test using an aliased namespace
const char* msg_out_func_alias = R"###(
namespace fgpu = flamegpu;
FLAMEGPU_AGENT_FUNCTION(msg_out_func_alias, fgpu::MsgNone, fgpu::MsgBruteForce) {
    FLAMEGPU->message_out.setVariable("x", FLAMEGPU->getVariable<int>("x"));
    return fgpu::ALIVE;
}
)###";

const char* msg_in_func_alias = R"###(
namespace fgpu = flamegpu;
FLAMEGPU_AGENT_FUNCTION(msg_in_func_alias, fgpu::MsgBruteForce, fgpu::MsgNone) {
    int sum = 0;
    for (auto& message : FLAMEGPU->message_in) {
        const int x = message.getVariable<int>("x");
        sum += x;
    }
    FLAMEGPU->setVariable<int>("sum", sum);
    return fgpu::ALIVE;
}
)###";


TEST(RTCNamespaceTest, AgentFunctionsAlias) {
    ModelDescription m("model");
    MsgBruteForce::Description& msg = m.newMessage("message_x");
    msg.newVariable<int>("x");
    AgentDescription& a = m.newAgent("agent");
    a.newVariable<int>("x");
    a.newVariable<int>("sum");
    a.newVariable<int>("product");
    AgentFunctionDescription& fo = a.newRTCFunction("msg_out_func_alias", msg_out_func_alias);
    fo.setMessageOutput(msg);
    AgentFunctionDescription& fi = a.newRTCFunction("msg_in_func_alias", msg_in_func_alias);
    fi.setMessageInput(msg);
    std::default_random_engine rng(static_cast<unsigned int>(time(nullptr)));
    std::uniform_int_distribution<int> dist(-3, 3);
    AgentVector pop(a, (unsigned int)AGENT_COUNT);
    int sum = 0;
    for (AgentVector::Agent ai : pop) {
        const int x = dist(rng);
        sum += x;
        ai.setVariable<int>("x", x);
    }
    LayerDescription& lo = m.newLayer("output_layer");
    lo.addAgentFunction(fo);
    LayerDescription& li = m.newLayer("input_layer");
    li.addAgentFunction(fi);
    CUDASimulation c(m);
    c.SimulationConfig().steps = 1;
    c.setPopulationData(pop);
    c.simulate();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        ASSERT_EQ(ai.getVariable<int>("sum"), sum);
    }
}

// Test aliasing the flamegpu namespace, but accessing using a mix (to check that message type comparisons work)
const char* msg_out_func_alias_mixed = R"###(
namespace fgpu = flamegpu;
FLAMEGPU_AGENT_FUNCTION(msg_out_func_alias_mixed, fgpu::MsgNone, fgpu::MsgBruteForce) {
    FLAMEGPU->message_out.setVariable("x", FLAMEGPU->getVariable<int>("x"));
    return fgpu::ALIVE;
}
)###";

const char* msg_in_func_alias_mixed = R"###(
namespace fgpu = flamegpu;
FLAMEGPU_AGENT_FUNCTION(msg_in_func_alias_mixed, flamegpu::MsgBruteForce, flamegpu::MsgNone) {
    int sum = 0;
    for (auto& message : FLAMEGPU->message_in) {
        const int x = message.getVariable<int>("x");
        sum += x;
    }
    FLAMEGPU->setVariable<int>("sum", sum);
    return fgpu::ALIVE;
}
)###";


TEST(RTCNamespaceTest, AgentFunctionsAliasMixed) {
    ModelDescription m("model");
    MsgBruteForce::Description& msg = m.newMessage("message_x");
    msg.newVariable<int>("x");
    AgentDescription& a = m.newAgent("agent");
    a.newVariable<int>("x");
    a.newVariable<int>("sum");
    a.newVariable<int>("product");
    AgentFunctionDescription& fo = a.newRTCFunction("msg_out_func_alias_mixed", msg_out_func_alias_mixed);
    fo.setMessageOutput(msg);
    AgentFunctionDescription& fi = a.newRTCFunction("msg_in_func_alias_mixed", msg_in_func_alias_mixed);
    fi.setMessageInput(msg);
    std::default_random_engine rng(static_cast<unsigned int>(time(nullptr)));
    std::uniform_int_distribution<int> dist(-3, 3);
    AgentVector pop(a, (unsigned int)AGENT_COUNT);
    int sum = 0;
    for (AgentVector::Agent ai : pop) {
        const int x = dist(rng);
        sum += x;
        ai.setVariable<int>("x", x);
    }
    LayerDescription& lo = m.newLayer("output_layer");
    lo.addAgentFunction(fo);
    LayerDescription& li = m.newLayer("input_layer");
    li.addAgentFunction(fi);
    CUDASimulation c(m);
    c.SimulationConfig().steps = 1;
    c.setPopulationData(pop);
    c.simulate();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        ASSERT_EQ(ai.getVariable<int>("sum"), sum);
    }
}


}  // namespace test_rtc_namespaces
}  // namespace flamegpu
