#include "hip/hip_runtime.h"
#ifndef TESTS_TEST_CASES_RUNTIME_TEST_HOST_REDUCTIONS_H_
#define TESTS_TEST_CASES_RUNTIME_TEST_HOST_REDUCTIONS_H_

#include <array>
#include <random>
#include <numeric>
#include <algorithm>

#include "gtest/gtest.h"

#include "flamegpu/flame_api.h"
#include "flamegpu/runtime/flamegpu_api.h"

namespace {
const unsigned int TEST_LEN = 256;
float float_out = 0;
double double_out = 0;
char char_out = 0;
unsigned char uchar_out = 0;
uint16_t uint16_t_out = 0;
int16_t int16_t_out = 0;
uint32_t uint32_t_out = 0;
int32_t int32_t_out = 0;
uint64_t uint64_t_out = 0;
int64_t int64_t_out = 0;
FLAMEGPU_STEP_FUNCTION(step_minfloat) {
    float_out = FLAMEGPU->agent("agent").min<float>("float");
}
FLAMEGPU_STEP_FUNCTION(step_maxfloat) {
    float_out = FLAMEGPU->agent("agent").max<float>("float");
}
FLAMEGPU_STEP_FUNCTION(step_sumfloat) {
    float_out = FLAMEGPU->agent("agent").sum<float>("float");
}
FLAMEGPU_STEP_FUNCTION(step_mindouble) {
    double_out = FLAMEGPU->agent("agent").min<double>("double");
}
FLAMEGPU_STEP_FUNCTION(step_maxdouble) {
    double_out = FLAMEGPU->agent("agent").max<double>("double");
}
FLAMEGPU_STEP_FUNCTION(step_sumdouble) {
    double_out = FLAMEGPU->agent("agent").sum<double>("double");
}
FLAMEGPU_STEP_FUNCTION(step_minuchar) {
    uchar_out = FLAMEGPU->agent("agent").min<unsigned char>("uchar");
}
FLAMEGPU_STEP_FUNCTION(step_maxuchar) {
    uchar_out = FLAMEGPU->agent("agent").max<unsigned char>("uchar");
}
FLAMEGPU_STEP_FUNCTION(step_sumuchar) {
    uchar_out = FLAMEGPU->agent("agent").sum<unsigned char>("uchar");
    uint64_t_out = FLAMEGPU->agent("agent").sum<unsigned char, int64_t>("uchar");
}
FLAMEGPU_STEP_FUNCTION(step_minchar) {
    char_out = FLAMEGPU->agent("agent").min<char>("char");
}
FLAMEGPU_STEP_FUNCTION(step_maxchar) {
    char_out = FLAMEGPU->agent("agent").max<char>("char");
}
FLAMEGPU_STEP_FUNCTION(step_sumchar) {
    char_out = FLAMEGPU->agent("agent").sum<char>("char");
    int64_t_out = FLAMEGPU->agent("agent").sum<char, int64_t>("char");
}
FLAMEGPU_STEP_FUNCTION(step_minuint16_t) {
    uint16_t_out = FLAMEGPU->agent("agent").min<uint16_t>("uint16_t");
}
FLAMEGPU_STEP_FUNCTION(step_maxuint16_t) {
    uint16_t_out = FLAMEGPU->agent("agent").max<uint16_t>("uint16_t");
}
FLAMEGPU_STEP_FUNCTION(step_sumuint16_t) {
    uint16_t_out = FLAMEGPU->agent("agent").sum<uint16_t>("uint16_t");
    uint64_t_out = FLAMEGPU->agent("agent").sum<uint16_t, int64_t>("uint16_t");
}
FLAMEGPU_STEP_FUNCTION(step_minint16_t) {
    int16_t_out = FLAMEGPU->agent("agent").min<int16_t>("int16_t");
}
FLAMEGPU_STEP_FUNCTION(step_maxint16_t) {
    int16_t_out = FLAMEGPU->agent("agent").max<int16_t>("int16_t");
}
FLAMEGPU_STEP_FUNCTION(step_sumint16_t) {
    int16_t_out = FLAMEGPU->agent("agent").sum<int16_t>("int16_t");
    int64_t_out = FLAMEGPU->agent("agent").sum<int16_t, int64_t>("int16_t");
}
FLAMEGPU_STEP_FUNCTION(step_minuint32_t) {
    uint32_t_out = FLAMEGPU->agent("agent").min<uint32_t>("uint32_t");
}
FLAMEGPU_STEP_FUNCTION(step_maxuint32_t) {
    uint32_t_out = FLAMEGPU->agent("agent").max<uint32_t>("uint32_t");
}
FLAMEGPU_STEP_FUNCTION(step_sumuint32_t) {
    uint32_t_out = FLAMEGPU->agent("agent").sum<uint32_t>("uint32_t");
    uint64_t_out = FLAMEGPU->agent("agent").sum<uint32_t, int64_t>("uint32_t");
}
FLAMEGPU_STEP_FUNCTION(step_minint32_t) {
    int32_t_out = FLAMEGPU->agent("agent").min<int32_t>("int32_t");
}
FLAMEGPU_STEP_FUNCTION(step_maxint32_t) {
    int32_t_out = FLAMEGPU->agent("agent").max<int32_t>("int32_t");
}
FLAMEGPU_STEP_FUNCTION(step_sumint32_t) {
    int32_t_out = FLAMEGPU->agent("agent").sum<int32_t>("int32_t");
    int64_t_out = FLAMEGPU->agent("agent").sum<int32_t, int64_t>("int32_t");
}
FLAMEGPU_STEP_FUNCTION(step_minuint64_t) {
    uint64_t_out = FLAMEGPU->agent("agent").min<uint64_t>("uint64_t");
}
FLAMEGPU_STEP_FUNCTION(step_maxuint64_t) {
    uint64_t_out = FLAMEGPU->agent("agent").max<uint64_t>("uint64_t");
}
FLAMEGPU_STEP_FUNCTION(step_sumuint64_t) {
    uint64_t_out = FLAMEGPU->agent("agent").sum<uint64_t>("uint64_t");
}
FLAMEGPU_STEP_FUNCTION(step_minint64_t) {
    int64_t_out = FLAMEGPU->agent("agent").min<int64_t>("int64_t");
}
FLAMEGPU_STEP_FUNCTION(step_maxint64_t) {
    int64_t_out = FLAMEGPU->agent("agent").max<int64_t>("int64_t");
}
FLAMEGPU_STEP_FUNCTION(step_sumint64_t) {
    int64_t_out = FLAMEGPU->agent("agent").sum<int64_t>("int64_t");
}

FLAMEGPU_CUSTOM_REDUCTION(customMax, a, b) {
    return a > b ? a : b;
}
FLAMEGPU_STEP_FUNCTION(step_reducefloat) {
    float_out = FLAMEGPU->agent("agent").reduce<float>("float", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reducedouble) {
    double_out = FLAMEGPU->agent("agent").reduce<double>("double", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reduceuchar) {
    uchar_out = FLAMEGPU->agent("agent").reduce<unsigned char>("uchar", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reducechar) {
    char_out = FLAMEGPU->agent("agent").reduce<char>("char", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reduceuint16_t) {
    uint16_t_out = FLAMEGPU->agent("agent").reduce<uint16_t>("uint16_t", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reduceint16_t) {
    int16_t_out = FLAMEGPU->agent("agent").reduce<int16_t>("int16_t", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reduceuint32_t) {
    uint32_t_out = FLAMEGPU->agent("agent").reduce<uint32_t>("uint32_t", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reduceint32_t) {
    int32_t_out = FLAMEGPU->agent("agent").reduce<int32_t>("int32_t", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reduceuint64_t) {
    uint64_t_out = FLAMEGPU->agent("agent").reduce<uint64_t>("uint64_t", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reduceint64_t) {
    int64_t_out = FLAMEGPU->agent("agent").reduce<int64_t>("int64_t", customMax, 0);
}

std::vector<unsigned int> uint_vec;
std::vector<int> int_vec;
FLAMEGPU_STEP_FUNCTION(step_histogramEvenfloat) {
    uint_vec = FLAMEGPU->agent("agent").histogramEven<float, unsigned int>("float", 10, 0.0f, 20.0f);
}
FLAMEGPU_STEP_FUNCTION(step_histogramEvendouble) {
    int_vec = FLAMEGPU->agent("agent").histogramEven<double, int>("double", 10, 0.0, 20.0);
}
FLAMEGPU_STEP_FUNCTION(step_histogramEvenchar) {
    uint_vec = FLAMEGPU->agent("agent").histogramEven<char, unsigned int>("char", 10, 0, 20);
}
FLAMEGPU_STEP_FUNCTION(step_histogramEvenuchar) {
    int_vec = FLAMEGPU->agent("agent").histogramEven<unsigned char, int>("uchar", 10, 0, 20);
}
FLAMEGPU_STEP_FUNCTION(step_histogramEvenint16_t) {
    uint_vec = FLAMEGPU->agent("agent").histogramEven<int16_t, unsigned int>("int16_t", 10, 0, 20);
}
FLAMEGPU_STEP_FUNCTION(step_histogramEvenuint16_t) {
    int_vec = FLAMEGPU->agent("agent").histogramEven<uint16_t, int>("uint16_t", 10, 0, 20);
}
FLAMEGPU_STEP_FUNCTION(step_histogramEvenint32_t) {
    uint_vec = FLAMEGPU->agent("agent").histogramEven<int32_t, unsigned int>("int32_t", 10, 0, 20);
}
FLAMEGPU_STEP_FUNCTION(step_histogramEvenuint32_t) {
    int_vec = FLAMEGPU->agent("agent").histogramEven<uint32_t, int>("uint32_t", 10, 0, 20);
}
FLAMEGPU_STEP_FUNCTION(step_histogramEvenint64_t) {
    uint_vec = FLAMEGPU->agent("agent").histogramEven<int64_t, unsigned int>("int64_t", 10, 0, 20);
}
FLAMEGPU_STEP_FUNCTION(step_histogramEvenuint64_t) {
    int_vec = FLAMEGPU->agent("agent").histogramEven<uint64_t, int>("uint64_t", 10, 0, 20);
}

FLAMEGPU_STEP_FUNCTION(step_countfloat) {
    uint32_t_out = FLAMEGPU->agent("agent").count<float>("float", 0);
}
FLAMEGPU_STEP_FUNCTION(step_countdouble) {
    uint32_t_out = FLAMEGPU->agent("agent").count<double>("double", 0);
}
FLAMEGPU_STEP_FUNCTION(step_countchar) {
    uint32_t_out = FLAMEGPU->agent("agent").count<char>("char", 0);
}
FLAMEGPU_STEP_FUNCTION(step_countuchar) {
    uint32_t_out = FLAMEGPU->agent("agent").count<unsigned char>("uchar", 0);
}
FLAMEGPU_STEP_FUNCTION(step_countint16_t) {
    uint32_t_out = FLAMEGPU->agent("agent").count<int16_t>("int16_t", 0);
}
FLAMEGPU_STEP_FUNCTION(step_countuint16_t) {
    uint32_t_out = FLAMEGPU->agent("agent").count<uint16_t>("uint16_t", 0);
}
FLAMEGPU_STEP_FUNCTION(step_countint32_t) {
    uint32_t_out = FLAMEGPU->agent("agent").count<int32_t>("int32_t", 0);
}
FLAMEGPU_STEP_FUNCTION(step_countuint32_t) {
    uint32_t_out = FLAMEGPU->agent("agent").count<uint32_t>("uint32_t", 0);
}
FLAMEGPU_STEP_FUNCTION(step_countint64_t) {
    uint32_t_out = FLAMEGPU->agent("agent").count<int64_t>("int64_t", 0);
}
FLAMEGPU_STEP_FUNCTION(step_countuint64_t) {
    uint32_t_out = FLAMEGPU->agent("agent").count<uint64_t>("uint64_t", 0);
}

FLAMEGPU_CUSTOM_REDUCTION(customSum, a, b) {
    return a + b;
}
FLAMEGPU_CUSTOM_TRANSFORM(customTransform, a) {
    return a <= 0 ? 1 : 0;
}
FLAMEGPU_STEP_FUNCTION(step_transformReduceFloat) {
    uint32_t_out = FLAMEGPU->agent("agent").transformReduce<float, uint32_t>("float", customTransform, customSum, 0);
}
FLAMEGPU_STEP_FUNCTION(step_transformReduceDouble) {
    int32_t_out = FLAMEGPU->agent("agent").transformReduce<double, int32_t>("double", customTransform, customSum, 0);
}
FLAMEGPU_STEP_FUNCTION(step_transformReducechar) {
    int32_t_out = FLAMEGPU->agent("agent").transformReduce<char, int32_t>("char", customTransform, customSum, 0);
}
FLAMEGPU_STEP_FUNCTION(step_transformReduceuchar) {
    int32_t_out = FLAMEGPU->agent("agent").transformReduce<unsigned char, int32_t>("uchar", customTransform, customSum, 0);
}
FLAMEGPU_STEP_FUNCTION(step_transformReduceint16_t) {
    int32_t_out = FLAMEGPU->agent("agent").transformReduce<int16_t, int32_t>("int16_t", customTransform, customSum, 0);
}
FLAMEGPU_STEP_FUNCTION(step_transformReduceuint16_t) {
    int32_t_out = FLAMEGPU->agent("agent").transformReduce<uint16_t, int32_t>("uint16_t", customTransform, customSum, 0);
}
FLAMEGPU_STEP_FUNCTION(step_transformReduceint32_t) {
    int32_t_out = FLAMEGPU->agent("agent").transformReduce<int32_t, int32_t>("int32_t", customTransform, customSum, 0);
}
FLAMEGPU_STEP_FUNCTION(step_transformReduceuint32_t) {
    int32_t_out = FLAMEGPU->agent("agent").transformReduce<uint32_t, int32_t>("uint32_t", customTransform, customSum, 0);
}
FLAMEGPU_STEP_FUNCTION(step_transformReduceint64_t) {
    int32_t_out = FLAMEGPU->agent("agent").transformReduce<int64_t, int32_t>("int64_t", customTransform, customSum, 0);
}
FLAMEGPU_STEP_FUNCTION(step_transformReduceuint64_t) {
    int32_t_out = FLAMEGPU->agent("agent").transformReduce<uint64_t, int32_t>("uint64_t", customTransform, customSum, 0);
}
FLAMEGPU_STEP_FUNCTION(step_sumException) {
    EXPECT_THROW(FLAMEGPU->agent("agedddnt"), InvalidCudaAgent);
    EXPECT_THROW(FLAMEGPU->agent("agent").sum<unsigned char>("float"), InvalidVarType);
    EXPECT_THROW(FLAMEGPU->agent("agent").sum<int64_t>("uint64_t"), InvalidVarType);
    EXPECT_THROW(FLAMEGPU->agent("agent").sum<double>("intsssssssss16_t"), InvalidAgentVar);
    EXPECT_THROW(FLAMEGPU->agent("agent").sum<uint64_t>("isssssssssssnt"), InvalidAgentVar);
}
FLAMEGPU_STEP_FUNCTION(step_minException) {
    EXPECT_THROW(FLAMEGPU->agent("agsssedddnt"), InvalidCudaAgent);
    EXPECT_THROW(FLAMEGPU->agent("agent").min<uint64_t>("char"), InvalidVarType);
    EXPECT_THROW(FLAMEGPU->agent("agent").min<int64_t>("uint64_t"), InvalidVarType);
    EXPECT_THROW(FLAMEGPU->agent("agent").min<double>("intssssssssssssss16_t"), InvalidAgentVar);
    EXPECT_THROW(FLAMEGPU->agent("agent").min<uint64_t>("issssssssssssnt"), InvalidAgentVar);
}
FLAMEGPU_STEP_FUNCTION(step_maxException) {
    EXPECT_THROW(FLAMEGPU->agent("ageaadddnt"), InvalidCudaAgent);
    EXPECT_THROW(FLAMEGPU->agent("agent").max<double>("float"), InvalidVarType);
    EXPECT_THROW(FLAMEGPU->agent("agent").max<float>("uint64_t"), InvalidVarType);
    EXPECT_THROW(FLAMEGPU->agent("agent").max<double>("intsssssssssss16_t"), InvalidAgentVar);
    EXPECT_THROW(FLAMEGPU->agent("agent").max<uint64_t>("ssssssssssssssint"), InvalidAgentVar);
}
FLAMEGPU_STEP_FUNCTION(step_customReductionException) {
    EXPECT_THROW(FLAMEGPU->agent("ageaadddnt"), InvalidCudaAgent);
    EXPECT_THROW(FLAMEGPU->agent("agent").reduce<double>("float", customMax, 0), InvalidVarType);
    EXPECT_THROW(FLAMEGPU->agent("agent").reduce<float>("uint64_t", customMax, 0), InvalidVarType);
    EXPECT_THROW(FLAMEGPU->agent("agent").reduce<double>("intsssssssssss16_t", customMax, 0), InvalidAgentVar);
    EXPECT_THROW(FLAMEGPU->agent("agent").reduce<uint64_t>("ssssssssssssssint", customMax, 0), InvalidAgentVar);
}
FLAMEGPU_STEP_FUNCTION(step_histogramEvenException) {
    EXPECT_THROW(FLAMEGPU->agent("ageaadddnt"), InvalidCudaAgent);
    EXPECT_THROW(FLAMEGPU->agent("agent").histogramEven<double>("float", 10, 0, 10), InvalidVarType);
    EXPECT_THROW(FLAMEGPU->agent("agent").histogramEven<float>("uint64_t", 10, 0, 10), InvalidVarType);
    EXPECT_THROW(FLAMEGPU->agent("agent").histogramEven<double>("intsssssssssss16_t", 10, 0, 10), InvalidAgentVar);
    EXPECT_THROW(FLAMEGPU->agent("agent").histogramEven<uint64_t>("ssssssssssssssint", 10, 0, 10), InvalidAgentVar);
    EXPECT_THROW(FLAMEGPU->agent("agent").histogramEven<int>("int", 10, 0, 0), InvalidArgument);
    EXPECT_THROW(FLAMEGPU->agent("agent").histogramEven<double>("double", 10, 11, 10), InvalidArgument);
}
FLAMEGPU_STEP_FUNCTION(step_transformReduceException) {
    EXPECT_THROW(FLAMEGPU->agent("ageaadddnt"), InvalidCudaAgent);
    EXPECT_THROW(FLAMEGPU->agent("agent").transformReduce<int32_t>("uint16_t", customTransform, customSum, 0), InvalidVarType);
    EXPECT_THROW(FLAMEGPU->agent("agent").transformReduce<float>("uint64_t", customTransform, customSum, 0), InvalidVarType);
    EXPECT_THROW(FLAMEGPU->agent("agent").transformReduce<double>("intsssssssssss16_t", customTransform, customSum, 0), InvalidAgentVar);
    EXPECT_THROW(FLAMEGPU->agent("agent").transformReduce<uint64_t>("ssssssssssssssint", customTransform, customSum, 0), InvalidAgentVar);
}
FLAMEGPU_STEP_FUNCTION(step_countException) {
    EXPECT_THROW(FLAMEGPU->agent("ageaadddnt"), InvalidCudaAgent);
    EXPECT_THROW(FLAMEGPU->agent("agent").count<int32_t>("double", 0), InvalidVarType);
    EXPECT_THROW(FLAMEGPU->agent("agent").count<float>("uint64_t", 0), InvalidVarType);
    EXPECT_THROW(FLAMEGPU->agent("agent").count<double>("intsssssssssss16_t", 0), InvalidAgentVar);
    EXPECT_THROW(FLAMEGPU->agent("agent").count<uint64_t>("ssssssssssssssint", 0), InvalidAgentVar);
}

class MiniSim {
 public:
    MiniSim() :
      model("model"),
      agent("agent"),
      simulation(model),
      population(nullptr) {
        agent.addAgentVariable<float>("float");
        agent.addAgentVariable<double>("double");
        agent.addAgentVariable<char>("char");
        agent.addAgentVariable<unsigned char>("uchar");
        agent.addAgentVariable<uint16_t>("uint16_t");
        agent.addAgentVariable<int16_t>("int16_t");
        agent.addAgentVariable<uint32_t>("uint32_t");
        agent.addAgentVariable<int32_t>("int32_t");
        agent.addAgentVariable<uint64_t>("uint64_t");
        agent.addAgentVariable<int64_t>("int64_t");
        population = new AgentPopulation(agent, TEST_LEN);
        simulation.setSimulationSteps(1);
    }
    void run() {
        model.addAgent(agent);
        // CudaModel must be declared here
        // As the initial call to constructor fixes the agent population
        // This means if we haven't called model.addAgent(agent) first
        CUDAAgentModel cuda_model(model);
        // This fails as agentMap is empty
        cuda_model.setInitialPopulationData(*population);
        ASSERT_NO_THROW(cuda_model.simulate(simulation));
        // The negative of this, is that cuda_model is inaccessible within the test!
        // So copy across population data here
        ASSERT_NO_THROW(cuda_model.getPopulationData(*population));
    }
    ModelDescription model;
    AgentDescription agent;
    Simulation simulation;
    AgentPopulation *population;
};
/**
* This defines a common fixture used as a base for all test cases in the file
* @see https://github.com/google/googletest/blob/master/googletest/samples/sample5_unittest.cc
*/
class HostReductionTest : public testing::Test {
 protected:
    void SetUp() override {
        ms = new MiniSim();
        uint_vec.clear();
        int_vec.clear();
    }

    void TearDown() override {
        delete ms;
    }

    MiniSim *ms = nullptr;
};

/**
 * Poor attempt to mimic hipcub::histogram::histogramEven()
 * Doesn't work great with odd boundaries and ranges near integer limits
 */
template<typename InT, typename OutT>
std::vector<OutT> histogramEven(const std::array<InT, TEST_LEN> &variables, const unsigned int &histogramBins, const InT &lowerBound, const InT &upperBound) {
    assert(upperBound > lowerBound);
    std::vector<OutT> rtn(histogramBins);
    for (auto &i : rtn)
        i = static_cast<OutT>(0);
    const InT diff = upperBound - lowerBound;
    const double diffP = diff / static_cast<double>(histogramBins);
    for (auto &i : variables) {
        if (i >= lowerBound && i < upperBound) {
            ++rtn[static_cast<int>(i/ diffP)];
        }
    }
    return rtn;
}
}  // namespace

/**
 * Float
 */
TEST_F(HostReductionTest, MinFloat) {
    ms->simulation.addStepFunction(&step_minfloat);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <float> dist(FLT_MIN, FLT_MAX);
    std::array<float, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<float>("float", in[i]);
    }
    ms->run();
    EXPECT_EQ(float_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxFloat) {
    ms->simulation.addStepFunction(&step_maxfloat);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <float> dist(FLT_MIN, FLT_MAX);
    std::array<float, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<float>("float", in[i]);
    }
    ms->run();
    EXPECT_EQ(float_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumFloat) {
    ms->simulation.addStepFunction(&step_sumfloat);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <float> dist(FLT_MIN, FLT_MAX);
    std::array<float, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<float>("float", in[i]);
    }
    ms->run();
    EXPECT_EQ(float_out, std::accumulate(in.begin(), in.end(), 0.0f));
}
TEST_F(HostReductionTest, CustomReduceFloat) {
    ms->simulation.addStepFunction(&step_reducefloat);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <float> dist(FLT_MIN, FLT_MAX);
    std::array<float, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<float>("float", in[i]);
    }
    ms->run();
    EXPECT_EQ(float_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, HistogramEvenFloat) {
    ms->simulation.addStepFunction(&step_histogramEvenfloat);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <float> dist(0, 20);
    std::array<float, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<float>("float", in[i]);
    }
    ms->run();
    auto check = histogramEven<float, unsigned int>(in, 10, 0, 20);
    for (unsigned int i = 0; i < uint_vec.size(); ++i) {
        EXPECT_EQ(uint_vec[i], check[i]);
    }
}
TEST_F(HostReductionTest, CustomTransformReduceFloat) {
    ms->simulation.addStepFunction(&step_transformReduceFloat);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <float> dist(FLT_MIN, FLT_MAX);
    std::array<float, TEST_LEN> in;
    std::array<int, TEST_LEN> inTransform;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<float>("float", in[i]);
    }
    ms->run();
    std::transform(in.begin(), in.end(), inTransform.begin(), customTransform_impl::unary_function<float, uint32_t>());
    EXPECT_EQ(uint32_t_out, std::count(inTransform.begin(), inTransform.end(), static_cast<uint32_t>(1)));
}
TEST_F(HostReductionTest, CountFloat) {
    ms->simulation.addStepFunction(&step_countfloat);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <float> dist(FLT_MIN, FLT_MAX);
    std::array<float, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < TEST_LEN/2) {
            in[i] = dist(rd);
        } else {
            in[i] = 0;
        }
        instance.setVariable<float>("float", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, std::count(in.begin(), in.end(), static_cast<float>(0)));
}

/**
 * Double
 */
TEST_F(HostReductionTest, MinDouble) {
    ms->simulation.addStepFunction(&step_mindouble);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <double> dist(DBL_MIN, DBL_MAX);
    std::array<double, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<double>("double", in[i]);
    }
    ms->run();
    EXPECT_EQ(double_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxDouble) {
    ms->simulation.addStepFunction(&step_maxdouble);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <double> dist(DBL_MIN, DBL_MAX);
    std::array<double, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<double>("double", in[i]);
    }
    ms->run();
    EXPECT_EQ(double_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumDouble) {
    ms->simulation.addStepFunction(&step_sumdouble);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <double> dist(DBL_MIN, DBL_MAX);
    std::array<double, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<double>("double", in[i]);
    }
    ms->run();
    EXPECT_EQ(double_out, std::accumulate(in.begin(), in.end(), 0.0));
}
TEST_F(HostReductionTest, CustomReduceDouble) {
    ms->simulation.addStepFunction(&step_reducedouble);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <double> dist(DBL_MIN, DBL_MAX);
    std::array<double, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<double>("double", in[i]);
    }
    ms->run();
    EXPECT_EQ(double_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, HistogramEvenDouble) {
    ms->simulation.addStepFunction(&step_histogramEvendouble);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <double> dist(0, 20);
    std::array<double, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<double>("double", in[i]);
    }
    ms->run();
    auto check = histogramEven<double, int>(in, 10, 0, 20);
    for (unsigned int i = 0; i < int_vec.size(); ++i) {
        EXPECT_EQ(int_vec[i], check[i]);
    }
}
TEST_F(HostReductionTest, CustomTransformReduceDouble) {
    ms->simulation.addStepFunction(&step_transformReduceDouble);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <double> dist(DBL_MIN, DBL_MAX);
    std::array<double, TEST_LEN> in;
    std::array<int, TEST_LEN> inTransform;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<double>("double", in[i]);
    }
    ms->run();
    std::transform(in.begin(), in.end(), inTransform.begin(), customTransform_impl::unary_function<double, int>());
    EXPECT_EQ(int32_t_out, std::count(inTransform.begin(), inTransform.end(), static_cast<int>(1)));
}
TEST_F(HostReductionTest, CountDouble) {
    ms->simulation.addStepFunction(&step_countdouble);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <double> dist(DBL_MIN, DBL_MAX);
    std::array<double, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < TEST_LEN / 2) {
            in[i] = dist(rd);
        } else {
            in[i] = 0;
        }
        instance.setVariable<double>("double", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, std::count(in.begin(), in.end(), static_cast<double>(0)));
}

/**
 * Char
 */
TEST_F(HostReductionTest, MinChar) {
    ms->simulation.addStepFunction(&step_minchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(CHAR_MIN, CHAR_MAX);
    std::array<char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = static_cast<char>(dist(rd));
        instance.setVariable<char>("char", in[i]);
    }
    ms->run();
    EXPECT_EQ(char_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxChar) {
    ms->simulation.addStepFunction(&step_maxchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(CHAR_MIN, CHAR_MAX);
    std::array<char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < 256) {
            in[i] = static_cast<char>(dist(rd));
        } else {
            in[i] = 0;
        }
        instance.setVariable<char>("char", in[i]);
    }
    ms->run();
    EXPECT_EQ(char_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumChar) {
    ms->simulation.addStepFunction(&step_sumchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(CHAR_MIN, CHAR_MAX);
    std::array<char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < 256) {
            in[i] = static_cast<char>(dist(rd));
        } else {
            in[i] = 0;
        }
        instance.setVariable<char>("char", in[i]);
    }
    ms->run();
    EXPECT_EQ(char_out, std::accumulate(in.begin(), in.end(), static_cast<char>(0)));
    EXPECT_EQ(int64_t_out, std::accumulate(in.begin(), in.end(), static_cast<int64_t>(0)));
}
TEST_F(HostReductionTest, CustomReduceChar) {
    ms->simulation.addStepFunction(&step_reducechar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(CHAR_MIN, CHAR_MAX);
    std::array<char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < 256) {
            in[i] = static_cast<char>(dist(rd));
        } else {
            in[i] = 0;
        }
        instance.setVariable<char>("char", in[i]);
    }
    ms->run();
    EXPECT_EQ(char_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, HistogramEvenChar) {
    ms->simulation.addStepFunction(&step_histogramEvenchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(0, 19);
    std::array<char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < 256) {
            in[i] = static_cast<char>(dist(rd));
        } else {
            in[i] = 0;
        }
        instance.setVariable<char>("char", in[i]);
    }
    ms->run();
    auto check = histogramEven<char, unsigned int>(in, 10, 0, 20);
    for (unsigned int i = 0; i < uint_vec.size(); ++i) {
        EXPECT_EQ(uint_vec[i], check[i]);
    }
}
TEST_F(HostReductionTest, CustomTransformReduceChar) {
    ms->simulation.addStepFunction(&step_transformReducechar);
    std::array<int, TEST_LEN> inTransform;
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(CHAR_MIN, CHAR_MAX);
    std::array<char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < 256) {
            in[i] = static_cast<char>(dist(rd));
        } else {
            in[i] = 0;
        }
        instance.setVariable<char>("char", in[i]);
    }
    ms->run();
    std::transform(in.begin(), in.end(), inTransform.begin(), customTransform_impl::unary_function<char, int>());
    EXPECT_EQ(int32_t_out, std::count(inTransform.begin(), inTransform.end(), static_cast<int>(1)));
}
TEST_F(HostReductionTest, CountChar) {
    ms->simulation.addStepFunction(&step_countchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(CHAR_MIN, CHAR_MAX);
    std::array<char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < TEST_LEN / 2) {
            in[i] = static_cast<char>(dist(rd));
        } else {
            in[i] = 0;
        }
        instance.setVariable<char>("char", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, std::count(in.begin(), in.end(), static_cast<char>(0)));
}

/**
 * Unsigned Char
 */
TEST_F(HostReductionTest, MinUnsignedChar) {
    ms->simulation.addStepFunction(&step_minuchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UCHAR_MAX);
    std::array<unsigned char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = static_cast<unsigned char>(dist(rd));
        instance.setVariable<unsigned char>("uchar", in[i]);
    }
    ms->run();
    EXPECT_EQ(uchar_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxUnsignedChar) {
    ms->simulation.addStepFunction(&step_maxuchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UCHAR_MAX);
    std::array<unsigned char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = static_cast<unsigned char>(dist(rd));
        instance.setVariable<unsigned char>("uchar", in[i]);
    }
    ms->run();
    EXPECT_EQ(uchar_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumUnsignedChar) {
    ms->simulation.addStepFunction(&step_sumuchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UCHAR_MAX);
    std::array<unsigned char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = static_cast<unsigned char>(dist(rd));
        instance.setVariable<unsigned char>("uchar", in[i]);
    }
    ms->run();
    EXPECT_EQ(uchar_out, std::accumulate(in.begin(), in.end(), static_cast<unsigned char>(0)));
    EXPECT_EQ(uint64_t_out, std::accumulate(in.begin(), in.end(), static_cast<uint64_t>(0)));
}
TEST_F(HostReductionTest, CustomReduceUnsignedChar) {
    ms->simulation.addStepFunction(&step_reduceuchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UCHAR_MAX);
    std::array<unsigned char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = static_cast<unsigned char>(dist(rd));
        instance.setVariable<unsigned char>("uchar", in[i]);
    }
    ms->run();
    EXPECT_EQ(uchar_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, HistogramEvenUnsignedChar) {
    ms->simulation.addStepFunction(&step_histogramEvenuchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, 19);
    std::array<unsigned char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = static_cast<unsigned char>(dist(rd));
        instance.setVariable<unsigned char>("uchar", in[i]);
    }
    ms->run();
    auto check = histogramEven<unsigned char, int>(in, 10, 0, 20);
    for (unsigned int i = 0; i < int_vec.size(); ++i) {
        EXPECT_EQ(int_vec[i], check[i]);
    }
}
TEST_F(HostReductionTest, CustomTransformReduceUnsignedChar) {
    ms->simulation.addStepFunction(&step_transformReduceuchar);
    std::array<int, TEST_LEN> inTransform;
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UCHAR_MAX);
    std::array<unsigned char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = static_cast<unsigned char>(dist(rd));
        instance.setVariable<unsigned char>("uchar", in[i]);
    }
    ms->run();
    std::transform(in.begin(), in.end(), inTransform.begin(), customTransform_impl::unary_function<unsigned char, int>());
    EXPECT_EQ(int32_t_out, std::count(inTransform.begin(), inTransform.end(), static_cast<int>(1)));
}
TEST_F(HostReductionTest, CountUnsignedChar) {
    ms->simulation.addStepFunction(&step_countuchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UCHAR_MAX);
    std::array<unsigned char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < TEST_LEN / 2) {
            in[i] = static_cast<unsigned char>(dist(rd));
        } else {
            in[i] = 0;
        }
        instance.setVariable<unsigned char>("uchar", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, std::count(in.begin(), in.end(), static_cast<unsigned char>(0)));
}

/**
 * int16_t
 */
TEST_F(HostReductionTest, MinInt16) {
    ms->simulation.addStepFunction(&step_minint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(INT16_MIN, INT16_MAX);
    std::array<int16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int16_t>("int16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int16_t_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxInt16) {
    ms->simulation.addStepFunction(&step_maxint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(INT16_MIN, INT16_MAX);
    std::array<int16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int16_t>("int16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int16_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumInt16) {
    ms->simulation.addStepFunction(&step_sumint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(INT16_MIN, INT16_MAX);
    std::array<int16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int16_t>("int16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int16_t_out, std::accumulate(in.begin(), in.end(), static_cast<int16_t>(0)));
    EXPECT_EQ(int64_t_out, std::accumulate(in.begin(), in.end(), static_cast<int64_t>(0)));
}
TEST_F(HostReductionTest, CustomReduceInt16) {
    ms->simulation.addStepFunction(&step_reduceint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(INT16_MIN, INT16_MAX);
    std::array<int16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int16_t>("int16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int16_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, HistogramEvenInt16) {
    ms->simulation.addStepFunction(&step_histogramEvenint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(0, 19);
    std::array<int16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int16_t>("int16_t", in[i]);
    }
    ms->run();
    auto check = histogramEven<int16_t, unsigned int>(in, 10, 0, 20);
    for (unsigned int i = 0; i < uint_vec.size(); ++i) {
        EXPECT_EQ(uint_vec[i], check[i]);
    }
}
TEST_F(HostReductionTest, CustomTransformReduceInt16) {
    ms->simulation.addStepFunction(&step_transformReduceint16_t);
    std::array<int, TEST_LEN> inTransform;
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(INT16_MIN, INT16_MAX);
    std::array<int16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int16_t>("int16_t", in[i]);
    }
    ms->run();
    std::transform(in.begin(), in.end(), inTransform.begin(), customTransform_impl::unary_function<int16_t, int>());
    EXPECT_EQ(int32_t_out, std::count(inTransform.begin(), inTransform.end(), static_cast<int>(1)));
}
TEST_F(HostReductionTest, CountInt16) {
    ms->simulation.addStepFunction(&step_countint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(INT16_MIN, INT16_MAX);
    std::array<int16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < TEST_LEN / 2) {
            in[i] = dist(rd);
        } else {
            in[i] = 0;
        }
        instance.setVariable<int16_t>("int16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, std::count(in.begin(), in.end(), static_cast<int16_t>(0)));
}

/**
 * uint16_t
 */
TEST_F(HostReductionTest, MinUnsignedInt16) {
    ms->simulation.addStepFunction(&step_minuint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UINT16_MAX);
    std::array<uint16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint16_t>("uint16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint16_t_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxUnsignedInt16) {
    ms->simulation.addStepFunction(&step_maxuint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UINT16_MAX);
    std::array<uint16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint16_t>("uint16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint16_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumUnsignedInt16) {
    ms->simulation.addStepFunction(&step_sumuint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UINT16_MAX);
    std::array<uint16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint16_t>("uint16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint16_t_out, std::accumulate(in.begin(), in.end(), static_cast<uint16_t>(0)));
    EXPECT_EQ(uint64_t_out, std::accumulate(in.begin(), in.end(), static_cast<uint64_t>(0)));
}
TEST_F(HostReductionTest, CustomReduceUnsignedInt16) {
    ms->simulation.addStepFunction(&step_reduceuint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UINT16_MAX);
    std::array<uint16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint16_t>("uint16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint16_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, HistogramEvenUnsignedInt16) {
    ms->simulation.addStepFunction(&step_histogramEvenuint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, 19);
    std::array<uint16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint16_t>("uint16_t", in[i]);
    }
    ms->run();
    auto check = histogramEven<uint16_t, int>(in, 10, 0, 20);
    for (unsigned int i = 0; i < int_vec.size(); ++i) {
        EXPECT_EQ(int_vec[i], check[i]);
    }
}
TEST_F(HostReductionTest, CustomTransformReduceUnsignedInt16) {
    ms->simulation.addStepFunction(&step_transformReduceuint16_t);
    std::array<int, TEST_LEN> inTransform;
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UINT16_MAX);
    std::array<uint16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint16_t>("uint16_t", in[i]);
    }
    ms->run();
    std::transform(in.begin(), in.end(), inTransform.begin(), customTransform_impl::unary_function<uint16_t, int>());
    EXPECT_EQ(int32_t_out, std::count(inTransform.begin(), inTransform.end(), static_cast<int>(1)));
}
TEST_F(HostReductionTest, CountUnsignedInt16) {
    ms->simulation.addStepFunction(&step_countuint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UINT16_MAX);
    std::array<uint16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < TEST_LEN / 2) {
            in[i] = dist(rd);
        } else {
            in[i] = 0;
        }
        instance.setVariable<uint16_t>("uint16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, std::count(in.begin(), in.end(), static_cast<uint16_t>(0)));
}

/**
 * int32_t
 */
TEST_F(HostReductionTest, MinInt32) {
    ms->simulation.addStepFunction(&step_minint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int32_t> dist(INT32_MIN, INT32_MAX);
    std::array<int32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int32_t>("int32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int32_t_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxInt32) {
    ms->simulation.addStepFunction(&step_maxint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int32_t> dist(INT32_MIN, INT32_MAX);
    std::array<int32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int32_t>("int32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int32_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumInt32) {
    ms->simulation.addStepFunction(&step_sumint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int32_t> dist(INT32_MIN, INT32_MAX);
    std::array<int32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int32_t>("int32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int32_t_out, std::accumulate(in.begin(), in.end(), static_cast<int32_t>(0)));
    EXPECT_EQ(int64_t_out, std::accumulate(in.begin(), in.end(), static_cast<int64_t>(0)));
}
TEST_F(HostReductionTest, CustomReduceInt32) {
    ms->simulation.addStepFunction(&step_reduceint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int32_t> dist(INT32_MIN, INT32_MAX);
    std::array<int32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int32_t>("int32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int32_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, HistogramEvenInt32) {
    ms->simulation.addStepFunction(&step_histogramEvenint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int32_t> dist(0, 19);
    std::array<int32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int32_t>("int32_t", in[i]);
    }
    ms->run();
    auto check = histogramEven<int32_t, unsigned int>(in, 10, 0, 20);
    for (unsigned int i = 0; i < uint_vec.size(); ++i) {
        EXPECT_EQ(uint_vec[i], check[i]);
    }
}
TEST_F(HostReductionTest, CustomTransformReduceInt32) {
    ms->simulation.addStepFunction(&step_transformReduceint32_t);
    std::array<int, TEST_LEN> inTransform;
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int32_t> dist(INT32_MIN, INT32_MAX);
    std::array<int32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int32_t>("int32_t", in[i]);
    }
    ms->run();
    std::transform(in.begin(), in.end(), inTransform.begin(), customTransform_impl::unary_function<int32_t, int>());
    EXPECT_EQ(int32_t_out, std::count(inTransform.begin(), inTransform.end(), static_cast<int>(1)));
}
TEST_F(HostReductionTest, CountInt32) {
    ms->simulation.addStepFunction(&step_countint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int32_t> dist(INT32_MIN, INT32_MAX);
    std::array<int32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < TEST_LEN / 2) {
            in[i] = dist(rd);
        } else {
            in[i] = 0;
        }
        instance.setVariable<int32_t>("int32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, std::count(in.begin(), in.end(), static_cast<int32_t>(0)));
}

/**
 * uint32_t
 */
TEST_F(HostReductionTest, MinUnsignedInt32) {
    ms->simulation.addStepFunction(&step_minuint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint32_t> dist(0, UINT32_MAX);
    std::array<uint32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint32_t>("uint32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxUnsignedInt32) {
    ms->simulation.addStepFunction(&step_maxuint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint32_t> dist(0, UINT32_MAX);
    std::array<uint32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint32_t>("uint32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumUnsignedInt32) {
    ms->simulation.addStepFunction(&step_sumuint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint32_t> dist(0, UINT32_MAX);
    std::array<uint32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint32_t>("uint32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, std::accumulate(in.begin(), in.end(), static_cast<uint32_t>(0)));
    EXPECT_EQ(uint64_t_out, std::accumulate(in.begin(), in.end(), static_cast<uint64_t>(0)));
}
TEST_F(HostReductionTest, CustomReduceUnsignedInt32) {
    ms->simulation.addStepFunction(&step_reduceuint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint32_t> dist(0, UINT32_MAX);
    std::array<uint32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint32_t>("uint32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, HistogramEvenUnsignedInt32) {
    ms->simulation.addStepFunction(&step_histogramEvenuint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint32_t> dist(0, 19);
    std::array<uint32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint32_t>("uint32_t", in[i]);
    }
    ms->run();
    auto check = histogramEven<uint32_t, int>(in, 10, 0, 20);
    for (unsigned int i = 0; i < int_vec.size(); ++i) {
        EXPECT_EQ(int_vec[i], check[i]);
    }
}
TEST_F(HostReductionTest, CustomTransformReduceUnsignedInt32) {
    ms->simulation.addStepFunction(&step_transformReduceuint32_t);
    std::array<int, TEST_LEN> inTransform;
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint32_t> dist(0, UINT32_MAX);
    std::array<uint32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint32_t>("uint32_t", in[i]);
    }
    ms->run();
    std::transform(in.begin(), in.end(), inTransform.begin(), customTransform_impl::unary_function<uint32_t, int>());
    EXPECT_EQ(int32_t_out, std::count(inTransform.begin(), inTransform.end(), static_cast<int>(1)));
}
TEST_F(HostReductionTest, CountUnsignedInt32) {
    ms->simulation.addStepFunction(&step_countuint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint32_t> dist(0, UINT32_MAX);
    std::array<uint32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < TEST_LEN / 2) {
            in[i] = dist(rd);
        } else {
            in[i] = 0;
        }
        instance.setVariable<uint32_t>("uint32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, std::count(in.begin(), in.end(), static_cast<uint32_t>(0)));
}

/**
 * int64_t
 */
TEST_F(HostReductionTest, MinInt64) {
    ms->simulation.addStepFunction(&step_minint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int64_t> dist(INT64_MIN, INT64_MAX);
    std::array<int64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int64_t>("int64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int64_t_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxInt64) {
    ms->simulation.addStepFunction(&step_maxint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int64_t> dist(INT64_MIN, INT64_MAX);
    std::array<int64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int64_t>("int64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int64_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumInt64) {
    ms->simulation.addStepFunction(&step_sumint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int64_t> dist(INT64_MIN, INT64_MAX);
    std::array<int64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int64_t>("int64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int64_t_out, std::accumulate(in.begin(), in.end(), 0ll));
}
TEST_F(HostReductionTest, CustomReduceInt64) {
    ms->simulation.addStepFunction(&step_reduceint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int64_t> dist(INT64_MIN, INT64_MAX);
    std::array<int64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int64_t>("int64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int64_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, HistogramEvenInt64) {
    ms->simulation.addStepFunction(&step_histogramEvenint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int64_t> dist(0, 19);
    std::array<int64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int64_t>("int64_t", in[i]);
    }
    ms->run();
    auto check = histogramEven<int64_t, unsigned int>(in, 10, 0, 20);
    for (unsigned int i = 0; i < uint_vec.size(); ++i) {
        EXPECT_EQ(uint_vec[i], check[i]);
    }
}
TEST_F(HostReductionTest, CustomTransformReduceInt64) {
    ms->simulation.addStepFunction(&step_transformReduceint64_t);
    std::array<int, TEST_LEN> inTransform;
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int64_t> dist(INT64_MIN, INT64_MAX);
    std::array<int64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int64_t>("int64_t", in[i]);
    }
    ms->run();
    std::transform(in.begin(), in.end(), inTransform.begin(), customTransform_impl::unary_function<int64_t, int>());
    EXPECT_EQ(int32_t_out, std::count(inTransform.begin(), inTransform.end(), static_cast<int>(1)));
}
TEST_F(HostReductionTest, CountInt64) {
    ms->simulation.addStepFunction(&step_countint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int64_t> dist(INT64_MIN, INT64_MAX);
    std::array<int64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < TEST_LEN / 2) {
            in[i] = dist(rd);
        } else {
            in[i] = 0;
        }
        instance.setVariable<int64_t>("int64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, std::count(in.begin(), in.end(), static_cast<int64_t>(0)));
}

/**
 * uint64_t
 */
TEST_F(HostReductionTest, MinUnsignedInt64) {
    ms->simulation.addStepFunction(&step_minuint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint64_t> dist(0, UINT64_MAX);
    std::array<uint64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint64_t>("uint64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint64_t_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxUnsignedInt64) {
    ms->simulation.addStepFunction(&step_maxuint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint64_t> dist(0, UINT64_MAX);
    std::array<uint64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint64_t>("uint64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint64_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumUnsignedInt64) {
    ms->simulation.addStepFunction(&step_sumuint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint64_t> dist(0, UINT64_MAX);
    std::array<uint64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint64_t>("uint64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint64_t_out, std::accumulate(in.begin(), in.end(), 0llu));
}
TEST_F(HostReductionTest, CustomReduceUnsignedInt64) {
    ms->simulation.addStepFunction(&step_reduceuint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint64_t> dist(0, UINT64_MAX);
    std::array<uint64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint64_t>("uint64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint64_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, HistogramEvenUnsignedInt64) {
    ms->simulation.addStepFunction(&step_histogramEvenuint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint64_t> dist(0, 19);
    std::array<uint64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint64_t>("uint64_t", in[i]);
    }
    ms->run();
    auto check = histogramEven<uint64_t, int>(in, 10, 0, 20);
    for (unsigned int i = 0; i < int_vec.size(); ++i) {
        EXPECT_EQ(int_vec[i], check[i]);
    }
}
TEST_F(HostReductionTest, CustomTransformReduceUnsignedInt64) {
    ms->simulation.addStepFunction(&step_transformReduceuint64_t);
    std::array<int, TEST_LEN> inTransform;
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint64_t> dist(0, UINT64_MAX);
    std::array<uint64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint64_t>("uint64_t", in[i]);
    }
    ms->run();
    std::transform(in.begin(), in.end(), inTransform.begin(), customTransform_impl::unary_function<uint64_t, int>());
    EXPECT_EQ(int32_t_out, std::count(inTransform.begin(), inTransform.end(), static_cast<int>(1)));
}
TEST_F(HostReductionTest, CountUnsignedInt64) {
    ms->simulation.addStepFunction(&step_countuint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint64_t> dist(0, UINT64_MAX);
    std::array<uint64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < TEST_LEN / 2) {
            in[i] = dist(rd);
        } else {
            in[i] = 0;
        }
        instance.setVariable<uint64_t>("uint64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, std::count(in.begin(), in.end(), static_cast<uint64_t>(0)));
}

/**
 * Bad Types
 */
TEST_F(HostReductionTest, SumException) {
    ms->simulation.addStepFunction(&step_sumException);
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
    }
    ms->run();
}
TEST_F(HostReductionTest, MinException) {
    ms->simulation.addStepFunction(&step_minException);
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
    }
    ms->run();
}
TEST_F(HostReductionTest, MaxException) {
    ms->simulation.addStepFunction(&step_maxException);
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
    }
    ms->run();
}
TEST_F(HostReductionTest, CustomReductionException) {
    ms->simulation.addStepFunction(&step_customReductionException);
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
    }
    ms->run();
}
TEST_F(HostReductionTest, HistogramEvenException) {
    ms->simulation.addStepFunction(&step_histogramEvenException);
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
    }
    ms->run();
}
TEST_F(HostReductionTest, CustomTransformException) {
    ms->simulation.addStepFunction(&step_transformReduceException);
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
    }
    ms->run();
}
TEST_F(HostReductionTest, CountException) {
    ms->simulation.addStepFunction(&step_countException);
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
    }
    ms->run();
}

#endif  // TESTS_TEST_CASES_RUNTIME_TEST_HOST_REDUCTIONS_H_
