#include "hip/hip_runtime.h"
#include <numeric>
#ifndef TESTS_TEST_CASES_RUNTIME_TEST_HOST_REDUCTIONS_H_
#define TESTS_TEST_CASES_RUNTIME_TEST_HOST_REDUCTIONS_H_

#include <array>
#include <random>

#include "gtest/gtest.h"

#include "flamegpu/flame_api.h"
#include "flamegpu/runtime/flamegpu_api.h"

namespace {
const unsigned int TEST_LEN = 256;
float float_out = 0;
double double_out = 0;
char char_out = 0;
unsigned char uchar_out = 0;
uint16_t uint16_t_out = 0;
int16_t int16_t_out = 0;
uint32_t uint32_t_out = 0;
int32_t int32_t_out = 0;
uint64_t uint64_t_out = 0;
int64_t int64_t_out = 0;
FLAMEGPU_STEP_FUNCTION(step_minfloat) {
    float_out = FLAMEGPU->agent("agent").min<float>("float");
}
FLAMEGPU_STEP_FUNCTION(step_maxfloat) {
    float_out = FLAMEGPU->agent("agent").max<float>("float");
}
FLAMEGPU_STEP_FUNCTION(step_sumfloat) {
    float_out = FLAMEGPU->agent("agent").sum<float>("float");
}
FLAMEGPU_STEP_FUNCTION(step_mindouble) {
    double_out = FLAMEGPU->agent("agent").min<double>("double");
}
FLAMEGPU_STEP_FUNCTION(step_maxdouble) {
    double_out = FLAMEGPU->agent("agent").max<double>("double");
}
FLAMEGPU_STEP_FUNCTION(step_sumdouble) {
    double_out = FLAMEGPU->agent("agent").sum<double>("double");
}
FLAMEGPU_STEP_FUNCTION(step_minuchar) {
    uchar_out = FLAMEGPU->agent("agent").min<unsigned char>("uchar");
}
FLAMEGPU_STEP_FUNCTION(step_maxuchar) {
    uchar_out = FLAMEGPU->agent("agent").max<unsigned char>("uchar");
}
FLAMEGPU_STEP_FUNCTION(step_sumuchar) {
    uchar_out = FLAMEGPU->agent("agent").sum<unsigned char>("uchar");
    uint64_t_out = FLAMEGPU->agent("agent").sum<unsigned char, int64_t>("uchar");
}
FLAMEGPU_STEP_FUNCTION(step_minchar) {
    char_out = FLAMEGPU->agent("agent").min<char>("char");
}
FLAMEGPU_STEP_FUNCTION(step_maxchar) {
    char_out = FLAMEGPU->agent("agent").max<char>("char");
}
FLAMEGPU_STEP_FUNCTION(step_sumchar) {
    char_out = FLAMEGPU->agent("agent").sum<char>("char");
    int64_t_out = FLAMEGPU->agent("agent").sum<char, int64_t>("char");
}
FLAMEGPU_STEP_FUNCTION(step_minuint16_t) {
    uint16_t_out = FLAMEGPU->agent("agent").min<uint16_t>("uint16_t");
}
FLAMEGPU_STEP_FUNCTION(step_maxuint16_t) {
    uint16_t_out = FLAMEGPU->agent("agent").max<uint16_t>("uint16_t");
}
FLAMEGPU_STEP_FUNCTION(step_sumuint16_t) {
    uint16_t_out = FLAMEGPU->agent("agent").sum<uint16_t>("uint16_t");
    uint64_t_out = FLAMEGPU->agent("agent").sum<uint16_t, int64_t>("uint16_t");
}
FLAMEGPU_STEP_FUNCTION(step_minint16_t) {
    int16_t_out = FLAMEGPU->agent("agent").min<int16_t>("int16_t");
}
FLAMEGPU_STEP_FUNCTION(step_maxint16_t) {
    int16_t_out = FLAMEGPU->agent("agent").max<int16_t>("int16_t");
}
FLAMEGPU_STEP_FUNCTION(step_sumint16_t) {
    int16_t_out = FLAMEGPU->agent("agent").sum<int16_t>("int16_t");
    int64_t_out = FLAMEGPU->agent("agent").sum<int16_t, int64_t>("int16_t");
}
FLAMEGPU_STEP_FUNCTION(step_minuint32_t) {
    uint32_t_out = FLAMEGPU->agent("agent").min<uint32_t>("uint32_t");
}
FLAMEGPU_STEP_FUNCTION(step_maxuint32_t) {
    uint32_t_out = FLAMEGPU->agent("agent").max<uint32_t>("uint32_t");
}
FLAMEGPU_STEP_FUNCTION(step_sumuint32_t) {
    uint32_t_out = FLAMEGPU->agent("agent").sum<uint32_t>("uint32_t");
    uint64_t_out = FLAMEGPU->agent("agent").sum<uint32_t, int64_t>("uint32_t");
}
FLAMEGPU_STEP_FUNCTION(step_minint32_t) {
    int32_t_out = FLAMEGPU->agent("agent").min<int32_t>("int32_t");
}
FLAMEGPU_STEP_FUNCTION(step_maxint32_t) {
    int32_t_out = FLAMEGPU->agent("agent").max<int32_t>("int32_t");
}
FLAMEGPU_STEP_FUNCTION(step_sumint32_t) {
    int32_t_out = FLAMEGPU->agent("agent").sum<int32_t>("int32_t");
    int64_t_out = FLAMEGPU->agent("agent").sum<int32_t, int64_t>("int32_t");
}
FLAMEGPU_STEP_FUNCTION(step_minuint64_t) {
    uint64_t_out = FLAMEGPU->agent("agent").min<uint64_t>("uint64_t");
}
FLAMEGPU_STEP_FUNCTION(step_maxuint64_t) {
    uint64_t_out = FLAMEGPU->agent("agent").max<uint64_t>("uint64_t");
}
FLAMEGPU_STEP_FUNCTION(step_sumuint64_t) {
    uint64_t_out = FLAMEGPU->agent("agent").sum<uint64_t>("uint64_t");
}
FLAMEGPU_STEP_FUNCTION(step_minint64_t) {
    int64_t_out = FLAMEGPU->agent("agent").min<int64_t>("int64_t");
}
FLAMEGPU_STEP_FUNCTION(step_maxint64_t) {
    int64_t_out = FLAMEGPU->agent("agent").max<int64_t>("int64_t");
}
FLAMEGPU_STEP_FUNCTION(step_sumint64_t) {
    int64_t_out = FLAMEGPU->agent("agent").sum<int64_t>("int64_t");
}

FLAMEGPU_CUSTOM_REDUCTION(customMax, a, b) {
    return a > b ? a : b;
}
FLAMEGPU_STEP_FUNCTION(step_reducefloat) {
    float_out = FLAMEGPU->agent("agent").reduce<float>("float", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reducedouble) {
    double_out = FLAMEGPU->agent("agent").reduce<double>("double", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reduceuchar) {
    uchar_out = FLAMEGPU->agent("agent").reduce<unsigned char>("uchar", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reducechar) {
    char_out = FLAMEGPU->agent("agent").reduce<char>("char", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reduceuint16_t) {
    uint16_t_out = FLAMEGPU->agent("agent").reduce<uint16_t>("uint16_t", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reduceint16_t) {
    int16_t_out = FLAMEGPU->agent("agent").reduce<int16_t>("int16_t", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reduceuint32_t) {
    uint32_t_out = FLAMEGPU->agent("agent").reduce<uint32_t>("uint32_t", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reduceint32_t) {
    int32_t_out = FLAMEGPU->agent("agent").reduce<int32_t>("int32_t", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reduceuint64_t) {
    uint64_t_out = FLAMEGPU->agent("agent").reduce<uint64_t>("uint64_t", customMax, 0);
}
FLAMEGPU_STEP_FUNCTION(step_reduceint64_t) {
    int64_t_out = FLAMEGPU->agent("agent").reduce<int64_t>("int64_t", customMax, 0);
}

std::vector<unsigned int> uint_vec;
FLAMEGPU_STEP_FUNCTION(step_histogramEvenfloat) {
    uint_vec = FLAMEGPU->agent("agent").histogramEven<float, unsigned int>("float", TEST_LEN/10, FLT_MIN, FLT_MAX);
}

class MiniSim {
 public:
    MiniSim() :
      model("model"),
      agent("agent"),
      simulation(model),
      population(nullptr) {
        agent.addAgentVariable<float>("float");
        agent.addAgentVariable<double>("double");
        agent.addAgentVariable<char>("char");
        agent.addAgentVariable<unsigned char>("uchar");
        agent.addAgentVariable<uint16_t>("uint16_t");
        agent.addAgentVariable<int16_t>("int16_t");
        agent.addAgentVariable<uint32_t>("uint32_t");
        agent.addAgentVariable<int32_t>("int32_t");
        agent.addAgentVariable<uint64_t>("uint64_t");
        agent.addAgentVariable<int64_t>("int64_t");
        population = new AgentPopulation(agent, TEST_LEN);
        simulation.setSimulationSteps(1);
    }
    void run() {
        model.addAgent(agent);
        // CudaModel must be declared here
        // As the initial call to constructor fixes the agent population
        // This means if we haven't called model.addAgent(agent) first
        CUDAAgentModel cuda_model(model);
        // This fails as agentMap is empty
        cuda_model.setInitialPopulationData(*population);
        ASSERT_NO_THROW(cuda_model.simulate(simulation));
        // The negative of this, is that cuda_model is inaccessible within the test!
        // So copy across population data here
        ASSERT_NO_THROW(cuda_model.getPopulationData(*population));
    }
    ModelDescription model;
    AgentDescription agent;
    Simulation simulation;
    AgentPopulation *population;
};
/**
* This defines a common fixture used as a base for all test cases in the file
* @see https://github.com/google/googletest/blob/master/googletest/samples/sample5_unittest.cc
*/
class HostReductionTest : public testing::Test {
 protected:
    void SetUp() override {
        ms = new MiniSim();
    }

    void TearDown() override {
        delete ms;
    }

    MiniSim *ms = nullptr;
};

template<typename InT, typename OutT>
std::vector<OutT> histogramEven(const std::array<InT, TEST_LEN> &variables, const unsigned int &histogramBins, const InT &lowerBound, const InT &upperBound) {
    assert(upperBound > lowerBound);
    std::vector<OutT> rtn(histogramBins);
    for (auto &i : rtn)
        i = static_cast<OutT>(0);
    const InT diff = upperBound - lowerBound;
    const double diffP = diff / histogramBins;
    for (auto &i : variables) {
        if (i >= lowerBound && i <= upperBound) {
            ++rtn[static_cast<int>((i - lowerBound) / diffP)];
        }
    }
    return rtn;
}
}  // namespace

/**
 * Float
 */
TEST_F(HostReductionTest, MinFloat) {
    ms->simulation.addStepFunction(&step_minfloat);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <float> dist(FLT_MIN, FLT_MAX);
    std::array<float, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<float>("float", in[i]);
    }
    ms->run();
    EXPECT_EQ(float_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxFloat) {
    ms->simulation.addStepFunction(&step_maxfloat);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <float> dist(FLT_MIN, FLT_MAX);
    std::array<float, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<float>("float", in[i]);
    }
    ms->run();
    EXPECT_EQ(float_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumFloat) {
    ms->simulation.addStepFunction(&step_sumfloat);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <float> dist(FLT_MIN, FLT_MAX);
    std::array<float, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<float>("float", in[i]);
    }
    ms->run();
    EXPECT_EQ(float_out, std::accumulate(in.begin(), in.end(), 0.0f));
}
TEST_F(HostReductionTest, CustomReduceFloat) {
    ms->simulation.addStepFunction(&step_reducefloat);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <float> dist(FLT_MIN, FLT_MAX);
    std::array<float, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<float>("float", in[i]);
    }
    ms->run();
    EXPECT_EQ(float_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, HistogramEvenFloat) {
    ms->simulation.addStepFunction(&step_histogramEvenfloat);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <float> dist(FLT_MIN, FLT_MAX);
    std::array<float, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<float>("float", in[i]);
    }
    ms->run();
    auto check = histogramEven<float, unsigned int>(in, TEST_LEN / 10, FLT_MIN, FLT_MAX);
    for (int i = 0; i < uint_vec.size(); ++i) {
        EXPECT_EQ(uint_vec[i], check[i]);
    }
}

/**
 * Double
 */
TEST_F(HostReductionTest, MinDouble) {
    ms->simulation.addStepFunction(&step_mindouble);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <double> dist(DBL_MIN, DBL_MAX);
    std::array<double, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<double>("double", in[i]);
    }
    ms->run();
    EXPECT_EQ(double_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxDouble) {
    ms->simulation.addStepFunction(&step_maxdouble);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <double> dist(DBL_MIN, DBL_MAX);
    std::array<double, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<double>("double", in[i]);
    }
    ms->run();
    EXPECT_EQ(double_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumDouble) {
    ms->simulation.addStepFunction(&step_sumdouble);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <double> dist(DBL_MIN, DBL_MAX);
    std::array<double, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<double>("double", in[i]);
    }
    ms->run();
    EXPECT_EQ(double_out, std::accumulate(in.begin(), in.end(), 0.0));
}
TEST_F(HostReductionTest, CustomReduceDouble) {
    ms->simulation.addStepFunction(&step_reducedouble);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_real_distribution <double> dist(DBL_MIN, DBL_MAX);
    std::array<double, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<double>("double", in[i]);
    }
    ms->run();
    EXPECT_EQ(double_out, *std::max_element(in.begin(), in.end()));
}

/**
 * Char
 */
TEST_F(HostReductionTest, MinChar) {
    ms->simulation.addStepFunction(&step_minchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(CHAR_MIN, CHAR_MAX);
    std::array<char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = static_cast<char>(dist(rd));
        instance.setVariable<char>("char", in[i]);
    }
    ms->run();
    EXPECT_EQ(char_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxChar) {
    ms->simulation.addStepFunction(&step_maxchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(0, 1);
    std::array<char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < 256) {
            in[i] = static_cast<char>(dist(rd));
        } else {
            in[i] = 0;
        }
        instance.setVariable<char>("char", in[i]);
    }
    ms->run();
    EXPECT_EQ(char_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumChar) {
    ms->simulation.addStepFunction(&step_sumchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(CHAR_MIN, CHAR_MAX);
    std::array<char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < 256) {
            in[i] = static_cast<char>(dist(rd));
        } else {
            in[i] = 0;
        }
        instance.setVariable<char>("char", in[i]);
    }
    ms->run();
    EXPECT_EQ(char_out, std::accumulate(in.begin(), in.end(), static_cast<char>(0)));
    EXPECT_EQ(int64_t_out, std::accumulate(in.begin(), in.end(), static_cast<int64_t>(0)));
}
TEST_F(HostReductionTest, CustomReduceChar) {
    ms->simulation.addStepFunction(&step_reducechar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(0, 1);
    std::array<char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        if (i < 256) {
            in[i] = static_cast<char>(dist(rd));
        } else {
            in[i] = 0;
        }
        instance.setVariable<char>("char", in[i]);
    }
    ms->run();
    EXPECT_EQ(char_out, *std::max_element(in.begin(), in.end()));
}

/**
 * Unsigned Char
 */
TEST_F(HostReductionTest, MinUnsignedChar) {
    ms->simulation.addStepFunction(&step_minuchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UCHAR_MAX);
    std::array<unsigned char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = static_cast<unsigned char>(dist(rd));
        instance.setVariable<unsigned char>("uchar", in[i]);
    }
    ms->run();
    EXPECT_EQ(uchar_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxUnsignedChar) {
    ms->simulation.addStepFunction(&step_maxuchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UCHAR_MAX);
    std::array<unsigned char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = static_cast<unsigned char>(dist(rd));
        instance.setVariable<unsigned char>("uchar", in[i]);
    }
    ms->run();
    EXPECT_EQ(uchar_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumUnsignedChar) {
    ms->simulation.addStepFunction(&step_sumuchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UCHAR_MAX);
    std::array<unsigned char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = static_cast<unsigned char>(dist(rd));
        instance.setVariable<unsigned char>("uchar", in[i]);
    }
    ms->run();
    EXPECT_EQ(uchar_out, std::accumulate(in.begin(), in.end(), static_cast<unsigned char>(0)));
    EXPECT_EQ(uint64_t_out, std::accumulate(in.begin(), in.end(), static_cast<uint64_t>(0)));
}
TEST_F(HostReductionTest, CustomReduceUnsignedChar) {
    ms->simulation.addStepFunction(&step_reduceuchar);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UCHAR_MAX);
    std::array<unsigned char, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = static_cast<unsigned char>(dist(rd));
        instance.setVariable<unsigned char>("uchar", in[i]);
    }
    ms->run();
    EXPECT_EQ(uchar_out, *std::max_element(in.begin(), in.end()));
}

/**
 * int16_t
 */
TEST_F(HostReductionTest, MinInt16) {
    ms->simulation.addStepFunction(&step_minint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(INT16_MIN, INT16_MAX);
    std::array<int16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int16_t>("int16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int16_t_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxInt16) {
    ms->simulation.addStepFunction(&step_maxint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(INT16_MIN, INT16_MAX);
    std::array<int16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int16_t>("int16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int16_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumInt16) {
    ms->simulation.addStepFunction(&step_sumint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(INT16_MIN, INT16_MAX);
    std::array<int16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int16_t>("int16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int16_t_out, std::accumulate(in.begin(), in.end(), static_cast<int16_t>(0)));
    EXPECT_EQ(int64_t_out, std::accumulate(in.begin(), in.end(), static_cast<int64_t>(0)));
}
TEST_F(HostReductionTest, CustomReduceInt16) {
    ms->simulation.addStepFunction(&step_reduceint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int16_t> dist(INT16_MIN, INT16_MAX);
    std::array<int16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int16_t>("int16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int16_t_out, *std::max_element(in.begin(), in.end()));
}

/**
 * uint16_t
 */
TEST_F(HostReductionTest, MinUnsignedInt16) {
    ms->simulation.addStepFunction(&step_minuint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UINT16_MAX);
    std::array<uint16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint16_t>("uint16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint16_t_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxUnsignedInt16) {
    ms->simulation.addStepFunction(&step_maxuint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UINT16_MAX);
    std::array<uint16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint16_t>("uint16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint16_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumUnsignedInt16) {
    ms->simulation.addStepFunction(&step_sumuint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UINT16_MAX);
    std::array<uint16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint16_t>("uint16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint16_t_out, std::accumulate(in.begin(), in.end(), static_cast<uint16_t>(0)));
    EXPECT_EQ(uint64_t_out, std::accumulate(in.begin(), in.end(), static_cast<uint64_t>(0)));
}
TEST_F(HostReductionTest, CustomReduceUnsignedInt16) {
    ms->simulation.addStepFunction(&step_reduceuint16_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint16_t> dist(0, UINT16_MAX);
    std::array<uint16_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint16_t>("uint16_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint16_t_out, *std::max_element(in.begin(), in.end()));
}

/**
 * int32_t
 */
TEST_F(HostReductionTest, MinInt32) {
    ms->simulation.addStepFunction(&step_minint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int32_t> dist(INT32_MIN, INT32_MAX);
    std::array<int32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int32_t>("int32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int32_t_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxInt32) {
    ms->simulation.addStepFunction(&step_maxint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int32_t> dist(INT32_MIN, INT32_MAX);
    std::array<int32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int32_t>("int32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int32_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumInt32) {
    ms->simulation.addStepFunction(&step_sumint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int32_t> dist(INT32_MIN, INT32_MAX);
    std::array<int32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int32_t>("int32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int32_t_out, std::accumulate(in.begin(), in.end(), static_cast<int32_t>(0)));
    EXPECT_EQ(int64_t_out, std::accumulate(in.begin(), in.end(), static_cast<int64_t>(0)));
}
TEST_F(HostReductionTest, CustomReduceInt32) {
    ms->simulation.addStepFunction(&step_reduceint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int32_t> dist(INT32_MIN, INT32_MAX);
    std::array<int32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int32_t>("int32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int32_t_out, *std::max_element(in.begin(), in.end()));
}

/**
 * uint32_t
 */
TEST_F(HostReductionTest, MinUnsignedInt32) {
    ms->simulation.addStepFunction(&step_minuint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint32_t> dist(0, UINT32_MAX);
    std::array<uint32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint32_t>("uint32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxUnsignedInt32) {
    ms->simulation.addStepFunction(&step_maxuint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint32_t> dist(0, UINT32_MAX);
    std::array<uint32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint32_t>("uint32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumUnsignedInt32) {
    ms->simulation.addStepFunction(&step_sumuint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint32_t> dist(0, UINT32_MAX);
    std::array<uint32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint32_t>("uint32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, std::accumulate(in.begin(), in.end(), static_cast<uint32_t>(0)));
    EXPECT_EQ(uint64_t_out, std::accumulate(in.begin(), in.end(), static_cast<uint64_t>(0)));
}
TEST_F(HostReductionTest, CustomReduceUnsignedInt32) {
    ms->simulation.addStepFunction(&step_reduceuint32_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint32_t> dist(0, UINT32_MAX);
    std::array<uint32_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint32_t>("uint32_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint32_t_out, *std::max_element(in.begin(), in.end()));
}

/**
 * int64_t
 */
TEST_F(HostReductionTest, MinInt64) {
    ms->simulation.addStepFunction(&step_minint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int64_t> dist(INT64_MIN, INT64_MAX);
    std::array<int64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int64_t>("int64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int64_t_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxInt64) {
    ms->simulation.addStepFunction(&step_maxint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int64_t> dist(INT64_MIN, INT64_MAX);
    std::array<int64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int64_t>("int64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int64_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumInt64) {
    ms->simulation.addStepFunction(&step_sumint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int64_t> dist(INT64_MIN, INT64_MAX);
    std::array<int64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int64_t>("int64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int64_t_out, std::accumulate(in.begin(), in.end(), 0ll));
}
TEST_F(HostReductionTest, CustomReduceInt64) {
    ms->simulation.addStepFunction(&step_reduceint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <int64_t> dist(INT64_MIN, INT64_MAX);
    std::array<int64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<int64_t>("int64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(int64_t_out, *std::max_element(in.begin(), in.end()));
}

/**
 * uint64_t
 */
TEST_F(HostReductionTest, MinUnsignedInt64) {
    ms->simulation.addStepFunction(&step_minuint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint64_t> dist(0, UINT64_MAX);
    std::array<uint64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint64_t>("uint64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint64_t_out, *std::min_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, MaxUnsignedInt64) {
    ms->simulation.addStepFunction(&step_maxuint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint64_t> dist(0, UINT64_MAX);
    std::array<uint64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint64_t>("uint64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint64_t_out, *std::max_element(in.begin(), in.end()));
}
TEST_F(HostReductionTest, SumUnsignedInt64) {
    ms->simulation.addStepFunction(&step_sumuint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint64_t> dist(0, UINT64_MAX);
    std::array<uint64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint64_t>("uint64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint64_t_out, std::accumulate(in.begin(), in.end(), 0llu));
}
TEST_F(HostReductionTest, CustomReduceUnsignedInt64) {
    ms->simulation.addStepFunction(&step_reduceuint64_t);
    std::mt19937 rd;  // Seed does not matter
    std::uniform_int_distribution <uint64_t> dist(0, UINT64_MAX);
    std::array<uint64_t, TEST_LEN> in;
    for (unsigned int i = 0; i < TEST_LEN; i++) {
        AgentInstance instance = ms->population->getNextInstance();
        in[i] = dist(rd);
        instance.setVariable<uint64_t>("uint64_t", in[i]);
    }
    ms->run();
    EXPECT_EQ(uint64_t_out, *std::max_element(in.begin(), in.end()));
}

#endif  // TESTS_TEST_CASES_RUNTIME_TEST_HOST_REDUCTIONS_H_
