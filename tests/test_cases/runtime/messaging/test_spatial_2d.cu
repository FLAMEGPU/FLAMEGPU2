#include "hip/hip_runtime.h"
/**
* Tests of feature Spatial 3D messaging
*
* Tests cover:
* > mandatory messaging, send/recieve
*/
#include "flamegpu/flame_api.h"
#include "flamegpu/runtime/flamegpu_api.h"

#include "gtest/gtest.h"


namespace test_message_spatial2d {

FLAMEGPU_AGENT_FUNCTION(out_mandatory2D, MsgNone, MsgSpatial2D) {
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
    FLAMEGPU->message_out.setLocation(
        FLAMEGPU->getVariable<float>("x"),
        FLAMEGPU->getVariable<float>("y"));
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(out_optional2D, MsgNone, MsgSpatial2D) {
    if (FLAMEGPU->getVariable<int>("do_output")) {
        FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
        FLAMEGPU->message_out.setLocation(
            FLAMEGPU->getVariable<float>("x"),
            FLAMEGPU->getVariable<float>("y"));
    }
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(out_optional2DNone, MsgNone, MsgSpatial2D) {
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(in2D, MsgSpatial2D, MsgNone) {
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    unsigned int count = 0;
    unsigned int badCount = 0;
     unsigned int myBin[2] = {
         static_cast<unsigned int>(x1),
         static_cast<unsigned int>(y1)
     };
    // Count how many messages we recieved (including our own)
    // This is all those which fall within the 3x3x3 Moore neighbourhood
    // Not our search radius
    for (const auto &message : FLAMEGPU->message_in(x1, y1)) {
         unsigned int msgBin[2] = {
             static_cast<unsigned int>(message.getVariable<float>("x")),
             static_cast<unsigned int>(message.getVariable<float>("y"))
         };
         bool isBad = false;
         for (unsigned int i = 0; i < 2; ++i) {  // Iterate axis
             int binDiff = myBin[i] - msgBin[i];
             if (binDiff > 1 || binDiff < -1) {
                 isBad = true;
             }
         }
        count++;
        badCount = isBad ? badCount + 1 : badCount;
    }
    FLAMEGPU->setVariable<unsigned int>("count", count);
    FLAMEGPU->setVariable<unsigned int>("badCount", badCount);
    return ALIVE;
}
TEST(Spatial2DMsgTest, Mandatory) {
    std::unordered_map<int, unsigned int> bin_counts;
    // Construct model
    ModelDescription model("Spatial2DMsgTestModel");
    {   // Location message
        MsgSpatial2D::Description &message = model.newMessage<MsgSpatial2D>("location");
        message.setMin(0, 0);
        message.setMax(11, 11);
        message.setRadius(1);
        // 11x11 bins, total 121
        message.newVariable<int>("id");  // unused by current test
    }
    {   // Circle agent
        AgentDescription &agent = model.newAgent("agent");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<unsigned int>("myBin");  // This will be presumed bin index of the agent, might not use this
        agent.newVariable<unsigned int>("count");  // Store the distance moved here, for validation
        agent.newVariable<unsigned int>("badCount");  // Store how many messages are out of range
        agent.newFunction("out", out_mandatory2D).setMessageOutput("location");
        agent.newFunction("in", in2D).setMessageInput("location");
    }
    {   // Layer #1
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(out_mandatory2D);
    }
    {   // Layer #2
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(in2D);
    }
    CUDASimulation cuda_model(model);

    const int AGENT_COUNT = 2049;
    AgentPopulation population(model.Agent("agent"), AGENT_COUNT);
    // Initialise agents (TODO)
    {
        // Currently population has not been init, so generate an agent population on the fly
        std::default_random_engine rng;
        std::uniform_real_distribution<float> dist(0.0f, 11.0f);
        for (unsigned int i = 0; i < AGENT_COUNT; i++) {
            AgentInstance instance = population.getNextInstance();
            instance.setVariable<int>("id", i);
            float pos[3] = { dist(rng), dist(rng), dist(rng) };
            instance.setVariable<float>("x", pos[0]);
            instance.setVariable<float>("y", pos[1]);
            // Solve the bin index
            const unsigned int bin_pos[2] = {
                (unsigned int)(pos[0] / 1),
                (unsigned int)(pos[1] / 1)
            };
            const unsigned int bin_index =
                bin_pos[1] * 11 +
                bin_pos[0];
            instance.setVariable<unsigned int>("myBin", bin_index);
            // Create it if it doesn't already exist
            if (bin_counts.find(bin_index) == bin_counts.end()) {
                bin_counts.emplace(bin_index, 0);
            }
            bin_counts[bin_index] += 1;
        }
        cuda_model.setPopulationData(population);
    }

    // Generate results expectation
    std::unordered_map<int, unsigned int> bin_results;
    // Iterate host bin
    for (unsigned int x1 = 0; x1 < 11; x1++) {
        for (unsigned int y1 = 0; y1 < 11; y1++) {
            // Solve the bin index
            const unsigned int bin_pos1[3] = {
                x1,
                y1
            };
            const unsigned int bin_index1 =
                bin_pos1[1] * 11 +
                bin_pos1[0];
            // Count our neighbours
            unsigned int count_sum = 0;
            for (int x2 = -1; x2 <= 1; x2++) {
                int bin_pos2[2] = {
                    static_cast<int>(bin_pos1[0]) + x2,
                    0
                };
                for (int y2 = -1; y2 <= 1; y2++) {
                    bin_pos2[1] = static_cast<int>(bin_pos1[1]) + y2;
                    // Ensure bin is in bounds
                    if (
                        bin_pos2[0] >= 0 &&
                        bin_pos2[1] >= 0 &&
                        bin_pos2[0] < 11 &&
                        bin_pos2[1] < 11
                        ) {
                        const unsigned int bin_index2 =
                            bin_pos2[1] * 11 +
                            bin_pos2[0];
                        count_sum += bin_counts[bin_index2];
                    }
                }
            }
            bin_results.emplace(bin_index1, count_sum);
         }
    }

    // Execute a single step of the model
    cuda_model.step();

    // Recover the results and check they match what was expected

    cuda_model.getPopulationData(population);
    // Validate each agent has same result
    unsigned int badCountWrong = 0;
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = population.getInstanceAt(i);
        unsigned int myBin = ai.getVariable<unsigned int>("myBin");
        unsigned int myResult = ai.getVariable<unsigned int>("count");
        EXPECT_EQ(myResult, bin_results.at(myBin));
        if (ai.getVariable<unsigned int>("badCount"))
            badCountWrong++;
    }
    EXPECT_EQ(badCountWrong, 0u);
}

TEST(Spatial2DMsgTest, Optional) {
    /**
     * This test is same as Mandatory, however extra flag has been added to block certain agents from outputting messages
     * Look for NEW!
     */
    std::unordered_map<int, unsigned int> bin_counts;
    std::unordered_map<int, unsigned int> bin_counts_optional;
    // Construct model
    ModelDescription model("Spatial2DMsgTestModel");
    {   // Location message
        MsgSpatial2D::Description &message = model.newMessage<MsgSpatial2D>("location");
        message.setMin(0, 0);
        message.setMax(11, 11);
        message.setRadius(1);
        // 11x11 bins, total 121
        message.newVariable<int>("id");  // unused by current test
    }
    {   // Circle agent
        AgentDescription &agent = model.newAgent("agent");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<int>("do_output");  // NEW!
        agent.newVariable<unsigned int>("myBin");  // This will be presumed bin index of the agent, might not use this
        agent.newVariable<unsigned int>("count");  // Store the distance moved here, for validation
        agent.newVariable<unsigned int>("badCount");  // Store how many messages are out of range
        auto &af = agent.newFunction("out", out_optional2D);  // NEW!
        af.setMessageOutput("location");
        af.setMessageOutputOptional(true);  // NEW!
        agent.newFunction("in", in2D).setMessageInput("location");
    }
    {   // Layer #1
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(out_optional2D);  // NEW!
    }
    {   // Layer #2
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(in2D);
    }
    CUDASimulation cuda_model(model);

    const int AGENT_COUNT = 2049;
    AgentPopulation population(model.Agent("agent"), AGENT_COUNT);
    // Initialise agents (TODO)
    {
        // Currently population has not been init, so generate an agent population on the fly
        std::default_random_engine rng;
        std::uniform_real_distribution<float> dist(0.0f, 11.0f);
        std::uniform_real_distribution<float> dist5(0.0f, 5.0f);
        for (unsigned int i = 0; i < AGENT_COUNT; i++) {
            AgentInstance instance = population.getNextInstance();
            instance.setVariable<int>("id", i);
            float pos[3] = { dist(rng), dist(rng), dist(rng) };
            int do_output = dist5(rng) < 4 ? 1 : 0;  // 80% chance of output  // NEW!
            instance.setVariable<float>("x", pos[0]);
            instance.setVariable<float>("y", pos[1]);
            instance.setVariable<int>("do_output", do_output);  // NEW!
            // Solve the bin index
            const unsigned int bin_pos[2] = {
                (unsigned int)(pos[0] / 1),
                (unsigned int)(pos[1] / 1)
            };
            const unsigned int bin_index =
                bin_pos[1] * 11 +
                bin_pos[0];
            instance.setVariable<unsigned int>("myBin", bin_index);
            // Create it if it doesn't already exist
            bin_counts[bin_index] += 1;
            if (do_output) {  // NEW!
                bin_counts_optional[bin_index] += 1;  // NEW!
            }
        }
        cuda_model.setPopulationData(population);
    }

    // Generate results expectation
    std::unordered_map<int, unsigned int> bin_results;
    std::unordered_map<int, unsigned int> bin_results_optional;
    // Iterate host bin
    for (unsigned int x1 = 0; x1 < 11; x1++) {
        for (unsigned int y1 = 0; y1 < 11; y1++) {
            // Solve the bin index
            const unsigned int bin_pos1[3] = {
                x1,
                y1
            };
            const unsigned int bin_index1 =
                bin_pos1[1] * 11 +
                bin_pos1[0];
            // Count our neighbours
            unsigned int count_sum = 0;
            unsigned int count_sum_optional = 0;  // NEW!
            for (int x2 = -1; x2 <= 1; x2++) {
                int bin_pos2[2] = {
                    static_cast<int>(bin_pos1[0]) + x2,
                    0
                };
                for (int y2 = -1; y2 <= 1; y2++) {
                    bin_pos2[1] = static_cast<int>(bin_pos1[1]) + y2;
                    // Ensure bin is in bounds
                    if (
                        bin_pos2[0] >= 0 &&
                        bin_pos2[1] >= 0 &&
                        bin_pos2[0] < 11 &&
                        bin_pos2[1] < 11
                        ) {
                        const unsigned int bin_index2 =
                            bin_pos2[1] * 11 +
                            bin_pos2[0];
                        count_sum += bin_counts[bin_index2];
                        count_sum_optional += bin_counts_optional[bin_index2];  // NEW!
                    }
                }
            }
            bin_results.emplace(bin_index1, count_sum);
            bin_results_optional.emplace(bin_index1, count_sum_optional);  // NEW!
        }
    }

    // Execute a single step of the model
    cuda_model.step();

    // Recover the results and check they match what was expected

    cuda_model.getPopulationData(population);
    // Validate each agent has same result
    unsigned int badCountWrong = 0;
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = population.getInstanceAt(i);
        unsigned int myBin = ai.getVariable<unsigned int>("myBin");
        unsigned int myResult = ai.getVariable<unsigned int>("count");
        if (ai.getVariable<unsigned int>("badCount"))
            badCountWrong++;
        EXPECT_EQ(myResult, bin_results_optional.at(myBin));  // NEW!
    }
    EXPECT_EQ(badCountWrong, 0u);
}
TEST(Spatial2DMsgTest, OptionalNone) {
    /**
     * This test is same as Mandatory, however extra flag has been added to block certain agents from outputting messages
     * Look for NEW!
     */
    std::unordered_map<int, unsigned int> bin_counts;
    std::unordered_map<int, unsigned int> bin_counts_optional;
    // Construct model
    ModelDescription model("Spatial2DMsgTestModel");
    {   // Location message
        MsgSpatial2D::Description &message = model.newMessage<MsgSpatial2D>("location");
        message.setMin(0, 0);
        message.setMax(11, 11);
        message.setRadius(1);
        // 11x11 bins, total 121
        message.newVariable<int>("id");  // unused by current test
    }
    {   // Circle agent
        AgentDescription &agent = model.newAgent("agent");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<int>("do_output");  // NEW!
        agent.newVariable<unsigned int>("myBin");  // This will be presumed bin index of the agent, might not use this
        agent.newVariable<unsigned int>("count");  // Store the distance moved here, for validation
        agent.newVariable<unsigned int>("badCount");  // Store how many messages are out of range
        auto &af = agent.newFunction("out", out_optional2DNone);  // NEW!
        af.setMessageOutput("location");
        af.setMessageOutputOptional(true);  // NEW!
        agent.newFunction("in", in2D).setMessageInput("location");
    }
    {   // Layer #1
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(out_optional2DNone);  // NEW!
    }
    {   // Layer #2
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(in2D);
    }
    CUDASimulation cuda_model(model);

    const int AGENT_COUNT = 2049;
    AgentPopulation population(model.Agent("agent"), AGENT_COUNT);
    // Initialise agents (TODO)
    {
        // Currently population has not been init, so generate an agent population on the fly
        std::default_random_engine rng;
        std::uniform_real_distribution<float> dist(0.0f, 11.0f);
        std::uniform_real_distribution<float> dist5(0.0f, 5.0f);
        for (unsigned int i = 0; i < AGENT_COUNT; i++) {
            AgentInstance instance = population.getNextInstance();
            instance.setVariable<int>("id", i);
            float pos[3] = { dist(rng), dist(rng), dist(rng) };
            int do_output = dist5(rng) < 4 ? 1 : 0;  // 80% chance of output  // NEW!
            instance.setVariable<float>("x", pos[0]);
            instance.setVariable<float>("y", pos[1]);
            instance.setVariable<int>("do_output", do_output);  // NEW!
            // Solve the bin index
            const unsigned int bin_pos[2] = {
                (unsigned int)(pos[0] / 1),
                (unsigned int)(pos[1] / 1)
            };
            const unsigned int bin_index =
                bin_pos[1] * 11 +
                bin_pos[0];
            instance.setVariable<unsigned int>("myBin", bin_index);
            // Create it if it doesn't already exist
            bin_counts[bin_index] += 1;
            if (do_output) {  // NEW!
                bin_counts_optional[bin_index] += 1;  // NEW!
            }
        }
        cuda_model.setPopulationData(population);
    }

    // Execute a single step of the model
    cuda_model.step();

    // Recover the results and check they match what was expected

    cuda_model.getPopulationData(population);
    // Validate each agent has same result
    unsigned int badCountWrong = 0;
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = population.getInstanceAt(i);
        unsigned int myResult = ai.getVariable<unsigned int>("count");
        if (ai.getVariable<unsigned int>("badCount"))
            badCountWrong++;
        EXPECT_EQ(myResult, 0u);  // NEW!
    }
    EXPECT_EQ(badCountWrong, 0u);
}

TEST(Spatial2DMsgTest, BadRadius) {
    ModelDescription model("Spatial2DMsgTestModel");
    MsgSpatial2D::Description &message = model.newMessage<MsgSpatial2D>("location");
    EXPECT_THROW(message.setRadius(0), InvalidArgument);
    EXPECT_THROW(message.setRadius(-10), InvalidArgument);
}
TEST(Spatial2DMsgTest, BadMin) {
    ModelDescription model("Spatial2DMsgTestModel");
    MsgSpatial2D::Description &message = model.newMessage<MsgSpatial2D>("location");
    message.setMax(5, 5);
    EXPECT_THROW(message.setMin(5, 0), InvalidArgument);
    EXPECT_THROW(message.setMin(0, 5), InvalidArgument);
    EXPECT_THROW(message.setMin(6, 0), InvalidArgument);
    EXPECT_THROW(message.setMin(0, 6), InvalidArgument);
}
TEST(Spatial2DMsgTest, BadMax) {
    ModelDescription model("Spatial2DMsgTestModel");
    MsgSpatial2D::Description &message = model.newMessage<MsgSpatial2D>("location");
    message.setMin(5, 5);
    EXPECT_THROW(message.setMax(5, 0), InvalidArgument);
    EXPECT_THROW(message.setMax(0, 5), InvalidArgument);
    EXPECT_THROW(message.setMax(4, 0), InvalidArgument);
    EXPECT_THROW(message.setMax(0, 4), InvalidArgument);
}
TEST(Spatial2DMsgTest, UnsetMax) {
    ModelDescription model("Spatial2DMsgTestModel");
    MsgSpatial2D::Description &message = model.newMessage<MsgSpatial2D>("location");
    message.setMin(5, 5);
    EXPECT_THROW(CUDASimulation m(model), InvalidMessage);
}
TEST(Spatial2DMsgTest, UnsetMin) {
    ModelDescription model("Spatial2DMsgTestModel");
    MsgSpatial2D::Description &message = model.newMessage<MsgSpatial2D>("location");
    message.setMin(5, 5);
    EXPECT_THROW(CUDASimulation m(model), InvalidMessage);
}
TEST(Spatial2DMsgTest, reserved_name) {
    ModelDescription model("Spatial2DMsgTestModel");
    MsgSpatial2D::Description &message = model.newMessage<MsgSpatial2D>("location");
    EXPECT_THROW(message.newVariable<int>("_"), ReservedName);
}

FLAMEGPU_AGENT_FUNCTION(count2D, MsgSpatial2D, MsgNone) {
    unsigned int count = 0;
    // Count how many messages we received (including our own)
    // This is all those which fall within the 3x3 Moore neighbourhood
    for (const auto &message : FLAMEGPU->message_in(0, 0)) {
        count++;
    }
    FLAMEGPU->setVariable<unsigned int>("count", count);
    return ALIVE;
}
TEST(Spatial2DMsgTest, ReadEmpty) {
// What happens if we read a message list before it has been output?
    ModelDescription model("Model");
    {   // Location message
        MsgSpatial2D::Description &message = model.newMessage<MsgSpatial2D>("location");
        message.setMin(-3, -3);
        message.setMax(3, 3);
        message.setRadius(2);
        message.newVariable<int>("id");  // unused by current test
    }
    {   // Circle agent
        AgentDescription &agent = model.newAgent("agent");
        agent.newVariable<unsigned int>("count", 0);  // Count the number of messages read
        agent.newFunction("in", count2D).setMessageInput("location");
    }
    {   // Layer #1
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(count2D);
    }
    // Create 1 agent
    AgentPopulation pop_in(model.Agent("agent"), 1);
    pop_in.getNextInstance();
    CUDASimulation cuda_model(model);
    cuda_model.setPopulationData(pop_in);
    // Execute model
    EXPECT_NO_THROW(cuda_model.step());
    // Check result
    AgentPopulation pop_out(model.Agent("agent"), 1);
    pop_out.getNextInstance().setVariable<unsigned int>("count", 1);
    cuda_model.getPopulationData(pop_out);
    EXPECT_EQ(pop_out.getCurrentListSize(), 1u);
    auto ai = pop_out.getInstanceAt(0);
    EXPECT_EQ(ai.getVariable<unsigned int>("count"), 0u);
}
}  // namespace test_message_spatial2d
