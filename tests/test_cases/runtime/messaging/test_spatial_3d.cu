#include "hip/hip_runtime.h"
/**
* Tests of feature Spatial 3D messaging
*
* Tests cover:
* > mandatory messaging, send/recieve
*/
#include "flamegpu/flamegpu.h"

#include "gtest/gtest.h"

namespace flamegpu {


namespace test_message_spatial3d {

FLAMEGPU_AGENT_FUNCTION(out_mandatory3D, MessageNone, MessageSpatial3D) {
    FLAMEGPU->message_out.setVariable<flamegpu::id_t>("id", FLAMEGPU->getID());
    FLAMEGPU->message_out.setLocation(
        FLAMEGPU->getVariable<float>("x"),
        FLAMEGPU->getVariable<float>("y"),
        FLAMEGPU->getVariable<float>("z"));
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(out_optional3D, MessageNone, MessageSpatial3D) {
    if (FLAMEGPU->getVariable<int>("do_output")) {
        FLAMEGPU->message_out.setVariable<flamegpu::id_t>("id", FLAMEGPU->getID());
        FLAMEGPU->message_out.setLocation(
            FLAMEGPU->getVariable<float>("x"),
            FLAMEGPU->getVariable<float>("y"),
            FLAMEGPU->getVariable<float>("z"));
    }
    return ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(out_optional3DNone, MessageNone, MessageSpatial3D) {
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(in3D, MessageSpatial3D, MessageNone) {
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    const float z1 = FLAMEGPU->getVariable<float>("z");
    unsigned int count = 0;
    unsigned int badCount = 0;
     int myBin[3] = {
         static_cast<int>(x1),
         static_cast<int>(y1),
         static_cast<int>(z1)
     };
    // Count how many messages we received (including our own)
    // This is all those which fall within the 3x3x3 Moore neighbourhood
    // Not our search radius
    for (const auto &message : FLAMEGPU->message_in(x1, y1, z1)) {
         int messageBin[3] = {
             static_cast<int>(message.getVariable<float>("x")),
             static_cast<int>(message.getVariable<float>("y")),
             static_cast<int>(message.getVariable<float>("z"))
        };
        bool isBad = false;
        for (unsigned int i = 0; i < 3; ++i) {  // Iterate axis
            int binDiff = myBin[i] - messageBin[i];
            if (binDiff > 1 || binDiff < -1) {
                isBad = true;
            }
        }
        count++;
        badCount = isBad ? badCount + 1 : badCount;
    }
    FLAMEGPU->setVariable<unsigned int>("count", count);
    FLAMEGPU->setVariable<unsigned int>("badCount", badCount);
    return ALIVE;
}
TEST(Spatial3DMessageTest, Mandatory) {
    std::unordered_map<int, unsigned int> bin_counts;
    // Construct model
    ModelDescription model("Spatial3DMessageTestModel");
    {   // Location message
        MessageSpatial3D::Description message = model.newMessage<MessageSpatial3D>("location");
        message.setMin(0, 0, 0);
        message.setMax(5, 5, 5);
        message.setRadius(1);
        // 5x5x5 bins, total 125
        message.newVariable<flamegpu::id_t>("id");  // unused by current test
    }
    {   // Circle agent
        AgentDescription agent = model.newAgent("agent");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("z");
        agent.newVariable<unsigned int>("myBin");  // This will be presumed bin index of the agent, might not use this
        agent.newVariable<unsigned int>("count");  // Store the distance moved here, for validation
        agent.newVariable<unsigned int>("badCount");  // Store how many messages are out of range
        agent.newFunction("out", out_mandatory3D).setMessageOutput("location");
        agent.newFunction("in", in3D).setMessageInput("location");
    }
    {   // Layer #1
        LayerDescription layer = model.newLayer();
        layer.addAgentFunction(out_mandatory3D);
    }
    {   // Layer #2
        LayerDescription layer = model.newLayer();
        layer.addAgentFunction(in3D);
    }
    CUDASimulation cudaSimulation(model);

    const int AGENT_COUNT = 2049;
    AgentVector population(model.Agent("agent"), AGENT_COUNT);
    // Initialise agents (TODO)
    {
        // Currently population has not been init, so generate an agent population on the fly
        std::mt19937_64 rng;
        std::uniform_real_distribution<float> dist(0.0f, 5.0f);
        for (unsigned int i = 0; i < AGENT_COUNT; i++) {
            AgentVector::Agent instance = population[i];
            float pos[3] = { dist(rng), dist(rng), dist(rng) };
            instance.setVariable<float>("x", pos[0]);
            instance.setVariable<float>("y", pos[1]);
            instance.setVariable<float>("z", pos[2]);
            // Solve the bin index
            const unsigned int bin_pos[3] = {
                (unsigned int)(pos[0] / 1),
                (unsigned int)(pos[1] / 1),
                (unsigned int)(pos[2] / 1)
            };
            const unsigned int bin_index =
                bin_pos[2] * 5 * 5 +
                bin_pos[1] * 5 +
                bin_pos[0];
            instance.setVariable<unsigned int>("myBin", bin_index);
            // Create it if it doesn't already exist
            if (bin_counts.find(bin_index) == bin_counts.end()) {
                bin_counts.emplace(bin_index, 0);
            }
            bin_counts[bin_index] += 1;
        }
        cudaSimulation.setPopulationData(population);
    }

    // Generate results expectation
    std::unordered_map<int, unsigned int> bin_results;
    // Iterate host bin
    for (unsigned int x1 = 0; x1 < 5; x1++) {
        for (unsigned int y1 = 0; y1 < 5; y1++) {
            for (unsigned int z1 = 0; z1 < 5; z1++) {
                // Solve the bin index
                const unsigned int bin_pos1[3] = {
                    x1,
                    y1,
                    z1
                };
                const unsigned int bin_index1 =
                    bin_pos1[2] * 5 * 5 +
                    bin_pos1[1] * 5 +
                    bin_pos1[0];
                // Count our neighbours
                unsigned int count_sum = 0;
                for (int x2 = -1; x2 <= 1; x2++) {
                    int bin_pos2[3] = {
                        static_cast<int>(bin_pos1[0]) + x2,
                        0,
                        0
                    };
                    for (int y2 = -1; y2 <= 1; y2++) {
                        bin_pos2[1] = static_cast<int>(bin_pos1[1]) + y2;
                        for (int z2 = -1; z2 <= 1; z2++) {
                            bin_pos2[2] = static_cast<int>(bin_pos1[2]) + z2;
                            // Ensure bin is in bounds
                            if (
                                bin_pos2[0] >= 0 &&
                                bin_pos2[1] >= 0 &&
                                bin_pos2[2] >= 0 &&
                                bin_pos2[0] < 5 &&
                                bin_pos2[1] < 5 &&
                                bin_pos2[2] < 5
                                ) {
                                const unsigned int bin_index2 =
                                    bin_pos2[2] * 5 * 5 +
                                    bin_pos2[1] * 5 +
                                    bin_pos2[0];
                                count_sum += bin_counts[bin_index2];
                            }
                        }
                    }
                }
                bin_results.emplace(bin_index1, count_sum);
            }
        }
    }

    // Execute a single step of the model
    cudaSimulation.step();

    // Recover the results and check they match what was expected

    cudaSimulation.getPopulationData(population);
    // Validate each agent has same result
    unsigned int badCountWrong = 0;
    for (AgentVector::Agent ai : population) {
        unsigned int myBin = ai.getVariable<unsigned int>("myBin");
        unsigned int myResult = ai.getVariable<unsigned int>("count");
        EXPECT_EQ(myResult, bin_results.at(myBin));
        if (ai.getVariable<unsigned int>("badCount"))
            badCountWrong++;
    }
    EXPECT_EQ(badCountWrong, 0u);
}

TEST(Spatial3DMessageTest, Optional) {
    /**
     * This test is same as Mandatory, however extra flag has been added to block certain agents from outputting messages
     * Look for NEW!
     */
    std::unordered_map<int, unsigned int> bin_counts;
    std::unordered_map<int, unsigned int> bin_counts_optional;
    // Construct model
    ModelDescription model("Spatial3DMessageTestModel");
    {   // Location message
        MessageSpatial3D::Description message = model.newMessage<MessageSpatial3D>("location");
        message.setMin(0, 0, 0);
        message.setMax(5, 5, 5);
        message.setRadius(1);
        // 5x5x5 bins, total 125
        message.newVariable<flamegpu::id_t>("id");  // unused by current test
    }
    {   // Circle agent
        AgentDescription agent = model.newAgent("agent");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("z");
        agent.newVariable<int>("do_output");  // NEW!
        agent.newVariable<unsigned int>("myBin");  // This will be presumed bin index of the agent, might not use this
        agent.newVariable<unsigned int>("count");  // Store the distance moved here, for validation
        agent.newVariable<unsigned int>("badCount");  // Store how many messages are out of range
        auto af = agent.newFunction("out", out_optional3D);  // NEW!
        af.setMessageOutput("location");
        af.setMessageOutputOptional(true);  // NEW!
        agent.newFunction("in", in3D).setMessageInput("location");
    }
    {   // Layer #1
        LayerDescription layer = model.newLayer();
        layer.addAgentFunction(out_optional3D);  // NEW!
    }
    {   // Layer #2
        LayerDescription layer = model.newLayer();
        layer.addAgentFunction(in3D);
    }
    CUDASimulation cudaSimulation(model);

    const int AGENT_COUNT = 2049;
    AgentVector population(model.Agent("agent"), AGENT_COUNT);
    // Initialise agents (TODO)
    {
        // Currently population has not been init, so generate an agent population on the fly
        std::mt19937_64 rng;
        std::uniform_real_distribution<float> dist(0.0f, 5.0f);
        for (unsigned int i = 0; i < AGENT_COUNT; i++) {
            AgentVector::Agent instance = population[i];
            float pos[3] = { dist(rng), dist(rng), dist(rng) };
            int do_output = dist(rng) < 4 ? 1 : 0;  // 80% chance of output  // NEW!
            instance.setVariable<float>("x", pos[0]);
            instance.setVariable<float>("y", pos[1]);
            instance.setVariable<float>("z", pos[2]);
            instance.setVariable<int>("do_output", do_output);  // NEW!
            // Solve the bin index
            const unsigned int bin_pos[3] = {
                (unsigned int)(pos[0] / 1),
                (unsigned int)(pos[1] / 1),
                (unsigned int)(pos[2] / 1)
            };
            const unsigned int bin_index =
                bin_pos[2] * 5 * 5 +
                bin_pos[1] * 5 +
                bin_pos[0];
            instance.setVariable<unsigned int>("myBin", bin_index);
            // Create it if it doesn't already exist
            bin_counts[bin_index] += 1;
            if (do_output) {  // NEW!
                bin_counts_optional[bin_index] += 1;  // NEW!
            }
        }
        cudaSimulation.setPopulationData(population);
    }

    // Generate results expectation
    std::unordered_map<int, unsigned int> bin_results;
    std::unordered_map<int, unsigned int> bin_results_optional;
    // Iterate host bin
    for (unsigned int x1 = 0; x1 < 5; x1++) {
        for (unsigned int y1 = 0; y1 < 5; y1++) {
            for (unsigned int z1 = 0; z1 < 5; z1++) {
                // Solve the bin index
                const unsigned int bin_pos1[3] = {
                    x1,
                    y1,
                    z1
                };
                const unsigned int bin_index1 =
                    bin_pos1[2] * 5 * 5 +
                    bin_pos1[1] * 5 +
                    bin_pos1[0];
                // Count our neighbours
                unsigned int count_sum = 0;
                unsigned int count_sum_optional = 0;  // NEW!
                for (int x2 = -1; x2 <= 1; x2++) {
                    int bin_pos2[3] = {
                        static_cast<int>(bin_pos1[0]) + x2,
                        0,
                        0
                    };
                    for (int y2 = -1; y2 <= 1; y2++) {
                        bin_pos2[1] = static_cast<int>(bin_pos1[1]) + y2;
                        for (int z2 = -1; z2 <= 1; z2++) {
                            bin_pos2[2] = static_cast<int>(bin_pos1[2]) + z2;
                            // Ensure bin is in bounds
                            if (
                                bin_pos2[0] >= 0 &&
                                bin_pos2[1] >= 0 &&
                                bin_pos2[2] >= 0 &&
                                bin_pos2[0] < 5 &&
                                bin_pos2[1] < 5 &&
                                bin_pos2[2] < 5
                                ) {
                                const unsigned int bin_index2 =
                                    bin_pos2[2] * 5 * 5 +
                                    bin_pos2[1] * 5 +
                                    bin_pos2[0];
                                count_sum += bin_counts[bin_index2];
                                count_sum_optional += bin_counts_optional[bin_index2];  // NEW!
                            }
                        }
                    }
                }
                bin_results.emplace(bin_index1, count_sum);
                bin_results_optional.emplace(bin_index1, count_sum_optional);  // NEW!
            }
        }
    }

    // Execute a single step of the model
    cudaSimulation.step();

    // Recover the results and check they match what was expected

    cudaSimulation.getPopulationData(population);
    // Validate each agent has same result
    unsigned int badCountWrong = 0;
    for (AgentVector::Agent ai : population) {
        unsigned int myBin = ai.getVariable<unsigned int>("myBin");
        unsigned int myResult = ai.getVariable<unsigned int>("count");
        if (ai.getVariable<unsigned int>("badCount"))
            badCountWrong++;
        EXPECT_EQ(myResult, bin_results_optional.at(myBin));  // NEW!
    }
    EXPECT_EQ(badCountWrong, 0u);
}
// Test optional message output, with no messaegs
TEST(Spatial3DMessageTest, OptionalNone) {
    /**
     * This test is same as Mandatory, however extra flag has been added to block certain agents from outputting messages
     * Look for NEW!
     */
    std::unordered_map<int, unsigned int> bin_counts;
    std::unordered_map<int, unsigned int> bin_counts_optional;
    // Construct model
    ModelDescription model("Spatial3DMessageTestModel");
    {   // Location message
        MessageSpatial3D::Description message = model.newMessage<MessageSpatial3D>("location");
        message.setMin(0, 0, 0);
        message.setMax(5, 5, 5);
        message.setRadius(1);
        // 5x5x5 bins, total 125
        message.newVariable<flamegpu::id_t>("id");  // unused by current test
    }
    {   // Circle agent
        AgentDescription agent = model.newAgent("agent");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("z");
        agent.newVariable<int>("do_output");  // NEW!
        agent.newVariable<unsigned int>("myBin");  // This will be presumed bin index of the agent, might not use this
        agent.newVariable<unsigned int>("count");  // Store the distance moved here, for validation
        agent.newVariable<unsigned int>("badCount");  // Store how many messages are out of range
        auto af = agent.newFunction("out", out_optional3DNone);  // NEW!
        af.setMessageOutput("location");
        af.setMessageOutputOptional(true);  // NEW!
        agent.newFunction("in", in3D).setMessageInput("location");
    }
    {   // Layer #1
        LayerDescription layer = model.newLayer();
        layer.addAgentFunction(out_optional3DNone);  // NEW!
    }
    {   // Layer #2
        LayerDescription layer = model.newLayer();
        layer.addAgentFunction(in3D);
    }
    CUDASimulation cudaSimulation(model);

    const int AGENT_COUNT = 2049;
    AgentVector population(model.Agent("agent"), AGENT_COUNT);
    // Initialise agents (TODO)
    {
        // Currently population has not been init, so generate an agent population on the fly
        std::mt19937_64 rng;
        std::uniform_real_distribution<float> dist(0.0f, 5.0f);
        for (unsigned int i = 0; i < AGENT_COUNT; i++) {
            AgentVector::Agent instance = population[i];
            float pos[3] = { dist(rng), dist(rng), dist(rng) };
            int do_output = dist(rng) < 4 ? 1 : 0;  // 80% chance of output  // NEW!
            instance.setVariable<float>("x", pos[0]);
            instance.setVariable<float>("y", pos[1]);
            instance.setVariable<float>("z", pos[2]);
            instance.setVariable<int>("do_output", do_output);  // NEW!
            // Solve the bin index
            const unsigned int bin_pos[3] = {
                (unsigned int)(pos[0] / 1),
                (unsigned int)(pos[1] / 1),
                (unsigned int)(pos[2] / 1)
            };
            const unsigned int bin_index =
                bin_pos[2] * 5 * 5 +
                bin_pos[1] * 5 +
                bin_pos[0];
            instance.setVariable<unsigned int>("myBin", bin_index);
            // Create it if it doesn't already exist
            bin_counts[bin_index] += 1;
            if (do_output) {  // NEW!
                bin_counts_optional[bin_index] += 1;  // NEW!
            }
        }
        cudaSimulation.setPopulationData(population);
    }

    // Execute a single step of the model
    cudaSimulation.step();

    // Recover the results and check they match what was expected

    cudaSimulation.getPopulationData(population);
    // Validate each agent has same result
    unsigned int badCountWrong = 0;
    for (AgentVector::Agent ai : population) {
        unsigned int myResult = ai.getVariable<unsigned int>("count");
        if (ai.getVariable<unsigned int>("badCount"))
            badCountWrong++;
        EXPECT_EQ(myResult, 0u);  // NEW!
    }
    EXPECT_EQ(badCountWrong, 0u);
}



TEST(Spatial3DMessageTest, BadRadius) {
    ModelDescription model("Spatial3DMessageTestModel");
    MessageSpatial3D::Description message = model.newMessage<MessageSpatial3D>("location");
    EXPECT_THROW(message.setRadius(0), exception::InvalidArgument);
    EXPECT_THROW(message.setRadius(-10), exception::InvalidArgument);
}
TEST(Spatial3DMessageTest, BadMin) {
    ModelDescription model("Spatial3DMessageTestModel");
    MessageSpatial3D::Description message = model.newMessage<MessageSpatial3D>("location");
    message.setMax(5, 5, 5);
    EXPECT_THROW(message.setMin(5, 0, 0), exception::InvalidArgument);
    EXPECT_THROW(message.setMin(0, 5, 0), exception::InvalidArgument);
    EXPECT_THROW(message.setMin(0, 0, 5), exception::InvalidArgument);
    EXPECT_THROW(message.setMin(6, 0, 0), exception::InvalidArgument);
    EXPECT_THROW(message.setMin(0, 6, 0), exception::InvalidArgument);
    EXPECT_THROW(message.setMin(0, 0, 6), exception::InvalidArgument);
}
TEST(Spatial3DMessageTest, BadMax) {
    ModelDescription model("Spatial3DMessageTestModel");
    MessageSpatial3D::Description message = model.newMessage<MessageSpatial3D>("location");
    message.setMin(5, 5, 5);
    EXPECT_THROW(message.setMax(5, 0, 0), exception::InvalidArgument);
    EXPECT_THROW(message.setMax(0, 5, 0), exception::InvalidArgument);
    EXPECT_THROW(message.setMax(0, 0, 5), exception::InvalidArgument);
    EXPECT_THROW(message.setMax(4, 0, 0), exception::InvalidArgument);
    EXPECT_THROW(message.setMax(0, 4, 0), exception::InvalidArgument);
    EXPECT_THROW(message.setMax(0, 0, 4), exception::InvalidArgument);
}
TEST(Spatial3DMessageTest, UnsetMax) {
    ModelDescription model("Spatial23MessageTestModel");
    MessageSpatial3D::Description message = model.newMessage<MessageSpatial3D>("location");
    message.setMin(5, 5, 5);
    EXPECT_THROW(CUDASimulation m(model), exception::InvalidMessage);
}
TEST(Spatial3DMessageTest, UnsetMin) {
    ModelDescription model("Spatial3DMessageTestModel");
    MessageSpatial3D::Description message = model.newMessage<MessageSpatial3D>("location");
    message.setMin(5, 5, 5);
    EXPECT_THROW(CUDASimulation m(model), exception::InvalidMessage);
}
TEST(Spatial3DMessageTest, reserved_name) {
    ModelDescription model("Spatial3DMessageTestModel");
    MessageSpatial3D::Description message = model.newMessage<MessageSpatial3D>("location");
    EXPECT_THROW(message.newVariable<int>("_"), exception::ReservedName);
}

FLAMEGPU_AGENT_FUNCTION(count3D, MessageSpatial3D, MessageNone) {
    unsigned int count = 0;
    // Count how many messages we received (including our own)
    // This is all those which fall within the 3x3x3 Moore neighbourhood
    for (const auto &message : FLAMEGPU->message_in(0, 0, 0)) {
        count++;
    }
    FLAMEGPU->setVariable<unsigned int>("count", count);
    return ALIVE;
}
TEST(Spatial3DMessageTest, ReadEmpty) {
// What happens if we read a message list before it has been output?
    ModelDescription model("Model");
    {   // Location message
        MessageSpatial3D::Description message = model.newMessage<MessageSpatial3D>("location");
        message.setMin(-3, -3, -3);
        message.setMax(3, 3, 3);
        message.setRadius(2);
        message.newVariable<flamegpu::id_t>("id");  // unused by current test
    }
    {   // Circle agent
        AgentDescription agent = model.newAgent("agent");
        agent.newVariable<unsigned int>("count", 0);  // Count the number of messages read
        agent.newFunction("in", count3D).setMessageInput("location");
    }
    {   // Layer #1
        LayerDescription layer = model.newLayer();
        layer.addAgentFunction(count3D);
    }
    // Create 1 agent
    AgentVector pop_in(model.Agent("agent"), 1);
    CUDASimulation cudaSimulation(model);
    cudaSimulation.setPopulationData(pop_in);
    // Execute model
    EXPECT_NO_THROW(cudaSimulation.step());
    // Check result
    AgentVector pop_out(model.Agent("agent"), 1);
    pop_out[0].setVariable<unsigned int>("count", 1);
    cudaSimulation.getPopulationData(pop_out);
    EXPECT_EQ(pop_out.size(), 1u);
    EXPECT_EQ(pop_out[0].getVariable<unsigned int>("count"), 0u);
}



FLAMEGPU_AGENT_FUNCTION(ArrayOut, MessageNone, MessageSpatial3D) {
    const unsigned int x = FLAMEGPU->getVariable<unsigned int, 3>("index", 0);
    const unsigned int y = FLAMEGPU->getVariable<unsigned int, 3>("index", 1);
    const unsigned int z = FLAMEGPU->getVariable<unsigned int, 3>("index", 2);
    FLAMEGPU->message_out.setVariable<unsigned int, 3>("v", 0, x * 3);
    FLAMEGPU->message_out.setVariable<unsigned int, 3>("v", 1, y * 7);
    FLAMEGPU->message_out.setVariable<unsigned int, 3>("v", 2, z * 11);
    FLAMEGPU->message_out.setLocation(static_cast<float>(x), static_cast<float>(y), static_cast<float>(z));
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(ArrayIn, MessageSpatial3D, MessageNone) {
    const unsigned int x = FLAMEGPU->getVariable<unsigned int, 3>("index", 0);
    const unsigned int y = FLAMEGPU->getVariable<unsigned int, 3>("index", 1);
    const unsigned int z = FLAMEGPU->getVariable<unsigned int, 3>("index", 2);
    for (auto &message : FLAMEGPU->message_in(static_cast<float>(x), static_cast<float>(y), static_cast<float>(z))) {
        if (static_cast<unsigned int>(message.getVariable<float>("x")) == x &&
            static_cast<unsigned int>(message.getVariable<float>("y")) == y &&
            static_cast<unsigned int>(message.getVariable<float>("z")) == z) {
            FLAMEGPU->setVariable<unsigned int, 3>("message_read", 0, message.getVariable<unsigned int, 3>("v", 0));
            FLAMEGPU->setVariable<unsigned int, 3>("message_read", 1, message.getVariable<unsigned int, 3>("v", 1));
            FLAMEGPU->setVariable<unsigned int, 3>("message_read", 2, message.getVariable<unsigned int, 3>("v", 2));
            break;
        }
    }
    return ALIVE;
}
TEST(Spatial3DMessageTest, ArrayVariable) {
    const char* MODEL_NAME = "Model";
    const char* AGENT_NAME = "Agent";
    const char* MESSAGE_NAME = "Message";
    const char* IN_FUNCTION_NAME = "InFunction";
    const char* OUT_FUNCTION_NAME = "OutFunction";
    const char* IN_LAYER_NAME = "InLayer";
    const char* OUT_LAYER_NAME = "OutLayer";
    const unsigned int CBRT_AGENT_COUNT = 11;
    ModelDescription m(MODEL_NAME);
    MessageSpatial3D::Description message = m.newMessage<MessageSpatial3D>(MESSAGE_NAME);
    message.setMin(0, 0, 0);
    message.setMax(static_cast<float>(CBRT_AGENT_COUNT), static_cast<float>(CBRT_AGENT_COUNT), static_cast<float>(CBRT_AGENT_COUNT));
    message.setRadius(1);
    message.newVariable<unsigned int, 3>("v");
    AgentDescription a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int, 3>("index");
    a.newVariable<unsigned int, 3>("message_read", {UINT_MAX, UINT_MAX, UINT_MAX});
    AgentFunctionDescription fo = a.newFunction(OUT_FUNCTION_NAME, ArrayOut);
    fo.setMessageOutput(message);
    AgentFunctionDescription fi = a.newFunction(IN_FUNCTION_NAME, ArrayIn);
    fi.setMessageInput(message);
    LayerDescription lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    AgentVector pop(a, CBRT_AGENT_COUNT * CBRT_AGENT_COUNT * CBRT_AGENT_COUNT);
    int t = 0;
    for (unsigned int i = 0; i < CBRT_AGENT_COUNT; ++i) {
        for (unsigned int j = 0; j < CBRT_AGENT_COUNT; ++j) {
            for (unsigned int k = 0; k < CBRT_AGENT_COUNT; ++k) {
                AgentVector::Agent ai = pop[t++];
                ai.setVariable<unsigned int, 3>("index", { i, j, k });
            }
        }
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        const std::array<unsigned int, 3> index = ai.getVariable<unsigned int, 3>("index");
        std::array<unsigned int, 3> v = ai.getVariable<unsigned int, 3>("message_read");
        ASSERT_EQ(v[0], index[0] * 3);
        ASSERT_EQ(v[1], index[1] * 7);
        ASSERT_EQ(v[2], index[2] * 11);
    }
}
const char* rtc_ArrayOut_func = R"###(
FLAMEGPU_AGENT_FUNCTION(ArrayOut, flamegpu::MessageNone, flamegpu::MessageSpatial3D) {
    const unsigned int x = FLAMEGPU->getVariable<unsigned int, 3>("index", 0);
    const unsigned int y = FLAMEGPU->getVariable<unsigned int, 3>("index", 1);
    const unsigned int z = FLAMEGPU->getVariable<unsigned int, 3>("index", 2);
    FLAMEGPU->message_out.setVariable<unsigned int, 3>("v", 0, x * 3);
    FLAMEGPU->message_out.setVariable<unsigned int, 3>("v", 1, y * 7);
    FLAMEGPU->message_out.setVariable<unsigned int, 3>("v", 2, z * 11);
    FLAMEGPU->message_out.setLocation(static_cast<float>(x), static_cast<float>(y), static_cast<float>(z));
    return flamegpu::ALIVE;
}
)###";
const char* rtc_ArrayIn_func = R"###(
FLAMEGPU_AGENT_FUNCTION(ArrayIn, flamegpu::MessageSpatial3D, flamegpu::MessageNone) {
    const unsigned int x = FLAMEGPU->getVariable<unsigned int, 3>("index", 0);
    const unsigned int y = FLAMEGPU->getVariable<unsigned int, 3>("index", 1);
    const unsigned int z = FLAMEGPU->getVariable<unsigned int, 3>("index", 2);
    for (auto &message : FLAMEGPU->message_in(static_cast<float>(x), static_cast<float>(y), static_cast<float>(z))) {
        if (static_cast<unsigned int>(message.getVariable<float>("x")) == x &&
            static_cast<unsigned int>(message.getVariable<float>("y")) == y &&
            static_cast<unsigned int>(message.getVariable<float>("z")) == z) {
            FLAMEGPU->setVariable<unsigned int, 3>("message_read", 0, message.getVariable<unsigned int, 3>("v", 0));
            FLAMEGPU->setVariable<unsigned int, 3>("message_read", 1, message.getVariable<unsigned int, 3>("v", 1));
            FLAMEGPU->setVariable<unsigned int, 3>("message_read", 2, message.getVariable<unsigned int, 3>("v", 2));
            break;
        }
    }
    return flamegpu::ALIVE;
}
)###";
TEST(RTCSpatial3DMessageTest, ArrayVariable) {
    const char* MODEL_NAME = "Model";
    const char* AGENT_NAME = "Agent";
    const char* MESSAGE_NAME = "Message";
    const char* IN_FUNCTION_NAME = "InFunction";
    const char* OUT_FUNCTION_NAME = "OutFunction";
    const char* IN_LAYER_NAME = "InLayer";
    const char* OUT_LAYER_NAME = "OutLayer";
    const unsigned int CBRT_AGENT_COUNT = 11;
    ModelDescription m(MODEL_NAME);
    MessageSpatial3D::Description message = m.newMessage<MessageSpatial3D>(MESSAGE_NAME);
    message.setMin(0, 0, 0);
    message.setMax(static_cast<float>(CBRT_AGENT_COUNT), static_cast<float>(CBRT_AGENT_COUNT), static_cast<float>(CBRT_AGENT_COUNT));
    message.setRadius(1);
    message.newVariable<unsigned int, 3>("v");
    AgentDescription a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int, 3>("index");
    a.newVariable<unsigned int, 3>("message_read", { UINT_MAX, UINT_MAX, UINT_MAX });
    AgentFunctionDescription fo = a.newRTCFunction(OUT_FUNCTION_NAME, rtc_ArrayOut_func);
    fo.setMessageOutput(message);
    AgentFunctionDescription fi = a.newRTCFunction(IN_FUNCTION_NAME, rtc_ArrayIn_func);
    fi.setMessageInput(message);
    LayerDescription lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    AgentVector pop(a, CBRT_AGENT_COUNT * CBRT_AGENT_COUNT * CBRT_AGENT_COUNT);
    int t = 0;
    for (unsigned int i = 0; i < CBRT_AGENT_COUNT; ++i) {
        for (unsigned int j = 0; j < CBRT_AGENT_COUNT; ++j) {
            for (unsigned int k = 0; k < CBRT_AGENT_COUNT; ++k) {
                AgentVector::Agent ai = pop[t++];
                ai.setVariable<unsigned int, 3>("index", { i, j, k });
            }
        }
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        const std::array<unsigned int, 3> index = ai.getVariable<unsigned int, 3>("index");
        std::array<unsigned int, 3> v = ai.getVariable<unsigned int, 3>("message_read");
        ASSERT_EQ(v[0], index[0] * 3);
        ASSERT_EQ(v[1], index[1] * 7);
        ASSERT_EQ(v[2], index[2] * 11);
    }
}

#if defined(FLAMEGPU_USE_GLM)
FLAMEGPU_AGENT_FUNCTION(ArrayOut_glm, MessageNone, MessageSpatial3D) {
    const unsigned int x = FLAMEGPU->getVariable<unsigned int, 3>("index", 0);
    const unsigned int y = FLAMEGPU->getVariable<unsigned int, 3>("index", 1);
    const unsigned int z = FLAMEGPU->getVariable<unsigned int, 3>("index", 2);
    glm::uvec3 t = glm::uvec3(x * 3, y * 7, z * 11);
    FLAMEGPU->message_out.setVariable<glm::uvec3>("v", t);
    FLAMEGPU->message_out.setLocation(static_cast<float>(x), static_cast<float>(y), static_cast<float>(z));
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(ArrayIn_glm, MessageSpatial3D, MessageNone) {
    const unsigned int x = FLAMEGPU->getVariable<unsigned int, 3>("index", 0);
    const unsigned int y = FLAMEGPU->getVariable<unsigned int, 3>("index", 1);
    const unsigned int z = FLAMEGPU->getVariable<unsigned int, 3>("index", 2);
    for (auto &message : FLAMEGPU->message_in(static_cast<float>(x), static_cast<float>(y), static_cast<float>(z))) {
        if (static_cast<unsigned int>(message.getVariable<float>("x")) == x &&
            static_cast<unsigned int>(message.getVariable<float>("y")) == y &&
            static_cast<unsigned int>(message.getVariable<float>("z")) == z) {
            FLAMEGPU->setVariable<glm::uvec3>("message_read", message.getVariable<glm::uvec3>("v"));
            break;
        }
    }
    return ALIVE;
}
TEST(Spatial3DMessageTest, ArrayVariable_glm) {
    const char* MODEL_NAME = "Model";
    const char* AGENT_NAME = "Agent";
    const char* MESSAGE_NAME = "Message";
    const char* IN_FUNCTION_NAME = "InFunction";
    const char* OUT_FUNCTION_NAME = "OutFunction";
    const char* IN_LAYER_NAME = "InLayer";
    const char* OUT_LAYER_NAME = "OutLayer";
    const unsigned int CBRT_AGENT_COUNT = 11;
    ModelDescription m(MODEL_NAME);
    MessageSpatial3D::Description message = m.newMessage<MessageSpatial3D>(MESSAGE_NAME);
    message.setMin(0, 0, 0);
    message.setMax(static_cast<float>(CBRT_AGENT_COUNT), static_cast<float>(CBRT_AGENT_COUNT), static_cast<float>(CBRT_AGENT_COUNT));
    message.setRadius(1);
    message.newVariable<unsigned int, 3>("v");
    AgentDescription a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int, 3>("index");
    a.newVariable<unsigned int, 3>("message_read", {UINT_MAX, UINT_MAX, UINT_MAX});
    AgentFunctionDescription fo = a.newFunction(OUT_FUNCTION_NAME, ArrayOut_glm);
    fo.setMessageOutput(message);
    AgentFunctionDescription fi = a.newFunction(IN_FUNCTION_NAME, ArrayIn_glm);
    fi.setMessageInput(message);
    LayerDescription lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    AgentVector pop(a, CBRT_AGENT_COUNT * CBRT_AGENT_COUNT * CBRT_AGENT_COUNT);
    int t = 0;
    for (unsigned int i = 0; i < CBRT_AGENT_COUNT; ++i) {
        for (unsigned int j = 0; j < CBRT_AGENT_COUNT; ++j) {
            for (unsigned int k = 0; k < CBRT_AGENT_COUNT; ++k) {
                AgentVector::Agent ai = pop[t++];
                ai.setVariable<unsigned int, 3>("index", { i, j, k });
            }
        }
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        const std::array<unsigned int, 3> index = ai.getVariable<unsigned int, 3>("index");
        std::array<unsigned int, 3> v = ai.getVariable<unsigned int, 3>("message_read");
        ASSERT_EQ(v[0], index[0] * 3);
        ASSERT_EQ(v[1], index[1] * 7);
        ASSERT_EQ(v[2], index[2] * 11);
    }
}
const char* rtc_ArrayOut_func_glm = R"###(
FLAMEGPU_AGENT_FUNCTION(ArrayOut, flamegpu::MessageNone, flamegpu::MessageSpatial3D) {
    const unsigned int x = FLAMEGPU->getVariable<unsigned int, 3>("index", 0);
    const unsigned int y = FLAMEGPU->getVariable<unsigned int, 3>("index", 1);
    const unsigned int z = FLAMEGPU->getVariable<unsigned int, 3>("index", 2);
    glm::uvec3 t = glm::uvec3(x * 3, y * 7, z * 11);
    FLAMEGPU->message_out.setVariable<glm::uvec3>("v", t);
    FLAMEGPU->message_out.setLocation(static_cast<float>(x), static_cast<float>(y), static_cast<float>(z));
    return flamegpu::ALIVE;
}
)###";
const char* rtc_ArrayIn_func_glm = R"###(
FLAMEGPU_AGENT_FUNCTION(ArrayIn, flamegpu::MessageSpatial3D, flamegpu::MessageNone) {
    const unsigned int x = FLAMEGPU->getVariable<unsigned int, 3>("index", 0);
    const unsigned int y = FLAMEGPU->getVariable<unsigned int, 3>("index", 1);
    const unsigned int z = FLAMEGPU->getVariable<unsigned int, 3>("index", 2);
    for (auto &message : FLAMEGPU->message_in(static_cast<float>(x), static_cast<float>(y), static_cast<float>(z))) {
        if (static_cast<unsigned int>(message.getVariable<float>("x")) == x &&
            static_cast<unsigned int>(message.getVariable<float>("y")) == y &&
            static_cast<unsigned int>(message.getVariable<float>("z")) == z) {
            FLAMEGPU->setVariable<glm::uvec3>("message_read", message.getVariable<glm::uvec3>("v"));
            break;
        }
    }
    return flamegpu::ALIVE;
}
)###";
TEST(RTCSpatial3DMessageTest, ArrayVariable_glm) {
    const char* MODEL_NAME = "Model";
    const char* AGENT_NAME = "Agent";
    const char* MESSAGE_NAME = "Message";
    const char* IN_FUNCTION_NAME = "InFunction";
    const char* OUT_FUNCTION_NAME = "OutFunction";
    const char* IN_LAYER_NAME = "InLayer";
    const char* OUT_LAYER_NAME = "OutLayer";
    const unsigned int CBRT_AGENT_COUNT = 11;
    ModelDescription m(MODEL_NAME);
    MessageSpatial3D::Description message = m.newMessage<MessageSpatial3D>(MESSAGE_NAME);
    message.setMin(0, 0, 0);
    message.setMax(static_cast<float>(CBRT_AGENT_COUNT), static_cast<float>(CBRT_AGENT_COUNT), static_cast<float>(CBRT_AGENT_COUNT));
    message.setRadius(1);
    message.newVariable<unsigned int, 3>("v");
    AgentDescription a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int, 3>("index");
    a.newVariable<unsigned int, 3>("message_read", { UINT_MAX, UINT_MAX, UINT_MAX });
    AgentFunctionDescription fo = a.newRTCFunction(OUT_FUNCTION_NAME, rtc_ArrayOut_func_glm);
    fo.setMessageOutput(message);
    AgentFunctionDescription fi = a.newRTCFunction(IN_FUNCTION_NAME, rtc_ArrayIn_func_glm);
    fi.setMessageInput(message);
    LayerDescription lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    AgentVector pop(a, CBRT_AGENT_COUNT * CBRT_AGENT_COUNT * CBRT_AGENT_COUNT);
    int t = 0;
    for (unsigned int i = 0; i < CBRT_AGENT_COUNT; ++i) {
        for (unsigned int j = 0; j < CBRT_AGENT_COUNT; ++j) {
            for (unsigned int k = 0; k < CBRT_AGENT_COUNT; ++k) {
                AgentVector::Agent ai = pop[t++];
                ai.setVariable<unsigned int, 3>("index", { i, j, k });
            }
        }
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        const std::array<unsigned int, 3> index = ai.getVariable<unsigned int, 3>("index");
        std::array<unsigned int, 3> v = ai.getVariable<unsigned int, 3>("message_read");
        ASSERT_EQ(v[0], index[0] * 3);
        ASSERT_EQ(v[1], index[1] * 7);
        ASSERT_EQ(v[2], index[2] * 11);
    }
}
#else
TEST(Spatial3DMessageTest, DISABLED_ArrayVariable_glm) { }
TEST(RTCSpatial3DMessageTest, DISABLED_ArrayVariable_glm) { }
#endif

FLAMEGPU_AGENT_FUNCTION(inWrapped3D, MessageSpatial3D, MessageNone) {
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    const float z1 = FLAMEGPU->getVariable<float>("z");
    const flamegpu::id_t ID = FLAMEGPU->getID();
    unsigned int count = 0;
    unsigned int badCount = 0;
    float xSum = 0;
    float ySum = 0;
    float zSum = 0;
    // Count how many messages we recieved (including our own)
    // This is all those which fall within the 3x3x3 Moore neighbourhood
    // Not our search radius
    for (const auto& message : FLAMEGPU->message_in.wrap(x1, y1, z1)) {
        const float x2 = message.getVirtualX(x1);
        const float y2 = message.getVirtualY(y1);
        const float z2 = message.getVirtualZ(z1);
        float x21 = x2 - x1;
        float y21 = y2 - y1;
        float z21 = z2 - z1;
        const float distance = sqrt(x21 * x21 + y21 * y21 + z21 * z21);
        if (distance > FLAMEGPU->message_in.radius() ||
            (abs(x21) != 2.0f && x2 != x1) ||
            (abs(y21) != 2.0f && y2 != y1) ||
            (abs(z21) != 2.0f && z2 != z1)
        ) {
            badCount++;
        } else {
            count++;
            if (message.getVariable<flamegpu::id_t>("id") != ID) {
                xSum += (x21);
                ySum += (y21);
                zSum += (z21);
            }
        }
    }
    FLAMEGPU->setVariable<unsigned int>("count", count);
    FLAMEGPU->setVariable<unsigned int>("badCount", badCount);
    FLAMEGPU->setVariable<float>("result_x", xSum);
    FLAMEGPU->setVariable<float>("result_y", ySum);
    FLAMEGPU->setVariable<float>("result_z", zSum);
    return ALIVE;
}
void wrapped_3d_test(const float x_offset, const float y_offset, const float z_offset, const float out_of_bounds = 0) {
    std::unordered_map<int, unsigned int> bin_counts;
    // Construct model
    ModelDescription model("Spatial2DMessageTestModel");
    {   // Location message
        MessageSpatial3D::Description message = model.newMessage<MessageSpatial3D>("location");
        message.setMin(0 + x_offset, 0 + y_offset, 0 + z_offset);
        message.setMax(70 + x_offset, 70 + y_offset, 70 + z_offset);
        message.setRadius(3.5);  // With a grid of agents spaced 2 units apart, this configuration should give each agent 8 neighbours (assuming my basic maths guessing works out)
        message.newVariable<flamegpu::id_t>("id");  // unused by current test
    }
    {   // Circle agent
        AgentDescription agent = model.newAgent("agent");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("z");
        agent.newVariable<float>("result_x");  // Sum all virtual X values, and this should equal 0 (or very close)
        agent.newVariable<float>("result_y");  // Sum all virtual X values, and this should equal 0 (or very close)
        agent.newVariable<float>("result_z");  // Sum all virtual X values, and this should equal 0 (or very close)
        agent.newVariable<unsigned int>("count");  // Count how many messages we receive
        agent.newVariable<unsigned int>("badCount");  // Count how many messages we receive that have bad data
        agent.newFunction("out", out_mandatory3D).setMessageOutput("location");
        agent.newFunction("in", inWrapped3D).setMessageInput("location");
    }
    {   // Layer #1
        LayerDescription layer = model.newLayer();
        layer.addAgentFunction(out_mandatory3D);
    }
    {   // Layer #2
        LayerDescription layer = model.newLayer();
        layer.addAgentFunction(inWrapped3D);
    }
    CUDASimulation cudaSimulation(model);

    AgentVector population(model.Agent("agent"), 35u * 35u * 35u);  // This must fit the env dims/radius set out above
    // Initialise agents (TODO)
    {
        // Currently population has not been init, so generate an agent population on the fly
        for (unsigned int i = 0; i < 35u; i++) {
            for (unsigned int j = 0; j < 35u; j++) {
                for (unsigned int k = 0; k < 35u; k++) {
                    unsigned int w =  (i * 35u+ j) * 35u + k;
                    AgentVector::Agent instance = population[w];
                    instance.setVariable<float>("x", i * 2.0f + x_offset + out_of_bounds);
                    instance.setVariable<float>("y", j * 2.0f + y_offset);
                    instance.setVariable<float>("z", k * 2.0f + z_offset);
                }
            }
        }
        cudaSimulation.setPopulationData(population);
    }

    // Execute a single step of the model
    cudaSimulation.step();

    // Recover the results and check they match what was expected
    cudaSimulation.getPopulationData(population);
    // Validate each agent has same result
    for (AgentVector::Agent ai : population) {
        EXPECT_EQ(0.0f, ai.getVariable<float>("result_x"));
        EXPECT_EQ(0.0f, ai.getVariable<float>("result_y"));
        EXPECT_EQ(0.0f, ai.getVariable<float>("result_z"));
        EXPECT_LE(ai.getVariable<unsigned int>("badCount"), 189u);
        EXPECT_EQ(27u, ai.getVariable<unsigned int>("count"));
    }
}
TEST(Spatial3DMessageTest, Wrapped) {
    wrapped_3d_test(0.0f, 0.0f, 0.0f);
}
// Test that it doesn't fall over if the environment min is not 0, with a few configurations
TEST(Spatial3DMessageTest, Wrapped2) {
    wrapped_3d_test(141.0f, -540.0f, 200.0f);
}
TEST(Spatial3DMessageTest, Wrapped3) {
    wrapped_3d_test(-1401.5f, 5640.3f, -2008.8f);
}
#if !defined(FLAMEGPU_SEATBELTS) || FLAMEGPU_SEATBELTS
// Test that FLAMEGPU_SEATBELTS catches out of bounds messages
TEST(Spatial3DMessageTest, Wrapped_OutOfBounds) {
    EXPECT_THROW(wrapped_3d_test(141.0f, -540.0f, 0.0f, 200.0f), exception::DeviceError);
}
#else
TEST(Spatial3DMessageTest, DISABLED_Wrapped_OutOfBounds) { }
#endif
FLAMEGPU_AGENT_FUNCTION(out_mandatory3D_OddStep, MessageNone, MessageSpatial3D) {
    if (FLAMEGPU->getStepCounter() % 2 == 0) {
        FLAMEGPU->message_out.setLocation(
            FLAMEGPU->getVariable<float>("x"),
            FLAMEGPU->getVariable<float>("y"),
            FLAMEGPU->getVariable<float>("z"));
    }
    return ALIVE;
}
FLAMEGPU_HOST_FUNCTION(create_agents_step_zero) {
    if (FLAMEGPU->getStepCounter() == 1) {
        auto agent = FLAMEGPU->agent("agent");
        std::mt19937_64 rng;
        std::uniform_real_distribution<float> dist(0.0f, 5.0f);
        for (unsigned int i = 0; i < 2049; ++i) {
            auto instance = agent.newAgent();
            float pos[3] = { dist(rng), dist(rng), dist(rng) };
            instance.setVariable<float>("x", pos[0]);
            instance.setVariable<float>("y", pos[1]);
            instance.setVariable<float>("z", pos[2]);
        }
    }
}
TEST(Spatial3DMessageTest, buffer_not_init) {
    // This tests that a bug is fixed
    // The bug occurred when a message list, yet to have messages output to it was used as a message input
    // This requires no agents at the first message output function during the second iteration
    // It does 4 iterations to ensure PBM is reset too.
    ModelDescription m("model");
    MessageSpatial3D::Description message = m.newMessage<MessageSpatial3D>("location");
    message.setMin(0, 0, 0);
    message.setMax(5, 5, 5);
    message.setRadius(1);
    AgentDescription agent = m.newAgent("agent");
    agent.newVariable<float>("x");
    agent.newVariable<float>("y");
    agent.newVariable<float>("z");
    agent.newVariable<unsigned int>("count");  // Store the distance moved here, for validation
    agent.newVariable<unsigned int>("badCount");  // Store how many messages are out of range
    AgentFunctionDescription fo = agent.newFunction("out", out_mandatory3D_OddStep);
    fo.setMessageOutput(message);
    fo.setMessageOutputOptional(true);
    AgentFunctionDescription fi = agent.newFunction("in", in3D);
    fi.setMessageInput(message);
    LayerDescription lo = m.newLayer();
    lo.addAgentFunction(fo);
    LayerDescription la = m.newLayer();
    la.addHostFunction(create_agents_step_zero);
    LayerDescription li = m.newLayer();
    li.addAgentFunction(fi);
    // Set pop in model
    CUDASimulation c(m);
    c.SimulationConfig().steps = 4;
    EXPECT_NO_THROW(c.simulate());
}
}  // namespace test_message_spatial3d
}  // namespace flamegpu
