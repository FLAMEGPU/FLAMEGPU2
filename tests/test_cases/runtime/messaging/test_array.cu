#include <chrono>
#include <algorithm>

#include "flamegpu/flame_api.h"
#include "flamegpu/runtime/flamegpu_api.h"

#include "gtest/gtest.h"

namespace test_message_array {
    const char *MODEL_NAME = "Model";
    const char *AGENT_NAME = "Agent";
    const char *MESSAGE_NAME = "Message";
    const char *IN_FUNCTION_NAME = "InFunction";
    const char *OUT_FUNCTION_NAME = "OutFunction";
    const char *IN_LAYER_NAME = "InLayer";
    const char *OUT_LAYER_NAME = "OutLayer";
    const unsigned int AGENT_COUNT = 128;
FLAMEGPU_AGENT_FUNCTION(OutFunction, MsgNone, MsgArray) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("message_write");
    FLAMEGPU->message_out.setVariable<unsigned int>("index_times_3", index * 3);
    FLAMEGPU->message_out.setIndex(index);
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(OutOptionalFunction, MsgNone, MsgArray) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("message_write");
    if (index % 2 == 0) {
        FLAMEGPU->message_out.setVariable<unsigned int>("index_times_3", index * 3);
        FLAMEGPU->message_out.setIndex(index);
    }
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(OutOptionalNoneFunction, MsgNone, MsgArray) {
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(OutBad, MsgNone, MsgArray) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("message_write");
    FLAMEGPU->message_out.setVariable<unsigned int>("index_times_3", index * 3);
    FLAMEGPU->message_out.setIndex(index == 13 ? 0 : index);
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(InFunction, MsgArray, MsgNone) {
    const unsigned int my_index = FLAMEGPU->getVariable<unsigned int>("index");
    const auto &message = FLAMEGPU->message_in.at(my_index);
    FLAMEGPU->setVariable("message_read", message.getVariable<unsigned int>("index_times_3"));
    return ALIVE;
}
TEST(TestMessage_Array, Mandatory) {
    ModelDescription m(MODEL_NAME);
    MsgArray::Description &msg = m.newMessage<MsgArray>(MESSAGE_NAME);
    msg.setLength(AGENT_COUNT);
    msg.newVariable<unsigned int>("index_times_3");
    AgentDescription &a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    a.newVariable<unsigned int>("message_write");
    AgentFunctionDescription &fo = a.newFunction(OUT_FUNCTION_NAME, OutFunction);
    fo.setMessageOutput(msg);
    AgentFunctionDescription &fi = a.newFunction(IN_FUNCTION_NAME, InFunction);
    fi.setMessageInput(msg);
    LayerDescription &lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription &li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Create a list of numbers
    std::array<unsigned int, AGENT_COUNT> numbers;
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        numbers[i] = i;
    }
    // Shuffle the list of numbers
    const unsigned seed = static_cast<unsigned int>(std::chrono::system_clock::now().time_since_epoch().count());
    std::shuffle(numbers.begin(), numbers.end(), std::default_random_engine(seed));
    // Assign the numbers in shuffled order to agents
    AgentPopulation pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = pop.getNextInstance();
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
        ai.setVariable<unsigned int>("message_write", numbers[i]);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = pop.getInstanceAt(i);
        const unsigned int index = ai.getVariable<unsigned int>("index");
        const unsigned int message_read = ai.getVariable<unsigned int>("message_read");
        EXPECT_EQ(index * 3, message_read);
    }
}
TEST(TestMessage_Array, Optional) {
    ModelDescription m(MODEL_NAME);
    MsgArray::Description &msg = m.newMessage<MsgArray>(MESSAGE_NAME);
    msg.setLength(AGENT_COUNT);
    msg.newVariable<unsigned int>("index_times_3");
    AgentDescription &a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    a.newVariable<unsigned int>("message_write");
    AgentFunctionDescription &fo = a.newFunction(OUT_FUNCTION_NAME, OutOptionalFunction);
    fo.setMessageOutput(msg);
    fo.setMessageOutputOptional(true);
    AgentFunctionDescription &fi = a.newFunction(IN_FUNCTION_NAME, InFunction);
    fi.setMessageInput(msg);
    LayerDescription &lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription &li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Create a list of numbers
    std::array<unsigned int, AGENT_COUNT> numbers;
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        numbers[i] = i;
    }
    // Shuffle the list of numbers
    const unsigned seed = static_cast<unsigned int>(std::chrono::system_clock::now().time_since_epoch().count());
    std::shuffle(numbers.begin(), numbers.end(), std::default_random_engine(seed));
    // Assign the numbers in shuffled order to agents
    AgentPopulation pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = pop.getNextInstance();
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
        ai.setVariable<unsigned int>("message_write", numbers[i]);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has same result
    // Validate each agent has same result
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = pop.getInstanceAt(i);
        unsigned int index = ai.getVariable<unsigned int>("index");
        const unsigned int message_read = ai.getVariable<unsigned int>("message_read");
        index = index % 2 == 0 ? index : 0;
        EXPECT_EQ(index * 3, message_read);
    }
}

// Test optional message output, wehre no messages are output.
TEST(TestMessage_Array, OptionalNone) {
    ModelDescription m(MODEL_NAME);
    MsgArray::Description &msg = m.newMessage<MsgArray>(MESSAGE_NAME);
    msg.setLength(AGENT_COUNT);
    msg.newVariable<unsigned int>("index_times_3");
    AgentDescription &a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    a.newVariable<unsigned int>("message_write");
    AgentFunctionDescription &fo = a.newFunction(OUT_FUNCTION_NAME, OutOptionalNoneFunction);
    fo.setMessageOutput(msg);
    fo.setMessageOutputOptional(true);
    AgentFunctionDescription &fi = a.newFunction(IN_FUNCTION_NAME, InFunction);
    fi.setMessageInput(msg);
    LayerDescription &lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription &li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);

    // Generate an arbitrary population.
    AgentPopulation pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = pop.getNextInstance();
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = pop.getInstanceAt(i);
        const unsigned int message_read = ai.getVariable<unsigned int>("message_read");
        // no messages should have been read.
        EXPECT_EQ(0u, message_read);
    }
}

FLAMEGPU_AGENT_FUNCTION(OutSimple, MsgNone, MsgArray) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("index");
    FLAMEGPU->message_out.setIndex(index);
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(MooreTest1, MsgArray, MsgNone) {
    const unsigned int my_index = FLAMEGPU->getVariable<unsigned int>("index");

    // Iterate and check it aligns
    auto filter = FLAMEGPU->message_in(my_index);
    auto msg = filter.begin();
    unsigned int message_read = 0;
    for (int i = -1; i <= 1; ++i) {
        // Skip ourself
        if (i != 0) {
            // Wrap over boundaries
            const unsigned int their_x = (my_index + i + FLAMEGPU->message_in.size()) % FLAMEGPU->message_in.size();
            if (msg->getX() == their_x)
                message_read++;
            ++msg;
        }
    }
    if (msg == filter.end())
        message_read++;
    FLAMEGPU->setVariable<unsigned int>("message_read", message_read);
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(MooreTest2, MsgArray, MsgNone) {
    const unsigned int my_index = FLAMEGPU->getVariable<unsigned int>("index");

    // Iterate and check it aligns
    auto filter = FLAMEGPU->message_in(my_index, 2);
    auto msg = filter.begin();
    unsigned int message_read = 0;
    for (int i = -2; i <= 2; ++i) {
        // Skip ourself
        if (i != 0) {
            // Wrap over boundaries
            const unsigned int their_x = (my_index + i + FLAMEGPU->message_in.size()) % FLAMEGPU->message_in.size();
            if (msg->getX() == their_x)
                message_read++;
            ++msg;
        }
    }
    if (msg == filter.end())
        message_read++;
    FLAMEGPU->setVariable<unsigned int>("message_read", message_read);
    return ALIVE;
}
TEST(TestMessage_Array, Moore1) {
    ModelDescription m(MODEL_NAME);
    MsgArray::Description &msg = m.newMessage<MsgArray>(MESSAGE_NAME);
    msg.setLength(AGENT_COUNT);
    AgentDescription &a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    AgentFunctionDescription &fo = a.newFunction(OUT_FUNCTION_NAME, OutSimple);
    fo.setMessageOutput(msg);
    AgentFunctionDescription &fi = a.newFunction(IN_FUNCTION_NAME, MooreTest1);
    fi.setMessageInput(msg);
    LayerDescription &lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription &li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Assign the numbers in shuffled order to agents
    AgentPopulation pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = pop.getNextInstance();
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has read 8 correct messages
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = pop.getInstanceAt(i);
        const unsigned int message_read = ai.getVariable<unsigned int>("message_read");
        EXPECT_EQ(3u, message_read);
    }
}
TEST(TestMessage_Array, Moore2) {
    ModelDescription m(MODEL_NAME);
    MsgArray::Description &msg = m.newMessage<MsgArray>(MESSAGE_NAME);
    msg.setLength(AGENT_COUNT);
    AgentDescription &a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    AgentFunctionDescription &fo = a.newFunction(OUT_FUNCTION_NAME, OutSimple);
    fo.setMessageOutput(msg);
    AgentFunctionDescription &fi = a.newFunction(IN_FUNCTION_NAME, MooreTest2);
    fi.setMessageInput(msg);
    LayerDescription &lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription &li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Assign the numbers in shuffled order to agents
    AgentPopulation pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = pop.getNextInstance();
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has read 8 correct messages
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = pop.getInstanceAt(i);
        const unsigned int message_read = ai.getVariable<unsigned int>("message_read");
        EXPECT_EQ(5u, message_read);
    }
}
// Exception tests
#ifndef NO_SEATBELTS
TEST(TestMessage_Array, DuplicateOutputException) {
#else
TEST(TestMessage_Array, DISABLED_DuplicateOutputException) {
#endif
    ModelDescription m(MODEL_NAME);
    MsgArray::Description &msg = m.newMessage<MsgArray>(MESSAGE_NAME);
    msg.setLength(AGENT_COUNT);
    msg.newVariable<unsigned int>("index_times_3");
    AgentDescription &a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    a.newVariable<unsigned int>("message_write");
    AgentFunctionDescription &fo = a.newFunction(OUT_FUNCTION_NAME, OutBad);
    fo.setMessageOutput(msg);
    AgentFunctionDescription &fi = a.newFunction(IN_FUNCTION_NAME, InFunction);
    fi.setMessageInput(msg);
    LayerDescription &lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription &li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Create a list of numbers
    std::array<unsigned int, AGENT_COUNT> numbers;
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        numbers[i] = i;
    }
    // Shuffle the list of numbers
    const unsigned seed = static_cast<unsigned int>(std::chrono::system_clock::now().time_since_epoch().count());
    std::shuffle(numbers.begin(), numbers.end(), std::default_random_engine(seed));
    // Assign the numbers in shuffled order to agents
    AgentPopulation pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentInstance ai = pop.getNextInstance();
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
        ai.setVariable<unsigned int>("message_write", numbers[i]);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    EXPECT_THROW(c.step(), ArrayMessageWriteConflict);
}
TEST(TestMessage_Array, ArrayLenZeroException) {
    ModelDescription m(MODEL_NAME);
    MsgArray::Description &msg = m.newMessage<MsgArray>(MESSAGE_NAME);
    EXPECT_THROW(msg.setLength(0), InvalidArgument);
}
TEST(TestMessage_Array, UnsetLength) {
    ModelDescription model(MODEL_NAME);
    model.newMessage<MsgArray>(MESSAGE_NAME);
    // message.setLength(5);  // Intentionally commented out
    EXPECT_THROW(CUDASimulation m(model), InvalidMessage);
}
TEST(TestMessage_Array, reserved_name) {
    ModelDescription model(MODEL_NAME);
    MsgArray::Description &message = model.newMessage<MsgArray>(MESSAGE_NAME);
    EXPECT_THROW(message.newVariable<int>("_"), ReservedName);
}
FLAMEGPU_AGENT_FUNCTION(countArray, MsgArray, MsgNone) {
    unsigned int value = FLAMEGPU->message_in.at(0).getVariable<unsigned int>("value");
    FLAMEGPU->setVariable<unsigned int>("value", value);
    return ALIVE;
}
TEST(TestMessage_Array, ReadEmpty) {
// What happens if we read a message list before it has been output?
    ModelDescription model("Model");
    {   // Location message
        MsgArray::Description &message = model.newMessage<MsgArray>("location");
        message.setLength(2);
        message.newVariable<int>("id");  // unused by current test
        message.newVariable<unsigned int>("value");
    }
    {   // Circle agent
        AgentDescription &agent = model.newAgent("agent");
        agent.newVariable<unsigned int>("value", 32323);  // Count the number of messages read
        agent.newFunction("in", countArray).setMessageInput("location");
    }
    {   // Layer #1
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(countArray);
    }
    // Create 1 agent
    AgentPopulation pop_in(model.Agent("agent"), 1);
    pop_in.getNextInstance();
    CUDASimulation cuda_model(model);
    cuda_model.setPopulationData(pop_in);
    // Execute model
    EXPECT_NO_THROW(cuda_model.step());
    // Check result
    AgentPopulation pop_out(model.Agent("agent"), 1);
    pop_out.getNextInstance().setVariable<unsigned int>("value", 22221);
    cuda_model.getPopulationData(pop_out);
    EXPECT_EQ(pop_out.getCurrentListSize(), 1u);
    auto ai = pop_out.getInstanceAt(0);
    EXPECT_EQ(ai.getVariable<unsigned int>("value"), 0u);  // Unset array msgs should be 0
}

}  // namespace test_message_array
