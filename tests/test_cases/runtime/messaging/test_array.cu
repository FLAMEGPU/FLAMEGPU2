#include "hip/hip_runtime.h"
#include <chrono>
#include <algorithm>

#include "flamegpu/flamegpu.h"

#include "gtest/gtest.h"
namespace flamegpu {


namespace test_message_array {
    const char *MODEL_NAME = "Model";
    const char *AGENT_NAME = "Agent";
    const char *MESSAGE_NAME = "Message";
    const char *IN_FUNCTION_NAME = "InFunction";
    const char *OUT_FUNCTION_NAME = "OutFunction";
    const char *IN_LAYER_NAME = "InLayer";
    const char *OUT_LAYER_NAME = "OutLayer";
    const unsigned int AGENT_COUNT = 128;
    __device__ const unsigned int dAGENT_COUNT = 128;
FLAMEGPU_AGENT_FUNCTION(OutFunction, MessageNone, MessageArray) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("message_write");
    FLAMEGPU->message_out.setVariable<unsigned int>("index_times_3", index * 3);
    FLAMEGPU->message_out.setIndex(index);
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(OutOptionalFunction, MessageNone, MessageArray) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("message_write");
    if (index % 2 == 0) {
        FLAMEGPU->message_out.setVariable<unsigned int>("index_times_3", index * 3);
        FLAMEGPU->message_out.setIndex(index);
    }
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(OutOptionalNoneFunction, MessageNone, MessageArray) {
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(OutBad, MessageNone, MessageArray) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("message_write");
    FLAMEGPU->message_out.setVariable<unsigned int>("index_times_3", index * 3);
    FLAMEGPU->message_out.setIndex(index == 13 ? 0 : index);
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(InFunction, MessageArray, MessageNone) {
    const unsigned int my_index = FLAMEGPU->getVariable<unsigned int>("index");
    const auto &message = FLAMEGPU->message_in.at(my_index);
    FLAMEGPU->setVariable("message_read", message.getVariable<unsigned int>("index_times_3"));
    return ALIVE;
}
TEST(TestMessage_Array, Mandatory) {
    ModelDescription m(MODEL_NAME);
    MessageArray::Description &message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    message.newVariable<unsigned int>("index_times_3");
    AgentDescription &a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    a.newVariable<unsigned int>("message_write");
    AgentFunctionDescription &fo = a.newFunction(OUT_FUNCTION_NAME, OutFunction);
    fo.setMessageOutput(message);
    AgentFunctionDescription &fi = a.newFunction(IN_FUNCTION_NAME, InFunction);
    fi.setMessageInput(message);
    LayerDescription &lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription &li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Create a list of numbers
    std::array<unsigned int, AGENT_COUNT> numbers;
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        numbers[i] = i;
    }
    // Shuffle the list of numbers
    const unsigned seed = static_cast<unsigned int>(std::chrono::system_clock::now().time_since_epoch().count());
    std::shuffle(numbers.begin(), numbers.end(), std::default_random_engine(seed));
    // Assign the numbers in shuffled order to agents
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
        ai.setVariable<unsigned int>("message_write", numbers[i]);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        const unsigned int index = ai.getVariable<unsigned int>("index");
        const unsigned int message_read = ai.getVariable<unsigned int>("message_read");
        EXPECT_EQ(index * 3, message_read);
    }
}
TEST(TestMessage_Array, Optional) {
    ModelDescription m(MODEL_NAME);
    MessageArray::Description &message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    message.newVariable<unsigned int>("index_times_3");
    AgentDescription &a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    a.newVariable<unsigned int>("message_write");
    AgentFunctionDescription &fo = a.newFunction(OUT_FUNCTION_NAME, OutOptionalFunction);
    fo.setMessageOutput(message);
    fo.setMessageOutputOptional(true);
    AgentFunctionDescription &fi = a.newFunction(IN_FUNCTION_NAME, InFunction);
    fi.setMessageInput(message);
    LayerDescription &lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription &li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Create a list of numbers
    std::array<unsigned int, AGENT_COUNT> numbers;
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        numbers[i] = i;
    }
    // Shuffle the list of numbers
    const unsigned seed = static_cast<unsigned int>(std::chrono::system_clock::now().time_since_epoch().count());
    std::shuffle(numbers.begin(), numbers.end(), std::default_random_engine(seed));
    // Assign the numbers in shuffled order to agents
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
        ai.setVariable<unsigned int>("message_write", numbers[i]);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        unsigned int index = ai.getVariable<unsigned int>("index");
        const unsigned int message_read = ai.getVariable<unsigned int>("message_read");
        index = index % 2 == 0 ? index : 0;
        EXPECT_EQ(index * 3, message_read);
    }
}

// Test optional message output, wehre no messages are output.
TEST(TestMessage_Array, OptionalNone) {
    ModelDescription m(MODEL_NAME);
    MessageArray::Description &message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    message.newVariable<unsigned int>("index_times_3");
    AgentDescription &a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    a.newVariable<unsigned int>("message_write");
    AgentFunctionDescription &fo = a.newFunction(OUT_FUNCTION_NAME, OutOptionalNoneFunction);
    fo.setMessageOutput(message);
    fo.setMessageOutputOptional(true);
    AgentFunctionDescription &fi = a.newFunction(IN_FUNCTION_NAME, InFunction);
    fi.setMessageInput(message);
    LayerDescription &lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription &li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);

    // Generate an arbitrary population.
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        const unsigned int message_read = ai.getVariable<unsigned int>("message_read");
        // no messages should have been read.
        EXPECT_EQ(0u, message_read);
    }
}

FLAMEGPU_AGENT_FUNCTION(OutSimple, MessageNone, MessageArray) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("index");
    FLAMEGPU->message_out.setIndex(index);
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(MooreTest1W, MessageArray, MessageNone) {
    const unsigned int my_index = FLAMEGPU->getVariable<unsigned int>("index");

    // Iterate and check it aligns
    auto filter = FLAMEGPU->message_in.wrap(my_index);
    auto message = filter.begin();
    unsigned int message_read = 0;
    for (int i = -1; i <= 1; ++i) {
        // Skip ourself
        if (i != 0) {
            // Wrap over boundaries
            const unsigned int their_x = (my_index + i + FLAMEGPU->message_in.size()) % FLAMEGPU->message_in.size();
            if (message->getX() == their_x)
                message_read++;
            ++message;
        }
    }
    FLAMEGPU->setVariable<unsigned int>("message_read", message_read);
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(MooreTest2W, MessageArray, MessageNone) {
    const unsigned int my_index = FLAMEGPU->getVariable<unsigned int>("index");

    // Iterate and check it aligns
    auto filter = FLAMEGPU->message_in.wrap(my_index, 2);
    auto message = filter.begin();
    unsigned int message_read = 0;
    for (int i = -2; i <= 2; ++i) {
        // Skip ourself
        if (i != 0) {
            // Wrap over boundaries
            const unsigned int their_x = (my_index + i + FLAMEGPU->message_in.size()) % FLAMEGPU->message_in.size();
            if (message->getX() == their_x)
                message_read++;
            ++message;
        }
    }
    FLAMEGPU->setVariable<unsigned int>("message_read", message_read);
    return ALIVE;
}
TEST(TestMessage_Array, Moore1W) {
    ModelDescription m(MODEL_NAME);
    MessageArray::Description &message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    AgentDescription &a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    AgentFunctionDescription &fo = a.newFunction(OUT_FUNCTION_NAME, OutSimple);
    fo.setMessageOutput(message);
    AgentFunctionDescription &fi = a.newFunction(IN_FUNCTION_NAME, MooreTest1W);
    fi.setMessageInput(message);
    LayerDescription &lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription &li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Assign the numbers in shuffled order to agents
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has read 8 correct messages
    for (AgentVector::Agent ai : pop) {
        const unsigned int message_read = ai.getVariable<unsigned int>("message_read");
        EXPECT_EQ(2u, message_read);
    }
}
TEST(TestMessage_Array, Moore2W) {
    ModelDescription m(MODEL_NAME);
    MessageArray::Description &message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    AgentDescription &a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    AgentFunctionDescription &fo = a.newFunction(OUT_FUNCTION_NAME, OutSimple);
    fo.setMessageOutput(message);
    AgentFunctionDescription &fi = a.newFunction(IN_FUNCTION_NAME, MooreTest2W);
    fi.setMessageInput(message);
    LayerDescription &lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription &li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Assign the numbers in shuffled order to agents
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has read 8 correct messages
    for (AgentVector::Agent ai : pop) {
        const unsigned int message_read = ai.getVariable<unsigned int>("message_read");
        EXPECT_EQ(4u, message_read);
    }
}
// Exception tests
#if !defined(SEATBELTS) || SEATBELTS
TEST(TestMessage_Array, DuplicateOutputException) {
#else
TEST(TestMessage_Array, DISABLED_DuplicateOutputException) {
#endif
    ModelDescription m(MODEL_NAME);
    MessageArray::Description &message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    message.newVariable<unsigned int>("index_times_3");
    AgentDescription &a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("message_write");
    AgentFunctionDescription &fo = a.newFunction(OUT_FUNCTION_NAME, OutBad);
    fo.setMessageOutput(message);
    AgentFunctionDescription &fi = a.newFunction(IN_FUNCTION_NAME, InFunction);
    fi.setMessageInput(message);
    LayerDescription &lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription &li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Create a list of numbers
    std::array<unsigned int, AGENT_COUNT> numbers;
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        numbers[i] = i;
    }
    // Shuffle the list of numbers
    const unsigned seed = static_cast<unsigned int>(std::chrono::system_clock::now().time_since_epoch().count());
    std::shuffle(numbers.begin(), numbers.end(), std::default_random_engine(seed));
    // Assign the numbers in shuffled order to agents
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("message_write", i);  // numbers[i]
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    EXPECT_THROW(c.step(), exception::ArrayMessageWriteConflict);
}
TEST(TestMessage_Array, ArrayLenZeroException) {
    ModelDescription m(MODEL_NAME);
    MessageArray::Description &message = m.newMessage<MessageArray>(MESSAGE_NAME);
    EXPECT_THROW(message.setLength(0), exception::InvalidArgument);
}
TEST(TestMessage_Array, UnsetLength) {
    ModelDescription model(MODEL_NAME);
    model.newMessage<MessageArray>(MESSAGE_NAME);
    // message.setLength(5);  // Intentionally commented out
    EXPECT_THROW(CUDASimulation m(model), exception::InvalidMessage);
}
TEST(TestMessage_Array, reserved_name) {
    ModelDescription model(MODEL_NAME);
    MessageArray::Description &message = model.newMessage<MessageArray>(MESSAGE_NAME);
    EXPECT_THROW(message.newVariable<int>("_"), exception::ReservedName);
}
FLAMEGPU_AGENT_FUNCTION(countArray, MessageArray, MessageNone) {
    unsigned int value = FLAMEGPU->message_in.at(0).getVariable<unsigned int>("value");
    FLAMEGPU->setVariable<unsigned int>("value", value);
    return ALIVE;
}
TEST(TestMessage_Array, ReadEmpty) {
// What happens if we read a message list before it has been output?
    ModelDescription model("Model");
    {   // Location message
        MessageArray::Description &message = model.newMessage<MessageArray>("location");
        message.setLength(2);
        message.newVariable<int>("id");  // unused by current test
        message.newVariable<unsigned int>("value");
    }
    {   // Circle agent
        AgentDescription &agent = model.newAgent("agent");
        agent.newVariable<unsigned int>("value", 32323);  // Count the number of messages read
        agent.newFunction("in", countArray).setMessageInput("location");
    }
    {   // Layer #1
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(countArray);
    }
    // Create 1 agent
    AgentVector pop_in(model.Agent("agent"), 1);
    CUDASimulation cudaSimulation(model);
    cudaSimulation.setPopulationData(pop_in);
    // Execute model
    EXPECT_NO_THROW(cudaSimulation.step());
    // Check result
    AgentVector pop_out(model.Agent("agent"), 1);
    pop_out[0].setVariable<unsigned int>("value", 22221);
    cudaSimulation.getPopulationData(pop_out);
    EXPECT_EQ(pop_out.size(), 1u);
    auto ai = pop_out[0];
    EXPECT_EQ(ai.getVariable<unsigned int>("value"), 0u);  // Unset array messages should be 0
}
#if !defined(SEATBELTS) || SEATBELTS
FLAMEGPU_AGENT_FUNCTION(InMooreWrapOutOfBoundsX, MessageArray, MessageNone) {
    for (auto a : FLAMEGPU->message_in.wrap(dAGENT_COUNT)) {
        FLAMEGPU->setVariable<unsigned int>("index", a.getVariable<unsigned int>("index_times_3"));
    }
    return ALIVE;
}
TEST(TestMessage_Array, MooreWrap_InitOutOfBoundsX) {
#else
TEST(TestMessage_Array, DISABLED_MooreWrap_InitOutOfBoundsX) {
#endif
    ModelDescription m(MODEL_NAME);
    MessageArray::Description& message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    message.newVariable<unsigned int>("index_times_3");
    AgentDescription& a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    a.newVariable<unsigned int>("message_write");
    AgentFunctionDescription& fo = a.newFunction(OUT_FUNCTION_NAME, OutFunction);
    fo.setMessageOutput(message);
    AgentFunctionDescription& fi = a.newFunction(IN_FUNCTION_NAME, InMooreWrapOutOfBoundsX);
    fi.setMessageInput(message);
    LayerDescription& lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription& li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Assign the numbers in shuffled order to agents
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
        ai.setVariable<unsigned int>("message_write", i);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    EXPECT_THROW(c.step(), flamegpu::exception::DeviceError);
}
#if !defined(SEATBELTS) || SEATBELTS
FLAMEGPU_AGENT_FUNCTION(InMooreWrapBadRadius1, MessageArray, MessageNone) {
    for (auto a : FLAMEGPU->message_in.wrap(0, 0)) {
        FLAMEGPU->setVariable<unsigned int>("index", a.getVariable<unsigned int>("index_times_3"));
    }
    return ALIVE;
}
TEST(TestMessage_Array, MooreWrap_BadRadius1) {
#else
TEST(TestMessage_Array, DISABLED_MooreWrap_BadRadius1) {
#endif
    ModelDescription m(MODEL_NAME);
    MessageArray::Description& message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    message.newVariable<unsigned int>("index_times_3");
    AgentDescription& a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    a.newVariable<unsigned int>("message_write");
    AgentFunctionDescription& fo = a.newFunction(OUT_FUNCTION_NAME, OutFunction);
    fo.setMessageOutput(message);
    AgentFunctionDescription& fi = a.newFunction(IN_FUNCTION_NAME, InMooreWrapBadRadius1);
    fi.setMessageInput(message);
    LayerDescription& lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription& li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Assign the numbers in shuffled order to agents
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
        ai.setVariable<unsigned int>("message_write", i);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    EXPECT_THROW(c.step(), flamegpu::exception::DeviceError);
}
#if !defined(SEATBELTS) || SEATBELTS
FLAMEGPU_AGENT_FUNCTION(InMooreWrapBadRadius2, MessageArray, MessageNone) {
    for (auto a : FLAMEGPU->message_in.wrap(0, 64)) {
        FLAMEGPU->setVariable<unsigned int>("index", a.getVariable<unsigned int>("index_times_3"));
    }
    return ALIVE;
}
TEST(TestMessage_Array, MooreWrap_BadRadius2) {
#else
TEST(TestMessage_Array, DISABLED_MooreWrap_BadRadius2) {
#endif
    ModelDescription m(MODEL_NAME);
    MessageArray::Description& message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    message.newVariable<unsigned int>("index_times_3");
    AgentDescription& a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    a.newVariable<unsigned int>("message_write");
    AgentFunctionDescription& fo = a.newFunction(OUT_FUNCTION_NAME, OutFunction);
    fo.setMessageOutput(message);
    AgentFunctionDescription& fi = a.newFunction(IN_FUNCTION_NAME, InMooreWrapBadRadius2);
    fi.setMessageInput(message);
    LayerDescription& lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription& li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Assign the numbers in shuffled order to agents
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
        ai.setVariable<unsigned int>("message_write", i);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    EXPECT_THROW(c.step(), flamegpu::exception::DeviceError);
}
#if !defined(SEATBELTS) || SEATBELTS
FLAMEGPU_AGENT_FUNCTION(InMooreOutOfBoundsX, MessageArray, MessageNone) {
    for (auto a : FLAMEGPU->message_in(dAGENT_COUNT)) {
        FLAMEGPU->setVariable<unsigned int>("index", a.getVariable<unsigned int>("index_times_3"));
    }
    return ALIVE;
}
TEST(TestMessage_Array, Moore_InitOutOfBoundsX) {
#else
TEST(TestMessage_Array, DISABLED_Moore_InitOutOfBoundsX) {
#endif
    ModelDescription m(MODEL_NAME);
    MessageArray::Description& message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    message.newVariable<unsigned int>("index_times_3");
    AgentDescription& a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    a.newVariable<unsigned int>("message_write");
    AgentFunctionDescription& fo = a.newFunction(OUT_FUNCTION_NAME, OutFunction);
    fo.setMessageOutput(message);
    AgentFunctionDescription& fi = a.newFunction(IN_FUNCTION_NAME, InMooreOutOfBoundsX);
    fi.setMessageInput(message);
    LayerDescription& lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription& li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Assign the numbers in shuffled order to agents
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
        ai.setVariable<unsigned int>("message_write", i);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    EXPECT_THROW(c.step(), flamegpu::exception::DeviceError);
}
#if !defined(SEATBELTS) || SEATBELTS
FLAMEGPU_AGENT_FUNCTION(InMooreBadRadius, MessageArray, MessageNone) {
    for (auto a : FLAMEGPU->message_in(0, 0)) {
        FLAMEGPU->setVariable<unsigned int>("index", a.getVariable<unsigned int>("index_times_3"));
    }
    return ALIVE;
}
TEST(TestMessage_Array, Moore_BadRadius) {
#else
TEST(TestMessage_Array, DISABLED_Moore_BadRadius) {
#endif
    ModelDescription m(MODEL_NAME);
    MessageArray::Description& message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    message.newVariable<unsigned int>("index_times_3");
    AgentDescription& a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int>("message_read", UINT_MAX);
    a.newVariable<unsigned int>("message_write");
    AgentFunctionDescription& fo = a.newFunction(OUT_FUNCTION_NAME, OutFunction);
    fo.setMessageOutput(message);
    AgentFunctionDescription& fi = a.newFunction(IN_FUNCTION_NAME, InMooreBadRadius);
    fi.setMessageInput(message);
    LayerDescription& lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription& li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Assign the numbers in shuffled order to agents
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("index", i);
        ai.setVariable<unsigned int>("message_read", UINT_MAX);
        ai.setVariable<unsigned int>("message_write", i);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    EXPECT_THROW(c.step(), flamegpu::exception::DeviceError);
}

/*
 * Test for fixed size grids with various com radii to check edge cases + expected cases.
 * 3x3x3 issue highlighted by see https://github.com/FLAMEGPU/FLAMEGPU2/issues/547
 */
FLAMEGPU_AGENT_FUNCTION(OutSimpleX, MessageNone, MessageArray) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("index");
    const unsigned int x = FLAMEGPU->getVariable<unsigned int>("x");
    FLAMEGPU->message_out.setVariable("index", index);
    FLAMEGPU->message_out.setIndex(x);
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(MooreWrapTestXC, MessageArray, MessageNone) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("index");
    const unsigned int x = FLAMEGPU->getVariable<unsigned int>("x");
    const unsigned int COMRADIUS = FLAMEGPU->environment.getProperty<unsigned int>("COMRADIUS");
    // Iterate message list counting how many messages were read
    unsigned int count = 0;
    for (const auto &message : FLAMEGPU->message_in.wrap(x, COMRADIUS)) {
        // @todo - check its the correct messages?
        count++;
    }
    FLAMEGPU->setVariable<unsigned int>("message_read", count);
    return ALIVE;
}

void test_moore_wrap_comradius(
    const unsigned int GRID_WIDTH,
    const unsigned int COMRADIUS
    ) {
    // Calc the population
    const unsigned int agentCount = GRID_WIDTH;

    // Define the model
    ModelDescription model("MooreXR");

    // Use an env var for the communication radius to use, rather than a __device__ or a #define.
    EnvironmentDescription &env = model.Environment();
    env.newProperty<unsigned int>("COMRADIUS", COMRADIUS);

    // Define the message
    MessageArray::Description &message = model.newMessage<MessageArray>(MESSAGE_NAME);
    message.newVariable<unsigned int>("index");
    message.setLength(GRID_WIDTH);
    AgentDescription &agent = model.newAgent(AGENT_NAME);
    agent.newVariable<unsigned int>("index");
    agent.newVariable<unsigned int>("x");
    agent.newVariable<unsigned int>("message_read", UINT_MAX);
    // Define the function and layers.
    AgentFunctionDescription &outputFunction = agent.newFunction("OutSimpleX", OutSimpleX);
    outputFunction.setMessageOutput(message);
    AgentFunctionDescription &inputFunction = agent.newFunction("MooreWrapTestXC", MooreWrapTestXC);
    inputFunction.setMessageInput(message);
    model.newLayer().addAgentFunction(outputFunction);
    LayerDescription &li = model.newLayer();
    li.addAgentFunction(inputFunction);
    // Assign the numbers in shuffled order to agents
    AgentVector population(agent, agentCount);
    for (unsigned int x = 0; x < GRID_WIDTH; x++) {
        unsigned int idx = x;
        AgentVector::Agent instance = population[idx];
        instance.setVariable<unsigned int>("index", idx);
        instance.setVariable<unsigned int>("x", x);
        instance.setVariable<unsigned int>("message_read", UINT_MAX);
    }
    // Set pop in model
    CUDASimulation simulation(model);
    simulation.setPopulationData(population);

    if ((COMRADIUS * 2) + 1 <= GRID_WIDTH) {
        simulation.step();
        simulation.getPopulationData(population);
        // Validate each agent has read correct messages

        // Calc the expected number of messages. This depoends on comm radius for wrapped moore neighbourhood
        const unsigned int expected_count = COMRADIUS * 2;

        for (AgentVector::Agent instance : population) {
            const unsigned int message_read = instance.getVariable<unsigned int>("message_read");
            ASSERT_EQ(expected_count, message_read);
        }
    } else {
        // If the comradius would lead to double message reads, a device error is thrown when SEATBELTS is enabled
        // Behaviour is otherwise undefined
#if !defined(SEATBELTS) || SEATBELTS
        EXPECT_THROW(simulation.step(), flamegpu::exception::DeviceError);
#endif
    }
}
// Test a range of environment sizes for comradius of 1, including small sizes which are an edge case, with wrapping.
TEST(TestMessage_Array, MooreWrapR1) {
    test_moore_wrap_comradius(1, 1);
    test_moore_wrap_comradius(2, 1);
    test_moore_wrap_comradius(3, 1);
    test_moore_wrap_comradius(4, 1);
}

// Test a range of environment sizes for comradius of 2, including small sizes which are an edge case, with wrapped communication.
TEST(TestMessage_Array, MooreWrapR2) {
    test_moore_wrap_comradius(1, 2);
    test_moore_wrap_comradius(2, 2);
    test_moore_wrap_comradius(3, 2);
    test_moore_wrap_comradius(4, 2);
    test_moore_wrap_comradius(5, 2);
    test_moore_wrap_comradius(6, 2);
}

FLAMEGPU_AGENT_FUNCTION(MooreTestXC, MessageArray, MessageNone) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("index");
    const unsigned int x = FLAMEGPU->getVariable<unsigned int>("x");
    const unsigned int COMRADIUS = FLAMEGPU->environment.getProperty<unsigned int>("COMRADIUS");
    // Iterate message list counting how many messages were read
    unsigned int count = 0;
    for (const auto &message : FLAMEGPU->message_in(x, COMRADIUS)) {
        // @todo - check its the correct messages?
        count++;
    }
    FLAMEGPU->setVariable<unsigned int>("message_read", count);
    return ALIVE;
}

void test_mooore_comradius(
    const unsigned int GRID_WIDTH,
    const unsigned int COMRADIUS
    ) {
    // Calc the population
    const unsigned int agentCount = GRID_WIDTH;

    // Define the model
    ModelDescription model("MooreXR");

    // Use an env var for the communication radius to use, rather than a __device__ or a #define.
    EnvironmentDescription &env = model.Environment();
    env.newProperty<unsigned int>("COMRADIUS", COMRADIUS);

    // Define the message
    MessageArray::Description &message = model.newMessage<MessageArray>(MESSAGE_NAME);
    message.newVariable<unsigned int>("index");
    message.setLength(GRID_WIDTH);
    AgentDescription &agent = model.newAgent(AGENT_NAME);
    agent.newVariable<unsigned int>("index");
    agent.newVariable<unsigned int>("x");
    agent.newVariable<unsigned int>("message_read", UINT_MAX);
    // Define the function and layers.
    AgentFunctionDescription &outputFunction = agent.newFunction("OutSimpleX", OutSimpleX);
    outputFunction.setMessageOutput(message);
    AgentFunctionDescription &inputFunction = agent.newFunction("MooreTestXC", MooreTestXC);
    inputFunction.setMessageInput(message);
    model.newLayer().addAgentFunction(outputFunction);
    LayerDescription &li = model.newLayer();
    li.addAgentFunction(inputFunction);
    // Assign the numbers in shuffled order to agents
    AgentVector population(agent, agentCount);
    for (unsigned int x = 0; x < GRID_WIDTH; x++) {
        unsigned int idx = x;
        AgentVector::Agent instance = population[idx];
        instance.setVariable<unsigned int>("index", idx);
        instance.setVariable<unsigned int>("x", x);
        instance.setVariable<unsigned int>("message_read", UINT_MAX);
    }
    // Set pop in model
    CUDASimulation simulation(model);
    simulation.setPopulationData(population);
    simulation.step();
    simulation.getPopulationData(population);
    unsigned int right_count = 0;
    // Validate each agent has read correct number of messages
    for (AgentVector::Agent instance : population) {
        const unsigned int x = instance.getVariable<unsigned int>("x");
        const unsigned int message_read = instance.getVariable<unsigned int>("message_read");

        unsigned int expected_read = 1;
        expected_read *= (std::min<int>(static_cast<int>(x + COMRADIUS), static_cast<int>(GRID_WIDTH) - 1) - std::max<int>(static_cast<int>(x) - static_cast<int>(COMRADIUS), 0) + 1);
        expected_read--;
        // ASSERT_EQ(message_read, expected_read);
        if (message_read == expected_read)
            right_count++;
    }
    ASSERT_EQ(right_count, population.size());
}
// Test a range of environment sizes for comradius of 1, including small sizes which are an edge case.
TEST(TestMessage_Array, MooreR1) {
    test_mooore_comradius(1, 1);
    test_mooore_comradius(2, 1);
    test_mooore_comradius(3, 1);
    test_mooore_comradius(4, 1);
}

// Test a range of environment sizes for comradius of 2, including small sizes which are an edge case.
TEST(TestMessage_Array, MooreR2) {
    test_mooore_comradius(1, 2);
    test_mooore_comradius(2, 2);
    test_mooore_comradius(3, 2);
    test_mooore_comradius(4, 2);
    test_mooore_comradius(5, 2);
    test_mooore_comradius(6, 2);
}

FLAMEGPU_AGENT_FUNCTION(ArrayOut, MessageNone, MessageArray) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("index");
    FLAMEGPU->message_out.setVariable<unsigned int, 3>("v", 0, index * 3);
    FLAMEGPU->message_out.setVariable<unsigned int, 3>("v", 1, index * 7);
    FLAMEGPU->message_out.setVariable<unsigned int, 3>("v", 2, index * 11);
    FLAMEGPU->message_out.setIndex(index);
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(ArrayIn, MessageArray, MessageNone) {
    const unsigned int my_index = FLAMEGPU->getVariable<unsigned int>("index");
    const auto &message = FLAMEGPU->message_in.at(my_index);
    FLAMEGPU->setVariable<unsigned int, 3>("message_read", 0, message.getVariable<unsigned int, 3>("v", 0));
    FLAMEGPU->setVariable<unsigned int, 3>("message_read", 1, message.getVariable<unsigned int, 3>("v", 1));
    FLAMEGPU->setVariable<unsigned int, 3>("message_read", 2, message.getVariable<unsigned int, 3>("v", 2));
    return ALIVE;
}
TEST(TestMessage_Array, ArrayVariable) {
    ModelDescription m(MODEL_NAME);
    MessageArray::Description &message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    message.newVariable<unsigned int, 3>("v");
    AgentDescription &a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int, 3>("message_read", {UINT_MAX, UINT_MAX, UINT_MAX});
    AgentFunctionDescription &fo = a.newFunction(OUT_FUNCTION_NAME, ArrayOut);
    fo.setMessageOutput(message);
    AgentFunctionDescription &fi = a.newFunction(IN_FUNCTION_NAME, ArrayIn);
    fi.setMessageInput(message);
    LayerDescription &lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription &li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Assign the numbers in shuffled order to agents
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("index", i);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        const unsigned int index = ai.getVariable<unsigned int>("index");
        std::array<unsigned int, 3> v = ai.getVariable<unsigned int, 3>("message_read");
        ASSERT_EQ(v[0], index * 3);
        ASSERT_EQ(v[1], index * 7);
        ASSERT_EQ(v[2], index * 11);
    }
}
const char* rtc_ArrayOut_func = R"###(
FLAMEGPU_AGENT_FUNCTION(ArrayOut, flamegpu::MessageNone, flamegpu::MessageArray) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("index");
    FLAMEGPU->message_out.setVariable<unsigned int, 3>("v", 0, index * 3);
    FLAMEGPU->message_out.setVariable<unsigned int, 3>("v", 1, index * 7);
    FLAMEGPU->message_out.setVariable<unsigned int, 3>("v", 2, index * 11);
    FLAMEGPU->message_out.setIndex(index);
    return flamegpu::ALIVE;
}
)###";
const char* rtc_ArrayIn_func = R"###(
FLAMEGPU_AGENT_FUNCTION(ArrayIn, flamegpu::MessageArray, flamegpu::MessageNone) {
    const unsigned int my_index = FLAMEGPU->getVariable<unsigned int>("index");
    const auto& message = FLAMEGPU->message_in.at(my_index);
    FLAMEGPU->setVariable<unsigned int, 3>("message_read", 0, message.getVariable<unsigned int, 3>("v", 0));
    FLAMEGPU->setVariable<unsigned int, 3>("message_read", 1, message.getVariable<unsigned int, 3>("v", 1));
    FLAMEGPU->setVariable<unsigned int, 3>("message_read", 2, message.getVariable<unsigned int, 3>("v", 2));
    return flamegpu::ALIVE;
}
)###";
TEST(TestRTCMessage_Array, ArrayVariable) {
    ModelDescription m(MODEL_NAME);
    MessageArray::Description& message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    message.newVariable<unsigned int, 3>("v");
    AgentDescription& a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int, 3>("message_read", { UINT_MAX, UINT_MAX, UINT_MAX });
    AgentFunctionDescription& fo = a.newRTCFunction(OUT_FUNCTION_NAME, rtc_ArrayOut_func);
    fo.setMessageOutput(message);
    AgentFunctionDescription& fi = a.newRTCFunction(IN_FUNCTION_NAME, rtc_ArrayIn_func);
    fi.setMessageInput(message);
    LayerDescription& lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription& li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Assign the numbers in shuffled order to agents
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("index", i);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        const unsigned int index = ai.getVariable<unsigned int>("index");
        std::array<unsigned int, 3> v = ai.getVariable<unsigned int, 3>("message_read");
        ASSERT_EQ(v[0], index * 3);
        ASSERT_EQ(v[1], index * 7);
        ASSERT_EQ(v[2], index * 11);
    }
}

#ifdef USE_GLM
FLAMEGPU_AGENT_FUNCTION(ArrayOut_glm, MessageNone, MessageArray) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("index");
    glm::uvec3 t = glm::uvec3(index * 3, index * 7, index * 11);
    FLAMEGPU->message_out.setVariable<glm::uvec3>("v", t);
    FLAMEGPU->message_out.setIndex(index);
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(ArrayIn_glm, MessageArray, MessageNone) {
    const unsigned int my_index = FLAMEGPU->getVariable<unsigned int>("index");
    const auto& message = FLAMEGPU->message_in.at(my_index);
    FLAMEGPU->setVariable<glm::uvec3>("message_read", message.getVariable<glm::uvec3>("v"));
    return ALIVE;
}
TEST(TestMessage_Array, ArrayVariable_glm) {
    ModelDescription m(MODEL_NAME);
    MessageArray::Description& message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    message.newVariable<unsigned int, 3>("v");
    AgentDescription& a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int, 3>("message_read", { UINT_MAX, UINT_MAX, UINT_MAX });
    AgentFunctionDescription& fo = a.newFunction(OUT_FUNCTION_NAME, ArrayOut_glm);
    fo.setMessageOutput(message);
    AgentFunctionDescription& fi = a.newFunction(IN_FUNCTION_NAME, ArrayIn_glm);
    fi.setMessageInput(message);
    LayerDescription& lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription& li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Assign the numbers in shuffled order to agents
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("index", i);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        const unsigned int index = ai.getVariable<unsigned int>("index");
        std::array<unsigned int, 3> v = ai.getVariable<unsigned int, 3>("message_read");
        ASSERT_EQ(v[0], index * 3);
        ASSERT_EQ(v[1], index * 7);
        ASSERT_EQ(v[2], index * 11);
    }
}
const char* rtc_ArrayOut_func_glm = R"###(
FLAMEGPU_AGENT_FUNCTION(ArrayOut, flamegpu::MessageNone, flamegpu::MessageArray) {
    const unsigned int index = FLAMEGPU->getVariable<unsigned int>("index");
    glm::uvec3 t = glm::uvec3(index * 3, index * 7, index * 11);
    FLAMEGPU->message_out.setVariable<glm::uvec3>("v", t);
    FLAMEGPU->message_out.setIndex(index);
    return flamegpu::ALIVE;
}
)###";
const char* rtc_ArrayIn_func_glm = R"###(
FLAMEGPU_AGENT_FUNCTION(ArrayIn, flamegpu::MessageArray, flamegpu::MessageNone) {
    const unsigned int my_index = FLAMEGPU->getVariable<unsigned int>("index");
    const auto& message = FLAMEGPU->message_in.at(my_index);
    FLAMEGPU->setVariable<glm::uvec3>("message_read", message.getVariable<glm::uvec3>("v"));
    return flamegpu::ALIVE;
}
)###";
TEST(TestRTCMessage_Array, ArrayVariable_glm) {
    ModelDescription m(MODEL_NAME);
    MessageArray::Description& message = m.newMessage<MessageArray>(MESSAGE_NAME);
    message.setLength(AGENT_COUNT);
    message.newVariable<unsigned int, 3>("v");
    AgentDescription& a = m.newAgent(AGENT_NAME);
    a.newVariable<unsigned int>("index");
    a.newVariable<unsigned int, 3>("message_read", { UINT_MAX, UINT_MAX, UINT_MAX });
    AgentFunctionDescription& fo = a.newRTCFunction(OUT_FUNCTION_NAME, rtc_ArrayOut_func_glm);
    fo.setMessageOutput(message);
    AgentFunctionDescription& fi = a.newRTCFunction(IN_FUNCTION_NAME, rtc_ArrayIn_func_glm);
    fi.setMessageInput(message);
    LayerDescription& lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription& li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    // Assign the numbers in shuffled order to agents
    AgentVector pop(a, AGENT_COUNT);
    for (unsigned int i = 0; i < AGENT_COUNT; ++i) {
        AgentVector::Agent ai = pop[i];
        ai.setVariable<unsigned int>("index", i);
    }
    // Set pop in model
    CUDASimulation c(m);
    c.setPopulationData(pop);
    c.step();
    c.getPopulationData(pop);
    // Validate each agent has same result
    for (AgentVector::Agent ai : pop) {
        const unsigned int index = ai.getVariable<unsigned int>("index");
        std::array<unsigned int, 3> v = ai.getVariable<unsigned int, 3>("message_read");
        ASSERT_EQ(v[0], index * 3);
        ASSERT_EQ(v[1], index * 7);
        ASSERT_EQ(v[2], index * 11);
    }
}
#else
TEST(TestMessage_Array, DISABLED_ArrayVariable_glm) { }
TEST(TestRTCMessage_Array, DISABLED_ArrayVariable_glm) { }
#endif

}  // namespace test_message_array
}  // namespace flamegpu
