#include "hip/hip_runtime.h"
#include <thread>
#include <chrono>

#include "flamegpu/flamegpu.h"

#include "gtest/gtest.h"

namespace flamegpu {
namespace tests {
namespace test_cuda_ensemble {

TEST(TestCUDAEnsemble, constructor) {
    // Create a model
    flamegpu::ModelDescription model("test");
    // Declare a pointer
    flamegpu::CUDAEnsemble * ensemble = nullptr;
    // Use the ctor
    // explicit CUDAEnsemble(const ModelDescription& model, int argc = 0, const char** argv = nullptr);
    EXPECT_NO_THROW(ensemble = new flamegpu::CUDAEnsemble(model, 0, nullptr));
    EXPECT_NE(ensemble, nullptr);
    // Check a property
    EXPECT_EQ(ensemble->Config().timing, false);
    // Run the destructor ~CUDAEnsemble
    EXPECT_NO_THROW(delete ensemble);
    ensemble = nullptr;
    // Check with simple argparsing.
    const char *argv[2] = { "prog.exe", "--timing" };
    EXPECT_NO_THROW(ensemble = new flamegpu::CUDAEnsemble(model, sizeof(argv) / sizeof(char*), argv));
    EXPECT_EQ(ensemble->Config().timing, true);
    EXPECT_NO_THROW(delete ensemble);
    ensemble = nullptr;
}
TEST(TestCUDAEnsemble, EnsembleConfig) {
    // Create a model
    flamegpu::ModelDescription model("test");
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Get a config object.
    // EnsembleConfig &Config()
    EXPECT_NO_THROW(ensemble.Config());
    auto &mutableConfig = ensemble.Config();
    // Get a const'd version.
    // const EnsembleConfig &getConfig()
    EXPECT_NO_THROW(ensemble.getConfig());
    auto &immutableConfig = ensemble.getConfig();
    // Check the default values are correct.
    EXPECT_EQ(immutableConfig.out_directory, "");
    EXPECT_EQ(immutableConfig.out_format, "json");
    EXPECT_EQ(immutableConfig.concurrent_runs, 4u);
    EXPECT_EQ(immutableConfig.devices, std::set<int>());  // @todo - this will need to change.
    EXPECT_EQ(immutableConfig.quiet, false);
    EXPECT_EQ(immutableConfig.timing, false);
    // Mutate the config. Note we cannot mutate the return from getConfig, and connot test this as it is a compialtion failure (requires ctest / standalone .cpp file)
    mutableConfig.out_directory = std::string("test");
    mutableConfig.out_format = std::string("xml");
    mutableConfig.concurrent_runs = 1;
    mutableConfig.devices = std::set<int>({0});
    mutableConfig.quiet = true;
    mutableConfig.timing = true;
    // Check via the const ref, this should show the same value as config was a reference, not a copy.
    EXPECT_EQ(immutableConfig.out_directory, "test");
    EXPECT_EQ(immutableConfig.out_format, "xml");
    EXPECT_EQ(immutableConfig.concurrent_runs, 1u);
    EXPECT_EQ(immutableConfig.devices, std::set<int>({0}));  // @todo - this will need to change.
    EXPECT_EQ(immutableConfig.quiet, true);
    EXPECT_EQ(immutableConfig.timing, true);
}
// This test causes `exit` so cannot be used.
/* TEST(TestCUDAEnsemble, DISABLED_initialise_help) {
    // Create a model
    flamegpu::ModelDescription model("test");
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Call initialise with differnt cli arguments, which will mutate values. Check they have the new value.
    const char *argv[2] = { "prog.exe", "--help" };
    ensemble.initialise(sizeof(argv) / sizeof(char*), argv);
} */
TEST(TestCUDAEnsemble, initialise_out) {
    // Create a model
    flamegpu::ModelDescription model("test");
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Call initialise with differnt cli arguments, which will mutate values. Check they have the new value.
    EXPECT_EQ(ensemble.getConfig().out_directory, "");
    EXPECT_EQ(ensemble.getConfig().out_format, "json");
    const char *argv[4] = { "prog.exe", "--out", "test", "xml" };
    ensemble.initialise(sizeof(argv) / sizeof(char*), argv);
    EXPECT_EQ(ensemble.getConfig().out_directory, "test");
    EXPECT_EQ(ensemble.getConfig().out_format, "xml");
}
TEST(TestCUDAEnsemble, initialise_concurrent_runs) {
    // Create a model
    flamegpu::ModelDescription model("test");
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Call initialise with differnt cli arguments, which will mutate values. Check they have the new value.
    EXPECT_EQ(ensemble.getConfig().concurrent_runs, 4u);
    const char *argv[3] = { "prog.exe", "--concurrent", "2" };
    ensemble.initialise(sizeof(argv) / sizeof(char*), argv);
    EXPECT_EQ(ensemble.getConfig().concurrent_runs, 2u);
}
TEST(TestCUDAEnsemble, initialise_devices) {
    // Create a model
    flamegpu::ModelDescription model("test");
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Call initialise with differnt cli arguments, which will mutate values. Check they have the new value.
    EXPECT_EQ(ensemble.getConfig().devices, std::set<int>({}));
    const char *argv[3] = { "prog.exe", "--devices", "0" };
    ensemble.initialise(sizeof(argv) / sizeof(char*), argv);
    EXPECT_EQ(ensemble.getConfig().devices, std::set<int>({0}));
}
TEST(TestCUDAEnsemble, initialise_quiet) {
    // Create a model
    flamegpu::ModelDescription model("test");
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Call initialise with differnt cli arguments, which will mutate values. Check they have the new value.
    EXPECT_EQ(ensemble.getConfig().quiet, false);
    const char *argv[2] = { "prog.exe", "--quiet" };
    ensemble.initialise(sizeof(argv) / sizeof(char*), argv);
    EXPECT_EQ(ensemble.getConfig().quiet, true);
}
TEST(TestCUDAEnsemble, initialise_timing) {
    // Create a model
    flamegpu::ModelDescription model("test");
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Call initialise with differnt cli arguments, which will mutate values. Check they have the new value.
    EXPECT_EQ(ensemble.getConfig().timing, false);
    const char *argv[2] = { "prog.exe", "--timing" };
    ensemble.initialise(sizeof(argv) / sizeof(char*), argv);
    EXPECT_EQ(ensemble.getConfig().timing, true);
}
TEST(TestCUDAEnsemble, initialise_error_level) {
    // Create a model
    flamegpu::ModelDescription model("test");
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Call initialise with different cli arguments, which will mutate values. Check they have the new value.
    EXPECT_EQ(ensemble.getConfig().error_level, CUDAEnsemble::EnsembleConfig::Slow);
    {
        const char* argv[3] = { "prog.exe", "-e", "0" };
        ensemble.initialise(sizeof(argv) / sizeof(char*), argv);
        EXPECT_EQ(ensemble.getConfig().error_level, CUDAEnsemble::EnsembleConfig::Off);
    }
    {
        const char* argv[3] = { "prog.exe", "--error", "1" };
        ensemble.initialise(sizeof(argv) / sizeof(char*), argv);
        EXPECT_EQ(ensemble.getConfig().error_level, CUDAEnsemble::EnsembleConfig::Slow);
    }
    {
        const char* argv[3] = { "prog.exe", "-e", "2" };
        ensemble.initialise(sizeof(argv) / sizeof(char*), argv);
        EXPECT_EQ(ensemble.getConfig().error_level, CUDAEnsemble::EnsembleConfig::Fast);
    }
    {
        const char* argv[3] = { "prog.exe", "--error", "Off" };
        ensemble.initialise(sizeof(argv) / sizeof(char*), argv);
        EXPECT_EQ(ensemble.getConfig().error_level, CUDAEnsemble::EnsembleConfig::Off);
    }
    {
        const char* argv[3] = { "prog.exe", "-e", "SLOW" };
        ensemble.initialise(sizeof(argv) / sizeof(char*), argv);
        EXPECT_EQ(ensemble.getConfig().error_level, CUDAEnsemble::EnsembleConfig::Slow);
    }
    {
        const char* argv[3] = { "prog.exe", "--error", "fast" };
        ensemble.initialise(sizeof(argv) / sizeof(char*), argv);
        EXPECT_EQ(ensemble.getConfig().error_level, CUDAEnsemble::EnsembleConfig::Fast);
    }
}
// Agent function used to check the ensemble runs.
FLAMEGPU_AGENT_FUNCTION(simulateAgentFn, flamegpu::MessageNone, flamegpu::MessageNone) {
    // Increment agent's counter by 1.
    FLAMEGPU->setVariable<int>("counter", FLAMEGPU->getVariable<int>("counter") + 1);
    return flamegpu::ALIVE;
}
FLAMEGPU_INIT_FUNCTION(simulateInit) {
    // Generate a basic pop
    const uint32_t POPULATION_TO_GENERATE = FLAMEGPU->environment.getProperty<uint32_t>("POPULATION_TO_GENERATE");
    auto agent = FLAMEGPU->agent("Agent");
    for (uint32_t i = 0; i < POPULATION_TO_GENERATE; ++i) {
        agent.newAgent().setVariable<uint32_t>("counter", 0);
    }
}
// File scoped variables to allow non-loggin based ensemble validation.
std::atomic<uint64_t> testSimulateSumOfSums = {0};
// File scoped atomics
FLAMEGPU_EXIT_FUNCTION(simulateExit) {
    uint64_t totalCounters = FLAMEGPU->agent("Agent").sum<uint32_t>("counter");
    // Add to the  file scoped atomic sum of sums.
    testSimulateSumOfSums += totalCounters;
}
TEST(TestCUDAEnsemble, simulate) {
    // Reset the atomic sum of sums to 0. Just in case.
    testSimulateSumOfSums = 0;
    // Number of simulations to run.
    constexpr uint32_t planCount = 2u;
    constexpr uint32_t populationSize = 32u;
    // Create a model containing atleast one agent type and function.
    flamegpu::ModelDescription model("test");
    // Environmental constant for initial population
    model.Environment().newProperty<uint32_t>("POPULATION_TO_GENERATE", populationSize, true);
    // Agent(s)
    flamegpu::AgentDescription &agent = model.newAgent("Agent");
    agent.newVariable<uint32_t>("counter", 0);
    agent.newFunction("simulateAgentFn", simulateAgentFn);
    // Control flow
    model.newLayer().addAgentFunction(simulateAgentFn);
    model.addInitFunction(simulateInit);
    model.addExitFunction(simulateExit);
    // Crete a small runplan, using a different number of steps per sim.
    uint64_t expectedResult = 0;
    flamegpu::RunPlanVector plans(model, planCount);
    for (uint32_t idx = 0; idx < plans.size(); idx++) {
        auto &plan = plans[idx];
        plan.setSteps(idx + 1);  // Can't have 0 steps without exit condition
        // Increment the expected result based on the number of steps.
        expectedResult += (idx + 1) * populationSize;
    }
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Make it quiet to avoid outputting during the test suite
    ensemble.Config().quiet = true;
    ensemble.Config().out_format = "";  // Suppress warning
    // Simulate the ensemble,
    EXPECT_NO_THROW(ensemble.simulate(plans));
    // Get the sum of sums from the atomic.
    uint64_t atomicResult = testSimulateSumOfSums.load();
    // Compare against the epxected value
    EXPECT_EQ(atomicResult, expectedResult);

    // An exception should be thrown if the Plan and Ensemble are for different models.
    flamegpu::ModelDescription modelTwo("two");
    flamegpu::RunPlanVector modelTwoPlans(modelTwo, 1);
    EXPECT_THROW(ensemble.simulate(modelTwoPlans), flamegpu::exception::InvalidArgument);
    // Exceptions can also be thrown if output_directory cannot be created, but I'm unsure how to reliably test this cross platform.
}
// Logging is more thoroughly tested in Logging. Here just make sure the methods work
TEST(TestCUDAEnsemble, setStepLog) {
    // Create a model containing atleast one agent type and function.
    flamegpu::ModelDescription model("test");
    // Environmental constant for initial population
    model.Environment().newProperty<float>("f", 0.f);
    // Add an agent so that the simulation can be ran, to check for presence of logs
    flamegpu::AgentDescription &agent = model.newAgent("Agent");
    agent.newVariable<uint32_t>("counter", 0);
    // Define the logging configuraiton.
    LoggingConfig lcfg(model);
    lcfg.logEnvironment("f");
    StepLoggingConfig slcfg(lcfg);
    slcfg.setFrequency(1);
    // Create a single run.
    auto plans = flamegpu::RunPlanVector(model, 1);
    plans[0].setSteps(1);
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Make it quiet to avoid outputting during the test suite
    ensemble.Config().quiet = true;
    ensemble.Config().out_format = "";  // Suppress warning
    // Set the StepLog config.
    EXPECT_NO_THROW(ensemble.setStepLog(slcfg));
    // Run the ensemble, generating logs
    ensemble.simulate(plans);
    // Get the logs, checking the correct number are present.
    const auto &runLogs = ensemble.getLogs();
    EXPECT_EQ(runLogs.size(), plans.size());
    for (auto &log : runLogs) {
        auto &stepLogs = log.getStepLog();
        EXPECT_EQ(stepLogs.size(), 1 + 1);  // This is 1 + 1 due to the always present init log.
        uint32_t expectedStepCount = 0;
        for (const auto &stepLog : stepLogs) {
            ASSERT_EQ(stepLog.getStepCount(), expectedStepCount);
            expectedStepCount++;
        }
    }

    // An exception will be thrown if the step log config is for a different model.
    flamegpu::ModelDescription modelTwo("two");
    LoggingConfig lcfgTwo(modelTwo);
    StepLoggingConfig slcfgTwo(lcfgTwo);
    slcfgTwo.setFrequency(1);
    flamegpu::RunPlanVector modelTwoPlans(modelTwo, 1);
    EXPECT_THROW(ensemble.setStepLog(slcfgTwo), flamegpu::exception::InvalidArgument);
}
TEST(TestCUDAEnsemble, setExitLog) {
    // Create a model containing atleast one agent type and function.
    flamegpu::ModelDescription model("test");
    // Environmental constant for initial population
    model.Environment().newProperty<float>("f", 0.f);
    // Add an agent so that the simulation can be ran, to check for presence of logs
    flamegpu::AgentDescription &agent = model.newAgent("Agent");
    agent.newVariable<uint32_t>("counter", 0u);
    // Define the logging configuraiton.
    LoggingConfig lcfg(model);
    lcfg.logEnvironment("f");
    // Create a single run.
    auto plans = flamegpu::RunPlanVector(model, 1u);
    plans[0].setSteps(1);
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Make it quiet to avoid outputting during the test suite
    ensemble.Config().quiet = true;
    ensemble.Config().out_format = "";  // Suppress warning
    // Set the StepLog config.
    EXPECT_NO_THROW(ensemble.setExitLog(lcfg));
    // Run the ensemble, generating logs
    ensemble.simulate(plans);
    // Get the logs, checking the correct number are present.
    const auto &runLogs = ensemble.getLogs();
    EXPECT_EQ(runLogs.size(), plans.size());
    for (auto &log : runLogs) {
        const auto &exitLog = log.getExitLog();
        ASSERT_EQ(exitLog.getStepCount(), 1u);
    }

    // An exception will be thrown if the step log config is for a different model.
    flamegpu::ModelDescription modelTwo("two");
    LoggingConfig lcfgTwo(modelTwo);
    flamegpu::RunPlanVector modelTwoPlans(modelTwo, 1u);
    EXPECT_THROW(ensemble.setExitLog(lcfgTwo), flamegpu::exception::InvalidArgument);
}
TEST(TestCUDAEnsemble, getLogs) {
    // Create an ensemble with no logging enabled, but call getLogs
    // Create a model containing atleast one agent type and function.
    flamegpu::ModelDescription model("test");
    auto plans = flamegpu::RunPlanVector(model, 1);
    plans[0].setSteps(1);
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    EXPECT_NO_THROW(ensemble.getLogs());
    const auto &runLogs = ensemble.getLogs();
    EXPECT_EQ(runLogs.size(), 0u);
}
// Agent function used to check the ensemble runs.
FLAMEGPU_AGENT_FUNCTION(elapsedAgentFn, flamegpu::MessageNone, flamegpu::MessageNone) {
    // Increment agent's counter by 1.
    FLAMEGPU->setVariable<int>("counter", FLAMEGPU->getVariable<int>("counter") + 1);
    return flamegpu::ALIVE;
}
FLAMEGPU_INIT_FUNCTION(elapsedInit) {
    // Generate a basic pop
    const uint32_t POPULATION_TO_GENERATE = FLAMEGPU->environment.getProperty<uint32_t>("POPULATION_TO_GENERATE");
    auto agent = FLAMEGPU->agent("Agent");
    for (uint32_t i = 0; i < POPULATION_TO_GENERATE; ++i) {
        agent.newAgent().setVariable<uint32_t>("counter", 0u);
    }
}
constexpr double sleepDurationSeconds = 0.5;
// File scoped atomics
FLAMEGPU_STEP_FUNCTION(elapsedStep) {
    // Sleep each thread for a duration of time.
    std::this_thread::sleep_for(std::chrono::duration<double>(sleepDurationSeconds));
}
TEST(TestCUDAEnsemble, getEnsembleElapsedTime) {
    // Create a model containing atleast one agent type and function.
    flamegpu::ModelDescription model("test");
    // Environmental constant for initial population
    model.Environment().newProperty<uint32_t>("POPULATION_TO_GENERATE", 1, true);
    // Agent(s)
    flamegpu::AgentDescription &agent = model.newAgent("Agent");
    agent.newVariable<uint32_t>("counter", 0);
    agent.newFunction("elapsedAgentFn", elapsedAgentFn);
    // Control flow
    model.newLayer().addAgentFunction(elapsedAgentFn);
    model.addInitFunction(elapsedInit);
    model.addStepFunction(elapsedStep);
    // Create a single run.
    auto plans = flamegpu::RunPlanVector(model, 1);
    plans[0].setSteps(1);
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Make it quiet to avoid outputting during the test suite
    ensemble.Config().quiet = true;
    ensemble.Config().out_format = "";  // Suppress warning
    // Get the elapsed seconds before the sim has been executed
    EXPECT_NO_THROW(ensemble.getEnsembleElapsedTime());
    // Assert that it is LE zero.
    EXPECT_LE(ensemble.getEnsembleElapsedTime(), 0.);
    // Simulate the ensemble,
    EXPECT_NO_THROW(ensemble.simulate(plans));
    // Get the elapsed seconds before the sim has been executed
    double elapsedSeconds = 0.f;
    EXPECT_NO_THROW(elapsedSeconds = ensemble.getEnsembleElapsedTime());
    // Ensure the elapsed time is larger than a threshold.
    double threshold = sleepDurationSeconds * 0.8;
    EXPECT_GE(elapsedSeconds, threshold);
}
unsigned int tracked_err_ct;
unsigned int tracked_runs_ct;
FLAMEGPU_STEP_FUNCTION(throwException) {
    ++tracked_runs_ct;
    static int i = 0;
    if (++i % 2 == 0) {
        ++tracked_err_ct;
        THROW exception::UnknownInternalError("Dummy Exception");
    }
}
TEST(TestCUDAEnsemble, ErrorOff) {
    tracked_err_ct = 0;
    tracked_runs_ct = 0;
    // Create a model containing atleast one agent type and function.
    flamegpu::ModelDescription model("test");
    // Environmental constant for initial population
    model.Environment().newProperty<uint32_t>("POPULATION_TO_GENERATE", 1, true);
    // Agent(s)
    flamegpu::AgentDescription& agent = model.newAgent("Agent");
    agent.newVariable<uint32_t>("counter", 0);
    // Control flow
    model.addInitFunction(elapsedInit);
    model.addStepFunction(throwException);
    // Create a set of 10 Run plans
    const unsigned int ENSEMBLE_COUNT = 10;
    auto plans = flamegpu::RunPlanVector(model, ENSEMBLE_COUNT);
    plans.setSteps(1);
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Make it quiet to avoid outputting during the test suite
    ensemble.Config().quiet = true;
    ensemble.Config().out_format = "";  // Suppress warning
    ensemble.Config().error_level = CUDAEnsemble::EnsembleConfig::Off;
    ensemble.Config().concurrent_runs = 1;  // Single device/no concurrency to ensure we get consistent data
    ensemble.Config().devices = {0};
    unsigned int reported_err_ct = 0;
    // Simulate the ensemble,
    EXPECT_NO_THROW(reported_err_ct = ensemble.simulate(plans));
    // Check correct number of fails is reported
    EXPECT_EQ(reported_err_ct, ENSEMBLE_COUNT / 2);
    EXPECT_EQ(tracked_err_ct, ENSEMBLE_COUNT / 2);
    EXPECT_EQ(tracked_runs_ct, ENSEMBLE_COUNT);
}
TEST(TestCUDAEnsemble, ErrorSlow) {
    tracked_err_ct = 0;
    tracked_runs_ct = 0;
    // Create a model containing atleast one agent type and function.
    flamegpu::ModelDescription model("test");
    // Environmental constant for initial population
    model.Environment().newProperty<uint32_t>("POPULATION_TO_GENERATE", 1, true);
    // Agent(s)
    flamegpu::AgentDescription& agent = model.newAgent("Agent");
    agent.newVariable<uint32_t>("counter", 0);
    // Control flow
    model.addInitFunction(elapsedInit);
    model.addStepFunction(throwException);
    // Create a set of 10 Run plans
    const unsigned int ENSEMBLE_COUNT = 10;
    auto plans = flamegpu::RunPlanVector(model, ENSEMBLE_COUNT);
    plans.setSteps(1);
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Make it quiet to avoid outputting during the test suite
    ensemble.Config().quiet = true;
    ensemble.Config().out_format = "";  // Suppress warning
    ensemble.Config().error_level = CUDAEnsemble::EnsembleConfig::Slow;
    ensemble.Config().concurrent_runs = 1;  // Single device/no concurrency to ensure we get consistent data
    ensemble.Config().devices = { 0 };
    // Simulate the ensemble,
    EXPECT_THROW(ensemble.simulate(plans), exception::EnsembleError);
    // Check correct number of fails occurred (Unable to retrieve actual error count except from stderr with SLOW)
    EXPECT_EQ(tracked_err_ct, ENSEMBLE_COUNT / 2);
    EXPECT_EQ(tracked_runs_ct, ENSEMBLE_COUNT);
}
TEST(TestCUDAEnsemble, ErrorFast) {
    tracked_err_ct = 0;
    tracked_runs_ct = 0;
    // Create a model containing atleast one agent type and function.
    flamegpu::ModelDescription model("test");
    // Environmental constant for initial population
    model.Environment().newProperty<uint32_t>("POPULATION_TO_GENERATE", 1, true);
    // Agent(s)
    flamegpu::AgentDescription& agent = model.newAgent("Agent");
    agent.newVariable<uint32_t>("counter", 0);
    // Control flow
    model.addInitFunction(elapsedInit);
    model.addStepFunction(throwException);
    // Create a set of 10 Run plans
    const unsigned int ENSEMBLE_COUNT = 10;
    auto plans = flamegpu::RunPlanVector(model, ENSEMBLE_COUNT);
    plans.setSteps(1);
    // Create an ensemble
    flamegpu::CUDAEnsemble ensemble(model);
    // Make it quiet to avoid outputting during the test suite
    ensemble.Config().quiet = true;
    ensemble.Config().out_format = "";  // Suppress warning
    ensemble.Config().error_level = CUDAEnsemble::EnsembleConfig::Fast;
    ensemble.Config().concurrent_runs = 1;  // Single device/no concurrency to ensure we get consistent data
    ensemble.Config().devices = { 0 };
    // Simulate the ensemble,
    EXPECT_THROW(ensemble.simulate(plans), exception::EnsembleError);
    // Check correct number of fails occurred (Fast kills ensemble as soon as first error occurs)
    EXPECT_EQ(tracked_err_ct, 1u);
    // The first run does not throw
    EXPECT_EQ(tracked_runs_ct, 2u);
}

}  // namespace test_cuda_ensemble
}  // namespace tests
}  // namespace flamegpu
