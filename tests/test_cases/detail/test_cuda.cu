#include <hip/hip_runtime.h>

#include <vector>
#include "flamegpu/detail/cuda.cuh"
#include "flamegpu/simulation/detail/CUDAErrorChecking.cuh"

#include "gtest/gtest.h"
namespace flamegpu {


// Test that wrapped hipFree works.
TEST(TestUtilDetailCuda, hipFree) {
    int * d_ptr = nullptr;
    hipError_t status = hipSuccess;
    // manually allocate a device pointer
    gpuErrchk(hipMalloc(&d_ptr, sizeof(int)));
    // Validate that the ptr is a valid device pointer
    hipPointerAttribute_t attributes = {};
    gpuErrchk(hipPointerGetAttributes(&attributes, d_ptr));
    EXPECT_EQ(attributes.type, hipMemoryTypeDevice);
    // call the wrapped cuda free method
    status = detail::cuda::hipFree(d_ptr);
    // It should not have thrown any cuda errors in normal use.
    EXPECT_EQ(status, hipSuccess);
    // The pointer will still have a non nullptr value, but it will no longer be a valid device ptr.
    EXPECT_NE(d_ptr, nullptr);
    gpuErrchk(hipPointerGetAttributes(&attributes, d_ptr));
    EXPECT_EQ(attributes.type, cudaMemoryTypeUnregistered);
    // Try a double free.
    status = detail::cuda::hipFree(d_ptr);
    // This will appear to succeed (a double free is identical to a device reset then free according from hipPointerGetAttributes' perspective), which is a difference from actual hipFree which would return hipErrorInvalidValue.
    EXPECT_EQ(status, hipSuccess);
    // reset the ptr
    d_ptr = nullptr;
    // Allocate the pointer again
    gpuErrchk(hipMalloc(&d_ptr, sizeof(int)));
    // Validate that the ptr is a valid device pointer
    attributes = {};
    gpuErrchk(hipPointerGetAttributes(&attributes, d_ptr));
    EXPECT_EQ(attributes.type, hipMemoryTypeDevice);
    // Trigger a device reset
    hipDeviceReset();
    // Attempt to free the ptr, this method should claim all things are fine (as the dev ptr has implicitly been freed)
    status = detail::cuda::hipFree(d_ptr);
    EXPECT_EQ(status, hipSuccess);
}

// Test that the wrapped hipHostFree works.
TEST(TestUtilDetailCuda, hipHostFree) {
    int * p_ptr = nullptr;
    hipError_t status = hipSuccess;
    // manually allocate a page-locked host pointer
    gpuErrchk(hipHostMalloc(&p_ptr, sizeof(int)));
    // Validate that the ptr is a valid page-locked host pointer
    hipPointerAttribute_t attributes = {};
    gpuErrchk(hipPointerGetAttributes(&attributes, p_ptr));
    // this appears to return hipMemoryTypeHost, even though it should return hipMemoryTypeHost
    EXPECT_EQ(attributes.type, hipMemoryTypeHost);
    // call the wrapped cuda free method
    status = detail::cuda::hipHostFree(p_ptr);
    // It should not have thrown any cuda errors in normal use.
    EXPECT_EQ(status, hipSuccess);
    // The pointer will still have a non nullptr value, but it will no longer be a valid page-locked ptr.
    EXPECT_NE(p_ptr, nullptr);
    gpuErrchk(hipPointerGetAttributes(&attributes, p_ptr));
    EXPECT_EQ(attributes.type, cudaMemoryTypeUnregistered);

    // Try a double free.
    status = detail::cuda::hipHostFree(p_ptr);
    // This will appear to succeed (a double free is identical to a device reset then free according from hipPointerGetAttributes' perspective), which is a difference from actual hipHostFree which would return hipErrorInvalidValue.
    EXPECT_EQ(status, hipSuccess);
    // reset the ptr
    p_ptr = nullptr;
    // Allocate the pointer again
    gpuErrchk(hipHostMalloc(&p_ptr, sizeof(int)));
    // Validate that the ptr is a valid page-locked host pointer
    attributes = {};
    gpuErrchk(hipPointerGetAttributes(&attributes, p_ptr));
    // this appears to return hipMemoryTypeHost, even though it should return hipMemoryTypeHost
    EXPECT_EQ(attributes.type, hipMemoryTypeHost);
    // Trigger a device reset
    hipDeviceReset();
    // Attempt to free the ptr, this method should claim all things are fine (as the dev ptr has implicitly been freed)
    status = detail::cuda::hipHostFree(p_ptr);
    EXPECT_EQ(status, hipSuccess);
}

// Test that getting the primary context works, Difficult to trigger failure cases for this method, so coverage is subpar.
TEST(TestUtilDetailCuda, cuDevicePrimaryContextIsActive) {
    // Make sure device 0 is active for this test.
    gpuErrchk(hipSetDevice(0));
    // Initialise a cudaContext, incase it somehow hasn't been already.
    gpuErrchk(hipFree(0));
    // check if the primary context is active or not for device 0, it shoudl be.
    bool isActive = false;
    isActive = detail::cuda::cuDevicePrimaryContextIsActive(0);
    EXPECT_EQ(isActive, true);
    // Call device reset and check again without establishing a new context, it should not be active.
    gpuErrchk(hipDeviceReset());
    isActive = detail::cuda::cuDevicePrimaryContextIsActive(0);
    EXPECT_EQ(isActive, false);
    // Check that exceptions will be raised correctly when passing bad device ordinals.
    // Expect an exception if the ordinal is negative
    EXPECT_THROW(detail::cuda::cuDevicePrimaryContextIsActive(-1), exception::InvalidCUDAdevice);
    // First grab the device count, to check for exceptions when the device ordinal is too big.
    int deviceCount = 0;
    gpuErrchk(hipGetDeviceCount(&deviceCount));
    if (deviceCount > 0) {
        // Expect an exception if the ordinal is too big.
        EXPECT_THROW(detail::cuda::cuDevicePrimaryContextIsActive(deviceCount), exception::InvalidCUDAdevice);
    }
}

}  // namespace flamegpu
