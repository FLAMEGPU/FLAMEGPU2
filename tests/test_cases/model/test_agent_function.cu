#include "gtest/gtest.h"

#include "flamegpu/flame_api.h"
#include "flamegpu/runtime/flamegpu_api.h"

namespace test_agent_function {
FLAMEGPU_AGENT_FUNCTION(agent_fn1) {
    // do nothing
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(agent_fn2) {
    // do nothing
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(agent_fn3) {
    // do nothing
    return ALIVE;
}

const char *MODEL_NAME = "Model";
const char *WRONG_MODEL_NAME = "Model2";
const char *AGENT_NAME = "Agent1";
const char *AGENT_NAME2 = "Agent2";
const char *AGENT_NAME3 = "Agent3";
const char *MESSAGE_NAME1 = "Message1";
const char *MESSAGE_NAME2 = "Message2";
const char *VARIABLE_NAME1 = "Var1";
const char *VARIABLE_NAME2 = "Var2";
const char *VARIABLE_NAME3 = "Var3";
const char *FUNCTION_NAME1 = "Function1";
const char *FUNCTION_NAME2 = "Function2";
const char *FUNCTION_NAME3 = "Function3";
const char *STATE_NAME = "State1";
const char *NEW_STATE_NAME = "State2";
const char *WRONG_STATE_NAME = "State3";

TEST(AgentFunctionDescriptionTest, InitialState) {
    ModelDescription _m(MODEL_NAME);
    AgentDescription &a = _m.newAgent(AGENT_NAME);
    AgentDescription &a2 = _m.newAgent(AGENT_NAME2);
    AgentDescription &a3 = _m.newAgent(AGENT_NAME3);
    a2.newState(STATE_NAME);
    a3.newState(ModelData::DEFAULT_STATE);
    a2.newState(NEW_STATE_NAME);
    a3.newState(NEW_STATE_NAME);
    AgentFunctionDescription &f = a.newFunction(FUNCTION_NAME1, agent_fn1);
    AgentFunctionDescription &f2 = a2.newFunction(FUNCTION_NAME2, agent_fn2);
    AgentFunctionDescription &f3 = a3.newFunction(FUNCTION_NAME3, agent_fn3);
    // Initial state begins whatever agent's initial state is
    EXPECT_EQ(f.getInitialState(), a.getInitialState());
    EXPECT_EQ(f2.getInitialState(), a2.getInitialState());
    EXPECT_EQ(f3.getInitialState(), a3.getInitialState());
    // Can change the initial state
    // f.setInitialState(NEW_STATE_NAME); // Don't perform this here, would need to change from default state
    f2.setInitialState(NEW_STATE_NAME);
    f3.setInitialState(NEW_STATE_NAME);
    // Returned value is same
    // EXPECT_EQ(f.getInitialState(), NEW_STATE_NAME);
    EXPECT_EQ(f2.getInitialState(), NEW_STATE_NAME);
    EXPECT_EQ(f3.getInitialState(), NEW_STATE_NAME);
    // Replacing agent's default state will replace their initial state
    a.newState(NEW_STATE_NAME);
    EXPECT_EQ(f.getInitialState(), NEW_STATE_NAME);
    // Can't set state to one not held by parent agent
    EXPECT_THROW(f.setInitialState(WRONG_STATE_NAME), InvalidStateName);
    EXPECT_THROW(f2.setInitialState(WRONG_STATE_NAME), InvalidStateName);
    EXPECT_THROW(f3.setInitialState(WRONG_STATE_NAME), InvalidStateName);
}
TEST(AgentFunctionDescriptionTest, EndState) {
    ModelDescription _m(MODEL_NAME);
    AgentDescription &a = _m.newAgent(AGENT_NAME);
    AgentDescription &a2 = _m.newAgent(AGENT_NAME2);
    AgentDescription &a3 = _m.newAgent(AGENT_NAME3);
    a2.newState(STATE_NAME);
    a3.newState(ModelData::DEFAULT_STATE);
    a2.newState(NEW_STATE_NAME);
    a3.newState(NEW_STATE_NAME);
    AgentFunctionDescription &f = a.newFunction(FUNCTION_NAME1, agent_fn1);
    AgentFunctionDescription &f2 = a2.newFunction(FUNCTION_NAME2, agent_fn2);
    AgentFunctionDescription &f3 = a3.newFunction(FUNCTION_NAME3, agent_fn3);
    // End state begins whatever agent's end state is
    EXPECT_EQ(f.getEndState(), a.getInitialState());
    EXPECT_EQ(f2.getEndState(), a2.getInitialState());
    EXPECT_EQ(f3.getEndState(), a3.getInitialState());
    // Can change the end state
    // f.setEndState(NEW_STATE_NAME); // Don't perform this here, would need to change from default state
    f2.setEndState(NEW_STATE_NAME);
    f3.setEndState(NEW_STATE_NAME);
    // Returned value is same
    // EXPECT_EQ(f.getEndState(), NEW_STATE_NAME);
    EXPECT_EQ(f2.getEndState(), NEW_STATE_NAME);
    EXPECT_EQ(f3.getEndState(), NEW_STATE_NAME);
    // Replacing agent's default state will replace their end state
    a.newState(NEW_STATE_NAME);
    EXPECT_EQ(f.getEndState(), NEW_STATE_NAME);
    // Can't set state to one not held by parent agent
    EXPECT_THROW(f.setEndState(WRONG_STATE_NAME), InvalidStateName);
    EXPECT_THROW(f2.setEndState(WRONG_STATE_NAME), InvalidStateName);
    EXPECT_THROW(f3.setEndState(WRONG_STATE_NAME), InvalidStateName);
}
TEST(AgentFunctionDescriptionTest, MessageInput) {
    ModelDescription _m(MODEL_NAME);
    AgentDescription &a = _m.newAgent(AGENT_NAME);
    MessageDescription &m = _m.newMessage(MESSAGE_NAME1);
    MessageDescription &m2 = _m.newMessage(MESSAGE_NAME2);
    AgentFunctionDescription &f = a.newFunction(FUNCTION_NAME1, agent_fn1);
    // Begins empty
    EXPECT_FALSE(f.hasMessageInput());
    EXPECT_THROW(f.getMessageInput(), OutOfBoundsException);
    // Can be set
    f.setMessageInput(m);
    EXPECT_TRUE(f.hasMessageInput());
    // Returns the expected value
    EXPECT_EQ(f.getMessageInput(), m);
    // Can be updated
    f.setMessageInput(m2);
    EXPECT_TRUE(f.hasMessageInput());
    // Returns the expected value
    EXPECT_EQ(f.getMessageInput(), m2);
}
TEST(AgentFunctionDescriptionTest, MessageOutput) {
    ModelDescription _m(MODEL_NAME);
    AgentDescription &a = _m.newAgent(AGENT_NAME);
    MessageDescription &m = _m.newMessage(MESSAGE_NAME1);
    MessageDescription &m2 = _m.newMessage(MESSAGE_NAME2);
    AgentFunctionDescription &f = a.newFunction(FUNCTION_NAME1, agent_fn1);
    // Begins empty
    EXPECT_FALSE(f.hasMessageOutput());
    EXPECT_THROW(f.getMessageOutput(), OutOfBoundsException);
    // Can be set
    f.setMessageOutput(m);
    EXPECT_TRUE(f.hasMessageOutput());
    // Returns the expected value
    EXPECT_EQ(f.getMessageOutput(), m);
    // Can be updated
    f.setMessageOutput(m2);
    EXPECT_TRUE(f.hasMessageOutput());
    // Returns the expected value
    EXPECT_EQ(f.getMessageOutput(), m2);
}
TEST(AgentFunctionDescriptionTest, MessageOutputOptional) {
    ModelDescription _m(MODEL_NAME);
    AgentDescription &a = _m.newAgent(AGENT_NAME);
    AgentFunctionDescription &f = a.newFunction(FUNCTION_NAME1, agent_fn1);
    // Begins disabled
    EXPECT_FALSE(f.getMessageOutputOptional());
    EXPECT_FALSE(f.MessageOutputOptional());
    // Can be updated
    f.MessageOutputOptional() = true;
    EXPECT_TRUE(f.getMessageOutputOptional());
    EXPECT_TRUE(f.MessageOutputOptional());
    f.setMessageOutputOptional(false);
    EXPECT_FALSE(f.getMessageOutputOptional());
    EXPECT_FALSE(f.MessageOutputOptional());
}
TEST(AgentFunctionDescriptionTest, AgentOutput) {
    ModelDescription _m(MODEL_NAME);
    AgentDescription &a = _m.newAgent(AGENT_NAME);
    AgentDescription &a2 = _m.newAgent(AGENT_NAME2);
    AgentFunctionDescription &f = a.newFunction(FUNCTION_NAME1, agent_fn1);
    // Begins empty
    EXPECT_FALSE(f.hasAgentOutput());
    EXPECT_THROW(f.getAgentOutput(), OutOfBoundsException);
    // Can be set
    f.setAgentOutput(a);
    EXPECT_TRUE(f.hasAgentOutput());
    // Returns the expected value
    EXPECT_EQ(f.getAgentOutput(), a);
    // Can be updated
    f.setAgentOutput(a2);
    EXPECT_TRUE(f.hasAgentOutput());
    // Returns the expected value
    EXPECT_EQ(f.getAgentOutput(), a2);
}
TEST(AgentFunctionDescriptionTest, AllowAgentDeath) {
    ModelDescription _m(MODEL_NAME);
    AgentDescription &a = _m.newAgent(AGENT_NAME);
    AgentFunctionDescription &f = a.newFunction(FUNCTION_NAME1, agent_fn1);
    // Begins disabled
    EXPECT_FALSE(f.getAllowAgentDeath());
    EXPECT_FALSE(f.AllowAgentDeath());
    // Can be updated
    f.AllowAgentDeath() = true;
    EXPECT_TRUE(f.getAllowAgentDeath());
    EXPECT_TRUE(f.AllowAgentDeath());
    f.setAllowAgentDeath(false);
    EXPECT_FALSE(f.getAllowAgentDeath());
    EXPECT_FALSE(f.AllowAgentDeath());
}

TEST(AgentFunctionDescriptionTest, MessageInput_WrongModel) {
    ModelDescription _m(MODEL_NAME);
    ModelDescription _m2(WRONG_MODEL_NAME);
    AgentDescription &a = _m.newAgent(AGENT_NAME);
    MessageDescription &m1 = _m.newMessage(MESSAGE_NAME1);
    MessageDescription &m2 = _m2.newMessage(MESSAGE_NAME2);
    AgentFunctionDescription &f = a.newFunction(FUNCTION_NAME1, agent_fn1);

    EXPECT_THROW(f.setMessageInput(m2), DifferentModel);
    EXPECT_NO_THROW(f.setMessageInput(m1));
}
TEST(AgentFunctionDescriptionTest, MessageOutput_WrongModel) {
    ModelDescription _m(MODEL_NAME);
    ModelDescription _m2(WRONG_MODEL_NAME);
    AgentDescription &a = _m.newAgent(AGENT_NAME);
    MessageDescription &m1 = _m.newMessage(MESSAGE_NAME1);
    MessageDescription &m2 = _m2.newMessage(MESSAGE_NAME2);
    AgentFunctionDescription &f = a.newFunction(FUNCTION_NAME1, agent_fn1);

    EXPECT_THROW(f.setMessageOutput(m2), DifferentModel);
    EXPECT_NO_THROW(f.setMessageOutput(m1));
}
TEST(AgentFunctionDescriptionTest, AgentOutput_WrongModel) {
    ModelDescription _m(MODEL_NAME);
    ModelDescription _m2(WRONG_MODEL_NAME);
    AgentDescription &a = _m.newAgent(AGENT_NAME);
    AgentDescription &a2 = _m2.newAgent(AGENT_NAME2);
    AgentFunctionDescription &f = a.newFunction(FUNCTION_NAME1, agent_fn1);

    EXPECT_THROW(f.setAgentOutput(a2), DifferentModel);
    EXPECT_NO_THROW(f.setAgentOutput(a));
}
TEST(AgentFunctionDescriptionTest, MessageInputOutput) {
    ModelDescription _m(MODEL_NAME);
    AgentDescription &a = _m.newAgent(AGENT_NAME);
    MessageDescription &m = _m.newMessage(MESSAGE_NAME1);
    MessageDescription &m2 = _m.newMessage(MESSAGE_NAME2);
    AgentFunctionDescription &f = a.newFunction(FUNCTION_NAME1, agent_fn1);
    // Cannot bind same message to input and output
    EXPECT_NO_THROW(f.setMessageInput(m));
    EXPECT_THROW(f.setMessageOutput(m), InvalidMessageName);
    EXPECT_NO_THROW(f.setMessageOutput(m2));
}
TEST(AgentFunctionDescriptionTest, MessageOutputInput) {
    ModelDescription _m(MODEL_NAME);
    AgentDescription &a = _m.newAgent(AGENT_NAME);
    MessageDescription &m = _m.newMessage(MESSAGE_NAME1);
    MessageDescription &m2 = _m.newMessage(MESSAGE_NAME2);
    AgentFunctionDescription &f = a.newFunction(FUNCTION_NAME1, agent_fn1);
    // Cannot bind same message to output and input
    EXPECT_NO_THROW(f.setMessageOutput(m));
    EXPECT_THROW(f.setMessageInput(m), InvalidMessageName);
    EXPECT_NO_THROW(f.setMessageInput(m2));
}
}  // namespace test_agent_function
