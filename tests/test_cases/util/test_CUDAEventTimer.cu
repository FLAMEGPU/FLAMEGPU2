#include <thread>
#include <chrono>
#include "flamegpu/util/detail/CUDAEventTimer.cuh"
#include "flamegpu/util/detail/wddm.cuh"
#include "flamegpu/gpu/detail/CUDAErrorChecking.cuh"

#include "gtest/gtest.h"
namespace flamegpu {


namespace test_CUDAEventTimer {

/**
 * This tests if the cudaEventTimer correctly times an event. 
 * GPUs using WDDM driver can be inaccurate.
 *   - eventRecord appear to be buffered to the device. 
 *   - So need to device sync before the threadSleep.
 *   - they appear to be accurate for timing the actual device work, just not wrt. the host (even in the default stream)
 *   - Needs further investigation at some point (@todo). May be worth falling back to chrono::steady_clock and notify the reduced precision if wddm? 
 */
TEST(TestUtilCUDAEventTimer, CUDAEventTimer) {
    // Create an event timer, time should be 0 initially.
    util::detail::Timer * timer = nullptr;
    EXPECT_NO_THROW(timer = new util::detail::CUDAEventTimer());
    // Expect an exception if sync is called via getElapsed* if start() has not yet been called.
    EXPECT_THROW(timer->getElapsedMilliseconds(), exception::TimerException);
    // Time an arbitrary event, and check the value is approximately correct.
    timer->start();
    // Expect an exception if sync is called via getElapsed* if stop() has not yet been called.
    EXPECT_THROW(timer->getElapsedMilliseconds(), exception::TimerException);
    const int sleep_duration_seconds = 1;
    const double min_expected_seconds = sleep_duration_seconds * 0.9;
    const double min_expected_millis = min_expected_seconds * 1000.0;
    // If the WDDM driver is being used, this test is only accurate if the  start event is synchronised (pushed to the device) prior to the sleep.
    // Essentially, CUDAEventTimers should not be used to time host code, they are only accurate for  the device code which they wrap.
    if (util::detail::wddm::deviceIsWDDM()) {
        gpuErrchk(hipDeviceSynchronize());
    }
    // Sleep for some amount of time.
    std::this_thread::sleep_for(std::chrono::seconds(sleep_duration_seconds));
    // Stop the timer.
    timer->stop();
    // Get the elapsed time. This implicitly synchronises the timer.
    EXPECT_GE(timer->getElapsedMilliseconds(), min_expected_millis);
    // Also check the seconds method.
    EXPECT_GE(timer->getElapsedSeconds(), min_expected_seconds);
    // Trigger the destructor.
    EXPECT_NO_THROW(delete timer);
    // Reset the device for profiling?
    gpuErrchk(hipDeviceReset());
}

}  // namespace test_CUDAEventTimer
}  // namespace flamegpu
