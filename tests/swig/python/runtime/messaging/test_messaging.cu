#include "flamegpu/flame_api.h"

#include "gtest/gtest.h"

namespace test_messaging {
    const char *MODEL_NAME = "Model";
    const char *AGENT_NAME = "Agent";
    const char *MESSAGE_NAME = "Message";
    const char *IN_FUNCTION_NAME = "InFunction";
    const char *OUT_FUNCTION_NAME = "OutFunction";
    const char *IN_LAYER_NAME = "InLayer";
    const char *OUT_LAYER_NAME = "OutLayer";

FLAMEGPU_AGENT_FUNCTION(OutFunction, MsgNone, MsgBruteForce) {
    FLAMEGPU->message_out.setVariable("x", FLAMEGPU->getVariable<int>("x"));
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(InFunction, MsgBruteForce, MsgNone) {
    int sum = FLAMEGPU->getVariable<int>("sum");
    int product = FLAMEGPU->getVariable<int>("product");
    for (auto &message : FLAMEGPU->message_in) {
        const int x = message.getVariable<int>("x");
        sum += x;
        product *= x;
        product = product > 1000000 ? 1 : product;
    }
    FLAMEGPU->setVariable<int>("sum", sum);
    FLAMEGPU->setVariable<int>("product", product);
    return ALIVE;
}

TEST(TestMessage, NoAgents) {
    // There was a bug whereby having message output with 0 agents lead to a segfault
    // This test confirms that it nolonger exists

    ModelDescription m(MODEL_NAME);
    MsgBruteForce::Description &msg = m.newMessage(MESSAGE_NAME);
    msg.newVariable<int>("x");
    AgentDescription &a = m.newAgent(AGENT_NAME);
    a.newVariable<int>("x");
    a.newVariable<int>("sum");
    a.newVariable<int>("product");
    AgentFunctionDescription &fo = a.newFunction(OUT_FUNCTION_NAME, OutFunction);
    fo.setMessageOutput(msg);
    AgentFunctionDescription &fi = a.newFunction(IN_FUNCTION_NAME, InFunction);
    fi.setMessageInput(msg);
    LayerDescription &lo = m.newLayer(OUT_LAYER_NAME);
    lo.addAgentFunction(fo);
    LayerDescription &li = m.newLayer(IN_LAYER_NAME);
    li.addAgentFunction(fi);
    CUDASimulation c(m);
    c.SimulationConfig().steps = 10;
    EXPECT_NO_THROW(c.simulate());
}
}  // namespace test_messaging
