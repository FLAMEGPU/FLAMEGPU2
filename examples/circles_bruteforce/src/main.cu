#include "hip/hip_runtime.h"
#include "flamegpu/flame_api.h"
#include "flamegpu/util/nvtx.h"



FLAMEGPU_AGENT_FUNCTION(output_message, MsgNone, MsgBruteForce) {
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
    FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
    FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
    FLAMEGPU->message_out.setVariable<float>("z", FLAMEGPU->getVariable<float>("z"));
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(move, MsgBruteForce, MsgNone) {
    const int ID = FLAMEGPU->getVariable<int>("id");
    const float REPULSE_FACTOR = FLAMEGPU->environment.get<float>("repulse");
    const float RADIUS = FLAMEGPU->environment.get<float>("radius");
    float fx = 0.0;
    float fy = 0.0;
    float fz = 0.0;
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    const float z1 = FLAMEGPU->getVariable<float>("z");
    int count = 0;
    for (const auto &message : FLAMEGPU->message_in) {
        if (message.getVariable<int>("id") != ID) {
            const float x2 = message.getVariable<float>("x");
            const float y2 = message.getVariable<float>("y");
            const float z2 = message.getVariable<float>("z");
            float x21 = x2 - x1;
            float y21 = y2 - y1;
            float z21 = z2 - z1;
            const float separation = cbrt(x21*x21 + y21*y21 + z21*z21);
            if (separation < RADIUS && separation > 0.0f) {
                float k = sinf((separation / RADIUS)*3.141*-2)*REPULSE_FACTOR;
                // Normalise without recalculating separation
                x21 /= separation;
                y21 /= separation;
                z21 /= separation;
                fx += k * x21;
                fy += k * y21;
                fz += k * z21;
                count++;
            }
        }
    }
    fx /= count > 0 ? count : 1;
    fy /= count > 0 ? count : 1;
    fz /= count > 0 ? count : 1;
    FLAMEGPU->setVariable<float>("x", x1 + fx);
    FLAMEGPU->setVariable<float>("y", y1 + fy);
    FLAMEGPU->setVariable<float>("z", z1 + fz);
    FLAMEGPU->setVariable<float>("drift", cbrt(fx*fx + fy*fy + fz*fz));
    return ALIVE;
}
FLAMEGPU_STEP_FUNCTION(Validation) {
    static float prevTotalDrift = FLT_MAX;
    static unsigned int driftDropped = 0;
    static unsigned int driftIncreased = 0;
    // This value should decline? as the model moves towards a steady equlibrium state
    // Once an equilibrium state is reached, it is likely to oscillate between 2-4? values
    float totalDrift = FLAMEGPU->agent("Circle").sum<float>("drift");
    if (totalDrift <= prevTotalDrift)
        driftDropped++;
    else
        driftIncreased++;
    prevTotalDrift = totalDrift;
    // printf("Avg Drift: %g\n", totalDrift / FLAMEGPU->agent("Circle").count());
    printf("%.2f%% Drift correct\n", 100 * driftDropped / static_cast<float>(driftDropped + driftIncreased));
}
int main(int argc, const char ** argv) {
    NVTX_RANGE("main");
    NVTX_PUSH("ModelDescription");
    ModelDescription model("Circles_BruteForce_example");

    const unsigned int AGENT_COUNT = 16384;
    const float ENV_MAX = static_cast<float>(floor(cbrt(AGENT_COUNT)));
    {   // Location message
        MsgBruteForce::Description &message = model.newMessage("location");
        message.newVariable<int>("id");
        message.newVariable<float>("x");
        message.newVariable<float>("y");
        message.newVariable<float>("z");
    }
    {   // Circle agent
        AgentDescription &agent = model.newAgent("Circle");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("z");
        agent.newVariable<float>("drift");  // Store the distance moved here, for validation
        agent.newFunction("output_message", output_message).setMessageOutput("location");
        agent.newFunction("move", move).setMessageInput("location");
    }


    /**
     * GLOBALS
     */
    {
        EnvironmentDescription &env = model.Environment();
        env.add("repulse", 0.05f);
        env.add("radius", 2.0f);
    }

    /**
     * Control flow
     */     
    {   // Attach init/step/exit functions and exit condition
        model.addStepFunction(Validation);
    }

    {   // Layer #1
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(output_message);
    }
    {   // Layer #2
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(move);
    }

    NVTX_POP();

    /**
     * Create Model Runner
     */
    NVTX_PUSH("CUDASimulation creation");
    CUDASimulation cuda_model(model, argc, argv);
    NVTX_POP();

    /**
     * Create visualisation
     */
#ifdef VISUALISATION
    ModelVis &m_vis = cuda_model.getVisualisation();
    {
        const float INIT_CAM = ENV_MAX * 1.25F;
        m_vis.setInitialCameraLocation(INIT_CAM, INIT_CAM, INIT_CAM);
        m_vis.setCameraSpeed(0.02f);
        auto &circ_agt = m_vis.addAgent("Circle");
        // Position vars are named x, y, z; so they are used by default
        circ_agt.setModel(Stock::Models::ICOSPHERE);
        circ_agt.setModelScale(1/10.0f);
    }
    m_vis.activate();
#endif

    /**
     * Initialisation
     */
    if (cuda_model.getSimulationConfig().input_file.empty()) {
        // Currently population has not been init, so generate an agent population on the fly
        std::default_random_engine rng;
        std::uniform_real_distribution<float> dist(0.0f, ENV_MAX);
        AgentPopulation population(model.Agent("Circle"), AGENT_COUNT);
        for (unsigned int i = 0; i < AGENT_COUNT; i++) {
            AgentInstance instance = population.getNextInstance();
            instance.setVariable<int>("id", i);
            instance.setVariable<float>("x", dist(rng));
            instance.setVariable<float>("y", dist(rng));
            instance.setVariable<float>("z", dist(rng));
        }
        cuda_model.setPopulationData(population);
    }

    /**
     * Execution
     */
    cuda_model.simulate();

    /**
     * Export Pop
     */
    cuda_model.exportData("end.xml");

#ifdef VISUALISATION
    m_vis.join();
#endif
    return 0;
}
