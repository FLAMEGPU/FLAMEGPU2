#include "hip/hip_runtime.h"
#include "flamegpu/flamegpu.h"

FLAMEGPU_AGENT_FUNCTION(output_message, flamegpu::MessageNone, flamegpu::MessageBruteForce) {
    FLAMEGPU->message_out.setVariable<flamegpu::id_t>("id", FLAMEGPU->getID());
    FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
    FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
    FLAMEGPU->message_out.setVariable<float>("z", FLAMEGPU->getVariable<float>("z"));
    return flamegpu::ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(move, flamegpu::MessageBruteForce, flamegpu::MessageNone) {
    const flamegpu::id_t ID = FLAMEGPU->getID();
    const float REPULSE_FACTOR = FLAMEGPU->environment.getProperty<float>("repulse");
    const float RADIUS = FLAMEGPU->environment.getProperty<float>("radius");
    float fx = 0.0;
    float fy = 0.0;
    float fz = 0.0;
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    const float z1 = FLAMEGPU->getVariable<float>("z");
    int count = 0;
    for (const auto &message : FLAMEGPU->message_in) {
        if (message.getVariable<flamegpu::id_t>("id") != ID) {
            const float x2 = message.getVariable<float>("x");
            const float y2 = message.getVariable<float>("y");
            const float z2 = message.getVariable<float>("z");
            float x21 = x2 - x1;
            float y21 = y2 - y1;
            float z21 = z2 - z1;
            const float separation = sqrtf(x21*x21 + y21*y21 + z21*z21);
            if (separation < RADIUS && separation > 0.0f) {
                float k = sinf((separation / RADIUS)*3.141f*-2)*REPULSE_FACTOR;
                // Normalise without recalculating separation
                x21 /= separation;
                y21 /= separation;
                z21 /= separation;
                fx += k * x21;
                fy += k * y21;
                fz += k * z21;
                count++;
            }
        }
    }
    fx /= count > 0 ? count : 1;
    fy /= count > 0 ? count : 1;
    fz /= count > 0 ? count : 1;
    FLAMEGPU->setVariable<float>("x", x1 + fx);
    FLAMEGPU->setVariable<float>("y", y1 + fy);
    FLAMEGPU->setVariable<float>("z", z1 + fz);
    FLAMEGPU->setVariable<float>("drift", sqrtf(fx*fx + fy*fy + fz*fz));
    return flamegpu::ALIVE;
}
FLAMEGPU_STEP_FUNCTION(Validation) {
    static float prevTotalDrift = FLT_MAX;
    static unsigned int driftDropped = 0;
    static unsigned int driftIncreased = 0;
    // This value should decline? as the model moves towards a steady equlibrium state
    // Once an equilibrium state is reached, it is likely to oscillate between 2-4? values
    float totalDrift = FLAMEGPU->agent("Circle").sum<float>("drift");
    if (totalDrift <= prevTotalDrift)
        driftDropped++;
    else
        driftIncreased++;
    prevTotalDrift = totalDrift;
    // printf("Avg Drift: %g\n", totalDrift / FLAMEGPU->agent("Circle").count());
    printf("%.2f%% Drift correct\n", 100 * driftDropped / static_cast<float>(driftDropped + driftIncreased));
}
int main(int argc, const char ** argv) {
    NVTX_RANGE("main");
    NVTX_PUSH("ModelDescription");
    flamegpu::ModelDescription model("Circles_BruteForce_example");

    const unsigned int AGENT_COUNT = 16384;
    const float ENV_MAX = static_cast<float>(floor(cbrt(AGENT_COUNT)));
    {   // Location message
        flamegpu::MessageBruteForce::Description &message = model.newMessage("location");
        message.newVariable<flamegpu::id_t>("id");
        message.newVariable<float>("x");
        message.newVariable<float>("y");
        message.newVariable<float>("z");
    }
    {   // Circle agent
        flamegpu::AgentDescription  &agent = model.newAgent("Circle");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("z");
        agent.newVariable<float>("drift");  // Store the distance moved here, for validation
        agent.newFunction("output_message", output_message).setMessageOutput("location");
        agent.newFunction("move", move).setMessageInput("location");
    }


    /**
     * GLOBALS
     */
    {
        flamegpu::EnvironmentDescription  &env = model.Environment();
        env.newProperty("repulse", 0.05f);
        env.newProperty("radius", 2.0f);
    }

    /**
     * Control flow
     */     
    {   // Attach init/step/exit functions and exit condition
        model.addStepFunction(Validation);
    }

    {   // Layer #1
        flamegpu::LayerDescription  &layer = model.newLayer();
        layer.addAgentFunction(output_message);
    }
    {   // Layer #2
        flamegpu::LayerDescription  &layer = model.newLayer();
        layer.addAgentFunction(move);
    }

    NVTX_POP();

    /**
     * Create Model Runner
     */
    NVTX_PUSH("CUDASimulation creation");
    flamegpu::CUDASimulation  cudaSimulation(model, argc, argv);
    NVTX_POP();

    /**
     * Create visualisation
     */
#ifdef VISUALISATION
    flamegpu::visualiser::ModelVis  &m_vis = cudaSimulation.getVisualisation();
    {
        const float INIT_CAM = ENV_MAX * 1.25F;
        m_vis.setInitialCameraLocation(INIT_CAM, INIT_CAM, INIT_CAM);
        m_vis.setCameraSpeed(0.02f);
        auto &circ_agt = m_vis.addAgent("Circle");
        // Position vars are named x, y, z; so they are used by default
        circ_agt.setModel(flamegpu::visualiser::Stock::Models::ICOSPHERE);
        circ_agt.setModelScale(1/10.0f);
    }
    m_vis.activate();
#endif

    /**
     * Initialisation
     */
    if (cudaSimulation.getSimulationConfig().input_file.empty()) {
        // Currently population has not been init, so generate an agent population on the fly
        std::mt19937_64 rng;
        std::uniform_real_distribution<float> dist(0.0f, ENV_MAX);
        flamegpu::AgentVector population(model.Agent("Circle"), AGENT_COUNT);
        for (unsigned int i = 0; i < AGENT_COUNT; i++) {
            flamegpu::AgentVector::Agent instance = population[i];
            instance.setVariable<float>("x", dist(rng));
            instance.setVariable<float>("y", dist(rng));
            instance.setVariable<float>("z", dist(rng));
        }
        cudaSimulation.setPopulationData(population);
    }

    /**
     * Execution
     */
    cudaSimulation.simulate();

    /**
     * Export Pop
     */
    cudaSimulation.exportData("end.xml");

#ifdef VISUALISATION
    m_vis.join();
#endif
    return 0;
}
