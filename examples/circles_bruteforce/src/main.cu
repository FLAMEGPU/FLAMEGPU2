#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>


#include "flamegpu/flame_api.h"
#include "flamegpu/runtime/flamegpu_api.h"
#include "flamegpu/io/factory.h"
#include "flamegpu/util/nvtx.h"



FLAMEGPU_AGENT_FUNCTION(output_message, MsgNone, MsgBruteForce) {
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
    FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
    FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
    FLAMEGPU->message_out.setVariable<float>("z", FLAMEGPU->getVariable<float>("z"));
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(move, MsgBruteForce, MsgNone) {
    const int ID = FLAMEGPU->getVariable<int>("id");
    const float REPULSE_FACTOR = FLAMEGPU->environment.get<float>("repulse");
    const float RADIUS = FLAMEGPU->environment.get<float>("radius");
    float fx = 0.0;
    float fy = 0.0;
    float fz = 0.0;
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    const float z1 = FLAMEGPU->getVariable<float>("z");
    int count = 0;
    for (const auto &message : FLAMEGPU->message_in) {
        if (message.getVariable<int>("id") != ID) {
            const float x2 = message.getVariable<float>("x");
            const float y2 = message.getVariable<float>("y");
            const float z2 = message.getVariable<float>("z");
            float x21 = x2 - x1;
            float y21 = y2 - y1;
            float z21 = z2 - z1;
            const float separation = cbrt(x21*x21 + y21*y21 + z21*z21);
            if (separation < RADIUS && separation > 0.0f) {
                float k = sinf((separation / RADIUS)*3.141*-2)*REPULSE_FACTOR;
                // Normalise without recalculating separation
                x21 /= separation;
                y21 /= separation;
                z21 /= separation;
                fx += k * x21;
                fy += k * y21;
                fz += k * z21;
                count++;
            }
        }
    }
    fx /= count > 0 ? count : 1;
    fy /= count > 0 ? count : 1;
    fz /= count > 0 ? count : 1;
    FLAMEGPU->setVariable<float>("x", x1 + fx);
    FLAMEGPU->setVariable<float>("y", y1 + fy);
    FLAMEGPU->setVariable<float>("z", z1 + fz);
    FLAMEGPU->setVariable<float>("drift", cbrt(fx*fx + fy*fy + fz*fz));
    return ALIVE;
}
FLAMEGPU_STEP_FUNCTION(Validation) {
    static float prevTotalDrift = FLT_MAX;
    static unsigned int driftDropped = 0;
    static unsigned int driftIncreased = 0;
    // This value should decline? as the model moves towards a steady equlibrium state
    // Once an equilibrium state is reached, it is likely to oscillate between 2-4? values
    float totalDrift = FLAMEGPU->agent("Circle").sum<float>("drift");
    if (totalDrift <= prevTotalDrift)
        driftDropped++;
    else
        driftIncreased++;
    prevTotalDrift = totalDrift;
    // printf("Avg Drift: %g\n", totalDrift / FLAMEGPU->agent("Circle").count());
    printf("%.2f%% Drift correct\n", 100 * driftDropped / static_cast<float>(driftDropped + driftIncreased));
}
void export_data(std::shared_ptr<AgentPopulation> pop, const char *filename);
int main(int argc, const char ** argv) {
    NVTX_RANGE("main");
    NVTX_PUSH("ModelDescription");
    ModelDescription model("Circles_BruteForce_example");

    {   // Location message
        MsgBruteForce::Description &message = model.newMessage("location");
        message.newVariable<int>("id");
        message.newVariable<float>("x");
        message.newVariable<float>("y");
        message.newVariable<float>("z");
    }
    {   // Circle agent
        AgentDescription &agent = model.newAgent("Circle");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("z");
        agent.newVariable<float>("drift");  // Store the distance moved here, for validation
        agent.newFunction("output_message", output_message).setMessageOutput("location");
        agent.newFunction("move", move).setMessageInput("location");
    }


    /**
     * GLOBALS
     */
    {
        EnvironmentDescription &env = model.Environment();
        env.add("repulse", 0.05f);
        env.add("radius", 1.0f);
    }

    /**
     * Control flow
     */     
    {   // Attach init/step/exit functions and exit condition
        model.addStepFunction(Validation);
    }

    {   // Layer #1
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(output_message);
    }
    {   // Layer #2
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(move);
    }

    NVTX_POP();

    /**
     * Create Model Runner
     */
    NVTX_PUSH("CUDAAgentModel creation");
    CUDAAgentModel cuda_model(model);
    NVTX_POP();

    /**
     * Initialisation
     */
    cuda_model.initialise(argc, argv);
    if (cuda_model.getSimulationConfig().xml_input_file.empty()) {
        // Currently population has not been init, so generate an agent population on the fly
        const unsigned int AGENT_COUNT = 16384;
        std::default_random_engine rng;
        std::uniform_real_distribution<float> dist(0.0f, static_cast<float>(floor(cbrt(AGENT_COUNT))));
        AgentPopulation population(model.Agent("Circle"), AGENT_COUNT);
        for (unsigned int i = 0; i < AGENT_COUNT; i++) {
            AgentInstance instance = population.getNextInstance();
            instance.setVariable<int>("id", i);
            instance.setVariable<float>("x", dist(rng));
            instance.setVariable<float>("y", dist(rng));
            instance.setVariable<float>("z", dist(rng));
        }
        cuda_model.setPopulationData(population);
    }

    /**
     * Execution
     */
     // This mode of execution allows the PRIMAGE visualiser to be used (2020-01-07)
     while (cuda_model.getStepCounter() < cuda_model.getSimulationConfig().steps && cuda_model.step()) {
        std::unordered_map<std::string, std::shared_ptr<AgentPopulation>> pops;
        auto a = std::make_shared<AgentPopulation>(model.getAgent("Circle"));
        cuda_model.getPopulationData(*a);
        export_data(a, (std::to_string(cuda_model.getStepCounter()-1)+".bin").c_str());
     }

    cuda_model.simulate();


    /**
     * Export Pop
     */
    // cuda_model.output();
    // Based on Simulation::output() // That can't currently be called
    std::unordered_map<std::string, std::shared_ptr<AgentPopulation>> pops;
    auto a = std::make_shared<AgentPopulation>(model.getAgent("Circle"));
    cuda_model.getPopulationData(*a);
    pops.emplace("Circle", a);
    StateWriter *write__ = WriterFactory::createWriter(pops, cuda_model.getStepCounter(), "end.xml");
    write__->writeStates();
    // export_data(a, "test.bin");
    // getchar();
    return 0;
}

void export_data(std::shared_ptr<AgentPopulation> pop, const char *filename) {
    // Basic binary export function, so that I can use the visualiser i made for kenneths model
    std::ofstream ofs;
    ofs.open(filename, std::ofstream::out | std::ofstream::trunc | std::ofstream::binary);
    if (ofs.is_open()) {
        float garbage[6];  // Need to begin with 6 floats, doesn't matter what they are
        // Write data
        ofs.write(reinterpret_cast<const char*>(garbage), sizeof(float)*6);
        ofs.write(reinterpret_cast<const char*>(pop->getReadOnlyStateMemory().getReadOnlyMemoryVector("x").getReadOnlyDataPtr()), sizeof(float)*pop->getCurrentListSize());
        ofs.write(reinterpret_cast<const char*>(pop->getReadOnlyStateMemory().getReadOnlyMemoryVector("y").getReadOnlyDataPtr()), sizeof(float)*pop->getCurrentListSize());
        ofs.write(reinterpret_cast<const char*>(pop->getReadOnlyStateMemory().getReadOnlyMemoryVector("z").getReadOnlyDataPtr()), sizeof(float)*pop->getCurrentListSize());
        ofs.write(reinterpret_cast<const char*>(pop->getReadOnlyStateMemory().getReadOnlyMemoryVector("drift").getReadOnlyDataPtr()), sizeof(float)*pop->getCurrentListSize());
        ofs.close();
    }
}
