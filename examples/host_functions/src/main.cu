#include "hip/hip_runtime.h"
#include "flamegpu/flame_api.h"

const unsigned int AGENT_COUNT = 1024;

FLAMEGPU_AGENT_FUNCTION(device_function, MsgNone, MsgNone) {
    const float &prop_float = FLAMEGPU->environment.get<float>("float");
    const int16_t &prop_int16 = FLAMEGPU->environment.get<int16_t>("int16_t");
    const uint64_t &prop_uint64_0 = FLAMEGPU->environment.get<uint64_t>("uint64_t", 0);
    const uint64_t &prop_uint64_1 = FLAMEGPU->environment.get<uint64_t>("uint64_t", 1);
    const uint64_t &prop_uint64_2 = FLAMEGPU->environment.get<uint64_t>("uint64_t", 2);
    if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
        printf("Agent Function[Thread 0]! Properties(Float: %g, int16: %hd, uint64[3]: {%llu, %llu, %llu})\n", prop_float, prop_int16, prop_uint64_0, prop_uint64_1, prop_uint64_2);
    }
    return ALIVE;
}
FLAMEGPU_INIT_FUNCTION(init_function) {
    float min_x = FLAMEGPU->agent("agent").min<float>("x");
    float max_x = FLAMEGPU->agent("agent").max<float>("x");
    printf("Init Function! (AgentCount: %u, Min: %g, Max: %g)\n", FLAMEGPU->agent("agent").count(), min_x, max_x);
    for (unsigned int i = AGENT_COUNT / 2; i < AGENT_COUNT; i++) {
        FLAMEGPU_HOST_NEW_AGENT_API instance = FLAMEGPU->newAgent("agent");
        instance.setVariable<float>("x", static_cast<float>(i));
        instance.setVariable<int>("a", i % 2 == 0 ? 1 : 0);
    }
    printf("Init Function! Created %u more agents\n", AGENT_COUNT / 2);
}
FLAMEGPU_CUSTOM_REDUCTION(customSum, a, b) {
    return a + b;
}
FLAMEGPU_CUSTOM_TRANSFORM(customTransform, a) {
    return (a == 0 || a == 1) ? 1 : 0;
}
FLAMEGPU_STEP_FUNCTION(step_function) {
    auto agent = FLAMEGPU->agent("agent");
    int sum_a = agent.sum<int>("a");
    int custom_sum_a = agent.reduce<int>("a", customSum, 0);
    unsigned int count_a = agent.count<int>("a", 1);
    unsigned int countif_a = agent.transformReduce<int, unsigned int>("a", customTransform, customSum, 0u);
    printf("Step Function! (AgentCount: %u, Sum: %d, CustomSum: %d, Count: %u, CustomCountIf: %u)\n", agent.count(), sum_a, custom_sum_a, count_a, countif_a);
}
FLAMEGPU_EXIT_FUNCTION(exit_function) {
    float uniform_real = FLAMEGPU->random.uniform<float>();
    int uniform_int = FLAMEGPU->random.uniform<int>(1, 10);
    float normal = FLAMEGPU->random.normal<float>();
    float logNormal = FLAMEGPU->random.logNormal<float>(1, 1);
    printf("Exit Function! (%g, %i, %g, %g)\n",
        uniform_real, uniform_int, normal, logNormal);
}
FLAMEGPU_HOST_FUNCTION(host_function) {
    std::vector<unsigned int> hist_x = FLAMEGPU->agent("agent").histogramEven<float>("x", 8, -0.5, 1023.5);
    printf("Host Function! (Hist: [%u, %u, %u, %u, %u, %u, %u, %u]\n",
        hist_x[0], hist_x[1], hist_x[2], hist_x[3], hist_x[4], hist_x[5], hist_x[6], hist_x[7]);
    FLAMEGPU->environment.set<int16_t>("int16_t", FLAMEGPU->environment.get<int16_t>("int16_t") + 1);
}
FLAMEGPU_EXIT_CONDITION(exit_condition) {
    const float CHANCE = 0.15f;
    float uniform_real = FLAMEGPU->random.uniform<float>();
    printf("Exit Condition! (Rolled: %g)\n", uniform_real);
    if (uniform_real < CHANCE) {
        printf("Rolled number is less than %g, exiting!\n", CHANCE);
        return EXIT;
    }
    return CONTINUE;
}


int main(int argc, const char ** argv) {
    ModelDescription model("host_functions_example");

    {  // agent
        AgentDescription &agent = model.newAgent("agent");
        agent.newVariable<float>("x");
        agent.newVariable<int>("a");
        agent.newFunction("device_function", device_function);
    }

    /**
     * GLOBALS
     */
    {
        EnvironmentDescription &envProperties = model.Environment();
        envProperties.add<float>("float", 12.0f);
        envProperties.add<int16_t>("int16_t", 0);
        envProperties.add<uint64_t, 3>("uint64_t", {11llu, 12llu, 13llu});
    }
    /**
     * Control flow
     */     
     {  // Attach init/step/exit functions and exit condition
        model.addInitFunction(init_function);
        model.addStepFunction(step_function);
        model.addExitFunction(exit_function);
        model.addExitCondition(exit_condition);
     }

     {
        LayerDescription &devicefn_layer = model.newLayer("devicefn_layer");
        devicefn_layer.addAgentFunction(device_function);
     }

     {
        LayerDescription &hostfn_layer = model.newLayer("hostfn_layer");
        hostfn_layer.addHostFunction(host_function);
     }

    /**
     * Initialisation
     */
    AgentPopulation population(model.Agent("agent"), AGENT_COUNT/2);
    for (unsigned int i = 0; i < AGENT_COUNT/2; i++) {
        AgentInstance instance = population.getNextInstance();
        instance.setVariable<float>("x", static_cast<float>(i));
        instance.setVariable<int>("a", i % 2 == 0 ? 1 : 0);
    }

    /**
     * Execution
     */
    CUDASimulation cuda_model(model);
    cuda_model.SimulationConfig().steps = 0;
    cuda_model.setPopulationData(population);
    cuda_model.initialise(argc, argv);
    cuda_model.simulate();

    cuda_model.getPopulationData(population);

    getchar();
    return 0;
}
