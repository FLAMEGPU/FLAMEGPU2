#include "hip/hip_runtime.h"
#include "flamegpu/flamegpu.h"

const unsigned int AGENT_COUNT = 1024;

FLAMEGPU_AGENT_FUNCTION(device_function, flamegpu::MessageNone, flamegpu::MessageNone) {
    const float prop_float = FLAMEGPU->environment.getProperty<float>("float");
    const int16_t prop_int16 = FLAMEGPU->environment.getProperty<int16_t>("int16_t");
    const uint64_t prop_uint64_0 = FLAMEGPU->environment.getProperty<uint64_t, 3>("uint64_t", 0);
    const uint64_t prop_uint64_1 = FLAMEGPU->environment.getProperty<uint64_t, 3>("uint64_t", 1);
    const uint64_t prop_uint64_2 = FLAMEGPU->environment.getProperty<uint64_t, 3>("uint64_t", 2);
    if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
        printf("Agent Function[Thread 0]! Properties(Float: %g, int16: %hd, uint64[3]: {%llu, %llu, %llu})\n", prop_float, prop_int16, prop_uint64_0, prop_uint64_1, prop_uint64_2);
    }
    return flamegpu::ALIVE;
}
FLAMEGPU_INIT_FUNCTION(init_function) {
    flamegpu::HostAgentAPI agent = FLAMEGPU->agent("agent");
    float min_x = agent.min<float>("x");
    float max_x = agent.max<float>("x");
    printf("Init Function! (AgentCount: %u, Min: %g, Max: %g)\n", FLAMEGPU->agent("agent").count(), min_x, max_x);
    for (unsigned int i = AGENT_COUNT / 2; i < AGENT_COUNT; i++) {
        flamegpu::HostNewAgentAPI instance = agent.newAgent();
        instance.setVariable<float>("x", static_cast<float>(i));
        instance.setVariable<int>("a", i % 2 == 0 ? 1 : 0);
    }
    printf("Init Function! Created %u more agents\n", AGENT_COUNT / 2);
}
FLAMEGPU_CUSTOM_REDUCTION(customSum, a, b) {
    return a + b;
}
FLAMEGPU_CUSTOM_TRANSFORM(customTransform, a) {
    return (a == 0 || a == 1) ? 1 : 0;
}
FLAMEGPU_STEP_FUNCTION(step_function) {
    auto agent = FLAMEGPU->agent("agent");
    int sum_a = agent.sum<int>("a");
    int custom_sum_a = agent.reduce<int>("a", customSum, 0);
    unsigned int count_a = agent.count<int>("a", 1);
    unsigned int countif_a = agent.transformReduce<int, unsigned int>("a", customTransform, customSum, 0u);
    printf("Step Function! (AgentCount: %u, Sum: %d, CustomSum: %d, Count: %u, CustomCountIf: %u)\n", agent.count(), sum_a, custom_sum_a, count_a, countif_a);
}
FLAMEGPU_EXIT_FUNCTION(exit_function) {
    float uniform_real = FLAMEGPU->random.uniform<float>();
    int uniform_int = FLAMEGPU->random.uniform<int>(1, 10);
    float normal = FLAMEGPU->random.normal<float>();
    float logNormal = FLAMEGPU->random.logNormal<float>(1, 1);
    printf("Exit Function! (%g, %i, %g, %g)\n",
        uniform_real, uniform_int, normal, logNormal);
}
FLAMEGPU_HOST_FUNCTION(host_function) {
    std::vector<unsigned int> hist_x = FLAMEGPU->agent("agent").histogramEven<float>("x", 8, -0.5, 1023.5);
    printf("Host Function! (Hist: [%u, %u, %u, %u, %u, %u, %u, %u]\n",
        hist_x[0], hist_x[1], hist_x[2], hist_x[3], hist_x[4], hist_x[5], hist_x[6], hist_x[7]);
    FLAMEGPU->environment.setProperty<int16_t>("int16_t", FLAMEGPU->environment.getProperty<int16_t>("int16_t") + 1);
}
FLAMEGPU_EXIT_CONDITION(exit_condition) {
    const float CHANCE = 0.15f;
    float uniform_real = FLAMEGPU->random.uniform<float>();
    printf("Exit Condition! (Rolled: %g)\n", uniform_real);
    if (uniform_real < CHANCE) {
        printf("Rolled number is less than %g, exiting!\n", CHANCE);
        return flamegpu::EXIT;
    }
    return flamegpu::CONTINUE;
}


int main(int argc, const char ** argv) {
    flamegpu::ModelDescription model("host_functions_example");

    {  // agent
        flamegpu::AgentDescription  agent = model.newAgent("agent");
        agent.newVariable<float>("x");
        agent.newVariable<int>("a");
        agent.newFunction("device_function", device_function);
    }

    /**
     * GLOBALS
     */
    {
        flamegpu::EnvironmentDescription  envProperties = model.Environment();
        envProperties.newProperty<float>("float", 12.0f);
        envProperties.newProperty<int16_t>("int16_t", 0);
        envProperties.newProperty<uint64_t, 3>("uint64_t", {11llu, 12llu, 13llu});
    }
    /**
     * Control flow
     */     
     {  // Attach init/step/exit functions and exit condition
        model.addInitFunction(init_function);
        model.addStepFunction(step_function);
        model.addExitFunction(exit_function);
        model.addExitCondition(exit_condition);
     }

     {
        flamegpu::LayerDescription devicefn_layer = model.newLayer("devicefn_layer");
        devicefn_layer.addAgentFunction(device_function);
     }

     {
        flamegpu::LayerDescription hostfn_layer = model.newLayer("hostfn_layer");
        hostfn_layer.addHostFunction(host_function);
     }

    /**
     * Initialisation
     */
    flamegpu::AgentVector population(model.Agent("agent"), AGENT_COUNT/2);
    for (unsigned int i = 0; i < AGENT_COUNT/2; i++) {
        flamegpu::AgentVector::Agent instance = population[i];
        instance.setVariable<float>("x", static_cast<float>(i));
        instance.setVariable<int>("a", i % 2 == 0 ? 1 : 0);
    }

    /**
     * Execution
     */
    flamegpu::CUDASimulation cudaSimulation(model);
    cudaSimulation.SimulationConfig().steps = 0;
    cudaSimulation.setPopulationData(population);
    cudaSimulation.initialise(argc, argv);
    cudaSimulation.simulate();

    cudaSimulation.getPopulationData(population);

    // Ensure profiling / memcheck work correctly
    flamegpu::util::cleanup();

    return 0;
}
