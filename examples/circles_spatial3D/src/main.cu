#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>


#include "flamegpu/flame_api.h"
#include "flamegpu/runtime/flamegpu_api.h"
#include "flamegpu/io/factory.h"
#include "flamegpu/visualiser/ModelVis.h"

FLAMEGPU_AGENT_FUNCTION(output_message, MsgNone, MsgSpatial3D) {
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
    FLAMEGPU->message_out.setLocation(
        FLAMEGPU->getVariable<float>("x"),
        FLAMEGPU->getVariable<float>("y"),
        FLAMEGPU->getVariable<float>("z"));
    return ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(move, MsgSpatial3D, MsgNone) {
    const int ID = FLAMEGPU->getVariable<int>("id");
    const float REPULSE_FACTOR = FLAMEGPU->environment.get<float>("repulse");
    const float RADIUS = FLAMEGPU->message_in.radius();
    float fx = 0.0;
    float fy = 0.0;
    float fz = 0.0;
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    const float z1 = FLAMEGPU->getVariable<float>("z");
    int count = 0;
    for (const auto &message : FLAMEGPU->message_in(x1, y1, z1)) {
        if (message.getVariable<int>("id") != ID) {
            const float x2 = message.getVariable<float>("x");
            const float y2 = message.getVariable<float>("y");
            const float z2 = message.getVariable<float>("z");
            float x21 = x2 - x1;
            float y21 = y2 - y1;
            float z21 = z2 - z1;
            const float separation = cbrt(x21*x21 + y21*y21 + z21*z21);
            if (separation < RADIUS && separation > 0.0f) {
                float k = sinf((separation / RADIUS)*3.141*-2)*REPULSE_FACTOR;
                // Normalise without recalculating separation
                x21 /= separation;
                y21 /= separation;
                z21 /= separation;
                fx += k * x21;
                fy += k * y21;
                fz += k * z21;
                count++;
            }
        }
    }
    fx /= count > 0 ? count : 1;
    fy /= count > 0 ? count : 1;
    fz /= count > 0 ? count : 1;
    FLAMEGPU->setVariable<float>("x", x1 + fx);
    FLAMEGPU->setVariable<float>("y", y1 + fy);
    FLAMEGPU->setVariable<float>("z", z1 + fz);
    FLAMEGPU->setVariable<float>("drift", cbrt(fx*fx + fy*fy + fz*fz));
    return ALIVE;
}
FLAMEGPU_STEP_FUNCTION(Validation) {
    static float prevTotalDrift = FLT_MAX;
    static unsigned int driftDropped = 0;
    static unsigned int driftIncreased = 0;
    // This value should decline? as the model moves towards a steady equlibrium state
    // Once an equilibrium state is reached, it is likely to oscillate between 2-4? values
    float totalDrift = FLAMEGPU->agent("Circle").sum<float>("drift");
    if (totalDrift <= prevTotalDrift)
        driftDropped++;
    else
        driftIncreased++;
    prevTotalDrift = totalDrift;
    // printf("Avg Drift: %g\n", totalDrift / FLAMEGPU->agent("Circle").count());
    printf("%.2f%% Drift correct\n", 100 * driftDropped / static_cast<float>(driftDropped + driftIncreased));
}
void export_data(std::shared_ptr<AgentPopulation> pop, const char *filename);
int main(int argc, const char ** argv) {
    const unsigned int AGENT_COUNT = 16384;
    ModelDescription model("Circles_BruteForce_example");

    {   // Location message
        MsgSpatial3D::Description &message = model.newMessage<MsgSpatial3D>("location");
        message.newVariable<int>("id");
        const float max_bound = static_cast<float>(floor(cbrt(AGENT_COUNT)));
        message.setRadius(1.0f);
        message.setMin(0, 0, 0);
        message.setMax(max_bound, max_bound, max_bound);
    }
    {   // Circle agent
        AgentDescription &agent = model.newAgent("Circle");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("z");
        agent.newVariable<float>("drift");  // Store the distance moved here, for validation
        agent.newFunction("output_message", output_message).setMessageOutput("location");
        agent.newFunction("move", move).setMessageInput("location");
    }

    /**
     * GLOBALS
     */
    {
        EnvironmentDescription &env = model.Environment();
        env.add("repulse", 0.05f);
    }

    /**
     * Control flow
     */     
    {   // Attach init/step/exit functions and exit condition
        model.addStepFunction(Validation);
    }

    {   // Layer #1
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(output_message);
    }
    {   // Layer #2
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(move);
    }

    /**
     * Create Model Runner
     */
    CUDAAgentModel cuda_model(model);

    /**
     * Create visualisation
     */
#ifdef VISUALISATION
    ModelVis &m_vis = cuda_model.getVisualisation();
    {
        auto &circ_agt = m_vis.addAgent("Circle");
        // Position vars are named x, y, z; so they are used by default
        circ_agt.setModel(Stock::Models::ICOSPHERE);
        // circ_agt.setModelScale(1/20.0f);  // Not worth using till we can affect camera position/speed
    }
    m_vis.activate();
#endif
    /**
     * Initialisation
     */
    cuda_model.initialise(argc, argv);
    if (cuda_model.getSimulationConfig().xml_input_file.empty()) {
        // Currently population has not been init, so generate an agent population on the fly
        std::default_random_engine rng;
        std::uniform_real_distribution<float> dist(0.0f, static_cast<float>(floor(cbrt(AGENT_COUNT))));
        AgentPopulation population(model.Agent("Circle"), AGENT_COUNT);
        for (unsigned int i = 0; i < AGENT_COUNT; i++) {
            AgentInstance instance = population.getNextInstance();
            instance.setVariable<int>("id", i);
            instance.setVariable<float>("x", dist(rng));
            instance.setVariable<float>("y", dist(rng));
            instance.setVariable<float>("z", dist(rng));
        }
        cuda_model.setPopulationData(population);
    }

    /**
     * Execution
     */
    cuda_model.simulate();

    /**
     * Export Pop
     */
    cuda_model.exportData("end.xml");

#ifdef VISUALISATION
    m_vis.join();
#endif
    return 0;
}
