#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <array>

#include "flamegpu/flamegpu.h"

// Grid Size (the product of these is the agent count)
#define GRID_WIDTH 256
#define GRID_HEIGHT 256

// Agent state variables
#define AGENT_STATUS_UNOCCUPIED 0
#define AGENT_STATUS_OCCUPIED 1
#define AGENT_STATUS_MOVEMENT_REQUESTED 2
#define AGENT_STATUS_MOVEMENT_UNRESOLVED 3

// Growback variables
#define SUGAR_GROWBACK_RATE 1
#define SUGAR_MAX_CAPACITY 7

// Visualisation mode (0=occupied/move status, 1=occupied/sugar/level)
#define VIS_MODE 1


FLAMEGPU_AGENT_FUNCTION(metabolise_and_growback, flamegpu::MessageNone, flamegpu::MessageNone) {
    int sugar_level = FLAMEGPU->getVariable<int>("sugar_level");
    int env_sugar_level = FLAMEGPU->getVariable<int>("env_sugar_level");
    int env_max_sugar_level = FLAMEGPU->getVariable<int>("env_max_sugar_level");
    int status = FLAMEGPU->getVariable<int>("status");
    // metabolise if occupied
    if (status == AGENT_STATUS_OCCUPIED || status == AGENT_STATUS_MOVEMENT_UNRESOLVED) {
        // store any sugar present in the cell
        if (env_sugar_level > 0) {
            sugar_level += env_sugar_level;
            // Occupied cells are marked as -1 sugar.
            env_sugar_level = -1;
        }

        // metabolise
        sugar_level -= FLAMEGPU->getVariable<int>("metabolism");

        // check if agent dies
        if (sugar_level == 0) {
            status = AGENT_STATUS_UNOCCUPIED;
            FLAMEGPU->setVariable<int>("agent_id", -1);
            env_sugar_level = 0;
            FLAMEGPU->setVariable<int>("metabolism", 0);
        }
    }

    // growback if unoccupied
    if (status == AGENT_STATUS_UNOCCUPIED) {
        env_sugar_level += SUGAR_GROWBACK_RATE;
        if (env_sugar_level > env_max_sugar_level) {
            env_sugar_level = env_max_sugar_level;
        }
    }

    // set all active agents to unresolved as they may now want to move
    if (status == AGENT_STATUS_OCCUPIED) {
        status = AGENT_STATUS_MOVEMENT_UNRESOLVED;
    }
    FLAMEGPU->setVariable<int>("sugar_level", sugar_level);
    FLAMEGPU->setVariable<int>("env_sugar_level", env_sugar_level);
    FLAMEGPU->setVariable<int>("status", status);

    return flamegpu::ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(output_cell_status, flamegpu::MessageNone, flamegpu::MessageArray2D) {
    unsigned int agent_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0);
    unsigned int agent_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);
    FLAMEGPU->message_out.setVariable("location_id", FLAMEGPU->getID());
    FLAMEGPU->message_out.setVariable("status", FLAMEGPU->getVariable<int>("status"));
    FLAMEGPU->message_out.setVariable("env_sugar_level", FLAMEGPU->getVariable<int>("env_sugar_level"));
    FLAMEGPU->message_out.setIndex(agent_x, agent_y);
    return flamegpu::ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(movement_request, flamegpu::MessageArray2D, flamegpu::MessageArray2D) {
    int best_sugar_level = -1;
    float best_sugar_random = -1;
    flamegpu::id_t best_location_id = flamegpu::ID_NOT_SET;

    // if occupied then look for empty cells {
    // find the best location to move to (ensure we don't just pick first cell with max value)
    int status = FLAMEGPU->getVariable<int>("status");

    unsigned int agent_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0);
    unsigned int agent_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);

    // if occupied then look for empty cells
    if (status == AGENT_STATUS_MOVEMENT_UNRESOLVED) {
        for (auto current_message : FLAMEGPU->message_in.wrap(agent_x, agent_y)) {
            // if location is unoccupied then check for empty locations
            if (current_message.getVariable<int>("status") == AGENT_STATUS_UNOCCUPIED) {
                // if the sugar level at current location is better than currently stored then update
                int message_env_sugar_level = current_message.getVariable<int>("env_sugar_level");
                float message_priority = FLAMEGPU->random.uniform<float>();
                if ((message_env_sugar_level > best_sugar_level) ||
                    (message_env_sugar_level == best_sugar_level && message_priority > best_sugar_random)) {
                    best_sugar_level = message_env_sugar_level;
                    best_sugar_random = message_priority;
                    best_location_id = current_message.getVariable<flamegpu::id_t>("location_id");
                }
            }
        }

        // if the agent has found a better location to move to then update its state
        // if there is a better location to move to then state indicates a movement request
        status = best_location_id != flamegpu::ID_NOT_SET ? AGENT_STATUS_MOVEMENT_REQUESTED : AGENT_STATUS_OCCUPIED;
        FLAMEGPU->setVariable<int>("status", status);
    }

    // add a movement request
    FLAMEGPU->message_out.setVariable<int>("agent_id", FLAMEGPU->getVariable<int>("agent_id"));
    FLAMEGPU->message_out.setVariable<flamegpu::id_t>("location_id", best_location_id);
    FLAMEGPU->message_out.setVariable<int>("sugar_level", FLAMEGPU->getVariable<int>("sugar_level"));
    FLAMEGPU->message_out.setVariable<int>("metabolism", FLAMEGPU->getVariable<int>("metabolism"));
    FLAMEGPU->message_out.setIndex(agent_x, agent_y);

    return flamegpu::ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(movement_response, flamegpu::MessageArray2D, flamegpu::MessageArray2D) {
    int best_request_id = -1;
    float best_request_priority = -1;
    int best_request_sugar_level = -1;
    int best_request_metabolism = -1;

    int status = FLAMEGPU->getVariable<int>("status");
    const flamegpu::id_t location_id = FLAMEGPU->getID();
    const unsigned int agent_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0);
    const unsigned int agent_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);

    for (auto current_message : FLAMEGPU->message_in.wrap(agent_x, agent_y)) {
        // if the location is unoccupied then check for agents requesting to move here
        if (status == AGENT_STATUS_UNOCCUPIED) {
            // check if request is to move to this location
            if (current_message.getVariable<flamegpu::id_t>("location_id") == location_id) {
                // check the priority and maintain the best ranked agent
                float message_priority = FLAMEGPU->random.uniform<float>();
                if (message_priority > best_request_priority) {
                    best_request_id = current_message.getVariable<int>("agent_id");
                    best_request_priority = message_priority;
                }
            }
        }
    }

    // if the location is unoccupied and an agent wants to move here then do so and send a response
    if ((status == AGENT_STATUS_UNOCCUPIED) && (best_request_id >= 0))    {
        FLAMEGPU->setVariable<int>("status", AGENT_STATUS_OCCUPIED);
        // move the agent to here and consume the cell's sugar
        best_request_sugar_level += FLAMEGPU->getVariable<int>("env_sugar_level");
        FLAMEGPU->setVariable<int>("agent_id", best_request_id);
        FLAMEGPU->setVariable<int>("sugar_level", best_request_sugar_level);
        FLAMEGPU->setVariable<int>("metabolism", best_request_metabolism);
        FLAMEGPU->setVariable<int>("env_sugar_level", -1);
    }

    // add a movement response
    FLAMEGPU->message_out.setVariable<int>("agent_id", best_request_id);
    FLAMEGPU->message_out.setIndex(agent_x, agent_y);

    return flamegpu::ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(movement_transaction, flamegpu::MessageArray2D, flamegpu::MessageNone) {
    int status = FLAMEGPU->getVariable<int>("status");
    int agent_id = FLAMEGPU->getVariable<int>("agent_id");
    unsigned int agent_x = FLAMEGPU->getVariable<unsigned int, 2>("pos", 0);
    unsigned int agent_y = FLAMEGPU->getVariable<unsigned int, 2>("pos", 1);

    for (auto current_message : FLAMEGPU->message_in.wrap(agent_x, agent_y)) {
        // if location contains an agent wanting to move then look for responses allowing relocation
        if (status == AGENT_STATUS_MOVEMENT_REQUESTED) {  // if the movement response request came from this location
            if (current_message.getVariable<int>("agent_id") == agent_id) {
                // remove the agent and reset agent specific variables as it has now moved
                status = AGENT_STATUS_UNOCCUPIED;
                FLAMEGPU->setVariable<int>("agent_id", -1);
                FLAMEGPU->setVariable<int>("sugar_level", 0);
                FLAMEGPU->setVariable<int>("metabolism", 0);
                FLAMEGPU->setVariable<int>("env_sugar_level", 0);
            }
        }
    }

    // if request has not been responded to then agent is unresolved
    if (status == AGENT_STATUS_MOVEMENT_REQUESTED) {
        status = AGENT_STATUS_MOVEMENT_UNRESOLVED;
    }

    FLAMEGPU->setVariable<int>("status", status);

    return flamegpu::ALIVE;
}
FLAMEGPU_EXIT_CONDITION(MovementExitCondition) {
    static unsigned int iterations = 0;
    iterations++;

    // Max iterations 9
    if (iterations < 9) {
        // Agent movements still unresolved
        if (FLAMEGPU->agent("agent").count("status", AGENT_STATUS_MOVEMENT_UNRESOLVED)) {
            return flamegpu::CONTINUE;
        }
    }

    iterations = 0;
    return flamegpu::EXIT;
}
/**
 * Construct the common components of agent shared between both parent and submodel
 */
flamegpu::AgentDescription makeCoreAgent(flamegpu::ModelDescription &model) {
    flamegpu::AgentDescription  agent = model.newAgent("agent");
    agent.newVariable<unsigned int, 2>("pos");
    agent.newVariable<int>("agent_id");
    agent.newVariable<int>("status");
    // agent specific variables
    agent.newVariable<int>("sugar_level");
    agent.newVariable<int>("metabolism");
    // environment specific var
    agent.newVariable<int>("env_sugar_level");
    agent.newVariable<int>("env_max_sugar_level");
#ifdef FLAMEGPU_VISUALISATION
    // Redundant seperate floating point position vars for vis
    agent.newVariable<float>("x");
    agent.newVariable<float>("y");
#endif
    return agent;
}
int main(int argc, const char ** argv) {
    flamegpu::util::nvtx::Range range{"main"};
    flamegpu::util::nvtx::push("ModelDescription");
    flamegpu::ModelDescription submodel("Movement_model");
    {  // Define sub model for conflict resolution
        /**
         * Messages
         */
        {   // cell_status message
            flamegpu::MessageArray2D::Description message = submodel.newMessage<flamegpu::MessageArray2D>("cell_status");
            message.newVariable<flamegpu::id_t>("location_id");
            message.newVariable<int>("status");
            message.newVariable<int>("env_sugar_level");
            message.setDimensions(GRID_WIDTH, GRID_HEIGHT);
        }
        {   // movement_request message
            flamegpu::MessageArray2D::Description message = submodel.newMessage<flamegpu::MessageArray2D>("movement_request");
            message.newVariable<int>("agent_id");
            message.newVariable<flamegpu::id_t>("location_id");
            message.newVariable<int>("sugar_level");
            message.newVariable<int>("metabolism");
            message.setDimensions(GRID_WIDTH, GRID_HEIGHT);
        }
        {   // movement_response message
            flamegpu::MessageArray2D::Description message = submodel.newMessage<flamegpu::MessageArray2D>("movement_response");
            message.newVariable<flamegpu::id_t>("location_id");
            message.newVariable<int>("agent_id");
            message.setDimensions(GRID_WIDTH, GRID_HEIGHT);
        }
        /**
         * Agents
         */
        {
            flamegpu::AgentDescription  agent = makeCoreAgent(submodel);
            auto fn_output_cell_status = agent.newFunction("output_cell_status", output_cell_status);
            {
                fn_output_cell_status.setMessageOutput("cell_status");
            }
            auto fn_movement_request = agent.newFunction("movement_request", movement_request);
            {
                fn_movement_request.setMessageInput("cell_status");
                fn_movement_request.setMessageOutput("movement_request");
            }
            auto fn_movement_response = agent.newFunction("movement_response", movement_response);
            {
                fn_movement_response.setMessageInput("movement_request");
                fn_movement_response.setMessageOutput("movement_response");
            }
            auto fn_movement_transaction = agent.newFunction("movement_transaction", movement_transaction);
            {
                fn_movement_transaction.setMessageInput("movement_response");
            }
        }

        /**
         * Globals
         */
        {
            // flamegpu::EnvironmentDescription  env = model.Environment();
        }

        /**
         * Control flow
         */
        {   // Layer #1
            flamegpu::LayerDescription layer = submodel.newLayer();
            layer.addAgentFunction(output_cell_status);
        }
        {   // Layer #2
            flamegpu::LayerDescription layer = submodel.newLayer();
            layer.addAgentFunction(movement_request);
        }
        {   // Layer #3
            flamegpu::LayerDescription layer = submodel.newLayer();
            layer.addAgentFunction(movement_response);
        }
        {   // Layer #4
            flamegpu::LayerDescription layer = submodel.newLayer();
            layer.addAgentFunction(movement_transaction);
        }
        submodel.addExitCondition(MovementExitCondition);
    }

    flamegpu::ModelDescription model("Sugarscape");

    /**
     * Agents
     */
    {   // Per cell agent
        flamegpu::AgentDescription  agent = makeCoreAgent(model);
        // Functions
        agent.newFunction("metabolise_and_growback", metabolise_and_growback);
    }

    /**
     * Submodels
     */
    flamegpu::SubModelDescription movement_sub = model.newSubModel("movement_conflict_resolution_model", submodel);
    {
        movement_sub.bindAgent("agent", "agent", true, true);
    }

    /**
     * Globals
     */
    {
        // flamegpu::EnvironmentDescription  env = model.Environment();
    }

    /**
     * Control flow
     */
    {   // Layer #1
        flamegpu::LayerDescription layer = model.newLayer();
        layer.addAgentFunction(metabolise_and_growback);
    }
    {   // Layer #2
        flamegpu::LayerDescription layer = model.newLayer();
        layer.addSubModel(movement_sub);
    }
    flamegpu::util::nvtx::pop();

    /**
     * Create Model Runner
     */
    flamegpu::util::nvtx::push("CUDASimulation creation");
    flamegpu::CUDASimulation  cudaSimulation(model);
    flamegpu::util::nvtx::pop();

    /**
     * Create visualisation
     * @note FLAMEGPU2 doesn't currently have proper support for discrete/2d visualisations
     */
#ifdef FLAMEGPU_VISUALISATION
    flamegpu::visualiser::ModelVis visualisation = cudaSimulation.getVisualisation();
    {
        visualisation.setSimulationSpeed(2);
        visualisation.setInitialCameraLocation(GRID_WIDTH / 2.0f, GRID_HEIGHT / 2.0f, 225.0f);
        visualisation.setInitialCameraTarget(GRID_WIDTH / 2.0f, GRID_HEIGHT /2.0f, 0.0f);
        visualisation.setCameraSpeed(0.001f * GRID_WIDTH);
        visualisation.setOrthographic(true);
        visualisation.setOrthographicZoomModifier(0.365f);
        visualisation.setViewClips(0.1f, 5000);

        auto agt = visualisation.addAgent("agent");
        // Position vars are named x, y, z; so they are used by default
        agt.setModel(flamegpu::visualiser::Stock::Models::CUBE);  // 5 unwanted faces!
        agt.setModelScale(1.0f);
#if VIS_MODE == 0
            flamegpu::visualiser::DiscreteColor<int> cell_colors = flamegpu::visualiser::DiscreteColor<int>("status", flamegpu::visualiser::Color{"#666"});
            cell_colors[AGENT_STATUS_UNOCCUPIED] = flamegpu::visualiser::Stock::Colors::RED;
            cell_colors[AGENT_STATUS_OCCUPIED] = flamegpu::visualiser::Stock::Colors::GREEN;
            cell_colors[AGENT_STATUS_MOVEMENT_REQUESTED] = flamegpu::visualiser::Stock::Colors::BLUE;  // Not possible, only occurs inside the submodel
            cell_colors[AGENT_STATUS_MOVEMENT_UNRESOLVED] = flamegpu::visualiser::Stock::Colors::WHITE;
            agt.setColor(cell_colors);
#else
            flamegpu::visualiser::DiscreteColor<int> cell_colors = flamegpu::visualiser::DiscreteColor<int>("env_sugar_level", flamegpu::visualiser::Stock::Palettes::Viridis(SUGAR_MAX_CAPACITY + 1), flamegpu::visualiser::Color{"#f00"});
            agt.setColor(cell_colors);
#endif
    }
    visualisation.activate();
#endif

    /**
     * Initialisation
     */
    flamegpu::util::nvtx::push("CUDASimulation initialisation");
    cudaSimulation.initialise(argc, argv);
    if (cudaSimulation.getSimulationConfig().input_file.empty()) {
        std::mt19937_64 rng;
        // Pre init, decide the sugar hotspots
        std::vector<std::array<unsigned int, 4>> sugar_hotspots;
        {
            std::uniform_int_distribution<unsigned int> width_dist(0, GRID_WIDTH-1);
            std::uniform_int_distribution<unsigned int> height_dist(0, GRID_HEIGHT-1);
            // Each sugar hotspot has a radius of 3-15 blocks
            std::uniform_int_distribution<unsigned int> radius_dist(5, 30);
            // Hostpot area should cover around 50% of the map
            float hotspot_area = 0;
            while (hotspot_area < GRID_WIDTH * GRID_HEIGHT) {
                unsigned int rad = radius_dist(rng);
                std::array<unsigned int, 4> hs = {width_dist(rng), height_dist(rng), rad, SUGAR_MAX_CAPACITY };
                sugar_hotspots.push_back(hs);
                hotspot_area += 3.141f * rad * rad;
            }
        }


        // Currently population has not been init, so generate an agent population on the fly
        const unsigned int CELL_COUNT = GRID_WIDTH * GRID_HEIGHT;
        std::uniform_real_distribution<float> normal(0, 1);
        std::uniform_int_distribution<int> agent_sugar_dist(0, SUGAR_MAX_CAPACITY * 2);
        std::uniform_int_distribution<int> poor_env_sugar_dist(0, SUGAR_MAX_CAPACITY/2);
        unsigned int i = 0;
        unsigned int agent_id = 0;
        flamegpu::AgentVector init_pop(model.Agent("agent"), CELL_COUNT);
        for (unsigned int x = 0; x < GRID_WIDTH; ++x) {
            for (unsigned int y = 0; y < GRID_HEIGHT; ++y) {
                flamegpu::AgentVector::Agent instance = init_pop[i++];
                instance.setVariable<unsigned int, 2>("pos", { x, y });
                // TODO: How should these values be init?
                // agent specific variables
                // 10% chance of cell holding an agent
                if (normal(rng) < 0.1) {
                    instance.setVariable<int>("agent_id", agent_id++);
                    instance.setVariable<int>("status", AGENT_STATUS_OCCUPIED);
                    instance.setVariable<int>("sugar_level", agent_sugar_dist(rng)/2);  // Agent sugar dist 0-3, less chance of 0
                    instance.setVariable<int>("metabolism", 6);
                } else {
                    instance.setVariable<int>("agent_id", -1);
                    instance.setVariable<int>("status", AGENT_STATUS_UNOCCUPIED);
                    instance.setVariable<int>("sugar_level", 0);
                    instance.setVariable<int>("metabolism", 0);
                }
                // environment specific var
                unsigned int env_sugar_lvl = 0;
                const int hotspot_core_size = 5;
                for (auto &hs : sugar_hotspots) {
                    // Workout the highest sugar lvl from a nearby hotspot
                    int hs_x = static_cast<int>(std::get<0>(hs));
                    int hs_y = static_cast<int>(std::get<1>(hs));
                    unsigned int hs_rad = std::get<2>(hs);
                    unsigned int hs_level = std::get<3>(hs);
                    float hs_dist = static_cast<float>(sqrt(pow(hs_x-static_cast<int>(x), 2.0) + pow(hs_y-static_cast<int>(y), 2.0)));
                    if (hs_dist <= hotspot_core_size) {
                        unsigned int t = hs_level;
                        env_sugar_lvl = t > env_sugar_lvl ? t : env_sugar_lvl;
                    } else if (hs_dist <= hs_rad) {
                        int non_core_len = hs_rad - hotspot_core_size;
                        float dist_from_core = hs_dist - hotspot_core_size;
                        unsigned int t = static_cast<unsigned int>(hs_level * (non_core_len - dist_from_core) / non_core_len);
                        env_sugar_lvl = t > env_sugar_lvl ? t : env_sugar_lvl;
                    }
                }
                env_sugar_lvl = env_sugar_lvl < SUGAR_MAX_CAPACITY / 2 ? poor_env_sugar_dist(rng) : env_sugar_lvl;
                instance.setVariable<int>("env_max_sugar_level", env_sugar_lvl);  // All cells begin at their local max sugar
                instance.setVariable<int>("env_sugar_level", env_sugar_lvl);
#ifdef FLAMEGPU_VISUALISATION
                // Redundant separate floating point position vars for vis
                instance.setVariable<float>("x", static_cast<float>(x));
                instance.setVariable<float>("y", static_cast<float>(y));
#endif
            }
        }
        cudaSimulation.setPopulationData(init_pop);
    }
    flamegpu::util::nvtx::pop();

    /**
     * Execution
     */
    cudaSimulation.simulate();

    /**
     * Export Pop
     */
    // cudaSimulation.exportData("end.xml");

#ifdef FLAMEGPU_VISUALISATION
    visualisation.join();
#endif

    // Ensure profiling / memcheck work correctly
    flamegpu::util::cleanup();

    return 0;
}
